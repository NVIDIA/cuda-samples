#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

////////////////////////////////////////////////////////////////////////////////
//
//  simpleCUFFT_2d_MGPU.cu
//
//  This sample code demonstrate the use of CUFFT library for 2D data on multiple GPU.
//  Example showing the use of CUFFT for solving 2D-POISSON equation using FFT on multiple GPU.
//  For reference we have used the equation given in http://www.bu.edu/pasi/files/2011/07/
//  Lecture83.pdf
//
////////////////////////////////////////////////////////////////////////////////


// System includes
#include <stdlib.h>
#include <stdio.h>

#include <string.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>

//CUFFT Header file
#include <hipfft/hipfftXt.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Complex data type
typedef float2 Complex;

// Data configuration
const int GPU_COUNT = 2;
const int BSZ_Y = 4;
const int BSZ_X = 4;

// Forward Declaration
void solvePoissonEquation(hipLibXtDesc *, hipLibXtDesc *, float **, int, int);

__global__ void solvePoisson(hipfftComplex *, hipfftComplex *, float *, int, int,
                             int n_gpu);

///////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf(
      "\nPoisson equation using CUFFT library on Multiple GPUs is "
      "starting...\n\n");

  int GPU_N;
  checkCudaErrors(hipGetDeviceCount(&GPU_N));

  if (GPU_N < GPU_COUNT) {
    printf("No. of GPU on node %d\n", GPU_N);
    printf("Two GPUs are required to run simpleCUFFT_2d_MGPU sample code\n");
    exit(EXIT_WAIVED);
  }

  int *major_minor = (int *)malloc(sizeof(int) * GPU_N * 2);
  int found2IdenticalGPUs = 0;
  int nGPUs = 2;
  int *whichGPUs;
  whichGPUs = (int *)malloc(sizeof(int) * nGPUs);

  for (int i = 0; i < GPU_N; i++) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, i));
    major_minor[i * 2] = deviceProp.major;
    major_minor[i * 2 + 1] = deviceProp.minor;
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", i,
           deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  for (int i = 0; i < GPU_N; i++) {
    for (int j = i + 1; j < GPU_N; j++) {
      if ((major_minor[i * 2] == major_minor[j * 2]) &&
          (major_minor[i * 2 + 1] == major_minor[j * 2 + 1])) {
        whichGPUs[0] = i;
        whichGPUs[1] = j;
        found2IdenticalGPUs = 1;
        break;
      }
    }
    if (found2IdenticalGPUs) {
      break;
    }
  }

  free(major_minor);
  if (!found2IdenticalGPUs) {
    printf(
        "No Two GPUs with same architecture found\nWaiving simpleCUFFT_2d_MGPU "
        "sample\n");
    exit(EXIT_WAIVED);
  }

  int N = 64;
  float xMAX = 1.0f, xMIN = 0.0f, yMIN = 0.0f, h = (xMAX - xMIN) / ((float)N),
        s = 0.1f, s2 = s * s;
  float *x, *y, *f, *u_a, r2;

  x = (float *)malloc(sizeof(float) * N * N);
  y = (float *)malloc(sizeof(float) * N * N);
  f = (float *)malloc(sizeof(float) * N * N);
  u_a = (float *)malloc(sizeof(float) * N * N);

  for (int j = 0; j < N; j++)
    for (int i = 0; i < N; i++) {
      x[N * j + i] = xMIN + i * h;
      y[N * j + i] = yMIN + j * h;
      r2 = (x[N * j + i] - 0.5f) * (x[N * j + i] - 0.5f) +
           (y[N * j + i] - 0.5f) * (y[N * j + i] - 0.5f);
      f[N * j + i] = (r2 - 2 * s2) / (s2 * s2) * exp(-r2 / (2 * s2));
      u_a[N * j + i] = exp(-r2 / (2 * s2));  // analytical solution
    }

  float *k, *d_k[GPU_COUNT];
  k = (float *)malloc(sizeof(float) * N);
  for (int i = 0; i <= N / 2; i++) {
    k[i] = i * 2 * (float)M_PI;
  }
  for (int i = N / 2 + 1; i < N; i++) {
    k[i] = (i - N) * 2 * (float)M_PI;
  }

  // Create a complex variable on host
  Complex *h_f = (Complex *)malloc(sizeof(Complex) * N * N);

  // Initialize the memory for the signal
  for (int i = 0; i < (N * N); i++) {
    h_f[i].x = f[i];
    h_f[i].y = 0.0f;
  }

  // hipfftCreate() - Create an empty plan
  hipfftResult result;
  hipfftHandle planComplex;
  result = hipfftCreate(&planComplex);
  if (result != HIPFFT_SUCCESS) {
    printf("hipfftCreate failed\n");
    exit(EXIT_FAILURE);
  }

  // hipfftXtSetGPUs() - Define which GPUs to use
  result = hipfftXtSetGPUs(planComplex, nGPUs, whichGPUs);

  if (result == HIPFFT_INVALID_DEVICE) {
    printf("This sample requires two GPUs on the same board.\n");
    printf("No such board was found. Waiving sample.\n");
    exit(EXIT_WAIVED);
  } else if (result != HIPFFT_SUCCESS) {
    printf("hipfftXtSetGPUs failed\n");
    exit(EXIT_FAILURE);
  }

  // Print the device information to run the code
  printf("\nRunning on GPUs\n");
  for (int i = 0; i < 2; i++) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, whichGPUs[i]));
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n",
           whichGPUs[i], deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  size_t *worksize;
  worksize = (size_t *)malloc(sizeof(size_t) * nGPUs);

  // hipfftMakePlan2d() - Create the plan
  result = hipfftMakePlan2d(planComplex, N, N, HIPFFT_C2C, worksize);
  if (result != HIPFFT_SUCCESS) {
    printf("*MakePlan* failed\n");
    exit(EXIT_FAILURE);
  }

  for (int i = 0; i < nGPUs; i++) {
    hipSetDevice(whichGPUs[i]);
    hipMalloc((void **)&d_k[i], sizeof(float) * N);
    hipMemcpy(d_k[i], k, sizeof(float) * N, hipMemcpyHostToDevice);
  }

  // Create a variable on device
  // d_f - variable on device to store the input data
  // d_d_f - variable that store the natural order of d_f data
  // d_out - device output
  hipLibXtDesc *d_f, *d_d_f, *d_out;

  // hipfftXtMalloc() - Malloc data on multiple GPUs

  result = hipfftXtMalloc(planComplex, (hipLibXtDesc **)&d_f,
                         HIPFFT_XT_FORMAT_INPLACE);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtMalloc failed\n");
    exit(EXIT_FAILURE);
  }

  result = hipfftXtMalloc(planComplex, (hipLibXtDesc **)&d_d_f,
                         HIPFFT_XT_FORMAT_INPLACE);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtMalloc failed\n");
    exit(EXIT_FAILURE);
  }

  result = hipfftXtMalloc(planComplex, (hipLibXtDesc **)&d_out,
                         HIPFFT_XT_FORMAT_INPLACE);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtMalloc failed\n");
    exit(EXIT_FAILURE);
  }

  // hipfftXtMemcpy() - Copy the data from host to device
  result = hipfftXtMemcpy(planComplex, d_f, h_f, HIPFFT_COPY_HOST_TO_DEVICE);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtMemcpy failed\n");
    exit(EXIT_FAILURE);
  }

  // hipfftXtExecDescriptorC2C() - Execute FFT on data on multiple GPUs
  printf("Forward 2d FFT on multiple GPUs\n");
  result = hipfftXtExecDescriptorC2C(planComplex, d_f, d_f, HIPFFT_FORWARD);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtExecC2C  failed\n");
    exit(EXIT_FAILURE);
  }

  // hipfftXtMemcpy() - Copy the data to natural order on GPUs
  result = hipfftXtMemcpy(planComplex, d_d_f, d_f, HIPFFT_COPY_DEVICE_TO_DEVICE);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtMemcpy failed\n");
    exit(EXIT_FAILURE);
  }

  printf("Solve Poisson Equation\n");
  solvePoissonEquation(d_d_f, d_out, d_k, N, nGPUs);

  printf("Inverse 2d FFT on multiple GPUs\n");
  // hipfftXtExecDescriptorC2C() - Execute inverse  FFT on data on multiple GPUs
  result = hipfftXtExecDescriptorC2C(planComplex, d_out, d_out, HIPFFT_BACKWARD);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtExecC2C  failed\n");
    exit(EXIT_FAILURE);
  }

  // Create a variable on host to copy the data from device
  // h_d_out - variable store the output of device
  Complex *h_d_out = (Complex *)malloc(sizeof(Complex) * N * N);

  // hipfftXtMemcpy() - Copy data from multiple GPUs to host
  result =
      hipfftXtMemcpy(planComplex, h_d_out, d_out, HIPFFT_COPY_DEVICE_TO_HOST);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtMemcpy failed\n");
    exit(EXIT_FAILURE);
  }

  float *out = (float *)malloc(sizeof(float) * N * N);
  float constant = h_d_out[0].x / N * N;
  for (int i = 0; i < N * N; i++) {
    // subtract u[0] to force the arbitrary constant to be 0
    out[i] = (h_d_out[i].x / (N * N)) - constant;
  }

  // cleanup memory

  free(h_f);
  free(k);
  free(out);
  free(h_d_out);
  free(x);
  free(whichGPUs);
  free(y);
  free(f);
  free(u_a);
  free(worksize);

  // cudaXtFree() - Free GPU memory
  for (int i = 0; i < GPU_COUNT; i++) {
    hipFree(d_k[i]);
  }
  result = hipfftXtFree(d_out);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtFree failed\n");
    exit(EXIT_FAILURE);
  }
  result = hipfftXtFree(d_f);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtFree failed\n");
    exit(EXIT_FAILURE);
  }
  result = hipfftXtFree(d_d_f);
  if (result != HIPFFT_SUCCESS) {
    printf("*XtFree failed\n");
    exit(EXIT_FAILURE);
  }

  // hipfftDestroy() - Destroy FFT plan
  result = hipfftDestroy(planComplex);
  if (result != HIPFFT_SUCCESS) {
    printf("hipfftDestroy failed: code %d\n", (int)result);
    exit(EXIT_FAILURE);
  }

  exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////////
// Launch kernel on  multiple GPU
///////////////////////////////////////////////////////////////////////////////////
void solvePoissonEquation(hipLibXtDesc *d_ft, hipLibXtDesc *d_ft_k, float **k,
                          int N, int nGPUs) {
  int device;
  dim3 dimGrid(int(N / BSZ_X), int((N / 2) / BSZ_Y));
  dim3 dimBlock(BSZ_X, BSZ_Y);

  for (int i = 0; i < nGPUs; i++) {
    device = d_ft_k->descriptor->GPUs[i];
    hipSetDevice(device);
    solvePoisson<<<dimGrid, dimBlock>>>(
        (hipfftComplex *)d_ft->descriptor->data[i],
        (hipfftComplex *)d_ft_k->descriptor->data[i], k[i], N, i, nGPUs);
  }

  // Wait for device to finish all operation
  for (int i = 0; i < nGPUs; i++) {
    device = d_ft_k->descriptor->GPUs[i];
    hipSetDevice(device);
    hipDeviceSynchronize();

    // Check if kernel execution generated and error
    getLastCudaError("Kernel execution failed [ solvePoisson ]");
  }
}

////////////////////////////////////////////////////////////////////////////////
// Kernel for Solving Poisson equation on GPU
////////////////////////////////////////////////////////////////////////////////
__global__ void solvePoisson(hipfftComplex *ft, hipfftComplex *ft_k, float *k,
                             int N, int gpu_id, int n_gpu) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int index = j * N + i;
  if (i < N && j < N / n_gpu) {
    float k2 =
        k[i] * k[i] + k[j + gpu_id * N / n_gpu] * k[j + gpu_id * N / n_gpu];
    if (i == 0 && j == 0 && gpu_id == 0) {
      k2 = 1.0f;
    }

    ft_k[index].x = -ft[index].x * 1 / k2;
    ft_k[index].y = -ft[index].y * 1 / k2;
  }
}
