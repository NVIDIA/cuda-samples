#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Example showing the use of CUFFT for fast 1D-convolution using FFT. */

// System includes
#include <stdlib.h>
#include <stdio.h>

#include <string.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>

//CUFFT Header file
#include <hipfft/hipfftXt.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Complex data type
typedef float2 Complex;

static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void ComplexPointwiseMulAndScale(hipfftComplex *,
                                                   hipfftComplex *, int, float);

// Kernel for GPU
void multiplyCoefficient(hipLibXtDesc *, hipLibXtDesc *, int, float, int);

// Filtering functions
void Convolve(const Complex *, int, const Complex *, int, Complex *);

// Padding functions
int PadData(const Complex *, Complex **, int, const Complex *, Complex **, int);

////////////////////////////////////////////////////////////////////////////////
// Data configuration
// The filter size is assumed to be a number smaller than the signal size
///////////////////////////////////////////////////////////////////////////////
const int SIGNAL_SIZE = 1018;
const int FILTER_KERNEL_SIZE = 11;
const int GPU_COUNT = 2;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("\n[simpleCUFFT_MGPU] is starting...\n\n");

  int GPU_N;
  checkCudaErrors(hipGetDeviceCount(&GPU_N));

  if (GPU_N < GPU_COUNT) {
    printf("No. of GPU on node %d\n", GPU_N);
    printf("Two GPUs are required to run simpleCUFFT_MGPU sample code\n");
    exit(EXIT_WAIVED);
  }

  int *major_minor = (int *)malloc(sizeof(int) * GPU_N * 2);
  int found2IdenticalGPUs = 0;
  int nGPUs = 2;
  int *whichGPUs;
  whichGPUs = (int *)malloc(sizeof(int) * nGPUs);

  for (int i = 0; i < GPU_N; i++) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, i));
    major_minor[i * 2] = deviceProp.major;
    major_minor[i * 2 + 1] = deviceProp.minor;
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", i,
           deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  for (int i = 0; i < GPU_N; i++) {
    for (int j = i + 1; j < GPU_N; j++) {
      if ((major_minor[i * 2] == major_minor[j * 2]) &&
          (major_minor[i * 2 + 1] == major_minor[j * 2 + 1])) {
        whichGPUs[0] = i;
        whichGPUs[1] = j;
        found2IdenticalGPUs = 1;
        break;
      }
    }
    if (found2IdenticalGPUs) {
      break;
    }
  }

  free(major_minor);
  if (!found2IdenticalGPUs) {
    printf(
        "No Two GPUs with same architecture found\nWaiving simpleCUFFT_2d_MGPU "
        "sample\n");
    exit(EXIT_WAIVED);
  }

  // Allocate host memory for the signal
  Complex *h_signal = (Complex *)malloc(sizeof(Complex) * SIGNAL_SIZE);

  // Initialize the memory for the signal
  for (int i = 0; i < SIGNAL_SIZE; ++i) {
    h_signal[i].x = rand() / (float)RAND_MAX;
    h_signal[i].y = 0;
  }

  // Allocate host memory for the filter
  Complex *h_filter_kernel =
      (Complex *)malloc(sizeof(Complex) * FILTER_KERNEL_SIZE);

  // Initialize the memory for the filter
  for (int i = 0; i < FILTER_KERNEL_SIZE; ++i) {
    h_filter_kernel[i].x = rand() / (float)RAND_MAX;
    h_filter_kernel[i].y = 0;
  }

  // Pad signal and filter kernel
  Complex *h_padded_signal;
  Complex *h_padded_filter_kernel;
  int new_size =
      PadData(h_signal, &h_padded_signal, SIGNAL_SIZE, h_filter_kernel,
              &h_padded_filter_kernel, FILTER_KERNEL_SIZE);

  // hipfftCreate() - Create an empty plan
  hipfftResult result;
  hipfftHandle plan_input;
  checkCudaErrors(hipfftCreate(&plan_input));

  // hipfftXtSetGPUs() - Define which GPUs to use
  result = hipfftXtSetGPUs(plan_input, nGPUs, whichGPUs);

  if (result == HIPFFT_INVALID_DEVICE) {
    printf("This sample requires two GPUs on the same board.\n");
    printf("No such board was found. Waiving sample.\n");
    exit(EXIT_WAIVED);
  } else if (result != HIPFFT_SUCCESS) {
    printf("hipfftXtSetGPUs failed\n");
    exit(EXIT_FAILURE);
  }

  // Print the device information to run the code
  printf("\nRunning on GPUs\n");
  for (int i = 0; i < nGPUs; i++) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, whichGPUs[i]));
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n",
           whichGPUs[i], deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  size_t *worksize;
  worksize = (size_t *)malloc(sizeof(size_t) * nGPUs);

  // hipfftMakePlan1d() - Create the plan
  checkCudaErrors(
      hipfftMakePlan1d(plan_input, new_size, HIPFFT_C2C, 1, worksize));

  // hipfftXtMalloc() - Malloc data on multiple GPUs
  hipLibXtDesc *d_signal;
  checkCudaErrors(hipfftXtMalloc(plan_input, (hipLibXtDesc **)&d_signal,
                                HIPFFT_XT_FORMAT_INPLACE));
  hipLibXtDesc *d_out_signal;
  checkCudaErrors(hipfftXtMalloc(plan_input, (hipLibXtDesc **)&d_out_signal,
                                HIPFFT_XT_FORMAT_INPLACE));
  hipLibXtDesc *d_filter_kernel;
  checkCudaErrors(hipfftXtMalloc(plan_input, (hipLibXtDesc **)&d_filter_kernel,
                                HIPFFT_XT_FORMAT_INPLACE));
  hipLibXtDesc *d_out_filter_kernel;
  checkCudaErrors(hipfftXtMalloc(plan_input,
                                (hipLibXtDesc **)&d_out_filter_kernel,
                                HIPFFT_XT_FORMAT_INPLACE));

  // hipfftXtMemcpy() - Copy data from host to multiple GPUs
  checkCudaErrors(hipfftXtMemcpy(plan_input, d_signal, h_padded_signal,
                                HIPFFT_COPY_HOST_TO_DEVICE));
  checkCudaErrors(hipfftXtMemcpy(plan_input, d_filter_kernel,
                                h_padded_filter_kernel,
                                HIPFFT_COPY_HOST_TO_DEVICE));

  // hipfftXtExecDescriptorC2C() - Execute FFT on data on multiple GPUs
  checkCudaErrors(
      hipfftXtExecDescriptorC2C(plan_input, d_signal, d_signal, HIPFFT_FORWARD));
  checkCudaErrors(hipfftXtExecDescriptorC2C(plan_input, d_filter_kernel,
                                           d_filter_kernel, HIPFFT_FORWARD));

  // hipfftXtMemcpy() - Copy the data to natural order on GPUs
  checkCudaErrors(hipfftXtMemcpy(plan_input, d_out_signal, d_signal,
                                HIPFFT_COPY_DEVICE_TO_DEVICE));
  checkCudaErrors(hipfftXtMemcpy(plan_input, d_out_filter_kernel,
                                d_filter_kernel, HIPFFT_COPY_DEVICE_TO_DEVICE));

  printf("\n\nValue of Library Descriptor\n");
  printf("Number of GPUs %d\n", d_out_signal->descriptor->nGPUs);
  printf("Device id  %d %d\n", d_out_signal->descriptor->GPUs[0],
         d_out_signal->descriptor->GPUs[1]);
  printf("Data size on GPU %ld %ld\n",
         (long)(d_out_signal->descriptor->size[0] / sizeof(hipfftComplex)),
         (long)(d_out_signal->descriptor->size[1] / sizeof(hipfftComplex)));

  // Multiply the coefficients together and normalize the result
  printf("Launching ComplexPointwiseMulAndScale<<< >>>\n");
  multiplyCoefficient(d_out_signal, d_out_filter_kernel, new_size,
                      1.0f / new_size, nGPUs);

  // hipfftXtExecDescriptorC2C() - Execute inverse  FFT on data on multiple GPUs
  printf("Transforming signal back hipfftExecC2C\n");
  checkCudaErrors(hipfftXtExecDescriptorC2C(plan_input, d_out_signal,
                                           d_out_signal, HIPFFT_BACKWARD));

  // Create host pointer pointing to padded signal
  Complex *h_convolved_signal = h_padded_signal;

  // Allocate host memory for the convolution result
  Complex *h_convolved_signal_ref =
      (Complex *)malloc(sizeof(Complex) * SIGNAL_SIZE);

  // hipfftXtMemcpy() - Copy data from multiple GPUs to host
  checkCudaErrors(hipfftXtMemcpy(plan_input, h_convolved_signal, d_out_signal,
                                HIPFFT_COPY_DEVICE_TO_HOST));

  // Convolve on the host
  Convolve(h_signal, SIGNAL_SIZE, h_filter_kernel, FILTER_KERNEL_SIZE,
           h_convolved_signal_ref);

  // Compare CPU and GPU result
  bool bTestResult =
      sdkCompareL2fe((float *)h_convolved_signal_ref,
                     (float *)h_convolved_signal, 2 * SIGNAL_SIZE, 1e-5f);
  printf("\nvalue of TestResult %d\n", bTestResult);

  // Cleanup memory
  free(whichGPUs);
  free(worksize);
  free(h_signal);
  free(h_filter_kernel);
  free(h_padded_signal);
  free(h_padded_filter_kernel);
  free(h_convolved_signal_ref);

  // cudaXtFree() - Free GPU memory
  checkCudaErrors(hipfftXtFree(d_signal));
  checkCudaErrors(hipfftXtFree(d_filter_kernel));
  checkCudaErrors(hipfftXtFree(d_out_signal));
  checkCudaErrors(hipfftXtFree(d_out_filter_kernel));

  // hipfftDestroy() - Destroy FFT plan
  checkCudaErrors(hipfftDestroy(plan_input));

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

///////////////////////////////////////////////////////////////////////////////////
// Function for padding original data
//////////////////////////////////////////////////////////////////////////////////
int PadData(const Complex *signal, Complex **padded_signal, int signal_size,
            const Complex *filter_kernel, Complex **padded_filter_kernel,
            int filter_kernel_size) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;
  int new_size = signal_size + maxRadius;

  // Pad signal
  Complex *new_data = (Complex *)malloc(sizeof(Complex) * new_size);
  memcpy(new_data + 0, signal, signal_size * sizeof(Complex));
  memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(Complex));
  *padded_signal = new_data;

  // Pad filter
  new_data = (Complex *)malloc(sizeof(Complex) * new_size);
  memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(Complex));
  memset(new_data + maxRadius, 0,
         (new_size - filter_kernel_size) * sizeof(Complex));
  memcpy(new_data + new_size - minRadius, filter_kernel,
         minRadius * sizeof(Complex));
  *padded_filter_kernel = new_data;

  return new_size;
}

////////////////////////////////////////////////////////////////////////////////
// Filtering operations - Computing Convolution on the host
////////////////////////////////////////////////////////////////////////////////
void Convolve(const Complex *signal, int signal_size,
              const Complex *filter_kernel, int filter_kernel_size,
              Complex *filtered_signal) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;

  // Loop over output element indices
  for (int i = 0; i < signal_size; ++i) {
    filtered_signal[i].x = filtered_signal[i].y = 0;

    // Loop over convolution indices
    for (int j = -maxRadius + 1; j <= minRadius; ++j) {
      int k = i + j;

      if (k >= 0 && k < signal_size) {
        filtered_signal[i] =
            ComplexAdd(filtered_signal[i],
                       ComplexMul(signal[k], filter_kernel[minRadius - j]));
      }
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
//  Launch Kernel on multiple GPU
////////////////////////////////////////////////////////////////////////////////
void multiplyCoefficient(hipLibXtDesc *d_signal,
                         hipLibXtDesc *d_filter_kernel, int new_size,
                         float val, int nGPUs) {
  int device;
  // Launch the ComplexPointwiseMulAndScale<<< >>> kernel on multiple GPU
  for (int i = 0; i < nGPUs; i++) {
    device = d_signal->descriptor->GPUs[i];

    // Set device
    checkCudaErrors(hipSetDevice(device));

    // Perform GPU computations
    ComplexPointwiseMulAndScale<<<32, 256>>>(
        (hipfftComplex *)d_signal->descriptor->data[i],
        (hipfftComplex *)d_filter_kernel->descriptor->data[i],
        int(d_signal->descriptor->size[i] / sizeof(hipfftComplex)), val);
  }

  // Wait for device to finish all operation
  for (int i = 0; i < nGPUs; i++) {
    device = d_signal->descriptor->GPUs[i];
    checkCudaErrors(hipSetDevice(device));
    hipDeviceSynchronize();
    // Check if kernel execution generated and error
    getLastCudaError("Kernel execution failed [ ComplexPointwiseMulAndScale ]");
  }
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
  Complex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
  Complex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}
// Complex pointwise multiplication
static __global__ void ComplexPointwiseMulAndScale(hipfftComplex *a,
                                                   hipfftComplex *b, int size,
                                                   float scale) {
  const int numThreads = blockDim.x * gridDim.x;
  const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
  for (int i = threadID; i < size; i += numThreads) {
    a[i] = ComplexScale(ComplexMul(a[i], b[i]), scale);
  }
}
