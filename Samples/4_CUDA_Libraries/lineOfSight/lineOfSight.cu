#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// This sample is an implementation of a simple line-of-sight algorithm:
// Given a height map and a ray originating at some observation point,
// it computes all the points along the ray that are visible from the
// observation point.
// It is based on the description made in "Guy E. Blelloch.  Vector models
// for data-parallel computing. MIT Press, 1990" and uses open source CUDA
// Thrust Library

#ifdef _WIN32
#define NOMINMAX
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

// includes, library
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/scan.h>
#include <thrust/copy.h>

////////////////////////////////////////////////////////////////////////////////
// declaration, types

// Boolean
typedef unsigned char Bool;
enum { False = 0, True = 1 };

// 2D height field
struct HeightField {
  int width;
  float *height;
};

// Ray
struct Ray {
  float3 origin;
  float2 dir;
  int length;
  float oneOverLength;
};

////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest(int argc, char **argv);
__global__ void computeAngles_kernel(const Ray, float *, hipTextureObject_t);
__global__ void computeVisibilities_kernel(const float *, const float *, int,
                                           Bool *);
void lineOfSight_gold(const HeightField, const Ray, Bool *);
__device__ __host__ float2 getLocation(const Ray, int);
__device__ __host__ float getAngle(const Ray, float2, float);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  int res = runTest(argc, argv);

  if (res != 1) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a line-of-sight test for CUDA
////////////////////////////////////////////////////////////////////////////////
int runTest(int argc, char **argv) {
  ////////////////////////////////////////////////////////////////////////////
  // Device initialization

  printf("[%s] - Starting...\n", argv[0]);

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  findCudaDevice(argc, (const char **)argv);

  ////////////////////////////////////////////////////////////////////////////
  // Timer

  // Create
  StopWatchInterface *timer;
  sdkCreateTimer(&timer);

  // Number of iterations to get accurate timing
  uint numIterations = 100;

  ////////////////////////////////////////////////////////////////////////////
  // Height field

  HeightField heightField;

  // Allocate in host memory
  int2 dim = make_int2(10000, 100);
  heightField.width = dim.x;
  thrust::host_vector<float> height(dim.x * dim.y);
  heightField.height = (float *)&height[0];

  //
  // Fill in with an arbitrary sine surface
  for (int x = 0; x < dim.x; ++x)
    for (int y = 0; y < dim.y; ++y) {
      float amp = 0.1f * (x + y);
      float period = 2.0f + amp;
      *(heightField.height + dim.x * y + x) =
          amp * (sinf(sqrtf((float)(x * x + y * y)) * 2.0f * 3.1416f / period) +
                 1.0f);
    }

  // Allocate CUDA array in device memory
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray *heightFieldArray;
  checkCudaErrors(
      hipMallocArray(&heightFieldArray, &channelDesc, dim.x, dim.y));

  // Initialize device memory
  checkCudaErrors(hipMemcpy2DToArray(
      heightFieldArray, 0, 0, heightField.height, dim.x * sizeof(float),
      dim.x * sizeof(float), dim.y, hipMemcpyHostToDevice));

  hipTextureObject_t heightFieldTex;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = heightFieldArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));
  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.addressMode[1] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&heightFieldTex, &texRes, &texDescr, NULL));

  //////////////////////////////////////////////////////////////////////////////
  // Ray (starts at origin and traverses the height field diagonally)

  Ray ray;
  ray.origin = make_float3(0, 0, 2.0f);
  int2 dir = make_int2(dim.x - 1, dim.y - 1);
  ray.dir = make_float2((float)dir.x, (float)dir.y);
  ray.length = max(abs(dir.x), abs(dir.y));
  ray.oneOverLength = 1.0f / ray.length;

  //////////////////////////////////////////////////////////////////////////////
  // View angles

  // Allocate view angles for each point along the ray
  thrust::device_vector<float> d_angles(ray.length);

  // Allocate result of max-scan operation on the array of view angles
  thrust::device_vector<float> d_scannedAngles(ray.length);

  //////////////////////////////////////////////////////////////////////////////
  // Visibility results

  // Allocate visibility results for each point along the ray
  thrust::device_vector<Bool> d_visibilities(ray.length);
  thrust::host_vector<Bool> h_visibilities(ray.length);
  thrust::host_vector<Bool> h_visibilitiesRef(ray.length);

  //////////////////////////////////////////////////////////////////////////////
  // Reference solution
  lineOfSight_gold(heightField, ray, (Bool *)&h_visibilitiesRef[0]);

  //////////////////////////////////////////////////////////////////////////////
  // Device solution

  // Execution configuration
  dim3 block(256);
  dim3 grid((uint)ceil(ray.length / (double)block.x));

  // Compute device solution
  printf("Line of sight\n");
  sdkStartTimer(&timer);

  for (uint i = 0; i < numIterations; ++i) {
    // Compute view angle for each point along the ray
    computeAngles_kernel<<<grid, block>>>(
        ray, thrust::raw_pointer_cast(&d_angles[0]), heightFieldTex);
    getLastCudaError("Kernel execution failed");

    // Perform a max-scan operation on the array of view angles
    thrust::inclusive_scan(d_angles.begin(), d_angles.end(),
                           d_scannedAngles.begin(), thrust::maximum<float>());
    getLastCudaError("Kernel execution failed");

    // Compute visibility results based on the array of view angles
    // and its scanned version
    computeVisibilities_kernel<<<grid, block>>>(
        thrust::raw_pointer_cast(&d_angles[0]),
        thrust::raw_pointer_cast(&d_scannedAngles[0]), ray.length,
        thrust::raw_pointer_cast(&d_visibilities[0]));
    getLastCudaError("Kernel execution failed");
  }

  hipDeviceSynchronize();
  sdkStopTimer(&timer);
  getLastCudaError("Kernel execution failed");

  // Copy visibility results back to the host
  thrust::copy(d_visibilities.begin(), d_visibilities.end(),
               h_visibilities.begin());

  // Compare device visibility results against reference results
  bool res = compareData(thrust::raw_pointer_cast(&h_visibilitiesRef[0]),
                         thrust::raw_pointer_cast(&h_visibilities[0]),
                         ray.length, 0.0f, 0.0f);
  printf("Average time: %f ms\n\n", sdkGetTimerValue(&timer) / numIterations);
  sdkResetTimer(&timer);

  // Cleanup memory
  checkCudaErrors(hipFreeArray(heightFieldArray));
  return res;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute view angles for each point along the ray
//! @param ray         ray
//! @param angles      view angles
////////////////////////////////////////////////////////////////////////////////
__global__ void computeAngles_kernel(const Ray ray, float *angles,
                                     hipTextureObject_t HeightFieldTex) {
  uint i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < ray.length) {
    float2 location = getLocation(ray, i + 1);
    float height = tex2D<float>(HeightFieldTex, location.x, location.y);
    float angle = getAngle(ray, location, height);
    angles[i] = angle;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute visibility for each point along the ray
//! @param angles          view angles
//! @param scannedAngles   max-scanned view angles
//! @param numAngles       number of view angles
//! @param visibilities    boolean array indicating the visibility of each point
//!                        along the ray
////////////////////////////////////////////////////////////////////////////////
__global__ void computeVisibilities_kernel(const float *angles,
                                           const float *scannedAngles,
                                           int numAngles, Bool *visibilities) {
  uint i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numAngles) {
    visibilities[i] = scannedAngles[i] <= angles[i];
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set
//! @param heightField     height field
//! @param ray             ray
//! @param visibilities    boolean array indicating the visibility of each point
//!                        along the ray
////////////////////////////////////////////////////////////////////////////////
void lineOfSight_gold(const HeightField heightField, const Ray ray,
                      Bool *visibilities) {
  float angleMax = asinf(-1.0f);

  for (int i = 0; i < ray.length; ++i) {
    float2 location = getLocation(ray, i + 1);
    float height =
        *(heightField.height + heightField.width * (int)floorf(location.y) +
          (int)floorf(location.x));
    float angle = getAngle(ray, location, height);

    if (angle > angleMax) {
      angleMax = angle;
      visibilities[i] = True;
    } else {
      visibilities[i] = False;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the 2D coordinates of the point located at i steps from the origin
//! of the ray
//! @param ray      ray
//! @param i        integer offset along the ray
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float2 getLocation(const Ray ray, int i) {
  float step = i * ray.oneOverLength;
  return make_float2(ray.origin.x, ray.origin.y) + ray.dir * step;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the angle of view between a 3D point and the origin of the ray
//! @param ray        ray
//! @param location   2D coordinates of the input point
//! @param height     height of the input point
////////////////////////////////////////////////////////////////////////////////
__device__ __host__ float getAngle(const Ray ray, float2 location,
                                   float height) {
  float2 dir = location - make_float2(ray.origin.x, ray.origin.y);
  return atanf((height - ray.origin.z) / length(dir));
}
