#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


/* 
 * Example showing the use of CUFFT for fast 1D-convolution using FFT. 
 * This sample is the same as simpleCUFFT, except that it uses a callback
 * function to perform the pointwise multiply and scale, on input to the
 * inverse transform.
 * 
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Complex data type
typedef float2 Complex;
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);

// This is the callback routine prototype
static __device__ hipfftComplex ComplexPointwiseMulAndScale(void *a,
                                                           size_t index,
                                                           void *cb_info,
                                                           void *sharedmem);

typedef struct _cb_params {
  Complex *filter;
  float scale;
} cb_params;

// This is the callback routine. It does complex pointwise multiplication with
// scaling.
static __device__ hipfftComplex ComplexPointwiseMulAndScale(void *a,
                                                           size_t index,
                                                           void *cb_info,
                                                           void *sharedmem) {
  cb_params *my_params = (cb_params *)cb_info;
  return (hipfftComplex)ComplexScale(
      ComplexMul(((Complex *)a)[index], (my_params->filter)[index]),
      my_params->scale);
}

// Define the device pointer to the callback routine. The host code will fetch
// this and pass it to CUFFT
__device__ hipfftCallbackLoadC myOwnCallbackPtr = ComplexPointwiseMulAndScale;
// Filtering functions
void Convolve(const Complex *, int, const Complex *, int, Complex *);

// Padding functions
int PadData(const Complex *, Complex **, int, const Complex *, Complex **, int);

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest(int argc, char **argv);

// The filter size is assumed to be a number smaller than the signal size
#define SIGNAL_SIZE 50
#define FILTER_KERNEL_SIZE 11

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  struct hipDeviceProp_t properties;
  int device;
  checkCudaErrors(hipGetDevice(&device));
  checkCudaErrors(hipGetDeviceProperties(&properties, device));
  if (!(properties.major >= 2)) {
    printf("simpleCUFFT_callback requires CUDA architecture SM2.0 or higher\n");
    return EXIT_WAIVED;
  }

  return runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUFFT callbacks
////////////////////////////////////////////////////////////////////////////////
int runTest(int argc, char **argv) {
  printf("[simpleCUFFT_callback] is starting...\n");

  findCudaDevice(argc, (const char **)argv);

  // Allocate host memory for the signal
  Complex *h_signal = (Complex *)malloc(sizeof(Complex) * SIGNAL_SIZE);

  // Initialize the memory for the signal
  for (unsigned int i = 0; i < SIGNAL_SIZE; ++i) {
    h_signal[i].x = rand() / (float)RAND_MAX;
    h_signal[i].y = 0;
  }

  // Allocate host memory for the filter
  Complex *h_filter_kernel =
      (Complex *)malloc(sizeof(Complex) * FILTER_KERNEL_SIZE);

  // Initialize the memory for the filter
  for (unsigned int i = 0; i < FILTER_KERNEL_SIZE; ++i) {
    h_filter_kernel[i].x = rand() / (float)RAND_MAX;
    h_filter_kernel[i].y = 0;
  }

  // Pad signal and filter kernel
  Complex *h_padded_signal;
  Complex *h_padded_filter_kernel;
  int new_size =
      PadData(h_signal, &h_padded_signal, SIGNAL_SIZE, h_filter_kernel,
              &h_padded_filter_kernel, FILTER_KERNEL_SIZE);
  int mem_size = sizeof(Complex) * new_size;

  // Allocate device memory for signal
  Complex *d_signal;
  checkCudaErrors(hipMalloc((void **)&d_signal, mem_size));
  // Copy host memory to device
  checkCudaErrors(
      hipMemcpy(d_signal, h_padded_signal, mem_size, hipMemcpyHostToDevice));

  // Allocate device memory for filter kernel
  Complex *d_filter_kernel;
  checkCudaErrors(hipMalloc((void **)&d_filter_kernel, mem_size));

  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_filter_kernel, h_padded_filter_kernel, mem_size,
                             hipMemcpyHostToDevice));

  // Create one CUFFT plan for the forward transforms, and one for the reverse
  // transform with load callback.
  hipfftHandle plan, cb_plan;
  size_t work_size;

  checkCudaErrors(hipfftCreate(&plan));
  checkCudaErrors(hipfftCreate(&cb_plan));

  checkCudaErrors(hipfftMakePlan1d(plan, new_size, HIPFFT_C2C, 1, &work_size));
  checkCudaErrors(hipfftMakePlan1d(cb_plan, new_size, HIPFFT_C2C, 1, &work_size));

  // Define a structure used to pass in the device address of the filter kernel,
  // and the scale factor
  cb_params h_params;

  h_params.filter = d_filter_kernel;
  h_params.scale = 1.0f / new_size;

  // Allocate device memory for parameters
  cb_params *d_params;
  checkCudaErrors(hipMalloc((void **)&d_params, sizeof(cb_params)));

  // Copy host memory to device
  checkCudaErrors(hipMemcpy(d_params, &h_params, sizeof(cb_params),
                             hipMemcpyHostToDevice));

  // The host needs to get a copy of the device pointer to the callback
  hipfftCallbackLoadC hostCopyOfCallbackPtr;

  checkCudaErrors(hipMemcpyFromSymbol(&hostCopyOfCallbackPtr, HIP_SYMBOL(myOwnCallbackPtr),
                                       sizeof(hostCopyOfCallbackPtr)));

  // Now associate the load callback with the plan.
  hipfftResult status =
      hipfftXtSetCallback(cb_plan, (void **)&hostCopyOfCallbackPtr,
                         HIPFFT_CB_LD_COMPLEX, (void **)&d_params);
  if (status == CUFFT_LICENSE_ERROR) {
    printf("This sample requires a valid license file.\n");
    printf(
        "The file was either not found, out of date, or otherwise invalid.\n");
    return EXIT_WAIVED;
  }

  checkCudaErrors(hipfftXtSetCallback(cb_plan, (void **)&hostCopyOfCallbackPtr,
                                     HIPFFT_CB_LD_COMPLEX, (void **)&d_params));

  // Transform signal and kernel
  printf("Transforming signal hipfftExecC2C\n");
  checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_signal,
                               (hipfftComplex *)d_signal, HIPFFT_FORWARD));
  checkCudaErrors(hipfftExecC2C(plan, (hipfftComplex *)d_filter_kernel,
                               (hipfftComplex *)d_filter_kernel, HIPFFT_FORWARD));

  // Transform signal back, using the callback to do the pointwise multiply on
  // the way in.
  printf("Transforming signal back hipfftExecC2C\n");
  checkCudaErrors(hipfftExecC2C(cb_plan, (hipfftComplex *)d_signal,
                               (hipfftComplex *)d_signal, HIPFFT_BACKWARD));

  // Copy device memory to host
  Complex *h_convolved_signal = h_padded_signal;
  checkCudaErrors(hipMemcpy(h_convolved_signal, d_signal, mem_size,
                             hipMemcpyDeviceToHost));

  // Allocate host memory for the convolution result
  Complex *h_convolved_signal_ref =
      (Complex *)malloc(sizeof(Complex) * SIGNAL_SIZE);

  // Convolve on the host
  Convolve(h_signal, SIGNAL_SIZE, h_filter_kernel, FILTER_KERNEL_SIZE,
           h_convolved_signal_ref);

  // check result
  bool bTestResult =
      sdkCompareL2fe((float *)h_convolved_signal_ref,
                     (float *)h_convolved_signal, 2 * SIGNAL_SIZE, 1e-5f);

  // Destroy CUFFT context
  checkCudaErrors(hipfftDestroy(plan));
  checkCudaErrors(hipfftDestroy(cb_plan));

  // cleanup memory
  free(h_signal);
  free(h_filter_kernel);
  free(h_padded_signal);
  free(h_padded_filter_kernel);
  free(h_convolved_signal_ref);
  checkCudaErrors(hipFree(d_signal));
  checkCudaErrors(hipFree(d_filter_kernel));
  checkCudaErrors(hipFree(d_params));

  return bTestResult ? EXIT_SUCCESS : EXIT_FAILURE;
}

// Pad data
int PadData(const Complex *signal, Complex **padded_signal, int signal_size,
            const Complex *filter_kernel, Complex **padded_filter_kernel,
            int filter_kernel_size) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;
  int new_size = signal_size + maxRadius;

  // Pad signal
  Complex *new_data = (Complex *)malloc(sizeof(Complex) * new_size);
  memcpy(new_data + 0, signal, signal_size * sizeof(Complex));
  memset(new_data + signal_size, 0, (new_size - signal_size) * sizeof(Complex));
  *padded_signal = new_data;

  // Pad filter
  new_data = (Complex *)malloc(sizeof(Complex) * new_size);
  memcpy(new_data + 0, filter_kernel + minRadius, maxRadius * sizeof(Complex));
  memset(new_data + maxRadius, 0,
         (new_size - filter_kernel_size) * sizeof(Complex));
  memcpy(new_data + new_size - minRadius, filter_kernel,
         minRadius * sizeof(Complex));
  *padded_filter_kernel = new_data;

  return new_size;
}

////////////////////////////////////////////////////////////////////////////////
// Filtering operations
////////////////////////////////////////////////////////////////////////////////

// Computes convolution on the host
void Convolve(const Complex *signal, int signal_size,
              const Complex *filter_kernel, int filter_kernel_size,
              Complex *filtered_signal) {
  int minRadius = filter_kernel_size / 2;
  int maxRadius = filter_kernel_size - minRadius;

  // Loop over output element indices
  for (int i = 0; i < signal_size; ++i) {
    filtered_signal[i].x = filtered_signal[i].y = 0;

    // Loop over convolution indices
    for (int j = -maxRadius + 1; j <= minRadius; ++j) {
      int k = i + j;

      if (k >= 0 && k < signal_size) {
        filtered_signal[i] =
            ComplexAdd(filtered_signal[i],
                       ComplexMul(signal[k], filter_kernel[minRadius - j]));
      }
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
  Complex c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
  Complex c;
  c.x = s * a.x;
  c.y = s * a.y;
  return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
  Complex c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}
