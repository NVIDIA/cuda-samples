#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on GPU using
 * Multi Block Cooperative Groups, also uses Unified Memory.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

const char *sSDKname = "conjugateGradientMultiBlockCG";

#define ENABLE_CPU_DEBUG_CODE 0
#define THREADS_PER_BLOCK 512

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz) {
  I[0] = 0, J[0] = 0, J[1] = 1;
  val[0] = static_cast<float>(rand()) / RAND_MAX + 10.0f;
  val[1] = static_cast<float>(rand()) / RAND_MAX;
  int start;

  for (int i = 1; i < N; i++) {
    if (i > 1) {
      I[i] = I[i - 1] + 3;
    } else {
      I[1] = 2;
    }

    start = (i - 1) * 3 + 2;
    J[start] = i - 1;
    J[start + 1] = i;

    if (i < N - 1) {
      J[start + 2] = i + 1;
    }

    val[start] = val[start - 1];
    val[start + 1] = static_cast<float>(rand()) / RAND_MAX + 10.0f;

    if (i < N - 1) {
      val[start + 2] = static_cast<float>(rand()) / RAND_MAX;
    }
  }

  I[N] = nz;
}

// I - contains location of the given non-zero element in the row of the matrix
// J - contains location of the given non-zero element in the column of the
// matrix val - contains values of the given non-zero elements of the matrix
// inputVecX - input vector to be multiplied
// outputVecY - resultant vector
void cpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha,
             float *inputVecX, float *outputVecY) {
  for (int i = 0; i < num_rows; i++) {
    int num_elems_this_row = I[i + 1] - I[i];

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++) {
      output += alpha * val[I[i] + j] * inputVecX[J[I[i] + j]];
    }
    outputVecY[i] = output;
  }

  return;
}

double dotProduct(float *vecA, float *vecB, int size) {
  double result = 0.0;

  for (int i = 0; i < size; i++) {
    result = result + (vecA[i] * vecB[i]);
  }

  return result;
}

void scaleVector(float *vec, float alpha, int size) {
  for (int i = 0; i < size; i++) {
    vec[i] = alpha * vec[i];
  }
}

void saxpy(float *x, float *y, float a, int size) {
  for (int i = 0; i < size; i++) {
    y[i] = a * x[i] + y[i];
  }
}

void cpuConjugateGrad(int *I, int *J, float *val, float *x, float *Ax, float *p,
                      float *r, int nnz, int N, float tol) {
  int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, b, a, na;

  cpuSpMV(I, J, val, nnz, N, alpha, x, Ax);
  saxpy(Ax, r, alpham1, N);

  float r1 = dotProduct(r, r, N);

  int k = 1;

  while (r1 > tol * tol && k <= max_iter) {
    if (k > 1) {
      b = r1 / r0;
      scaleVector(p, b, N);

      saxpy(r, p, alpha, N);
    } else {
      for (int i = 0; i < N; i++) p[i] = r[i];
    }

    cpuSpMV(I, J, val, nnz, N, alpha, p, Ax);

    float dot = dotProduct(p, Ax, N);
    a = r1 / dot;

    saxpy(p, x, a, N);
    na = -a;
    saxpy(Ax, r, na, N);

    r0 = r1;
    r1 = dotProduct(r, r, N);

    printf("\nCPU code iteration = %3d, residual = %e\n", k, sqrt(r1));
    k++;
  }
}

__device__ void gpuSpMV(int *I, int *J, float *val, int nnz, int num_rows,
                        float alpha, float *inputVecX, float *outputVecY,
                        cg::thread_block &cta, const cg::grid_group &grid) {
  for (int i = grid.thread_rank(); i < num_rows; i += grid.size()) {
    int row_elem = I[i];
    int next_row_elem = I[i + 1];
    int num_elems_this_row = next_row_elem - row_elem;

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++) {
      // I or J or val arrays - can be put in shared memory
      // as the access is random and reused in next calls of gpuSpMV function.
      output += alpha * val[row_elem + j] * inputVecX[J[row_elem + j]];
    }

    outputVecY[i] = output;
  }
}

__device__ void gpuSaxpy(float *x, float *y, float a, int size,
                         const cg::grid_group &grid) {
  for (int i = grid.thread_rank(); i < size; i += grid.size()) {
    y[i] = a * x[i] + y[i];
  }
}

__device__ void gpuDotProduct(float *vecA, float *vecB, double *result,
                              int size, const cg::thread_block &cta,
                              const cg::grid_group &grid) {
  extern __shared__ double tmp[];

  double temp_sum = 0.0;
  for (int i = grid.thread_rank(); i < size; i += grid.size()) {
    temp_sum += static_cast<double>(vecA[i] * vecB[i]);
  }

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

  if (tile32.thread_rank() == 0) {
    tmp[tile32.meta_group_rank()] = temp_sum;
  }

  cg::sync(cta);

  if (tile32.meta_group_rank() == 0) {
     temp_sum = tile32.thread_rank() < tile32.meta_group_size() ? tmp[tile32.thread_rank()] : 0.0;
     temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
      atomicAdd(result, temp_sum);
    }
  }
}

__device__ void gpuCopyVector(float *srcA, float *destB, int size,
                              const cg::grid_group &grid) {
  for (int i = grid.thread_rank(); i < size; i += grid.size()) {
    destB[i] = srcA[i];
  }
}

__device__ void gpuScaleVectorAndSaxpy(const float *x, float *y, float a, float scale, int size,
                         const cg::grid_group &grid) {
  for (int i = grid.thread_rank(); i < size; i += grid.size()) {
    y[i] = a * x[i] + scale * y[i];
  }
}

extern "C" __global__ void gpuConjugateGradient(int *I, int *J, float *val,
                                                float *x, float *Ax, float *p,
                                                float *r, double *dot_result,
                                                int nnz, int N, float tol) {
  cg::thread_block cta = cg::this_thread_block();
  cg::grid_group grid = cg::this_grid();

  int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, r1, b, a, na;

  gpuSpMV(I, J, val, nnz, N, alpha, x, Ax, cta, grid);

  cg::sync(grid);

  gpuSaxpy(Ax, r, alpham1, N, grid);

  cg::sync(grid);

  gpuDotProduct(r, r, dot_result, N, cta, grid);

  cg::sync(grid);

  r1 = *dot_result;

  int k = 1;
  while (r1 > tol * tol && k <= max_iter) {
    if (k > 1) {
      b = r1 / r0;
      gpuScaleVectorAndSaxpy(r, p, alpha, b, N, grid);
    } else {
      gpuCopyVector(r, p, N, grid);
    }

    cg::sync(grid);

    gpuSpMV(I, J, val, nnz, N, alpha, p, Ax, cta, grid);

    if (threadIdx.x == 0 && blockIdx.x == 0) *dot_result = 0.0;

    cg::sync(grid);

    gpuDotProduct(p, Ax, dot_result, N, cta, grid);

    cg::sync(grid);

    a = r1 / *dot_result;

    gpuSaxpy(p, x, a, N, grid);
    na = -a;
    gpuSaxpy(Ax, r, na, N, grid);

    r0 = r1;

    cg::sync(grid);
    if (threadIdx.x == 0 && blockIdx.x == 0) *dot_result = 0.0;

    cg::sync(grid);

    gpuDotProduct(r, r, dot_result, N, cta, grid);

    cg::sync(grid);

    r1 = *dot_result;
    k++;
  }
}

bool areAlmostEqual(float a, float b, float maxRelDiff) {
  float diff = fabsf(a - b);
  float abs_a = fabsf(a);
  float abs_b = fabsf(b);
  float largest = abs_a > abs_b ? abs_a : abs_b;

  if (diff <= largest * maxRelDiff) {
    return true;
  } else {
    printf("maxRelDiff = %.8e\n", maxRelDiff);
    printf(
        "diff %.8e > largest * maxRelDiff %.8e therefore %.8e and %.8e are not "
        "same\n",
        diff, largest * maxRelDiff, a, b);
    return false;
  }
}

int main(int argc, char **argv) {
  int N = 0, nz = 0, *I = NULL, *J = NULL;
  float *val = NULL;
  const float tol = 1e-5f;
  float *x;
  float *rhs;
  float r1;
  float *r, *p, *Ax;
  hipEvent_t start, stop;

  printf("Starting [%s]...\n", sSDKname);

  // This will pick the best possible CUDA capable device
  hipDeviceProp_t deviceProp;
  int devID = findCudaDevice(argc, (const char **)argv);
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

  if (!deviceProp.managedMemory) {
    // This sample requires being run on a device that supports Unified Memory
    fprintf(stderr, "Unified Memory not supported on this device\n");
    exit(EXIT_WAIVED);
  }

  // This sample requires being run on a device that supports Cooperative Kernel
  // Launch
  if (!deviceProp.cooperativeLaunch) {
    printf(
        "\nSelected GPU (%d) does not support Cooperative Kernel Launch, "
        "Waiving the run\n",
        devID);
    exit(EXIT_WAIVED);
  }

  // Statistics about the GPU device
  printf(
      "> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
      deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

  /* Generate a random tridiagonal symmetric matrix in CSR format */
  N = 1048576;
  nz = (N - 2) * 3 + 4;

  hipMallocManaged(reinterpret_cast<void **>(&I), sizeof(int) * (N + 1));
  hipMallocManaged(reinterpret_cast<void **>(&J), sizeof(int) * nz);
  hipMallocManaged(reinterpret_cast<void **>(&val), sizeof(float) * nz);

  genTridiag(I, J, val, N, nz);

  hipMallocManaged(reinterpret_cast<void **>(&x), sizeof(float) * N);
  hipMallocManaged(reinterpret_cast<void **>(&rhs), sizeof(float) * N);

  double *dot_result;

  hipMallocManaged(reinterpret_cast<void **>(&dot_result), sizeof(double));

  *dot_result = 0.0;

  // temp memory for CG
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&r), N * sizeof(float)));
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&p), N * sizeof(float)));
  checkCudaErrors(
      hipMallocManaged(reinterpret_cast<void **>(&Ax), N * sizeof(float)));

  hipDeviceSynchronize();

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

#if ENABLE_CPU_DEBUG_CODE
  float *Ax_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  float *r_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  float *p_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * N));
  float *x_cpu = reinterpret_cast<float *>(malloc(sizeof(float) * N));

  for (int i = 0; i < N; i++) {
    r_cpu[i] = 1.0;
    Ax_cpu[i] = x_cpu[i] = 0.0;
  }

#endif

  for (int i = 0; i < N; i++) {
    r[i] = rhs[i] = 1.0;
    x[i] = 0.0;
  }

  void *kernelArgs[] = {
      (void *)&I,  (void *)&J, (void *)&val, (void *)&x,
      (void *)&Ax, (void *)&p, (void *)&r,   (void *)&dot_result,
      (void *)&nz, (void *)&N, (void *)&tol,
  };

  int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK/32) + 1);
  int numBlocksPerSm = 0;
  int numThreads = THREADS_PER_BLOCK;

  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &numBlocksPerSm, gpuConjugateGradient, numThreads, sMemSize));

  int numSms = deviceProp.multiProcessorCount;
  dim3 dimGrid(numSms * numBlocksPerSm, 1, 1),
      dimBlock(THREADS_PER_BLOCK, 1, 1);
  checkCudaErrors(hipEventRecord(start, 0));
  checkCudaErrors(hipLaunchCooperativeKernel((void *)gpuConjugateGradient,
                                              dimGrid, dimBlock, kernelArgs,
                                              sMemSize, NULL));
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipDeviceSynchronize());

  float time;
  checkCudaErrors(hipEventElapsedTime(&time, start, stop));

  r1 = *dot_result;

  printf("GPU Final, residual = %e, kernel execution time = %f ms\n", sqrt(r1),
         time);

#if ENABLE_CPU_DEBUG_CODE
  cpuConjugateGrad(I, J, val, x_cpu, Ax_cpu, p_cpu, r_cpu, nz, N, tol);
#endif

  float rsum, diff, err = 0.0;

  for (int i = 0; i < N; i++) {
    rsum = 0.0;

    for (int j = I[i]; j < I[i + 1]; j++) {
      rsum += val[j] * x[J[j]];
    }

    diff = fabs(rsum - rhs[i]);

    if (diff > err) {
      err = diff;
    }
  }

  checkCudaErrors(hipFree(I));
  checkCudaErrors(hipFree(J));
  checkCudaErrors(hipFree(val));
  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(rhs));
  checkCudaErrors(hipFree(r));
  checkCudaErrors(hipFree(p));
  checkCudaErrors(hipFree(Ax));
  checkCudaErrors(hipFree(dot_result));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

#if ENABLE_CPU_DEBUG_CODE
  free(Ax_cpu);
  free(r_cpu);
  free(p_cpu);
  free(x_cpu);
#endif

  printf("Test Summary:  Error amount = %f \n", err);
  fprintf(stdout, "&&&& conjugateGradientMultiBlockCG %s\n",
          (sqrt(r1) < tol) ? "PASSED" : "FAILED");
  exit((sqrt(r1) < tol) ? EXIT_SUCCESS : EXIT_FAILURE);
}
