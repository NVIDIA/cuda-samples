/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "cudla.h"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/stat.h>
#include <fstream>
#include <sstream>

#define DPRINTF(...) printf(__VA_ARGS__)

static void printTensorDesc(cudlaModuleTensorDescriptor* tensorDesc) {
  DPRINTF("\tTENSOR NAME : %s\n", tensorDesc->name);
  DPRINTF("\tsize: %lu\n", tensorDesc->size);

  DPRINTF("\tdims: [%lu, %lu, %lu, %lu]\n", tensorDesc->n, tensorDesc->c,
          tensorDesc->h, tensorDesc->w);

  DPRINTF("\tdata fmt: %d\n", tensorDesc->dataFormat);
  DPRINTF("\tdata type: %d\n", tensorDesc->dataType);
  DPRINTF("\tdata category: %d\n", tensorDesc->dataCategory);
  DPRINTF("\tpixel fmt: %d\n", tensorDesc->pixelFormat);
  DPRINTF("\tpixel mapping: %d\n", tensorDesc->pixelMapping);
  DPRINTF("\tstride[0]: %d\n", tensorDesc->stride[0]);
  DPRINTF("\tstride[1]: %d\n", tensorDesc->stride[1]);
  DPRINTF("\tstride[2]: %d\n", tensorDesc->stride[2]);
  DPRINTF("\tstride[3]: %d\n", tensorDesc->stride[3]);
}

static int initializeInputBuffers(char* filePath,
                                  cudlaModuleTensorDescriptor* tensorDesc,
                                  unsigned char* buf) {
  // Read the file in filePath and fill up 'buf' according to format
  // specified by the user.

  return 0;
}

typedef struct {
  cudlaDevHandle devHandle;
  cudlaModule moduleHandle;
  unsigned char* loadableData;
  hipStream_t stream;
  unsigned char* inputBuffer;
  unsigned char* outputBuffer;
  void* inputBufferGPU;
  void* outputBufferGPU;
  cudlaModuleTensorDescriptor* inputTensorDesc;
  cudlaModuleTensorDescriptor* outputTensorDesc;
} ResourceList;

void cleanUp(ResourceList* resourceList);

void cleanUp(ResourceList* resourceList) {
  if (resourceList->inputTensorDesc != NULL) {
    free(resourceList->inputTensorDesc);
    resourceList->inputTensorDesc = NULL;
  }
  if (resourceList->outputTensorDesc != NULL) {
    free(resourceList->outputTensorDesc);
    resourceList->outputTensorDesc = NULL;
  }

  if (resourceList->loadableData != NULL) {
    free(resourceList->loadableData);
    resourceList->loadableData = NULL;
  }

  if (resourceList->moduleHandle != NULL) {
    cudlaModuleUnload(resourceList->moduleHandle, 0);
    resourceList->moduleHandle = NULL;
  }

  if (resourceList->devHandle != NULL) {
    cudlaDestroyDevice(resourceList->devHandle);
    resourceList->devHandle = NULL;
  }

  if (resourceList->inputBufferGPU != 0) {
    hipFree(resourceList->inputBufferGPU);
    resourceList->inputBufferGPU = 0;
  }
  if (resourceList->outputBufferGPU != 0) {
    hipFree(resourceList->outputBufferGPU);
    resourceList->outputBufferGPU = 0;
  }

  if (resourceList->inputBuffer != NULL) {
    free(resourceList->inputBuffer);
    resourceList->inputBuffer = NULL;
  }
  if (resourceList->outputBuffer != NULL) {
    free(resourceList->outputBuffer);
    resourceList->outputBuffer = NULL;
  }

  if (resourceList->stream != NULL) {
    hipStreamDestroy(resourceList->stream);
    resourceList->stream = NULL;
  }
}

int main(int argc, char** argv) {
  cudlaDevHandle devHandle;
  cudlaModule moduleHandle;
  cudlaStatus err;
  FILE* fp = NULL;
  struct stat st;
  size_t file_size;
  size_t actually_read = 0;
  unsigned char* loadableData = NULL;

  hipStream_t stream;
  hipError_t result;
  const char* errPtr = NULL;

  ResourceList resourceList;

  memset(&resourceList, 0x00, sizeof(ResourceList));

  if (argc != 3) {
    DPRINTF("Usage : ./cuDLAHybridMode <loadable> <imageFile>\n");
    return 1;
  }

  // Read loadable into buffer.
  fp = fopen(argv[1], "rb");
  if (fp == NULL) {
    DPRINTF("Cannot open file %s\n", argv[1]);
    return 1;
  }

  if (stat(argv[1], &st) != 0) {
    DPRINTF("Cannot stat file\n");
    return 1;
  }

  file_size = st.st_size;
  DPRINTF("The file size = %ld\n", file_size);

  loadableData = (unsigned char*)malloc(file_size);
  if (loadableData == NULL) {
    DPRINTF("Cannot Allocate memory for loadable\n");
    return 1;
  }

  actually_read = fread(loadableData, 1, file_size, fp);
  if (actually_read != file_size) {
    free(loadableData);
    DPRINTF("Read wrong size\n");
    return 1;
  }
  fclose(fp);

  resourceList.loadableData = loadableData;

  // Initialize CUDA.
  result = hipFree(0);
  if (result != hipSuccess) {
    errPtr = hipGetErrorName(result);
    DPRINTF("Error in creating hipFree = %s\n", errPtr);
    cleanUp(&resourceList);
    return 1;
  }
  result = hipSetDevice(0);
  if (result != hipSuccess) {
    errPtr = hipGetErrorName(result);
    DPRINTF("Error in creating hipSetDevice = %s\n", errPtr);
    cleanUp(&resourceList);
    return 1;
  }

  err = cudlaCreateDevice(0, &devHandle, CUDLA_CUDA_DLA);
  if (err != cudlaSuccess) {
    DPRINTF("Error in cuDLA create device = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }

  DPRINTF("Device created successfully\n");
  resourceList.devHandle = devHandle;

  err = cudlaModuleLoadFromMemory(devHandle, loadableData, file_size,
                                  &moduleHandle, 0);
  if (err != cudlaSuccess) {
    DPRINTF("Error in cudlaModuleLoadFromMemory = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  } else {
    DPRINTF("Successfully loaded module\n");
  }

  resourceList.moduleHandle = moduleHandle;

  // Create CUDA stream.
  result = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

  if (result != hipSuccess) {
    errPtr = hipGetErrorName(result);
    DPRINTF("Error in creating cuda stream = %s\n", errPtr);
    cleanUp(&resourceList);
    return 1;
  }

  resourceList.stream = stream;

  // Get tensor attributes.
  uint32_t numInputTensors = 0;
  uint32_t numOutputTensors = 0;
  cudlaModuleAttribute attribute;

  err = cudlaModuleGetAttributes(moduleHandle, CUDLA_NUM_INPUT_TENSORS,
                                 &attribute);
  if (err != cudlaSuccess) {
    DPRINTF("Error in getting numInputTensors = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }
  numInputTensors = attribute.numInputTensors;
  DPRINTF("numInputTensors = %d\n", numInputTensors);

  err = cudlaModuleGetAttributes(moduleHandle, CUDLA_NUM_OUTPUT_TENSORS,
                                 &attribute);
  if (err != cudlaSuccess) {
    DPRINTF("Error in getting numOutputTensors = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }
  numOutputTensors = attribute.numOutputTensors;
  DPRINTF("numOutputTensors = %d\n", numOutputTensors);

  cudlaModuleTensorDescriptor* inputTensorDesc =
      (cudlaModuleTensorDescriptor*)malloc(sizeof(cudlaModuleTensorDescriptor) *
                                           numInputTensors);
  cudlaModuleTensorDescriptor* outputTensorDesc =
      (cudlaModuleTensorDescriptor*)malloc(sizeof(cudlaModuleTensorDescriptor) *
                                           numOutputTensors);

  if ((inputTensorDesc == NULL) || (outputTensorDesc == NULL)) {
    if (inputTensorDesc != NULL) {
      free(inputTensorDesc);
      inputTensorDesc = NULL;
    }

    if (outputTensorDesc != NULL) {
      free(outputTensorDesc);
      outputTensorDesc = NULL;
    }

    cleanUp(&resourceList);
    return 1;
  }

  resourceList.inputTensorDesc = inputTensorDesc;
  resourceList.outputTensorDesc = outputTensorDesc;

  attribute.inputTensorDesc = inputTensorDesc;
  err = cudlaModuleGetAttributes(moduleHandle, CUDLA_INPUT_TENSOR_DESCRIPTORS,
                                 &attribute);
  if (err != cudlaSuccess) {
    DPRINTF("Error in getting input tensor descriptor = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }
  DPRINTF("Printing input tensor descriptor\n");
  printTensorDesc(inputTensorDesc);

  attribute.outputTensorDesc = outputTensorDesc;
  err = cudlaModuleGetAttributes(moduleHandle, CUDLA_OUTPUT_TENSOR_DESCRIPTORS,
                                 &attribute);
  if (err != cudlaSuccess) {
    DPRINTF("Error in getting output tensor descriptor = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }
  DPRINTF("Printing output tensor descriptor\n");
  printTensorDesc(outputTensorDesc);

  // Setup the input and output buffers which will be used as an input to CUDA.
  unsigned char* inputBuffer = (unsigned char*)malloc(inputTensorDesc[0].size);
  if (inputBuffer == NULL) {
    DPRINTF("Error in allocating input memory\n");
    cleanUp(&resourceList);
    return 1;
  }

  resourceList.inputBuffer = inputBuffer;

  unsigned char* outputBuffer =
      (unsigned char*)malloc(outputTensorDesc[0].size);
  if (outputBuffer == NULL) {
    DPRINTF("Error in allocating output memory\n");
    cleanUp(&resourceList);
    return 1;
  }

  resourceList.outputBuffer = outputBuffer;

  memset(inputBuffer, 0x00, inputTensorDesc[0].size);
  memset(outputBuffer, 0x00, outputTensorDesc[0].size);

  // Fill up the buffers with data.
  if (initializeInputBuffers(argv[2], inputTensorDesc, inputBuffer) != 0) {
    DPRINTF("Error in initializing input buffer\n");
    cleanUp(&resourceList);
    return 1;
  }

  // Allocate memory on GPU.
  void* inputBufferGPU;
  void* outputBufferGPU;
  result = hipMalloc(&inputBufferGPU, inputTensorDesc[0].size);
  if (result != hipSuccess) {
    DPRINTF("Error in allocating input memory on GPU\n");
    cleanUp(&resourceList);
    return 1;
  }

  resourceList.inputBufferGPU = inputBufferGPU;

  result = hipMalloc(&outputBufferGPU, outputTensorDesc[0].size);
  if (result != hipSuccess) {
    DPRINTF("Error in allocating output memory on GPU\n");
    cleanUp(&resourceList);
    return 1;
  }

  resourceList.outputBufferGPU = outputBufferGPU;

  // Register the CUDA-allocated buffers.
  uint64_t* inputBufferRegisteredPtr = NULL;
  uint64_t* outputBufferRegisteredPtr = NULL;

  err = cudlaMemRegister(devHandle, (uint64_t*)inputBufferGPU,
                         inputTensorDesc[0].size, &inputBufferRegisteredPtr, 0);
  if (err != cudlaSuccess) {
    DPRINTF("Error in registering input memory = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }

  err =
      cudlaMemRegister(devHandle, (uint64_t*)outputBufferGPU,
                       outputTensorDesc[0].size, &outputBufferRegisteredPtr, 0);
  if (err != cudlaSuccess) {
    DPRINTF("Error in registering output memory = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }
  DPRINTF("ALL MEMORY REGISTERED SUCCESSFULLY\n");

  // Copy data from CPU buffers to GPU buffers.
  result = hipMemcpyAsync(inputBufferGPU, inputBuffer, inputTensorDesc[0].size,
                           hipMemcpyHostToDevice, stream);
  if (result != hipSuccess) {
    DPRINTF("Error in enqueueing memcpy for input\n");
    cleanUp(&resourceList);
    return 1;
  }
  result =
      hipMemsetAsync(outputBufferGPU, 0, outputTensorDesc[0].size, stream);
  if (result != hipSuccess) {
    DPRINTF("Error in enqueueing memset for output\n");
    cleanUp(&resourceList);
    return 1;
  }

  // Enqueue a cuDLA task.
  cudlaTask task;
  task.moduleHandle = moduleHandle;
  task.outputTensor = &outputBufferRegisteredPtr;
  task.numOutputTensors = 1;
  task.numInputTensors = 1;
  task.inputTensor = &inputBufferRegisteredPtr;
  task.waitEvents = NULL;
  task.signalEvents = NULL;
  err = cudlaSubmitTask(devHandle, &task, 1, stream, 0);
  if (err != cudlaSuccess) {
    DPRINTF("Error in submitting task\n");
    cleanUp(&resourceList);
    return 1;
  }
  DPRINTF("SUBMIT IS DONE !!!\n");

  // Wait for stream operations to finish and bring output buffer to CPU.
  result =
      hipMemcpyAsync(outputBuffer, outputBufferGPU, outputTensorDesc[0].size,
                      hipMemcpyDeviceToHost, stream);
  if (result != hipSuccess) {
    DPRINTF("Error in bringing result back to CPU\n");
    cleanUp(&resourceList);
    return 1;
  }
  result = hipStreamSynchronize(stream);
  if (result != hipSuccess) {
    DPRINTF("Error in synchronizing stream\n");
    cleanUp(&resourceList);
    return 1;
  }

  // Output is available in outputBuffer.

  // Teardown.
  err = cudlaMemUnregister(devHandle, inputBufferRegisteredPtr);
  if (err != cudlaSuccess) {
    DPRINTF("Error in unregistering input memory = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }

  err = cudlaMemUnregister(devHandle, outputBufferRegisteredPtr);
  if (err != cudlaSuccess) {
    DPRINTF("Error in registering output memory = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  }
  DPRINTF("ALL MEMORY UNREGISTERED SUCCESSFULLY\n");

  free(inputTensorDesc);
  free(outputTensorDesc);
  free(loadableData);
  free(inputBuffer);
  free(outputBuffer);
  hipFree(inputBufferGPU);
  hipFree(outputBufferGPU);

  resourceList.inputTensorDesc = NULL;
  resourceList.outputTensorDesc = NULL;
  resourceList.loadableData = NULL;
  resourceList.inputBuffer = NULL;
  resourceList.outputBuffer = NULL;
  resourceList.inputBufferGPU = 0;
  resourceList.outputBufferGPU = 0;

  result = hipStreamDestroy(stream);
  if (result != hipSuccess) {
    errPtr = hipGetErrorName(result);
    DPRINTF("Error in destroying cuda stream = %s\n", errPtr);
    cleanUp(&resourceList);
    return 1;
  }

  resourceList.stream = NULL;

  err = cudlaModuleUnload(moduleHandle, 0);
  if (err != cudlaSuccess) {
    DPRINTF("Error in cudlaModuleUnload = %d\n", err);
    cleanUp(&resourceList);
    return 1;
  } else {
    DPRINTF("Successfully unloaded module\n");
  }

  resourceList.moduleHandle = NULL;

  err = cudlaDestroyDevice(devHandle);
  if (err != cudlaSuccess) {
    DPRINTF("Error in cuDLA destroy device = %d\n", err);
    return 1;
  }
  DPRINTF("Device destroyed successfully\n");

  resourceList.devHandle = NULL;

  DPRINTF("cuDLAHybridMode DONE !!!\n");

  return 0;
}
