#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_image.h>

// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return ((unsigned int)(rgba.w * 255.0f) << 24) |
         ((unsigned int)(rgba.z * 255.0f) << 16) |
         ((unsigned int)(rgba.y * 255.0f) << 8) |
         ((unsigned int)(rgba.x * 255.0f));
}

////////////////////////////////////////////////////////////////////////////////
//! Rotate an image using texture lookups
//! @param outputData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
static __global__ void transformKernel(unsigned int *outputData, int width,
                                       int height, float theta,
                                       hipTextureObject_t tex) {
  // calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = (float)x - (float)width / 2;
  float v = (float)y - (float)height / 2;
  float tu = u * cosf(theta) - v * sinf(theta);
  float tv = v * cosf(theta) + u * sinf(theta);

  tu /= (float)width;
  tv /= (float)height;

  // read from texture and write to global memory
  float4 pix = tex2D<float4>(tex, tu + 0.5f, tv + 0.5f);
  unsigned int pixelInt = rgbaFloatToInt(pix);
  outputData[y * width + x] = pixelInt;
}

static __global__ void rgbToGrayscaleKernel(unsigned int *rgbaImage,
                                            size_t imageWidth,
                                            size_t imageHeight) {
  size_t gidX = blockDim.x * blockIdx.x + threadIdx.x;

  uchar4 *pixArray = (uchar4 *)rgbaImage;

  for (int pixId = gidX; pixId < imageWidth * imageHeight;
       pixId += gridDim.x * blockDim.x) {
    uchar4 dataA = pixArray[pixId];
    unsigned char grayscale =
        (unsigned char)(dataA.x * 0.3 + dataA.y * 0.59 + dataA.z * 0.11);
    uchar4 dataB = make_uchar4(grayscale, grayscale, grayscale, 0);
    pixArray[pixId] = dataB;
  }
}

void launchGrayScaleKernel(unsigned int *d_rgbaImage,
                           std::string image_filename, size_t imageWidth,
                           size_t imageHeight, hipStream_t stream) {
  int numThreadsPerBlock = 1024;
  int numOfBlocks = (imageWidth * imageHeight) / numThreadsPerBlock;

  rgbToGrayscaleKernel<<<numOfBlocks, numThreadsPerBlock, 0, stream>>>(
      d_rgbaImage, imageWidth, imageHeight);

  unsigned int *outputData;
  checkCudaErrors(hipHostMalloc((void**)&outputData, sizeof(unsigned int) * imageWidth * imageHeight));
  checkCudaErrors(hipMemcpyAsync(
      outputData, d_rgbaImage, sizeof(unsigned int) * imageWidth * imageHeight,
      hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  char outputFilename[1024];
  strcpy(outputFilename, image_filename.c_str());
  strcpy(outputFilename + image_filename.length() - 4, "_out.ppm");
  sdkSavePPM4ub(outputFilename, (unsigned char *)outputData, imageWidth,
                imageHeight);
  printf("Wrote '%s'\n", outputFilename);

  checkCudaErrors(hipHostFree(outputData));
}

void rotateKernel(hipTextureObject_t &texObj, const float angle,
                  unsigned int *d_outputData, const int imageWidth,
                  const int imageHeight, hipStream_t stream) {
  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(imageWidth / dimBlock.x, imageHeight / dimBlock.y, 1);

  transformKernel<<<dimGrid, dimBlock, 0, stream>>>(d_outputData, imageWidth,
                                                    imageHeight, angle, texObj);
}
