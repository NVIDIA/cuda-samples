#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on GPU
 * using CUBLAS and CUSPARSE with CUDA Graphs
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

const char *sSDKname = "conjugateGradientCudaGraphs";

#ifndef WITH_GRAPH
#define WITH_GRAPH 1
#endif

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz) {
  I[0] = 0, J[0] = 0, J[1] = 1;
  val[0] = (float)rand() / RAND_MAX + 10.0f;
  val[1] = (float)rand() / RAND_MAX;
  int start;

  for (int i = 1; i < N; i++) {
    if (i > 1) {
      I[i] = I[i - 1] + 3;
    } else {
      I[1] = 2;
    }

    start = (i - 1) * 3 + 2;
    J[start] = i - 1;
    J[start + 1] = i;

    if (i < N - 1) {
      J[start + 2] = i + 1;
    }

    val[start] = val[start - 1];
    val[start + 1] = (float)rand() / RAND_MAX + 10.0f;

    if (i < N - 1) {
      val[start + 2] = (float)rand() / RAND_MAX;
    }
  }

  I[N] = nz;
}

__global__ void initVectors(float *rhs, float *x, int N) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  for (size_t i = gid; i < N; i += gridDim.x * blockDim.x) {
    rhs[i] = 1.0;
    x[i] = 0.0;
  }
}

__global__ void r1_div_x(float *r1, float *r0, float *b) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    b[0] = r1[0] / r0[0];
  }
}

__global__ void a_minus(float *a, float *na) {
  int gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid == 0) {
    na[0] = -(a[0]);
  }
}

int main(int argc, char **argv) {
  int N = 0, nz = 0, *I = NULL, *J = NULL;
  float *val = NULL;
  const float tol = 1e-5f;
  const int max_iter = 10000;
  float *x;
  float *rhs;
  float r1;

  int *d_col, *d_row;
  float *d_val, *d_x;
  float *d_r, *d_p, *d_Ax;
  int k;
  float alpha, beta, alpham1;

  hipStream_t stream1, streamForGraph;

  // This will pick the best possible CUDA capable device
  hipDeviceProp_t deviceProp;
  int devID = findCudaDevice(argc, (const char **)argv);

  if (devID < 0) {
    printf("exiting...\n");
    exit(EXIT_SUCCESS);
  }

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

  // Statistics about the GPU device
  printf(
      "> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
      deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

  /* Generate a random tridiagonal symmetric matrix in CSR format */
  N = 1048576;
  nz = (N - 2) * 3 + 4;
  checkCudaErrors(hipHostMalloc(&I, sizeof(int) * (N + 1)));
  checkCudaErrors(hipHostMalloc(&J, sizeof(int) * nz));
  checkCudaErrors(hipHostMalloc(&val, sizeof(float) * nz));
  genTridiag(I, J, val, N, nz);

  checkCudaErrors(hipHostMalloc(&x, sizeof(float) * N));
  rhs = (float *)malloc(sizeof(float) * N);

  for (int i = 0; i < N; i++) {
    rhs[i] = 1.0;
    x[i] = 0.0;
  }

  /* Get handle to the CUBLAS context */
  hipblasHandle_t cublasHandle = 0;
  hipblasStatus_t hipblasStatus_t;
  hipblasStatus_t = hipblasCreate(&cublasHandle);

  checkCudaErrors(hipblasStatus_t);

  /* Get handle to the CUSPARSE context */
  hipsparseHandle_t cusparseHandle = 0;
  hipsparseStatus_t cusparseStatus;
  cusparseStatus = hipsparseCreate(&cusparseHandle);

  checkCudaErrors(cusparseStatus);

  checkCudaErrors(hipStreamCreate(&stream1));

  checkCudaErrors(hipMalloc((void **)&d_col, nz * sizeof(int)));
  checkCudaErrors(hipMalloc((void **)&d_row, (N + 1) * sizeof(int)));
  checkCudaErrors(hipMalloc((void **)&d_val, nz * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_x, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_r, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_p, N * sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_Ax, N * sizeof(float)));

  float *d_r1, *d_r0, *d_dot, *d_a, *d_na, *d_b;
  checkCudaErrors(hipMalloc((void **)&d_r1, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_r0, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_dot, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_a, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_na, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_b, sizeof(float)));

  /* Wrap raw data into cuSPARSE generic API objects */
  hipsparseSpMatDescr_t matA = NULL;
  checkCudaErrors(hipsparseCreateCsr(&matA, N, N, nz, d_row, d_col, d_val,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
  hipsparseDnVecDescr_t vecx = NULL;
  checkCudaErrors(hipsparseCreateDnVec(&vecx, N, d_x, HIP_R_32F));
  hipsparseDnVecDescr_t vecp = NULL;
  checkCudaErrors(hipsparseCreateDnVec(&vecp, N, d_p, HIP_R_32F));
  hipsparseDnVecDescr_t vecAx = NULL;
  checkCudaErrors(hipsparseCreateDnVec(&vecAx, N, d_Ax, HIP_R_32F));

  /* Allocate workspace for cuSPARSE */
  size_t bufferSize = 0;
  checkCudaErrors(hipsparseSpMV_bufferSize(
      cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecx,
      &beta, vecAx, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
  void *buffer = NULL;
  checkCudaErrors(hipMalloc(&buffer, bufferSize));

  hipsparseMatDescr_t descr = 0;
  checkCudaErrors(hipsparseCreateMatDescr(&descr));

  checkCudaErrors(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
  checkCudaErrors(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));

  int numBlocks = 0, blockSize = 0;
  checkCudaErrors(
      hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, initVectors));

  checkCudaErrors(hipMemcpyAsync(d_col, J, nz * sizeof(int),
                                  hipMemcpyHostToDevice, stream1));
  checkCudaErrors(hipMemcpyAsync(d_row, I, (N + 1) * sizeof(int),
                                  hipMemcpyHostToDevice, stream1));
  checkCudaErrors(hipMemcpyAsync(d_val, val, nz * sizeof(float),
                                  hipMemcpyHostToDevice, stream1));

  initVectors<<<numBlocks, blockSize, 0, stream1>>>(d_r, d_x, N);

  alpha = 1.0;
  alpham1 = -1.0;
  beta = 0.0;

  checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
  checkCudaErrors(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecx, &beta, vecAx, HIP_R_32F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

  checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
  checkCudaErrors(hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1));

  checkCudaErrors(
      hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE));
  checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));

  k = 1;
  // First Iteration when k=1 starts
  checkCudaErrors(hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1));
  checkCudaErrors(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecp, &beta, vecAx, HIP_R_32F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

  checkCudaErrors(hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));

  r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);

  checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));

  a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);

  checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));

  checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),
                                  hipMemcpyDeviceToDevice, stream1));

  checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));

  checkCudaErrors(hipMemcpyAsync(&r1, d_r1, sizeof(float),
                                  hipMemcpyDeviceToHost, stream1));
  checkCudaErrors(hipStreamSynchronize(stream1));
  printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
  // First Iteration when k=1 ends
  k++;

#if WITH_GRAPH
  hipGraph_t initGraph;
  checkCudaErrors(hipStreamCreate(&streamForGraph));
  checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
  checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));
  checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));

  r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_r0, d_b);
  hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
  checkCudaErrors(hipblasSscal(cublasHandle, N, d_b, d_p, 1));
  hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);
  checkCudaErrors(hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1));
  hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);

  checkCudaErrors(
      hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST));
  checkCudaErrors(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, vecp, &beta, vecAx, HIP_R_32F,
                               HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

  checkCudaErrors(hipMemsetAsync(d_dot, 0, sizeof(float), stream1));
  checkCudaErrors(hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));

  r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);

  checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));

  a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);

  checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));

  checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),
                                  hipMemcpyDeviceToDevice, stream1));
  checkCudaErrors(hipMemsetAsync(d_r1, 0, sizeof(float), stream1));

  checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));

  checkCudaErrors(hipMemcpyAsync((float *)&r1, d_r1, sizeof(float),
                                  hipMemcpyDeviceToHost, stream1));

  checkCudaErrors(hipStreamEndCapture(stream1, &initGraph));
  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, initGraph, NULL, NULL, 0));
#endif

  checkCudaErrors(hipblasSetStream(cublasHandle, stream1));
  checkCudaErrors(hipsparseSetStream(cusparseHandle, stream1));

  while (r1 > tol * tol && k <= max_iter) {
#if WITH_GRAPH
    checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
    checkCudaErrors(hipStreamSynchronize(streamForGraph));
#else
    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_r0, d_b);
    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    checkCudaErrors(hipblasSscal(cublasHandle, N, d_b, d_p, 1));

    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST);
    checkCudaErrors(hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1));

    checkCudaErrors(hipsparseSpMV(
        cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecp,
        &beta, vecAx, HIP_R_32F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));

    hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_DEVICE);
    checkCudaErrors(hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, d_dot));

    r1_div_x<<<1, 1, 0, stream1>>>(d_r1, d_dot, d_a);

    checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_a, d_p, 1, d_x, 1));

    a_minus<<<1, 1, 0, stream1>>>(d_a, d_na);
    checkCudaErrors(hipblasSaxpy(cublasHandle, N, d_na, d_Ax, 1, d_r, 1));

    checkCudaErrors(hipMemcpyAsync(d_r0, d_r1, sizeof(float),
                                    hipMemcpyDeviceToDevice, stream1));

    checkCudaErrors(hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, d_r1));
    checkCudaErrors(hipMemcpyAsync((float *)&r1, d_r1, sizeof(float),
                                    hipMemcpyDeviceToHost, stream1));
    checkCudaErrors(hipStreamSynchronize(stream1));
#endif
    printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
    k++;
  }

#if WITH_GRAPH
  checkCudaErrors(hipMemcpyAsync(x, d_x, N * sizeof(float),
                                  hipMemcpyDeviceToHost, streamForGraph));
  checkCudaErrors(hipStreamSynchronize(streamForGraph));
#else
  checkCudaErrors(hipMemcpyAsync(x, d_x, N * sizeof(float),
                                  hipMemcpyDeviceToHost, stream1));
  checkCudaErrors(hipStreamSynchronize(stream1));
#endif

  float rsum, diff, err = 0.0;

  for (int i = 0; i < N; i++) {
    rsum = 0.0;

    for (int j = I[i]; j < I[i + 1]; j++) {
      rsum += val[j] * x[J[j]];
    }

    diff = fabs(rsum - rhs[i]);

    if (diff > err) {
      err = diff;
    }
  }

#if WITH_GRAPH
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphDestroy(initGraph));
  checkCudaErrors(hipStreamDestroy(streamForGraph));
#endif
  checkCudaErrors(hipStreamDestroy(stream1));
  hipsparseDestroy(cusparseHandle);
  hipblasDestroy(cublasHandle);

  if (matA) {
    checkCudaErrors(hipsparseDestroySpMat(matA));
  }
  if (vecx) {
    checkCudaErrors(hipsparseDestroyDnVec(vecx));
  }
  if (vecAx) {
    checkCudaErrors(hipsparseDestroyDnVec(vecAx));
  }
  if (vecp) {
    checkCudaErrors(hipsparseDestroyDnVec(vecp));
  }

  checkCudaErrors(hipHostFree(I));
  checkCudaErrors(hipHostFree(J));
  checkCudaErrors(hipHostFree(val));
  checkCudaErrors(hipHostFree(x));
  free(rhs);
  checkCudaErrors(hipFree(d_col));
  checkCudaErrors(hipFree(d_row));
  checkCudaErrors(hipFree(d_val));
  checkCudaErrors(hipFree(d_x));
  checkCudaErrors(hipFree(d_r));
  checkCudaErrors(hipFree(d_p));
  checkCudaErrors(hipFree(d_Ax));

  printf("Test Summary:  Error amount = %f\n", err);
  exit((k <= max_iter) ? 0 : 1);
}
