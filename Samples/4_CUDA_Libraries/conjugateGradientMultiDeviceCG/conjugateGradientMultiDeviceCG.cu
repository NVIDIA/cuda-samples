#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements a conjugate gradient solver on multiple GPU using
 * Unified Memory optimized prefetching and usage hints.
 *
 */

// includes, system
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <map>
#include <iostream>
#include <set>
#include <utility>

#include <hip/hip_runtime.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

const char *sSDKname = "conjugateGradientMultiDeviceCG";

#define ENABLE_CPU_DEBUG_CODE 0
#define THREADS_PER_BLOCK 512

__device__ double grid_dot_result = 0.0;

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, float *val, int N, int nz) {
  I[0] = 0, J[0] = 0, J[1] = 1;
  val[0] = (float)rand() / RAND_MAX + 10.0f;
  val[1] = (float)rand() / RAND_MAX;
  int start;

  for (int i = 1; i < N; i++) {
    if (i > 1) {
      I[i] = I[i - 1] + 3;
    } else {
      I[1] = 2;
    }

    start = (i - 1) * 3 + 2;
    J[start] = i - 1;
    J[start + 1] = i;

    if (i < N - 1) {
      J[start + 2] = i + 1;
    }

    val[start] = val[start - 1];
    val[start + 1] = (float)rand() / RAND_MAX + 10.0f;

    if (i < N - 1) {
      val[start + 2] = (float)rand() / RAND_MAX;
    }
  }

  I[N] = nz;
}

// I - contains location of the given non-zero element in the row of the matrix
// J - contains location of the given non-zero element in the column of the
// matrix val - contains values of the given non-zero elements of the matrix
// inputVecX - input vector to be multiplied
// outputVecY - resultant vector
void cpuSpMV(int *I, int *J, float *val, int nnz, int num_rows, float alpha,
             float *inputVecX, float *outputVecY) {
  for (int i = 0; i < num_rows; i++) {
    int num_elems_this_row = I[i + 1] - I[i];

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++) {
      output += alpha * val[I[i] + j] * inputVecX[J[I[i] + j]];
    }
    outputVecY[i] = output;
  }

  return;
}

float dotProduct(float *vecA, float *vecB, int size) {
  float result = 0.0;

  for (int i = 0; i < size; i++) {
    result = result + (vecA[i] * vecB[i]);
  }

  return result;
}

void scaleVector(float *vec, float alpha, int size) {
  for (int i = 0; i < size; i++) {
    vec[i] = alpha * vec[i];
  }
}

void saxpy(float *x, float *y, float a, int size) {
  for (int i = 0; i < size; i++) {
    y[i] = a * x[i] + y[i];
  }
}

void cpuConjugateGrad(int *I, int *J, float *val, float *x, float *Ax, float *p,
                      float *r, int nnz, int N, float tol) {
  int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, b, a, na;

  cpuSpMV(I, J, val, nnz, N, alpha, x, Ax);
  saxpy(Ax, r, alpham1, N);

  float r1 = dotProduct(r, r, N);

  int k = 1;

  while (r1 > tol * tol && k <= max_iter) {
    if (k > 1) {
      b = r1 / r0;
      scaleVector(p, b, N);

      saxpy(r, p, alpha, N);
    } else {
      for (int i = 0; i < N; i++) p[i] = r[i];
    }

    cpuSpMV(I, J, val, nnz, N, alpha, p, Ax);

    float dot = dotProduct(p, Ax, N);
    a = r1 / dot;

    saxpy(p, x, a, N);
    na = -a;
    saxpy(Ax, r, na, N);

    r0 = r1;
    r1 = dotProduct(r, r, N);

    printf("\nCPU code iteration = %3d, residual = %e\n", k, sqrt(r1));
    k++;
  }
}

// Data filled on CPU needed for MultiGPU operations.
struct MultiDeviceData {
  unsigned char *hostMemoryArrivedList;
  unsigned int numDevices;
  unsigned int deviceRank;
};

// Class used for coordination of multiple devices.
class PeerGroup {
  const MultiDeviceData &data;
  const cg::grid_group &grid;

  __device__ unsigned char load_arrived(unsigned char *arrived) const {
#if __CUDA_ARCH__ < 700
    return *(volatile unsigned char *)arrived;
#else
    unsigned int result;
    asm volatile("ld.acquire.sys.global.u8 %0, [%1];"
                 : "=r"(result)
                 : "l"(arrived)
                 : "memory");
    return result;
#endif
  }

  __device__ void store_arrived(unsigned char *arrived,
                                unsigned char val) const {
#if __CUDA_ARCH__ < 700
    *(volatile unsigned char *)arrived = val;
#else
    unsigned int reg_val = val;
    asm volatile(
        "st.release.sys.global.u8 [%1], %0;" ::"r"(reg_val) "l"(arrived)
        : "memory");

    // Avoids compiler warnings from unused variable val.
    (void)(reg_val = reg_val);
#endif
  }

 public:
  __device__ PeerGroup(const MultiDeviceData &data, const cg::grid_group &grid)
      : data(data), grid(grid){};

  __device__ unsigned int size() const { return data.numDevices * grid.size(); }

  __device__ unsigned int thread_rank() const {
    return data.deviceRank * grid.size() + grid.thread_rank();
  }

  __device__ void sync() const {
    grid.sync();

    // One thread from each grid participates in the sync.
    if (grid.thread_rank() == 0) {
      if (data.deviceRank == 0) {
        // Leader grid waits for others to join and then releases them.
        // Other GPUs can arrive in any order, so the leader have to wait for
        // all others.
        for (int i = 0; i < data.numDevices - 1; i++) {
          while (load_arrived(&data.hostMemoryArrivedList[i]) == 0)
            ;
        }
        for (int i = 0; i < data.numDevices - 1; i++) {
          store_arrived(&data.hostMemoryArrivedList[i], 0);
        }
        __threadfence_system();
      } else {
        // Other grids note their arrival and wait to be released.
        store_arrived(&data.hostMemoryArrivedList[data.deviceRank - 1], 1);
        while (load_arrived(&data.hostMemoryArrivedList[data.deviceRank - 1]) ==
               1)
          ;
      }
    }

    grid.sync();
  }
};

__device__ void gpuSpMV(int *I, int *J, float *val, int nnz, int num_rows,
                        float alpha, float *inputVecX, float *outputVecY,
                        const PeerGroup &peer_group) {
  for (int i = peer_group.thread_rank(); i < num_rows; i += peer_group.size()) {
    int row_elem = I[i];
    int next_row_elem = I[i + 1];
    int num_elems_this_row = next_row_elem - row_elem;

    float output = 0.0;
    for (int j = 0; j < num_elems_this_row; j++) {
      output += alpha * val[row_elem + j] * inputVecX[J[row_elem + j]];
    }

    outputVecY[i] = output;
  }
}

__device__ void gpuSaxpy(float *x, float *y, float a, int size,
                         const PeerGroup &peer_group) {
  for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
    y[i] = a * x[i] + y[i];
  }
}

__device__ void gpuDotProduct(float *vecA, float *vecB, int size,
                              const cg::thread_block &cta,
                              const PeerGroup &peer_group) {
  extern __shared__ double tmp[];

  double temp_sum = 0.0;

  for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
    temp_sum += (double)(vecA[i] * vecB[i]);
  }

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

  if (tile32.thread_rank() == 0) {
    tmp[tile32.meta_group_rank()] = temp_sum;
  }

  cg::sync(cta);

  if (tile32.meta_group_rank() == 0) {
    temp_sum = tile32.thread_rank() < tile32.meta_group_size()
                   ? tmp[tile32.thread_rank()]
                   : 0.0;
    temp_sum = cg::reduce(tile32, temp_sum, cg::plus<double>());

    if (tile32.thread_rank() == 0) {
      atomicAdd(&grid_dot_result, temp_sum);
    }
  }
}

__device__ void gpuCopyVector(float *srcA, float *destB, int size,
                              const PeerGroup &peer_group) {
  for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
    destB[i] = srcA[i];
  }
}

__device__ void gpuScaleVectorAndSaxpy(float *x, float *y, float a, float scale,
                                       int size, const PeerGroup &peer_group) {
  for (int i = peer_group.thread_rank(); i < size; i += peer_group.size()) {
    y[i] = a * x[i] + scale * y[i];
  }
}

extern "C" __global__ void multiGpuConjugateGradient(
    int *I, int *J, float *val, float *x, float *Ax, float *p, float *r,
    double *dot_result, int nnz, int N, float tol,
    MultiDeviceData multi_device_data) {
  cg::thread_block cta = cg::this_thread_block();
  cg::grid_group grid = cg::this_grid();
  PeerGroup peer_group(multi_device_data, grid);

  const int max_iter = 10000;

  float alpha = 1.0;
  float alpham1 = -1.0;
  float r0 = 0.0, r1, b, a, na;

  for (int i = peer_group.thread_rank(); i < N; i += peer_group.size()) {
    r[i] = 1.0;
    x[i] = 0.0;
  }

  cg::sync(grid);

  gpuSpMV(I, J, val, nnz, N, alpha, x, Ax, peer_group);

  cg::sync(grid);

  gpuSaxpy(Ax, r, alpham1, N, peer_group);

  cg::sync(grid);

  gpuDotProduct(r, r, N, cta, peer_group);

  cg::sync(grid);

  if (grid.thread_rank() == 0) {
    atomicAdd_system(dot_result, grid_dot_result);
    grid_dot_result = 0.0;
  }
  peer_group.sync();

  r1 = *dot_result;

  int k = 1;
  while (r1 > tol * tol && k <= max_iter) {
    if (k > 1) {
      b = r1 / r0;
      gpuScaleVectorAndSaxpy(r, p, alpha, b, N, peer_group);
    } else {
      gpuCopyVector(r, p, N, peer_group);
    }

    peer_group.sync();

    gpuSpMV(I, J, val, nnz, N, alpha, p, Ax, peer_group);

    if (peer_group.thread_rank() == 0) {
      *dot_result = 0.0;
    }
    peer_group.sync();

    gpuDotProduct(p, Ax, N, cta, peer_group);

    cg::sync(grid);

    if (grid.thread_rank() == 0) {
      atomicAdd_system(dot_result, grid_dot_result);
      grid_dot_result = 0.0;
    }
    peer_group.sync();

    a = r1 / *dot_result;

    gpuSaxpy(p, x, a, N, peer_group);

    na = -a;

    gpuSaxpy(Ax, r, na, N, peer_group);

    r0 = r1;

    peer_group.sync();

    if (peer_group.thread_rank() == 0) {
      *dot_result = 0.0;
    }

    peer_group.sync();

    gpuDotProduct(r, r, N, cta, peer_group);

    cg::sync(grid);

    if (grid.thread_rank() == 0) {
      atomicAdd_system(dot_result, grid_dot_result);
      grid_dot_result = 0.0;
    }
    peer_group.sync();

    r1 = *dot_result;
    k++;
  }
}

// Map of device version to device number
std::multimap<std::pair<int, int>, int> getIdenticalGPUs() {
  int numGpus = 0;
  checkCudaErrors(hipGetDeviceCount(&numGpus));

  std::multimap<std::pair<int, int>, int> identicalGpus;

  for (int i = 0; i < numGpus; i++) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, i));

    // Filter unsupported devices
    if (deviceProp.cooperativeLaunch && deviceProp.concurrentManagedAccess) {
      identicalGpus.emplace(std::make_pair(deviceProp.major, deviceProp.minor),
                            i);
    }
    printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", i,
           deviceProp.name, deviceProp.major, deviceProp.minor);
  }

  return identicalGpus;
}

int main(int argc, char **argv) {
  constexpr size_t kNumGpusRequired = 2;
  int N = 0, nz = 0, *I = NULL, *J = NULL;
  float *val = NULL;
  const float tol = 1e-5f;
  float *x;
  float rhs = 1.0;
  float r1;
  float *r, *p, *Ax;

  printf("Starting [%s]...\n", sSDKname);
  auto gpusByArch = getIdenticalGPUs();

  auto it = gpusByArch.begin();
  auto end = gpusByArch.end();

  auto bestFit = std::make_pair(it, it);
  // use std::distance to find the largest number of GPUs amongst architectures
  auto distance = [](decltype(bestFit) p) {
    return std::distance(p.first, p.second);
  };

  // Read each unique key/pair element in order
  for (; it != end; it = gpusByArch.upper_bound(it->first)) {
    // first and second are iterators bounded within the architecture group
    auto testFit = gpusByArch.equal_range(it->first);
    // Always use devices with highest architecture version or whichever has the
    // most devices available
    if (distance(bestFit) <= distance(testFit)) bestFit = testFit;
  }

  if (distance(bestFit) < kNumGpusRequired) {
    printf(
        "No two or more GPUs with same architecture capable of "
        "concurrentManagedAccess found. "
        "\nWaiving the sample\n");
    exit(EXIT_WAIVED);
  }

  std::set<int> bestFitDeviceIds;

  // Check & select peer-to-peer access capable GPU devices as enabling p2p
  // access between participating GPUs gives better performance.
  for (auto itr = bestFit.first; itr != bestFit.second; itr++) {
    int deviceId = itr->second;
    checkCudaErrors(hipSetDevice(deviceId));

    std::for_each(
        itr, bestFit.second,
        [&deviceId, &bestFitDeviceIds,
         &kNumGpusRequired](decltype(*itr) mapPair) {
          if (deviceId != mapPair.second) {
            int access = 0;
            checkCudaErrors(
                hipDeviceCanAccessPeer(&access, deviceId, mapPair.second));
            printf("Device=%d %s Access Peer Device=%d\n", deviceId,
                   access ? "CAN" : "CANNOT", mapPair.second);
            if (access && bestFitDeviceIds.size() < kNumGpusRequired) {
              bestFitDeviceIds.emplace(deviceId);
              bestFitDeviceIds.emplace(mapPair.second);
            } else {
              printf("Ignoring device %i (max devices exceeded)\n",
                     mapPair.second);
            }
          }
        });

    if (bestFitDeviceIds.size() >= kNumGpusRequired) {
      printf("Selected p2p capable devices - ");
      for (auto devicesItr = bestFitDeviceIds.begin();
           devicesItr != bestFitDeviceIds.end(); devicesItr++) {
        printf("deviceId = %d  ", *devicesItr);
      }
      printf("\n");
      break;
    }
  }

  // if bestFitDeviceIds.size() == 0 it means the GPUs in system are not p2p
  // capable, hence we add it without p2p capability check.
  if (!bestFitDeviceIds.size()) {
    printf("Devices involved are not p2p capable.. selecting %zu of them\n",
           kNumGpusRequired);
    std::for_each(bestFit.first, bestFit.second,
                  [&bestFitDeviceIds,
                   &kNumGpusRequired](decltype(*bestFit.first) mapPair) {
                    if (bestFitDeviceIds.size() < kNumGpusRequired) {
                      bestFitDeviceIds.emplace(mapPair.second);
                    } else {
                      printf("Ignoring device %i (max devices exceeded)\n",
                             mapPair.second);
                    }
                    // Insert the sequence into the deviceIds set
                  });
  } else {
    // perform hipDeviceEnablePeerAccess in both directions for all
    // participating devices.
    for (auto p1_itr = bestFitDeviceIds.begin();
         p1_itr != bestFitDeviceIds.end(); p1_itr++) {
      checkCudaErrors(hipSetDevice(*p1_itr));
      for (auto p2_itr = bestFitDeviceIds.begin();
           p2_itr != bestFitDeviceIds.end(); p2_itr++) {
        if (*p1_itr != *p2_itr) {
          checkCudaErrors(hipDeviceEnablePeerAccess(*p2_itr, 0));
          checkCudaErrors(hipSetDevice(*p1_itr));
        }
      }
    }
  }

  /* Generate a random tridiagonal symmetric matrix in CSR format */
  N = 10485760 * 2;
  nz = (N - 2) * 3 + 4;

  checkCudaErrors(hipMallocManaged((void **)&I, sizeof(int) * (N + 1)));
  checkCudaErrors(hipMallocManaged((void **)&J, sizeof(int) * nz));
  checkCudaErrors(hipMallocManaged((void **)&val, sizeof(float) * nz));

  float *val_cpu = (float *)malloc(sizeof(float) * nz);

  genTridiag(I, J, val_cpu, N, nz);

  memcpy(val, val_cpu, sizeof(float) * nz);
  checkCudaErrors(
      hipMemAdvise(I, sizeof(int) * (N + 1), hipMemAdviseSetReadMostly, 0));
  checkCudaErrors(
      hipMemAdvise(J, sizeof(int) * nz, hipMemAdviseSetReadMostly, 0));
  checkCudaErrors(
      hipMemAdvise(val, sizeof(float) * nz, hipMemAdviseSetReadMostly, 0));

  checkCudaErrors(hipMallocManaged((void **)&x, sizeof(float) * N));

  double *dot_result;
  checkCudaErrors(hipMallocManaged((void **)&dot_result, sizeof(double)));

  checkCudaErrors(hipMemset(dot_result, 0, sizeof(double)));

  // temp memory for ConjugateGradient
  checkCudaErrors(hipMallocManaged((void **)&r, N * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&p, N * sizeof(float)));
  checkCudaErrors(hipMallocManaged((void **)&Ax, N * sizeof(float)));

  std::cout << "\nRunning on GPUs = " << kNumGpusRequired << std::endl;
  hipStream_t nStreams[kNumGpusRequired];

  int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK / 32) + 1);
  int numBlocksPerSm = INT_MAX;
  int numThreads = THREADS_PER_BLOCK;
  int numSms = INT_MAX;
  auto deviceId = bestFitDeviceIds.begin();

  // set numSms & numBlocksPerSm to be lowest of 2 devices
  while (deviceId != bestFitDeviceIds.end()) {
    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, *deviceId));

    int numBlocksPerSm_current = 0;
    checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
        &numBlocksPerSm_current, multiGpuConjugateGradient, numThreads,
        sMemSize));

    if (numBlocksPerSm > numBlocksPerSm_current) {
      numBlocksPerSm = numBlocksPerSm_current;
    }
    if (numSms > deviceProp.multiProcessorCount) {
      numSms = deviceProp.multiProcessorCount;
    }
    deviceId++;
  }

  if (!numBlocksPerSm) {
    printf(
        "Max active blocks per SM is returned as 0.\n Hence, Waiving the "
        "sample\n");
    exit(EXIT_WAIVED);
  }

  int device_count = 0;
  int totalThreadsPerGPU = numSms * numBlocksPerSm * THREADS_PER_BLOCK;
  deviceId = bestFitDeviceIds.begin();
  while (deviceId != bestFitDeviceIds.end()) {
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipStreamCreate(&nStreams[device_count]));

    int perGPUIter = N / (totalThreadsPerGPU * kNumGpusRequired);
    int offset_Ax = device_count * totalThreadsPerGPU;
    int offset_r = device_count * totalThreadsPerGPU;
    int offset_p = device_count * totalThreadsPerGPU;
    int offset_x = device_count * totalThreadsPerGPU;

    checkCudaErrors(hipMemPrefetchAsync(I, sizeof(int) * N, *deviceId,
                                         nStreams[device_count]));
    checkCudaErrors(hipMemPrefetchAsync(val, sizeof(float) * nz, *deviceId,
                                         nStreams[device_count]));
    checkCudaErrors(hipMemPrefetchAsync(J, sizeof(float) * nz, *deviceId,
                                         nStreams[device_count]));

    if (offset_Ax <= N) {
      for (int i = 0; i < perGPUIter; i++) {
        hipMemAdvise(Ax + offset_Ax, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetPreferredLocation, *deviceId);
        hipMemAdvise(r + offset_r, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetPreferredLocation, *deviceId);
        hipMemAdvise(x + offset_x, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetPreferredLocation, *deviceId);
        hipMemAdvise(p + offset_p, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetPreferredLocation, *deviceId);

        hipMemAdvise(Ax + offset_Ax, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetAccessedBy, *deviceId);
        hipMemAdvise(r + offset_r, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetAccessedBy, *deviceId);
        hipMemAdvise(p + offset_p, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetAccessedBy, *deviceId);
        hipMemAdvise(x + offset_x, sizeof(float) * totalThreadsPerGPU,
                      hipMemAdviseSetAccessedBy, *deviceId);

        offset_Ax += totalThreadsPerGPU * kNumGpusRequired;
        offset_r += totalThreadsPerGPU * kNumGpusRequired;
        offset_p += totalThreadsPerGPU * kNumGpusRequired;
        offset_x += totalThreadsPerGPU * kNumGpusRequired;

        if (offset_Ax >= N) {
          break;
        }
      }
    }

    device_count++;
    deviceId++;
  }

#if ENABLE_CPU_DEBUG_CODE
  float *Ax_cpu = (float *)malloc(sizeof(float) * N);
  float *r_cpu = (float *)malloc(sizeof(float) * N);
  float *p_cpu = (float *)malloc(sizeof(float) * N);
  float *x_cpu = (float *)malloc(sizeof(float) * N);

  for (int i = 0; i < N; i++) {
    r_cpu[i] = 1.0;
    Ax_cpu[i] = x_cpu[i] = 0.0;
  }
#endif

  printf("Total threads per GPU = %d numBlocksPerSm  = %d\n",
         numSms * numBlocksPerSm * THREADS_PER_BLOCK, numBlocksPerSm);
  dim3 dimGrid(numSms * numBlocksPerSm, 1, 1),
      dimBlock(THREADS_PER_BLOCK, 1, 1);

  // Structure used for cross-grid synchronization.
  MultiDeviceData multi_device_data;
  checkCudaErrors(hipHostAlloc(
      &multi_device_data.hostMemoryArrivedList,
      (kNumGpusRequired - 1) * sizeof(*multi_device_data.hostMemoryArrivedList),
      hipHostMallocPortable));
  memset(multi_device_data.hostMemoryArrivedList, 0,
         (kNumGpusRequired - 1) *
             sizeof(*multi_device_data.hostMemoryArrivedList));
  multi_device_data.numDevices = kNumGpusRequired;
  multi_device_data.deviceRank = 0;

  void *kernelArgs[] = {
      (void *)&I,  (void *)&J, (void *)&val, (void *)&x,
      (void *)&Ax, (void *)&p, (void *)&r,   (void *)&dot_result,
      (void *)&nz, (void *)&N, (void *)&tol, (void *)&multi_device_data,
  };

  printf("Launching kernel\n");

  deviceId = bestFitDeviceIds.begin();
  device_count = 0;
  while (deviceId != bestFitDeviceIds.end()) {
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipLaunchCooperativeKernel(
        (void *)multiGpuConjugateGradient, dimGrid, dimBlock, kernelArgs,
        sMemSize, nStreams[device_count++]));
    multi_device_data.deviceRank++;
    deviceId++;
  }

  checkCudaErrors(hipMemPrefetchAsync(x, sizeof(float) * N, hipCpuDeviceId));
  checkCudaErrors(
      hipMemPrefetchAsync(dot_result, sizeof(double), hipCpuDeviceId));

  deviceId = bestFitDeviceIds.begin();
  device_count = 0;
  while (deviceId != bestFitDeviceIds.end()) {
    checkCudaErrors(hipSetDevice(*deviceId));
    checkCudaErrors(hipStreamSynchronize(nStreams[device_count++]));
    deviceId++;
  }

  r1 = (float)*dot_result;

  printf("GPU Final, residual = %e \n  ", sqrt(r1));

#if ENABLE_CPU_DEBUG_CODE
  cpuConjugateGrad(I, J, val, x_cpu, Ax_cpu, p_cpu, r_cpu, nz, N, tol);
#endif

  float rsum, diff, err = 0.0;

  for (int i = 0; i < N; i++) {
    rsum = 0.0;

    for (int j = I[i]; j < I[i + 1]; j++) {
      rsum += val_cpu[j] * x[J[j]];
    }

    diff = fabs(rsum - rhs);

    if (diff > err) {
      err = diff;
    }
  }

  checkCudaErrors(hipHostFree(multi_device_data.hostMemoryArrivedList));
  checkCudaErrors(hipFree(I));
  checkCudaErrors(hipFree(J));
  checkCudaErrors(hipFree(val));
  checkCudaErrors(hipFree(x));
  checkCudaErrors(hipFree(r));
  checkCudaErrors(hipFree(p));
  checkCudaErrors(hipFree(Ax));
  checkCudaErrors(hipFree(dot_result));
  free(val_cpu);

#if ENABLE_CPU_DEBUG_CODE
  free(Ax_cpu);
  free(r_cpu);
  free(p_cpu);
  free(x_cpu);
#endif

  printf("Test Summary:  Error amount = %f \n", err);
  fprintf(stdout, "&&&& conjugateGradientMultiDeviceCG %s\n",
          (sqrt(r1) < tol) ? "PASSED" : "FAILED");
  exit((sqrt(r1) < tol) ? EXIT_SUCCESS : EXIT_FAILURE);
}
