/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

///////////////////////////////////////////////////////////////////////////////

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>

// Round a / b to nearest higher integer value
int cuda_iDivUp(int a, int b) { return (a + (b - 1)) / b; }

// complex math functions
__device__ float2 conjugate(float2 arg) { return make_float2(arg.x, -arg.y); }

__device__ float2 complex_exp(float arg) {
  return make_float2(cosf(arg), sinf(arg));
}

__device__ float2 complex_add(float2 a, float2 b) {
  return make_float2(a.x + b.x, a.y + b.y);
}

__device__ float2 complex_mult(float2 ab, float2 cd) {
  return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x);
}

// generate wave heightfield at time t based on initial heightfield and
// dispersion relationship
__global__ void generateSpectrumKernel(float2 *h0, float2 *ht,
                                       unsigned int in_width,
                                       unsigned int out_width,
                                       unsigned int out_height, float t,
                                       float patchSize) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int in_index = y * in_width + x;
  unsigned int in_mindex =
      (out_height - y) * in_width + (out_width - x);  // mirrored
  unsigned int out_index = y * out_width + x;

  // calculate wave vector
  float2 k;
  k.x = (-(int)out_width / 2.0f + x) * (2.0f * HIP_PI_F / patchSize);
  k.y = (-(int)out_width / 2.0f + y) * (2.0f * HIP_PI_F / patchSize);

  // calculate dispersion w(k)
  float k_len = sqrtf(k.x * k.x + k.y * k.y);
  float w = sqrtf(9.81f * k_len);

  if ((x < out_width) && (y < out_height)) {
    float2 h0_k = h0[in_index];
    float2 h0_mk = h0[in_mindex];

    // output frequency-space complex values
    ht[out_index] =
        complex_add(complex_mult(h0_k, complex_exp(w * t)),
                    complex_mult(conjugate(h0_mk), complex_exp(-w * t)));
    // ht[out_index] = h0_k;
  }
}

// update height map values based on output of FFT
__global__ void updateHeightmapKernel(float *heightMap, float2 *ht,
                                      unsigned int width) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int i = y * width + x;

  // cos(pi * (m1 + m2))
  float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;

  heightMap[i] = ht[i].x * sign_correction;
}

// update height map values based on output of FFT
__global__ void updateHeightmapKernel_y(float *heightMap, float2 *ht,
                                        unsigned int width) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int i = y * width + x;

  // cos(pi * (m1 + m2))
  float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;

  heightMap[i] = ht[i].y * sign_correction;
}

// generate slope by partial differences in spatial domain
__global__ void calculateSlopeKernel(float *h, float2 *slopeOut,
                                     unsigned int width, unsigned int height) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int i = y * width + x;

  float2 slope = make_float2(0.0f, 0.0f);

  if ((x > 0) && (y > 0) && (x < width - 1) && (y < height - 1)) {
    slope.x = h[i + 1] - h[i - 1];
    slope.y = h[i + width] - h[i - width];
  }

  slopeOut[i] = slope;
}

// wrapper functions
extern "C" void cudaGenerateSpectrumKernel(float2 *d_h0, float2 *d_ht,
                                           unsigned int in_width,
                                           unsigned int out_width,
                                           unsigned int out_height,
                                           float animTime, float patchSize) {
  dim3 block(8, 8, 1);
  dim3 grid(cuda_iDivUp(out_width, block.x), cuda_iDivUp(out_height, block.y),
            1);
  generateSpectrumKernel<<<grid, block>>>(d_h0, d_ht, in_width, out_width,
                                          out_height, animTime, patchSize);
}

extern "C" void cudaUpdateHeightmapKernel(float *d_heightMap, float2 *d_ht,
                                          unsigned int width,
                                          unsigned int height, bool autoTest) {
  dim3 block(8, 8, 1);
  dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
  if (autoTest) {
    updateHeightmapKernel_y<<<grid, block>>>(d_heightMap, d_ht, width);
  } else {
    updateHeightmapKernel<<<grid, block>>>(d_heightMap, d_ht, width);
  }
}

extern "C" void cudaCalculateSlopeKernel(float *hptr, float2 *slopeOut,
                                         unsigned int width,
                                         unsigned int height) {
  dim3 block(8, 8, 1);
  dim3 grid2(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
  calculateSlopeKernel<<<grid2, block>>>(hptr, slopeOut, width, height);
}
