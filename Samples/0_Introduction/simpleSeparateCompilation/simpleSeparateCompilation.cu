#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes.
#include <stdio.h>
#include <iostream>

// STL.
#include <vector>

// CUDA runtime.
#include <hip/hip_runtime.h>

// Helper functions and utilities to work with CUDA.
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Device library includes.
#include "simpleDeviceLibrary.cuh"

using std::cout;
using std::endl;

using std::vector;

#define EPS 1e-5

typedef unsigned int uint;
typedef float (*deviceFunc)(float);

const char *sampleName = "simpleSeparateCompilation";

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Static device pointers to __device__ functions.
__device__ deviceFunc dMultiplyByTwoPtr = multiplyByTwo;
__device__ deviceFunc dDivideByTwoPtr = divideByTwo;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Transforms vector.
//! Applies the __device__ function "f" to each element of the vector "v".
////////////////////////////////////////////////////////////////////////////////
__global__ void transformVector(float *v, deviceFunc f, uint size) {
  uint tid = blockIdx.x * blockDim.x + threadIdx.x;

  if (tid < size) {
    v[tid] = (*f)(v[tid]);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, const char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  cout << sampleName << " starting..." << endl;

  runTest(argc, (const char **)argv);

  cout << sampleName << " completed, returned " << (testResult ? "OK" : "ERROR")
       << endl;

  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

void runTest(int argc, const char **argv) {
  try {
    // This will pick the best possible CUDA capable device.
    findCudaDevice(argc, (const char **)argv);

    // Create host vector.
    const uint kVectorSize = 1000;

    vector<float> hVector(kVectorSize);

    for (uint i = 0; i < kVectorSize; ++i) {
      hVector[i] = rand() / static_cast<float>(RAND_MAX);
    }

    // Create and populate device vector.
    float *dVector;
    checkCudaErrors(hipMalloc(&dVector, kVectorSize * sizeof(float)));

    checkCudaErrors(hipMemcpy(dVector, &hVector[0],
                               kVectorSize * sizeof(float),
                               hipMemcpyHostToDevice));

    // Kernel configuration, where a one-dimensional
    // grid and one-dimensional blocks are configured.
    const int nThreads = 1024;
    const int nBlocks = 1;

    dim3 dimGrid(nBlocks);
    dim3 dimBlock(nThreads);

    // Test library functions.
    deviceFunc hFunctionPtr;

    hipMemcpyFromSymbol(&hFunctionPtr, HIP_SYMBOL(dMultiplyByTwoPtr), sizeof(deviceFunc));
    transformVector<<<dimGrid, dimBlock>>>(dVector, hFunctionPtr, kVectorSize);
    checkCudaErrors(hipGetLastError());

    hipMemcpyFromSymbol(&hFunctionPtr, HIP_SYMBOL(dDivideByTwoPtr), sizeof(deviceFunc));
    transformVector<<<dimGrid, dimBlock>>>(dVector, hFunctionPtr, kVectorSize);
    checkCudaErrors(hipGetLastError());

    // Download results.
    vector<float> hResultVector(kVectorSize);

    checkCudaErrors(hipMemcpy(&hResultVector[0], dVector,
                               kVectorSize * sizeof(float),
                               hipMemcpyDeviceToHost));

    // Check results.
    for (int i = 0; i < kVectorSize; ++i) {
      if (fabs(hVector[i] - hResultVector[i]) > EPS) {
        cout << "Computations were incorrect..." << endl;
        testResult = false;
        break;
      }
    }

    // Free resources.
    if (dVector) checkCudaErrors(hipFree(dVector));
  } catch (...) {
    cout << "Error occured, exiting..." << endl;

    exit(EXIT_FAILURE);
  }
}
