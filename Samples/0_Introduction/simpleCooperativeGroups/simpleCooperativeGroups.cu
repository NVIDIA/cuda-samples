#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 *
 * This sample is a simple code that illustrates basic usage of
 * cooperative groups within the thread block. The code launches a single
 * thread block, creates a cooperative group of all threads in the block,
 * and a set of tiled partition cooperative groups. For each, it uses a
 * generic reduction function to calculate the sum of all the ranks in
 * that group. In each case the result is printed, together with the
 * expected answer (which is calculated using the analytical formula
 * (n-1)*n)/2, noting that the ranks start at zero).
 *
 */

#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

using namespace cooperative_groups;

/**
 * CUDA device function
 *
 * calculates the sum of val across the group g. The workspace array, x,
 * must be large enough to contain g.size() integers.
 */
__device__ int sumReduction(thread_group g, int *x, int val) {
  // rank of this thread in the group
  int lane = g.thread_rank();

  // for each iteration of this loop, the number of threads active in the
  // reduction, i, is halved, and each active thread (with index [lane])
  // performs a single summation of it's own value with that
  // of a "partner" (with index [lane+i]).
  for (int i = g.size() / 2; i > 0; i /= 2) {
    // store value for this thread in temporary array
    x[lane] = val;

    // synchronize all threads in group
    g.sync();

    if (lane < i)
      // active threads perform summation of their value with
      // their partner's value
      val += x[lane + i];

    // synchronize all threads in group
    g.sync();
  }

  // master thread in group returns result, and others return -1.
  if (g.thread_rank() == 0)
    return val;
  else
    return -1;
}

/**
 * CUDA kernel device code
 *
 * Creates cooperative groups and performs reductions
 */
__global__ void cgkernel() {
  // threadBlockGroup includes all threads in the block
  thread_block threadBlockGroup = this_thread_block();
  int threadBlockGroupSize = threadBlockGroup.size();

  // workspace array in shared memory required for reduction
  extern __shared__ int workspace[];

  int input, output, expectedOutput;

  // input to reduction, for each thread, is its' rank in the group
  input = threadBlockGroup.thread_rank();

  // expected output from analytical formula (n-1)(n)/2
  // (noting that indexing starts at 0 rather than 1)
  expectedOutput = (threadBlockGroupSize - 1) * threadBlockGroupSize / 2;

  // perform reduction
  output = sumReduction(threadBlockGroup, workspace, input);

  // master thread in group prints out result
  if (threadBlockGroup.thread_rank() == 0) {
    printf(
        " Sum of all ranks 0..%d in threadBlockGroup is %d (expected %d)\n\n",
        (int)threadBlockGroup.size() - 1, output, expectedOutput);

    printf(" Now creating %d groups, each of size 16 threads:\n\n",
           (int)threadBlockGroup.size() / 16);
  }

  threadBlockGroup.sync();

  // each tiledPartition16 group includes 16 threads
  thread_block_tile<16> tiledPartition16 =
      tiled_partition<16>(threadBlockGroup);

  // This offset allows each group to have its own unique area in the workspace
  // array
  int workspaceOffset =
      threadBlockGroup.thread_rank() - tiledPartition16.thread_rank();

  // input to reduction, for each thread, is its' rank in the group
  input = tiledPartition16.thread_rank();

  // expected output from analytical formula (n-1)(n)/2
  // (noting that indexing starts at 0 rather than 1)
  expectedOutput = 15 * 16 / 2;

  // Perform reduction
  output = sumReduction(tiledPartition16, workspace + workspaceOffset, input);

  // each master thread prints out result
  if (tiledPartition16.thread_rank() == 0)
    printf(
        "   Sum of all ranks 0..15 in this tiledPartition16 group is %d "
        "(expected %d)\n",
        output, expectedOutput);

  return;
}

/**
 * Host main routine
 */
int main() {
  // Error code to check return values for CUDA calls
  hipError_t err;

  // Launch the kernel

  int blocksPerGrid = 1;
  int threadsPerBlock = 64;

  printf("\nLaunching a single block with %d threads...\n\n", threadsPerBlock);

  // we use the optional third argument to specify the size
  // of shared memory required in the kernel
  cgkernel<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(int)>>>();
  err = hipDeviceSynchronize();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  printf("\n...Done.\n\n");

  return 0;
}
