#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
* This sample demonstrates how to use texture fetches from layered 2D textures
* in CUDA C
*
* This sample first generates a 3D input data array for the layered texture
* and the expected output. Then it starts CUDA C kernels, one for each layer,
* which fetch their layer's texture data (using normalized texture coordinates)
* transform it to the expected output, and write it to a 3D output data array.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, kernels
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper for shared that are common to CUDA Samples

static const char *sSDKname = "simpleLayeredTexture";

////////////////////////////////////////////////////////////////////////////////
//! Transform a layer of a layered 2D texture using texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void transformKernel(float *g_odata, int width, int height,
                                int layer, hipTextureObject_t tex) {
  // calculate this thread's data point
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  // 0.5f offset and division are necessary to access the original data points
  // in the texture (such that bilinear interpolation will not be activated).
  // For details, see also CUDA Programming Guide, Appendix D
  float u = (x + 0.5f) / (float)width;
  float v = (y + 0.5f) / (float)height;

  // read from texture, do expected transformation and write to global memory
  g_odata[layer * width * height + y * width + x] =
      -tex2DLayered<float>(tex, u, v, layer) + layer;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("[%s] - Starting...\n", sSDKname);

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  int devID = findCudaDevice(argc, (const char **)argv);

  bool bResult = true;

  // get number of SMs on this GPU
  hipDeviceProp_t deviceProps;

  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s] has %d Multi-Processors ", deviceProps.name,
         deviceProps.multiProcessorCount);
  printf("SM %d.%d\n", deviceProps.major, deviceProps.minor);

  // generate input data for layered texture
  unsigned int width = 512, height = 512, num_layers = 5;
  unsigned int size = width * height * num_layers * sizeof(float);
  float *h_data = (float *)malloc(size);

  for (unsigned int layer = 0; layer < num_layers; layer++)
    for (int i = 0; i < (int)(width * height); i++) {
      h_data[layer * width * height + i] = (float)i;
    }

  // this is the expected transformation of the input data (the expected output)
  float *h_data_ref = (float *)malloc(size);

  for (unsigned int layer = 0; layer < num_layers; layer++)
    for (int i = 0; i < (int)(width * height); i++) {
      h_data_ref[layer * width * height + i] =
          -h_data[layer * width * height + i] + layer;
    }

  // allocate device memory for result
  float *d_data = NULL;
  checkCudaErrors(hipMalloc((void **)&d_data, size));

  // allocate array and copy image data
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray *cu_3darray;
  checkCudaErrors(hipMalloc3DArray(&cu_3darray, &channelDesc,
                                    make_hipExtent(width, height, num_layers),
                                    hipArrayLayered));
  hipMemcpy3DParms myparms = {0};
  myparms.srcPos = make_hipPos(0, 0, 0);
  myparms.dstPos = make_hipPos(0, 0, 0);
  myparms.srcPtr =
      make_hipPitchedPtr(h_data, width * sizeof(float), width, height);
  myparms.dstArray = cu_3darray;
  myparms.extent = make_hipExtent(width, height, num_layers);
  myparms.kind = hipMemcpyHostToDevice;
  checkCudaErrors(hipMemcpy3D(&myparms));

  hipTextureObject_t tex;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = cu_3darray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

  printf(
      "Covering 2D data array of %d x %d: Grid size is %d x %d, each block has "
      "8 x 8 threads\n",
      width, height, dimGrid.x, dimGrid.y);

  transformKernel<<<dimGrid, dimBlock>>>(d_data, width, height, 0,
                                         tex);  // warmup (for better timing)

  // check if kernel execution generated an error
  getLastCudaError("warmup Kernel execution failed");

  checkCudaErrors(hipDeviceSynchronize());

  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  // execute the kernel
  for (unsigned int layer = 0; layer < num_layers; layer++)
    transformKernel<<<dimGrid, dimBlock, 0>>>(d_data, width, height, layer,
                                              tex);

  // check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  printf("Processing time: %.3f msec\n", sdkGetTimerValue(&timer));
  printf("%.2f Mtexlookups/sec\n",
         (width * height * num_layers / (sdkGetTimerValue(&timer) / 1000.0f) /
          1e6));
  sdkDeleteTimer(&timer);

  // allocate mem for the result on host side
  float *h_odata = (float *)malloc(size);
  // copy result from device to host
  checkCudaErrors(hipMemcpy(h_odata, d_data, size, hipMemcpyDeviceToHost));

  // write regression file if necessary
  if (checkCmdLineFlag(argc, (const char **)argv, "regression")) {
    // write file for regression test
    sdkWriteFile<float>("./data/regression.dat", h_odata, width * height, 0.0f,
                        false);
  } else {
    printf("Comparing kernel output to expected data\n");

#define MIN_EPSILON_ERROR 5e-3f
    bResult = compareData(h_odata, h_data_ref, width * height * num_layers,
                          MIN_EPSILON_ERROR, 0.0f);
  }

  // cleanup memory
  free(h_data);
  free(h_data_ref);
  free(h_odata);

  checkCudaErrors(hipDestroyTextureObject(tex));
  checkCudaErrors(hipFree(d_data));
  checkCudaErrors(hipFreeArray(cu_3darray));

  exit(bResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
