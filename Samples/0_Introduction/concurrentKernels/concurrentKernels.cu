#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//
// This sample demonstrates the use of streams for concurrent execution. It also
// illustrates how to introduce dependencies between CUDA streams with the
// hipStreamWaitEvent function.
//

// Devices of compute capability 2.0 or higher can overlap the kernels
//
#include <hip/hip_cooperative_groups.h>
#include <stdio.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// This is a kernel that does no real work but runs at least for a specified
// number of clocks
__global__ void clock_block(clock_t *d_o, clock_t clock_count) {
  unsigned int start_clock = (unsigned int)clock();

  clock_t clock_offset = 0;

  while (clock_offset < clock_count) {
    unsigned int end_clock = (unsigned int)clock();

    // The code below should work like
    // this (thanks to modular arithmetics):
    //
    // clock_offset = (clock_t) (end_clock > start_clock ?
    //                           end_clock - start_clock :
    //                           end_clock + (0xffffffffu - start_clock));
    //
    // Indeed, let m = 2^32 then
    // end - start = end + m - start (mod m).

    clock_offset = (clock_t)(end_clock - start_clock);
  }

  d_o[0] = clock_offset;
}

// Single warp reduction kernel
__global__ void sum(clock_t *d_clocks, int N) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ clock_t s_clocks[32];

  clock_t my_sum = 0;

  for (int i = threadIdx.x; i < N; i += blockDim.x) {
    my_sum += d_clocks[i];
  }

  s_clocks[threadIdx.x] = my_sum;
  cg::sync(cta);

  for (int i = 16; i > 0; i /= 2) {
    if (threadIdx.x < i) {
      s_clocks[threadIdx.x] += s_clocks[threadIdx.x + i];
    }

    cg::sync(cta);
  }

  d_clocks[0] = s_clocks[0];
}

int main(int argc, char **argv) {
  int nkernels = 8;             // number of concurrent kernels
  int nstreams = nkernels + 1;  // use one more stream than concurrent kernel
  int nbytes = nkernels * sizeof(clock_t);  // number of data bytes
  float kernel_time = 10;                   // time the kernel should run in ms
  float elapsed_time;                       // timing variables
  int cuda_device = 0;

  printf("[%s] - Starting...\n", argv[0]);

  // get number of kernels if overridden on the command line
  if (checkCmdLineFlag(argc, (const char **)argv, "nkernels")) {
    nkernels = getCmdLineArgumentInt(argc, (const char **)argv, "nkernels");
    nstreams = nkernels + 1;
  }

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  cuda_device = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDevice(&cuda_device));

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

  if ((deviceProp.concurrentKernels == 0)) {
    printf("> GPU does not support concurrent kernel execution\n");
    printf("  CUDA kernel runs will be serialized\n");
  }

  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

  // allocate host memory
  clock_t *a = 0;  // pointer to the array data in host memory
  checkCudaErrors(hipHostMalloc((void **)&a, nbytes));

  // allocate device memory
  clock_t *d_a = 0;  // pointers to data and init value in the device memory
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));

  // allocate and initialize an array of stream handles
  hipStream_t *streams =
      (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

  for (int i = 0; i < nstreams; i++) {
    checkCudaErrors(hipStreamCreate(&(streams[i])));
  }

  // create CUDA event handles
  hipEvent_t start_event, stop_event;
  checkCudaErrors(hipEventCreate(&start_event));
  checkCudaErrors(hipEventCreate(&stop_event));

  // the events are used for synchronization only and hence do not need to
  // record timings this also makes events not introduce global sync points when
  // recorded which is critical to get overlap
  hipEvent_t *kernelEvent;
  kernelEvent = (hipEvent_t *)malloc(nkernels * sizeof(hipEvent_t));

  for (int i = 0; i < nkernels; i++) {
    checkCudaErrors(
        hipEventCreateWithFlags(&(kernelEvent[i]), hipEventDisableTiming));
  }

  //////////////////////////////////////////////////////////////////////
  // time execution with nkernels streams
  clock_t total_clocks = 0;
#if defined(__arm__) || defined(__aarch64__)
  // the kernel takes more time than the channel reset time on arm archs, so to
  // prevent hangs reduce time_clocks.
  clock_t time_clocks = (clock_t)(kernel_time * (deviceProp.clockRate / 100));
#else
  clock_t time_clocks = (clock_t)(kernel_time * deviceProp.clockRate);
#endif

  hipEventRecord(start_event, 0);

  // queue nkernels in separate streams and record when they are done
  for (int i = 0; i < nkernels; ++i) {
    clock_block<<<1, 1, 0, streams[i]>>>(&d_a[i], time_clocks);
    total_clocks += time_clocks;
    checkCudaErrors(hipEventRecord(kernelEvent[i], streams[i]));

    // make the last stream wait for the kernel event to be recorded
    checkCudaErrors(
        hipStreamWaitEvent(streams[nstreams - 1], kernelEvent[i], 0));
  }

  // queue a sum kernel and a copy back to host in the last stream.
  // the commands in this stream get dispatched as soon as all the kernel events
  // have been recorded
  sum<<<1, 32, 0, streams[nstreams - 1]>>>(d_a, nkernels);
  checkCudaErrors(hipMemcpyAsync(
      a, d_a, sizeof(clock_t), hipMemcpyDeviceToHost, streams[nstreams - 1]));

  // at this point the CPU has dispatched all work for the GPU and can continue
  // processing other tasks in parallel

  // in this sample we just wait until the GPU is done
  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(stop_event));
  checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));

  printf("Expected time for serial execution of %d kernels = %.3fs\n", nkernels,
         nkernels * kernel_time / 1000.0f);
  printf("Expected time for concurrent execution of %d kernels = %.3fs\n",
         nkernels, kernel_time / 1000.0f);
  printf("Measured time for sample = %.3fs\n", elapsed_time / 1000.0f);

  bool bTestResult = (a[0] > total_clocks);

  // release resources
  for (int i = 0; i < nkernels; i++) {
    hipStreamDestroy(streams[i]);
    hipEventDestroy(kernelEvent[i]);
  }

  free(streams);
  free(kernelEvent);

  hipEventDestroy(start_event);
  hipEventDestroy(stop_event);
  hipHostFree(a);
  hipFree(d_a);

  if (!bTestResult) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
