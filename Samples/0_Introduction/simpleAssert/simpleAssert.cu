#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#else
#include <sys/utsname.h>
#endif

// Includes, system
#include <stdio.h>
#include <cassert>

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

const char *sampleName = "simpleAssert";

////////////////////////////////////////////////////////////////////////////////
// Auto-Verification Code
bool testResult = true;

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Tests assert function.
//! Thread whose id > N will print assertion failed error message.
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel(int N) {
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  assert(gtid < N);
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("%s starting...\n", sampleName);

  runTest(argc, argv);

  printf("%s completed, returned %s\n", sampleName,
         testResult ? "OK" : "ERROR!");
  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

void runTest(int argc, char **argv) {
  int Nblocks = 2;
  int Nthreads = 32;
  hipError_t error;

#ifndef _WIN32
  utsname OS_System_Type;
  uname(&OS_System_Type);

  printf("OS_System_Type.release = %s\n", OS_System_Type.release);

  if (!strcasecmp(OS_System_Type.sysname, "Darwin")) {
    printf("simpleAssert is not current supported on Mac OSX\n\n");
    exit(EXIT_SUCCESS);
  } else {
    printf("OS Info: <%s>\n\n", OS_System_Type.version);
  }

#endif

  // This will pick the best possible CUDA capable device
  findCudaDevice(argc, (const char **)argv);

  // Kernel configuration, where a one-dimensional
  // grid and one-dimensional blocks are configured.
  dim3 dimGrid(Nblocks);
  dim3 dimBlock(Nthreads);

  printf("Launch kernel to generate assertion failures\n");
  testKernel<<<dimGrid, dimBlock>>>(60);

  // Synchronize (flushes assert output).
  printf("\n-- Begin assert output\n\n");
  error = hipDeviceSynchronize();
  printf("\n-- End assert output\n\n");

  // Check for errors and failed asserts in asynchronous kernel launch.
  if (error == hipErrorAssert) {
    printf(
        "Device assert failed as expected, "
        "CUDA error message is: %s\n\n",
        hipGetErrorString(error));
  }

  testResult = error == hipErrorAssert;
}
