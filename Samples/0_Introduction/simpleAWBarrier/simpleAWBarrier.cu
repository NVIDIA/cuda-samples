#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Includes, system
#include <stdio.h>

// Includes CUDA
#include <hip/hip_runtime.h>
#include <cuda/barrier>
#include <hip/hip_cooperative_groups.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

namespace cg = cooperative_groups;

#if __CUDA_ARCH__ >= 700
template <bool writeSquareRoot>
__device__ void reduceBlockData(
    cuda::barrier<cuda::thread_scope_block> &barrier,
    cg::thread_block_tile<32> &tile32, double &threadSum, double *result) {
  extern __shared__ double tmp[];

#pragma unroll
  for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
    threadSum += tile32.shfl_down(threadSum, offset);
  }
  if (tile32.thread_rank() == 0) {
    tmp[tile32.meta_group_rank()] = threadSum;
  }

  auto token = barrier.arrive();

  barrier.wait(std::move(token));

  // The warp 0 will perform last round of reduction
  if (tile32.meta_group_rank() == 0) {
    double beta = tile32.thread_rank() < tile32.meta_group_size()
                      ? tmp[tile32.thread_rank()]
                      : 0.0;

#pragma unroll
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      beta += tile32.shfl_down(beta, offset);
    }

    if (tile32.thread_rank() == 0) {
      if (writeSquareRoot)
        *result = sqrt(beta);
      else
        *result = beta;
    }
  }
}
#endif

__global__ void normVecByDotProductAWBarrier(float *vecA, float *vecB,
                                             double *partialResults, int size) {
#if __CUDA_ARCH__ >= 700
#pragma diag_suppress static_var_with_dynamic_init
  cg::thread_block cta = cg::this_thread_block();
  cg::grid_group grid = cg::this_grid();
  ;
  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  __shared__ cuda::barrier<cuda::thread_scope_block> barrier;

  if (threadIdx.x == 0) {
    init(&barrier, blockDim.x);
  }

  cg::sync(cta);

  double threadSum = 0.0;
  for (int i = grid.thread_rank(); i < size; i += grid.size()) {
    threadSum += (double)(vecA[i] * vecB[i]);
  }

  // Each thread block performs reduction of partial dotProducts and writes to
  // global mem.
  reduceBlockData<false>(barrier, tile32, threadSum,
                         &partialResults[blockIdx.x]);

  cg::sync(grid);

  // One block performs the final summation of partial dot products
  // of all the thread blocks and writes the sqrt of final dot product.
  if (blockIdx.x == 0) {
    threadSum = 0.0;
    for (int i = cta.thread_rank(); i < gridDim.x; i += cta.size()) {
      threadSum += partialResults[i];
    }
    reduceBlockData<true>(barrier, tile32, threadSum, &partialResults[0]);
  }

  cg::sync(grid);

  const double finalValue = partialResults[0];

  // Perform normalization of vecA & vecB.
  for (int i = grid.thread_rank(); i < size; i += grid.size()) {
    vecA[i] = (float)vecA[i] / finalValue;
    vecB[i] = (float)vecB[i] / finalValue;
  }
#endif
}

int runNormVecByDotProductAWBarrier(int argc, char **argv, int deviceId);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("%s starting...\n", argv[0]);

  // This will pick the best possible CUDA capable device
  int dev = findCudaDevice(argc, (const char **)argv);

  int major = 0;
  checkCudaErrors(
      hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, dev));

  // Arrive-Wait Barrier require a GPU of Volta (SM7X) architecture or higher.
  if (major < 7) {
    printf("simpleAWBarrier requires SM 7.0 or higher.  Exiting...\n");
    exit(EXIT_WAIVED);
  }

  int supportsCooperativeLaunch = 0;
  checkCudaErrors(hipDeviceGetAttribute(&supportsCooperativeLaunch,
                                         hipDeviceAttributeCooperativeLaunch, dev));

  if (!supportsCooperativeLaunch) {
    printf(
        "\nSelected GPU (%d) does not support Cooperative Kernel Launch, "
        "Waiving the run\n",
        dev);
    exit(EXIT_WAIVED);
  }

  int testResult = runNormVecByDotProductAWBarrier(argc, argv, dev);

  printf("%s completed, returned %s\n", argv[0], testResult ? "OK" : "ERROR!");
  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

int runNormVecByDotProductAWBarrier(int argc, char **argv, int deviceId) {
  float *vecA, *d_vecA;
  float *vecB, *d_vecB;
  double *d_partialResults;
  int size = 10000000;

  checkCudaErrors(hipHostMalloc(&vecA, sizeof(float) * size));
  checkCudaErrors(hipHostMalloc(&vecB, sizeof(float) * size));

  checkCudaErrors(hipMalloc(&d_vecA, sizeof(float) * size));
  checkCudaErrors(hipMalloc(&d_vecB, sizeof(float) * size));

  float baseVal = 2.0;
  for (int i = 0; i < size; i++) {
    vecA[i] = vecB[i] = baseVal;
  }

  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  checkCudaErrors(hipMemcpyAsync(d_vecA, vecA, sizeof(float) * size,
                                  hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipMemcpyAsync(d_vecB, vecB, sizeof(float) * size,
                                  hipMemcpyHostToDevice, stream));

  // Kernel configuration, where a one-dimensional
  // grid and one-dimensional blocks are configured.
  int minGridSize = 0, blockSize = 0;
  checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
      &minGridSize, &blockSize, (void *)normVecByDotProductAWBarrier, 0, size));

  int smemSize = ((blockSize / 32) + 1) * sizeof(double);

  int numBlocksPerSm = 0;
  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &numBlocksPerSm, normVecByDotProductAWBarrier, blockSize, smemSize));

  int multiProcessorCount = 0;
  checkCudaErrors(hipDeviceGetAttribute(
      &multiProcessorCount, hipDeviceAttributeMultiprocessorCount, deviceId));

  minGridSize = multiProcessorCount * numBlocksPerSm;
  checkCudaErrors(hipMalloc(&d_partialResults, minGridSize * sizeof(double)));

  printf(
      "Launching normVecByDotProductAWBarrier kernel with numBlocks = %d "
      "blockSize = %d\n",
      minGridSize, blockSize);

  dim3 dimGrid(minGridSize, 1, 1), dimBlock(blockSize, 1, 1);

  void *kernelArgs[] = {(void *)&d_vecA, (void *)&d_vecB,
                        (void *)&d_partialResults, (void *)&size};

  checkCudaErrors(
      hipLaunchCooperativeKernel((void *)normVecByDotProductAWBarrier, dimGrid,
                                  dimBlock, kernelArgs, smemSize, stream));

  checkCudaErrors(hipMemcpyAsync(vecA, d_vecA, sizeof(float) * size,
                                  hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  float expectedResult = (baseVal / sqrt(size * baseVal * baseVal));
  unsigned int matches = 0;
  for (int i = 0; i < size; i++) {
    if ((vecA[i] - expectedResult) > 0.00001) {
      printf("mismatch at i = %d\n", i);
      break;
    } else {
      matches++;
    }
  }

  printf("Result = %s\n", matches == size ? "PASSED" : "FAILED");
  checkCudaErrors(hipFree(d_vecA));
  checkCudaErrors(hipFree(d_vecB));
  checkCudaErrors(hipFree(d_partialResults));

  checkCudaErrors(hipHostFree(vecA));
  checkCudaErrors(hipHostFree(vecB));
  return matches == size;
}
