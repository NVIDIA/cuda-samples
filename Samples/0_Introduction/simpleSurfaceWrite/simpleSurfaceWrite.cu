#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample demonstrates how use texture fetches in CUDA
 *
 * This sample takes an input PGM image (imageFilename) and generates
 * an output PGM image (imageFilename_out).  This CUDA kernel performs
 * a simple 2D transform (rotation) on the texture coordinates (u,v).
 */

// Includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

#define MIN_EPSILON_ERROR 5e-3f

////////////////////////////////////////////////////////////////////////////////
// Define the files that are to be save and the reference images for validation
const char *imageFilename = "teapot512.pgm";
const char *refFilename = "ref_rotated.pgm";
float angle = 0.5f;  // angle to rotate image by (in radians)

// Auto-Verification Code
bool testResult = true;

static const char *sampleName = "simpleSurfaceWrite";

////////////////////////////////////////////////////////////////////////////////
// Kernels
////////////////////////////////////////////////////////////////////////////////
//! Write to a cuArray (texture data source) using surface writes
//! @param gIData input data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void surfaceWriteKernel(float *gIData, int width, int height,
                                   hipSurfaceObject_t outputSurface) {
  // calculate surface coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  // read from global memory and write to cuarray (via surface reference)
  surf2Dwrite(gIData[y * width + x], outputSurface, x * 4, y,
              hipBoundaryModeTrap);
}

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param gOData  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void transformKernel(float *gOData, int width, int height,
                                float theta, hipTextureObject_t tex) {
  // calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = x / (float)width;
  float v = y / (float)height;

  // transform coordinates
  u -= 0.5f;
  v -= 0.5f;
  float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
  float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

  // read from texture and write to global memory
  gOData[y * width + x] = tex2D<float>(tex, tu, tv);
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

extern "C" void computeGold(float *reference, float *idata,
                            const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("%s starting...\n", sampleName);

  // Process command-line arguments
  if (argc > 1) {
    if (checkCmdLineFlag(argc, (const char **)argv, "input")) {
      getCmdLineArgumentString(argc, (const char **)argv, "input",
                               (char **)&imageFilename);

      if (checkCmdLineFlag(argc, (const char **)argv, "reference")) {
        getCmdLineArgumentString(argc, (const char **)argv, "reference",
                                 (char **)&refFilename);
      } else {
        printf("-input flag should be used with -reference flag");
        exit(EXIT_FAILURE);
      }
    } else if (checkCmdLineFlag(argc, (const char **)argv, "reference")) {
      printf("-reference flag should be used with -input flag");
      exit(EXIT_FAILURE);
    }
  }

  runTest(argc, argv);

  printf("%s completed, returned %s\n", sampleName,
         testResult ? "OK" : "ERROR!");
  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  // Use command-line specified CUDA device,
  // otherwise use device with highest Gflops/s
  int devID = findCudaDevice(argc, (const char **)argv);

  // Get number of SMs on this GPU
  hipDeviceProp_t deviceProps;

  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s] has %d Multi-Processors, SM %d.%d\n",
         deviceProps.name, deviceProps.multiProcessorCount, deviceProps.major,
         deviceProps.minor);

  // Load image from disk
  float *hData = NULL;
  unsigned int width, height;
  char *imagePath = sdkFindFilePath(imageFilename, argv[0]);

  if (imagePath == NULL) {
    printf("Unable to source image input file: %s\n", imageFilename);
    exit(EXIT_FAILURE);
  }

  sdkLoadPGM(imagePath, &hData, &width, &height);

  unsigned int size = width * height * sizeof(float);
  printf("Loaded '%s', %d x %d pixels\n", imageFilename, width, height);

  // Load reference image from image (output)
  float *hDataRef = (float *)malloc(size);
  char *refPath = sdkFindFilePath(refFilename, argv[0]);

  if (refPath == NULL) {
    printf("Unable to find reference image file: %s\n", refFilename);
    exit(EXIT_FAILURE);
  }

  sdkLoadPGM(refPath, &hDataRef, &width, &height);

  // Allocate device memory for result
  float *dData = NULL;
  checkCudaErrors(hipMalloc((void **)&dData, size));

  // Allocate array and copy image data
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray *cuArray;
  checkCudaErrors(hipMallocArray(&cuArray, &channelDesc, width, height,
                                  hipArraySurfaceLoadStore));

  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(width / dimBlock.x, height / dimBlock.y, 1);

  hipSurfaceObject_t outputSurface;
  hipResourceDesc surfRes;
  memset(&surfRes, 0, sizeof(hipResourceDesc));
  surfRes.resType = hipResourceTypeArray;
  surfRes.res.array.array = cuArray;

  checkCudaErrors(hipCreateSurfaceObject(&outputSurface, &surfRes));
#if 1
  checkCudaErrors(hipMemcpy(dData, hData, size, hipMemcpyHostToDevice));
  surfaceWriteKernel<<<dimGrid, dimBlock>>>(dData, width, height,
                                            outputSurface);
#else  // This is what differs from the example simpleTexture
  checkCudaErrors(
      hipMemcpyToArray(cuArray, 0, 0, hData, size, hipMemcpyHostToDevice));
#endif

  hipTextureObject_t tex;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = cuArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

  // Warmup
  transformKernel<<<dimGrid, dimBlock, 0>>>(dData, width, height, angle, tex);

  checkCudaErrors(hipDeviceSynchronize());

  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  // Execute the kernel
  transformKernel<<<dimGrid, dimBlock, 0>>>(dData, width, height, angle, tex);

  // Check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");

  hipDeviceSynchronize();
  sdkStopTimer(&timer);
  printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
  printf("%.2f Mpixels/sec\n",
         (width * height / (sdkGetTimerValue(&timer) / 1000.0f)) / 1e6);
  sdkDeleteTimer(&timer);

  // Allocate mem for the result on host side
  float *hOData = (float *)malloc(size);
  // copy result from device to host
  checkCudaErrors(hipMemcpy(hOData, dData, size, hipMemcpyDeviceToHost));

  // Write result to file
  char outputFilename[1024];
  strcpy(outputFilename, "output.pgm");
  sdkSavePGM("output.pgm", hOData, width, height);
  printf("Wrote '%s'\n", outputFilename);

  // Write regression file if necessary
  if (checkCmdLineFlag(argc, (const char **)argv, "regression")) {
    // Write file for regression test
    sdkWriteFile<float>("./data/regression.dat", hOData, width * height, 0.0f,
                        false);
  } else {
    // We need to reload the data from disk,
    // because it is inverted upon output
    sdkLoadPGM(outputFilename, &hOData, &width, &height);

    printf("Comparing files\n");
    printf("\toutput:    <%s>\n", outputFilename);
    printf("\treference: <%s>\n", refPath);
    testResult =
        compareData(hOData, hDataRef, width * height, MIN_EPSILON_ERROR, 0.0f);
  }

  checkCudaErrors(hipDestroySurfaceObject(outputSurface));
  checkCudaErrors(hipDestroyTextureObject(tex));
  checkCudaErrors(hipFree(dData));
  checkCudaErrors(hipFreeArray(cuArray));
  free(imagePath);
  free(refPath);
}
