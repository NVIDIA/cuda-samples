#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include <assert.h>
#include "mergeSort_common.h"

inline __device__ void Comparator(uint &keyA, uint &valA, uint &keyB,
                                  uint &valB, uint arrowDir) {
  uint t;

  if ((keyA > keyB) == arrowDir) {
    t = keyA;
    keyA = keyB;
    keyB = t;
    t = valA;
    valA = valB;
    valB = t;
  }
}

__global__ void bitonicSortSharedKernel(uint *d_DstKey, uint *d_DstVal,
                                        uint *d_SrcKey, uint *d_SrcVal,
                                        uint arrayLength, uint sortDir) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  // Shared memory storage for one or more short vectors
  __shared__ uint s_key[SHARED_SIZE_LIMIT];
  __shared__ uint s_val[SHARED_SIZE_LIMIT];

  // Offset to the beginning of subbatch and load data
  d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  s_key[threadIdx.x + 0] = d_SrcKey[0];
  s_val[threadIdx.x + 0] = d_SrcVal[0];
  s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
  s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

  for (uint size = 2; size < arrayLength; size <<= 1) {
    // Bitonic merge
    uint dir = (threadIdx.x & (size / 2)) != 0;

    for (uint stride = size / 2; stride > 0; stride >>= 1) {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride],
                 s_val[pos + stride], dir);
    }
  }

  // ddd == sortDir for the last bitonic merge step
  {
    for (uint stride = arrayLength / 2; stride > 0; stride >>= 1) {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride],
                 s_val[pos + stride], sortDir);
    }
  }

  cg::sync(cta);
  d_DstKey[0] = s_key[threadIdx.x + 0];
  d_DstVal[0] = s_val[threadIdx.x + 0];
  d_DstKey[(SHARED_SIZE_LIMIT / 2)] =
      s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
  d_DstVal[(SHARED_SIZE_LIMIT / 2)] =
      s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

// Helper function (also used by odd-even merge sort)
extern "C" uint factorRadix2(uint *log2L, uint L) {
  if (!L) {
    *log2L = 0;
    return 0;
  } else {
    for (*log2L = 0; (L & 1) == 0; L >>= 1, *log2L++)
      ;

    return L;
  }
}

extern "C" void bitonicSortShared(uint *d_DstKey, uint *d_DstVal,
                                  uint *d_SrcKey, uint *d_SrcVal,
                                  uint batchSize, uint arrayLength,
                                  uint sortDir) {
  // Nothing to sort
  if (arrayLength < 2) {
    return;
  }

  // Only power-of-two array lengths are supported by this implementation
  uint log2L;
  uint factorizationRemainder = factorRadix2(&log2L, arrayLength);
  assert(factorizationRemainder == 1);

  uint blockCount = batchSize * arrayLength / SHARED_SIZE_LIMIT;
  uint threadCount = SHARED_SIZE_LIMIT / 2;

  assert(arrayLength <= SHARED_SIZE_LIMIT);
  assert((batchSize * arrayLength) % SHARED_SIZE_LIMIT == 0);

  bitonicSortSharedKernel<<<blockCount, threadCount>>>(
      d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, arrayLength, sortDir);
  getLastCudaError("bitonicSortSharedKernel<<<>>> failed!\n");
}

////////////////////////////////////////////////////////////////////////////////
// Merge step 3: merge elementary intervals
////////////////////////////////////////////////////////////////////////////////
static inline __host__ __device__ uint iDivUp(uint a, uint b) {
  return ((a % b) == 0) ? (a / b) : (a / b + 1);
}

static inline __host__ __device__ uint getSampleCount(uint dividend) {
  return iDivUp(dividend, SAMPLE_STRIDE);
}

template <uint sortDir>
static inline __device__ void ComparatorExtended(uint &keyA, uint &valA,
                                                 uint &flagA, uint &keyB,
                                                 uint &valB, uint &flagB,
                                                 uint arrowDir) {
  uint t;

  if ((!(flagA || flagB) && ((keyA > keyB) == arrowDir)) ||
      ((arrowDir == sortDir) && (flagA == 1)) ||
      ((arrowDir != sortDir) && (flagB == 1))) {
    t = keyA;
    keyA = keyB;
    keyB = t;
    t = valA;
    valA = valB;
    valB = t;
    t = flagA;
    flagA = flagB;
    flagB = t;
  }
}

template <uint sortDir>
__global__ void bitonicMergeElementaryIntervalsKernel(
    uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *d_SrcVal,
    uint *d_LimitsA, uint *d_LimitsB, uint stride, uint N) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ uint s_key[2 * SAMPLE_STRIDE];
  __shared__ uint s_val[2 * SAMPLE_STRIDE];
  __shared__ uint s_inf[2 * SAMPLE_STRIDE];

  const uint intervalI = blockIdx.x & ((2 * stride) / SAMPLE_STRIDE - 1);
  const uint segmentBase = (blockIdx.x - intervalI) * SAMPLE_STRIDE;
  d_SrcKey += segmentBase;
  d_SrcVal += segmentBase;
  d_DstKey += segmentBase;
  d_DstVal += segmentBase;

  // Set up threadblock-wide parameters
  __shared__ uint startSrcA, lenSrcA, startSrcB, lenSrcB, startDst;

  if (threadIdx.x == 0) {
    uint segmentElementsA = stride;
    uint segmentElementsB = umin(stride, N - segmentBase - stride);
    uint segmentSamplesA = stride / SAMPLE_STRIDE;
    uint segmentSamplesB = getSampleCount(segmentElementsB);
    uint segmentSamples = segmentSamplesA + segmentSamplesB;

    startSrcA = d_LimitsA[blockIdx.x];
    startSrcB = d_LimitsB[blockIdx.x];
    startDst = startSrcA + startSrcB;

    uint endSrcA = (intervalI + 1 < segmentSamples) ? d_LimitsA[blockIdx.x + 1]
                                                    : segmentElementsA;
    uint endSrcB = (intervalI + 1 < segmentSamples) ? d_LimitsB[blockIdx.x + 1]
                                                    : segmentElementsB;
    lenSrcA = endSrcA - startSrcA;
    lenSrcB = endSrcB - startSrcB;
  }

  s_inf[threadIdx.x + 0] = 1;
  s_inf[threadIdx.x + SAMPLE_STRIDE] = 1;

  // Load input data
  cg::sync(cta);

  if (threadIdx.x < lenSrcA) {
    s_key[threadIdx.x] = d_SrcKey[0 + startSrcA + threadIdx.x];
    s_val[threadIdx.x] = d_SrcVal[0 + startSrcA + threadIdx.x];
    s_inf[threadIdx.x] = 0;
  }

  // Prepare for bitonic merge by inversing the ordering
  if (threadIdx.x < lenSrcB) {
    s_key[2 * SAMPLE_STRIDE - 1 - threadIdx.x] =
        d_SrcKey[stride + startSrcB + threadIdx.x];
    s_val[2 * SAMPLE_STRIDE - 1 - threadIdx.x] =
        d_SrcVal[stride + startSrcB + threadIdx.x];
    s_inf[2 * SAMPLE_STRIDE - 1 - threadIdx.x] = 0;
  }

  //"Extended" bitonic merge
  for (uint stride = SAMPLE_STRIDE; stride > 0; stride >>= 1) {
    cg::sync(cta);
    uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
    ComparatorExtended<sortDir>(s_key[pos + 0], s_val[pos + 0], s_inf[pos + 0],
                                s_key[pos + stride], s_val[pos + stride],
                                s_inf[pos + stride], sortDir);
  }

  // Store sorted data
  cg::sync(cta);
  d_DstKey += startDst;
  d_DstVal += startDst;

  if (threadIdx.x < lenSrcA) {
    d_DstKey[threadIdx.x] = s_key[threadIdx.x];
    d_DstVal[threadIdx.x] = s_val[threadIdx.x];
  }

  if (threadIdx.x < lenSrcB) {
    d_DstKey[lenSrcA + threadIdx.x] = s_key[lenSrcA + threadIdx.x];
    d_DstVal[lenSrcA + threadIdx.x] = s_val[lenSrcA + threadIdx.x];
  }
}

extern "C" void bitonicMergeElementaryIntervals(uint *d_DstKey, uint *d_DstVal,
                                                uint *d_SrcKey, uint *d_SrcVal,
                                                uint *d_LimitsA,
                                                uint *d_LimitsB, uint stride,
                                                uint N, uint sortDir) {
  uint lastSegmentElements = N % (2 * stride);

  uint mergePairs = (lastSegmentElements > stride)
                        ? getSampleCount(N)
                        : (N - lastSegmentElements) / SAMPLE_STRIDE;

  if (sortDir) {
    bitonicMergeElementaryIntervalsKernel<1U><<<mergePairs, SAMPLE_STRIDE>>>(
        d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, d_LimitsA, d_LimitsB, stride,
        N);
    getLastCudaError("mergeElementaryIntervalsKernel<1> failed\n");
  } else {
    bitonicMergeElementaryIntervalsKernel<0U><<<mergePairs, SAMPLE_STRIDE>>>(
        d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, d_LimitsA, d_LimitsB, stride,
        N);
    getLastCudaError("mergeElementaryIntervalsKernel<0> failed\n");
  }
}
