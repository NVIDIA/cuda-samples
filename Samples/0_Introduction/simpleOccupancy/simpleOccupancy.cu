#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

const int manualBlockSize = 32;

////////////////////////////////////////////////////////////////////////////////
// Test kernel
//
// This kernel squares each array element. Each thread addresses
// himself with threadIdx and blockIdx, so that it can handle any
// execution configuration, including anything the launch configurator
// API suggests.
////////////////////////////////////////////////////////////////////////////////
__global__ void square(int *array, int arrayCount) {
  extern __shared__ int dynamicSmem[];
  int idx = threadIdx.x + blockIdx.x * blockDim.x;

  if (idx < arrayCount) {
    array[idx] *= array[idx];
  }
}

////////////////////////////////////////////////////////////////////////////////
// Potential occupancy calculator
//
// The potential occupancy is calculated according to the kernel and
// execution configuration the user desires. Occupancy is defined in
// terms of active blocks per multiprocessor, and the user can convert
// it to other metrics.
//
// This wrapper routine computes the occupancy of kernel, and reports
// it in terms of active warps / maximum warps per SM.
////////////////////////////////////////////////////////////////////////////////
static double reportPotentialOccupancy(void *kernel, int blockSize,
                                       size_t dynamicSMem) {
  int device;
  hipDeviceProp_t prop;

  int numBlocks;
  int activeWarps;
  int maxWarps;

  double occupancy;

  checkCudaErrors(hipGetDevice(&device));
  checkCudaErrors(hipGetDeviceProperties(&prop, device));

  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &numBlocks, kernel, blockSize, dynamicSMem));

  activeWarps = numBlocks * blockSize / prop.warpSize;
  maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

  occupancy = (double)activeWarps / maxWarps;

  return occupancy;
}

////////////////////////////////////////////////////////////////////////////////
// Occupancy-based launch configurator
//
// The launch configurator, hipOccupancyMaxPotentialBlockSize and
// hipOccupancyMaxPotentialBlockSizeVariableSMem, suggests a block
// size that achieves the best theoretical occupancy. It also returns
// the minimum number of blocks needed to achieve the occupancy on the
// whole device.
//
// This launch configurator is purely occupancy-based. It doesn't
// translate directly to performance, but the suggestion should
// nevertheless be a good starting point for further optimizations.
//
// This function configures the launch based on the "automatic"
// argument, records the runtime, and reports occupancy and runtime.
////////////////////////////////////////////////////////////////////////////////
static int launchConfig(int *array, int arrayCount, bool automatic) {
  int blockSize;
  int minGridSize;
  int gridSize;
  size_t dynamicSMemUsage = 0;

  hipEvent_t start;
  hipEvent_t end;

  float elapsedTime;

  double potentialOccupancy;

  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&end));

  if (automatic) {
    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
        &minGridSize, &blockSize, (void *)square, dynamicSMemUsage,
        arrayCount));

    std::cout << "Suggested block size: " << blockSize << std::endl
              << "Minimum grid size for maximum occupancy: " << minGridSize
              << std::endl;
  } else {
    // This block size is too small. Given limited number of
    // active blocks per multiprocessor, the number of active
    // threads will be limited, and thus unable to achieve maximum
    // occupancy.
    //
    blockSize = manualBlockSize;
  }

  // Round up
  //
  gridSize = (arrayCount + blockSize - 1) / blockSize;

  // Launch and profile
  //
  checkCudaErrors(hipEventRecord(start));
  square<<<gridSize, blockSize, dynamicSMemUsage>>>(array, arrayCount);
  checkCudaErrors(hipEventRecord(end));

  checkCudaErrors(hipDeviceSynchronize());

  // Calculate occupancy
  //
  potentialOccupancy =
      reportPotentialOccupancy((void *)square, blockSize, dynamicSMemUsage);

  std::cout << "Potential occupancy: " << potentialOccupancy * 100 << "%"
            << std::endl;

  // Report elapsed time
  //
  checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
  std::cout << "Elapsed time: " << elapsedTime << "ms" << std::endl;

  return 0;
}

////////////////////////////////////////////////////////////////////////////////
// The test
//
// The test generates an array and squares it with a CUDA kernel, then
// verifies the result.
////////////////////////////////////////////////////////////////////////////////
static int test(bool automaticLaunchConfig, const int count = 1000000) {
  int *array;
  int *dArray;
  int size = count * sizeof(int);

  array = new int[count];

  for (int i = 0; i < count; i += 1) {
    array[i] = i;
  }

  checkCudaErrors(hipMalloc(&dArray, size));
  checkCudaErrors(hipMemcpy(dArray, array, size, hipMemcpyHostToDevice));

  for (int i = 0; i < count; i += 1) {
    array[i] = 0;
  }

  launchConfig(dArray, count, automaticLaunchConfig);

  checkCudaErrors(hipMemcpy(array, dArray, size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipFree(dArray));

  // Verify the return data
  //
  for (int i = 0; i < count; i += 1) {
    if (array[i] != i * i) {
      std::cout << "element " << i << " expected " << i * i << " actual "
                << array[i] << std::endl;
      return 1;
    }
  }
  delete[] array;

  return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Sample Main
//
// The sample runs the test with manually configured launch and
// automatically configured launch, and reports the occupancy and
// performance.
////////////////////////////////////////////////////////////////////////////////
int main() {
  int status;

  std::cout << "starting Simple Occupancy" << std::endl << std::endl;

  std::cout << "[ Manual configuration with " << manualBlockSize
            << " threads per block ]" << std::endl;

  status = test(false);
  if (status) {
    std::cerr << "Test failed\n" << std::endl;
    return -1;
  }

  std::cout << std::endl;

  std::cout << "[ Automatic, occupancy-based configuration ]" << std::endl;
  status = test(true);
  if (status) {
    std::cerr << "Test failed\n" << std::endl;
    return -1;
  }

  std::cout << std::endl;
  std::cout << "Test PASSED\n" << std::endl;

  return 0;
}
