/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

static const char *sSDKsample = "[simpleVoteIntrinsics]\0";

////////////////////////////////////////////////////////////////////////////////
// Global types and parameters
////////////////////////////////////////////////////////////////////////////////
#define VOTE_DATA_GROUP 4

////////////////////////////////////////////////////////////////////////////////
// CUDA Voting Kernel functions
////////////////////////////////////////////////////////////////////////////////
#include "simpleVote_kernel.cuh"

// Generate the test pattern for Tests 1 and 2
void genVoteTestPattern(unsigned int *VOTE_PATTERN, int size) {
  // For testing VOTE.Any (all of these threads will return 0)
  for (int i = 0; i < size / 4; i++) {
    VOTE_PATTERN[i] = 0x00000000;
  }

  // For testing VOTE.Any (1/2 these threads will return 1)
  for (int i = 2 * size / 8; i < 4 * size / 8; i++) {
    VOTE_PATTERN[i] = (i & 0x01) ? i : 0;
  }

  // For testing VOTE.all (1/2 of these threads will return 0)
  for (int i = 2 * size / 4; i < 3 * size / 4; i++) {
    VOTE_PATTERN[i] = (i & 0x01) ? 0 : i;
  }

  // For testing VOTE.all (all of these threads will return 1)
  for (int i = 3 * size / 4; i < 4 * size / 4; i++) {
    VOTE_PATTERN[i] = 0xffffffff;
  }
}

int checkErrors1(unsigned int *h_result, int start, int end, int warp_size,
                 const char *voteType) {
  int i, sum = 0;

  for (sum = 0, i = start; i < end; i++) {
    sum += h_result[i];
  }

  if (sum > 0) {
    printf("\t<%s>[%d - %d] = ", voteType, start, end - 1);

    for (i = start; i < end; i++) {
      printf("%d", h_result[i]);
    }

    printf("%d values FAILED\n", sum);
  }

  return (sum > 0);
}

int checkErrors2(unsigned int *h_result, int start, int end, int warp_size,
                 const char *voteType) {
  int i, sum = 0;

  for (sum = 0, i = start; i < end; i++) {
    sum += h_result[i];
  }

  if (sum != warp_size) {
    printf("\t<%s>[%d - %d] = ", voteType, start, end - 1);

    for (i = start; i < end; i++) {
      printf("%d", h_result[i]);
    }

    printf(" - FAILED\n");
  }

  return (sum != warp_size);
}

// Verification code for Kernel #1
int checkResultsVoteAnyKernel1(unsigned int *h_result, int size,
                               int warp_size) {
  int error_count = 0;

  error_count += checkErrors1(h_result, 0, VOTE_DATA_GROUP * warp_size / 4,
                              warp_size, "Vote.Any");
  error_count +=
      checkErrors2(h_result, VOTE_DATA_GROUP * warp_size / 4,
                   2 * VOTE_DATA_GROUP * warp_size / 4, warp_size, "Vote.Any");
  error_count +=
      checkErrors2(h_result, 2 * VOTE_DATA_GROUP * warp_size / 4,
                   3 * VOTE_DATA_GROUP * warp_size / 4, warp_size, "Vote.Any");
  error_count +=
      checkErrors2(h_result, 3 * VOTE_DATA_GROUP * warp_size / 4,
                   4 * VOTE_DATA_GROUP * warp_size / 4, warp_size, "Vote.Any");

  printf((error_count == 0) ? "\tOK\n" : "\tERROR\n");
  return error_count;
}

// Verification code for Kernel #2
int checkResultsVoteAllKernel2(unsigned int *h_result, int size,
                               int warp_size) {
  int error_count = 0;

  error_count += checkErrors1(h_result, 0, VOTE_DATA_GROUP * warp_size / 4,
                              warp_size, "Vote.All");
  error_count +=
      checkErrors1(h_result, VOTE_DATA_GROUP * warp_size / 4,
                   2 * VOTE_DATA_GROUP * warp_size / 4, warp_size, "Vote.All");
  error_count +=
      checkErrors1(h_result, 2 * VOTE_DATA_GROUP * warp_size / 4,
                   3 * VOTE_DATA_GROUP * warp_size / 4, warp_size, "Vote.All");
  error_count +=
      checkErrors2(h_result, 3 * VOTE_DATA_GROUP * warp_size / 4,
                   4 * VOTE_DATA_GROUP * warp_size / 4, warp_size, "Vote.All");

  printf((error_count == 0) ? "\tOK\n" : "\tERROR\n");
  return error_count;
}

// Verification code for Kernel #3
int checkResultsVoteAnyKernel3(bool *hinfo, int size) {
  int i, error_count = 0;

  for (i = 0; i < size * 3; i++) {
    switch (i % 3) {
      case 0:

        // First warp should be all zeros.
        if (hinfo[i] != (i >= size * 1)) {
          error_count++;
        }

        break;

      case 1:

        // First warp and half of second should be all zeros.
        if (hinfo[i] != (i >= size * 3 / 2)) {
          error_count++;
        }

        break;

      case 2:

        // First two warps should be all zeros.
        if (hinfo[i] != (i >= size * 2)) {
          error_count++;
        }

        break;
    }
  }

  printf((error_count == 0) ? "\tOK\n" : "\tERROR\n");
  return error_count;
}

int main(int argc, char **argv) {
  unsigned int *h_input, *h_result;
  unsigned int *d_input, *d_result;

  bool *dinfo = NULL, *hinfo = NULL;
  int error_count[3] = {0, 0, 0};

  hipDeviceProp_t deviceProp;
  int devID, warp_size = 32;

  printf("%s\n", sSDKsample);

  // This will pick the best possible CUDA capable device
  devID = findCudaDevice(argc, (const char **)argv);

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

  // Statistics about the GPU device
  printf(
      "> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
      deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

  h_input = (unsigned int *)malloc(VOTE_DATA_GROUP * warp_size *
                                   sizeof(unsigned int));
  h_result = (unsigned int *)malloc(VOTE_DATA_GROUP * warp_size *
                                    sizeof(unsigned int));
  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&d_input),
                 VOTE_DATA_GROUP * warp_size * sizeof(unsigned int)));
  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&d_result),
                 VOTE_DATA_GROUP * warp_size * sizeof(unsigned int)));
  genVoteTestPattern(h_input, VOTE_DATA_GROUP * warp_size);
  checkCudaErrors(hipMemcpy(d_input, h_input,
                             VOTE_DATA_GROUP * warp_size * sizeof(unsigned int),
                             hipMemcpyHostToDevice));

  // Start of Vote Any Test Kernel #1
  printf("[VOTE Kernel Test 1/3]\n");
  printf("\tRunning <<Vote.Any>> kernel1 ...\n");
  {
    checkCudaErrors(hipDeviceSynchronize());
    dim3 gridBlock(1, 1);
    dim3 threadBlock(VOTE_DATA_GROUP * warp_size, 1);
    VoteAnyKernel1<<<gridBlock, threadBlock>>>(d_input, d_result,
                                               VOTE_DATA_GROUP * warp_size);
    getLastCudaError("VoteAnyKernel() execution failed\n");
    checkCudaErrors(hipDeviceSynchronize());
  }
  checkCudaErrors(hipMemcpy(h_result, d_result,
                             VOTE_DATA_GROUP * warp_size * sizeof(unsigned int),
                             hipMemcpyDeviceToHost));
  error_count[0] += checkResultsVoteAnyKernel1(
      h_result, VOTE_DATA_GROUP * warp_size, warp_size);

  // Start of Vote All Test Kernel #2
  printf("\n[VOTE Kernel Test 2/3]\n");
  printf("\tRunning <<Vote.All>> kernel2 ...\n");
  {
    checkCudaErrors(hipDeviceSynchronize());
    dim3 gridBlock(1, 1);
    dim3 threadBlock(VOTE_DATA_GROUP * warp_size, 1);
    VoteAllKernel2<<<gridBlock, threadBlock>>>(d_input, d_result,
                                               VOTE_DATA_GROUP * warp_size);
    getLastCudaError("VoteAllKernel() execution failed\n");
    checkCudaErrors(hipDeviceSynchronize());
  }
  checkCudaErrors(hipMemcpy(h_result, d_result,
                             VOTE_DATA_GROUP * warp_size * sizeof(unsigned int),
                             hipMemcpyDeviceToHost));
  error_count[1] += checkResultsVoteAllKernel2(
      h_result, VOTE_DATA_GROUP * warp_size, warp_size);

  // Second Vote Kernel Test #3 (both Any/All)
  hinfo = reinterpret_cast<bool *>(calloc(warp_size * 3 * 3, sizeof(bool)));
  hipMalloc(reinterpret_cast<void **>(&dinfo),
             warp_size * 3 * 3 * sizeof(bool));
  hipMemcpy(dinfo, hinfo, warp_size * 3 * 3 * sizeof(bool),
             hipMemcpyHostToDevice);

  printf("\n[VOTE Kernel Test 3/3]\n");
  printf("\tRunning <<Vote.Any>> kernel3 ...\n");
  {
    checkCudaErrors(hipDeviceSynchronize());
    VoteAnyKernel3<<<1, warp_size * 3>>>(dinfo, warp_size);
    checkCudaErrors(hipDeviceSynchronize());
  }

  hipMemcpy(hinfo, dinfo, warp_size * 3 * 3 * sizeof(bool),
             hipMemcpyDeviceToHost);

  error_count[2] = checkResultsVoteAnyKernel3(hinfo, warp_size * 3);

  // Now free these resources for Test #1,2
  checkCudaErrors(hipFree(d_input));
  checkCudaErrors(hipFree(d_result));
  free(h_input);
  free(h_result);

  // Free resources from Test #3
  free(hinfo);
  hipFree(dinfo);

  printf("\tShutting down...\n");

  return (error_count[0] == 0 && error_count[1] == 0 && error_count[2] == 0)
             ? EXIT_SUCCESS
             : EXIT_FAILURE;
}
