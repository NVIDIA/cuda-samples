#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#define THREAD_N 256
#define N 1024
#define DIV_UP(a, b) (((a) + (b) - 1) / (b))

// Includes, system
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>
#include <hip/hip_vector_types.h>
#include "cppOverload_kernel.cuh"

const char *sampleName = "C++ Function Overloading";

#define OUTPUT_ATTR(attr)                                         \
  printf("Shared Size:   %d\n", (int)attr.sharedSizeBytes);       \
  printf("Constant Size: %d\n", (int)attr.constSizeBytes);        \
  printf("Local Size:    %d\n", (int)attr.localSizeBytes);        \
  printf("Max Threads Per Block: %d\n", attr.maxThreadsPerBlock); \
  printf("Number of Registers: %d\n", attr.numRegs);              \
  printf("PTX Version: %d\n", attr.ptxVersion);                   \
  printf("Binary Version: %d\n", attr.binaryVersion);

bool check_func1(int *hInput, int *hOutput, int a) {
  for (int i = 0; i < N; ++i) {
    int cpuRes = hInput[i] * a + i;

    if (hOutput[i] != cpuRes) {
      return false;
    }
  }

  return true;
}

bool check_func2(int2 *hInput, int *hOutput, int a) {
  for (int i = 0; i < N; i++) {
    int cpuRes = (hInput[i].x + hInput[i].y) * a + i;

    if (hOutput[i] != cpuRes) {
      return false;
    }
  }

  return true;
}

bool check_func3(int *hInput1, int *hInput2, int *hOutput, int a) {
  for (int i = 0; i < N; i++) {
    if (hOutput[i] != (hInput1[i] + hInput2[i]) * a + i) {
      return false;
    }
  }

  return true;
}

int main(int argc, const char *argv[]) {
  int *hInput = NULL;
  int *hOutput = NULL;
  int *dInput = NULL;
  int *dOutput = NULL;

  printf("%s starting...\n", sampleName);

  int deviceCount;
  checkCudaErrors(hipGetDeviceCount(&deviceCount));
  printf("Device Count: %d\n", deviceCount);

  int deviceID = findCudaDevice(argc, argv);
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDeviceProperties(&prop, deviceID));
  if (prop.major < 2) {
    printf(
        "ERROR: cppOverload requires GPU devices with compute SM 2.0 or "
        "higher.\n");
    printf("Current GPU device has compute SM%d.%d, Exiting...", prop.major,
           prop.minor);
    exit(EXIT_WAIVED);
  }

  checkCudaErrors(hipSetDevice(deviceID));

  // Allocate device memory
  checkCudaErrors(hipMalloc(&dInput, sizeof(int) * N * 2));
  checkCudaErrors(hipMalloc(&dOutput, sizeof(int) * N));

  // Allocate host memory
  checkCudaErrors(hipHostMalloc(&hInput, sizeof(int) * N * 2));
  checkCudaErrors(hipHostMalloc(&hOutput, sizeof(int) * N));

  for (int i = 0; i < N * 2; i++) {
    hInput[i] = i;
  }

  // Copy data from host to device
  checkCudaErrors(
      hipMemcpy(dInput, hInput, sizeof(int) * N * 2, hipMemcpyHostToDevice));

  // Test C++ overloading
  bool testResult = true;
  bool funcResult = true;
  int a = 1;

  void (*func1)(const int *, int *, int);
  void (*func2)(const int2 *, int *, int);
  void (*func3)(const int *, const int *, int *, int);
  struct hipFuncAttributes attr;

  // overload function 1
  func1 = simple_kernel;
  memset(&attr, 0, sizeof(attr));
  checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*func1), hipFuncCachePreferShared));
  checkCudaErrors(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(*func1)));
  OUTPUT_ATTR(attr);
  (*func1)<<<DIV_UP(N, THREAD_N), THREAD_N>>>(dInput, dOutput, a);
  checkCudaErrors(
      hipMemcpy(hOutput, dOutput, sizeof(int) * N, hipMemcpyDeviceToHost));
  funcResult = check_func1(hInput, hOutput, a);
  printf("simple_kernel(const int *pIn, int *pOut, int a) %s\n\n",
         funcResult ? "PASSED" : "FAILED");
  testResult &= funcResult;

  // overload function 2
  func2 = simple_kernel;
  memset(&attr, 0, sizeof(attr));
  checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*func2), hipFuncCachePreferShared));
  checkCudaErrors(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(*func2)));
  OUTPUT_ATTR(attr);
  (*func2)<<<DIV_UP(N, THREAD_N), THREAD_N>>>((int2 *)dInput, dOutput, a);
  checkCudaErrors(
      hipMemcpy(hOutput, dOutput, sizeof(int) * N, hipMemcpyDeviceToHost));
  funcResult = check_func2(reinterpret_cast<int2 *>(hInput), hOutput, a);
  printf("simple_kernel(const int2 *pIn, int *pOut, int a) %s\n\n",
         funcResult ? "PASSED" : "FAILED");
  testResult &= funcResult;

  // overload function 3
  func3 = simple_kernel;
  memset(&attr, 0, sizeof(attr));
  checkCudaErrors(hipFuncSetCacheConfig(reinterpret_cast<const void*>(*func3), hipFuncCachePreferShared));
  checkCudaErrors(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(*func3)));
  OUTPUT_ATTR(attr);
  (*func3)<<<DIV_UP(N, THREAD_N), THREAD_N>>>(dInput, dInput + N, dOutput, a);
  checkCudaErrors(
      hipMemcpy(hOutput, dOutput, sizeof(int) * N, hipMemcpyDeviceToHost));
  funcResult = check_func3(&hInput[0], &hInput[N], hOutput, a);
  printf(
      "simple_kernel(const int *pIn1, const int *pIn2, int *pOut, int a) "
      "%s\n\n",
      funcResult ? "PASSED" : "FAILED");
  testResult &= funcResult;

  checkCudaErrors(hipFree(dInput));
  checkCudaErrors(hipFree(dOutput));
  checkCudaErrors(hipHostFree(hOutput));
  checkCudaErrors(hipHostFree(hInput));

  checkCudaErrors(hipDeviceSynchronize());

  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
