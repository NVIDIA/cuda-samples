/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _SIMPLETEXTURE_KERNEL_H_
#define _SIMPLETEXTURE_KERNEL_H_
#include <hip/hip_runtime.h>

////////////////////////////////////////////////////////////////////////////////
//! Transform an image using texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void transformKernel(float *g_odata, int width,
                                           int height, float theta,
                                           hipTextureObject_t tex) {
  // calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = (float)x - (float)width / 2;
  float v = (float)y - (float)height / 2;
  float tu = u * cosf(theta) - v * sinf(theta);
  float tv = v * cosf(theta) + u * sinf(theta);

  tu /= (float)width;
  tv /= (float)height;

  // read from texture and write to global memory
  g_odata[y * width + x] = tex2D<float>(tex, tu + 0.5f, tv + 0.5f);
}

#endif  // #ifndef _SIMPLETEXTURE_KERNEL_H_
