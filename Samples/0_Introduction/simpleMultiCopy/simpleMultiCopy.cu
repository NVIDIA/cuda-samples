#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Quadro and Tesla GPUs with compute capability >= 2.0 can overlap two
 * memcopies with kernel execution. This sample illustrates the usage of CUDA
 * streams to achieve overlapping of kernel execution with copying data to and
 * from the device.
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
*/

const char *sSDKname = "simpleMultiCopy";

// includes, system
#include <stdio.h>

// include CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper for shared that are common to CUDA Samples

// includes, kernels
// Declare the CUDA kernels here and main() code that is needed to launch
// Compute workload on the system
__global__ void incKernel(int *g_out, int *g_in, int N, int inner_reps) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    for (int i = 0; i < inner_reps; ++i) {
      g_out[idx] = g_in[idx] + 1;
    }
  }
}

#define STREAM_COUNT 4

// Uncomment to simulate data source/sink IO times
//#define SIMULATE_IO

int *h_data_source;
int *h_data_sink;

int *h_data_in[STREAM_COUNT];
int *d_data_in[STREAM_COUNT];

int *h_data_out[STREAM_COUNT];
int *d_data_out[STREAM_COUNT];

hipEvent_t cycleDone[STREAM_COUNT];
hipStream_t stream[STREAM_COUNT];

hipEvent_t start, stop;

int N = 1 << 22;
int nreps = 10;  // number of times each experiment is repeated
int inner_reps = 5;

int memsize;

dim3 block(512);
dim3 grid;

int thread_blocks;

float processWithStreams(int streams_used);
void init();
bool test();

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]) {
  int cuda_device = 0;
  float scale_factor;
  hipDeviceProp_t deviceProp;

  printf("[%s] - Starting...\n", sSDKname);

  if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
    cuda_device = getCmdLineArgumentInt(argc, (const char **)argv, "device=");

    if (cuda_device < 0) {
      printf("Invalid command line parameters\n");
      exit(EXIT_FAILURE);
    } else {
      printf("cuda_device = %d\n", cuda_device);
      cuda_device = gpuDeviceInit(cuda_device);

      if (cuda_device < 0) {
        printf("No CUDA Capable devices found, exiting...\n");
        exit(EXIT_SUCCESS);
      }
    }
  } else {
    // Otherwise pick the device with the highest Gflops/s
    cuda_device = gpuGetMaxGflopsDeviceId();
    checkCudaErrors(hipSetDevice(cuda_device));
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
    printf("> Using CUDA device [%d]: %s\n", cuda_device, deviceProp.name);
  }

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));
  printf("[%s] has %d MP(s) x %d (Cores/MP) = %d (Cores)\n", deviceProp.name,
         deviceProp.multiProcessorCount,
         _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
         _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
             deviceProp.multiProcessorCount);

  // Anything that is less than 32 Cores will have scaled down workload
  scale_factor =
      max((32.0f / (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
                    (float)deviceProp.multiProcessorCount)),
          1.0f);
  N = (int)((float)N / scale_factor);

  printf("> Device name: %s\n", deviceProp.name);
  printf("> CUDA Capability %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);
  printf("> scale_factor = %.2f\n", 1.0f / scale_factor);
  printf("> array_size   = %d\n\n", N);

  memsize = N * sizeof(int);

  thread_blocks = N / block.x;

  grid.x = thread_blocks % 65535;
  grid.y = (thread_blocks / 65535 + 1);

  // Allocate resources

  h_data_source = (int *)malloc(memsize);
  h_data_sink = (int *)malloc(memsize);

  for (int i = 0; i < STREAM_COUNT; ++i) {
    checkCudaErrors(
        hipHostAlloc(&h_data_in[i], memsize, hipHostMallocDefault));
    checkCudaErrors(hipMalloc(&d_data_in[i], memsize));
    checkCudaErrors(hipMemset(d_data_in[i], 0, memsize));

    checkCudaErrors(
        hipHostAlloc(&h_data_out[i], memsize, hipHostMallocDefault));
    checkCudaErrors(hipMalloc(&d_data_out[i], memsize));

    checkCudaErrors(hipStreamCreate(&stream[i]));
    checkCudaErrors(hipEventCreate(&cycleDone[i]));

    hipEventRecord(cycleDone[i], stream[i]);
  }

  hipEventCreate(&start);
  hipEventCreate(&stop);

  init();

  // Kernel warmup
  incKernel<<<grid, block>>>(d_data_out[0], d_data_in[0], N, inner_reps);

  // Time copies and kernel
  hipEventRecord(start, 0);
  checkCudaErrors(hipMemcpyAsync(d_data_in[0], h_data_in[0], memsize,
                                  hipMemcpyHostToDevice, 0));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float memcpy_h2d_time;
  hipEventElapsedTime(&memcpy_h2d_time, start, stop);

  hipEventRecord(start, 0);
  checkCudaErrors(hipMemcpyAsync(h_data_out[0], d_data_out[0], memsize,
                                  hipMemcpyDeviceToHost, 0));
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float memcpy_d2h_time;
  hipEventElapsedTime(&memcpy_d2h_time, start, stop);

  hipEventRecord(start, 0);
  incKernel<<<grid, block, 0, 0>>>(d_data_out[0], d_data_in[0], N, inner_reps);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  float kernel_time;
  hipEventElapsedTime(&kernel_time, start, stop);

  printf("\n");
  printf("Relevant properties of this CUDA device\n");
  printf(
      "(%s) Can overlap one CPU<>GPU data transfer with GPU kernel execution "
      "(device property \"deviceOverlap\")\n",
      deviceProp.deviceOverlap ? "X" : " ");
  // printf("(%s) Can execute several GPU kernels simultaneously (compute
  // capability >= 2.0)\n", deviceProp.major >= 2 ? "X": " ");
  printf(
      "(%s) Can overlap two CPU<>GPU data transfers with GPU kernel execution\n"
      "    (Compute Capability >= 2.0 AND (Tesla product OR Quadro "
      "4000/5000/6000/K5000)\n",
      (deviceProp.major >= 2 && deviceProp.asyncEngineCount > 1) ? "X" : " ");

  printf("\n");
  printf("Measured timings (throughput):\n");
  printf(" Memcpy host to device\t: %f ms (%f GB/s)\n", memcpy_h2d_time,
         (memsize * 1e-6) / memcpy_h2d_time);
  printf(" Memcpy device to host\t: %f ms (%f GB/s)\n", memcpy_d2h_time,
         (memsize * 1e-6) / memcpy_d2h_time);
  printf(" Kernel\t\t\t: %f ms (%f GB/s)\n", kernel_time,
         (inner_reps * memsize * 2e-6) / kernel_time);

  printf("\n");
  printf(
      "Theoretical limits for speedup gained from overlapped data "
      "transfers:\n");
  printf("No overlap at all (transfer-kernel-transfer): %f ms \n",
         memcpy_h2d_time + memcpy_d2h_time + kernel_time);
  printf("Compute can overlap with one transfer: %f ms\n",
         max((memcpy_h2d_time + memcpy_d2h_time), kernel_time));
  printf("Compute can overlap with both data transfers: %f ms\n",
         max(max(memcpy_h2d_time, memcpy_d2h_time), kernel_time));

  // Process pipelined work
  float serial_time = processWithStreams(1);
  float overlap_time = processWithStreams(STREAM_COUNT);

  printf("\nAverage measured timings over %d repetitions:\n", nreps);
  printf(" Avg. time when execution fully serialized\t: %f ms\n",
         serial_time / nreps);
  printf(" Avg. time when overlapped using %d streams\t: %f ms\n", STREAM_COUNT,
         overlap_time / nreps);
  printf(" Avg. speedup gained (serialized - overlapped)\t: %f ms\n",
         (serial_time - overlap_time) / nreps);

  printf("\nMeasured throughput:\n");
  printf(" Fully serialized execution\t\t: %f GB/s\n",
         (nreps * (memsize * 2e-6)) / serial_time);
  printf(" Overlapped using %d streams\t\t: %f GB/s\n", STREAM_COUNT,
         (nreps * (memsize * 2e-6)) / overlap_time);

  // Verify the results, we will use the results for final output
  bool bResults = test();

  // Free resources

  free(h_data_source);
  free(h_data_sink);

  for (int i = 0; i < STREAM_COUNT; ++i) {
    hipHostFree(h_data_in[i]);
    hipFree(d_data_in[i]);

    hipHostFree(h_data_out[i]);
    hipFree(d_data_out[i]);

    hipStreamDestroy(stream[i]);
    hipEventDestroy(cycleDone[i]);
  }

  hipEventDestroy(start);
  hipEventDestroy(stop);

  // Test result
  exit(bResults ? EXIT_SUCCESS : EXIT_FAILURE);
}

float processWithStreams(int streams_used) {
  int current_stream = 0;

  float time;

  // Do processing in a loop
  //
  // Note: All memory commands are processed in the order  they are issued,
  // independent of the stream they are enqueued in. Hence the pattern by
  // which the copy and kernel commands are enqueued in the stream
  // has an influence on the achieved overlap.

  hipEventRecord(start, 0);

  for (int i = 0; i < nreps; ++i) {
    int next_stream = (current_stream + 1) % streams_used;

#ifdef SIMULATE_IO
    // Store the result
    memcpy(h_data_sink, h_data_out[current_stream], memsize);

    // Read new input
    memcpy(h_data_in[next_stream], h_data_source, memsize);
#endif

    // Ensure that processing and copying of the last cycle has finished
    hipEventSynchronize(cycleDone[next_stream]);

    // Process current frame
    incKernel<<<grid, block, 0, stream[current_stream]>>>(
        d_data_out[current_stream], d_data_in[current_stream], N, inner_reps);

    // Upload next frame
    checkCudaErrors(
        hipMemcpyAsync(d_data_in[next_stream], h_data_in[next_stream], memsize,
                        hipMemcpyHostToDevice, stream[next_stream]));

    // Download current frame
    checkCudaErrors(hipMemcpyAsync(
        h_data_out[current_stream], d_data_out[current_stream], memsize,
        hipMemcpyDeviceToHost, stream[current_stream]));

    checkCudaErrors(
        hipEventRecord(cycleDone[current_stream], stream[current_stream]));

    current_stream = next_stream;
  }

  hipEventRecord(stop, 0);

  hipDeviceSynchronize();

  hipEventElapsedTime(&time, start, stop);

  return time;
}

void init() {
  for (int i = 0; i < N; ++i) {
    h_data_source[i] = 0;
  }

  for (int i = 0; i < STREAM_COUNT; ++i) {
    memcpy(h_data_in[i], h_data_source, memsize);
  }
}

bool test() {
  bool passed = true;

  for (int j = 0; j < STREAM_COUNT; ++j) {
    for (int i = 0; i < N; ++i) {
      passed &= (h_data_out[j][i] == 1);
    }
  }

  return passed;
}
