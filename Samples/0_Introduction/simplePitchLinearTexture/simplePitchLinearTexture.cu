#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* pitchLinearTexture
*
* This example demonstrates how to use textures bound to pitch linear memory.
* It performs a shift of matrix elements using wrap addressing mode (aka
* periodic boundary conditions) on two arrays, a pitch linear and a CUDA array,
* in order to highlight the differences in using each.
*
* Textures binding to pitch linear memory is a new feature in CUDA 2.2,
* and allows use of texture features such as wrap addressing mode and
* filtering which are not possible with textures bound to regular linear memory
*/

// includes, system
#include <stdio.h>

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// Includes CUDA
#include <hip/hip_runtime.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check

#define NUM_REPS 100  // number of repetitions performed
#define TILE_DIM 16   // tile/block size

const char *sSDKsample = "simplePitchLinearTexture";

// Auto-Verification Code
bool bTestResult = true;

////////////////////////////////////////////////////////////////////////////////
// NB: (1) The second argument "pitch" is in elements, not bytes
//     (2) normalized coordinates are used (required for wrap address mode)
////////////////////////////////////////////////////////////////////////////////
//! Shifts matrix elements using pitch linear array
//! @param odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void shiftPitchLinear(float *odata, int pitch, int width, int height,
                                 int shiftX, int shiftY,
                                 hipTextureObject_t texRefPL) {
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  int yid = blockIdx.y * blockDim.y + threadIdx.y;

  odata[yid * pitch + xid] = tex2D<float>(
      texRefPL, (xid + shiftX) / (float)width, (yid + shiftY) / (float)height);
}

////////////////////////////////////////////////////////////////////////////////
//! Shifts matrix elements using regular array
//! @param odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void shiftArray(float *odata, int pitch, int width, int height,
                           int shiftX, int shiftY,
                           hipTextureObject_t texRefArray) {
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  int yid = blockIdx.y * blockDim.y + threadIdx.y;

  odata[yid * pitch + xid] =
      tex2D<float>(texRefArray, (xid + shiftX) / (float)width,
                   (yid + shiftY) / (float)height);
}

////////////////////////////////////////////////////////////////////////////////
// Declaration, forward
void runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("%s starting...\n\n", sSDKsample);

  runTest(argc, argv);

  printf("%s completed, returned %s\n", sSDKsample,
         bTestResult ? "OK" : "ERROR!");
  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  // Set array size
  const int nx = 2048;
  const int ny = 2048;

  // Setup shifts applied to x and y data
  const int x_shift = 5;
  const int y_shift = 7;

  if ((nx % TILE_DIM != 0) || (ny % TILE_DIM != 0)) {
    printf("nx and ny must be multiples of TILE_DIM\n");
    exit(EXIT_FAILURE);
  }

  // Setup execution configuration parameters
  dim3 dimGrid(nx / TILE_DIM, ny / TILE_DIM), dimBlock(TILE_DIM, TILE_DIM);

  // This will pick the best possible CUDA capable device
  int devID = findCudaDevice(argc, (const char **)argv);

  // CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Host allocation and initialization
  float *h_idata = (float *)malloc(sizeof(float) * nx * ny);
  float *h_odata = (float *)malloc(sizeof(float) * nx * ny);
  float *gold = (float *)malloc(sizeof(float) * nx * ny);

  for (int i = 0; i < nx * ny; ++i) {
    h_idata[i] = (float)i;
  }

  // Device memory allocation
  // Pitch linear input data
  float *d_idataPL;
  size_t d_pitchBytes;

  checkCudaErrors(hipMallocPitch((void **)&d_idataPL, &d_pitchBytes,
                                  nx * sizeof(float), ny));

  // Array input data
  hipArray *d_idataArray;
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  checkCudaErrors(hipMallocArray(&d_idataArray, &channelDesc, nx, ny));

  // Pitch linear output data
  float *d_odata;
  checkCudaErrors(hipMallocPitch((void **)&d_odata, &d_pitchBytes,
                                  nx * sizeof(float), ny));

  // Copy host data to device
  // Pitch linear
  size_t h_pitchBytes = nx * sizeof(float);

  checkCudaErrors(hipMemcpy2D(d_idataPL, d_pitchBytes, h_idata, h_pitchBytes,
                               nx * sizeof(float), ny, hipMemcpyHostToDevice));

  // Array
  checkCudaErrors(hipMemcpyToArray(d_idataArray, 0, 0, h_idata,
                                    nx * ny * sizeof(float),
                                    hipMemcpyHostToDevice));

  hipTextureObject_t texRefPL;
  hipTextureObject_t texRefArray;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypePitch2D;
  texRes.res.pitch2D.devPtr = d_idataPL;
  texRes.res.pitch2D.desc = channelDesc;
  texRes.res.pitch2D.width = nx;
  texRes.res.pitch2D.height = ny;
  texRes.res.pitch2D.pitchInBytes = h_pitchBytes;
  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&texRefPL, &texRes, &texDescr, NULL));
  memset(&texRes, 0, sizeof(hipResourceDesc));
  memset(&texDescr, 0, sizeof(hipTextureDesc));
  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_idataArray;
  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;
  checkCudaErrors(
      hipCreateTextureObject(&texRefArray, &texRes, &texDescr, NULL));

  // Reference calculation
  for (int j = 0; j < ny; ++j) {
    int jshift = (j + y_shift) % ny;

    for (int i = 0; i < nx; ++i) {
      int ishift = (i + x_shift) % nx;
      gold[j * nx + i] = h_idata[jshift * nx + ishift];
    }
  }

  // Run ShiftPitchLinear kernel
  checkCudaErrors(
      hipMemset2D(d_odata, d_pitchBytes, 0, nx * sizeof(float), ny));

  checkCudaErrors(hipEventRecord(start, 0));

  for (int i = 0; i < NUM_REPS; ++i) {
    shiftPitchLinear<<<dimGrid, dimBlock>>>(d_odata,
                                            (int)(d_pitchBytes / sizeof(float)),
                                            nx, ny, x_shift, y_shift, texRefPL);
  }

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float timePL;
  checkCudaErrors(hipEventElapsedTime(&timePL, start, stop));

  // Check results
  checkCudaErrors(hipMemcpy2D(h_odata, h_pitchBytes, d_odata, d_pitchBytes,
                               nx * sizeof(float), ny, hipMemcpyDeviceToHost));

  bool res = compareData(gold, h_odata, nx * ny, 0.0f, 0.15f);

  bTestResult = true;

  if (res == false) {
    printf("*** shiftPitchLinear failed ***\n");
    bTestResult = false;
  }

  // Run ShiftArray kernel
  checkCudaErrors(
      hipMemset2D(d_odata, d_pitchBytes, 0, nx * sizeof(float), ny));
  checkCudaErrors(hipEventRecord(start, 0));

  for (int i = 0; i < NUM_REPS; ++i) {
    shiftArray<<<dimGrid, dimBlock>>>(d_odata,
                                      (int)(d_pitchBytes / sizeof(float)), nx,
                                      ny, x_shift, y_shift, texRefArray);
  }

  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float timeArray;
  checkCudaErrors(hipEventElapsedTime(&timeArray, start, stop));

  // Check results
  checkCudaErrors(hipMemcpy2D(h_odata, h_pitchBytes, d_odata, d_pitchBytes,
                               nx * sizeof(float), ny, hipMemcpyDeviceToHost));
  res = compareData(gold, h_odata, nx * ny, 0.0f, 0.15f);

  if (res == false) {
    printf("*** shiftArray failed ***\n");
    bTestResult = false;
  }

  float bandwidthPL =
      2.f * 1000.f * nx * ny * sizeof(float) / (1.e+9f) / (timePL / NUM_REPS);
  float bandwidthArray = 2.f * 1000.f * nx * ny * sizeof(float) / (1.e+9f) /
                         (timeArray / NUM_REPS);

  printf("\nBandwidth (GB/s) for pitch linear: %.2e; for array: %.2e\n",
         bandwidthPL, bandwidthArray);

  float fetchRatePL = nx * ny / 1.e+6f / (timePL / (1000.0f * NUM_REPS));
  float fetchRateArray = nx * ny / 1.e+6f / (timeArray / (1000.0f * NUM_REPS));

  printf(
      "\nTexture fetch rate (Mpix/s) for pitch linear: "
      "%.2e; for array: %.2e\n\n",
      fetchRatePL, fetchRateArray);

  // Cleanup
  free(h_idata);
  free(h_odata);
  free(gold);

  checkCudaErrors(hipDestroyTextureObject(texRefPL));
  checkCudaErrors(hipDestroyTextureObject(texRefArray));
  checkCudaErrors(hipFree(d_idataPL));
  checkCudaErrors(hipFreeArray(d_idataArray));
  checkCudaErrors(hipFree(d_odata));

  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
}
