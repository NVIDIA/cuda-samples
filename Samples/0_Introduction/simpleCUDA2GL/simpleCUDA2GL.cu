#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Utilities and system includes

#include <hip/hip_runtime_api.h>

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b) { return max(a, min(b, x)); }

__device__ int clamp(int x, int a, int b) { return max(a, min(b, x)); }

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b) {
  r = clamp(r, 0.0f, 255.0f);
  g = clamp(g, 0.0f, 255.0f);
  b = clamp(b, 0.0f, 255.0f);
  return (int(b) << 16) | (int(g) << 8) | int(r);
}

__global__ void cudaProcess(unsigned int *g_odata, int imgw) {
  extern __shared__ uchar4 sdata[];

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bw = blockDim.x;
  int bh = blockDim.y;
  int x = blockIdx.x * bw + tx;
  int y = blockIdx.y * bh + ty;

  uchar4 c4 = make_uchar4((x & 0x20) ? 100 : 0, 0, (y & 0x20) ? 100 : 0, 0);
  g_odata[y * imgw + x] = rgbToInt(c4.z, c4.y, c4.x);
}

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                                   unsigned int *g_odata, int imgw) {
  cudaProcess<<<grid, block, sbytes>>>(g_odata, imgw);
}
