
#include <hip/hip_runtime.h>
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This example shows how to use the clock function to measure the performance
 * of block of threads of a kernel accurately. Blocks are executed in parallel
 * and out of order. Since there's no synchronization mechanism between blocks,
 * we measure the clock once for each block. The clock samples are written to
 * device memory.
 */

// This kernel computes a standard parallel reduction and evaluates the
// time it takes to do that for each block. The timing results are stored
// in device memory.

extern "C" __global__ void timedReduction(const float *input, float *output,
                                          clock_t *timer) {
  // __shared__ float shared[2 * blockDim.x];
  extern __shared__ float shared[];

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;

  if (tid == 0) timer[bid] = clock();

  // Copy input.
  shared[tid] = input[tid];
  shared[tid + blockDim.x] = input[tid + blockDim.x];

  // Perform reduction to find minimum.
  for (int d = blockDim.x; d > 0; d /= 2) {
    __syncthreads();

    if (tid < d) {
      float f0 = shared[tid];
      float f1 = shared[tid + d];

      if (f1 < f0) {
        shared[tid] = f1;
      }
    }
  }

  // Write result.
  if (tid == 0) output[bid] = shared[0];

  __syncthreads();

  if (tid == 0) timer[bid + gridDim.x] = clock();
}
