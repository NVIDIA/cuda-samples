#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

__global__ void testKernel(int val) {
  printf("[%d, %d]:\t\tValue is:%d\n", blockIdx.y * gridDim.x + blockIdx.x,
         threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x +
             threadIdx.x,
         val);
}

int main(int argc, char **argv) {
  int devID;
  hipDeviceProp_t props;

  // This will pick the best possible CUDA capable device
  devID = findCudaDevice(argc, (const char **)argv);

  // Get GPU information
  checkCudaErrors(hipGetDevice(&devID));
  checkCudaErrors(hipGetDeviceProperties(&props, devID));
  printf("Device %d: \"%s\" with Compute %d.%d capability\n", devID, props.name,
         props.major, props.minor);

  printf("printf() is called. Output:\n\n");

  // Kernel configuration, where a two-dimensional grid and
  // three-dimensional blocks are configured.
  dim3 dimGrid(2, 2);
  dim3 dimBlock(2, 2, 2);
  testKernel<<<dimGrid, dimBlock>>>(10);
  hipDeviceSynchronize();

  return EXIT_SUCCESS;
}
