#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "hip/hip_fp16.h"
#include "hip/hip_runtime_api.h"

#include <cstdio>
#include <cstdlib>
#include <ctime>

#define NUM_OF_BLOCKS 128
#define NUM_OF_THREADS 128

__forceinline__ __device__ void reduceInShared_intrinsics(half2 *const v) {
  if (threadIdx.x < 64)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 64]);
  __syncthreads();
  if (threadIdx.x < 32)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 32]);
  __syncthreads();
  if (threadIdx.x < 16)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 16]);
  __syncthreads();
  if (threadIdx.x < 8)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 8]);
  __syncthreads();
  if (threadIdx.x < 4)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 4]);
  __syncthreads();
  if (threadIdx.x < 2)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 2]);
  __syncthreads();
  if (threadIdx.x < 1)
    v[threadIdx.x] = __hadd2(v[threadIdx.x], v[threadIdx.x + 1]);
  __syncthreads();
}

__forceinline__ __device__ void reduceInShared_native(half2 *const v) {
  if (threadIdx.x < 64) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 64];
  __syncthreads();
  if (threadIdx.x < 32) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 32];
  __syncthreads();
  if (threadIdx.x < 16) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 16];
  __syncthreads();
  if (threadIdx.x < 8) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 8];
  __syncthreads();
  if (threadIdx.x < 4) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 4];
  __syncthreads();
  if (threadIdx.x < 2) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 2];
  __syncthreads();
  if (threadIdx.x < 1) v[threadIdx.x] = v[threadIdx.x] + v[threadIdx.x + 1];
  __syncthreads();
}

__global__ void scalarProductKernel_intrinsics(half2 const *const a,
                                               half2 const *const b,
                                               float *const results,
                                               size_t const size) {
  const int stride = gridDim.x * blockDim.x;
  __shared__ half2 shArray[NUM_OF_THREADS];

  shArray[threadIdx.x] = __float2half2_rn(0.f);
  half2 value = __float2half2_rn(0.f);

  for (int i = threadIdx.x + blockDim.x + blockIdx.x; i < size; i += stride) {
    value = __hfma2(a[i], b[i], value);
  }

  shArray[threadIdx.x] = value;
  __syncthreads();
  reduceInShared_intrinsics(shArray);

  if (threadIdx.x == 0) {
    half2 result = shArray[0];
    float f_result = __low2float(result) + __high2float(result);
    results[blockIdx.x] = f_result;
  }
}

__global__ void scalarProductKernel_native(half2 const *const a,
                                           half2 const *const b,
                                           float *const results,
                                           size_t const size) {
  const int stride = gridDim.x * blockDim.x;
  __shared__ half2 shArray[NUM_OF_THREADS];

  half2 value(0.f, 0.f);
  shArray[threadIdx.x] = value;

  for (int i = threadIdx.x + blockDim.x + blockIdx.x; i < size; i += stride) {
    value = a[i] * b[i] + value;
  }

  shArray[threadIdx.x] = value;
  __syncthreads();
  reduceInShared_native(shArray);

  if (threadIdx.x == 0) {
    half2 result = shArray[0];
    float f_result = (float)result.y + (float)result.x;
    results[blockIdx.x] = f_result;
  }
}

void generateInput(half2 *a, size_t size) {
  for (size_t i = 0; i < size; ++i) {
    half2 temp;
    temp.x = static_cast<float>(rand() % 4);
    temp.y = static_cast<float>(rand() % 2);
    a[i] = temp;
  }
}

int main(int argc, char *argv[]) {
  srand((unsigned int)time(NULL));
  size_t size = NUM_OF_BLOCKS * NUM_OF_THREADS * 16;

  half2 *vec[2];
  half2 *devVec[2];

  float *results;
  float *devResults;

  int devID = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t devProp;
  checkCudaErrors(hipGetDeviceProperties(&devProp, devID));

  if (devProp.major < 5 || (devProp.major == 5 && devProp.minor < 3)) {
    printf(
        "ERROR: fp16ScalarProduct requires GPU devices with compute SM 5.3 or "
        "higher.\n");
    return EXIT_WAIVED;
  }

  for (int i = 0; i < 2; ++i) {
    checkCudaErrors(hipHostMalloc((void **)&vec[i], size * sizeof *vec[i]));
    checkCudaErrors(hipMalloc((void **)&devVec[i], size * sizeof *devVec[i]));
  }

  checkCudaErrors(
      hipHostMalloc((void **)&results, NUM_OF_BLOCKS * sizeof *results));
  checkCudaErrors(
      hipMalloc((void **)&devResults, NUM_OF_BLOCKS * sizeof *devResults));

  for (int i = 0; i < 2; ++i) {
    generateInput(vec[i], size);
    checkCudaErrors(hipMemcpy(devVec[i], vec[i], size * sizeof *vec[i],
                               hipMemcpyHostToDevice));
  }

  scalarProductKernel_native<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(
      devVec[0], devVec[1], devResults, size);

  checkCudaErrors(hipMemcpy(results, devResults,
                             NUM_OF_BLOCKS * sizeof *results,
                             hipMemcpyDeviceToHost));

  float result_native = 0;
  for (int i = 0; i < NUM_OF_BLOCKS; ++i) {
    result_native += results[i];
  }
  printf("Result native operators\t: %f \n", result_native);

  scalarProductKernel_intrinsics<<<NUM_OF_BLOCKS, NUM_OF_THREADS>>>(
      devVec[0], devVec[1], devResults, size);

  checkCudaErrors(hipMemcpy(results, devResults,
                             NUM_OF_BLOCKS * sizeof *results,
                             hipMemcpyDeviceToHost));

  float result_intrinsics = 0;
  for (int i = 0; i < NUM_OF_BLOCKS; ++i) {
    result_intrinsics += results[i];
  }
  printf("Result intrinsics\t: %f \n", result_intrinsics);

  printf("&&&& fp16ScalarProduct %s\n",
         (fabs(result_intrinsics - result_native) < 0.00001) ? "PASSED"
                                                             : "FAILED");

  for (int i = 0; i < 2; ++i) {
    checkCudaErrors(hipFree(devVec[i]));
    checkCudaErrors(hipHostFree(vec[i]));
  }

  checkCudaErrors(hipFree(devResults));
  checkCudaErrors(hipHostFree(results));

  return EXIT_SUCCESS;
}
