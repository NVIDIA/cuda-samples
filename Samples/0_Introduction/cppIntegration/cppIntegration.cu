#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* 
 * Example of integrating CUDA functions into an existing
 * application / framework.
 * Host part of the device code.
 * Compiled with Cuda compiler.
 */

// System includes
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

extern "C" void computeGold(char *reference, char *idata,
                            const unsigned int len);
extern "C" void computeGold2(int2 *reference, int2 *idata,
                             const unsigned int len);

///////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel(int *g_data) {
  // write data to global memory
  const unsigned int tid = threadIdx.x;
  int data = g_data[tid];

  // use integer arithmetic to process all four bytes with one thread
  // this serializes the execution, but is the simplest solutions to avoid
  // bank conflicts for this very low number of threads
  // in general it is more efficient to process each byte by a separate thread,
  // to avoid bank conflicts the access pattern should be
  // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
  // and wid is the warp id
  // see also the programming guide for a more in depth discussion.
  g_data[tid] =
      ((((data << 0) >> 24) - 10) << 24) | ((((data << 8) >> 24) - 10) << 16) |
      ((((data << 16) >> 24) - 10) << 8) | ((((data << 24) >> 24) - 10) << 0);
}

///////////////////////////////////////////////////////////////////////////////
//! Demonstration that int2 data can be used in the cpp code
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel2(int2 *g_data) {
  // write data to global memory
  const unsigned int tid = threadIdx.x;
  int2 data = g_data[tid];

  // use integer arithmetic to process all four bytes with one thread
  // this serializes the execution, but is the simplest solutions to avoid
  // bank conflicts for this very low number of threads
  // in general it is more efficient to process each byte by a separate thread,
  // to avoid bank conflicts the access pattern should be
  // g_data[4 * wtid + wid], where wtid is the thread id within the half warp
  // and wid is the warp id
  // see also the programming guide for a more in depth discussion.
  g_data[tid].x = data.x - data.y;
}

////////////////////////////////////////////////////////////////////////////////
//! Entry point for Cuda functionality on host side
//! @param argc  command line argument count
//! @param argv  command line arguments
//! @param data  data to process on the device
//! @param len   len of \a data
////////////////////////////////////////////////////////////////////////////////
extern "C" bool runTest(const int argc, const char **argv, char *data,
                        int2 *data_int2, unsigned int len) {
  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  findCudaDevice(argc, (const char **)argv);

  const unsigned int num_threads = len / 4;
  assert(0 == (len % 4));
  const unsigned int mem_size = sizeof(char) * len;
  const unsigned int mem_size_int2 = sizeof(int2) * len;

  // allocate device memory
  char *d_data;
  checkCudaErrors(hipMalloc((void **)&d_data, mem_size));
  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_data, data, mem_size, hipMemcpyHostToDevice));
  // allocate device memory for int2 version
  int2 *d_data_int2;
  checkCudaErrors(hipMalloc((void **)&d_data_int2, mem_size_int2));
  // copy host memory to device
  checkCudaErrors(hipMemcpy(d_data_int2, data_int2, mem_size_int2,
                             hipMemcpyHostToDevice));

  // setup execution parameters
  dim3 grid(1, 1, 1);
  dim3 threads(num_threads, 1, 1);
  dim3 threads2(len, 1, 1);  // more threads needed fir separate int2 version
  // execute the kernel
  kernel<<<grid, threads>>>((int *)d_data);
  kernel2<<<grid, threads2>>>(d_data_int2);

  // check if kernel execution generated and error
  getLastCudaError("Kernel execution failed");

  // compute reference solutions
  char *reference = (char *)malloc(mem_size);
  computeGold(reference, data, len);
  int2 *reference2 = (int2 *)malloc(mem_size_int2);
  computeGold2(reference2, data_int2, len);

  // copy results from device to host
  checkCudaErrors(hipMemcpy(data, d_data, mem_size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(data_int2, d_data_int2, mem_size_int2,
                             hipMemcpyDeviceToHost));

  // check result
  bool success = true;

  for (unsigned int i = 0; i < len; i++) {
    if (reference[i] != data[i] || reference2[i].x != data_int2[i].x ||
        reference2[i].y != data_int2[i].y) {
      success = false;
    }
  }

  // cleanup memory
  checkCudaErrors(hipFree(d_data));
  checkCudaErrors(hipFree(d_data_int2));
  free(reference);
  free(reference2);

  return success;
}
