#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

hipAccessPolicyWindow initAccessPolicyWindow(void) {
  hipAccessPolicyWindow accessPolicyWindow = {0};
  accessPolicyWindow.base_ptr = (void *)0;
  accessPolicyWindow.num_bytes = 0;
  accessPolicyWindow.hitRatio = 0.f;
  accessPolicyWindow.hitProp = hipAccessPropertyNormal;
  accessPolicyWindow.missProp = hipAccessPropertyStreaming;
  return accessPolicyWindow;
}

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param data  input data in global memory
//! @param dataSize  input data size
//! @param bigData  input bigData in global memory
//! @param bigDataSize  input bigData size
//! @param hitcount how many data access are done within block
////////////////////////////////////////////////////////////////////////////////
static __global__ void kernCacheSegmentTest(int *data, int dataSize, int *trash,
                                            int bigDataSize, int hitCount) {
  __shared__ unsigned int hit;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int tID = row * blockDim.y + col;
  uint32_t psRand = tID;

  atomicExch(&hit, 0);
  __syncthreads();
  while (hit < hitCount) {
    psRand ^= psRand << 13;
    psRand ^= psRand >> 17;
    psRand ^= psRand << 5;

    int idx = tID - psRand;
    if (idx < 0) {
      idx = -idx;
    }

    if ((tID % 2) == 0) {
      data[psRand % dataSize] = data[psRand % dataSize] + data[idx % dataSize];
    } else {
      trash[psRand % bigDataSize] =
          trash[psRand % bigDataSize] + trash[idx % bigDataSize];
    }

    atomicAdd(&hit, 1);
  }
}
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { runTest(argc, argv); }

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  bool bTestResult = true;
  hipAccessPolicyWindow accessPolicyWindow;
  hipDeviceProp_t deviceProp;
  hipLaunchAttributeValue streamAttrValue;
  hipStream_t stream;
  hipLaunchAttributeID streamAttrID;
  dim3 threads(32, 32);
  int *dataDevicePointer;
  int *dataHostPointer;
  int dataSize;
  int *bigDataDevicePointer;
  int *bigDataHostPointer;
  int bigDataSize;
  StopWatchInterface *timer = 0;

  printf("%s Starting...\n\n", argv[0]);

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  int devID = findCudaDevice(argc, (const char **)argv);
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);
  // Get device properties
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
  dim3 blocks(deviceProp.maxGridSize[1], 1);

  // Make sure device the l2 optimization
  if (deviceProp.persistingL2CacheMaxSize == 0) {
    printf(
        "Waiving execution as device %d does not support persisting L2 "
        "Caching\n",
        devID);
    exit(EXIT_WAIVED);
  }

  // Create stream to assiocate with window
  checkCudaErrors(hipStreamCreate(&stream));

  // Set the amount of l2 cache that will be persisting to maximum the device
  // can support
  checkCudaErrors(hipDeviceSetLimit(cudaLimitPersistingL2CacheSize,
                                     deviceProp.persistingL2CacheMaxSize));

  // Stream attribute to set
  streamAttrID = hipLaunchAttributeAccessPolicyWindow;

  // Default window
  streamAttrValue.accessPolicyWindow = initAccessPolicyWindow();
  accessPolicyWindow = initAccessPolicyWindow();

  // Allocate size of both buffers
  bigDataSize = (deviceProp.l2CacheSize * 4) / sizeof(int);
  dataSize = (deviceProp.l2CacheSize / 4) / sizeof(int);

  // Allocate data
  checkCudaErrors(hipHostMalloc(&dataHostPointer, dataSize * sizeof(int)));
  checkCudaErrors(
      hipHostMalloc(&bigDataHostPointer, bigDataSize * sizeof(int)));

  for (int i = 0; i < bigDataSize; ++i) {
    if (i < dataSize) {
      dataHostPointer[i] = i;
    }

    bigDataHostPointer[bigDataSize - i - 1] = i;
  }

  checkCudaErrors(
      hipMalloc((void **)&dataDevicePointer, dataSize * sizeof(int)));
  checkCudaErrors(
      hipMalloc((void **)&bigDataDevicePointer, bigDataSize * sizeof(int)));
  checkCudaErrors(hipMemcpyAsync(dataDevicePointer, dataHostPointer,
                                  dataSize * sizeof(int),
                                  hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipMemcpyAsync(bigDataDevicePointer, bigDataHostPointer,
                                  bigDataSize * sizeof(int),
                                  hipMemcpyHostToDevice, stream));

  // Make a window for the buffer of interest
  accessPolicyWindow.base_ptr = (void *)dataDevicePointer;
  accessPolicyWindow.num_bytes = dataSize * sizeof(int);
  accessPolicyWindow.hitRatio = 1.f;
  accessPolicyWindow.hitProp = hipAccessPropertyPersisting;
  accessPolicyWindow.missProp = hipAccessPropertyNormal;
  streamAttrValue.accessPolicyWindow = accessPolicyWindow;

  // Assign window to stream
  checkCudaErrors(
      hipStreamSetAttribute(stream, streamAttrID, &streamAttrValue));

  // Demote any previous persisting lines
  checkCudaErrors(cudaCtxResetPersistingL2Cache());

  checkCudaErrors(hipStreamSynchronize(stream));
  kernCacheSegmentTest<<<blocks, threads, 0, stream>>>(
      dataDevicePointer, dataSize, bigDataDevicePointer, bigDataSize, 0xAFFFF);

  checkCudaErrors(hipStreamSynchronize(stream));
  // check if kernel execution generated and error
  getLastCudaError("Kernel execution failed");

  // Free memory
  checkCudaErrors(hipHostFree(dataHostPointer));
  checkCudaErrors(hipHostFree(bigDataHostPointer));
  checkCudaErrors(hipFree(dataDevicePointer));
  checkCudaErrors(hipFree(bigDataDevicePointer));

  sdkStopTimer(&timer);
  printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
