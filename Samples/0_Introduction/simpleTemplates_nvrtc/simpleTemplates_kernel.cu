#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// includes, kernels
#include "sharedmem.cuh"

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////

template <class T>
__device__ void testKernel(T *g_idata, T *g_odata) {
  // Shared mem size is determined by the host app at run time
  SharedMemory<T> smem;

  T *sdata = smem.getPointer();

  // access thread id
  const unsigned int tid = threadIdx.x;

  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;

  // read in input data from global memory
  sdata[tid] = g_idata[tid];

  __syncthreads();

  // perform some computations
  sdata[tid] = (T)num_threads * sdata[tid];

  __syncthreads();

  // write data to global memory
  g_odata[tid] = sdata[tid];
}

extern "C" __global__ void testFloat(float *p1, float *p2) {
  testKernel<float>(p1, p2);
}

extern "C" __global__ void testInt(int *p1, int *p2) {
  testKernel<int>(p1, p2);
}
