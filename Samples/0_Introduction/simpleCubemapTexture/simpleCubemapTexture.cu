#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
* This sample demonstrates how to use texture fetches from layered 2D textures
* in CUDA C
*
* This sample first generates a 3D input data array for the layered texture
* and the expected output. Then it starts CUDA C kernels, one for each layer,
* which fetch their layer's texture data (using normalized texture coordinates)
* transform it to the expected output, and write it to a 3D output data array.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

static const char *sSDKname = "simpleCubemapTexture";

// includes, kernels

////////////////////////////////////////////////////////////////////////////////
//! Transform a cubemap face of a linear buffe using cubemap texture lookups
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void transformKernel(float *g_odata, int width,
                                hipTextureObject_t tex) {
  // calculate this thread's data point
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  // 0.5f offset and division are necessary to access the original data points
  // in the texture (such that bilinear interpolation will not be activated).
  // For details, see also CUDA Programming Guide, Appendix D

  float u = ((x + 0.5f) / (float)width) * 2.f - 1.f;
  float v = ((y + 0.5f) / (float)width) * 2.f - 1.f;

  float cx, cy, cz;

  for (unsigned int face = 0; face < 6; face++) {
    // Layer 0 is positive X face
    if (face == 0) {
      cx = 1;
      cy = -v;
      cz = -u;
    }
    // Layer 1 is negative X face
    else if (face == 1) {
      cx = -1;
      cy = -v;
      cz = u;
    }
    // Layer 2 is positive Y face
    else if (face == 2) {
      cx = u;
      cy = 1;
      cz = v;
    }
    // Layer 3 is negative Y face
    else if (face == 3) {
      cx = u;
      cy = -1;
      cz = -v;
    }
    // Layer 4 is positive Z face
    else if (face == 4) {
      cx = u;
      cy = -v;
      cz = 1;
    }
    // Layer 4 is negative Z face
    else if (face == 5) {
      cx = -u;
      cy = -v;
      cz = -1;
    }

    // read from texture, do expected transformation and write to global memory
    g_odata[face * width * width + y * width + x] =
        -texCubemap<float>(tex, cx, cy, cz);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  int devID = findCudaDevice(argc, (const char **)argv);

  bool bResult = true;

  // get number of SMs on this GPU
  hipDeviceProp_t deviceProps;

  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s] has %d Multi-Processors ", deviceProps.name,
         deviceProps.multiProcessorCount);
  printf("SM %d.%d\n", deviceProps.major, deviceProps.minor);

  if (deviceProps.major < 2) {
    printf(
        "%s requires SM 2.0 or higher for support of Texture Arrays.  Test "
        "will exit... \n",
        sSDKname);

    exit(EXIT_WAIVED);
  }

  // generate input data for layered texture
  unsigned int width = 64, num_faces = 6, num_layers = 1;
  unsigned int cubemap_size = width * width * num_faces;
  unsigned int size = cubemap_size * num_layers * sizeof(float);
  float *h_data = (float *)malloc(size);

  for (int i = 0; i < (int)(cubemap_size * num_layers); i++) {
    h_data[i] = (float)i;
  }

  // this is the expected transformation of the input data (the expected output)
  float *h_data_ref = (float *)malloc(size);

  for (unsigned int layer = 0; layer < num_layers; layer++) {
    for (int i = 0; i < (int)(cubemap_size); i++) {
      h_data_ref[layer * cubemap_size + i] =
          -h_data[layer * cubemap_size + i] + layer;
    }
  }

  // allocate device memory for result
  float *d_data = NULL;
  checkCudaErrors(hipMalloc((void **)&d_data, size));

  // allocate array and copy image data
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  hipArray *cu_3darray;
  //    checkCudaErrors(hipMalloc3DArray( &cu_3darray, &channelDesc,
  //    make_hipExtent(width, height, num_layers), hipArrayLayered ));
  checkCudaErrors(hipMalloc3DArray(&cu_3darray, &channelDesc,
                                    make_hipExtent(width, width, num_faces),
                                    hipArrayCubemap));
  hipMemcpy3DParms myparms = {0};
  myparms.srcPos = make_hipPos(0, 0, 0);
  myparms.dstPos = make_hipPos(0, 0, 0);
  myparms.srcPtr =
      make_hipPitchedPtr(h_data, width * sizeof(float), width, width);
  myparms.dstArray = cu_3darray;
  myparms.extent = make_hipExtent(width, width, num_faces);
  myparms.kind = hipMemcpyHostToDevice;
  checkCudaErrors(hipMemcpy3D(&myparms));

  hipTextureObject_t tex;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = cu_3darray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.addressMode[2] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

  dim3 dimBlock(8, 8, 1);
  dim3 dimGrid(width / dimBlock.x, width / dimBlock.y, 1);

  printf(
      "Covering Cubemap data array of %d~3 x %d: Grid size is %d x %d, each "
      "block has 8 x 8 threads\n",
      width, num_layers, dimGrid.x, dimGrid.y);

  transformKernel<<<dimGrid, dimBlock>>>(d_data, width,
                                         tex);  // warmup (for better timing)

  // check if kernel execution generated an error
  getLastCudaError("warmup Kernel execution failed");

  checkCudaErrors(hipDeviceSynchronize());

  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  // execute the kernel
  transformKernel<<<dimGrid, dimBlock, 0>>>(d_data, width, tex);

  // check if kernel execution generated an error
  getLastCudaError("Kernel execution failed");

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  printf("Processing time: %.3f msec\n", sdkGetTimerValue(&timer));
  printf("%.2f Mtexlookups/sec\n",
         (cubemap_size / (sdkGetTimerValue(&timer) / 1000.0f) / 1e6));
  sdkDeleteTimer(&timer);

  // allocate mem for the result on host side
  float *h_odata = (float *)malloc(size);
  // copy result from device to host
  checkCudaErrors(hipMemcpy(h_odata, d_data, size, hipMemcpyDeviceToHost));

  // write regression file if necessary
  if (checkCmdLineFlag(argc, (const char **)argv, "regression")) {
    // write file for regression test
    sdkWriteFile<float>("./data/regression.dat", h_odata, width * width, 0.0f,
                        false);
  } else {
    printf("Comparing kernel output to expected data\n");

#define MIN_EPSILON_ERROR 5e-3f
    bResult =
        compareData(h_odata, h_data_ref, cubemap_size, MIN_EPSILON_ERROR, 0.0f);
  }

  // cleanup memory
  free(h_data);
  free(h_data_ref);
  free(h_odata);

  checkCudaErrors(hipDestroyTextureObject(tex));
  checkCudaErrors(hipFree(d_data));
  checkCudaErrors(hipFreeArray(cu_3darray));

  exit(bResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
