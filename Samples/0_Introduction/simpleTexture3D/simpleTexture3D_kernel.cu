#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _SIMPLETEXTURE3D_KERNEL_CU_
#define _SIMPLETEXTURE3D_KERNEL_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipTextureObject_t tex;  // 3D texture

__global__ void d_render(uint *d_output, uint imageW, uint imageH, float w,
                         hipTextureObject_t texObj) {
  uint x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
  uint y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;

  float u = x / (float)imageW;
  float v = y / (float)imageH;
  // read from 3D texture
  float voxel = tex3D<float>(texObj, u, v, w);

  if ((x < imageW) && (y < imageH)) {
    // write output color
    uint i = __umul24(y, imageW) + x;
    d_output[i] = voxel * 255;
  }
}

extern "C" void setTextureFilterMode(bool bLinearFilter) {
  if (tex) {
    checkCudaErrors(hipDestroyTextureObject(tex));
  }
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_volumeArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode =
      bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;
  ;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.addressMode[2] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));
}

extern "C" void initCuda(const uchar *h_volume, hipExtent volumeSize) {
  // create 3D array
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uchar>();
  checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

  // copy data to 3D array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr =
      make_hipPitchedPtr((void *)h_volume, volumeSize.width * sizeof(uchar),
                          volumeSize.width, volumeSize.height);
  copyParams.dstArray = d_volumeArray;
  copyParams.extent = volumeSize;
  copyParams.kind = hipMemcpyHostToDevice;
  checkCudaErrors(hipMemcpy3D(&copyParams));

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_volumeArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  // access with normalized texture coordinates
  texDescr.normalizedCoords = true;
  // linear interpolation
  texDescr.filterMode = hipFilterModeLinear;
  // wrap texture coordinates
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.addressMode[2] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));
}

extern "C" void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output,
                              uint imageW, uint imageH, float w) {
  d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, w, tex);
}

void cleanupCuda() {
  if (tex) {
    checkCudaErrors(hipDestroyTextureObject(tex));
  }
  if (d_volumeArray) {
    checkCudaErrors(hipFreeArray(d_volumeArray));
  }
}

#endif  // #ifndef _SIMPLETEXTURE3D_KERNEL_CU_
