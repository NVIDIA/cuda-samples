#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This example shows how to use the clock function to measure the performance
 * of block of threads of a kernel accurately. Blocks are executed in parallel
 * and out of order. Since there's no synchronization mechanism between blocks,
 * we measure the clock once for each block. The clock samples are written to
 * device memory.
 */

// System includes
#include <assert.h>
#include <stdint.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// This kernel computes a standard parallel reduction and evaluates the
// time it takes to do that for each block. The timing results are stored
// in device memory.
__global__ static void timedReduction(const float *input, float *output,
                                      clock_t *timer) {
  // __shared__ float shared[2 * blockDim.x];
  extern __shared__ float shared[];

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;

  if (tid == 0) timer[bid] = clock();

  // Copy input.
  shared[tid] = input[tid];
  shared[tid + blockDim.x] = input[tid + blockDim.x];

  // Perform reduction to find minimum.
  for (int d = blockDim.x; d > 0; d /= 2) {
    __syncthreads();

    if (tid < d) {
      float f0 = shared[tid];
      float f1 = shared[tid + d];

      if (f1 < f0) {
        shared[tid] = f1;
      }
    }
  }

  // Write result.
  if (tid == 0) output[bid] = shared[0];

  __syncthreads();

  if (tid == 0) timer[bid + gridDim.x] = clock();
}

#define NUM_BLOCKS 64
#define NUM_THREADS 256

// It's interesting to change the number of blocks and the number of threads to
// understand how to keep the hardware busy.
//
// Here are some numbers I get on my G80:
//    blocks - clocks
//    1 - 3096
//    8 - 3232
//    16 - 3364
//    32 - 4615
//    64 - 9981
//
// With less than 16 blocks some of the multiprocessors of the device are idle.
// With more than 16 you are using all the multiprocessors, but there's only one
// block per multiprocessor and that doesn't allow you to hide the latency of
// the memory. With more than 32 the speed scales linearly.

// Start the main CUDA Sample here
int main(int argc, char **argv) {
  printf("CUDA Clock sample\n");

  // This will pick the best possible CUDA capable device
  int dev = findCudaDevice(argc, (const char **)argv);

  float *dinput = NULL;
  float *doutput = NULL;
  clock_t *dtimer = NULL;

  clock_t timer[NUM_BLOCKS * 2];
  float input[NUM_THREADS * 2];

  for (int i = 0; i < NUM_THREADS * 2; i++) {
    input[i] = (float)i;
  }

  checkCudaErrors(
      hipMalloc((void **)&dinput, sizeof(float) * NUM_THREADS * 2));
  checkCudaErrors(hipMalloc((void **)&doutput, sizeof(float) * NUM_BLOCKS));
  checkCudaErrors(
      hipMalloc((void **)&dtimer, sizeof(clock_t) * NUM_BLOCKS * 2));

  checkCudaErrors(hipMemcpy(dinput, input, sizeof(float) * NUM_THREADS * 2,
                             hipMemcpyHostToDevice));

  timedReduction<<<NUM_BLOCKS, NUM_THREADS, sizeof(float) * 2 * NUM_THREADS>>>(
      dinput, doutput, dtimer);

  checkCudaErrors(hipMemcpy(timer, dtimer, sizeof(clock_t) * NUM_BLOCKS * 2,
                             hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(dinput));
  checkCudaErrors(hipFree(doutput));
  checkCudaErrors(hipFree(dtimer));

  long double avgElapsedClocks = 0;

  for (int i = 0; i < NUM_BLOCKS; i++) {
    avgElapsedClocks += (long double)(timer[i + NUM_BLOCKS] - timer[i]);
  }

  avgElapsedClocks = avgElapsedClocks / NUM_BLOCKS;
  printf("Average clocks/block = %Lf\n", avgElapsedClocks);

  return EXIT_SUCCESS;
}
