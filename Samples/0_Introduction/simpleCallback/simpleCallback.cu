#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample implements multi-threaded heterogeneous computing workloads with
 * the new CPU callbacks for CUDA streams and events introduced with CUDA 5.0.
 * Together with the thread safety of the CUDA API implementing heterogeneous
 * workloads that float between CPU threads and GPUs has become simple and
 * efficient.
 *
 * The workloads in the sample follow the form CPU preprocess -> GPU process ->
 * CPU postprocess.
 * Each CPU processing step is handled by its own dedicated thread. GPU
 * workloads are sent to all available GPUs in the system.
 *
 */

// System includes
#include <stdio.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include "multithreading.h"

const int N_workloads = 8;
const int N_elements_per_workload = 100000;

CUTBarrier thread_barrier;

void CUDART_CB myStreamCallback(hipStream_t event, hipError_t status,
                                void *data);

struct heterogeneous_workload {
  int id;
  int cudaDeviceID;

  int *h_data;
  int *d_data;
  hipStream_t stream;

  bool success;
};

__global__ void incKernel(int *data, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < N) data[i]++;
}

CUT_THREADPROC launch(void *void_arg) {
  heterogeneous_workload *workload = (heterogeneous_workload *)void_arg;

  // Select GPU for this CPU thread
  checkCudaErrors(hipSetDevice(workload->cudaDeviceID));

  // Allocate Resources
  checkCudaErrors(hipStreamCreate(&workload->stream));
  checkCudaErrors(
      hipMalloc(&workload->d_data, N_elements_per_workload * sizeof(int)));
  checkCudaErrors(hipHostAlloc(&workload->h_data,
                                N_elements_per_workload * sizeof(int),
                                hipHostMallocPortable));

  // CPU thread generates data
  for (int i = 0; i < N_elements_per_workload; ++i) {
    workload->h_data[i] = workload->id + i;
  }

  // Schedule work for GPU in CUDA stream without blocking the CPU thread
  // Note: Dedicated streams enable concurrent execution of workloads on the GPU
  dim3 block(512);
  dim3 grid((N_elements_per_workload + block.x - 1) / block.x);

  checkCudaErrors(hipMemcpyAsync(workload->d_data, workload->h_data,
                                  N_elements_per_workload * sizeof(int),
                                  hipMemcpyHostToDevice, workload->stream));
  incKernel<<<grid, block, 0, workload->stream>>>(workload->d_data,
                                                  N_elements_per_workload);
  checkCudaErrors(hipMemcpyAsync(workload->h_data, workload->d_data,
                                  N_elements_per_workload * sizeof(int),
                                  hipMemcpyDeviceToHost, workload->stream));

  // New in CUDA 5.0: Add a CPU callback which is called once all currently
  // pending operations in the CUDA stream have finished
  checkCudaErrors(
      hipStreamAddCallback(workload->stream, myStreamCallback, workload, 0));

  CUT_THREADEND;
  // CPU thread end of life, GPU continues to process data...
}

CUT_THREADPROC postprocess(void *void_arg) {
  heterogeneous_workload *workload = (heterogeneous_workload *)void_arg;
  // ... GPU is done with processing, continue on new CPU thread...

  // Select GPU for this CPU thread
  checkCudaErrors(hipSetDevice(workload->cudaDeviceID));

  // CPU thread consumes results from GPU
  workload->success = true;

  for (int i = 0; i < N_workloads; ++i) {
    workload->success &= workload->h_data[i] == i + workload->id + 1;
  }

  // Free Resources
  checkCudaErrors(hipFree(workload->d_data));
  checkCudaErrors(hipHostFree(workload->h_data));
  checkCudaErrors(hipStreamDestroy(workload->stream));

  // Signal the end of the heterogeneous workload to main thread
  cutIncrementBarrier(&thread_barrier);

  CUT_THREADEND;
}

void CUDART_CB myStreamCallback(hipStream_t stream, hipError_t status,
                                void *data) {
  // Check status of GPU after stream operations are done
  checkCudaErrors(status);

  // Spawn new CPU worker thread and continue processing on the CPU
  cutStartThread(postprocess, data);
}

int main(int argc, char **argv) {
  int N_gpus, max_gpus = 0;
  int gpuInfo[32];  // assume a maximum of 32 GPUs in a system configuration

  printf("Starting simpleCallback\n");

  checkCudaErrors(hipGetDeviceCount(&N_gpus));
  printf("Found %d CUDA capable GPUs\n", N_gpus);

  if (N_gpus > 32) {
    printf("simpleCallback only supports 32 GPU(s)\n");
  }

  for (int devid = 0; devid < N_gpus; devid++) {
    int SMversion;
    hipDeviceProp_t deviceProp;
    hipSetDevice(devid);
    hipGetDeviceProperties(&deviceProp, devid);
    SMversion = deviceProp.major << 4 + deviceProp.minor;
    printf("GPU[%d] %s supports SM %d.%d", devid, deviceProp.name,
           deviceProp.major, deviceProp.minor);
    printf(", %s GPU Callback Functions\n",
           (SMversion >= 0x11) ? "capable" : "NOT capable");

    if (SMversion >= 0x11) {
      gpuInfo[max_gpus++] = devid;
    }
  }

  printf("%d GPUs available to run Callback Functions\n", max_gpus);

  heterogeneous_workload *workloads;
  workloads = (heterogeneous_workload *)malloc(N_workloads *
                                               sizeof(heterogeneous_workload));
  ;
  thread_barrier = cutCreateBarrier(N_workloads);

  // Main thread spawns a CPU worker thread for each heterogeneous workload
  printf("Starting %d heterogeneous computing workloads\n", N_workloads);

  for (int i = 0; i < N_workloads; ++i) {
    workloads[i].id = i;
    workloads[i].cudaDeviceID = gpuInfo[i % max_gpus];  // i % N_gpus;

    cutStartThread(launch, &workloads[i]);
  }

  // Sleep until all workloads have finished
  cutWaitForBarrier(&thread_barrier);
  printf("Total of %d workloads finished:\n", N_workloads);

  bool success = true;

  for (int i = 0; i < N_workloads; ++i) {
    success &= workloads[i].success;
  }

  printf("%s\n", success ? "Success" : "Failure");

  free(workloads);

  exit(success ? EXIT_SUCCESS : EXIT_FAILURE);
}
