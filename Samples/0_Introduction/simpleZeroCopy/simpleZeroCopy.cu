#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(float *a, float *b, float *c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

// Allocate generic memory with malloc() and pin it laster instead of using
// hipHostAlloc()
bool bPinGenericMemory = false;

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT 4096
#define ALIGN_UP(x, size) (((size_t)x + (size - 1)) & (~(size - 1)))

int main(int argc, char **argv) {
  int n, nelem, deviceCount;
  int idev = 0;  // use default device 0
  char *device = NULL;
  unsigned int flags;
  size_t bytes;
  float *a, *b, *c;           // Pinned memory allocated on the CPU
  float *a_UA, *b_UA, *c_UA;  // Non-4K Aligned Pinned memory on the CPU
  float *d_a, *d_b, *d_c;     // Device pointers for mapped memory
  float errorNorm, refNorm, ref, diff;
  hipDeviceProp_t deviceProp;

  if (checkCmdLineFlag(argc, (const char **)argv, "help")) {
    printf("Usage:  simpleZeroCopy [OPTION]\n\n");
    printf("Options:\n");
    printf("  --device=[device #]  Specify the device to be used\n");
    printf(
        "  --use_generic_memory (optional) use generic page-aligned for system "
        "memory\n");
    return EXIT_SUCCESS;
  }

  /* Get the device selected by the user or default to 0, and then set it. */
  if (getCmdLineArgumentString(argc, (const char **)argv, "device", &device)) {
    hipGetDeviceCount(&deviceCount);
    idev = atoi(device);

    if (idev >= deviceCount || idev < 0) {
      fprintf(stderr,
              "Device number %d is invalid, will use default CUDA device 0.\n",
              idev);
      idev = 0;
    }
  }

  // if GPU found supports SM 1.2, then continue, otherwise we exit
  if (!checkCudaCapabilities(1, 2)) {
    exit(EXIT_SUCCESS);
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "use_generic_memory")) {
#if defined(__APPLE__) || defined(MACOSX)
    bPinGenericMemory = false;  // Generic Pinning of System Paged memory is not
                                // currently supported on Mac OSX
#else
    bPinGenericMemory = true;
#endif
  }

  if (bPinGenericMemory) {
    printf("> Using Generic System Paged Memory (malloc)\n");
  } else {
    printf("> Using CUDA Host Allocated (hipHostAlloc)\n");
  }

  checkCudaErrors(hipSetDevice(idev));

  /* Verify the selected device supports mapped memory and set the device
     flags for mapping host memory. */

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, idev));

#if CUDART_VERSION >= 2020

  if (!deviceProp.canMapHostMemory) {
    fprintf(stderr, "Device %d does not support mapping CPU host memory!\n",
            idev);

    exit(EXIT_SUCCESS);
  }

  checkCudaErrors(hipSetDeviceFlags(hipDeviceMapHost));
#else
  fprintf(stderr,
          "CUDART version %d.%d does not support "
          "<hipDeviceProp_t.canMapHostMemory> field\n",
          , CUDART_VERSION / 1000, (CUDART_VERSION % 100) / 10);

  exit(EXIT_SUCCESS);
#endif

#if CUDART_VERSION < 4000

  if (bPinGenericMemory) {
    fprintf(
        stderr,
        "CUDART version %d.%d does not support <hipHostRegister> function\n",
        CUDART_VERSION / 1000, (CUDART_VERSION % 100) / 10);

    exit(EXIT_SUCCESS);
  }

#endif

  /* Allocate mapped CPU memory. */

  nelem = 1048576;
  bytes = nelem * sizeof(float);

  if (bPinGenericMemory) {
#if CUDART_VERSION >= 4000
    a_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);
    b_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);
    c_UA = (float *)malloc(bytes + MEMORY_ALIGNMENT);

    // We need to ensure memory is aligned to 4K (so we will need to padd memory
    // accordingly)
    a = (float *)ALIGN_UP(a_UA, MEMORY_ALIGNMENT);
    b = (float *)ALIGN_UP(b_UA, MEMORY_ALIGNMENT);
    c = (float *)ALIGN_UP(c_UA, MEMORY_ALIGNMENT);

    checkCudaErrors(hipHostRegister(a, bytes, hipHostRegisterMapped));
    checkCudaErrors(hipHostRegister(b, bytes, hipHostRegisterMapped));
    checkCudaErrors(hipHostRegister(c, bytes, hipHostRegisterMapped));
#endif
  } else {
#if CUDART_VERSION >= 2020
    flags = hipHostMallocMapped;
    checkCudaErrors(hipHostAlloc((void **)&a, bytes, flags));
    checkCudaErrors(hipHostAlloc((void **)&b, bytes, flags));
    checkCudaErrors(hipHostAlloc((void **)&c, bytes, flags));
#endif
  }

  /* Initialize the vectors. */

  for (n = 0; n < nelem; n++) {
    a[n] = rand() / (float)RAND_MAX;
    b[n] = rand() / (float)RAND_MAX;
  }

    /* Get the device pointers for the pinned CPU memory mapped into the GPU
       memory space. */

#if CUDART_VERSION >= 2020
  checkCudaErrors(hipHostGetDevicePointer((void **)&d_a, (void *)a, 0));
  checkCudaErrors(hipHostGetDevicePointer((void **)&d_b, (void *)b, 0));
  checkCudaErrors(hipHostGetDevicePointer((void **)&d_c, (void *)c, 0));
#endif

  /* Call the GPU kernel using the CPU pointers residing in CPU mapped memory.
   */
  printf("> vectorAddGPU kernel will add vectors using mapped CPU memory...\n");
  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem / (float)block.x));
  vectorAddGPU<<<grid, block>>>(d_a, d_b, d_c, nelem);
  checkCudaErrors(hipDeviceSynchronize());
  getLastCudaError("vectorAddGPU() execution failed");

  /* Compare the results */

  printf("> Checking the results from vectorAddGPU() ...\n");
  errorNorm = 0.f;
  refNorm = 0.f;

  for (n = 0; n < nelem; n++) {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff * diff;
    refNorm += ref * ref;
  }

  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);

  /* Memory clean up */

  printf("> Releasing CPU memory...\n");

  if (bPinGenericMemory) {
#if CUDART_VERSION >= 4000
    checkCudaErrors(hipHostUnregister(a));
    checkCudaErrors(hipHostUnregister(b));
    checkCudaErrors(hipHostUnregister(c));
    free(a_UA);
    free(b_UA);
    free(c_UA);
#endif
  } else {
#if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostFree(a));
    checkCudaErrors(hipHostFree(b));
    checkCudaErrors(hipHostFree(c));
#endif
  }

  exit(errorNorm / refNorm < 1.e-6f ? EXIT_SUCCESS : EXIT_FAILURE);
}
