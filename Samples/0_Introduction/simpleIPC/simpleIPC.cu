#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample demonstrates Inter Process Communication
 * using one process per GPU for computation.
 */
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include "hip/hip_runtime_api.h"
#include "helper_multiprocess.h"
static const char shmName[] = "simpleIPCshm";
// For direct NVLINK and PCI-E peers, at max 8 simultaneous peers are allowed
// For NVSWITCH connected peers like DGX-2, simultaneous peers are not limited
// in the same way.
#define MAX_DEVICES (32)
#define DATA_SIZE (64ULL << 20ULL)  // 64MB

#if defined(__linux__)
#define cpu_atomic_add32(a, x) __sync_add_and_fetch(a, x)
#elif defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define cpu_atomic_add32(a, x) InterlockedAdd((volatile LONG *)a, x)
#else
#error Unsupported system
#endif

typedef struct shmStruct_st {
  size_t nprocesses;
  int barrier;
  int sense;
  int devices[MAX_DEVICES];
  hipIpcMemHandle_t memHandle[MAX_DEVICES];
  hipIpcEventHandle_t eventHandle[MAX_DEVICES];
} shmStruct;

__global__ void simpleKernel(char *ptr, int sz, char val) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (; idx < sz; idx += (gridDim.x * blockDim.x)) {
    ptr[idx] = val;
  }
}

static void barrierWait(volatile int *barrier, volatile int *sense,
                        unsigned int n) {
  int count;

  // Check-in
  count = cpu_atomic_add32(barrier, 1);
  if (count == n)  // Last one in
    *sense = 1;
  while (!*sense)
    ;

  // Check-out
  count = cpu_atomic_add32(barrier, -1);
  if (count == 0)  // Last one out
    *sense = 0;
  while (*sense)
    ;
}

static void childProcess(int id) {
  volatile shmStruct *shm = NULL;
  hipStream_t stream;
  sharedMemoryInfo info;
  size_t procCount, i;
  int blocks = 0;
  int threads = 128;
  hipDeviceProp_t prop;
  std::vector<void *> ptrs;
  std::vector<hipEvent_t> events;
  std::vector<char> verification_buffer(DATA_SIZE);

  if (sharedMemoryOpen(shmName, sizeof(shmStruct), &info) != 0) {
    printf("Failed to create shared memory slab\n");
    exit(EXIT_FAILURE);
  }
  shm = (volatile shmStruct *)info.addr;
  procCount = shm->nprocesses;

  printf("Process %d: Starting on device %d...\n", id, shm->devices[id]);

  checkCudaErrors(hipSetDevice(shm->devices[id]));
  checkCudaErrors(hipGetDeviceProperties(&prop, shm->devices[id]));
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &blocks, simpleKernel, threads, 0));
  blocks *= prop.multiProcessorCount;

  // Open and track all the allocations and events created in the master
  // process for use later
  for (i = 0; i < procCount; i++) {
    void *ptr = NULL;
    hipEvent_t event;

    // Notice, we don't need to explicitly enable peer access for
    // allocations on other devices.
    checkCudaErrors(
        hipIpcOpenMemHandle(&ptr, *(hipIpcMemHandle_t *)&shm->memHandle[i],
                             hipIpcMemLazyEnablePeerAccess));
    checkCudaErrors(hipIpcOpenEventHandle(
        &event, *(hipIpcEventHandle_t *)&shm->eventHandle[i]));

    ptrs.push_back(ptr);
    events.push_back(event);
  }

  // At each iteration of the loop, each sibling process will push work on
  // their respective devices accessing the next peer mapped buffer allocated
  // by the master process (these can come from other sibling processes as
  // well). To coordinate each process' access, we force the stream to wait for
  // the work already accessing this buffer asynchronously through IPC events,
  // allowing the CPU processes to continue to queue more work.
  for (i = 0; i < procCount; i++) {
    size_t bufferId = (i + id) % procCount;
    // Wait for the buffer to be accessed to be ready
    checkCudaErrors(hipStreamWaitEvent(stream, events[bufferId], 0));
    // Push a simple kernel on it
    simpleKernel<<<blocks, threads, 0, stream>>>((char *)ptrs[bufferId],
                                                 DATA_SIZE, id);
    checkCudaErrors(hipGetLastError());
    // Signal that this buffer is ready for the next consumer
    checkCudaErrors(hipEventRecord(events[bufferId], stream));
    // Wait for all my sibling processes to push this stage of their work
    // before proceeding to the next. This prevents siblings from racing
    // ahead and clobbering the recorded event or waiting on the wrong
    // recorded event.
    barrierWait(&shm->barrier, &shm->sense, (unsigned int)procCount);
    if (id == 0) {
      printf("Step %lld done\n", (unsigned long long)i);
    }
  }

  // Now wait for my buffer to be ready so I can copy it locally and verify it
  checkCudaErrors(hipStreamWaitEvent(stream, events[id], 0));
  checkCudaErrors(hipMemcpyAsync(&verification_buffer[0], ptrs[id], DATA_SIZE,
                                  hipMemcpyDeviceToHost, stream));
  // And wait for all the queued up work to complete
  checkCudaErrors(hipStreamSynchronize(stream));

  printf("Process %d: verifying...\n", id);

  // The contents should have the id of the sibling just after me
  char compareId = (char)((id + 1) % procCount);
  for (unsigned long long j = 0; j < DATA_SIZE; j++) {
    if (verification_buffer[j] != compareId) {
      printf("Process %d: Verification mismatch at %lld: %d != %d\n", id, j,
             (int)verification_buffer[j], (int)compareId);
    }
  }

  // Clean up!
  for (i = 0; i < procCount; i++) {
    checkCudaErrors(hipIpcCloseMemHandle(ptrs[i]));
    checkCudaErrors(hipEventDestroy(events[i]));
  }

  checkCudaErrors(hipStreamDestroy(stream));

  printf("Process %d complete!\n", id);
}

static void parentProcess(char *app) {
  sharedMemoryInfo info;
  int devCount, i;
  volatile shmStruct *shm = NULL;
  std::vector<void *> ptrs;
  std::vector<hipEvent_t> events;
  std::vector<Process> processes;

  checkCudaErrors(hipGetDeviceCount(&devCount));

  if (sharedMemoryCreate(shmName, sizeof(*shm), &info) != 0) {
    printf("Failed to create shared memory slab\n");
    exit(EXIT_FAILURE);
  }
  shm = (volatile shmStruct *)info.addr;
  memset((void *)shm, 0, sizeof(*shm));

  // Pick all the devices that can access each other's memory for this test
  // Keep in mind that CUDA has minimal support for fork() without a
  // corresponding exec() in the child process, but in this case our
  // spawnProcess will always exec, so no need to worry.
  for (i = 0; i < devCount; i++) {
    bool allPeers = true;
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, i));

    // CUDA IPC is only supported on devices with unified addressing
    if (!prop.unifiedAddressing) {
      printf("Device %d does not support unified addressing, skipping...\n", i);
      continue;
    }
    // This sample requires two processes accessing each device, so we need
    // to ensure exclusive or prohibited mode is not set
    if (prop.computeMode != hipComputeModeDefault) {
      printf("Device %d is in an unsupported compute mode for this sample\n",
             i);
      continue;
    }

    for (int j = 0; j < shm->nprocesses; j++) {
      int canAccessPeerIJ, canAccessPeerJI;
      checkCudaErrors(
          hipDeviceCanAccessPeer(&canAccessPeerJI, shm->devices[j], i));
      checkCudaErrors(
          hipDeviceCanAccessPeer(&canAccessPeerIJ, i, shm->devices[j]));
      if (!canAccessPeerIJ || !canAccessPeerJI) {
        allPeers = false;
        break;
      }
    }
    if (allPeers) {
      // Enable peers here.  This isn't necessary for IPC, but it will
      // setup the peers for the device.  For systems that only allow 8
      // peers per GPU at a time, this acts to remove devices from CanAccessPeer
      for (int j = 0; j < shm->nprocesses; j++) {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipDeviceEnablePeerAccess(shm->devices[j], 0));
        checkCudaErrors(hipSetDevice(shm->devices[j]));
        checkCudaErrors(hipDeviceEnablePeerAccess(i, 0));
      }
      shm->devices[shm->nprocesses++] = i;
      if (shm->nprocesses >= MAX_DEVICES) break;
    } else {
      printf(
          "Device %d is not peer capable with some other selected peers, "
          "skipping\n",
          i);
    }
  }

  if (shm->nprocesses == 0) {
    printf("No CUDA devices support IPC\n");
    exit(EXIT_WAIVED);
  }

  // Now allocate memory and an event for each process and fill the shared
  // memory buffer with the IPC handles to communicate
  for (i = 0; i < shm->nprocesses; i++) {
    void *ptr = NULL;
    hipEvent_t event;

    checkCudaErrors(hipSetDevice(shm->devices[i]));
    checkCudaErrors(hipMalloc(&ptr, DATA_SIZE));
    checkCudaErrors(
        hipIpcGetMemHandle((hipIpcMemHandle_t *)&shm->memHandle[i], ptr));
    checkCudaErrors(hipEventCreate(
        &event, hipEventDisableTiming | hipEventInterprocess));
    checkCudaErrors(hipIpcGetEventHandle(
        (hipIpcEventHandle_t *)&shm->eventHandle[i], event));

    ptrs.push_back(ptr);
    events.push_back(event);
  }

  // Launch the child processes!
  for (i = 0; i < shm->nprocesses; i++) {
    char devIdx[10];
    char *const args[] = {app, devIdx, NULL};
    Process process;

    SPRINTF(devIdx, "%d", i);

    if (spawnProcess(&process, app, args)) {
      printf("Failed to create process\n");
      exit(EXIT_FAILURE);
    }

    processes.push_back(process);
  }

  // And wait for them to finish
  for (i = 0; i < processes.size(); i++) {
    if (waitProcess(&processes[i]) != EXIT_SUCCESS) {
      printf("Process %d failed!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  // Clean up!
  for (i = 0; i < shm->nprocesses; i++) {
    checkCudaErrors(hipSetDevice(shm->devices[i]));
    checkCudaErrors(hipEventSynchronize(events[i]));
    checkCudaErrors(hipEventDestroy(events[i]));
    checkCudaErrors(hipFree(ptrs[i]));
  }

  sharedMemoryClose(&info);
}

int main(int argc, char **argv) {
#if defined(__arm__) || defined(__aarch64__)
  printf("Not supported on ARM\n");
  return EXIT_WAIVED;
#else
  if (argc == 1) {
    parentProcess(argv[0]);
  } else {
    childProcess(atoi(argv[1]));
  }
  return EXIT_SUCCESS;
#endif
}
