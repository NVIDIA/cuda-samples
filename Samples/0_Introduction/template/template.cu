#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Template project which demonstrates the basics on how to setup a project
 * example application.
 * Host code.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);

extern "C" void computeGold(float *reference, float *idata,
                            const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
__global__ void testKernel(float *g_idata, float *g_odata) {
  // shared memory
  // the size is determined by the host application
  extern __shared__ float sdata[];

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;

  // read in input data from global memory
  sdata[tid] = g_idata[tid];
  __syncthreads();

  // perform some computations
  sdata[tid] = (float)num_threads * sdata[tid];
  __syncthreads();

  // write data to global memory
  g_odata[tid] = sdata[tid];
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) { runTest(argc, argv); }

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  bool bTestResult = true;

  printf("%s Starting...\n\n", argv[0]);

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  int devID = findCudaDevice(argc, (const char **)argv);

  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  unsigned int num_threads = 32;
  unsigned int mem_size = sizeof(float) * num_threads;

  // allocate host memory
  float *h_idata = (float *)malloc(mem_size);

  // initalize the memory
  for (unsigned int i = 0; i < num_threads; ++i) {
    h_idata[i] = (float)i;
  }

  // allocate device memory
  float *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, mem_size));
  // copy host memory to device
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

  // allocate device memory for result
  float *d_odata;
  checkCudaErrors(hipMalloc((void **)&d_odata, mem_size));

  // setup execution parameters
  dim3 grid(1, 1, 1);
  dim3 threads(num_threads, 1, 1);

  // execute the kernel
  testKernel<<<grid, threads, mem_size>>>(d_idata, d_odata);

  // check if kernel execution generated and error
  getLastCudaError("Kernel execution failed");

  // allocate mem for the result on host side
  float *h_odata = (float *)malloc(mem_size);
  // copy result from device to host
  checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,
                             hipMemcpyDeviceToHost));

  sdkStopTimer(&timer);
  printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  // compute reference solution
  float *reference = (float *)malloc(mem_size);
  computeGold(reference, h_idata, num_threads);

  // check result
  if (checkCmdLineFlag(argc, (const char **)argv, "regression")) {
    // write file for regression test
    sdkWriteFile("./data/regression.dat", h_odata, num_threads, 0.0f, false);
  } else {
    // custom output handling when no regression test running
    // in this case check if the result is equivalent to the expected solution
    bTestResult = compareData(reference, h_odata, num_threads, 0.0f, 0.0f);
  }

  // cleanup memory
  free(h_idata);
  free(h_odata);
  free(reference);
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_odata));

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
