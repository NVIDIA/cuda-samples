#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* This sample is a templatized version of the template project.
* It also shows how to correctly templatize dynamically allocated shared
* memory arrays.
* Host code.
*/

// System includes
#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <math.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

// includes, kernels
#include "sharedmem.cuh"

int g_TotalFailures = 0;

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
template <class T>
__global__ void testKernel(T *g_idata, T *g_odata) {
  // Shared mem size is determined by the host app at run time
  SharedMemory<T> smem;
  T *sdata = smem.getPointer();

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;

  // read in input data from global memory
  sdata[tid] = g_idata[tid];
  __syncthreads();

  // perform some computations
  sdata[tid] = (T)num_threads * sdata[tid];
  __syncthreads();

  // write data to global memory
  g_odata[tid] = sdata[tid];
}

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
template <class T>
void runTest(int argc, char **argv, int len);

template <class T>
void computeGold(T *reference, T *idata, const unsigned int len) {
  const T T_len = static_cast<T>(len);

  for (unsigned int i = 0; i < len; ++i) {
    reference[i] = idata[i] * T_len;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("> runTest<float,32>\n");
  runTest<float>(argc, argv, 32);
  printf("> runTest<int,64>\n");
  runTest<int>(argc, argv, 64);

  printf("\n[simpleTemplates] -> Test Results: %d Failures\n", g_TotalFailures);

  exit(g_TotalFailures == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}

// To completely templatize runTest (below) with cutil, we need to use
// template specialization to wrap up CUTIL's array comparison and file writing
// functions for different types.

// Here's the generic wrapper for cutCompare*
template <class T>
class ArrayComparator {
 public:
  bool compare(const T *reference, T *data, unsigned int len) {
    fprintf(stderr,
            "Error: no comparison function implemented for this type\n");
    return false;
  }
};

// Here's the specialization for ints:
template <>
class ArrayComparator<int> {
 public:
  bool compare(const int *reference, int *data, unsigned int len) {
    return compareData(reference, data, len, 0.15f, 0.0f);
  }
};

// Here's the specialization for floats:
template <>
class ArrayComparator<float> {
 public:
  bool compare(const float *reference, float *data, unsigned int len) {
    return compareData(reference, data, len, 0.15f, 0.15f);
  }
};

// Here's the generic wrapper for cutWriteFile*
template <class T>
class ArrayFileWriter {
 public:
  bool write(const char *filename, T *data, unsigned int len, float epsilon) {
    fprintf(stderr,
            "Error: no file write function implemented for this type\n");
    return false;
  }
};

// Here's the specialization for ints:
template <>
class ArrayFileWriter<int> {
 public:
  bool write(const char *filename, int *data, unsigned int len, float epsilon) {
    return sdkWriteFile(filename, data, len, epsilon, false);
  }
};

// Here's the specialization for floats:
template <>
class ArrayFileWriter<float> {
 public:
  bool write(const char *filename, float *data, unsigned int len,
             float epsilon) {
    return sdkWriteFile(filename, data, len, epsilon, false);
  }
};

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
template <class T>
void runTest(int argc, char **argv, int len) {
  int devID;
  hipDeviceProp_t deviceProps;

  devID = findCudaDevice(argc, (const char **)argv);

  // get number of SMs on this GPU
  checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
  printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name,
         deviceProps.multiProcessorCount);

  // create and start timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);

  // start the timer
  sdkStartTimer(&timer);

  unsigned int num_threads = len;
  unsigned int mem_size = sizeof(float) * num_threads;

  // allocate host memory
  T *h_idata = (T *)malloc(mem_size);

  // initialize the memory
  for (unsigned int i = 0; i < num_threads; ++i) {
    h_idata[i] = (T)i;
  }

  // allocate device memory
  T *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, mem_size));
  // copy host memory to device
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

  // allocate device memory for result
  T *d_odata;
  checkCudaErrors(hipMalloc((void **)&d_odata, mem_size));

  // setup execution parameters
  dim3 grid(1, 1, 1);
  dim3 threads(num_threads, 1, 1);

  // execute the kernel
  testKernel<T><<<grid, threads, mem_size>>>(d_idata, d_odata);

  // check if kernel execution generated and error
  getLastCudaError("Kernel execution failed");

  // allocate mem for the result on host side
  T *h_odata = (T *)malloc(mem_size);
  // copy result from device to host
  checkCudaErrors(hipMemcpy(h_odata, d_odata, sizeof(T) * num_threads,
                             hipMemcpyDeviceToHost));

  sdkStopTimer(&timer);
  printf("Processing time: %f (ms)\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  // compute reference solution
  T *reference = (T *)malloc(mem_size);
  computeGold<T>(reference, h_idata, num_threads);

  ArrayComparator<T> comparator;
  ArrayFileWriter<T> writer;

  // check result
  if (checkCmdLineFlag(argc, (const char **)argv, "regression")) {
    // write file for regression test
    writer.write("./data/regression.dat", h_odata, num_threads, 0.0f);
  } else {
    // custom output handling when no regression test running
    // in this case check if the result is equivalent to the expected solution
    bool res = comparator.compare(reference, h_odata, num_threads);
    printf("Compare %s\n\n", (1 == res) ? "OK" : "MISMATCH");
    g_TotalFailures += (1 != res);
  }

  // cleanup memory
  free(h_idata);
  free(h_odata);
  free(reference);
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_odata));
}
