#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This application demonstrates how to use the CUDA API to use multiple GPUs,
 * with an emphasis on simple illustration of the techniques (not on
 * performance).
 *
 * Note that in order to detect multiple GPUs in your system you have to disable
 * SLI in the nvidia control panel. Otherwise only one GPU is visible to the
 * application. On the other side, you can still extend your desktop to screens
 * attached to both GPUs.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef MAX
#define MAX(a, b) (a > b ? a : b)
#endif

#include "simpleMultiGPU.h"

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int MAX_GPU_COUNT = 32;
const int DATA_N = 1048576 * 32;

////////////////////////////////////////////////////////////////////////////////
// Simple reduction kernel.
// Refer to the 'reduction' CUDA Sample describing
// reduction optimization strategies
////////////////////////////////////////////////////////////////////////////////
__global__ static void reduceKernel(float *d_Result, float *d_Input, int N) {
  const int tid = blockIdx.x * blockDim.x + threadIdx.x;
  const int threadN = gridDim.x * blockDim.x;
  float sum = 0;

  for (int pos = tid; pos < N; pos += threadN) sum += d_Input[pos];

  d_Result[tid] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  // Solver config
  TGPUplan plan[MAX_GPU_COUNT];

  // GPU reduction results
  float h_SumGPU[MAX_GPU_COUNT];

  float sumGPU;
  double sumCPU, diff;

  int i, j, gpuBase, GPU_N;

  const int BLOCK_N = 32;
  const int THREAD_N = 256;
  const int ACCUM_N = BLOCK_N * THREAD_N;

  printf("Starting simpleMultiGPU\n");
  checkCudaErrors(hipGetDeviceCount(&GPU_N));

  if (GPU_N > MAX_GPU_COUNT) {
    GPU_N = MAX_GPU_COUNT;
  }

  printf("CUDA-capable device count: %i\n", GPU_N);

  printf("Generating input data...\n\n");

  // Subdividing input data across GPUs
  // Get data sizes for each GPU
  for (i = 0; i < GPU_N; i++) {
    plan[i].dataN = DATA_N / GPU_N;
  }

  // Take into account "odd" data sizes
  for (i = 0; i < DATA_N % GPU_N; i++) {
    plan[i].dataN++;
  }

  // Assign data ranges to GPUs
  gpuBase = 0;

  for (i = 0; i < GPU_N; i++) {
    plan[i].h_Sum = h_SumGPU + i;
    gpuBase += plan[i].dataN;
  }

  // Create streams for issuing GPU command asynchronously and allocate memory
  // (GPU and System page-locked)
  for (i = 0; i < GPU_N; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipStreamCreate(&plan[i].stream));
    // Allocate memory
    checkCudaErrors(
        hipMalloc((void **)&plan[i].d_Data, plan[i].dataN * sizeof(float)));
    checkCudaErrors(
        hipMalloc((void **)&plan[i].d_Sum, ACCUM_N * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Sum_from_device,
                                   ACCUM_N * sizeof(float)));
    checkCudaErrors(hipHostMalloc((void **)&plan[i].h_Data,
                                   plan[i].dataN * sizeof(float)));

    for (j = 0; j < plan[i].dataN; j++) {
      plan[i].h_Data[j] = (float)rand() / (float)RAND_MAX;
    }
  }

  // Start timing and compute on GPU(s)
  printf("Computing with %d GPUs...\n", GPU_N);
  // create and start timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);

  // start the timer
  sdkStartTimer(&timer);

  // Copy data to GPU, launch the kernel and copy data back. All asynchronously
  for (i = 0; i < GPU_N; i++) {
    // Set device
    checkCudaErrors(hipSetDevice(i));

    // Copy input data from CPU
    checkCudaErrors(hipMemcpyAsync(plan[i].d_Data, plan[i].h_Data,
                                    plan[i].dataN * sizeof(float),
                                    hipMemcpyHostToDevice, plan[i].stream));

    // Perform GPU computations
    reduceKernel<<<BLOCK_N, THREAD_N, 0, plan[i].stream>>>(
        plan[i].d_Sum, plan[i].d_Data, plan[i].dataN);
    getLastCudaError("reduceKernel() execution failed.\n");

    // Read back GPU results
    checkCudaErrors(hipMemcpyAsync(plan[i].h_Sum_from_device, plan[i].d_Sum,
                                    ACCUM_N * sizeof(float),
                                    hipMemcpyDeviceToHost, plan[i].stream));
  }

  // Process GPU results
  for (i = 0; i < GPU_N; i++) {
    float sum;

    // Set device
    checkCudaErrors(hipSetDevice(i));

    // Wait for all operations to finish
    hipStreamSynchronize(plan[i].stream);

    // Finalize GPU reduction for current subvector
    sum = 0;

    for (j = 0; j < ACCUM_N; j++) {
      sum += plan[i].h_Sum_from_device[j];
    }

    *(plan[i].h_Sum) = (float)sum;

    // Shut down this GPU
    checkCudaErrors(hipHostFree(plan[i].h_Sum_from_device));
    checkCudaErrors(hipFree(plan[i].d_Sum));
    checkCudaErrors(hipFree(plan[i].d_Data));
    checkCudaErrors(hipStreamDestroy(plan[i].stream));
  }

  sumGPU = 0;

  for (i = 0; i < GPU_N; i++) {
    sumGPU += h_SumGPU[i];
  }

  sdkStopTimer(&timer);
  printf("  GPU Processing time: %f (ms)\n\n", sdkGetTimerValue(&timer));
  sdkDeleteTimer(&timer);

  // Compute on Host CPU
  printf("Computing with Host CPU...\n\n");

  sumCPU = 0;

  for (i = 0; i < GPU_N; i++) {
    for (j = 0; j < plan[i].dataN; j++) {
      sumCPU += plan[i].h_Data[j];
    }
  }

  // Compare GPU and CPU results
  printf("Comparing GPU and Host CPU results...\n");
  diff = fabs(sumCPU - sumGPU) / fabs(sumCPU);
  printf("  GPU sum: %f\n  CPU sum: %f\n", sumGPU, sumCPU);
  printf("  Relative difference: %E \n\n", diff);

  // Cleanup and shutdown
  for (i = 0; i < GPU_N; i++) {
    checkCudaErrors(hipSetDevice(i));
    checkCudaErrors(hipHostFree(plan[i].h_Data));
  }

  exit((diff < 1e-5) ? EXIT_SUCCESS : EXIT_FAILURE);
}
