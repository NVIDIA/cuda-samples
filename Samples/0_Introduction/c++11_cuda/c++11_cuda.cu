#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <thrust/device_ptr.h>
#include <thrust/count.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <hip/hip_runtime_api.h>

/////////////////////////////////////////////////////////////////
// Some utility code to define grid_stride_range
// Normally this would be in a header but it's here
// for didactic purposes. Uses
#include "range.hpp"
using namespace util::lang;

// type alias to simplify typing...
template <typename T>
using step_range = typename range_proxy<T>::step_range_proxy;

template <typename T>
__device__ step_range<T> grid_stride_range(T begin, T end) {
  begin += blockDim.x * blockIdx.x + threadIdx.x;
  return range(begin, end).step(gridDim.x * blockDim.x);
}
/////////////////////////////////////////////////////////////////

template <typename T, typename Predicate>
__device__ void count_if(int *count, T *data, int n, Predicate p) {
  for (auto i : grid_stride_range(0, n)) {
    if (p(data[i])) atomicAdd(count, 1);
  }
}

// Use count_if with a lambda function that searches for x, y, z or w
// Note the use of range-based for loop and initializer_list inside the functor
// We use auto so we don't have to know the type of the functor or array
__global__ void xyzw_frequency(int *count, char *text, int n) {
  const char letters[]{'x', 'y', 'z', 'w'};

  count_if(count, text, n, [&](char c) {
    for (const auto x : letters)
      if (c == x) return true;
    return false;
  });
}

__global__ void xyzw_frequency_thrust_device(int *count, char *text, int n) {
  const char letters[]{'x', 'y', 'z', 'w'};
  *count = thrust::count_if(thrust::device, text, text + n, [=](char c) {
    for (const auto x : letters)
      if (c == x) return true;
    return false;
  });
}

// a bug in Thrust 1.8 causes warnings when this is uncommented
// so commented out by default -- fixed in Thrust master branch
#if 0 
void xyzw_frequency_thrust_host(int *count, char *text, int n)
{
  const char letters[] { 'x','y','z','w' };
  *count = thrust::count_if(thrust::host, text, text+n, [&](char c) {
    for (const auto x : letters) 
      if (c == x) return true;
    return false;
  });
}
#endif

int main(int argc, char **argv) {
  const char *filename = sdkFindFilePath("warandpeace.txt", argv[0]);

  int numBytes = 16 * 1048576;
  char *h_text = (char *)malloc(numBytes);

  // find first CUDA device
  int devID = findCudaDevice(argc, (const char **)argv);

  char *d_text;
  checkCudaErrors(hipMalloc((void **)&d_text, numBytes));

  FILE *fp = fopen(filename, "r");
  if (fp == NULL) {
    printf("Cannot find the input text file\n. Exiting..\n");
    return EXIT_FAILURE;
  }
  int len = (int)fread(h_text, sizeof(char), numBytes, fp);
  fclose(fp);
  std::cout << "Read " << len << " byte corpus from " << filename << std::endl;

  checkCudaErrors(hipMemcpy(d_text, h_text, len, hipMemcpyHostToDevice));

  int count = 0;
  int *d_count;
  checkCudaErrors(hipMalloc(&d_count, sizeof(int)));
  checkCudaErrors(hipMemset(d_count, 0, sizeof(int)));

  // Try uncommenting one kernel call at a time
  xyzw_frequency<<<8, 256>>>(d_count, d_text, len);
  xyzw_frequency_thrust_device<<<1, 1>>>(d_count, d_text, len);
  checkCudaErrors(
      hipMemcpy(&count, d_count, sizeof(int), hipMemcpyDeviceToHost));

  // xyzw_frequency_thrust_host(&count, h_text, len);

  std::cout << "counted " << count
            << " instances of 'x', 'y', 'z', or 'w' in \"" << filename << "\""
            << std::endl;

  checkCudaErrors(hipFree(d_count));
  checkCudaErrors(hipFree(d_text));

  return EXIT_SUCCESS;
}
