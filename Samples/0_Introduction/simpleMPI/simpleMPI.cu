#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Simple example demonstrating how to use MPI with CUDA
*
*  Generate some random numbers on one node.
*  Dispatch them to all nodes.
*  Compute their square root on each node's GPU.
*  Compute the average of the results using MPI.
*
*  simpleMPI.cu: GPU part, compiled with nvcc
*/

#include <iostream>
using std::cerr;
using std::endl;

#include "simpleMPI.h"

// Error handling macro
#define CUDA_CHECK(call)                                                 \
  if ((call) != hipSuccess) {                                           \
    hipError_t err = hipGetLastError();                                \
    cerr << "CUDA error calling \"" #call "\", code is " << err << endl; \
    my_abort(err);                                                       \
  }

// Device code
// Very simple GPU Kernel that computes square roots of input numbers
__global__ void simpleMPIKernel(float *input, float *output) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  output[tid] = sqrt(input[tid]);
}

// Initialize an array with random data (between 0 and 1)
void initData(float *data, int dataSize) {
  for (int i = 0; i < dataSize; i++) {
    data[i] = (float)rand() / RAND_MAX;
  }
}

// CUDA computation on each node
// No MPI here, only CUDA
void computeGPU(float *hostData, int blockSize, int gridSize) {
  int dataSize = blockSize * gridSize;

  // Allocate data on GPU memory
  float *deviceInputData = NULL;
  CUDA_CHECK(hipMalloc((void **)&deviceInputData, dataSize * sizeof(float)));

  float *deviceOutputData = NULL;
  CUDA_CHECK(hipMalloc((void **)&deviceOutputData, dataSize * sizeof(float)));

  // Copy to GPU memory
  CUDA_CHECK(hipMemcpy(deviceInputData, hostData, dataSize * sizeof(float),
                        hipMemcpyHostToDevice));

  // Run kernel
  simpleMPIKernel<<<gridSize, blockSize>>>(deviceInputData, deviceOutputData);

  // Copy data back to CPU memory
  CUDA_CHECK(hipMemcpy(hostData, deviceOutputData, dataSize * sizeof(float),
                        hipMemcpyDeviceToHost));

  // Free GPU memory
  CUDA_CHECK(hipFree(deviceInputData));
  CUDA_CHECK(hipFree(deviceOutputData));
}

float sum(float *data, int size) {
  float accum = 0.f;

  for (int i = 0; i < size; i++) {
    accum += data[i];
  }

  return accum;
}
