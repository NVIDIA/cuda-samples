#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample illustrates the usage of CUDA streams for overlapping
 * kernel execution with device/host memcopies.  The kernel is used to
 * initialize an array to a specific value, after which the array is
 * copied to the host (CPU) memory.  To increase performance, multiple
 * kernel/memcopy pairs are launched asynchronously, each pair in its
 * own stream.  Devices with Compute Capability 1.1 can overlap a kernel
 * and a memcopy as long as they are issued in different streams.  Kernels
 * are serialized.  Thus, if n pairs are launched, streamed approach
 * can reduce the memcopy cost to the (1/n)th of a single copy of the entire
 * data set.
 *
 * Additionally, this sample uses CUDA events to measure elapsed time for
 * CUDA calls.  Events are a part of CUDA API and provide a system independent
 * way to measure execution times on CUDA devices with approximately 0.5
 * microsecond precision.
 *
 * Elapsed times are averaged over nreps repetitions (10 by default).
 *
*/

const char *sSDKsample = "simpleStreams";

const char *sEventSyncMethod[] = {"hipEventDefault", "hipEventBlockingSync",
                                  "hipEventDisableTiming", NULL};

const char *sDeviceSyncMethod[] = {
    "hipDeviceScheduleAuto",         "hipDeviceScheduleSpin",
    "hipDeviceScheduleYield",        "INVALID",
    "hipDeviceScheduleBlockingSync", NULL};

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#ifndef WIN32
#include <sys/mman.h>  // for mmap() / munmap()
#endif

// Macro to aligned up to the memory size in question
#define MEMORY_ALIGNMENT 4096
#define ALIGN_UP(x, size) (((size_t)x + (size - 1)) & (~(size - 1)))

__global__ void init_array(int *g_data, int *factor, int num_iterations) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = 0; i < num_iterations; i++) {
    g_data[idx] += *factor;  // non-coalesced on purpose, to burn time
  }
}

bool correct_data(int *a, const int n, const int c) {
  for (int i = 0; i < n; i++) {
    if (a[i] != c) {
      printf("%d: %d %d\n", i, a[i], c);
      return false;
    }
  }

  return true;
}

inline void AllocateHostMemory(bool bPinGenericMemory, int **pp_a,
                               int **ppAligned_a, int nbytes) {
#if CUDART_VERSION >= 4000
#if !defined(__arm__) && !defined(__aarch64__)
  if (bPinGenericMemory) {
// allocate a generic page-aligned chunk of system memory
#ifdef WIN32
    printf(
        "> VirtualAlloc() allocating %4.2f Mbytes of (generic page-aligned "
        "system memory)\n",
        (float)nbytes / 1048576.0f);
    *pp_a = (int *)VirtualAlloc(NULL, (nbytes + MEMORY_ALIGNMENT),
                                MEM_RESERVE | MEM_COMMIT, PAGE_READWRITE);
#else
    printf(
        "> mmap() allocating %4.2f Mbytes (generic page-aligned system "
        "memory)\n",
        (float)nbytes / 1048576.0f);
    *pp_a = (int *)mmap(NULL, (nbytes + MEMORY_ALIGNMENT),
                        PROT_READ | PROT_WRITE, MAP_PRIVATE | MAP_ANON, -1, 0);
#endif

    *ppAligned_a = (int *)ALIGN_UP(*pp_a, MEMORY_ALIGNMENT);

    printf(
        "> hipHostRegister() registering %4.2f Mbytes of generic allocated "
        "system memory\n",
        (float)nbytes / 1048576.0f);
    // pin allocate memory
    checkCudaErrors(
        hipHostRegister(*ppAligned_a, nbytes, hipHostRegisterMapped));
  } else
#endif
#endif
  {
    printf("> hipHostMalloc() allocating %4.2f Mbytes of system memory\n",
           (float)nbytes / 1048576.0f);
    // allocate host memory (pinned is required for achieve asynchronicity)
    checkCudaErrors(hipHostMalloc((void **)pp_a, nbytes));
    *ppAligned_a = *pp_a;
  }
}

inline void FreeHostMemory(bool bPinGenericMemory, int **pp_a,
                           int **ppAligned_a, int nbytes) {
#if CUDART_VERSION >= 4000
#if !defined(__arm__) && !defined(__aarch64__)
  // CUDA 4.0 support pinning of generic host memory
  if (bPinGenericMemory) {
    // unpin and delete host memory
    checkCudaErrors(hipHostUnregister(*ppAligned_a));
#ifdef WIN32
    VirtualFree(*pp_a, 0, MEM_RELEASE);
#else
    munmap(*pp_a, nbytes);
#endif
  } else
#endif
#endif
  {
    hipHostFree(*pp_a);
  }
}

static const char *sSyncMethod[] = {
    "0 (Automatic Blocking)",
    "1 (Spin Blocking)",
    "2 (Yield Blocking)",
    "3 (Undefined Blocking Method)",
    "4 (Blocking Sync Event) = low CPU utilization",
    NULL};

void printHelp() {
  printf("Usage: %s [options below]\n", sSDKsample);
  printf("\t--sync_method=n for CPU/GPU synchronization\n");
  printf("\t             n=%s\n", sSyncMethod[0]);
  printf("\t             n=%s\n", sSyncMethod[1]);
  printf("\t             n=%s\n", sSyncMethod[2]);
  printf("\t   <Default> n=%s\n", sSyncMethod[4]);
  printf(
      "\t--use_generic_memory (default) use generic page-aligned for system "
      "memory\n");
  printf(
      "\t--use_cuda_malloc_host (optional) use hipHostMalloc to allocate "
      "system memory\n");
}

#if defined(__APPLE__) || defined(MACOSX)
#define DEFAULT_PINNED_GENERIC_MEMORY false
#else
#define DEFAULT_PINNED_GENERIC_MEMORY true
#endif

int main(int argc, char **argv) {
  int cuda_device = 0;
  int nstreams = 4;              // number of streams for CUDA calls
  int nreps = 10;                // number of times each experiment is repeated
  int n = 16 * 1024 * 1024;      // number of ints in the data set
  int nbytes = n * sizeof(int);  // number of data bytes
  dim3 threads, blocks;          // kernel launch configuration
  float elapsed_time, time_memcpy, time_kernel;  // timing variables
  float scale_factor = 1.0f;

  // allocate generic memory and pin it laster instead of using hipHostAlloc()

  bool bPinGenericMemory =
      DEFAULT_PINNED_GENERIC_MEMORY;  // we want this to be the default behavior
  int device_sync_method =
      hipDeviceScheduleBlockingSync;  // by default we use BlockingSync

  int niterations;  // number of iterations for the loop inside the kernel

  printf("[ %s ]\n\n", sSDKsample);

  if (checkCmdLineFlag(argc, (const char **)argv, "help")) {
    printHelp();
    return EXIT_SUCCESS;
  }

  if ((device_sync_method = getCmdLineArgumentInt(argc, (const char **)argv,
                                                  "sync_method")) >= 0) {
    if (device_sync_method == 0 || device_sync_method == 1 ||
        device_sync_method == 2 || device_sync_method == 4) {
      printf("Device synchronization method set to = %s\n",
             sSyncMethod[device_sync_method]);
      printf("Setting reps to 100 to demonstrate steady state\n");
      nreps = 100;
    } else {
      printf("Invalid command line option sync_method=\"%d\"\n",
             device_sync_method);
      return EXIT_FAILURE;
    }
  } else {
    printHelp();
    return EXIT_SUCCESS;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "use_generic_memory")) {
#if defined(__APPLE__) || defined(MACOSX)
    bPinGenericMemory = false;  // Generic Pinning of System Paged memory not
                                // currently supported on Mac OSX
#else
    bPinGenericMemory = true;
#endif
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "use_cuda_malloc_host")) {
    bPinGenericMemory = false;
  }

  printf("\n> ");
  cuda_device = findCudaDevice(argc, (const char **)argv);

  // check the compute capability of the device
  int num_devices = 0;
  checkCudaErrors(hipGetDeviceCount(&num_devices));

  if (0 == num_devices) {
    printf(
        "your system does not have a CUDA capable device, waiving test...\n");
    return EXIT_WAIVED;
  }

  // check if the command-line chosen device ID is within range, exit if not
  if (cuda_device >= num_devices) {
    printf(
        "cuda_device=%d is invalid, must choose device ID between 0 and %d\n",
        cuda_device, num_devices - 1);
    return EXIT_FAILURE;
  }

  checkCudaErrors(hipSetDevice(cuda_device));

  // Checking for compute capabilities
  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

  niterations = 5;

  // Check if GPU can map host memory (Generic Method), if not then we override
  // bPinGenericMemory to be false
  if (bPinGenericMemory) {
    printf("Device: <%s> canMapHostMemory: %s\n", deviceProp.name,
           deviceProp.canMapHostMemory ? "Yes" : "No");

    if (deviceProp.canMapHostMemory == 0) {
      printf(
          "Using hipHostMalloc, CUDA device does not support mapping of "
          "generic host memory\n");
      bPinGenericMemory = false;
    }
  }

  // Anything that is less than 32 Cores will have scaled down workload
  scale_factor =
      max((32.0f / (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
                    (float)deviceProp.multiProcessorCount)),
          1.0f);
  n = (int)rint((float)n / scale_factor);

  printf("> CUDA Capable: SM %d.%d hardware\n", deviceProp.major,
         deviceProp.minor);
  printf("> %d Multiprocessor(s) x %d (Cores/Multiprocessor) = %d (Cores)\n",
         deviceProp.multiProcessorCount,
         _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
         _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
             deviceProp.multiProcessorCount);

  printf("> scale_factor = %1.4f\n", 1.0f / scale_factor);
  printf("> array_size   = %d\n\n", n);

  // enable use of blocking sync, to reduce CPU usage
  printf("> Using CPU/GPU Device Synchronization method (%s)\n",
         sDeviceSyncMethod[device_sync_method]);
  checkCudaErrors(hipSetDeviceFlags(
      device_sync_method | (bPinGenericMemory ? hipDeviceMapHost : 0)));

  // allocate host memory
  int c = 5;            // value to which the array will be initialized
  int *h_a = 0;         // pointer to the array data in host memory
  int *hAligned_a = 0;  // pointer to the array data in host memory (aligned to
                        // MEMORY_ALIGNMENT)

  // Allocate Host memory (could be using hipHostMalloc or VirtualAlloc/mmap if
  // using the new CUDA 4.0 features
  AllocateHostMemory(bPinGenericMemory, &h_a, &hAligned_a, nbytes);

  // allocate device memory
  int *d_a = 0,
      *d_c = 0;  // pointers to data and init value in the device memory
  checkCudaErrors(hipMalloc((void **)&d_a, nbytes));
  checkCudaErrors(hipMemset(d_a, 0x0, nbytes));
  checkCudaErrors(hipMalloc((void **)&d_c, sizeof(int)));
  checkCudaErrors(hipMemcpy(d_c, &c, sizeof(int), hipMemcpyHostToDevice));

  printf("\nStarting Test\n");

  // allocate and initialize an array of stream handles
  hipStream_t *streams =
      (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

  for (int i = 0; i < nstreams; i++) {
    checkCudaErrors(hipStreamCreate(&(streams[i])));
  }

  // create CUDA event handles
  // use blocking sync
  hipEvent_t start_event, stop_event;
  int eventflags =
      ((device_sync_method == hipDeviceScheduleBlockingSync) ? hipEventBlockingSync
                                                      : hipEventDefault);

  checkCudaErrors(hipEventCreateWithFlags(&start_event, eventflags));
  checkCudaErrors(hipEventCreateWithFlags(&stop_event, eventflags));

  // time memcopy from device
  checkCudaErrors(hipEventRecord(start_event, 0));  // record in stream-0, to
                                                     // ensure that all previous
                                                     // CUDA calls have
                                                     // completed
  checkCudaErrors(hipMemcpyAsync(hAligned_a, d_a, nbytes,
                                  hipMemcpyDeviceToHost, streams[0]));
  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(
      stop_event));  // block until the event is actually recorded
  checkCudaErrors(hipEventElapsedTime(&time_memcpy, start_event, stop_event));
  printf("memcopy:\t%.2f\n", time_memcpy);

  // time kernel
  threads = dim3(512, 1);
  blocks = dim3(n / threads.x, 1);
  checkCudaErrors(hipEventRecord(start_event, 0));
  init_array<<<blocks, threads, 0, streams[0]>>>(d_a, d_c, niterations);
  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(stop_event));
  checkCudaErrors(hipEventElapsedTime(&time_kernel, start_event, stop_event));
  printf("kernel:\t\t%.2f\n", time_kernel);

  //////////////////////////////////////////////////////////////////////
  // time non-streamed execution for reference
  threads = dim3(512, 1);
  blocks = dim3(n / threads.x, 1);
  checkCudaErrors(hipEventRecord(start_event, 0));

  for (int k = 0; k < nreps; k++) {
    init_array<<<blocks, threads>>>(d_a, d_c, niterations);
    checkCudaErrors(
        hipMemcpy(hAligned_a, d_a, nbytes, hipMemcpyDeviceToHost));
  }

  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(stop_event));
  checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));
  printf("non-streamed:\t%.2f\n", elapsed_time / nreps);

  //////////////////////////////////////////////////////////////////////
  // time execution with nstreams streams
  threads = dim3(512, 1);
  blocks = dim3(n / (nstreams * threads.x), 1);
  memset(hAligned_a, 255,
         nbytes);  // set host memory bits to all 1s, for testing correctness
  checkCudaErrors(hipMemset(
      d_a, 0, nbytes));  // set device memory to all 0s, for testing correctness
  checkCudaErrors(hipEventRecord(start_event, 0));

  for (int k = 0; k < nreps; k++) {
    // asynchronously launch nstreams kernels, each operating on its own portion
    // of data
    for (int i = 0; i < nstreams; i++) {
      init_array<<<blocks, threads, 0, streams[i]>>>(d_a + i * n / nstreams,
                                                     d_c, niterations);
    }

    // asynchronously launch nstreams memcopies.  Note that memcopy in stream x
    // will only
    //   commence executing when all previous CUDA calls in stream x have
    //   completed
    for (int i = 0; i < nstreams; i++) {
      checkCudaErrors(hipMemcpyAsync(hAligned_a + i * n / nstreams,
                                      d_a + i * n / nstreams, nbytes / nstreams,
                                      hipMemcpyDeviceToHost, streams[i]));
    }
  }

  checkCudaErrors(hipEventRecord(stop_event, 0));
  checkCudaErrors(hipEventSynchronize(stop_event));
  checkCudaErrors(hipEventElapsedTime(&elapsed_time, start_event, stop_event));
  printf("%d streams:\t%.2f\n", nstreams, elapsed_time / nreps);

  // check whether the output is correct
  printf("-------------------------------\n");
  bool bResults = correct_data(hAligned_a, n, c * nreps * niterations);

  // release resources
  for (int i = 0; i < nstreams; i++) {
    checkCudaErrors(hipStreamDestroy(streams[i]));
  }

  checkCudaErrors(hipEventDestroy(start_event));
  checkCudaErrors(hipEventDestroy(stop_event));

  // Free hipHostMalloc or Generic Host allocated memory (from CUDA 4.0)
  FreeHostMemory(bPinGenericMemory, &h_a, &hAligned_a, nbytes);

  checkCudaErrors(hipFree(d_a));
  checkCudaErrors(hipFree(d_c));

  return bResults ? EXIT_SUCCESS : EXIT_FAILURE;
}
