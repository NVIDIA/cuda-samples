#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Multi-GPU sample using OpenMP for threading on the CPU side
 * needs a compiler that supports OpenMP 2.0
 */

#include <hip/hip_runtime_api.h>
#include <omp.h>
#include <stdio.h>  // stdio functions are used since C++ streams aren't necessarily thread safe

using namespace std;

// a simple kernel that simply increments each array element by b
__global__ void kernelAddConstant(int *g_a, const int b) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  g_a[idx] += b;
}

// a predicate that checks whether each array element is set to its index plus b
int correctResult(int *data, const int n, const int b) {
  for (int i = 0; i < n; i++)
    if (data[i] != i + b) return 0;

  return 1;
}

int main(int argc, char *argv[]) {
  int num_gpus = 0;  // number of CUDA GPUs

  printf("%s Starting...\n\n", argv[0]);

  /////////////////////////////////////////////////////////////////
  // determine the number of CUDA capable GPUs
  //
  hipGetDeviceCount(&num_gpus);

  if (num_gpus < 1) {
    printf("no CUDA capable devices were detected\n");
    return 1;
  }

  /////////////////////////////////////////////////////////////////
  // display CPU and GPU configuration
  //
  printf("number of host CPUs:\t%d\n", omp_get_num_procs());
  printf("number of CUDA devices:\t%d\n", num_gpus);

  for (int i = 0; i < num_gpus; i++) {
    hipDeviceProp_t dprop;
    hipGetDeviceProperties(&dprop, i);
    printf("   %d: %s\n", i, dprop.name);
  }

  printf("---------------------------\n");

  /////////////////////////////////////////////////////////////////
  // initialize data
  //
  unsigned int n = num_gpus * 8192;
  unsigned int nbytes = n * sizeof(int);
  int *a = 0;  // pointer to data on the CPU
  int b = 3;   // value by which the array is incremented
  a = (int *)malloc(nbytes);

  if (0 == a) {
    printf("couldn't allocate CPU memory\n");
    return 1;
  }

  for (unsigned int i = 0; i < n; i++) a[i] = i;

  ////////////////////////////////////////////////////////////////
  // run as many CPU threads as there are CUDA devices
  //   each CPU thread controls a different device, processing its
  //   portion of the data.  It's possible to use more CPU threads
  //   than there are CUDA devices, in which case several CPU
  //   threads will be allocating resources and launching kernels
  //   on the same device.  For example, try omp_set_num_threads(2*num_gpus);
  //   Recall that all variables declared inside an "omp parallel" scope are
  //   local to each CPU thread
  //
  omp_set_num_threads(
      num_gpus);  // create as many CPU threads as there are CUDA devices
// omp_set_num_threads(2*num_gpus);// create twice as many CPU threads as there
// are CUDA devices
#pragma omp parallel
  {
    unsigned int cpu_thread_id = omp_get_thread_num();
    unsigned int num_cpu_threads = omp_get_num_threads();

    // set and check the CUDA device for this CPU thread
    int gpu_id = -1;
    checkCudaErrors(hipSetDevice(
        cpu_thread_id %
        num_gpus));  // "% num_gpus" allows more CPU threads than GPU devices
    checkCudaErrors(hipGetDevice(&gpu_id));
    printf("CPU thread %d (of %d) uses CUDA device %d\n", cpu_thread_id,
           num_cpu_threads, gpu_id);

    int *d_a =
        0;  // pointer to memory on the device associated with this CPU thread
    int *sub_a =
        a +
        cpu_thread_id * n /
            num_cpu_threads;  // pointer to this CPU thread's portion of data
    unsigned int nbytes_per_kernel = nbytes / num_cpu_threads;
    dim3 gpu_threads(128);  // 128 threads per block
    dim3 gpu_blocks(n / (gpu_threads.x * num_cpu_threads));

    checkCudaErrors(hipMalloc((void **)&d_a, nbytes_per_kernel));
    checkCudaErrors(hipMemset(d_a, 0, nbytes_per_kernel));
    checkCudaErrors(
        hipMemcpy(d_a, sub_a, nbytes_per_kernel, hipMemcpyHostToDevice));
    kernelAddConstant<<<gpu_blocks, gpu_threads>>>(d_a, b);

    checkCudaErrors(
        hipMemcpy(sub_a, d_a, nbytes_per_kernel, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(d_a));
  }
  printf("---------------------------\n");

  if (hipSuccess != hipGetLastError())
    printf("%s\n", hipGetErrorString(hipGetLastError()));

  ////////////////////////////////////////////////////////////////
  // check the result
  //
  bool bResult = correctResult(a, n, b);

  if (a) free(a);  // free CPU memory

  exit(bResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
