#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* A program demonstrating trivial use of system-wide atomics on migratable
 * memory.
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include <stdint.h>
#include <cstdio>
#include <ctime>

#define min(a, b) (a) < (b) ? (a) : (b)
#define max(a, b) (a) > (b) ? (a) : (b)

#define LOOP_NUM 50

__global__ void atomicKernel(int *atom_arr) {
  unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = 0; i < LOOP_NUM; i++) {
    // Atomic addition
    atomicAdd_system(&atom_arr[0], 10);

    // Atomic exchange
    atomicExch_system(&atom_arr[1], tid);

    // Atomic maximum
    atomicMax_system(&atom_arr[2], tid);

    // Atomic minimum
    atomicMin_system(&atom_arr[3], tid);

    // Atomic increment (modulo 17+1)
    atomicInc_system((unsigned int *)&atom_arr[4], 17);

    // Atomic decrement
    atomicDec_system((unsigned int *)&atom_arr[5], 137);

    // Atomic compare-and-swap
    atomicCAS_system(&atom_arr[6], tid - 1, tid);

    // Bitwise atomic instructions

    // Atomic AND
    atomicAnd_system(&atom_arr[7], 2 * tid + 7);

    // Atomic OR
    atomicOr_system(&atom_arr[8], 1 << tid);

    // Atomic XOR
    atomicXor_system(&atom_arr[9], tid);
  }
}

void atomicKernel_CPU(int *atom_arr, int no_of_threads) {
  for (int i = no_of_threads; i < 2 * no_of_threads; i++) {
    for (int j = 0; j < LOOP_NUM; j++) {
      // Atomic addition
      __sync_fetch_and_add(&atom_arr[0], 10);

      // Atomic exchange
      __sync_lock_test_and_set(&atom_arr[1], i);

      // Atomic maximum
      int old, expected;
      do {
        expected = atom_arr[2];
        old = __sync_val_compare_and_swap(&atom_arr[2], expected,
                                          max(expected, i));
      } while (old != expected);

      // Atomic minimum
      do {
        expected = atom_arr[3];
        old = __sync_val_compare_and_swap(&atom_arr[3], expected,
                                          min(expected, i));
      } while (old != expected);

      // Atomic increment (modulo 17+1)
      int limit = 17;
      do {
        expected = atom_arr[4];
        old = __sync_val_compare_and_swap(
            &atom_arr[4], expected, (expected >= limit) ? 0 : expected + 1);
      } while (old != expected);

      // Atomic decrement
      limit = 137;
      do {
        expected = atom_arr[5];
        old = __sync_val_compare_and_swap(
            &atom_arr[5], expected,
            ((expected == 0) || (expected > limit)) ? limit : expected - 1);
      } while (old != expected);

      // Atomic compare-and-swap
      __sync_val_compare_and_swap(&atom_arr[6], i - 1, i);

      // Bitwise atomic instructions

      // Atomic AND
      __sync_fetch_and_and(&atom_arr[7], 2 * i + 7);

      // Atomic OR
      __sync_fetch_and_or(&atom_arr[8], 1 << i);

      // Atomic XOR
      // 11th element should be 0xff
      __sync_fetch_and_xor(&atom_arr[9], i);
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set
//! Each element is multiplied with the number of threads / array length
//! @param reference  reference data, computed but preallocated
//! @param idata      input data as provided to device
//! @param len        number of elements in reference / idata
////////////////////////////////////////////////////////////////////////////////
int verify(int *testData, const int len) {
  int val = 0;

  for (int i = 0; i < len * LOOP_NUM; ++i) {
    val += 10;
  }

  if (val != testData[0]) {
    printf("atomicAdd failed val = %d testData = %d\n", val, testData[0]);
    return false;
  }

  val = 0;

  bool found = false;

  for (int i = 0; i < len; ++i) {
    // second element should be a member of [0, len)
    if (i == testData[1]) {
      found = true;
      break;
    }
  }

  if (!found) {
    printf("atomicExch failed\n");
    return false;
  }

  val = -(1 << 8);

  for (int i = 0; i < len; ++i) {
    // third element should be len-1
    val = max(val, i);
  }

  if (val != testData[2]) {
    printf("atomicMax failed\n");
    return false;
  }

  val = 1 << 8;

  for (int i = 0; i < len; ++i) {
    val = min(val, i);
  }

  if (val != testData[3]) {
    printf("atomicMin failed\n");
    return false;
  }

  int limit = 17;
  val = 0;

  for (int i = 0; i < len * LOOP_NUM; ++i) {
    val = (val >= limit) ? 0 : val + 1;
  }

  if (val != testData[4]) {
    printf("atomicInc failed\n");
    return false;
  }

  limit = 137;
  val = 0;

  for (int i = 0; i < len * LOOP_NUM; ++i) {
    val = ((val == 0) || (val > limit)) ? limit : val - 1;
  }

  if (val != testData[5]) {
    printf("atomicDec failed\n");
    return false;
  }

  found = false;

  for (int i = 0; i < len; ++i) {
    // seventh element should be a member of [0, len)
    if (i == testData[6]) {
      found = true;
      break;
    }
  }

  if (!found) {
    printf("atomicCAS failed\n");
    return false;
  }

  val = 0xff;

  for (int i = 0; i < len; ++i) {
    // 8th element should be 1
    val &= (2 * i + 7);
  }

  if (val != testData[7]) {
    printf("atomicAnd failed\n");
    return false;
  }

  val = 0;

  for (int i = 0; i < len; ++i) {
    // 9th element should be 0xff
    val |= (1 << i);
  }

  if (val != testData[8]) {
    printf("atomicOr failed\n");
    return false;
  }

  val = 0xff;

  for (int i = 0; i < len; ++i) {
    // 11th element should be 0xff
    val ^= i;
  }

  if (val != testData[9]) {
    printf("atomicXor failed\n");
    return false;
  }

  return true;
}

int main(int argc, char **argv) {
  // set device
  hipDeviceProp_t device_prop;
  int dev_id = findCudaDevice(argc, (const char **)argv);
  checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));

  if (!device_prop.managedMemory) {
    // This samples requires being run on a device that supports Unified Memory
    fprintf(stderr, "Unified Memory not supported on this device\n");
    exit(EXIT_WAIVED);
  }

  if (device_prop.computeMode == hipComputeModeProhibited) {
    // This sample requires being run with a default or process exclusive mode
    fprintf(stderr,
            "This sample requires a device in either default or process "
            "exclusive mode\n");
    exit(EXIT_WAIVED);
  }

  if (device_prop.major < 6) {
    printf(
        "%s: requires a minimum CUDA compute 6.0 capability, waiving "
        "testing.\n",
        argv[0]);
    exit(EXIT_WAIVED);
  }

  unsigned int numThreads = 256;
  unsigned int numBlocks = 64;
  unsigned int numData = 10;

  int *atom_arr;

  if (device_prop.pageableMemoryAccess) {
    printf("CAN access pageable memory\n");
    atom_arr = (int *)malloc(sizeof(int) * numData);
  } else {
    printf("CANNOT access pageable memory\n");
    checkCudaErrors(hipMallocManaged(&atom_arr, sizeof(int) * numData));
  }

  for (unsigned int i = 0; i < numData; i++) atom_arr[i] = 0;

  // To make the AND and XOR tests generate something other than 0...
  atom_arr[7] = atom_arr[9] = 0xff;

  atomicKernel<<<numBlocks, numThreads>>>(atom_arr);
  atomicKernel_CPU(atom_arr, numBlocks * numThreads);

  checkCudaErrors(hipDeviceSynchronize());

  // Compute & verify reference solution
  int testResult = verify(atom_arr, 2 * numThreads * numBlocks);

  if (device_prop.pageableMemoryAccess) {
    free(atom_arr);
  } else {
    hipFree(atom_arr);
  }

  printf("systemWideAtomics completed, returned %s \n",
         testResult ? "OK" : "ERROR!");
  exit(testResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
