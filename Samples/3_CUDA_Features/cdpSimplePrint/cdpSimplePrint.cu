#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#include <cstdio>
#include <cstdlib>
#include <iostream>

////////////////////////////////////////////////////////////////////////////////
// Variable on the GPU used to generate unique identifiers of blocks.
////////////////////////////////////////////////////////////////////////////////
__device__ int g_uids = 0;

////////////////////////////////////////////////////////////////////////////////
// Print a simple message to signal the block which is currently executing.
////////////////////////////////////////////////////////////////////////////////
__device__ void print_info(int depth, int thread, int uid, int parent_uid) {
  if (threadIdx.x == 0) {
    if (depth == 0)
      printf("BLOCK %d launched by the host\n", uid);
    else {
      char buffer[32];

      for (int i = 0; i < depth; ++i) {
        buffer[3 * i + 0] = '|';
        buffer[3 * i + 1] = ' ';
        buffer[3 * i + 2] = ' ';
      }

      buffer[3 * depth] = '\0';
      printf("%sBLOCK %d launched by thread %d of block %d\n", buffer, uid,
             thread, parent_uid);
    }
  }

  __syncthreads();
}

////////////////////////////////////////////////////////////////////////////////
// The kernel using CUDA dynamic parallelism.
//
// It generates a unique identifier for each block. Prints the information
// about that block. Finally, if the 'max_depth' has not been reached, the
// block launches new blocks directly from the GPU.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_kernel(int max_depth, int depth, int thread,
                           int parent_uid) {
  // We create a unique ID per block. Thread 0 does that and shares the value
  // with the other threads.
  __shared__ int s_uid;

  if (threadIdx.x == 0) {
    s_uid = atomicAdd(&g_uids, 1);
  }

  __syncthreads();

  // We print the ID of the block and information about its parent.
  print_info(depth, thread, s_uid, parent_uid);

  // We launch new blocks if we haven't reached the max_depth yet.
  if (++depth >= max_depth) {
    return;
  }

  cdp_kernel<<<gridDim.x, blockDim.x>>>(max_depth, depth, threadIdx.x, s_uid);
}

////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("starting Simple Print (CUDA Dynamic Parallelism)\n");

  // Parse a few command-line arguments.
  int max_depth = 2;

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "h")) {
    printf(
        "Usage: %s depth=<max_depth>\t(where max_depth is a value between 1 "
        "and 8).\n",
        argv[0]);
    exit(EXIT_SUCCESS);
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "depth")) {
    max_depth = getCmdLineArgumentInt(argc, (const char **)argv, "depth");

    if (max_depth < 1 || max_depth > 8) {
      printf("depth parameter has to be between 1 and 8\n");
      exit(EXIT_FAILURE);
    }
  }

  // Find/set the device.
  int device = -1;
  hipDeviceProp_t deviceProp;
  device = findCudaDevice(argc, (const char **)argv);
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));

  if (!(deviceProp.major > 3 ||
        (deviceProp.major == 3 && deviceProp.minor >= 5))) {
    printf("GPU %d - %s  does not support CUDA Dynamic Parallelism\n Exiting.",
           device, deviceProp.name);
    exit(EXIT_WAIVED);
  }

  // Print a message describing what the sample does.
  printf(
      "*********************************************************************"
      "******\n");
  printf(
      "The CPU launches 2 blocks of 2 threads each. On the device each thread "
      "will\n");
  printf(
      "launch 2 blocks of 2 threads each. The GPU we will do that "
      "recursively\n");
  printf("until it reaches max_depth=%d\n\n", max_depth);
  printf("In total 2");
  int num_blocks = 2, sum = 2;

  for (int i = 1; i < max_depth; ++i) {
    num_blocks *= 4;
    printf("+%d", num_blocks);
    sum += num_blocks;
  }

  printf("=%d blocks are launched!!! (%d from the GPU)\n", sum, sum - 2);
  printf(
      "************************************************************************"
      "***\n\n");

  // We set the recursion limit for CDP to max_depth.
  hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, max_depth);

  // Launch the kernel from the CPU.
  printf("Launching cdp_kernel() with CUDA Dynamic Parallelism:\n\n");
  cdp_kernel<<<2, 2>>>(max_depth, 0, 0, -1);
  checkCudaErrors(hipGetLastError());

  // Finalize.
  checkCudaErrors(hipDeviceSynchronize());

  exit(EXIT_SUCCESS);
}
