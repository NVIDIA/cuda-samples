#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


// CUDA sample demonstrating a Double precision GEMM computation using the Warp
//  Matrix Multiply and Accumulate API introduced in CUDA 11.0.

// In this program, the compute_dgemm kernel computes the result of a matrix multiplication
// and addition: D = alpha * A * B + beta * C. The dimensions of both C and D matrices
// are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x K_GLOBAL (row-major), the B matrix
// is K_GLOBAL x N_GLOBAL (column-major).
// In that kernel, each CTA computes one 64 x 64 tile of the resulting matrix
// per iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 64 x 64 tile to compute.
// Each CTA consists of eight warps. For the 64 x 64 tile, each warp computes eight
// 8 x 8 subtiles, organized in a 2 x 4 two-dimensional array.
// Warps compute the 8 x 8 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and accumulating
// the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 64 x 64 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments from
//   shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the A and B
//   data from shared memory, thus reducing the number of data copies from global memory.
// - The portions of the A and B matrices are stored in shared memory with an additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_DOUBLE macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory contents to
//   global memory, again avoiding redundant random global memory accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register utilization,
//   but carefully enough to avoid local memory use.

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <mma.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/memcpy_async.h>
#include <cuda/std/type_traits>
#include <cuda/barrier>
#include <cuda/pipeline>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Externally configurable parameters.

#ifndef CPU_DEBUG
// Set this to 1 to verify the correctness of the GPU-computed matrix.
#define CPU_DEBUG 0
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 0
#endif

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 8
#define N 8
#define K 4

// GEMM configuration.

#define M_TILES 1024
#define N_TILES 1024
#define K_TILES 1024

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit 8x16-tile chunks of each
// the A and B matrix data, that are (M = 8) * (K = 4) * 8 * (CHUNK_K = 16) * sizeof(double) = 32 Kb each
// But we cannot account the 4 Kb total skew overhead, without which the performance
// would be severely impacted. So we choose to reduce the chunk size in half,
// i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 8
#else
#define CHUNK_K 16
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(double))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B matrix
// in shared memory to minimize possible bank conflicts.
// Before performing the nvcuda::wmma::mma_sync operation, the warp must load the matrix
// data using the nvcuda::wmma::load_matrix_sync operation. Although the memory access pattern
// is not specified for that function, each lane in the warp can read one or multiple matrix
// elements from different matrix rows or columns.
// For shared memory, such access can result in bank conflicts if different rows / columns
// of the matrix map to the same bank. By shifting each row and column by a few bytes, we
// make sure that they map to different banks, thus reducing the number of possible bank
// conflicts.
// The number of 4 eight-byte "double" elements is chosen as the minimum possible shift because
// we must keep each row and column 256-bit aligned, as required by nvcuda::wmma::load_matrix_sync.
#define SKEW_DOUBLE 4

#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("Line %d: '%s' failed: %s\n", __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)

enum kernels
{
    dmma_shmem_gemm_async_copy      = 0, // DMMA shmem using kernel with async_copy
    dmma_shmem_gemm_cg_async_copy   = 1, // DMMA shmem using kernel with cooperative groups async_copy
    dmma_shmem_gemm                 = 2, // DMMA shmem using kernel normal copy (without async_copy).
    simple_dmma_gemm                = 3  // DMMA non-shmem using simple kernel.
};

const char* kernelNames[] = {"compute_dgemm_async_copy", "compute_dgemm_cg_async_copy",
                            "compute_dgemm", "simple_wmma_gemm"};

using namespace nvcuda;
namespace cg = cooperative_groups;

__host__ void init_host_matrices(double *a, double *b, double *c)
{
    for (int i = 0; i < M_GLOBAL; i++) {
        for (int j = 0; j < K_GLOBAL; j++) {
            a[i*K_GLOBAL+j] = (double) (rand() % 3);
        }
    }

    for (int i = 0; i < N_GLOBAL; i++) {
        for (int j = 0; j < K_GLOBAL; j++) {
            b[i*K_GLOBAL+j] = (double) (rand() % 3);
        }
    }

    for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
        c[t] =  (double) (rand() % 3);
    }
}

__global__ void compute_dgemm(const double *A, const double *B, const double *C, double *D, double alpha, double beta)
{
#if __CUDA_ARCH__ >= 800
    extern __shared__ double shmem[][CHUNK_K * K + SKEW_DOUBLE];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;


    // This pointer is used to access the C and D matrix tiles this warp computes.
    double *shmem_warp_tile_ptr = (double*)&shmem[0][0] + (warpId / BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS + (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
    double *shmem_warp_stream_ptr = (double*)&shmem[0][0] + warpId * SHMEM_STRIDE * N;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may result
    // in a loss of precision). Zero still needs to be specially handled though.
    beta /= alpha;

    // Each CTA slides along the 64 x 64 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const double *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < N; i++) {
            *((int4 *)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) =
                *((int4 *)(src_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId);
        }

        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, double> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Scale the C matrix.
#pragma unroll
       for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const double *warp_ptr = (warpId < (WARPS_PER_BLOCK/2)) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);

        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            size_t shmem_idx = warpId < (WARPS_PER_BLOCK/2) ? (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                                              (N * (warpId % (WARPS_PER_BLOCK/2)) * 2 + shmem_idx_b_off);

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            const double *lane_ptr = warp_ptr + tile_k * K + (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL;

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
            for(int i = 0; i < ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP); i++) {
                 // Copy 16 bytes at once in each lane.
                *((int4*)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) = *((int4*)lane_ptr +  (laneId % CHUNK_COPY_LINE_LANES));

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }

            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId/2) * M * 2 + (i * M);
                    const double *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);

#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const double *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);

                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }

            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
                // warp are well-defined even though element indices within fragment storage are not defined.
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global memory.
        double *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < N; i++) {
            *((int4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
#endif
}

__global__ void compute_dgemm_async_copy(const double *A, const double *B, const double *C, double *D, double alpha, double beta)
{
#if __CUDA_ARCH__ >= 800
    extern __shared__ double shmem[][CHUNK_K * K + SKEW_DOUBLE];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    constexpr size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    // This pointer is used to access the C and D matrix tiles this warp computes.
    double *shmem_warp_tile_ptr = &shmem[0][0] + (warpId/BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS + (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
    double *shmem_warp_stream_ptr = &shmem[0][0] + warpId * SHMEM_STRIDE * N;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may result
    // in a loss of precision). Zero still needs to be specially handled though.
    beta /= alpha;

    cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();

    const auto shape2 = cuda::aligned_size_t<alignof(double2)>(sizeof(double2));
    constexpr int loadStride = 1; // load 2 double, left-shift by 1.

    // Each CTA slides along the 64 x 64 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const double *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < N; i++) {
            pipe.producer_acquire();
            cuda::memcpy_async(&shmem_warp_stream_ptr[(SHMEM_STRIDE * i) + (laneId << loadStride)],
                                &src_gmem_warp_stream_ptr[(GLOBAL_MEM_STRIDE * i) + (laneId << loadStride)],
                                shape2, pipe);

            pipe.producer_commit();
        }
        // Now wait for all the above issued 8 batches to complete.
        cuda::pipeline_consumer_wait_prior<0>(pipe);
        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, double> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
                // Scale the C matrix.
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        pipe.consumer_release();
        // sync here so that shared memory can then be used for loading A & B matrices.
        __syncthreads();

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const double *warp_ptr = (warpId < (WARPS_PER_BLOCK/2)) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);

        const int stridePerLaneCopy = (laneId / CHUNK_COPY_LINE_LANES);
        constexpr int chunksPerLane = ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP);
        const int laneLoadElem = (laneId % CHUNK_COPY_LINE_LANES) << loadStride;

        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            // As for DMMA  M == N we use M for warp 4-7 + shmem_idx_b_off.
            size_t shmem_idx = (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) + (shmem_idx_b_off * (warpId/(WARPS_PER_BLOCK/2)));

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            const double *lane_ptr = warp_ptr + tile_k * K + stridePerLaneCopy * K_GLOBAL + laneLoadElem;

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += stridePerLaneCopy;
#pragma unroll
            for(int i = 0; i < chunksPerLane; i++) {
                 // Copy 16 bytes at once in each lane.
                pipe.producer_acquire();

                cuda::memcpy_async(&shmem[shmem_idx][laneLoadElem], lane_ptr, shape2, pipe);

                pipe.producer_commit();

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }

            cuda::pipeline_consumer_wait_prior<0>(pipe);
            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b[WARP_ROW_TILES];
#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId/2) * M * 2 + (i * M);
                    const double *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);
#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const double *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);
                        }
                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }
            pipe.consumer_release();
            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
                // warp are well-defined even though element indices within fragment storage are not defined.
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global memory.
        double *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < N; i++) {
            *((int4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
#endif
}

 __global__ void compute_dgemm_cg_async_copy(const double *A, const double *B, const double *C, double *D, double alpha, double beta)
{
#if __CUDA_ARCH__ >= 800
    extern __shared__ double shmem[][CHUNK_K * K + SKEW_DOUBLE];
    auto cta = cg::this_thread_block();
    auto tile32 = cg::tiled_partition<32>(cta);

    constexpr int tileChunkCopySize = WARP_SIZE / CHUNK_COPY_LINES_PER_WARP;
    auto tileChunkCopy = cg::tiled_partition<tileChunkCopySize>(cta);

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    constexpr size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    // This pointer is used to access the C and D matrix tiles this warp computes.
    double *shmem_warp_tile_ptr = (double*)&shmem[0][0] + (warpId/2) * SHMEM_STRIDE * N * 2 + (warpId%2) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
    double *shmem_warp_stream_ptr = (double*)&shmem[0][0] + warpId * SHMEM_STRIDE * N;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may result
    // in a loss of precision). Zero still needs to be specially handled though.
    beta /= alpha;

    // Each CTA slides along the 64 x 64 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const double *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < N; i++) {
            auto dst_ptr = &shmem_warp_stream_ptr[(SHMEM_STRIDE * i)];
            auto src_ptr = &src_gmem_warp_stream_ptr[(GLOBAL_MEM_STRIDE * i)];
            cg::memcpy_async(tile32, dst_ptr, src_ptr, cuda::aligned_size_t<alignof(double2)>{tile32.size() * sizeof(double2)});
        }

        cg::wait(cta);

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, double> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;
                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
            }
        }

        // Scale the C matrix.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        // sync here so that shared memory can then be used for loading A & B matrices.
        cg::wait(cta);
        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const double *warp_ptr = (warpId < 4) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
            (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);

        const int stridePerLaneCopy = (laneId / CHUNK_COPY_LINE_LANES);
        constexpr int chunksPerLane = ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP);
        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            // As for DMMA  M == N we use M for warp 4-7 + shmem_idx_b_off.
            size_t shmem_idx = (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) + (shmem_idx_b_off * (warpId/(WARPS_PER_BLOCK/2)));

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            auto lane_ptr = warp_ptr + tile_k * K + stridePerLaneCopy * K_GLOBAL;

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += stridePerLaneCopy;

#pragma unroll
            for(int i = 0; i < chunksPerLane; i++) {
                // Copy 16 bytes at once in each lane.
                auto dst_ptr = &shmem[shmem_idx][0];
                auto src_ptr = lane_ptr;

                cg::memcpy_async(tileChunkCopy, dst_ptr, src_ptr, 
                                cuda::aligned_size_t<alignof(double2)>{tileChunkCopySize * sizeof(double2)});

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }
            cg::wait(cta);

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId/2) * M * 2 + (i * M);
                    const double *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);

#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const double *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_DOUBLE);

                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }
            cg::sync(cta);
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
                // warp are well-defined even though element indices within fragment storage are not defined.
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                double *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        cg::sync(cta);

        // Now that shared memory contains all the D tiles, stream them to global memory.
        double *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < N; i++) {
            *((int4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }
        cg::sync(cta);
    }
#endif
}

// Performs an MxNxK DGEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 8, 8 and 4 respectively. 
//  3) A is row major, B is column major matrix.
// Note: This is a less performant version of the compute_dgemm kernel. It is designed for
//       demonstration purposes only to show the CUDA WMMA API use without relying on
//       availability of the shared memory.
__global__ void simple_wmma_gemm(double *a, double *b, double *c, double *d, int m_ld, int n_ld, int k_ld, double alpha, double beta)
{
#if __CUDA_ARCH__ >= 800
    // Leading dimensions. Packed with no transpositions.
    int lda = k_ld;
    int ldb = k_ld;
    int ldc = n_ld;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, M, N, K, double, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, M, N, K, double, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, M, N, K, double> acc_frag;
    wmma::fragment<wmma::accumulator, M, N, K, double> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < k_ld; i += K) {
        int aCol = i;
        int aRow = warpM * M;

        int bCol = warpN * N;
        int bRow = i;

        // Bounds checking
        if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
            // Load the inputs
            wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
            wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }
    }

    // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
    int cCol = warpN * N;
    int cRow = warpM * M;

    if (cRow < m_ld && cCol < n_ld) {
        wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc, wmma::mem_row_major);

        for(int i=0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc, wmma::mem_row_major);
    }
#endif
}

__host__ void matMultiplyOnHost(double *A, double *B, double *C,
                                float alpha, float beta,
                                int numARows, int numAColumns,
                                int numBRows, int numBColumns,
                                int numCRows, int numCColumns)
{
    for (int i = 0; i < numCRows; i++) {
        for (int j = 0; j < numCColumns; j++) {
            double temp = 0.0;

            for (int k = 0; k < numAColumns; k++) {
                // B matrix is column major. A matrix is row major.
                temp += A[i * numAColumns + k] * B[j * numBRows + k];
            }

            C[i*numCColumns + j] = temp * alpha + beta * C[i * numCColumns + j];
        }
    }
}

int main(int argc, char **argv)
{
    printf("Initializing...\n");

    int dev = findCudaDevice(argc, (const char **)argv);

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

    // Double precision Tensor cores require a GPU of Ampere (SM8X) architecture or higher.
    if (deviceProp.major < 8) {
        printf("dmmaTensorCoreGemm requires SM 8.0 or higher.  Exiting...\n");
        exit(EXIT_WAIVED);
    }

    printf("M: %d (%d x %d)\n", M_GLOBAL, M, M_TILES);
    printf("N: %d (%d x %d)\n", N_GLOBAL, N, N_TILES);
    printf("K: %d (%d x %d)\n", K_GLOBAL, K, K_TILES);

    double *A_h = NULL;
    double *B_h = NULL;
    double *C_h = NULL;
#if CPU_DEBUG
    double *result_hD = NULL;
    double *result_host = NULL;
#endif

    A_h = (double*) malloc(sizeof(double) * M_GLOBAL * K_GLOBAL);
    B_h = (double*) malloc(sizeof(double) * K_GLOBAL * N_GLOBAL);
    C_h = (double*) malloc(sizeof(double) * M_GLOBAL * N_GLOBAL);
#if CPU_DEBUG
    result_hD   = (double*) malloc(sizeof(double) * M_GLOBAL * N_GLOBAL);
    result_host = (double*) malloc(sizeof(double) * M_GLOBAL * N_GLOBAL);
#endif

    double *A = NULL;
    double *B = NULL;
    double *C = NULL;
    double *D = NULL;

    checkCudaErrors(hipMalloc((void**)&A, sizeof(double) * M_GLOBAL * K_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&B, sizeof(double) * N_GLOBAL * K_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&C, sizeof(double) * M_GLOBAL * N_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&D, sizeof(double) * M_GLOBAL * N_GLOBAL));

    assert(((unsigned long long)A) % 128 == 0);
    assert(((unsigned long long)B) % 128 == 0);
    assert(((unsigned long long)C) % 128 == 0);
    assert(((unsigned long long)D) % 128 == 0);

    init_host_matrices(A_h, B_h, C_h);

    printf("Preparing data for GPU...\n");

    checkCudaErrors(hipMemcpy(A, A_h, sizeof(double) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(B, B_h, sizeof(double) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(C, C_h, sizeof(double) * M_GLOBAL * N_GLOBAL, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(D, 0, sizeof(double) * M_GLOBAL * N_GLOBAL));

    enum {
        // Compute the right amount of shared memory to request.
        // We need shared memory to hold per-CTA C and D matrix tiles, and to cache per-CTA chunks
        // of the A and B matrices. Therefore, the right amount to request is the maximum of those
        // two numbers.
        SHMEM_SZ = MAX(sizeof(double) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_DOUBLE) * 2,
                       M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N * (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(double))
    };

    printf("Required shared memory size: %lu Kb\n", SHMEM_SZ / 1024UL);

    const double alpha = 1.1f;
    const double beta = 1.2f;

    hipEvent_t start, stop;

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));

    kernels selected_kernel = dmma_shmem_gemm_async_copy;

    // kernel to run - default (dmma_shmem_gemm_async_copy == 0)
    if (checkCmdLineFlag(argc, (const char **)argv, "kernel")) {
        int kernel_number = getCmdLineArgumentInt(argc, (const char **)argv, "kernel");
        if (kernel_number < 4)
        {
            selected_kernel = (kernels)kernel_number;
        }
        else
        {
            printf("Error: kernel number should be between 0 to 3, you have entered %d\n", kernel_number);
            exit(EXIT_FAILURE);
        }
    }

    // If enough shared memory available on the GPU use high performant kernel
    if ((deviceProp.sharedMemPerMultiprocessor >= SHMEM_SZ) && (selected_kernel != simple_dmma_gemm))
    {
        printf("Computing using high performance kernel = %d - %s\n", selected_kernel, kernelNames[selected_kernel]);

        switch (selected_kernel)
        {
            case dmma_shmem_gemm_async_copy :
            default:
                checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(compute_dgemm_async_copy), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));
                checkKernelErrors((compute_dgemm_async_copy<<<deviceProp.multiProcessorCount*3, THREADS_PER_BLOCK, SHMEM_SZ>>>(A, B, C, D, alpha, beta)));
                break;
            case dmma_shmem_gemm_cg_async_copy :
                checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(compute_dgemm_cg_async_copy), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));
                checkKernelErrors((compute_dgemm_cg_async_copy<<<deviceProp.multiProcessorCount*3, THREADS_PER_BLOCK, SHMEM_SZ>>>(A, B, C, D, alpha, beta)));
                break;
            case dmma_shmem_gemm :
                checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(compute_dgemm), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));
                checkKernelErrors((compute_dgemm<<<deviceProp.multiProcessorCount*2, THREADS_PER_BLOCK, SHMEM_SZ>>>(A, B, C, D, alpha, beta)));
                break;
        }

#if CPU_DEBUG
        checkCudaErrors(hipMemcpy(result_hD, D, sizeof(double)*M_GLOBAL*N_GLOBAL, hipMemcpyDeviceToHost));
#endif
    }
    else
    {
        dim3 gridDim;
        dim3 blockDim;

        // blockDim.x must be a multple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (M_GLOBAL + (M * blockDim.x / 32 - 1)) / (M * blockDim.x / 32);
        gridDim.y = (N_GLOBAL + N * blockDim.y - 1) / (N * blockDim.y);

        printf("Computing... using simple_wmma_gemm kernel\n");
        simple_wmma_gemm<<<gridDim, blockDim>>>(A, B, C, D, M_GLOBAL, N_GLOBAL, K_GLOBAL, alpha, beta);
#if CPU_DEBUG
        checkCudaErrors(hipMemcpy(result_hD, D, sizeof(double) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost));
#endif
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));

#if CPU_DEBUG
    printf("Verifying correctness of the computations...\n");

    memcpy(result_host, C_h, sizeof(double) * M_GLOBAL * N_GLOBAL);

    matMultiplyOnHost(A_h, B_h, result_host,
                      alpha, beta,
                      M_GLOBAL, K_GLOBAL,
                      K_GLOBAL, N_GLOBAL,
                      M_GLOBAL, N_GLOBAL);

    size_t number_of_matches = 0;
    for (int i = 0; i < N_GLOBAL*M_GLOBAL; i++) {
        if  (fabs(result_hD[i] - result_host[i]) > 0.1f)
        {
            printf("mismatch i=%d result_hD=%f result_host=%f\n", i, result_hD[i], result_host[i]);
            break;
        }
        else
        {
            number_of_matches++;
        }
    }
    printf("number_of_matches = %zu out of = %d \n", number_of_matches, N_GLOBAL*M_GLOBAL);
    free(result_hD);
    free(result_host);
#endif

    float milliseconds = 0;

    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    printf("Time: %f ms\n", milliseconds);
    printf("FP64 TFLOPS: %.2f\n", (((double)M_GLOBAL * N_GLOBAL * K_GLOBAL * 2)/(milliseconds/1000.)) / 1e12);

    free(A_h);
    free(B_h);
    free(C_h);
    checkCudaErrors(hipFree((void*)A));
    checkCudaErrors(hipFree((void*)B));
    checkCudaErrors(hipFree((void*)C));
    checkCudaErrors(hipFree((void*)D));

    return 0;
}
