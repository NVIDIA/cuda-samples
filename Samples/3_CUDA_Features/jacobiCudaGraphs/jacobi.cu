#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <vector>
#include "jacobi.h"

namespace cg = cooperative_groups;

// 8 Rows of square-matrix A processed by each CTA.
// This can be max 32 and only power of 2 (i.e., 2/4/8/16/32).
#define ROWS_PER_CTA 8

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double *address, double val) {
  unsigned long long int *address_as_ull = (unsigned long long int *)address;
  unsigned long long int old = *address_as_ull, assumed;

  do {
    assumed = old;
    old = atomicCAS(address_as_ull, assumed,
                    __double_as_longlong(val + __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN !=
    // NaN)
  } while (assumed != old);

  return __longlong_as_double(old);
}
#endif

static __global__ void JacobiMethod(const float *A, const double *b,
                                    const float conv_threshold, double *x,
                                    double *x_new, double *sum) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ double x_shared[N_ROWS];  // N_ROWS == n
  __shared__ double b_shared[ROWS_PER_CTA + 1];

  for (int i = threadIdx.x; i < N_ROWS; i += blockDim.x) {
    x_shared[i] = x[i];
  }

  if (threadIdx.x < ROWS_PER_CTA) {
    int k = threadIdx.x;
    for (int i = k + (blockIdx.x * ROWS_PER_CTA);
         (k < ROWS_PER_CTA) && (i < N_ROWS);
         k += ROWS_PER_CTA, i += ROWS_PER_CTA) {
      b_shared[i % (ROWS_PER_CTA + 1)] = b[i];
    }
  }

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  for (int k = 0, i = blockIdx.x * ROWS_PER_CTA;
       (k < ROWS_PER_CTA) && (i < N_ROWS); k++, i++) {
    double rowThreadSum = 0.0;
    for (int j = threadIdx.x; j < N_ROWS; j += blockDim.x) {
      rowThreadSum += (A[i * N_ROWS + j] * x_shared[j]);
    }

    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      rowThreadSum += tile32.shfl_down(rowThreadSum, offset);
    }

    if (tile32.thread_rank() == 0) {
      atomicAdd(&b_shared[i % (ROWS_PER_CTA + 1)], -rowThreadSum);
    }
  }

  cg::sync(cta);

  if (threadIdx.x < ROWS_PER_CTA) {
    cg::thread_block_tile<ROWS_PER_CTA> tile8 =
        cg::tiled_partition<ROWS_PER_CTA>(cta);
    double temp_sum = 0.0;

    int k = threadIdx.x;

    for (int i = k + (blockIdx.x * ROWS_PER_CTA);
         (k < ROWS_PER_CTA) && (i < N_ROWS);
         k += ROWS_PER_CTA, i += ROWS_PER_CTA) {
      double dx = b_shared[i % (ROWS_PER_CTA + 1)];
      dx /= A[i * N_ROWS + i];

      x_new[i] = (x_shared[i] + dx);
      temp_sum += fabs(dx);
    }

    for (int offset = tile8.size() / 2; offset > 0; offset /= 2) {
      temp_sum += tile8.shfl_down(temp_sum, offset);
    }

    if (tile8.thread_rank() == 0) {
      atomicAdd(sum, temp_sum);
    }
  }
}

// Thread block size for finalError kernel should be multiple of 32
static __global__ void finalError(double *x, double *g_sum) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  extern __shared__ double warpSum[];
  double sum = 0.0;

  int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

  for (int i = globalThreadId; i < N_ROWS; i += blockDim.x * gridDim.x) {
    double d = x[i] - 1.0;
    sum += fabs(d);
  }

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
    sum += tile32.shfl_down(sum, offset);
  }

  if (tile32.thread_rank() == 0) {
    warpSum[threadIdx.x / warpSize] = sum;
  }

  cg::sync(cta);

  double blockSum = 0.0;
  if (threadIdx.x < (blockDim.x / warpSize)) {
    blockSum = warpSum[threadIdx.x];
  }

  if (threadIdx.x < 32) {
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      blockSum += tile32.shfl_down(blockSum, offset);
    }
    if (tile32.thread_rank() == 0) {
      atomicAdd(g_sum, blockSum);
    }
  }
}

double JacobiMethodGpuCudaGraphExecKernelSetParams(
    const float *A, const double *b, const float conv_threshold,
    const int max_iter, double *x, double *x_new, hipStream_t stream) {
  // CTA size
  dim3 nthreads(256, 1, 1);
  // grid size
  dim3 nblocks((N_ROWS / ROWS_PER_CTA) + 2, 1, 1);
  hipGraph_t graph;
  hipGraphExec_t graphExec = NULL;

  double sum = 0.0;
  double *d_sum = NULL;
  checkCudaErrors(hipMalloc(&d_sum, sizeof(double)));

  std::vector<hipGraphNode_t> nodeDependencies;
  hipGraphNode_t memcpyNode, jacobiKernelNode, memsetNode;
  hipMemcpy3DParms memcpyParams = {0};
  hipMemsetParams memsetParams = {0};

  memsetParams.dst = (void *)d_sum;
  memsetParams.value = 0;
  memsetParams.pitch = 0;
  // elementSize can be max 4 bytes, so we take sizeof(float) and width=2
  memsetParams.elementSize = sizeof(float);
  memsetParams.width = 2;
  memsetParams.height = 1;

  checkCudaErrors(hipGraphCreate(&graph, 0));
  checkCudaErrors(
      hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams));
  nodeDependencies.push_back(memsetNode);

  hipKernelNodeParams NodeParams0, NodeParams1;
  NodeParams0.func = (void *)JacobiMethod;
  NodeParams0.gridDim = nblocks;
  NodeParams0.blockDim = nthreads;
  NodeParams0.sharedMemBytes = 0;
  void *kernelArgs0[6] = {(void *)&A, (void *)&b,     (void *)&conv_threshold,
                          (void *)&x, (void *)&x_new, (void *)&d_sum};
  NodeParams0.kernelParams = kernelArgs0;
  NodeParams0.extra = NULL;

  checkCudaErrors(
      hipGraphAddKernelNode(&jacobiKernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &NodeParams0));

  nodeDependencies.clear();
  nodeDependencies.push_back(jacobiKernelNode);

  memcpyParams.srcArray = NULL;
  memcpyParams.srcPos = make_hipPos(0, 0, 0);
  memcpyParams.srcPtr = make_hipPitchedPtr(d_sum, sizeof(double), 1, 1);
  memcpyParams.dstArray = NULL;
  memcpyParams.dstPos = make_hipPos(0, 0, 0);
  memcpyParams.dstPtr = make_hipPitchedPtr(&sum, sizeof(double), 1, 1);
  memcpyParams.extent = make_hipExtent(sizeof(double), 1, 1);
  memcpyParams.kind = hipMemcpyDeviceToHost;

  checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &memcpyParams));

  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

  NodeParams1.func = (void *)JacobiMethod;
  NodeParams1.gridDim = nblocks;
  NodeParams1.blockDim = nthreads;
  NodeParams1.sharedMemBytes = 0;
  void *kernelArgs1[6] = {(void *)&A,     (void *)&b, (void *)&conv_threshold,
                          (void *)&x_new, (void *)&x, (void *)&d_sum};
  NodeParams1.kernelParams = kernelArgs1;
  NodeParams1.extra = NULL;

  int k = 0;
  for (k = 0; k < max_iter; k++) {
    checkCudaErrors(hipGraphExecKernelNodeSetParams(
        graphExec, jacobiKernelNode,
        ((k & 1) == 0) ? &NodeParams0 : &NodeParams1));
    checkCudaErrors(hipGraphLaunch(graphExec, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    if (sum <= conv_threshold) {
      checkCudaErrors(hipMemsetAsync(d_sum, 0, sizeof(double), stream));
      nblocks.x = (N_ROWS / nthreads.x) + 1;
      size_t sharedMemSize = ((nthreads.x / 32) + 1) * sizeof(double);
      if ((k & 1) == 0) {
        finalError<<<nblocks, nthreads, sharedMemSize, stream>>>(x_new, d_sum);
      } else {
        finalError<<<nblocks, nthreads, sharedMemSize, stream>>>(x, d_sum);
      }

      checkCudaErrors(hipMemcpyAsync(&sum, d_sum, sizeof(double),
                                      hipMemcpyDeviceToHost, stream));
      checkCudaErrors(hipStreamSynchronize(stream));
      printf("GPU iterations : %d\n", k + 1);
      printf("GPU error : %.3e\n", sum);
      break;
    }
  }

  checkCudaErrors(hipFree(d_sum));
  return sum;
}

double JacobiMethodGpuCudaGraphExecUpdate(const float *A, const double *b,
                                          const float conv_threshold,
                                          const int max_iter, double *x,
                                          double *x_new, hipStream_t stream) {
  // CTA size
  dim3 nthreads(256, 1, 1);
  // grid size
  dim3 nblocks((N_ROWS / ROWS_PER_CTA) + 2, 1, 1);
  hipGraph_t graph;
  hipGraphExec_t graphExec = NULL;

  double sum = 0.0;
  double *d_sum;
  checkCudaErrors(hipMalloc(&d_sum, sizeof(double)));

  int k = 0;
  for (k = 0; k < max_iter; k++) {
    checkCudaErrors(
        hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
    checkCudaErrors(hipMemsetAsync(d_sum, 0, sizeof(double), stream));
    if ((k & 1) == 0) {
      JacobiMethod<<<nblocks, nthreads, 0, stream>>>(A, b, conv_threshold, x,
                                                     x_new, d_sum);
    } else {
      JacobiMethod<<<nblocks, nthreads, 0, stream>>>(A, b, conv_threshold,
                                                     x_new, x, d_sum);
    }
    checkCudaErrors(hipMemcpyAsync(&sum, d_sum, sizeof(double),
                                    hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamEndCapture(stream, &graph));

    if (graphExec == NULL) {
      checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
    } else {
      hipGraphExecUpdateResult updateResult_out;
      checkCudaErrors(
          hipGraphExecUpdate(graphExec, graph, NULL, &updateResult_out));
      if (updateResult_out != hipGraphExecUpdateSuccess) {
        if (graphExec != NULL) {
          checkCudaErrors(hipGraphExecDestroy(graphExec));
        }
        printf("k = %d graph update failed with error - %d\n", k,
               updateResult_out);
        checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));
      }
    }
    checkCudaErrors(hipGraphLaunch(graphExec, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    if (sum <= conv_threshold) {
      checkCudaErrors(hipMemsetAsync(d_sum, 0, sizeof(double), stream));
      nblocks.x = (N_ROWS / nthreads.x) + 1;
      size_t sharedMemSize = ((nthreads.x / 32) + 1) * sizeof(double);
      if ((k & 1) == 0) {
        finalError<<<nblocks, nthreads, sharedMemSize, stream>>>(x_new, d_sum);
      } else {
        finalError<<<nblocks, nthreads, sharedMemSize, stream>>>(x, d_sum);
      }

      checkCudaErrors(hipMemcpyAsync(&sum, d_sum, sizeof(double),
                                      hipMemcpyDeviceToHost, stream));
      checkCudaErrors(hipStreamSynchronize(stream));
      printf("GPU iterations : %d\n", k + 1);
      printf("GPU error : %.3e\n", sum);
      break;
    }
  }

  checkCudaErrors(hipFree(d_sum));
  return sum;
}

double JacobiMethodGpu(const float *A, const double *b,
                       const float conv_threshold, const int max_iter,
                       double *x, double *x_new, hipStream_t stream) {
  // CTA size
  dim3 nthreads(256, 1, 1);
  // grid size
  dim3 nblocks((N_ROWS / ROWS_PER_CTA) + 2, 1, 1);

  double sum = 0.0;
  double *d_sum;
  checkCudaErrors(hipMalloc(&d_sum, sizeof(double)));
  int k = 0;

  for (k = 0; k < max_iter; k++) {
    checkCudaErrors(hipMemsetAsync(d_sum, 0, sizeof(double), stream));
    if ((k & 1) == 0) {
      JacobiMethod<<<nblocks, nthreads, 0, stream>>>(A, b, conv_threshold, x,
                                                     x_new, d_sum);
    } else {
      JacobiMethod<<<nblocks, nthreads, 0, stream>>>(A, b, conv_threshold,
                                                     x_new, x, d_sum);
    }
    checkCudaErrors(hipMemcpyAsync(&sum, d_sum, sizeof(double),
                                    hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipStreamSynchronize(stream));

    if (sum <= conv_threshold) {
      checkCudaErrors(hipMemsetAsync(d_sum, 0, sizeof(double), stream));
      nblocks.x = (N_ROWS / nthreads.x) + 1;
      size_t sharedMemSize = ((nthreads.x / 32) + 1) * sizeof(double);
      if ((k & 1) == 0) {
        finalError<<<nblocks, nthreads, sharedMemSize, stream>>>(x_new, d_sum);
      } else {
        finalError<<<nblocks, nthreads, sharedMemSize, stream>>>(x, d_sum);
      }

      checkCudaErrors(hipMemcpyAsync(&sum, d_sum, sizeof(double),
                                      hipMemcpyDeviceToHost, stream));
      checkCudaErrors(hipStreamSynchronize(stream));
      printf("GPU iterations : %d\n", k + 1);
      printf("GPU error : %.3e\n", sum);
      break;
    }
  }

  checkCudaErrors(hipFree(d_sum));
  return sum;
}
