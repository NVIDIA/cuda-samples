#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// std::system includes
#include <cstdio>

// CUDA-C includes
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#define TOTAL_SIZE 256 * 1024 * 1024
#define EACH_SIZE 128 * 1024 * 1024

// # threadblocks
#define TBLOCKS 1024
#define THREADS 512

// throw error on equality
#define ERR_EQ(X, Y)                                                           \
  do {                                                                         \
    if ((X) == (Y)) {                                                          \
      fprintf(stderr, "Error in %s at %s:%d\n", __func__, __FILE__, __LINE__); \
      exit(-1);                                                                \
    }                                                                          \
  } while (0)

// throw error on difference
#define ERR_NE(X, Y)                                                           \
  do {                                                                         \
    if ((X) != (Y)) {                                                          \
      fprintf(stderr, "Error in %s at %s:%d\n", __func__, __FILE__, __LINE__); \
      exit(-1);                                                                \
    }                                                                          \
  } while (0)

// copy from source -> destination arrays
__global__ void memcpy_kernel(int *dst, int *src, size_t n) {
  int num = gridDim.x * blockDim.x;
  int id = blockDim.x * blockIdx.x + threadIdx.x;

  for (int i = id; i < n / sizeof(int); i += num) {
    dst[i] = src[i];
  }
}

// initialise memory
void mem_init(int *buf, size_t n) {
  for (int i = 0; i < n / sizeof(int); i++) {
    buf[i] = i;
  }
}

int main(int argc, char **argv) {
  hipDeviceProp_t device_prop;
  int dev_id;

  printf("Starting [%s]...\n", argv[0]);

  // set device
  dev_id = findCudaDevice(argc, (const char **)argv);
  checkCudaErrors(hipGetDeviceProperties(&device_prop, dev_id));

  if ((device_prop.major << 4) + device_prop.minor < 0x35) {
    fprintf(stderr,
            "%s requires Compute Capability of SM 3.5 or higher to "
            "run.\nexiting...\n",
            argv[0]);
    exit(EXIT_WAIVED);
  }

  // get the range of priorities available
  // [ greatest_priority, lowest_priority ]
  int priority_low;
  int priority_hi;
  checkCudaErrors(
      hipDeviceGetStreamPriorityRange(&priority_low, &priority_hi));

  printf("CUDA stream priority range: LOW: %d to HIGH: %d\n", priority_low,
         priority_hi);

  // create streams with highest and lowest available priorities
  hipStream_t st_low;
  hipStream_t st_hi;
  checkCudaErrors(hipStreamCreateWithPriority(&st_low, hipStreamNonBlocking,
                                               priority_low));
  checkCudaErrors(
      hipStreamCreateWithPriority(&st_hi, hipStreamNonBlocking, priority_hi));

  size_t size;
  size = TOTAL_SIZE;

  // initialise host data
  int *h_src_low;
  int *h_src_hi;
  ERR_EQ(h_src_low = (int *)malloc(size), NULL);
  ERR_EQ(h_src_hi = (int *)malloc(size), NULL);
  mem_init(h_src_low, size);
  mem_init(h_src_hi, size);

  // initialise device data
  int *h_dst_low;
  int *h_dst_hi;
  ERR_EQ(h_dst_low = (int *)malloc(size), NULL);
  ERR_EQ(h_dst_hi = (int *)malloc(size), NULL);
  memset(h_dst_low, 0, size);
  memset(h_dst_hi, 0, size);

  // copy source data -> device
  int *d_src_low;
  int *d_src_hi;
  checkCudaErrors(hipMalloc(&d_src_low, size));
  checkCudaErrors(hipMalloc(&d_src_hi, size));
  checkCudaErrors(
      hipMemcpy(d_src_low, h_src_low, size, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_src_hi, h_src_hi, size, hipMemcpyHostToDevice));

  // allocate memory for memcopy destination
  int *d_dst_low;
  int *d_dst_hi;
  checkCudaErrors(hipMalloc(&d_dst_low, size));
  checkCudaErrors(hipMalloc(&d_dst_hi, size));

  // create some events
  hipEvent_t ev_start_low;
  hipEvent_t ev_start_hi;
  hipEvent_t ev_end_low;
  hipEvent_t ev_end_hi;
  checkCudaErrors(hipEventCreate(&ev_start_low));
  checkCudaErrors(hipEventCreate(&ev_start_hi));
  checkCudaErrors(hipEventCreate(&ev_end_low));
  checkCudaErrors(hipEventCreate(&ev_end_hi));

  /* */

  // call pair of kernels repeatedly (with different priority streams)
  checkCudaErrors(hipEventRecord(ev_start_low, st_low));
  checkCudaErrors(hipEventRecord(ev_start_hi, st_hi));

  for (int i = 0; i < TOTAL_SIZE; i += EACH_SIZE) {
    int j = i / sizeof(int);
    memcpy_kernel<<<TBLOCKS, THREADS, 0, st_low>>>(d_dst_low + j, d_src_low + j,
                                                   EACH_SIZE);
    memcpy_kernel<<<TBLOCKS, THREADS, 0, st_hi>>>(d_dst_hi + j, d_src_hi + j,
                                                  EACH_SIZE);
  }

  checkCudaErrors(hipEventRecord(ev_end_low, st_low));
  checkCudaErrors(hipEventRecord(ev_end_hi, st_hi));

  checkCudaErrors(hipEventSynchronize(ev_end_low));
  checkCudaErrors(hipEventSynchronize(ev_end_hi));

  /* */

  size = TOTAL_SIZE;
  checkCudaErrors(
      hipMemcpy(h_dst_low, d_dst_low, size, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_dst_hi, d_dst_hi, size, hipMemcpyDeviceToHost));

  // check results of kernels
  ERR_NE(memcmp(h_dst_low, h_src_low, size), 0);
  ERR_NE(memcmp(h_dst_hi, h_src_hi, size), 0);

  // check timings
  float ms_low;
  float ms_hi;
  checkCudaErrors(hipEventElapsedTime(&ms_low, ev_start_low, ev_end_low));
  checkCudaErrors(hipEventElapsedTime(&ms_hi, ev_start_hi, ev_end_hi));

  printf("elapsed time of kernels launched to LOW priority stream: %.3lf ms\n",
         ms_low);
  printf("elapsed time of kernels launched to HI  priority stream: %.3lf ms\n",
         ms_hi);

  exit(EXIT_SUCCESS);
}
