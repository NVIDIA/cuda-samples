#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#include <hip/hip_runtime.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

#define NUM_ELEMS 10000000
#define NUM_THREADS_PER_BLOCK 512

// warp-aggregated atomic increment
__device__ int atomicAggInc(int *counter) {
  cg::coalesced_group active = cg::coalesced_threads();

  // leader does the update
  int res = 0;
  if (active.thread_rank() == 0) {
    res = atomicAdd(counter, active.size());
  }

  // broadcast result
  res = active.shfl(res, 0);

  // each thread computes its own value
  return res + active.thread_rank();
}

__global__ void filter_arr(int *dst, int *nres, const int *src, int n) {
  int id = threadIdx.x + blockIdx.x * blockDim.x;

  for (int i = id; i < n; i += gridDim.x * blockDim.x) {
    if (src[i] > 0) dst[atomicAggInc(nres)] = src[i];
  }
}

// warp-aggregated atomic multi bucket increment
#if __CUDA_ARCH__ >= 700
__device__ int atomicAggIncMulti(const int bucket, int *counter)
{
  cg::coalesced_group active = cg::coalesced_threads();
  // group all threads with same bucket value.
  auto labeledGroup = cg::labeled_partition(active, bucket);

  int res = 0;
  if (labeledGroup.thread_rank() == 0)
  {
    res = atomicAdd(&counter[bucket], labeledGroup.size());
  }

  // broadcast result
  res = labeledGroup.shfl(res, 0);

  // each thread computes its own value
  return res + labeledGroup.thread_rank();
}
#endif

// Places individual value indices into its corresponding buckets.
__global__ void mapToBuckets(const int *srcArr, int *indicesBuckets, int *bucketCounters, const int srcSize, const int numOfBuckets)
{
#if __CUDA_ARCH__ >= 700
  cg::grid_group grid = cg::this_grid();

  for (int i=grid.thread_rank(); i < srcSize; i += grid.size())
  {
    const int bucket = srcArr[i];
    if (bucket < numOfBuckets)
    {
      indicesBuckets[atomicAggIncMulti(bucket, bucketCounters)] = i;
    }
  }
#endif
}

int mapIndicesToBuckets(int *h_srcArr, int *d_srcArr, int numOfBuckets)
{
  int *d_indicesBuckets, *d_bucketCounters;
  int *cpuBucketCounters = new int[numOfBuckets];
  int *h_bucketCounters = new int[numOfBuckets];

  memset(cpuBucketCounters, 0, sizeof(int)*numOfBuckets);
  // Initialize each bucket counters.
  for (int i = 0; i < numOfBuckets; i++)
  {
    h_bucketCounters[i] = i*NUM_ELEMS;
  }

  checkCudaErrors(hipMalloc(&d_indicesBuckets, sizeof(int) * NUM_ELEMS * numOfBuckets));
  checkCudaErrors(hipMalloc(&d_bucketCounters, sizeof(int) * numOfBuckets));

  checkCudaErrors(hipMemcpy(d_bucketCounters, h_bucketCounters, sizeof(int)*numOfBuckets, hipMemcpyHostToDevice));

  dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1, 1);
  dim3 dimGrid((NUM_ELEMS / NUM_THREADS_PER_BLOCK), 1, 1);

  mapToBuckets<<<dimGrid, dimBlock>>>(d_srcArr, d_indicesBuckets, d_bucketCounters, NUM_ELEMS, numOfBuckets);

  checkCudaErrors(hipMemcpy(h_bucketCounters, d_bucketCounters, sizeof(int)*numOfBuckets, hipMemcpyDeviceToHost));

  for (int i=0; i < NUM_ELEMS; i++)
  {
    cpuBucketCounters[h_srcArr[i]]++;
  }

  bool allMatch = true;
  int finalElems = 0;
  for (int i=0; i < numOfBuckets; i++)
  {
    finalElems += (h_bucketCounters[i] - i*NUM_ELEMS);
    if (cpuBucketCounters[i] != (h_bucketCounters[i] - i*NUM_ELEMS))
    {
      allMatch = false;
      break;
    }
  }

  if (!allMatch && finalElems != NUM_ELEMS)
  {
      return EXIT_FAILURE;
  }
  return EXIT_SUCCESS;
}

// Warp-aggregated atomic Max in multi bucket
#if __CUDA_ARCH__ >= 700
__device__ void atomicAggMaxMulti(const int bucket, int *counter, const int valueForMax)
{
  cg::coalesced_group active = cg::coalesced_threads();
  // group all threads with same bucket value.
  auto labeledGroup = cg::labeled_partition(active, bucket);

  const int maxValueInGroup = cg::reduce(labeledGroup, valueForMax, cg::greater<int>());

  if (labeledGroup.thread_rank() == 0)
  {
    atomicMax(&counter[bucket], maxValueInGroup);
  }
}
#endif

// Performs max calculation in each buckets.
__global__ void calculateMaxInEachBuckets(const int *srcArr, const int *valueInBuckets, int *bucketsMax, const int srcSize, const int numOfBuckets)
{
#if __CUDA_ARCH__ >= 700
  cg::grid_group grid = cg::this_grid();

  for (int i=grid.thread_rank(); i < srcSize; i += grid.size())
  {
    const int bucket = srcArr[i];
    if (bucket < numOfBuckets)
    {
      atomicAggMaxMulti(bucket, bucketsMax, valueInBuckets[i]);
    }
  }
#endif
}

int calculateMaxInBuckets(int *h_srcArr, int *d_srcArr, int numOfBuckets)
{
  int *d_valueInBuckets, *d_bucketsMax;
  int *h_valueInBuckets = new int[NUM_ELEMS];
  int *cpuBucketsMax    = new int[numOfBuckets];
  int *h_bucketsMax     = new int[numOfBuckets];

  memset(cpuBucketsMax, 0, sizeof(int) * numOfBuckets);

  // Here we create values which is assumed to correspond to each 
  // buckets of srcArr at same array index.
  for (int i=0; i < NUM_ELEMS; i++)
  {
    h_valueInBuckets[i] = rand();
  }

  checkCudaErrors(hipMalloc(&d_valueInBuckets, sizeof(int) * NUM_ELEMS));
  checkCudaErrors(hipMalloc(&d_bucketsMax, sizeof(int) * numOfBuckets));

  checkCudaErrors(hipMemset(d_bucketsMax, 0, sizeof(int) * numOfBuckets));
  checkCudaErrors(hipMemcpy(d_valueInBuckets, h_valueInBuckets, sizeof(int) * NUM_ELEMS, hipMemcpyHostToDevice));

  dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1, 1);
  dim3 dimGrid((NUM_ELEMS / NUM_THREADS_PER_BLOCK), 1, 1);

  calculateMaxInEachBuckets<<<dimGrid, dimBlock>>>(d_srcArr, d_valueInBuckets, d_bucketsMax, NUM_ELEMS, numOfBuckets);

  checkCudaErrors(hipMemcpy(h_bucketsMax, d_bucketsMax, sizeof(int) * numOfBuckets, hipMemcpyDeviceToHost));

  for (int i = 0; i < NUM_ELEMS; i++)
  {
    if (cpuBucketsMax[h_srcArr[i]] < h_valueInBuckets[i])
    {
      cpuBucketsMax[h_srcArr[i]] = h_valueInBuckets[i];
    }
  }

  bool allMatch = true;
  int finalElems = 0;
  for (int i=0; i < numOfBuckets; i++)
  {
    if (cpuBucketsMax[i] != h_bucketsMax[i])
    {
      allMatch = false;
      printf("CPU i=%d  max = %d mismatches GPU max = %d\n", i, cpuBucketsMax[i], h_bucketsMax[i]);
      break;
    }
  }
  if (allMatch)
  {
    printf("CPU max matches GPU max\n"); 
  }

  delete[] h_valueInBuckets;
  delete[] cpuBucketsMax;
  delete[] h_bucketsMax;
  checkCudaErrors(hipFree(d_valueInBuckets));
  checkCudaErrors(hipFree(d_bucketsMax));

  if (!allMatch && finalElems != NUM_ELEMS)
  {
      return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}

int main(int argc, char **argv) {
  int *data_to_filter, *filtered_data, nres = 0;
  int *d_data_to_filter, *d_filtered_data, *d_nres;

  int numOfBuckets = 5;

  data_to_filter = reinterpret_cast<int *>(malloc(sizeof(int) * NUM_ELEMS));

  // Generate input data.
  for (int i = 0; i < NUM_ELEMS; i++) {
    data_to_filter[i] = rand() % numOfBuckets;
  }

  int devId = findCudaDevice(argc, (const char **)argv);

  checkCudaErrors(hipMalloc(&d_data_to_filter, sizeof(int) * NUM_ELEMS));
  checkCudaErrors(hipMalloc(&d_filtered_data, sizeof(int) * NUM_ELEMS));
  checkCudaErrors(hipMalloc(&d_nres, sizeof(int)));

  checkCudaErrors(hipMemcpy(d_data_to_filter, data_to_filter,
                             sizeof(int) * NUM_ELEMS, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemset(d_nres, 0, sizeof(int)));

  dim3 dimBlock(NUM_THREADS_PER_BLOCK, 1, 1);
  dim3 dimGrid((NUM_ELEMS / NUM_THREADS_PER_BLOCK) + 1, 1, 1);

  filter_arr<<<dimGrid, dimBlock>>>(d_filtered_data, d_nres, d_data_to_filter,
                                    NUM_ELEMS);

  checkCudaErrors(
      hipMemcpy(&nres, d_nres, sizeof(int), hipMemcpyDeviceToHost));

  filtered_data = reinterpret_cast<int *>(malloc(sizeof(int) * nres));

  checkCudaErrors(hipMemcpy(filtered_data, d_filtered_data, sizeof(int) * nres,
                             hipMemcpyDeviceToHost));

  int *host_filtered_data =
      reinterpret_cast<int *>(malloc(sizeof(int) * NUM_ELEMS));

  // Generate host output with host filtering code.
  int host_flt_count = 0;
  for (int i = 0; i < NUM_ELEMS; i++) {
    if (data_to_filter[i] > 0) {
      host_filtered_data[host_flt_count++] = data_to_filter[i];
    }
  }

  int major = 0;
  checkCudaErrors(hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, devId));

  int mapIndicesToBucketsStatus = EXIT_SUCCESS;
  int calculateMaxInBucketsStatus = EXIT_SUCCESS;
  // atomicAggIncMulti & atomicAggMaxMulti require a GPU of Volta (SM7X) architecture or higher,
  // so that it can take advantage of the new MATCH capability of Volta hardware
  if (major >= 7) {
    mapIndicesToBucketsStatus = mapIndicesToBuckets(data_to_filter, d_data_to_filter, numOfBuckets);
    calculateMaxInBucketsStatus = calculateMaxInBuckets(data_to_filter, d_data_to_filter, numOfBuckets);
  }

  printf("\nWarp Aggregated Atomics %s \n",
         (host_flt_count == nres) && (mapIndicesToBucketsStatus == EXIT_SUCCESS) && 
         (calculateMaxInBucketsStatus == EXIT_SUCCESS) ? "PASSED" : "FAILED");

  checkCudaErrors(hipFree(d_data_to_filter));
  checkCudaErrors(hipFree(d_filtered_data));
  checkCudaErrors(hipFree(d_nres));
  free(data_to_filter);
  free(filtered_data);
  free(host_filtered_data);
}
