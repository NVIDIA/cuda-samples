#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define MAX_DEPTH 16
#define INSERTION_SORT 32

////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort(unsigned int *data, int left, int right) {
  for (int i = left; i <= right; ++i) {
    unsigned min_val = data[i];
    int min_idx = i;

    // Find the smallest value in the range [left, right].
    for (int j = i + 1; j <= right; ++j) {
      unsigned val_j = data[j];

      if (val_j < min_val) {
        min_idx = j;
        min_val = val_j;
      }
    }

    // Swap the values.
    if (i != min_idx) {
      data[min_idx] = data[i];
      data[i] = min_val;
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(unsigned int *data, int left, int right,
                                     int depth) {
  // If we're too deep or there are few elements left, we use an insertion
  // sort...
  if (depth >= MAX_DEPTH || right - left <= INSERTION_SORT) {
    selection_sort(data, left, right);
    return;
  }

  unsigned int *lptr = data + left;
  unsigned int *rptr = data + right;
  unsigned int pivot = data[(left + right) / 2];

  // Do the partitioning.
  while (lptr <= rptr) {
    // Find the next left- and right-hand values to swap
    unsigned int lval = *lptr;
    unsigned int rval = *rptr;

    // Move the left pointer as long as the pointed element is smaller than the
    // pivot.
    while (lval < pivot) {
      lptr++;
      lval = *lptr;
    }

    // Move the right pointer as long as the pointed element is larger than the
    // pivot.
    while (rval > pivot) {
      rptr--;
      rval = *rptr;
    }

    // If the swap points are valid, do the swap!
    if (lptr <= rptr) {
      *lptr++ = rval;
      *rptr-- = lval;
    }
  }

  // Now the recursive part
  int nright = rptr - data;
  int nleft = lptr - data;

  // Launch a new block to sort the left part.
  if (left < (rptr - data)) {
    hipStream_t s;
    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
    cdp_simple_quicksort<<<1, 1, 0, s>>>(data, left, nright, depth + 1);
    hipStreamDestroy(s);
  }

  // Launch a new block to sort the right part.
  if ((lptr - data) < right) {
    hipStream_t s1;
    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    cdp_simple_quicksort<<<1, 1, 0, s1>>>(data, nleft, right, depth + 1);
    hipStreamDestroy(s1);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Call the quicksort kernel from the host.
////////////////////////////////////////////////////////////////////////////////
void run_qsort(unsigned int *data, unsigned int nitems) {
  // Prepare CDP for the max depth 'MAX_DEPTH'.
  checkCudaErrors(hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH));

  // Launch on device
  int left = 0;
  int right = nitems - 1;
  std::cout << "Launching kernel on the GPU" << std::endl;
  cdp_simple_quicksort<<<1, 1>>>(data, left, right, 0);
  checkCudaErrors(hipDeviceSynchronize());
}

////////////////////////////////////////////////////////////////////////////////
// Initialize data on the host.
////////////////////////////////////////////////////////////////////////////////
void initialize_data(unsigned int *dst, unsigned int nitems) {
  // Fixed seed for illustration
  srand(2047);

  // Fill dst with random values
  for (unsigned i = 0; i < nitems; i++) dst[i] = rand() % nitems;
}

////////////////////////////////////////////////////////////////////////////////
// Verify the results.
////////////////////////////////////////////////////////////////////////////////
void check_results(int n, unsigned int *results_d) {
  unsigned int *results_h = new unsigned[n];
  checkCudaErrors(hipMemcpy(results_h, results_d, n * sizeof(unsigned),
                             hipMemcpyDeviceToHost));

  for (int i = 1; i < n; ++i)
    if (results_h[i - 1] > results_h[i]) {
      std::cout << "Invalid item[" << i - 1 << "]: " << results_h[i - 1]
                << " greater than " << results_h[i] << std::endl;
      exit(EXIT_FAILURE);
    }

  std::cout << "OK" << std::endl;
  delete[] results_h;
}

////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  int num_items = 128;
  bool verbose = false;

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "h")) {
    std::cerr << "Usage: " << argv[0]
              << " num_items=<num_items>\twhere num_items is the number of "
                 "items to sort"
              << std::endl;
    exit(EXIT_SUCCESS);
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "v")) {
    verbose = true;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "num_items")) {
    num_items = getCmdLineArgumentInt(argc, (const char **)argv, "num_items");

    if (num_items < 1) {
      std::cerr << "ERROR: num_items has to be greater than 1" << std::endl;
      exit(EXIT_FAILURE);
    }
  }

  // Find/set device and get device properties
  int device = -1;
  hipDeviceProp_t deviceProp;
  device = findCudaDevice(argc, (const char **)argv);
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));

  if (!(deviceProp.major > 3 ||
        (deviceProp.major == 3 && deviceProp.minor >= 5))) {
    printf("GPU %d - %s  does not support CUDA Dynamic Parallelism\n Exiting.",
           device, deviceProp.name);
    exit(EXIT_WAIVED);
  }

  // Create input data
  unsigned int *h_data = 0;
  unsigned int *d_data = 0;

  // Allocate CPU memory and initialize data.
  std::cout << "Initializing data:" << std::endl;
  h_data = (unsigned int *)malloc(num_items * sizeof(unsigned int));
  initialize_data(h_data, num_items);

  if (verbose) {
    for (int i = 0; i < num_items; i++)
      std::cout << "Data [" << i << "]: " << h_data[i] << std::endl;
  }

  // Allocate GPU memory.
  checkCudaErrors(
      hipMalloc((void **)&d_data, num_items * sizeof(unsigned int)));
  checkCudaErrors(hipMemcpy(d_data, h_data, num_items * sizeof(unsigned int),
                             hipMemcpyHostToDevice));

  // Execute
  std::cout << "Running quicksort on " << num_items << " elements" << std::endl;
  run_qsort(d_data, num_items);

  // Check result
  std::cout << "Validating results: ";
  check_results(num_items, d_data);

  free(h_data);
  checkCudaErrors(hipFree(d_data));

  exit(EXIT_SUCCESS);
}
