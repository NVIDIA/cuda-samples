#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <assert.h>
#include <stdio.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define NUM_GRAPHS 8
#define THREADS_PER_BLOCK 512

void printMemoryFootprint(int device) {
  size_t footprint;
  checkCudaErrors(hipDeviceGetGraphMemAttribute(
      device, (hipGraphMemAttributeType)0, &footprint));
  printf("    FOOTPRINT: %lu bytes\n", footprint);
}

void prepareAllocParams(hipMemAllocNodeParams *allocParams, size_t bytes,
                        int device) {
  memset(allocParams, 0, sizeof(*allocParams));

  allocParams->bytesize = bytes;
  allocParams->poolProps.allocType = hipMemAllocationTypePinned;
  allocParams->poolProps.location.id = device;
  allocParams->poolProps.location.type = hipMemLocationTypeDevice;
}

void createVirtAddrReuseGraph(hipGraphExec_t *graphExec, size_t bytes,
                              int device) {
  hipGraph_t graph;
  hipGraphNode_t allocNodeA, allocNodeB, freeNodeA, freeNodeB;
  hipMemAllocNodeParams allocParams;
  float *d_a, *d_b;

  checkCudaErrors(hipGraphCreate(&graph, 0));
  prepareAllocParams(&allocParams, bytes, device);

  checkCudaErrors(
      hipGraphAddMemAllocNode(&allocNodeA, graph, NULL, 0, &allocParams));
  d_a = (float *)allocParams.dptr;
  checkCudaErrors(
      hipGraphAddMemFreeNode(&freeNodeA, graph, &allocNodeA, 1, (void *)d_a));

  // The dependency between the allocation of d_b and the free of d_a allows d_b
  // to reuse the same VA.
  checkCudaErrors(hipGraphAddMemAllocNode(&allocNodeB, graph, &freeNodeA, 1,
                                           &allocParams));
  d_b = (float *)allocParams.dptr;

  if (d_a == d_b) {
    printf("Check confirms that d_a and d_b share a virtual address.\n");
  } else {
    printf("Check shows that d_a and d_b DO NOT share a virtual address.\n");
  }

  checkCudaErrors(
      hipGraphAddMemFreeNode(&freeNodeB, graph, &allocNodeB, 1, (void *)d_b));

  checkCudaErrors(hipGraphInstantiate(graphExec, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphDestroy(graph));
}

void virtualAddressReuseSingleGraph(size_t bytes, int device) {
  hipStream_t stream;
  hipGraphExec_t graphExec;

  printf("================================\n");
  printf("Running virtual address reuse example.\n");
  printf(
      "Sequential allocations & frees within a single graph enable CUDA to "
      "reuse virtual addresses.\n\n");

  createVirtAddrReuseGraph(&graphExec, bytes, device);
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));
  printMemoryFootprint(device);

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipStreamDestroy(stream));
}

// This is a kernel that does no real work but runs at least for a specified
// number of clocks
__global__ void clockBlock(clock_t clock_count) {
  unsigned int start_clock = (unsigned int)clock();

  clock_t clock_offset = 0;

  while (clock_offset < clock_count) {
    unsigned int end_clock = (unsigned int)clock();

    // The code below should work like
    // this (thanks to modular arithmetics):
    //
    // clock_offset = (clock_t) (end_clock > start_clock ?
    //                           end_clock - start_clock :
    //                           end_clock + (0xffffffffu - start_clock));
    //
    // Indeed, let m = 2^32 then
    // end - start = end + m - start (mod m).

    clock_offset = (clock_t)(end_clock - start_clock);
  }
}

// A pointer to the allocated device buffer is returned in dPtr so the caller
// can compare virtual addresses. The kernel node is added to increase the
// graph's runtime.
void createSimpleAllocFreeGraph(hipGraphExec_t *graphExec, float **dPtr,
                                size_t bytes, int device) {
  hipGraph_t graph;
  hipGraphNode_t allocNodeA, freeNodeA, blockDeviceNode;
  hipMemAllocNodeParams allocParams;
  hipKernelNodeParams blockDeviceNodeParams = {0};
  int numElements = bytes / sizeof(float);
  float kernelTime = 5;  // time for each thread to run in microseconds

  checkCudaErrors(hipGraphCreate(&graph, 0));
  prepareAllocParams(&allocParams, bytes, device);

  checkCudaErrors(
      hipGraphAddMemAllocNode(&allocNodeA, graph, NULL, 0, &allocParams));
  *dPtr = (float *)allocParams.dptr;

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, device));
  clock_t time_clocks = (clock_t)((kernelTime / 1000.0) * deviceProp.clockRate);

  void *blockDeviceArgs[1] = {(void *)&time_clocks};

  size_t numBlocks = numElements / (size_t)THREADS_PER_BLOCK;
  blockDeviceNodeParams.gridDim = dim3(numBlocks, 1, 1);
  blockDeviceNodeParams.blockDim = dim3(THREADS_PER_BLOCK, 1, 1);
  blockDeviceNodeParams.sharedMemBytes = 0;
  blockDeviceNodeParams.extra = NULL;
  blockDeviceNodeParams.func = (void *)clockBlock;
  blockDeviceNodeParams.kernelParams = (void **)blockDeviceArgs;
  checkCudaErrors(hipGraphAddKernelNode(&blockDeviceNode, graph, &allocNodeA,
                                         1, &blockDeviceNodeParams));

  checkCudaErrors(hipGraphAddMemFreeNode(&freeNodeA, graph, &blockDeviceNode,
                                          1, (void *)*dPtr));

  checkCudaErrors(hipGraphInstantiate(graphExec, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphDestroy(graph));
}

void physicalMemoryReuseSingleStream(size_t bytes, int device) {
  hipStream_t stream;
  hipGraphExec_t graphExecs[NUM_GRAPHS];
  float *dPtrs[NUM_GRAPHS];
  bool virtualAddrDiffer = true;

  printf("================================\n");
  printf("Running physical memory reuse example.\n");
  printf(
      "CUDA reuses the same physical memory for allocations from separate "
      "graphs when the allocation lifetimes don't overlap.\n\n");

  for (int i = 0; i < NUM_GRAPHS; i++) {
    createSimpleAllocFreeGraph(&graphExecs[i], &dPtrs[i], bytes, device);
  }

  printf("Creating the graph execs does not reserve any physical memory.\n");
  printMemoryFootprint(device);

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  checkCudaErrors(hipGraphLaunch(graphExecs[0], stream));
  printf("\nThe first graph launched reserves the memory it needs.\n");
  printMemoryFootprint(device);

  checkCudaErrors(hipGraphLaunch(graphExecs[0], stream));
  printf(
      "A subsequent launch of the same graph in the same stream reuses the "
      "same physical memory. ");
  printf("Thus the memory footprint does not grow here.\n");
  printMemoryFootprint(device);

  printf(
      "\nSubsequent launches of other graphs in the same stream also reuse the "
      "physical memory. ");
  printf("Thus the memory footprint does not grow here.\n");
  for (int i = 1; i < NUM_GRAPHS; i++) {
    checkCudaErrors(hipGraphLaunch(graphExecs[i], stream));
    printf("%02d: ", i);
    printMemoryFootprint(device);
  }

  checkCudaErrors(hipStreamSynchronize(stream));

  for (int i = 0; i < NUM_GRAPHS; i++) {
    for (int j = i + 1; j < NUM_GRAPHS; j++) {
      if (dPtrs[i] == dPtrs[j]) {
        virtualAddrDiffer = false;
        printf("Error: Graph exec %d and %d have the same virtual address!\n",
               i - 1, i);
      }
    }
    checkCudaErrors(hipGraphExecDestroy(graphExecs[i]));
  }
  if (virtualAddrDiffer) {
    printf("\nCheck confirms all graphs use a different virtual address.\n");
  } else {
    printf(
        "\nAll graphs do NOT use different virtual addresses. Exiting test.\n");
    exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipStreamDestroy(stream));
}

void simultaneousStreams(size_t bytes, int device) {
  hipStream_t streams[NUM_GRAPHS];
  hipGraphExec_t graphExecs[NUM_GRAPHS];
  float *dPtrs[NUM_GRAPHS];

  printf("================================\n");
  printf("Running simultaneous streams example.\n");
  printf("Graphs that can run concurrently need separate physical memory. ");
  printf(
      "In this example, each graph launched in a separate stream increases the "
      "total memory footprint.\n\n");

  printf(
      "When launching a new graph, CUDA may reuse physical memory from a graph "
      "whose execution has already ");
  printf(
      "finished -- even if the new graph is being launched in a different "
      "stream from the completed graph. ");
  printf(
      "Therefore, a kernel node is added to the graphs to increase "
      "runtime.\n\n");

  for (int i = 0; i < NUM_GRAPHS; i++) {
    createSimpleAllocFreeGraph(&graphExecs[i], &dPtrs[i], bytes, device);
    checkCudaErrors(
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));
  }

  printf("Initial footprint:\n");
  printMemoryFootprint(device);

  printf(
      "\nEach graph launch in a seperate stream grows the memory footprint:\n");
  for (int i = 1; i < NUM_GRAPHS; i++) {
    checkCudaErrors(hipGraphLaunch(graphExecs[i], streams[i]));
    printf("%02d: ", i);
    printMemoryFootprint(device);
  }

  for (int i = 0; i < NUM_GRAPHS; i++) {
    checkCudaErrors(hipStreamSynchronize(streams[i]));
    checkCudaErrors(hipGraphExecDestroy(graphExecs[i]));
    checkCudaErrors(hipStreamDestroy(streams[i]));
  }
}

void createSimpleAllocNoFreeGraph(hipGraphExec_t *graphExec, float **dPtr,
                                  size_t bytes, int device) {
  hipGraph_t graph;
  hipGraphNode_t allocNodeA;
  hipMemAllocNodeParams allocParams;

  checkCudaErrors(hipGraphCreate(&graph, 0));
  prepareAllocParams(&allocParams, bytes, device);

  checkCudaErrors(
      hipGraphAddMemAllocNode(&allocNodeA, graph, NULL, 0, &allocParams));
  *dPtr = (float *)allocParams.dptr;

  checkCudaErrors(hipGraphInstantiate(graphExec, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphDestroy(graph));
}

void unfreedAllocations(size_t bytes, int device) {
  hipStream_t stream;
  hipGraphExec_t graphExecs[NUM_GRAPHS];
  float *dPtrs[NUM_GRAPHS];

  printf("================================\n");
  printf("Running unfreed streams example.\n");
  printf(
      "CUDA cannot reuse phyiscal memory from graphs which do not free their "
      "allocations.\n\n");

  for (int i = 0; i < NUM_GRAPHS; i++) {
    createSimpleAllocNoFreeGraph(&graphExecs[i], &dPtrs[i], bytes, device);
  }

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  printf(
      "Despite being launched in the same stream, each graph launch grows the "
      "memory footprint. ");
  printf(
      "Since the allocation is not freed, CUDA keeps the memory valid for "
      "use.\n");
  for (int i = 0; i < NUM_GRAPHS; i++) {
    checkCudaErrors(hipGraphLaunch(graphExecs[i], stream));
    printf("%02d: ", i);
    printMemoryFootprint(device);
  }

  checkCudaErrors(hipStreamSynchronize(stream));

  checkCudaErrors(hipDeviceGraphMemTrim(device));
  printf(
      "\nTrimming does not impact the memory footprint since the un-freed "
      "allocations are still holding onto the memory.\n");
  printMemoryFootprint(device);

  for (int i = 0; i < NUM_GRAPHS; i++) {
    checkCudaErrors(hipFree(dPtrs[i]));
  }
  printf("\nFreeing the allocations does not shrink the footprint.\n");
  printMemoryFootprint(device);

  checkCudaErrors(hipDeviceGraphMemTrim(device));
  printf(
      "\nSince the allocations are now freed, trimming does reduce the "
      "footprint even when the graph execs are not yet destroyed.\n");
  printMemoryFootprint(device);

  for (int i = 0; i < NUM_GRAPHS; i++) {
    checkCudaErrors(hipGraphExecDestroy(graphExecs[i]));
  }
  checkCudaErrors(hipStreamDestroy(stream));
}

void cleanupMemory(int device) {
  checkCudaErrors(hipDeviceGraphMemTrim(device));
  printf("\nCleaning up example by trimming device memory.\n");
  printMemoryFootprint(device);
  printf("\n");
}

int main(int argc, char **argv) {
  size_t bytes = 64 * 1024 * 1024;
  int device = findCudaDevice(argc, (const char **)argv);

  int driverVersion = 0;
  int deviceSupportsMemoryPools = 0;

  hipDriverGetVersion(&driverVersion);
  printf("Driver version is: %d.%d\n", driverVersion / 1000,
         (driverVersion % 100) / 10);

  if (driverVersion < 11040) {
    printf("Waiving execution as driver does not support Graph Memory Nodes\n");
    exit(EXIT_WAIVED);
  }

  hipDeviceGetAttribute(&deviceSupportsMemoryPools,
                         hipDeviceAttributeMemoryPoolsSupported, device);
  if (!deviceSupportsMemoryPools) {
    printf("Waiving execution as device does not support Memory Pools\n");
    exit(EXIT_WAIVED);
  } else {
    printf("Running sample.\n");
  }

  virtualAddressReuseSingleGraph(bytes, device);
  cleanupMemory(device);

  physicalMemoryReuseSingleStream(bytes, device);
  cleanupMemory(device);

  simultaneousStreams(bytes, device);
  cleanupMemory(device);

  unfreedAllocations(bytes, device);
  cleanupMemory(device);

  printf("================================\n");
  printf("Sample complete.\n");
}
