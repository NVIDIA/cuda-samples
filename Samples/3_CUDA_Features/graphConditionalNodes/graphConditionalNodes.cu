#include "hip/hip_runtime.h"
/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This file demonstrates the usage of conditional graph nodes with
 * a series of *simple* example graphs.
 *
 * For more information on conditional nodes, see the programming guide:
 *
 *   https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#conditional-graph-nodes
 *
 */

// System includes
#include <cassert>
#include <cstdio>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

/*
 * Create a graph containing two nodes.
 * The first node, A, is a kernel and the second node, B, is a conditional IF node.
 * The kernel sets the condition variable to true if a device memory location
 * contains an odd number. Otherwise the condition variable is set to false.
 * There is a single kernel, C, within the conditional body which prints a message.
 *
 * A -> B [ C ]
 *
 */

__global__ void ifGraphKernelA(char *dPtr, cudaGraphConditionalHandle handle)
{
    // In this example, condition is set if *dPtr is odd
    unsigned int value = *dPtr & 0x01;
    cudaGraphSetConditional(handle, value);
    printf("GPU: Handle set to %d\n", value);
}

// This kernel will only be executed if the condition is true
__global__ void ifGraphKernelC(void)
{
    printf("GPU: Hello from the GPU! The condition was true.\n");
}

// Setup and launch the graph
void simpleIfGraph(void)
{
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t kernelNode;
    hipGraphNode_t conditionalNode;

    void *kernelArgs[2];

    // Allocate a byte of device memory to use as input
    char *dPtr;
    checkCudaErrors(hipMalloc((void **)&dPtr, 1));

    printf("simpleIfGraph: Building graph...\n");
    hipGraphCreate(&graph, 0);

    // Create conditional handle.
    cudaGraphConditionalHandle handle;
    cudaGraphConditionalHandleCreate(&handle, graph);

    // Use a kernel upstream of the conditional to set the handle value
    hipGraphNodeParams params = {hipGraphNodeTypeKernel};
    params.kernel.func = (void *)ifGraphKernelA;
    params.kernel.gridDim = 1;
    params.kernel.blockDim = 1;
    params.kernel.kernelParams = kernelArgs;
    kernelArgs[0] = &dPtr;
    kernelArgs[1] = &handle;
    checkCudaErrors(hipGraphAddNode(&kernelNode, graph, NULL, 0, &params));

    hipGraphNodeParams cParams = {hipGraphNodeTypeConditional};
    cParams.conditional.handle = handle;
    cParams.conditional.type = cudaGraphCondTypeIf;
    cParams.conditional.size = 1;
    checkCudaErrors(hipGraphAddNode(&conditionalNode, graph, &kernelNode, 1, &cParams));

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    // Populate the body of the conditional node
    hipGraphNode_t bodyNode;
    params.kernel.func = (void *)ifGraphKernelC;
    params.kernel.kernelParams = nullptr;
    checkCudaErrors(hipGraphAddNode(&bodyNode, bodyGraph, NULL, 0, &params));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 0, 1)); // Set dPtr to 0
    printf("Host: Launching graph with device memory set to 0\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 1, 1)); // Set dPtr to 1
    printf("Host: Launching graph with device memory set to 1\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));

    printf("simpleIfGraph: Complete\n\n");
}

/*
 * Create a graph containing a single conditional while node.
 * The default value of the conditional variable is set to true, so this
 * effectively becomes a do-while loop as the conditional body will always
 * execute at least once. The body of the conditional contains 3 kernel nodes:
 * A [ B -> C -> D ]
 * Nodes B and C are just dummy nodes for demonstrative purposes. Node D
 * will decrement a device memory location and set the condition value to false
 * when the value reaches zero, terminating the loop.
 * In this example, stream capture is used to populate the conditional body.
 */

// This kernel will only be executed if the condition is true
__global__ void doWhileEmptyKernel(void)
{
    printf("GPU: doWhileEmptyKernel()\n");
    return;
}

__global__ void doWhileLoopKernel(char *dPtr, cudaGraphConditionalHandle handle)
{
    if (--(*dPtr) == 0)
    {
        cudaGraphSetConditional(handle, 0);
    }
    printf("GPU: counter = %d\n", *dPtr);
}

void simpleDoWhileGraph(void)
{
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t conditionalNode;

    // Allocate a byte of device memory to use as input
    char *dPtr;
    checkCudaErrors(hipMalloc((void **)&dPtr, 1));

    printf("simpleDoWhileGraph: Building graph...\n");
    checkCudaErrors(hipGraphCreate(&graph, 0));

    cudaGraphConditionalHandle handle;
    checkCudaErrors(cudaGraphConditionalHandleCreate(&handle, graph, 1, cudaGraphCondAssignDefault));

    hipGraphNodeParams cParams = {hipGraphNodeTypeConditional};
    cParams.conditional.handle = handle;
    cParams.conditional.type = cudaGraphCondTypeWhile;
    cParams.conditional.size = 1;
    checkCudaErrors(hipGraphAddNode(&conditionalNode, graph, NULL, 0, &cParams));

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    hipStream_t captureStream;
    checkCudaErrors(hipStreamCreate(&captureStream));

    checkCudaErrors(hipStreamBeginCaptureToGraph(captureStream, bodyGraph, nullptr, nullptr, 0, hipStreamCaptureModeGlobal));
    doWhileEmptyKernel<<<1, 1, 0, captureStream>>>();
    doWhileEmptyKernel<<<1, 1, 0, captureStream>>>();
    doWhileLoopKernel<<<1, 1, 0, captureStream>>>(dPtr, handle);
    checkCudaErrors(hipStreamEndCapture(captureStream, nullptr));
    checkCudaErrors(hipStreamDestroy(captureStream));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 10, 1)); // Set dPtr to 10
    printf("Host: Launching graph with loop counter set to 10\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));

    printf("simpleDoWhileGraph: Complete\n\n");
}

/*
 * Create a graph containing a conditional while loop using stream capture.
 * This demonstrates how to insert a conditional node into a stream which is
 * being captured. The graph consists of a kernel node, A, followed by a
 * conditional while node, B, followed by a kernel node, D. The conditional
 * body is populated by a single kernel node, C:
 *
 * A -> B [ C ] -> D
 *
 * The same kernel will be used for both nodes A and C. This kernel will test
 * a device memory location and set the condition when the location is non-zero.
 * We must run the kernel before the loop as well as inside the loop in order
 * to behave like a while loop as opposed to a do-while loop. We need to evaluate
 * the device memory location before the conditional node is evaluated in order
 * to set the condition variable properly. Because we're using a kernel upstream
 * of the conditional node, there is no need to use the handle default value to
 * initialize the conditional value.
 */

__global__ void capturedWhileKernel(char *dPtr, cudaGraphConditionalHandle handle)
{
    printf("GPU: counter = %d\n", *dPtr);
    if (*dPtr)
    {
        (*dPtr)--;
    }
    cudaGraphSetConditional(handle, *dPtr);
}

__global__ void capturedWhileEmptyKernel(void)
{
    printf("GPU: capturedWhileEmptyKernel()\n");
    return;
}

void capturedWhileGraph(void)
{
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    hipStreamCaptureStatus status;
    const hipGraphNode_t *dependencies;
    size_t numDependencies;

    // Allocate a byte of device memory to use as input
    char *dPtr;
    checkCudaErrors(hipMalloc((void **)&dPtr, 1));

    printf("capturedWhileGraph: Building graph...\n");
    hipStream_t captureStream;
    checkCudaErrors(hipStreamCreate(&captureStream));

    checkCudaErrors(hipStreamBeginCapture(captureStream, hipStreamCaptureModeGlobal));

    // Obtain the handle of the graph
    checkCudaErrors(hipStreamGetCaptureInfo(captureStream, &status, NULL, &graph, &dependencies, &numDependencies));

    // Create the conditional handle
    cudaGraphConditionalHandle handle;
    checkCudaErrors(cudaGraphConditionalHandleCreate(&handle, graph));

    // Insert kernel node A
    capturedWhileKernel<<<1, 1, 0, captureStream>>>(dPtr, handle);

    // Obtain the handle for node A
    checkCudaErrors(hipStreamGetCaptureInfo(captureStream, &status, NULL, &graph, &dependencies, &numDependencies));

    // Insert conditional node B
    hipGraphNode_t conditionalNode;
    hipGraphNodeParams cParams = {hipGraphNodeTypeConditional};
    cParams.conditional.handle = handle;
    cParams.conditional.type = cudaGraphCondTypeWhile;
    cParams.conditional.size = 1;
    checkCudaErrors(hipGraphAddNode(&conditionalNode, graph, dependencies, numDependencies, &cParams));

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    // Update stream capture dependencies to account for the node we manually added
    checkCudaErrors(hipStreamUpdateCaptureDependencies(captureStream, &conditionalNode, 1, hipStreamSetCaptureDependencies));

    // Insert kernel node D
    capturedWhileEmptyKernel<<<1, 1, 0, captureStream>>>();

    checkCudaErrors(hipStreamEndCapture(captureStream, &graph));
    checkCudaErrors(hipStreamDestroy(captureStream));

    // Populate conditional body graph using stream capture
    hipStream_t bodyStream;
    checkCudaErrors(hipStreamCreate(&bodyStream));

    checkCudaErrors(hipStreamBeginCaptureToGraph(bodyStream, bodyGraph, nullptr, nullptr, 0, hipStreamCaptureModeGlobal));

    // Insert kernel node C
    capturedWhileKernel<<<1, 1, 0, bodyStream>>>(dPtr, handle);
    checkCudaErrors(hipStreamEndCapture(bodyStream, nullptr));
    checkCudaErrors(hipStreamDestroy(bodyStream));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Initialize device memory and launch the graph
    // Device memory is zero, so the conditional node will not execute
    checkCudaErrors(hipMemset(dPtr, 0, 1)); // Set dPtr to 0
    printf("Host: Launching graph with loop counter set to 0\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 10, 1)); // Set dPtr to 10
    printf("Host: Launching graph with loop counter set to 10\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));

    printf("capturedWhileGraph: Complete\n\n");
}

/*
 * Create a graph containing two nodes.
 * The first node, A, is a kernel and the second node, B, is a conditional IF node containing two graphs.
 * The first graph within the conditional will be executed when the condition is true, while the second
 * graph will be executed when the conditional is false.
 * The kernel sets the condition variable to true if a device memory location
 * contains an odd number. Otherwise the condition variable is set to false.
 * There is a single kernel(C & D) within each conditional body which prints a message.
 *
 * A -> B [ C | D ]
 *
 * This example requires CUDA >= 12.8.
 */

// This kernel will only be executed if the condition is false
__global__ void ifGraphKernelD(void)
{
    printf("GPU: Hello from the GPU! The condition was false.\n");
}

// Setup and launch the graph
void simpleIfElseGraph(void)
{
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t kernelNode;
    hipGraphNode_t conditionalNode;

    void *kernelArgs[2];

    // Allocate a byte of device memory to use as input
    char *dPtr;
    checkCudaErrors(hipMalloc((void **)&dPtr, 1));

    printf("simpleIfElseGraph: Building graph...\n");
    hipGraphCreate(&graph, 0);

    // Create conditional handle.
    cudaGraphConditionalHandle handle;
    cudaGraphConditionalHandleCreate(&handle, graph);

    // Use a kernel upstream of the conditional to set the handle value
    hipGraphNodeParams params = {hipGraphNodeTypeKernel};
    params.kernel.func = (void *)ifGraphKernelA;
    params.kernel.gridDim = 1;
    params.kernel.blockDim = 1;
    params.kernel.kernelParams = kernelArgs;
    kernelArgs[0] = &dPtr;
    kernelArgs[1] = &handle;
    checkCudaErrors(hipGraphAddNode(&kernelNode, graph, NULL, 0, &params));

    hipGraphNodeParams cParams = {hipGraphNodeTypeConditional};
    cParams.conditional.handle = handle;
    cParams.conditional.type = cudaGraphCondTypeIf;
    cParams.conditional.size = 2; // Set size to 2 to indicate an ELSE graph will be used
    checkCudaErrors(hipGraphAddNode(&conditionalNode, graph, &kernelNode, 1, &cParams));

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    // Populate the body of the first graph in the conditional node, executed if the condition is true
    hipGraphNode_t trueBodyNode;
    params.kernel.func = (void *)ifGraphKernelC;
    params.kernel.kernelParams = nullptr;
    checkCudaErrors(hipGraphAddNode(&trueBodyNode, bodyGraph, NULL, 0, &params));

    // Populate the body of the second graph in the conditional node, executed if the condition is false
    bodyGraph = cParams.conditional.phGraph_out[1];

    hipGraphNode_t falseBodyNode;
    params.kernel.func = (void *)ifGraphKernelD;
    params.kernel.kernelParams = nullptr;
    checkCudaErrors(hipGraphAddNode(&falseBodyNode, bodyGraph, NULL, 0, &params));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 0, 1)); // Set dPtr to 0
    printf("Host: Launching graph with device memory set to 0\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 1, 1)); // Set dPtr to 1
    printf("Host: Launching graph with device memory set to 1\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));

    printf("simpleIfElseGraph: Complete\n\n");
}

/*
 * Create a graph containing two nodes.
 * The first node, A, is a kernel and the second node, B, is a conditional SWITCH node containing four graphs.
 * The nth graph within the conditional will be executed when the condition is n. If conditional >= n,
 * no graph will be executed.
 * Kernel A sets the condition variable to the value stored in a device memory location.
 * This device location is updated from the host with each launch to demonstrate the behavior.
 * There is a single kernel(nodes C, D, E and F) within each conditional body which prints a message.
 *
 * A -> B [ C | D | E | F ]
 *
 * This example requires CUDA >= 12.8.
 */

__global__ void switchGraphKernelA(char *dPtr, cudaGraphConditionalHandle handle)
{
    unsigned int value = *dPtr;
    cudaGraphSetConditional(handle, value);
    printf("GPU: Handle set to %d\n", value);
}

__global__ void switchGraphKernelC(void)
{
    printf("GPU: Hello from switchGraphKernelC(), running on the GPU!\n");
}

__global__ void switchGraphKernelD(void)
{
    printf("GPU: Hello from switchGraphKernelD(), running on the GPU!\n");
}

__global__ void switchGraphKernelE(void)
{
    printf("GPU: Hello from switchGraphKernelE(), running on the GPU!\n");
}

__global__ void switchGraphKernelF(void)
{
    printf("GPU: Hello from switchGraphKernelF(), running on the GPU!\n");
}

// Setup and launch the graph
void simpleSwitchGraph(void)
{
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t kernelNode;
    hipGraphNode_t conditionalNode;

    void *kernelArgs[2];

    // Allocate a byte of device memory to use as input
    char *dPtr;
    checkCudaErrors(hipMalloc((void **)&dPtr, 1));

    printf("simpleSwitchGraph: Building graph...\n");
    hipGraphCreate(&graph, 0);

    // Create conditional handle.
    cudaGraphConditionalHandle handle;
    cudaGraphConditionalHandleCreate(&handle, graph);

    // Use a kernel upstream of the conditional to set the handle value
    hipGraphNodeParams params = {hipGraphNodeTypeKernel};
    params.kernel.func = (void *)switchGraphKernelA;
    params.kernel.gridDim = 1;
    params.kernel.blockDim = 1;
    params.kernel.kernelParams = kernelArgs;
    kernelArgs[0] = &dPtr;
    kernelArgs[1] = &handle;
    checkCudaErrors(hipGraphAddNode(&kernelNode, graph, NULL, 0, &params));

    hipGraphNodeParams cParams = {hipGraphNodeTypeConditional};
    cParams.conditional.handle = handle;
    cParams.conditional.type = cudaGraphCondTypeSwitch;
    cParams.conditional.size = 4;
    checkCudaErrors(hipGraphAddNode(&conditionalNode, graph, &kernelNode, 1, &cParams));

    // Populate the four graph bodies within the SWITCH conditional graph
    hipGraphNode_t bodyNode;
    params.kernel.kernelParams = nullptr;
    params.kernel.func = (void *)switchGraphKernelC;
    checkCudaErrors(hipGraphAddNode(&bodyNode, cParams.conditional.phGraph_out[0], NULL, 0, &params));
    params.kernel.func = (void *)switchGraphKernelD;
    checkCudaErrors(hipGraphAddNode(&bodyNode, cParams.conditional.phGraph_out[1], NULL, 0, &params));
    params.kernel.func = (void *)switchGraphKernelE;
    checkCudaErrors(hipGraphAddNode(&bodyNode, cParams.conditional.phGraph_out[2], NULL, 0, &params));
    params.kernel.func = (void *)switchGraphKernelF;
    checkCudaErrors(hipGraphAddNode(&bodyNode, cParams.conditional.phGraph_out[3], NULL, 0, &params));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    for (char i = 0; i < 5; i++)
    {
        // Initialize device memory and launch the graph
        checkCudaErrors(hipMemset(dPtr, i, 1));
        printf("Host: Launching graph with device memory set to %d\n", i);
        checkCudaErrors(hipGraphLaunch(graphExec, 0));
        checkCudaErrors(hipDeviceSynchronize());
    }

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));

    printf("simpleSwitchGraph: Complete\n\n");
}

int main(int argc, char **argv)
{
    int device = findCudaDevice(argc, (const char **)argv);

    int driverVersion = 0;

    hipDriverGetVersion(&driverVersion);
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
           (driverVersion % 100) / 10);

    if (driverVersion < 12030)
    {
        printf("Skipping execution as driver does not support Graph Conditional Nodes\n");
        return 0;
    }

    simpleIfGraph();
    simpleDoWhileGraph();
    capturedWhileGraph();

    if (driverVersion < 12080)
    {
        printf("Skipping execution as driver does not support if/else and switch type Graph Conditional Nodes\n");
        return 0;
    }

    simpleIfElseGraph();
    simpleSwitchGraph();

    return 0;
}
