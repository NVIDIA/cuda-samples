#include "hip/hip_runtime.h"
/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This file demonstrates the usage of conditional graph nodes with
 * a series of *simple* example graphs.
 * 
 * For more information on conditional nodes, see the programming guide:
 * 
 *   https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#conditional-graph-nodes
 *
 */

// System includes
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

/*
 * Create a graph containing two nodes.
 * The first node, A, is a kernel and the second node, B, is a conditional IF node.
 * The kernel sets the condition variable to true if a device memory location
 * contains an odd number. Otherwise the condition variable is set to false.
 * There is a single kernel, C, within the conditional body which prints a message.
 *
 * A -> B [ C ]
 *
 */

__global__ void ifGraphKernelA(char *dPtr, cudaGraphConditionalHandle handle)
{
	// In this example, condition is set if *dPtr is odd
    unsigned int value = *dPtr & 0x01;
    cudaGraphSetConditional(handle, value);
    printf("GPU: Handle set to %d\n", value);
}

// This kernel will only be executed if the condition is true
__global__ void ifGraphKernelC(void)
{
    printf("GPU: Hello from the GPU!\n");
}

// Setup and launch the graph
void simpleIfGraph(void)
{
    hipGraph_t     graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t node;

    void *kernelArgs[2];

    // Allocate a byte of device memory to use as input
    char *dPtr;
    checkCudaErrors(hipMalloc((void**)&dPtr, 1));

    printf("simpleIfGraph: Building graph...\n");
    hipGraphCreate(&graph, 0);

    // Create conditional handle.
    cudaGraphConditionalHandle handle;
    cudaGraphConditionalHandleCreate(&handle, graph);

    // Use a kernel upstream of the conditional to set the handle value
    hipGraphNodeParams params = { hipGraphNodeTypeKernel };
    params.kernel.func         = (void *)ifGraphKernelA;
    params.kernel.gridDim.x    = params.kernel.gridDim.y = params.kernel.gridDim.z = 1;
    params.kernel.blockDim.x   = params.kernel.blockDim.y = params.kernel.blockDim.z = 1;
    params.kernel.kernelParams = kernelArgs;
    kernelArgs[0] = &dPtr;
    kernelArgs[1] = &handle;
    checkCudaErrors(hipGraphAddNode(&node, graph, NULL, 0, &params));

    hipGraphNodeParams cParams = { hipGraphNodeTypeConditional };
    cParams.conditional.handle = handle;
    cParams.conditional.type   = cudaGraphCondTypeIf;
    cParams.conditional.size   = 1;
    checkCudaErrors(hipGraphAddNode(&node, graph, &node, 1, &cParams));

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    // Populate the body of the conditional node
    hipGraphNode_t bodyNode;
    params.kernel.func         = (void *)ifGraphKernelC;
    params.kernel.kernelParams = nullptr;
    checkCudaErrors(hipGraphAddNode(&bodyNode, bodyGraph, NULL, 0, &params));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 0, 1)); // Set dPtr to 0
    printf("Host: Launching graph with conditional value set to false\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 1, 1)); // Set dPtr to 1
    printf("Host: Launching graph with conditional value set to true\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));

    printf("simpleIfGraph: Complete\n\n");
}

/*
 * Create a graph containing a single conditional while node.
 * The default value of the conditional variable is set to true, so this
 * effectively becomes a do-while loop as the conditional body will always
 * execute at least once. The body of the conditional contains 3 kernel nodes:
 * A [ B -> C -> D ]
 * Nodes B and C are just dummy nodes for demonstrative purposes. Node D
 * will decrement a device memory location and set the condition value to false
 * when the value reaches zero, terminating the loop.
 * In this example, stream capture is used to populate the conditional body.
 */

// This kernel will only be executed if the condition is true
__global__ void doWhileEmptyKernel(void)
{
    printf("GPU: doWhileEmptyKernel()\n");
    return;
}

__global__ void doWhileLoopKernel(char *dPtr, cudaGraphConditionalHandle handle)
{
    if (--(*dPtr) == 0) {
        cudaGraphSetConditional(handle, 0);
    }
    printf("GPU: counter = %d\n", *dPtr);
}

void simpleDoWhileGraph(void)
{
    hipGraph_t     graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t node;

    // Allocate a byte of device memory to use as input
    char *dPtr;
    checkCudaErrors(hipMalloc((void**)&dPtr, 1));

    printf("simpleDoWhileGraph: Building graph...\n");
    checkCudaErrors(hipGraphCreate(&graph, 0));

    cudaGraphConditionalHandle handle;
    checkCudaErrors(cudaGraphConditionalHandleCreate(&handle, graph, 1, cudaGraphCondAssignDefault));

    hipGraphNodeParams cParams = { hipGraphNodeTypeConditional };
    cParams.conditional.handle = handle;
    cParams.conditional.type   = cudaGraphCondTypeWhile;
    cParams.conditional.size   = 1;
    checkCudaErrors(hipGraphAddNode(&node, graph, NULL, 0, &cParams));

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    hipStream_t captureStream;
    checkCudaErrors(hipStreamCreate(&captureStream));
    
    checkCudaErrors(hipStreamBeginCaptureToGraph(captureStream, bodyGraph, nullptr, nullptr, 0, hipStreamCaptureModeRelaxed));
    doWhileEmptyKernel<<<1, 1, 0, captureStream>>>();
    doWhileEmptyKernel<<<1, 1, 0, captureStream>>>();
    doWhileLoopKernel<<<1, 1, 0, captureStream>>>(dPtr, handle);
    checkCudaErrors(hipStreamEndCapture(captureStream, nullptr));
    checkCudaErrors(hipStreamDestroy(captureStream));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 10, 1)); // Set dPtr to 10
    printf("Host: Launching graph with loop counter set to 10\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));

    printf("simpleDoWhileGraph: Complete\n\n");
}


/*
 * Create a graph containing a conditional while loop using stream capture.
 * This demonstrates how to insert a conditional node into a stream which is
 * being captured. The graph consists of a kernel node followed by a conditional
 * while node which contains a single kernel node:
 *
 * A -> B [ C ]
 *
 * The same kernel will be used for both nodes A and C. This kernel will test
 * a device memory location and set the condition when the location is non-zero.
 * We must run the kernel before the loop as well as inside the loop in order
 * to behave like a while loop. We need to evaluate the device memory location
 * before the conditional node is evaluated in order to set the condition variable
 * properly. Because we're using a kernel upstream of the conditional node,
 * there is no need to use the handle default value to initialize the conditional
 * value.
 */

__global__ void capturedWhileKernel(char *dPtr, cudaGraphConditionalHandle handle)
{
    printf("GPU: counter = %d\n", *dPtr);
    if (*dPtr) {
        (*dPtr)--;
    }
    cudaGraphSetConditional(handle, *dPtr);
}

void capturedWhileGraph(void)
{
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    hipStreamCaptureStatus status;
    const hipGraphNode_t *dependencies;
    size_t numDependencies;

    // Allocate a byte of device memory to use as input
    char *dPtr;
    checkCudaErrors(hipMalloc((void**)&dPtr, 1));

    printf("capturedWhileGraph: Building graph...\n");
    hipStream_t captureStream;
    checkCudaErrors(hipStreamCreate(&captureStream));

    checkCudaErrors(hipStreamBeginCapture(captureStream, hipStreamCaptureModeRelaxed));

    // Obtain the handle of the graph
    checkCudaErrors(hipStreamGetCaptureInfo(captureStream, &status, NULL, &graph, &dependencies, &numDependencies));

    // Create the conditional handle
    cudaGraphConditionalHandle handle;
    checkCudaErrors(cudaGraphConditionalHandleCreate(&handle, graph));

    // Insert kernel node A
    capturedWhileKernel<<<1, 1, 0, captureStream>>>(dPtr, handle);

    // Obtain the handle for node A
    checkCudaErrors(hipStreamGetCaptureInfo(captureStream, &status, NULL, &graph, &dependencies, &numDependencies));

    // Insert conditional node B
    hipGraphNode_t node;
    hipGraphNodeParams cParams = { hipGraphNodeTypeConditional };
    cParams.conditional.handle = handle;
    cParams.conditional.type   = cudaGraphCondTypeWhile;
    cParams.conditional.size   = 1;
    checkCudaErrors(hipGraphAddNode(&node, graph, dependencies, numDependencies, &cParams));

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    checkCudaErrors(hipStreamEndCapture(captureStream, &graph));
    checkCudaErrors(hipStreamDestroy(captureStream));

    // Populate conditional body graph using stream capture
    hipStream_t bodyStream;
    checkCudaErrors(hipStreamCreate(&bodyStream));

    checkCudaErrors(hipStreamBeginCaptureToGraph(bodyStream, bodyGraph, nullptr, nullptr, 0, hipStreamCaptureModeRelaxed));

    // Insert kernel node C
    capturedWhileKernel<<<1, 1, 0, bodyStream>>>(dPtr, handle);
    checkCudaErrors(hipStreamEndCapture(bodyStream, nullptr));
    checkCudaErrors(hipStreamDestroy(bodyStream));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Initialize device memory and launch the graph
    // Device memory is zero, so the conditional node will not execute
    checkCudaErrors(hipMemset(dPtr, 0, 1)); // Set dPtr to 0
    printf("Host: Launching graph with loop counter set to 0\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 10, 1)); // Set dPtr to 10
    printf("Host: Launching graph with loop counter set to 10\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));

    printf("capturedWhileGraph: Complete\n\n");
}


int main(int argc, char **argv) {
    int device = findCudaDevice(argc, (const char **)argv);

    int driverVersion = 0;

    hipDriverGetVersion(&driverVersion);
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
            (driverVersion % 100) / 10);

    if (driverVersion < 12030) {
        printf("Waiving execution as driver does not support Graph Conditional Nodes\n");
        exit(EXIT_WAIVED);
    }

    simpleIfGraph();
    simpleDoWhileGraph();
    capturedWhileGraph();

    return 0;
}
