#include "hip/hip_runtime.h"
/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This file demonstrates the usage of conditional graph nodes with
 * a series of *simple* example graphs.
 * 
 * For more information on conditional nodes, see the programming guide:
 * 
 *   https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#conditional-graph-nodes
 *
 */

// System includes
#include <assert.h>
#include <stdio.h>

#include <climits>
#include <vector>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

/*
 * Create a graph containing two nodes.
 * The first node, A, is a kernel and the second node, B, is a conditional IF node.
 * The kernel sets the condition variable to true if a device memory location
 * contains an odd number. Otherwise the condition variable is set to false.
 * There is a single kernel, C, within the conditional body which prints a message.
 *
 * A -> B [ C ]
 *
 */

__global__ void ifGraphKernelA(char *dPtr, cudaGraphConditionalHandle handle)
{
	// In this example, condition is set if *dPtr is odd
    unsigned int value = *dPtr & 0x01;
    cudaGraphSetConditional(handle, value);
    printf("GPU: Handle set to %d\n", value);
}

// This kernel will only be executed if the condition is true
__global__ void ifGraphKernelC(void)
{
    printf("GPU: Hello from the GPU!\n");
}

// Setup and launch the graph
void simpleIfGraph(void)
{
    hipGraph_t     graph;
    hipGraphExec_t graphExec;
    hipGraphNode_t node;

    void *kernelArgs[2];
    char *dPtr;      // Pointer to device memory location

    // Allocate a byte of device memory to use as input
    checkCudaErrors(hipMalloc((void**)&dPtr, 1));

    hipGraphCreate(&graph, 0);

    // Create conditional handle.
    cudaGraphConditionalHandle handle;
    cudaGraphConditionalHandleCreate(&handle, graph);

    // Use a kernel upstream of the conditional to set the handle value
    hipGraphNodeParams params = { hipGraphNodeTypeKernel };
    params.kernel.func         = (void *)ifGraphKernelA;
    params.kernel.gridDim.x    = params.kernel.gridDim.y = params.kernel.gridDim.z = 1;
    params.kernel.blockDim.x   = params.kernel.blockDim.y = params.kernel.blockDim.z = 1;
    params.kernel.kernelParams = kernelArgs;
    kernelArgs[0] = &dPtr;
    kernelArgs[1] = &handle;
    checkCudaErrors(hipGraphAddNode(&node, graph, NULL, 0, &params));

    hipGraphNodeParams cParams = { hipGraphNodeTypeConditional };
    cParams.conditional.handle = handle;
    cParams.conditional.type   = cudaGraphCondTypeIf;
    cParams.conditional.size   = 1;
    checkCudaErrors(hipGraphAddNode(&node, graph, &node, 1, &cParams));

    hipGraph_t bodyGraph = cParams.conditional.phGraph_out[0];

    // Populate the body of the conditional node
    hipGraphNode_t bodyNode;
    params.kernel.func         = (void *)ifGraphKernelC;
    params.kernel.kernelParams = nullptr;
    checkCudaErrors(hipGraphAddNode(&bodyNode, bodyGraph, NULL, 0, &params));

    checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 0, 1)); // Set dPtr to 0
    printf("Host: Launching graph with conditional value set to false\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Initialize device memory and launch the graph
    checkCudaErrors(hipMemset(dPtr, 1, 1)); // Set dPtr to 1
    printf("Host: Launching graph with conditional value set to true\n");
    checkCudaErrors(hipGraphLaunch(graphExec, 0));
    checkCudaErrors(hipDeviceSynchronize());

    // Cleanup
    checkCudaErrors(hipGraphExecDestroy(graphExec));
    checkCudaErrors(hipGraphDestroy(graph));
    checkCudaErrors(hipFree(dPtr));
}

int main(int argc, char **argv) {
    int device = findCudaDevice(argc, (const char **)argv);

    int driverVersion = 0;

    hipDriverGetVersion(&driverVersion);
    printf("Driver version is: %d.%d\n", driverVersion / 1000,
            (driverVersion % 100) / 10);

    if (driverVersion < 12030) {
        printf("Waiving execution as driver does not support Graph Conditional Nodes\n");
        exit(EXIT_WAIVED);
    }

    simpleIfGraph();

    return 0;
}
