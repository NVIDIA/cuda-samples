#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// CUDA sample demonstrating a __hip_bfloat16 (E8M7) GEMM computation using the Warp Matrix Multiply
// and Accumulate API introduced in CUDA 11.0.

// In this program, the compute_gemm kernel computes the result of a matrix multiplication
// and addition: D = alpha * A * B + beta * C. The dimensions of both C and D matrices
// are M_GLOBAL x N_GLOBAL. The A matrix is M_GLOBAL x K_GLOBAL (row-major), the B matrix
// is K_GLOBAL x N_GLOBAL (column-major).
// In that kernel, each CTA computes one 128 x 128 tile of the resulting matrix
// per iteration. When the tile is computed, the CTA stores it to the global memory
// and begins a new iteration, selecting a new 128 x 128 tile to compute.
// Each CTA consists of eight warps. For the 128 x 128 tile, each warp computes eight
// 16 x 16 subtiles, organized in a 2 x 4 two-dimensional array.
// Warps compute the 16 x 16 subtiles using nvcuda::wmma::mma_sync operations by
// moving through the K_GLOBAL dimension of the A and B matrices and accumulating
// the intermediate result in the local thread state.

// There are a number of simple optimizations used in the algorithm:
// - The CTA copies the 128 x 128 tile of the C matrix from the global memory to
//   shared memory. After that is done, each warp loads the C matrix fragments from
//   shared memory, thus avoiding a random global memory access.
// - On each internal iteration, the CTA copies a portion of the A and B matrices from
//   global memory to shared memory. After that, all warps in the CTA reuse the A and B
//   data from shared memory, thus reducing the number of data copies from global memory.
// - The portions of the A and B matrices are stored in shared memory with an additional
//   padding (skew) to reduce the number of shared memory access bank conflicts.
//   (See a detailed explanation near the SKEW_BF16 macro definition.)
// - When the CTA finishes computing the tiles of the resulting matrix, each warp stores
//   its subtiles to shared memory. The CTA then copies the shared memory contents to
//   global memory, again avoiding redundant random global memory accesses.
// - Note that the CTA tile size is chosen to maximize the GPU register utilization,
//   but carefully enough to avoid local memory use.

#include <assert.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <mma.h>
#include <cuda/pipeline>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// Externally configurable parameters.

#ifndef CPU_DEBUG
// Set this to 1 to verify the correctness of the GPU-computed matrix.
#define CPU_DEBUG 0
#endif

#ifndef SHARED_MEMORY_LIMIT_64K
// Set this to 0 to use more than 64 Kb of shared memory to cache data, to
// improve the performance of the computations on GPU.
// Note that you need a GPU that can have more than 64 Kb of shared memory
// per multiprocessor.
#define SHARED_MEMORY_LIMIT_64K 0
#endif

// GPU configuration.

#define WARP_SIZE 32

// MMA matrix tile dimensions.

#define M 16
#define N 16
#define K 16

// GEMM configuration.

#define M_TILES 512
#define N_TILES 512
#define K_TILES 512

#define M_GLOBAL (M * M_TILES)
#define N_GLOBAL (N * N_TILES)
#define K_GLOBAL (K * K_TILES)

#define C_LAYOUT wmma::mem_row_major

// Implementation constants.

#define WARPS_PER_BLOCK 8
#define THREADS_PER_BLOCK (WARP_SIZE * WARPS_PER_BLOCK)

#if SHARED_MEMORY_LIMIT_64K
// With only 64 Kb shared memory available, we can fit two 8-tile chunks of
// the A and B matrix data, that is (M = 16) * (K = 16) * 8 * (CHUNK_K = 8)
// * sizeof(__hip_bfloat16) = 32 Kb each.
// (i.e. two 8x8 arrays of tiles of 16x16 __hip_bfloat16-typed elements per CTA).
// But we cannot account the 8 Kb total skew overhead, without which the performance
// would be severely impacted. So we choose to reduce the chunk size in half,
// i.e. the amount of A and B matrix data we cache in shared memory.
// Accordingly, this doubles the number of outer iterations across the global K
// dimension, which only slightly impacts the performance.
#define CHUNK_K 4
#else
#define CHUNK_K 8
#endif

#define CHUNK_LINE_BYTES (CHUNK_K * K * sizeof(__hip_bfloat16))
#define WARP_COPY_BYTES (WARP_SIZE * sizeof(int4))
#define CHUNK_COPY_LINES_PER_WARP (WARP_COPY_BYTES / CHUNK_LINE_BYTES)
#define CHUNK_COPY_LINE_LANES (WARP_SIZE / CHUNK_COPY_LINES_PER_WARP)

#define BLOCK_ROW_WARPS 2
#define BLOCK_COL_WARPS 4

#define WARP_ROW_TILES 4
#define WARP_COL_TILES 2

#define BLOCK_ROW_TILES (WARP_ROW_TILES * BLOCK_ROW_WARPS)
#define BLOCK_COL_TILES (WARP_COL_TILES * BLOCK_COL_WARPS)

#define GLOBAL_MEM_STRIDE N_GLOBAL

#define SHMEM_STRIDE (N * BLOCK_ROW_TILES)
#define SHMEM_OFFSET (N * WARP_ROW_TILES)

// The macro below is used to shift rows of the A matrix and columns of the B matrix
// in shared memory to minimize possible bank conflicts.
// Before performing the nvcuda::wmma::mma_sync operation, the warp must load the matrix
// data using the nvcuda::wmma::load_matrix_sync operation. Although the memory access pattern
// is not specified for that function, each lane in the warp can read one or multiple matrix
// elements from different matrix rows or columns.
// For shared memory, such access can result in bank conflicts if different rows / columns
// of the matrix map to the same bank. By shifting each row and column by a few bytes, we
// make sure that they map to different banks, thus reducing the number of possible bank
// conflicts.
// The number of 16 two-byte "__hip_bfloat16" elements is chosen as the minimum possible shift because
// we must keep each row and column 256-bit aligned, as required by nvcuda::wmma::load_matrix_sync.
#define SKEW_BF16 16

#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("Line %d: '%s' failed: %s\n", __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)

enum kernels
{
    bf16mma_shmem_gemm_async_copy  = 0, // __hip_bfloat16 MMA shmem using kernel with async_copy 
    bf16mma_shmem_gemm             = 1, // __hip_bfloat16 MMA shmem using kernel normal copy (without async_copy).
    simple_bf16mma_gemm            = 2  // __hip_bfloat16 MMA non-shmem using simple kernel.
};

const char* kernelNames[] = {"compute_bf16gemm_async_copy", "compute_bf16gemm", 
                            "simple_wmma_bf16gemm"};

using namespace nvcuda;

__host__ void init_host_matrices(__hip_bfloat16 *a, __hip_bfloat16 *b, float *c)
{
    for (int i = 0; i < M_GLOBAL; i++) {
        for (int j = 0; j < K_GLOBAL; j++) {
            a[i*K_GLOBAL+j] = (__hip_bfloat16)(float)(rand() % 3);
        }
    }

    for (int i = 0; i < N_GLOBAL; i++) {
        for (int j = 0; j < K_GLOBAL; j++) {
            b[i*K_GLOBAL+j] = (__hip_bfloat16)(float)(rand() % 3);
        }
    }

    for (int t = 0; t < M_GLOBAL * N_GLOBAL; t++) {
        c[t] =  (float)(rand() % 3);
    }
}

__global__ void compute_bf16gemm(const __hip_bfloat16 *A, const __hip_bfloat16 *B, const float *C, float *D, float alpha, float beta)
{
#if __CUDA_ARCH__ >= 800
    extern __shared__ __hip_bfloat16 shmem[][CHUNK_K * K + SKEW_BF16];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    const size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    // This pointer is used to access the C and D matrix tiles this warp computes.
    float *shmem_warp_tile_ptr = (float*)&shmem[0][0] + (warpId / BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS + (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
    float *shmem_warp_stream_ptr = (float*)&shmem[0][0] + warpId * SHMEM_STRIDE * N;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may result
    // in a loss of precision). Zero still needs to be specially handled though.
    beta /= alpha;

    // Each CTA slides along the 128 x 128 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const float *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < N; i++) {
            *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId) = 
                *((int4*)(src_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId);
        }

        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Scale the C matrix.
#pragma unroll
       for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const __hip_bfloat16 *warp_ptr = (warpId < (WARPS_PER_BLOCK/2)) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);

        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            size_t shmem_idx = warpId < (WARPS_PER_BLOCK/2) ? (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) : 
                                                              (N * (warpId % (WARPS_PER_BLOCK/2)) * 2 + shmem_idx_b_off);

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            const __hip_bfloat16 *lane_ptr = (warp_ptr + tile_k * K + (laneId / CHUNK_COPY_LINE_LANES) * K_GLOBAL);

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += laneId / CHUNK_COPY_LINE_LANES;

#pragma unroll
            for(int i = 0; i < ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP) * 2; i++) {
                // Copy 16 bytes at once in each lane.
                *((int4*)&shmem[shmem_idx][0] + (laneId % CHUNK_COPY_LINE_LANES)) = *((int4*)lane_ptr +  (laneId % CHUNK_COPY_LINE_LANES));

                // Advance the global memory pointer and the shared memory index.
                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }

            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, __hip_bfloat16, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, __hip_bfloat16, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId/BLOCK_ROW_WARPS) * M * BLOCK_ROW_WARPS + (i * M);
                    const __hip_bfloat16 *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_BF16);

#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const __hip_bfloat16 *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_BF16);
                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }

            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
                // warp are well-defined even though element indices within fragment storage are not defined.
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global memory.
        float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < N; i++) {
            *((float4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((float4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
#endif
}

__global__ void compute_bf16gemm_async_copy(const __hip_bfloat16 *A, const __hip_bfloat16 *B, const float *C, float *D, float alpha, float beta)
{
#if __CUDA_ARCH__ >= 800
    extern __shared__ __hip_bfloat16 shmem[][CHUNK_K * K + SKEW_BF16];

    // Warp and lane identification.
    const unsigned int warpId = threadIdx.x / WARP_SIZE;
    const unsigned int laneId = threadIdx.x % WARP_SIZE;

    // Offset in shared memory from which the B matrix is stored.
    constexpr size_t shmem_idx_b_off = BLOCK_COL_TILES * M;

    // This pointer is used to access the C and D matrix tiles this warp computes.
    float *shmem_warp_tile_ptr = (float*)&shmem[0][0] + (warpId / BLOCK_ROW_WARPS) * SHMEM_STRIDE * N * BLOCK_ROW_WARPS + (warpId % BLOCK_ROW_WARPS) * SHMEM_OFFSET;

    // This pointer is used to stream the C and D matrices block-wide tile to and from shared memory.
    float *shmem_warp_stream_ptr = (float*)&shmem[0][0] + warpId * SHMEM_STRIDE * N;

    // Adjust the beta scaler, as it'll be multiplied by alpha at the end of
    // each tile computation. Technically this is not generally correct (may result
    // in a loss of precision). Zero still needs to be specially handled though.
    beta /= alpha;

    cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
    const auto shape4 = cuda::aligned_size_t<alignof(float4)>(sizeof(float4));
    constexpr int loadStride = 2; // load 4 floats, left-shift by 2.

    // Each CTA slides along the 128 x 128 tiles from the top left corner of the matrix to the
    // right and down, and selects the next tile to compute. Once there's no such tile,
    // all warps in this CTA exit.
    for(unsigned int block_pos = blockIdx.x;; block_pos += gridDim.x) {
        const unsigned int block_tile_i = ((block_pos * BLOCK_ROW_TILES) / N_TILES) * (BLOCK_COL_TILES);
        const unsigned int block_tile_j = (block_pos * BLOCK_COL_TILES) % N_TILES;

        // Stop when there are no more D matrix tiles to compute in this CTA.
        if (block_tile_i >= M_TILES) {
            break;
        }

        // This warp's pointer to the C matrix data to copy memory from to shared memory.
        const size_t gmem_idx = (block_tile_i + warpId) * M * GLOBAL_MEM_STRIDE + block_tile_j * N;
        const float *src_gmem_warp_stream_ptr = &C[gmem_idx];

        // Stream multiple C tiles to shared memory.
#pragma unroll
        for (int i = 0; i < N; i++) {
            pipe.producer_acquire();

            cuda::memcpy_async(&shmem_warp_stream_ptr[(SHMEM_STRIDE * i) + (laneId << loadStride)],
                                &src_gmem_warp_stream_ptr[(GLOBAL_MEM_STRIDE * i) + (laneId << loadStride)],
                                shape4, pipe);

            pipe.producer_commit();
        }

        // Now wait for all the above issued 8 batches to complete.
        cuda::pipeline_consumer_wait_prior<0>(pipe);
        __syncthreads();

        // These fragments will accumulate the result of A and B matrix fragment multiplications
        // along the K_GLOBAL dimension.
        wmma::fragment<wmma::accumulator, M, N, K, float> c[WARP_COL_TILES][WARP_ROW_TILES];

        // Load the C matrix tiles into fragments from shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
                const float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * N + j * N;

                wmma::load_matrix_sync(c[i][j], tile_ptr, SHMEM_STRIDE, C_LAYOUT);
                // Scale the C matrix.
#pragma unroll
                for (int t = 0; t < c[i][j].num_elements; t++) {
                    c[i][j].x[t] *= beta;
                }
            }
        }

        pipe.consumer_release();

        // sync here so that shared memory can then be used for loading A & B matrices.
        __syncthreads();
        // Select what warp copies what matrix to shared memory.
        // Warps 0-3 copy the A matrix, warps 4-7 copy the B matrix.
        const __hip_bfloat16 *warp_ptr = (warpId < (WARPS_PER_BLOCK/2)) ? (&A[block_tile_i * M * K_GLOBAL] + M * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2) :
                                              (&B[block_tile_j * N * K_GLOBAL] + N * K_GLOBAL * (warpId % (WARPS_PER_BLOCK/2)) * 2);

        constexpr int chunksPerLane = ((WARP_SIZE/2) / CHUNK_COPY_LINES_PER_WARP) * 2;
        constexpr int loadStrideBfloat8 = 3; // load 8 bfloats, left-shift by 3.
        const int laneLoadElem = (laneId % CHUNK_COPY_LINE_LANES) << loadStrideBfloat8;
        const int stridePerLaneCopy = (laneId / CHUNK_COPY_LINE_LANES);

        // Go through the global K dimension by a fixed step at a time.
#pragma unroll
        for (int tile_k = 0; tile_k < K_TILES; tile_k += CHUNK_K) {
            // Copy slices of the A and B matrices to shared memory.
            // The first half of the warps in the CTA copy the A matrix, the rest copy the B matrix.
            // As for bf16 MMA  M == N we use M for warp 4-7 + shmem_idx_b_off.
            size_t shmem_idx = (M * (warpId % (WARPS_PER_BLOCK/2)) * 2) + ((warpId / (WARPS_PER_BLOCK/2)) * shmem_idx_b_off);

            // First half of the warp copies the first row / column of the matrix,
            // the second half of the warp copies the next.
            const __hip_bfloat16 *lane_ptr = warp_ptr + tile_k * K + stridePerLaneCopy * K_GLOBAL + laneLoadElem;

            // Shift the second half of the warp to the next row / column in the shared memory.
            shmem_idx += stridePerLaneCopy;

#pragma unroll
            for(int i = 0; i < chunksPerLane; i++) {
                // Copy 16 bytes at once in each lane.
                pipe.producer_acquire();
                cuda::memcpy_async(&shmem[shmem_idx][laneLoadElem], lane_ptr, shape4, pipe);
                pipe.producer_commit();
                // Advance the global memory pointer and the shared memory index.
                lane_ptr = lane_ptr + K_GLOBAL * CHUNK_COPY_LINES_PER_WARP;
                shmem_idx += CHUNK_COPY_LINES_PER_WARP;
            }
            cuda::pipeline_consumer_wait_prior<0>(pipe);
            __syncthreads();

            // Compute a grid of C matrix tiles in each warp.
#pragma unroll
            for (int k_step = 0; k_step < CHUNK_K; k_step++) {
                wmma::fragment<wmma::matrix_a, M, N, K, __hip_bfloat16, wmma::row_major> a[WARP_COL_TILES];
                wmma::fragment<wmma::matrix_b, M, N, K, __hip_bfloat16, wmma::col_major> b[WARP_ROW_TILES];

#pragma unroll
                for (int i = 0; i < WARP_COL_TILES; i++) {
                    size_t shmem_idx_a = (warpId / BLOCK_ROW_WARPS) * M * BLOCK_ROW_WARPS + (i * M);
                    const __hip_bfloat16 *tile_ptr = &shmem[shmem_idx_a][k_step * K];

                    wmma::load_matrix_sync(a[i], tile_ptr, K * CHUNK_K + SKEW_BF16);

#pragma unroll
                    for (int j = 0; j < WARP_ROW_TILES; j++) {
                        if (i == 0) {
                            // Load the B matrix fragment once, because it is going to be reused
                            // against the other A matrix fragments.
                            size_t shmem_idx_b = shmem_idx_b_off + (WARP_ROW_TILES * N) * (warpId%2) + (j * N);
                            const __hip_bfloat16 *tile_ptr = &shmem[shmem_idx_b][k_step * K];

                            wmma::load_matrix_sync(b[j], tile_ptr, K * CHUNK_K + SKEW_BF16);
                        }

                        wmma::mma_sync(c[i][j], a[i], b[j], c[i][j]);
                    }
                }
            }

            pipe.consumer_release();
            __syncthreads();
        }

        // Store the D fragments to shared memory.
#pragma unroll
        for (int i = 0; i < WARP_COL_TILES; i++) {
#pragma unroll
            for (int j = 0; j < WARP_ROW_TILES; j++) {
#pragma unroll
                // Uniform, point-wise transformations of ALL fragment elements by ALL threads in the
                // warp are well-defined even though element indices within fragment storage are not defined.
                for (int t = 0; t < c[i][j].num_elements; t++)
                    c[i][j].x[t] *= alpha;

                float *tile_ptr = shmem_warp_tile_ptr + i * SHMEM_STRIDE * K + j * N;

                wmma::store_matrix_sync(tile_ptr, c[i][j], SHMEM_STRIDE, C_LAYOUT);
            }
        }

        __syncthreads();

        // Now that shared memory contains all the D tiles, stream them to global memory.
        float *dst_gmem_warp_stream_ptr = &D[gmem_idx];

#pragma unroll
        for (int i = 0; i < N; i++) {
            *((int4*)(dst_gmem_warp_stream_ptr + GLOBAL_MEM_STRIDE * i) + laneId) =
                *((int4*)(shmem_warp_stream_ptr + SHMEM_STRIDE * i) + laneId);
        }

        __syncthreads();
    }
#endif
}

// Performs an MxNxK bf16 GEMM (C=alpha*A*B + beta*C) assuming:
//  1) Matrices are packed in memory.
//  2) M, N and K are multiples of 16, 16 and 16 respectively. 
//  3) A is row major, B is column major matrix.
// Note: This is a less performant version of the compute_bf16gemm kernel. It is designed for
//       demonstration purposes only to show the CUDA WMMA API use without relying on
//       availability of the shared memory.
__global__ void simple_wmma_bf16gemm(__hip_bfloat16 *a, __hip_bfloat16 *b, float *c, float *d, int m_ld, int n_ld, int k_ld, float alpha, float beta)
{
#if __CUDA_ARCH__ >= 800
   // Leading dimensions. Packed with no transpositions.
    int lda = k_ld;
    int ldb = k_ld;
    int ldc = n_ld;

   // Tile using a 2D grid
   int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
   int warpN = (blockIdx.y * blockDim.y + threadIdx.y);
 
   // Declare the fragments
   wmma::fragment<wmma::matrix_a, M, N, K, __hip_bfloat16, wmma::row_major> a_frag;
   wmma::fragment<wmma::matrix_b, M, N, K, __hip_bfloat16, wmma::col_major> b_frag;
   wmma::fragment<wmma::accumulator, M, N, K, float> acc_frag;
   wmma::fragment<wmma::accumulator, M, N, K, float> c_frag;

   wmma::fill_fragment(acc_frag, 0.0f);

   // Loop over k
   for (int i = 0; i < k_ld; i += K) {
      int aCol = i; 
      int aRow = warpM * M;

      int bCol = i;
      int bRow = warpN * N;

      // Bounds checking
      if (aRow < m_ld && aCol < k_ld && bRow < k_ld && bCol < n_ld) {
         // Load the inputs
         wmma::load_matrix_sync(a_frag, a + aCol + aRow * lda, lda);
         wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);
 
         // Perform the matrix multiplication
         wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

      }
   }

   // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
   int cCol = warpN * N;
   int cRow = warpM * M;

   if (cRow < m_ld && cCol < n_ld) {
      wmma::load_matrix_sync(c_frag, c + cCol + cRow * ldc, ldc, wmma::mem_row_major);

      for(int i=0; i < c_frag.num_elements; i++) {
         c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
      }

      // Store the output
      wmma::store_matrix_sync(d + cCol + cRow * ldc, c_frag, ldc, wmma::mem_row_major);
   }
#endif
}

__host__ void matMultiplyOnHost(__hip_bfloat16 *A, __hip_bfloat16 *B, float *C,
                                float alpha, float beta,
                                int numARows, int numAColumns,
                                int numBRows, int numBColumns,
                                int numCRows, int numCColumns)
{
    for (int i = 0; i < numCRows; i++) {
        for (int j = 0; j < numCColumns; j++) {
            float temp = 0.0;

            for (int k = 0; k < numAColumns; k++) {
                temp += (float)A[i * numAColumns + k] * (float)B[j * numBRows + k];
            }

            C[i*numCColumns + j] = temp * alpha + beta * C[i * numCColumns + j];
        }
    }
}

int main(int argc, char **argv)
{
    printf("Initializing...\n");

    int dev = findCudaDevice(argc, (const char **)argv);

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

    // Tensor cores require a GPU of Volta (SM8X) architecture or higher.
    if (deviceProp.major < 8) {
        printf("bf16TensorCoreGemm requires requires SM 8.0 or higher to use Tensor Cores.  Exiting...\n");
        exit(EXIT_WAIVED);
    }

    printf("M: %d (%d x %d)\n", M_GLOBAL, M, M_TILES);
    printf("N: %d (%d x %d)\n", N_GLOBAL, N, N_TILES);
    printf("K: %d (%d x %d)\n", K_GLOBAL, K, K_TILES);

    __hip_bfloat16 *A_h = NULL;
    __hip_bfloat16 *B_h = NULL;
    float *C_h = NULL;
#if CPU_DEBUG
    float *result_hD = NULL;
    float *result_host = NULL;
#endif

    A_h = (__hip_bfloat16*) malloc(sizeof(__hip_bfloat16) * M_GLOBAL * K_GLOBAL);
    B_h = (__hip_bfloat16*) malloc(sizeof(__hip_bfloat16) * K_GLOBAL * N_GLOBAL);
    C_h = (float*) malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
#if CPU_DEBUG
    result_hD   = (float*) malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
    result_host = (float*) malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
#endif

    __hip_bfloat16 *A = NULL;
    __hip_bfloat16 *B = NULL;
    float *C = NULL;
    float *D = NULL;

    checkCudaErrors(hipMalloc((void**)&A, sizeof(__hip_bfloat16) * M_GLOBAL * K_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&B, sizeof(__hip_bfloat16) * N_GLOBAL * K_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&C, sizeof(float) * M_GLOBAL * N_GLOBAL));
    checkCudaErrors(hipMalloc((void**)&D, sizeof(float) * M_GLOBAL * N_GLOBAL));

    assert(((unsigned long long)A) % 128 == 0);
    assert(((unsigned long long)B) % 128 == 0);
    assert(((unsigned long long)C) % 128 == 0);
    assert(((unsigned long long)D) % 128 == 0);

    init_host_matrices(A_h, B_h, C_h);

    printf("Preparing data for GPU...\n");

    checkCudaErrors(hipMemcpy(A, A_h, sizeof(__hip_bfloat16) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(B, B_h, sizeof(__hip_bfloat16) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(C, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(D, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));

    enum {
        // Compute the right amount of shared memory to request.
        // We need shared memory to hold per-CTA C and D matrix tiles, and to cache per-CTA chunks
        // of the A and B matrices. Therefore, the right amount to request is the maximum of those
        // two numbers.
        SHMEM_SZ = MAX(sizeof(__hip_bfloat16) * (BLOCK_COL_TILES * M) * (CHUNK_K * K + SKEW_BF16) * 2,
                       M * (BLOCK_ROW_WARPS * WARP_ROW_TILES) * N * (BLOCK_COL_WARPS * WARP_COL_TILES) * sizeof(float))
    };

    printf("Required shared memory size: %lu Kb\n", SHMEM_SZ / 1024UL);

    const float alpha = 1.1f;
    const float beta = 1.2f;

    hipEvent_t start, stop;

    checkCudaErrors(hipEventCreate(&start));    
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));

    // kernel to run - default (b16mma_shmem_gemm_async_copy == 0)
    kernels selected_kernel = bf16mma_shmem_gemm_async_copy;

    if (checkCmdLineFlag(argc, (const char **)argv, "kernel")) {
        int kernel_number = getCmdLineArgumentInt(argc, (const char **)argv, "kernel");
        if (kernel_number < 3) {
            selected_kernel = (kernels)kernel_number;
        }
        else {
            printf("Error: kernel number should be between 0 to 2, you have entered %d\n", kernel_number);
            exit(EXIT_FAILURE);
        }
    }

    // If enough shared memory available on the GPU use high performant kernel
    if ((deviceProp.sharedMemPerMultiprocessor >= SHMEM_SZ) && (selected_kernel != simple_bf16mma_gemm)) {
        printf("Computing using high performance kernel = %d - %s\n", selected_kernel, kernelNames[selected_kernel]);

        switch (selected_kernel)
        {
            case bf16mma_shmem_gemm_async_copy :
            default:
                checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(compute_bf16gemm_async_copy), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));
                checkKernelErrors((compute_bf16gemm_async_copy<<<deviceProp.multiProcessorCount*2, THREADS_PER_BLOCK, SHMEM_SZ>>>(A, B, C, D, alpha, beta)));
                break;
            case bf16mma_shmem_gemm :
                checkCudaErrors(hipFuncSetAttribute(reinterpret_cast<const void*>(compute_bf16gemm), hipFuncAttributeMaxDynamicSharedMemorySize, SHMEM_SZ));
                checkKernelErrors((compute_bf16gemm<<<deviceProp.multiProcessorCount*2, THREADS_PER_BLOCK, SHMEM_SZ>>>(A, B, C, D, alpha, beta)));
                break;
        }
#if CPU_DEBUG
        checkCudaErrors(hipMemcpy(result_hD, D, sizeof(float)*M_GLOBAL*N_GLOBAL, hipMemcpyDeviceToHost));
#endif
    }
    else {
        dim3 gridDim;
        dim3 blockDim;
     
        // blockDim.x must be a multple of warpSize
        // 128x4 means we have 16 warps and a block computes a 64x64 output tile
        blockDim.x = 128;
        blockDim.y = 4;

        gridDim.x = (M_GLOBAL + (M * blockDim.x / 32 - 1)) / (M * blockDim.x / 32);
        gridDim.y = (N_GLOBAL + N * blockDim.y - 1) / (N * blockDim.y);

        printf("Computing... using simple_wmma_gemm kernel\n");
        simple_wmma_bf16gemm<<<gridDim, blockDim>>>(A, B, C, D, M_GLOBAL, N_GLOBAL, K_GLOBAL, alpha, beta);
#if CPU_DEBUG
        checkCudaErrors(hipMemcpy(result_hD, D, sizeof(float) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost));
#endif
    }

    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));

#if CPU_DEBUG
    printf("Verifying correctness of the computations...\n");

    memcpy(result_host, C_h, sizeof(float) * M_GLOBAL * N_GLOBAL);

    matMultiplyOnHost(A_h, B_h, result_host,
                      alpha, beta,
                      M_GLOBAL, K_GLOBAL,
                      K_GLOBAL, N_GLOBAL,
                      M_GLOBAL, N_GLOBAL);

    for (int i = 0; i < N_GLOBAL * M_GLOBAL; i++) {
        if (fabs(result_hD[i] - result_host[i]) > 0.1f) {
            printf("mismatch i=%d result_hD=%f result_host=%f\n", i, result_hD[i], result_host[i]);
        }
    }
    free(result_hD);
    free(result_host);
#endif

    float milliseconds = 0;

    checkCudaErrors(hipEventElapsedTime(&milliseconds, start, stop));

    printf("Time: %f ms\n", milliseconds);
    printf("TFLOPS: %.2f\n", (((double)M_GLOBAL * N_GLOBAL * K_GLOBAL * 2)/(milliseconds/1000.)) / 1e12);

    free(A_h);
    free(B_h);
    free(C_h);
    checkCudaErrors(hipFree((void*)A));
    checkCudaErrors(hipFree((void*)B));
    checkCudaErrors(hipFree((void*)C));
    checkCudaErrors(hipFree((void*)D));

    return 0;
}
