#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// System includes
#include <assert.h>
#include <stdio.h>

#include <climits>
#include <vector>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define THREADS_PER_BLOCK 512
#define ALLOWABLE_VARIANCE 1.e-6f
#define NUM_ELEMENTS 8000000

// Stores the square of each input element in output array
__global__ void squareArray(const float *input, float *output,
                            int numElements) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < numElements) {
    output[idx] = input[idx] * input[idx];
  }
}

// Stores the negative of each input element in output array
__global__ void negateArray(const float *input, float *output,
                            int numElements) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < numElements) {
    output[idx] = input[idx] * -1;
  }
}

struct negSquareArrays {
  float *input;
  float *square;
  float *negSquare;
  int numElements;
  size_t bytes;
  size_t numBlocks;
};

void fillRandomly(float *array, int numElements) {
  for (int n = 0; n < numElements; n++) {
    array[n] = rand() / (float)RAND_MAX;
  }
}

void resetOutputArrays(negSquareArrays *hostArrays) {
  fillRandomly(hostArrays->square, hostArrays->numElements);
  fillRandomly(hostArrays->negSquare, hostArrays->numElements);
}

void prepareHostArrays(negSquareArrays *hostArrays) {
  hostArrays->numElements = NUM_ELEMENTS;
  size_t bytes = hostArrays->numElements * sizeof(float);

  size_t numBlocks = hostArrays->numElements / (size_t)THREADS_PER_BLOCK;
  if ((numBlocks % (size_t)THREADS_PER_BLOCK) != 0) {
    numBlocks++;
  }

  hostArrays->input = (float *)malloc(bytes);
  hostArrays->square = (float *)malloc(bytes);
  hostArrays->negSquare = (float *)malloc(bytes);
  hostArrays->bytes = bytes;
  hostArrays->numBlocks = numBlocks;

  fillRandomly(hostArrays->input, hostArrays->numElements);
  fillRandomly(hostArrays->square, hostArrays->numElements);
  fillRandomly(hostArrays->negSquare, hostArrays->numElements);
}

void createFreeGraph(hipGraphExec_t *graphExec, float *dPtr) {
  hipGraph_t graph;
  hipGraphNode_t freeNode;

  checkCudaErrors(hipGraphCreate(&graph, 0));

  checkCudaErrors(
      hipGraphAddMemFreeNode(&freeNode, graph, NULL, 0, (void *)dPtr));

  checkCudaErrors(hipGraphInstantiate(graphExec, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphDestroy(graph));
}

/**
 * Demonstrates explicitly creating a CUDA graph including memory nodes.
 * createNegateSquaresGraphWithStreamCapture constructs an equivalent graph
 * using stream capture.
 *
 * If d_negSquare_out is non null, then:
 * 1) d_negSquare will not be freed;
 * 2) the value of d_negSquare_out will be set to d_negSquare.
 *
 * Diagram of the graph constructed by createNegateSquaresGraphExplicitly:
 *
 * alloc d_input
 *       |
 * alloc d_square
 *       |
 * Memcpy a to device
 *       |
 * launch kernel squareArray ------->---- Memcpy d_square to host
 *       |                                      |
 * free d_input                                 |
 *       |                                      |
 * allocate d_negSquare                         |
 *       |                                      |
 * launch kernel negateArray -------->--- free d_square
 *       |
 * Memcpy d_negSquare to host
 *       |
 * free d_negSquare
 */
void createNegateSquaresGraphExplicitly(hipGraphExec_t *graphExec, int device,
                                        negSquareArrays *hostArrays,
                                        float **d_negSquare_out = NULL) {
  // Array buffers on device
  float *d_input, *d_square, *d_negSquare;

  // Memory allocation parameters
  hipMemAllocNodeParams allocParams;
  memset(&allocParams, 0, sizeof(allocParams));
  allocParams.bytesize = hostArrays->bytes;
  allocParams.poolProps.allocType = hipMemAllocationTypePinned;
  allocParams.poolProps.location.id = device;
  allocParams.poolProps.location.type = hipMemLocationTypeDevice;

  // Kernel launch parameters
  hipKernelNodeParams kernelNodeParams = {0};
  kernelNodeParams.gridDim = dim3(hostArrays->numBlocks, 1, 1);
  kernelNodeParams.blockDim = dim3(THREADS_PER_BLOCK, 1, 1);
  kernelNodeParams.sharedMemBytes = 0;
  kernelNodeParams.extra = NULL;

  hipGraph_t graph;
  hipGraphNode_t allocNodeInput, allocNodeSquare, allocNodeNegSquare;
  hipGraphNode_t copyNodeInput, copyNodeSquare, copyNodeNegSquare;
  hipGraphNode_t squareKernelNode, negateKernelNode;
  hipGraphNode_t freeNodeInput, freeNodeSquare;

  // Buffer for storing graph node dependencies
  std::vector<hipGraphNode_t> nodeDependencies;

  checkCudaErrors(hipGraphCreate(&graph, 0));

  checkCudaErrors(
      hipGraphAddMemAllocNode(&allocNodeInput, graph, NULL, 0, &allocParams));
  d_input = (float *)allocParams.dptr;

  // To keep the graph structure simple (fewer branching dependencies),
  // allocNodeSquare should depend on allocNodeInput
  checkCudaErrors(hipGraphAddMemAllocNode(&allocNodeSquare, graph,
                                           &allocNodeInput, 1, &allocParams));
  d_square = (float *)allocParams.dptr;

  // copyNodeInput needs to depend on allocNodeInput because copyNodeInput
  // writes to d_input. It does so here indirectly through allocNodeSquare.
  checkCudaErrors(hipGraphAddMemcpyNode1D(
      &copyNodeInput, graph, &allocNodeSquare, 1, d_input, hostArrays->input,
      hostArrays->bytes, hipMemcpyHostToDevice));

  void *squareKernelArgs[3] = {(void *)&d_input, (void *)&d_square,
                               (void *)&(hostArrays->numElements)};
  kernelNodeParams.func = (void *)squareArray;
  kernelNodeParams.kernelParams = (void **)squareKernelArgs;

  // Square kernel depends on copyNodeInput to ensure all data is on the device
  // before kernel launch.
  checkCudaErrors(hipGraphAddKernelNode(&squareKernelNode, graph,
                                         &copyNodeInput, 1, &kernelNodeParams));

  checkCudaErrors(hipGraphAddMemcpyNode1D(
      &copyNodeSquare, graph, &squareKernelNode, 1, hostArrays->square,
      d_square, hostArrays->bytes, hipMemcpyDeviceToHost));

  // Free of d_input depends on the square kernel to ensure that d_input is not
  // freed while being read by the kernel. It also depends on the alloc of
  // d_input via squareKernelNode > copyNodeInput > allocNodeSquare >
  // allocNodeInput.
  checkCudaErrors(hipGraphAddMemFreeNode(&freeNodeInput, graph,
                                          &squareKernelNode, 1, d_input));

  // Allocation of C depends on free of A so CUDA can reuse the virtual address.
  checkCudaErrors(hipGraphAddMemAllocNode(&allocNodeNegSquare, graph,
                                           &freeNodeInput, 1, &allocParams));
  d_negSquare = (float *)allocParams.dptr;

  if (d_negSquare == d_input) {
    printf(
        "Check verified that d_negSquare and d_input share a virtual "
        "address.\n");
  }

  void *negateKernelArgs[3] = {(void *)&d_square, (void *)&d_negSquare,
                               (void *)&(hostArrays->numElements)};
  kernelNodeParams.func = (void *)negateArray;
  kernelNodeParams.kernelParams = (void **)negateKernelArgs;

  checkCudaErrors(hipGraphAddKernelNode(
      &negateKernelNode, graph, &allocNodeNegSquare, 1, &kernelNodeParams));

  nodeDependencies.push_back(copyNodeSquare);
  nodeDependencies.push_back(negateKernelNode);
  checkCudaErrors(hipGraphAddMemFreeNode(&freeNodeSquare, graph,
                                          nodeDependencies.data(),
                                          nodeDependencies.size(), d_square));
  nodeDependencies.clear();

  checkCudaErrors(hipGraphAddMemcpyNode1D(
      &copyNodeNegSquare, graph, &negateKernelNode, 1, hostArrays->negSquare,
      d_negSquare, hostArrays->bytes, hipMemcpyDeviceToHost));

  if (d_negSquare_out == NULL) {
    hipGraphNode_t freeNodeNegSquare;
    checkCudaErrors(hipGraphAddMemFreeNode(
        &freeNodeNegSquare, graph, &copyNodeNegSquare, 1, d_negSquare));
  } else {
    *d_negSquare_out = d_negSquare;
  }

  checkCudaErrors(hipGraphInstantiate(graphExec, graph, NULL, NULL, 0));
  checkCudaErrors(hipGraphDestroy(graph));
}

/**
 * Adds work to a CUDA stream which negates the square of values in the input
 * array.
 *
 * If d_negSquare_out is non null, then:
 * 1) d_negSquare will not be freed;
 * 2) the value of d_negSquare_out will be set to d_negSquare.
 *
 * Diagram of the stream operations in doNegateSquaresInStream
 * ---------------------------------------------------------------------
 * | STREAM                             | STREAM2                      |
 * ---------------------------------------------------------------------
 *
 * alloc d_input
 *       |
 * alloc d_square
 *       |
 * Memcpy a to device
 *       |
 * launch kernel squareArray
 *       |
 * record squareKernelCompleteEvent -->-- wait squareKernelCompleteEvent
 *       |                                      |
 * free d_input                                 |
 *       |                                      |
 * allocate d_negSquare                   Memcpy d_square to host
 *       |                                      |
 * launch kernel negateArray                    |
 *       |                                      |
 * record negateKernelCompleteEvent -->-- wait negateKernelCompleteEvent
 *       |                                      |
 * Memcpy d_negSquare to host                   |
 *       |                                free d_square
 * free d_negSquare                             |
 *       |                                      |
 * wait squareFreeEvent --------------<---- record squareFreeEvent
 */
void doNegateSquaresInStream(hipStream_t stream1, negSquareArrays *hostArrays,
                             float **d_negSquare_out = NULL) {
  float *d_input, *d_square, *d_negSquare;
  hipStream_t stream2;
  hipEvent_t squareKernelCompleteEvent, negateKernelCompleteEvent,
      squareFreeEvent;

  checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));

  checkCudaErrors(hipEventCreate(&squareKernelCompleteEvent));
  checkCudaErrors(hipEventCreate(&negateKernelCompleteEvent));
  checkCudaErrors(hipEventCreate(&squareFreeEvent));

  // Virtual addresses are assigned synchronously when hipMallocAsync is
  // called, thus there is no performace benefit gained by separating the
  // allocations into two streams.
  checkCudaErrors(hipMallocAsync(&d_input, hostArrays->bytes, stream1));
  checkCudaErrors(hipMallocAsync(&d_square, hostArrays->bytes, stream1));

  checkCudaErrors(hipMemcpyAsync(d_input, hostArrays->input, hostArrays->bytes,
                                  hipMemcpyHostToDevice, stream1));
  squareArray<<<hostArrays->numBlocks, THREADS_PER_BLOCK, 0, stream1>>>(
      d_input, d_square, hostArrays->numElements);
  checkCudaErrors(hipEventRecord(squareKernelCompleteEvent, stream1));

  checkCudaErrors(hipStreamWaitEvent(stream2, squareKernelCompleteEvent, 0));
  checkCudaErrors(hipMemcpyAsync(hostArrays->square, d_square,
                                  hostArrays->bytes, hipMemcpyDeviceToHost,
                                  stream2));

  checkCudaErrors(hipFreeAsync(d_input, stream1));
  checkCudaErrors(hipMallocAsync(&d_negSquare, hostArrays->bytes, stream1));
  negateArray<<<hostArrays->numBlocks, THREADS_PER_BLOCK, 0, stream1>>>(
      d_square, d_negSquare, hostArrays->numElements);
  checkCudaErrors(hipEventRecord(negateKernelCompleteEvent, stream1));
  checkCudaErrors(hipMemcpyAsync(hostArrays->negSquare, d_negSquare,
                                  hostArrays->bytes, hipMemcpyDeviceToHost,
                                  stream1));
  if (d_negSquare_out == NULL) {
    checkCudaErrors(hipFreeAsync(d_negSquare, stream1));
  } else {
    *d_negSquare_out = d_negSquare;
  }

  checkCudaErrors(hipStreamWaitEvent(stream2, negateKernelCompleteEvent, 0));
  checkCudaErrors(hipFreeAsync(d_square, stream2));
  checkCudaErrors(hipEventRecord(squareFreeEvent, stream2));

  checkCudaErrors(hipStreamWaitEvent(stream1, squareFreeEvent, 0));

  checkCudaErrors(hipStreamDestroy(stream2));
  checkCudaErrors(hipEventDestroy(squareKernelCompleteEvent));
  checkCudaErrors(hipEventDestroy(negateKernelCompleteEvent));
  checkCudaErrors(hipEventDestroy(squareFreeEvent));
}

/**
 * Demonstrates creating a CUDA graph including memory nodes using stream
 * capture. createNegateSquaresGraphExplicitly constructs an equivalent graph
 * without stream capture.
 */
void createNegateSquaresGraphWithStreamCapture(hipGraphExec_t *graphExec,
                                               negSquareArrays *hostArrays,
                                               float **d_negSquare_out = NULL) {
  hipGraph_t graph;
  hipStream_t stream;

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  checkCudaErrors(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  doNegateSquaresInStream(stream, hostArrays, d_negSquare_out);
  checkCudaErrors(hipStreamEndCapture(stream, &graph));

  checkCudaErrors(hipGraphInstantiate(graphExec, graph, NULL, NULL, 0));
  checkCudaErrors(hipStreamDestroy(stream));
  checkCudaErrors(hipGraphDestroy(graph));
}

void prepareRefArrays(negSquareArrays *hostArrays,
                      negSquareArrays *deviceRefArrays,
                      bool **foundValidationFailure) {
  deviceRefArrays->bytes = hostArrays->bytes;
  deviceRefArrays->numElements = hostArrays->numElements;

  for (int i = 0; i < hostArrays->numElements; i++) {
    hostArrays->square[i] = hostArrays->input[i] * hostArrays->input[i];
    hostArrays->negSquare[i] = hostArrays->square[i] * -1;
  }

  checkCudaErrors(
      hipMalloc((void **)&deviceRefArrays->negSquare, deviceRefArrays->bytes));
  checkCudaErrors(hipMemcpy(deviceRefArrays->negSquare, hostArrays->negSquare,
                             hostArrays->bytes, hipMemcpyHostToDevice));

  checkCudaErrors(
      hipMallocManaged((void **)foundValidationFailure, sizeof(bool)));
}

int checkValidationFailure(bool *foundValidationFailure) {
  if (*foundValidationFailure) {
    printf("Validation FAILURE!\n\n");
    *foundValidationFailure = false;
    return EXIT_FAILURE;
  } else {
    printf("Validation PASSED!\n\n");
    return EXIT_SUCCESS;
  }
}

__global__ void validateGPU(float *d_negSquare, negSquareArrays devRefArrays,
                            bool *foundValidationFailure) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  float ref, diff;

  if (idx < devRefArrays.numElements) {
    ref = devRefArrays.negSquare[idx];
    diff = d_negSquare[idx] - ref;
    diff *= diff;
    ref *= ref;
    if (diff / ref > ALLOWABLE_VARIANCE) {
      *foundValidationFailure = true;
    }
  }
}

void validateHost(negSquareArrays *hostArrays, bool *foundValidationFailure) {
  float ref, diff;

  for (int i = 0; i < hostArrays->numElements; i++) {
    ref = hostArrays->input[i] * hostArrays->input[i] * -1;
    diff = hostArrays->negSquare[i] - ref;
    diff *= diff;
    ref *= ref;
    if (diff / ref > ALLOWABLE_VARIANCE) {
      *foundValidationFailure = true;
    }
  }
}

int main(int argc, char **argv) {
  negSquareArrays hostArrays, deviceRefArrays;
  hipStream_t stream;
  hipGraphExec_t graphExec, graphExecFreeC;

  // Declare pointers for GPU buffers
  float *d_negSquare = NULL;
  bool *foundValidationFailure = NULL;

  srand(time(0));
  int device = findCudaDevice(argc, (const char **)argv);

  int driverVersion = 0;
  int deviceSupportsMemoryPools = 0;

  hipDriverGetVersion(&driverVersion);
  printf("Driver version is: %d.%d\n", driverVersion / 1000,
         (driverVersion % 100) / 10);

  if (driverVersion < 11040) {
    printf("Waiving execution as driver does not support Graph Memory Nodes\n");
    exit(EXIT_WAIVED);
  }

  hipDeviceGetAttribute(&deviceSupportsMemoryPools,
                         hipDeviceAttributeMemoryPoolsSupported, device);
  if (!deviceSupportsMemoryPools) {
    printf("Waiving execution as device does not support Memory Pools\n");
    exit(EXIT_WAIVED);
  } else {
    printf("Setting up sample.\n");
  }

  prepareHostArrays(&hostArrays);
  prepareRefArrays(&hostArrays, &deviceRefArrays, &foundValidationFailure);
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  printf("Setup complete.\n\n");

  printf("Running negateSquares in a stream.\n");
  doNegateSquaresInStream(stream, &hostArrays);
  checkCudaErrors(hipStreamSynchronize(stream));
  printf("Validating negateSquares in a stream...\n");
  validateHost(&hostArrays, foundValidationFailure);
  checkValidationFailure(foundValidationFailure);
  resetOutputArrays(&hostArrays);

  printf("Running negateSquares in a stream-captured graph.\n");
  createNegateSquaresGraphWithStreamCapture(&graphExec, &hostArrays);
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));
  printf("Validating negateSquares in a stream-captured graph...\n");
  validateHost(&hostArrays, foundValidationFailure);
  checkValidationFailure(foundValidationFailure);
  resetOutputArrays(&hostArrays);

  printf("Running negateSquares in an explicitly constructed graph.\n");
  createNegateSquaresGraphExplicitly(&graphExec, device, &hostArrays);
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  checkCudaErrors(hipStreamSynchronize(stream));
  printf("Validating negateSquares in an explicitly constructed graph...\n");
  validateHost(&hostArrays, foundValidationFailure);
  checkValidationFailure(foundValidationFailure);
  resetOutputArrays(&hostArrays);

  // Each of the three examples below free d_negSquare outside the graph. As
  // demonstrated by validateGPU, d_negSquare can be accessed by outside the
  // graph before d_negSquare is freed.

  printf("Running negateSquares with d_negSquare freed outside the stream.\n");
  createNegateSquaresGraphExplicitly(&graphExec, device, &hostArrays,
                                     &d_negSquare);
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  validateGPU<<<hostArrays.numBlocks, THREADS_PER_BLOCK, 0, stream>>>(
      d_negSquare, deviceRefArrays, foundValidationFailure);
  // Since hipFree is synchronous, the stream must synchronize before freeing
  // d_negSquare to ensure d_negSquare no longer being accessed.
  checkCudaErrors(hipStreamSynchronize(stream));
  checkCudaErrors(hipFree(d_negSquare));
  printf(
      "Validating negateSquares with d_negSquare freed outside the "
      "stream...\n");
  validateHost(&hostArrays, foundValidationFailure);
  checkValidationFailure(foundValidationFailure);
  resetOutputArrays(&hostArrays);

  printf("Running negateSquares with d_negSquare freed outside the graph.\n");
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  validateGPU<<<hostArrays.numBlocks, THREADS_PER_BLOCK, 0, stream>>>(
      d_negSquare, deviceRefArrays, foundValidationFailure);
  checkCudaErrors(hipFreeAsync(d_negSquare, stream));
  checkCudaErrors(hipStreamSynchronize(stream));
  printf(
      "Validating negateSquares with d_negSquare freed outside the graph...\n");
  checkValidationFailure(foundValidationFailure);
  resetOutputArrays(&hostArrays);

  printf(
      "Running negateSquares with d_negSquare freed in a different graph.\n");
  createFreeGraph(&graphExecFreeC, d_negSquare);
  checkCudaErrors(hipGraphLaunch(graphExec, stream));
  validateGPU<<<hostArrays.numBlocks, THREADS_PER_BLOCK, 0, stream>>>(
      d_negSquare, deviceRefArrays, foundValidationFailure);
  checkCudaErrors(hipGraphLaunch(graphExecFreeC, stream));
  checkCudaErrors(hipStreamSynchronize(stream));
  printf(
      "Validating negateSquares with d_negSquare freed in a different "
      "graph...\n");
  checkValidationFailure(foundValidationFailure);

  printf("Cleaning up sample.\n");
  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphExecDestroy(graphExecFreeC));
  checkCudaErrors(hipStreamDestroy(stream));
  checkCudaErrors(hipFree(foundValidationFailure));
  checkCudaErrors(hipFree(deviceRefArrays.negSquare));
  free(hostArrays.input);
  free(hostArrays.square);
  free(hostArrays.negSquare);
  printf("Cleanup complete. Exiting sample.\n");
}