#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////////////
// A structure of 2D points (structure of arrays).
////////////////////////////////////////////////////////////////////////////////
class Points {
  float *m_x;
  float *m_y;

 public:
  // Constructor.
  __host__ __device__ Points() : m_x(NULL), m_y(NULL) {}

  // Constructor.
  __host__ __device__ Points(float *x, float *y) : m_x(x), m_y(y) {}

  // Get a point.
  __host__ __device__ __forceinline__ float2 get_point(int idx) const {
    return make_float2(m_x[idx], m_y[idx]);
  }

  // Set a point.
  __host__ __device__ __forceinline__ void set_point(int idx, const float2 &p) {
    m_x[idx] = p.x;
    m_y[idx] = p.y;
  }

  // Set the pointers.
  __host__ __device__ __forceinline__ void set(float *x, float *y) {
    m_x = x;
    m_y = y;
  }
};

////////////////////////////////////////////////////////////////////////////////
// A 2D bounding box
////////////////////////////////////////////////////////////////////////////////
class Bounding_box {
  // Extreme points of the bounding box.
  float2 m_p_min;
  float2 m_p_max;

 public:
  // Constructor. Create a unit box.
  __host__ __device__ Bounding_box() {
    m_p_min = make_float2(0.0f, 0.0f);
    m_p_max = make_float2(1.0f, 1.0f);
  }

  // Compute the center of the bounding-box.
  __host__ __device__ void compute_center(float2 &center) const {
    center.x = 0.5f * (m_p_min.x + m_p_max.x);
    center.y = 0.5f * (m_p_min.y + m_p_max.y);
  }

  // The points of the box.
  __host__ __device__ __forceinline__ const float2 &get_max() const {
    return m_p_max;
  }

  __host__ __device__ __forceinline__ const float2 &get_min() const {
    return m_p_min;
  }

  // Does a box contain a point.
  __host__ __device__ bool contains(const float2 &p) const {
    return p.x >= m_p_min.x && p.x < m_p_max.x && p.y >= m_p_min.y &&
           p.y < m_p_max.y;
  }

  // Define the bounding box.
  __host__ __device__ void set(float min_x, float min_y, float max_x,
                               float max_y) {
    m_p_min.x = min_x;
    m_p_min.y = min_y;
    m_p_max.x = max_x;
    m_p_max.y = max_y;
  }
};

////////////////////////////////////////////////////////////////////////////////
// A node of a quadree.
////////////////////////////////////////////////////////////////////////////////
class Quadtree_node {
  // The identifier of the node.
  int m_id;
  // The bounding box of the tree.
  Bounding_box m_bounding_box;
  // The range of points.
  int m_begin, m_end;

 public:
  // Constructor.
  __host__ __device__ Quadtree_node() : m_id(0), m_begin(0), m_end(0) {}

  // The ID of a node at its level.
  __host__ __device__ int id() const { return m_id; }

  // The ID of a node at its level.
  __host__ __device__ void set_id(int new_id) { m_id = new_id; }

  // The bounding box.
  __host__ __device__ __forceinline__ const Bounding_box &bounding_box() const {
    return m_bounding_box;
  }

  // Set the bounding box.
  __host__ __device__ __forceinline__ void set_bounding_box(float min_x,
                                                            float min_y,
                                                            float max_x,
                                                            float max_y) {
    m_bounding_box.set(min_x, min_y, max_x, max_y);
  }

  // The number of points in the tree.
  __host__ __device__ __forceinline__ int num_points() const {
    return m_end - m_begin;
  }

  // The range of points in the tree.
  __host__ __device__ __forceinline__ int points_begin() const {
    return m_begin;
  }

  __host__ __device__ __forceinline__ int points_end() const { return m_end; }

  // Define the range for that node.
  __host__ __device__ __forceinline__ void set_range(int begin, int end) {
    m_begin = begin;
    m_end = end;
  }
};

////////////////////////////////////////////////////////////////////////////////
// Algorithm parameters.
////////////////////////////////////////////////////////////////////////////////
struct Parameters {
  // Choose the right set of points to use as in/out.
  int point_selector;
  // The number of nodes at a given level (2^k for level k).
  int num_nodes_at_this_level;
  // The recursion depth.
  int depth;
  // The max value for depth.
  const int max_depth;
  // The minimum number of points in a node to stop recursion.
  const int min_points_per_node;

  // Constructor set to default values.
  __host__ __device__ Parameters(int max_depth, int min_points_per_node)
      : point_selector(0),
        num_nodes_at_this_level(1),
        depth(0),
        max_depth(max_depth),
        min_points_per_node(min_points_per_node) {}

  // Copy constructor. Changes the values for next iteration.
  __host__ __device__ Parameters(const Parameters &params, bool)
      : point_selector((params.point_selector + 1) % 2),
        num_nodes_at_this_level(4 * params.num_nodes_at_this_level),
        depth(params.depth + 1),
        max_depth(params.max_depth),
        min_points_per_node(params.min_points_per_node) {}
};

////////////////////////////////////////////////////////////////////////////////
// Build a quadtree on the GPU. Use CUDA Dynamic Parallelism.
//
// The algorithm works as follows. The host (CPU) launches one block of
// NUM_THREADS_PER_BLOCK threads. That block will do the following steps:
//
// 1- Check the number of points and its depth.
//
// We impose a maximum depth to the tree and a minimum number of points per
// node. If the maximum depth is exceeded or the minimum number of points is
// reached. The threads in the block exit.
//
// Before exiting, they perform a buffer swap if it is needed. Indeed, the
// algorithm uses two buffers to permute the points and make sure they are
// properly distributed in the quadtree. By design we want all points to be
// in the first buffer of points at the end of the algorithm. It is the reason
// why we may have to swap the buffer before leavin (if the points are in the
// 2nd buffer).
//
// 2- Count the number of points in each child.
//
// If the depth is not too high and the number of points is sufficient, the
// block has to dispatch the points into four geometrical buckets: Its
// children. For that purpose, we compute the center of the bounding box and
// count the number of points in each quadrant.
//
// The set of points is divided into sections. Each section is given to a
// warp of threads (32 threads). Warps use __ballot and __popc intrinsics
// to count the points. See the Programming Guide for more information about
// those functions.
//
// 3- Scan the warps' results to know the "global" numbers.
//
// Warps work independently from each other. At the end, each warp knows the
// number of points in its section. To know the numbers for the block, the
// block has to run a scan/reduce at the block level. It's a traditional
// approach. The implementation in that sample is not as optimized as what
// could be found in fast radix sorts, for example, but it relies on the same
// idea.
//
// 4- Move points.
//
// Now that the block knows how many points go in each of its 4 children, it
// remains to dispatch the points. It is straightforward.
//
// 5- Launch new blocks.
//
// The block launches four new blocks: One per children. Each of the four blocks
// will apply the same algorithm.
////////////////////////////////////////////////////////////////////////////////
template <int NUM_THREADS_PER_BLOCK>
__global__ void build_quadtree_kernel(Quadtree_node *nodes, Points *points,
                                      Parameters params) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  // The number of warps in a block.
  const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / warpSize;

  // Shared memory to store the number of points.
  extern __shared__ int smem[];

  // s_num_pts[4][NUM_WARPS_PER_BLOCK];
  // Addresses of shared memory.
  volatile int *s_num_pts[4];

  for (int i = 0; i < 4; ++i)
    s_num_pts[i] = (volatile int *)&smem[i * NUM_WARPS_PER_BLOCK];

  // Compute the coordinates of the threads in the block.
  const int warp_id = threadIdx.x / warpSize;
  const int lane_id = threadIdx.x % warpSize;

  // Mask for compaction.
  // Same as: asm( "mov.u32 %0, %%lanemask_lt;" : "=r"(lane_mask_lt) );
  int lane_mask_lt = (1 << lane_id) - 1;

  // The current node.
  Quadtree_node &node = nodes[blockIdx.x];

  // The number of points in the node.
  int num_points = node.num_points();

  float2 center;
  int range_begin, range_end;
  int warp_cnts[4] = {0, 0, 0, 0};
  //
  // 1- Check the number of points and its depth.
  //

  // Stop the recursion here. Make sure points[0] contains all the points.
  if (params.depth >= params.max_depth ||
      num_points <= params.min_points_per_node) {
    if (params.point_selector == 1) {
      int it = node.points_begin(), end = node.points_end();

      for (it += threadIdx.x; it < end; it += NUM_THREADS_PER_BLOCK)
        if (it < end) points[0].set_point(it, points[1].get_point(it));
    }

    return;
  }

  // Compute the center of the bounding box of the points.
  const Bounding_box &bbox = node.bounding_box();

  bbox.compute_center(center);

  // Find how many points to give to each warp.
  int num_points_per_warp = max(
      warpSize, (num_points + NUM_WARPS_PER_BLOCK - 1) / NUM_WARPS_PER_BLOCK);

  // Each warp of threads will compute the number of points to move to each
  // quadrant.
  range_begin = node.points_begin() + warp_id * num_points_per_warp;
  range_end = min(range_begin + num_points_per_warp, node.points_end());

  //
  // 2- Count the number of points in each child.
  //

  // Input points.
  const Points &in_points = points[params.point_selector];

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
  // Compute the number of points.
  for (int range_it = range_begin + tile32.thread_rank();
       tile32.any(range_it < range_end); range_it += warpSize) {
    // Is it still an active thread?
    bool is_active = range_it < range_end;

    // Load the coordinates of the point.
    float2 p =
        is_active ? in_points.get_point(range_it) : make_float2(0.0f, 0.0f);

    // Count top-left points.
    int num_pts =
        __popc(tile32.ballot(is_active && p.x < center.x && p.y >= center.y));
    warp_cnts[0] += tile32.shfl(num_pts, 0);

    // Count top-right points.
    num_pts =
        __popc(tile32.ballot(is_active && p.x >= center.x && p.y >= center.y));
    warp_cnts[1] += tile32.shfl(num_pts, 0);

    // Count bottom-left points.
    num_pts =
        __popc(tile32.ballot(is_active && p.x < center.x && p.y < center.y));
    warp_cnts[2] += tile32.shfl(num_pts, 0);

    // Count bottom-right points.
    num_pts =
        __popc(tile32.ballot(is_active && p.x >= center.x && p.y < center.y));
    warp_cnts[3] += tile32.shfl(num_pts, 0);
  }

  if (tile32.thread_rank() == 0) {
    s_num_pts[0][warp_id] = warp_cnts[0];
    s_num_pts[1][warp_id] = warp_cnts[1];
    s_num_pts[2][warp_id] = warp_cnts[2];
    s_num_pts[3][warp_id] = warp_cnts[3];
  }

  // Make sure warps have finished counting.
  cg::sync(cta);

  //
  // 3- Scan the warps' results to know the "global" numbers.
  //

  // First 4 warps scan the numbers of points per child (inclusive scan).
  if (warp_id < 4) {
    int num_pts = tile32.thread_rank() < NUM_WARPS_PER_BLOCK
                      ? s_num_pts[warp_id][tile32.thread_rank()]
                      : 0;
#pragma unroll

    for (int offset = 1; offset < NUM_WARPS_PER_BLOCK; offset *= 2) {
      int n = tile32.shfl_up(num_pts, offset);

      if (tile32.thread_rank() >= offset) num_pts += n;
    }

    if (tile32.thread_rank() < NUM_WARPS_PER_BLOCK)
      s_num_pts[warp_id][tile32.thread_rank()] = num_pts;
  }

  cg::sync(cta);

  // Compute global offsets.
  if (warp_id == 0) {
    int sum = s_num_pts[0][NUM_WARPS_PER_BLOCK - 1];

    for (int row = 1; row < 4; ++row) {
      int tmp = s_num_pts[row][NUM_WARPS_PER_BLOCK - 1];
      cg::sync(tile32);

      if (tile32.thread_rank() < NUM_WARPS_PER_BLOCK)
        s_num_pts[row][tile32.thread_rank()] += sum;

      cg::sync(tile32);
      sum += tmp;
    }
  }

  cg::sync(cta);

  // Make the scan exclusive.
  int val = 0;
  if (threadIdx.x < 4 * NUM_WARPS_PER_BLOCK) {
    val = threadIdx.x == 0 ? 0 : smem[threadIdx.x - 1];
    val += node.points_begin();
  }

  cg::sync(cta);

  if (threadIdx.x < 4 * NUM_WARPS_PER_BLOCK) {
    smem[threadIdx.x] = val;
  }

  cg::sync(cta);

  //
  // 4- Move points.
  //
  if (!(params.depth >= params.max_depth ||
        num_points <= params.min_points_per_node)) {
    // Output points.
    Points &out_points = points[(params.point_selector + 1) % 2];

    warp_cnts[0] = s_num_pts[0][warp_id];
    warp_cnts[1] = s_num_pts[1][warp_id];
    warp_cnts[2] = s_num_pts[2][warp_id];
    warp_cnts[3] = s_num_pts[3][warp_id];

    const Points &in_points = points[params.point_selector];
    // Reorder points.
    for (int range_it = range_begin + tile32.thread_rank();
         tile32.any(range_it < range_end); range_it += warpSize) {
      // Is it still an active thread?
      bool is_active = range_it < range_end;

      // Load the coordinates of the point.
      float2 p =
          is_active ? in_points.get_point(range_it) : make_float2(0.0f, 0.0f);

      // Count top-left points.
      bool pred = is_active && p.x < center.x && p.y >= center.y;
      int vote = tile32.ballot(pred);
      int dest = warp_cnts[0] + __popc(vote & lane_mask_lt);

      if (pred) out_points.set_point(dest, p);

      warp_cnts[0] += tile32.shfl(__popc(vote), 0);

      // Count top-right points.
      pred = is_active && p.x >= center.x && p.y >= center.y;
      vote = tile32.ballot(pred);
      dest = warp_cnts[1] + __popc(vote & lane_mask_lt);

      if (pred) out_points.set_point(dest, p);

      warp_cnts[1] += tile32.shfl(__popc(vote), 0);

      // Count bottom-left points.
      pred = is_active && p.x < center.x && p.y < center.y;
      vote = tile32.ballot(pred);
      dest = warp_cnts[2] + __popc(vote & lane_mask_lt);

      if (pred) out_points.set_point(dest, p);

      warp_cnts[2] += tile32.shfl(__popc(vote), 0);

      // Count bottom-right points.
      pred = is_active && p.x >= center.x && p.y < center.y;
      vote = tile32.ballot(pred);
      dest = warp_cnts[3] + __popc(vote & lane_mask_lt);

      if (pred) out_points.set_point(dest, p);

      warp_cnts[3] += tile32.shfl(__popc(vote), 0);
    }
  }

  cg::sync(cta);

  if (tile32.thread_rank() == 0) {
    s_num_pts[0][warp_id] = warp_cnts[0];
    s_num_pts[1][warp_id] = warp_cnts[1];
    s_num_pts[2][warp_id] = warp_cnts[2];
    s_num_pts[3][warp_id] = warp_cnts[3];
  }

  cg::sync(cta);

  //
  // 5- Launch new blocks.
  //
  if (!(params.depth >= params.max_depth ||
        num_points <= params.min_points_per_node)) {
    // The last thread launches new blocks.
    if (threadIdx.x == NUM_THREADS_PER_BLOCK - 1) {
      // The children.
      Quadtree_node *children =
          &nodes[params.num_nodes_at_this_level - (node.id() & ~3)];

      // The offsets of the children at their level.
      int child_offset = 4 * node.id();

      // Set IDs.
      children[child_offset + 0].set_id(4 * node.id() + 0);
      children[child_offset + 1].set_id(4 * node.id() + 1);
      children[child_offset + 2].set_id(4 * node.id() + 2);
      children[child_offset + 3].set_id(4 * node.id() + 3);

      const Bounding_box &bbox = node.bounding_box();
      // Points of the bounding-box.
      const float2 &p_min = bbox.get_min();
      const float2 &p_max = bbox.get_max();

      // Set the bounding boxes of the children.
      children[child_offset + 0].set_bounding_box(p_min.x, center.y, center.x,
                                                  p_max.y);  // Top-left.
      children[child_offset + 1].set_bounding_box(center.x, center.y, p_max.x,
                                                  p_max.y);  // Top-right.
      children[child_offset + 2].set_bounding_box(p_min.x, p_min.y, center.x,
                                                  center.y);  // Bottom-left.
      children[child_offset + 3].set_bounding_box(center.x, p_min.y, p_max.x,
                                                  center.y);  // Bottom-right.

      // Set the ranges of the children.

      children[child_offset + 0].set_range(node.points_begin(),
                                           s_num_pts[0][warp_id]);
      children[child_offset + 1].set_range(s_num_pts[0][warp_id],
                                           s_num_pts[1][warp_id]);
      children[child_offset + 2].set_range(s_num_pts[1][warp_id],
                                           s_num_pts[2][warp_id]);
      children[child_offset + 3].set_range(s_num_pts[2][warp_id],
                                           s_num_pts[3][warp_id]);

      // Launch 4 children.
      build_quadtree_kernel<NUM_THREADS_PER_BLOCK><<<
          4, NUM_THREADS_PER_BLOCK, 4 * NUM_WARPS_PER_BLOCK * sizeof(int)>>>(
          &children[child_offset], points, Parameters(params, true));
    }
  }
}

////////////////////////////////////////////////////////////////////////////////
// Make sure a Quadtree is properly defined.
////////////////////////////////////////////////////////////////////////////////
bool check_quadtree(const Quadtree_node *nodes, int idx, int num_pts,
                    Points *pts, Parameters params) {
  const Quadtree_node &node = nodes[idx];
  int num_points = node.num_points();

  if (!(params.depth == params.max_depth ||
        num_points <= params.min_points_per_node)) {
    int num_points_in_children = 0;

    num_points_in_children +=
        nodes[params.num_nodes_at_this_level + 4 * idx + 0].num_points();
    num_points_in_children +=
        nodes[params.num_nodes_at_this_level + 4 * idx + 1].num_points();
    num_points_in_children +=
        nodes[params.num_nodes_at_this_level + 4 * idx + 2].num_points();
    num_points_in_children +=
        nodes[params.num_nodes_at_this_level + 4 * idx + 3].num_points();

    if (num_points_in_children != node.num_points()) return false;

    return check_quadtree(&nodes[params.num_nodes_at_this_level], 4 * idx + 0,
                          num_pts, pts, Parameters(params, true)) &&
           check_quadtree(&nodes[params.num_nodes_at_this_level], 4 * idx + 1,
                          num_pts, pts, Parameters(params, true)) &&
           check_quadtree(&nodes[params.num_nodes_at_this_level], 4 * idx + 2,
                          num_pts, pts, Parameters(params, true)) &&
           check_quadtree(&nodes[params.num_nodes_at_this_level], 4 * idx + 3,
                          num_pts, pts, Parameters(params, true));
  }

  const Bounding_box &bbox = node.bounding_box();

  for (int it = node.points_begin(); it < node.points_end(); ++it) {
    if (it >= num_pts) return false;

    float2 p = pts->get_point(it);

    if (!bbox.contains(p)) return false;
  }

  return true;
}

////////////////////////////////////////////////////////////////////////////////
// Parallel random number generator.
////////////////////////////////////////////////////////////////////////////////
struct Random_generator {
  int count;

  __host__ __device__ Random_generator() : count(0) {}
  __host__ __device__ unsigned int hash(unsigned int a) {
    a = (a + 0x7ed55d16) + (a << 12);
    a = (a ^ 0xc761c23c) ^ (a >> 19);
    a = (a + 0x165667b1) + (a << 5);
    a = (a + 0xd3a2646c) ^ (a << 9);
    a = (a + 0xfd7046c5) + (a << 3);
    a = (a ^ 0xb55a4f09) ^ (a >> 16);
    return a;
  }

  __host__ __device__ __forceinline__ thrust::tuple<float, float> operator()() {
#ifdef __CUDA_ARCH__
    unsigned seed = hash(blockIdx.x * blockDim.x + threadIdx.x + count);
    // thrust::generate may call operator() more than once per thread.
    // Hence, increment count by grid size to ensure uniqueness of seed
    count += blockDim.x * gridDim.x;
#else
    unsigned seed = hash(0);
#endif
    thrust::default_random_engine rng(seed);
    thrust::random::uniform_real_distribution<float> distrib;
    return thrust::make_tuple(distrib(rng), distrib(rng));
  }
};

////////////////////////////////////////////////////////////////////////////////
// Allocate GPU structs, launch kernel and clean up
////////////////////////////////////////////////////////////////////////////////
bool cdpQuadtree(int warp_size) {
  // Constants to control the algorithm.
  const int num_points = 1024;
  const int max_depth = 8;
  const int min_points_per_node = 16;

  // Allocate memory for points.
  thrust::device_vector<float> x_d0(num_points);
  thrust::device_vector<float> x_d1(num_points);
  thrust::device_vector<float> y_d0(num_points);
  thrust::device_vector<float> y_d1(num_points);

  // Generate random points.
  Random_generator rnd;
  thrust::generate(
      thrust::make_zip_iterator(thrust::make_tuple(x_d0.begin(), y_d0.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(x_d0.end(), y_d0.end())),
      rnd);

  // Host structures to analyze the device ones.
  Points points_init[2];
  points_init[0].set(thrust::raw_pointer_cast(&x_d0[0]),
                     thrust::raw_pointer_cast(&y_d0[0]));
  points_init[1].set(thrust::raw_pointer_cast(&x_d1[0]),
                     thrust::raw_pointer_cast(&y_d1[0]));

  // Allocate memory to store points.
  Points *points;
  checkCudaErrors(hipMalloc((void **)&points, 2 * sizeof(Points)));
  checkCudaErrors(hipMemcpy(points, points_init, 2 * sizeof(Points),
                             hipMemcpyHostToDevice));

  // We could use a close form...
  int max_nodes = 0;

  for (int i = 0, num_nodes_at_level = 1; i < max_depth;
       ++i, num_nodes_at_level *= 4)
    max_nodes += num_nodes_at_level;

  // Allocate memory to store the tree.
  Quadtree_node root;
  root.set_range(0, num_points);
  Quadtree_node *nodes;
  checkCudaErrors(
      hipMalloc((void **)&nodes, max_nodes * sizeof(Quadtree_node)));
  checkCudaErrors(
      hipMemcpy(nodes, &root, sizeof(Quadtree_node), hipMemcpyHostToDevice));

  // We set the recursion limit for CDP to max_depth.
  hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, max_depth);

  // Build the quadtree.
  Parameters params(max_depth, min_points_per_node);
  std::cout << "Launching CDP kernel to build the quadtree" << std::endl;
  const int NUM_THREADS_PER_BLOCK = 128;  // Do not use less than 128 threads.
  const int NUM_WARPS_PER_BLOCK = NUM_THREADS_PER_BLOCK / warp_size;
  const size_t smem_size = 4 * NUM_WARPS_PER_BLOCK * sizeof(int);
  build_quadtree_kernel<
      NUM_THREADS_PER_BLOCK><<<1, NUM_THREADS_PER_BLOCK, smem_size>>>(
      nodes, points, params);
  checkCudaErrors(hipGetLastError());

  // Copy points to CPU.
  thrust::host_vector<float> x_h(x_d0);
  thrust::host_vector<float> y_h(y_d0);
  Points host_points;
  host_points.set(thrust::raw_pointer_cast(&x_h[0]),
                  thrust::raw_pointer_cast(&y_h[0]));

  // Copy nodes to CPU.
  Quadtree_node *host_nodes = new Quadtree_node[max_nodes];
  checkCudaErrors(hipMemcpy(host_nodes, nodes,
                             max_nodes * sizeof(Quadtree_node),
                             hipMemcpyDeviceToHost));

  // Validate the results.
  bool ok = check_quadtree(host_nodes, 0, num_points, &host_points, params);
  std::cout << "Results: " << (ok ? "OK" : "FAILED") << std::endl;

  // Free CPU memory.
  delete[] host_nodes;

  // Free memory.
  checkCudaErrors(hipFree(nodes));
  checkCudaErrors(hipFree(points));

  return ok;
}

////////////////////////////////////////////////////////////////////////////////
// Main entry point.
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  // Find/set the device.
  // The test requires an architecture SM35 or greater (CDP capable).
  int cuda_device = findCudaDevice(argc, (const char **)argv);
  hipDeviceProp_t deviceProps;
  checkCudaErrors(hipGetDeviceProperties(&deviceProps, cuda_device));
  int cdpCapable = (deviceProps.major == 3 && deviceProps.minor >= 5) ||
                   deviceProps.major >= 4;

  printf("GPU device %s has compute capabilities (SM %d.%d)\n",
         deviceProps.name, deviceProps.major, deviceProps.minor);

  if (!cdpCapable) {
    std::cerr << "cdpQuadTree requires SM 3.5 or higher to use CUDA Dynamic "
                 "Parallelism.  Exiting...\n"
              << std::endl;
    exit(EXIT_WAIVED);
  }

  bool ok = cdpQuadtree(deviceProps.warpSize);

  return (ok ? EXIT_SUCCESS : EXIT_FAILURE);
}
