#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <string.h>

__forceinline__ __device__ float2 operator+(float2 a, float2 b) {
  float2 c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

__forceinline__ __device__ float2 operator-(float2 a, float2 b) {
  float2 c;
  c.x = a.x - b.x;
  c.y = a.y - b.y;
  return c;
}

__forceinline__ __device__ float2 operator*(float a, float2 b) {
  float2 c;
  c.x = a * b.x;
  c.y = a * b.y;
  return c;
}

__forceinline__ __device__ float length(float2 a) {
  return sqrtf(a.x * a.x + a.y * a.y);
}

#define MAX_TESSELLATION 32
struct BezierLine {
  float2 CP[3];
  float2 *vertexPos;
  int nVertices;
};

__global__ void computeBezierLinePositions(int lidx, BezierLine *bLines,
                                           int nTessPoints) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;

  if (idx < nTessPoints) {
    float u = (float)idx / (float)(nTessPoints - 1);
    float omu = 1.0f - u;

    float B3u[3];

    B3u[0] = omu * omu;
    B3u[1] = 2.0f * u * omu;
    B3u[2] = u * u;

    float2 position = {0, 0};

    for (int i = 0; i < 3; i++) {
      position = position + B3u[i] * bLines[lidx].CP[i];
    }

    bLines[lidx].vertexPos[idx] = position;
  }
}

__global__ void computeBezierLinesCDP(BezierLine *bLines, int nLines) {
  int lidx = threadIdx.x + blockDim.x * blockIdx.x;

  if (lidx < nLines) {
    float curvature = length(bLines[lidx].CP[1] -
                             0.5f * (bLines[lidx].CP[0] + bLines[lidx].CP[2])) /
                      length(bLines[lidx].CP[2] - bLines[lidx].CP[0]);
    int nTessPoints = min(max((int)(curvature * 16.0f), 4), MAX_TESSELLATION);

    if (bLines[lidx].vertexPos == NULL) {
      bLines[lidx].nVertices = nTessPoints;
      hipMalloc((void **)&bLines[lidx].vertexPos,
                 nTessPoints * sizeof(float2));
    }

    computeBezierLinePositions<<<ceilf((float)bLines[lidx].nVertices / 32.0f),
                                 32>>>(lidx, bLines, bLines[lidx].nVertices);
  }
}

__global__ void freeVertexMem(BezierLine *bLines, int nLines) {
  int lidx = threadIdx.x + blockDim.x * blockIdx.x;

  if (lidx < nLines) hipFree(bLines[lidx].vertexPos);
}

unsigned int checkCapableSM35Device(int argc, char **argv) {
  // Get device properties
  hipDeviceProp_t properties;
  int device_count = 0, device = -1;

  if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
    device = getCmdLineArgumentInt(argc, (const char **)argv, "device");

    hipDeviceProp_t properties;
    checkCudaErrors(hipGetDeviceProperties(&properties, device));

    if (properties.major > 3 ||
        (properties.major == 3 && properties.minor >= 5)) {
      printf("Running on GPU  %d (%s)\n", device, properties.name);
    } else {
      printf(
          "cdpBezierTessellation requires GPU devices with compute SM 3.5 or "
          "higher.");
      printf("Current GPU device has compute SM %d.%d. Exiting...\n",
             properties.major, properties.minor);
      return EXIT_FAILURE;
    }

  } else {
    checkCudaErrors(hipGetDeviceCount(&device_count));

    for (int i = 0; i < device_count; ++i) {
      checkCudaErrors(hipGetDeviceProperties(&properties, i));

      if (properties.major > 3 ||
          (properties.major == 3 && properties.minor >= 5)) {
        device = i;
        printf("Running on GPU %d (%s)\n", i, properties.name);
        break;
      }

      printf("GPU %d %s does not support CUDA Dynamic Parallelism\n", i,
             properties.name);
    }
  }
  if (device == -1) {
    fprintf(stderr,
            "cdpBezierTessellation requires GPU devices with compute SM 3.5 or "
            "higher.  Exiting...\n");
    return EXIT_WAIVED;
  }

  return EXIT_SUCCESS;
}

#define N_LINES 256
#define BLOCK_DIM 64
int main(int argc, char **argv) {
  BezierLine *bLines_h = new BezierLine[N_LINES];

  float2 last = {0, 0};

  for (int i = 0; i < N_LINES; i++) {
    bLines_h[i].CP[0] = last;

    for (int j = 1; j < 3; j++) {
      bLines_h[i].CP[j].x = (float)rand() / (float)RAND_MAX;
      bLines_h[i].CP[j].y = (float)rand() / (float)RAND_MAX;
    }

    last = bLines_h[i].CP[2];
    bLines_h[i].vertexPos = NULL;
    bLines_h[i].nVertices = 0;
  }

  unsigned int sm35Ret = checkCapableSM35Device(argc, argv);
  if (sm35Ret != EXIT_SUCCESS) {
    exit(sm35Ret);
  }

  BezierLine *bLines_d;
  checkCudaErrors(hipMalloc((void **)&bLines_d, N_LINES * sizeof(BezierLine)));
  checkCudaErrors(hipMemcpy(bLines_d, bLines_h, N_LINES * sizeof(BezierLine),
                             hipMemcpyHostToDevice));
  printf("Computing Bezier Lines (CUDA Dynamic Parallelism Version) ... ");
  computeBezierLinesCDP<<<(unsigned int)ceil((float)N_LINES / (float)BLOCK_DIM),
                          BLOCK_DIM>>>(bLines_d, N_LINES);
  printf("Done!\n");

  // Do something to draw the lines here

  freeVertexMem<<<(unsigned int)ceil((float)N_LINES / (float)BLOCK_DIM),
                  BLOCK_DIM>>>(bLines_d, N_LINES);
  checkCudaErrors(hipFree(bLines_d));
  delete[] bLines_h;

  exit(EXIT_SUCCESS);
}
