#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <vector>

namespace cg = cooperative_groups;

#define THREADS_PER_BLOCK 512
#define GRAPH_LAUNCH_ITERATIONS 3

typedef struct callBackData {
  const char *fn_name;
  double *data;
} callBackData_t;

__global__ void reduce(float *inputVec, double *outputVec, size_t inputSize,
                       size_t outputSize) {
  __shared__ double tmp[THREADS_PER_BLOCK];

  cg::thread_block cta = cg::this_thread_block();
  size_t globaltid = blockIdx.x * blockDim.x + threadIdx.x;

  double temp_sum = 0.0;
  for (int i = globaltid; i < inputSize; i += gridDim.x * blockDim.x) {
    temp_sum += (double)inputVec[i];
  }
  tmp[cta.thread_rank()] = temp_sum;

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  double beta = temp_sum;
  double temp;

  for (int i = tile32.size() / 2; i > 0; i >>= 1) {
    if (tile32.thread_rank() < i) {
      temp = tmp[cta.thread_rank() + i];
      beta += temp;
      tmp[cta.thread_rank()] = beta;
    }
    cg::sync(tile32);
  }
  cg::sync(cta);

  if (cta.thread_rank() == 0 && blockIdx.x < outputSize) {
    beta = 0.0;
    for (int i = 0; i < cta.size(); i += tile32.size()) {
      beta += tmp[i];
    }
    outputVec[blockIdx.x] = beta;
  }
}

__global__ void reduceFinal(double *inputVec, double *result,
                            size_t inputSize) {
  __shared__ double tmp[THREADS_PER_BLOCK];

  cg::thread_block cta = cg::this_thread_block();
  size_t globaltid = blockIdx.x * blockDim.x + threadIdx.x;

  double temp_sum = 0.0;
  for (int i = globaltid; i < inputSize; i += gridDim.x * blockDim.x) {
    temp_sum += (double)inputVec[i];
  }
  tmp[cta.thread_rank()] = temp_sum;

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  // do reduction in shared mem
  if ((blockDim.x >= 512) && (cta.thread_rank() < 256)) {
    tmp[cta.thread_rank()] = temp_sum = temp_sum + tmp[cta.thread_rank() + 256];
  }

  cg::sync(cta);

  if ((blockDim.x >= 256) && (cta.thread_rank() < 128)) {
    tmp[cta.thread_rank()] = temp_sum = temp_sum + tmp[cta.thread_rank() + 128];
  }

  cg::sync(cta);

  if ((blockDim.x >= 128) && (cta.thread_rank() < 64)) {
    tmp[cta.thread_rank()] = temp_sum = temp_sum + tmp[cta.thread_rank() + 64];
  }

  cg::sync(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockDim.x >= 64) temp_sum += tmp[cta.thread_rank() + 32];
    // Reduce final warp using shuffle
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      temp_sum += tile32.shfl_down(temp_sum, offset);
    }
  }
  // write result for this block to global mem
  if (cta.thread_rank() == 0) result[0] = temp_sum;
}

void init_input(float *a, size_t size) {
  for (size_t i = 0; i < size; i++) a[i] = (rand() & 0xFF) / (float)RAND_MAX;
}

void CUDART_CB myHostNodeCallback(void *data) {
  // Check status of GPU after stream operations are done
  callBackData_t *tmp = (callBackData_t *)(data);
  // checkCudaErrors(tmp->status);

  double *result = (double *)(tmp->data);
  char *function = (char *)(tmp->fn_name);
  printf("[%s] Host callback final reduced sum = %lf\n", function, *result);
  *result = 0.0;  // reset the result
}

void cudaGraphsManual(float *inputVec_h, float *inputVec_d, double *outputVec_d,
                      double *result_d, size_t inputSize, size_t numOfBlocks) {
  hipStream_t streamForGraph;
  hipGraph_t graph;
  std::vector<hipGraphNode_t> nodeDependencies;
  hipGraphNode_t memcpyNode, kernelNode, memsetNode;
  double result_h = 0.0;

  checkCudaErrors(hipStreamCreate(&streamForGraph));

  hipKernelNodeParams kernelNodeParams = {0};
  hipMemcpy3DParms memcpyParams = {0};
  hipMemsetParams memsetParams = {0};

  memcpyParams.srcArray = NULL;
  memcpyParams.srcPos = make_hipPos(0, 0, 0);
  memcpyParams.srcPtr =
      make_hipPitchedPtr(inputVec_h, sizeof(float) * inputSize, inputSize, 1);
  memcpyParams.dstArray = NULL;
  memcpyParams.dstPos = make_hipPos(0, 0, 0);
  memcpyParams.dstPtr =
      make_hipPitchedPtr(inputVec_d, sizeof(float) * inputSize, inputSize, 1);
  memcpyParams.extent = make_hipExtent(sizeof(float) * inputSize, 1, 1);
  memcpyParams.kind = hipMemcpyHostToDevice;

  memsetParams.dst = (void *)outputVec_d;
  memsetParams.value = 0;
  memsetParams.pitch = 0;
  memsetParams.elementSize = sizeof(float);  // elementSize can be max 4 bytes
  memsetParams.width = numOfBlocks * 2;
  memsetParams.height = 1;

  checkCudaErrors(hipGraphCreate(&graph, 0));
  checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, NULL, 0, &memcpyParams));
  checkCudaErrors(
      hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams));

  nodeDependencies.push_back(memsetNode);
  nodeDependencies.push_back(memcpyNode);

  void *kernelArgs[4] = {(void *)&inputVec_d, (void *)&outputVec_d, &inputSize,
                         &numOfBlocks};

  kernelNodeParams.func = (void *)reduce;
  kernelNodeParams.gridDim = dim3(numOfBlocks, 1, 1);
  kernelNodeParams.blockDim = dim3(THREADS_PER_BLOCK, 1, 1);
  kernelNodeParams.sharedMemBytes = 0;
  kernelNodeParams.kernelParams = (void **)kernelArgs;
  kernelNodeParams.extra = NULL;

  checkCudaErrors(
      hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));

  nodeDependencies.clear();
  nodeDependencies.push_back(kernelNode);

  memset(&memsetParams, 0, sizeof(memsetParams));
  memsetParams.dst = result_d;
  memsetParams.value = 0;
  memsetParams.elementSize = sizeof(float);
  memsetParams.width = 2;
  memsetParams.height = 1;
  checkCudaErrors(
      hipGraphAddMemsetNode(&memsetNode, graph, NULL, 0, &memsetParams));

  nodeDependencies.push_back(memsetNode);

  memset(&kernelNodeParams, 0, sizeof(kernelNodeParams));
  kernelNodeParams.func = (void *)reduceFinal;
  kernelNodeParams.gridDim = dim3(1, 1, 1);
  kernelNodeParams.blockDim = dim3(THREADS_PER_BLOCK, 1, 1);
  kernelNodeParams.sharedMemBytes = 0;
  void *kernelArgs2[3] = {(void *)&outputVec_d, (void *)&result_d,
                          &numOfBlocks};
  kernelNodeParams.kernelParams = kernelArgs2;
  kernelNodeParams.extra = NULL;

  checkCudaErrors(
      hipGraphAddKernelNode(&kernelNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &kernelNodeParams));
  nodeDependencies.clear();
  nodeDependencies.push_back(kernelNode);

  memset(&memcpyParams, 0, sizeof(memcpyParams));

  memcpyParams.srcArray = NULL;
  memcpyParams.srcPos = make_hipPos(0, 0, 0);
  memcpyParams.srcPtr = make_hipPitchedPtr(result_d, sizeof(double), 1, 1);
  memcpyParams.dstArray = NULL;
  memcpyParams.dstPos = make_hipPos(0, 0, 0);
  memcpyParams.dstPtr = make_hipPitchedPtr(&result_h, sizeof(double), 1, 1);
  memcpyParams.extent = make_hipExtent(sizeof(double), 1, 1);
  memcpyParams.kind = hipMemcpyDeviceToHost;
  checkCudaErrors(
      hipGraphAddMemcpyNode(&memcpyNode, graph, nodeDependencies.data(),
                             nodeDependencies.size(), &memcpyParams));
  nodeDependencies.clear();
  nodeDependencies.push_back(memcpyNode);

  hipGraphNode_t hostNode;
  hipHostNodeParams hostParams = {0};
  hostParams.fn = myHostNodeCallback;
  callBackData_t hostFnData;
  hostFnData.data = &result_h;
  hostFnData.fn_name = "cudaGraphsManual";
  hostParams.userData = &hostFnData;

  checkCudaErrors(hipGraphAddHostNode(&hostNode, graph,
                                       nodeDependencies.data(),
                                       nodeDependencies.size(), &hostParams));

  hipGraphNode_t *nodes = NULL;
  size_t numNodes = 0;
  checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
  printf("\nNum of nodes in the graph created manually = %zu\n", numNodes);

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

  hipGraph_t clonedGraph;
  hipGraphExec_t clonedGraphExec;
  checkCudaErrors(hipGraphClone(&clonedGraph, graph));
  checkCudaErrors(
      hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0));

  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
  }

  checkCudaErrors(hipStreamSynchronize(streamForGraph));

  printf("Cloned Graph Output.. \n");
  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(clonedGraphExec, streamForGraph));
  }
  checkCudaErrors(hipStreamSynchronize(streamForGraph));

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphExecDestroy(clonedGraphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipGraphDestroy(clonedGraph));
  checkCudaErrors(hipStreamDestroy(streamForGraph));
}

void cudaGraphsUsingStreamCapture(float *inputVec_h, float *inputVec_d,
                                  double *outputVec_d, double *result_d,
                                  size_t inputSize, size_t numOfBlocks) {
  hipStream_t stream1, stream2, stream3, streamForGraph;
  hipEvent_t forkStreamEvent, memsetEvent1, memsetEvent2;
  hipGraph_t graph;
  double result_h = 0.0;

  checkCudaErrors(hipStreamCreate(&stream1));
  checkCudaErrors(hipStreamCreate(&stream2));
  checkCudaErrors(hipStreamCreate(&stream3));
  checkCudaErrors(hipStreamCreate(&streamForGraph));

  checkCudaErrors(hipEventCreate(&forkStreamEvent));
  checkCudaErrors(hipEventCreate(&memsetEvent1));
  checkCudaErrors(hipEventCreate(&memsetEvent2));

  checkCudaErrors(hipStreamBeginCapture(stream1, hipStreamCaptureModeGlobal));

  checkCudaErrors(hipEventRecord(forkStreamEvent, stream1));
  checkCudaErrors(hipStreamWaitEvent(stream2, forkStreamEvent, 0));
  checkCudaErrors(hipStreamWaitEvent(stream3, forkStreamEvent, 0));

  checkCudaErrors(hipMemcpyAsync(inputVec_d, inputVec_h,
                                  sizeof(float) * inputSize, hipMemcpyDefault,
                                  stream1));

  checkCudaErrors(
      hipMemsetAsync(outputVec_d, 0, sizeof(double) * numOfBlocks, stream2));

  checkCudaErrors(hipEventRecord(memsetEvent1, stream2));

  checkCudaErrors(hipMemsetAsync(result_d, 0, sizeof(double), stream3));
  checkCudaErrors(hipEventRecord(memsetEvent2, stream3));

  checkCudaErrors(hipStreamWaitEvent(stream1, memsetEvent1, 0));

  reduce<<<numOfBlocks, THREADS_PER_BLOCK, 0, stream1>>>(
      inputVec_d, outputVec_d, inputSize, numOfBlocks);

  checkCudaErrors(hipStreamWaitEvent(stream1, memsetEvent2, 0));

  reduceFinal<<<1, THREADS_PER_BLOCK, 0, stream1>>>(outputVec_d, result_d,
                                                    numOfBlocks);
  checkCudaErrors(hipMemcpyAsync(&result_h, result_d, sizeof(double),
                                  hipMemcpyDefault, stream1));

  callBackData_t hostFnData = {0};
  hostFnData.data = &result_h;
  hostFnData.fn_name = "cudaGraphsUsingStreamCapture";
  hipHostFn_t fn = myHostNodeCallback;
  checkCudaErrors(hipLaunchHostFunc(stream1, fn, &hostFnData));
  checkCudaErrors(hipStreamEndCapture(stream1, &graph));

  hipGraphNode_t *nodes = NULL;
  size_t numNodes = 0;
  checkCudaErrors(hipGraphGetNodes(graph, nodes, &numNodes));
  printf("\nNum of nodes in the graph created using stream capture API = %zu\n",
         numNodes);

  hipGraphExec_t graphExec;
  checkCudaErrors(hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0));

  hipGraph_t clonedGraph;
  hipGraphExec_t clonedGraphExec;
  checkCudaErrors(hipGraphClone(&clonedGraph, graph));
  checkCudaErrors(
      hipGraphInstantiate(&clonedGraphExec, clonedGraph, NULL, NULL, 0));

  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(graphExec, streamForGraph));
  }

  checkCudaErrors(hipStreamSynchronize(streamForGraph));

  printf("Cloned Graph Output.. \n");
  for (int i = 0; i < GRAPH_LAUNCH_ITERATIONS; i++) {
    checkCudaErrors(hipGraphLaunch(clonedGraphExec, streamForGraph));
  }

  checkCudaErrors(hipStreamSynchronize(streamForGraph));

  checkCudaErrors(hipGraphExecDestroy(graphExec));
  checkCudaErrors(hipGraphExecDestroy(clonedGraphExec));
  checkCudaErrors(hipGraphDestroy(graph));
  checkCudaErrors(hipGraphDestroy(clonedGraph));
  checkCudaErrors(hipStreamDestroy(stream1));
  checkCudaErrors(hipStreamDestroy(stream2));
  checkCudaErrors(hipStreamDestroy(streamForGraph));
}

int main(int argc, char **argv) {
  size_t size = 1 << 24;  // number of elements to reduce
  size_t maxBlocks = 512;

  // This will pick the best possible CUDA capable device
  int devID = findCudaDevice(argc, (const char **)argv);

  printf("%zu elements\n", size);
  printf("threads per block  = %d\n", THREADS_PER_BLOCK);
  printf("Graph Launch iterations = %d\n", GRAPH_LAUNCH_ITERATIONS);

  float *inputVec_d = NULL, *inputVec_h = NULL;
  double *outputVec_d = NULL, *result_d;

  checkCudaErrors(hipHostMalloc(&inputVec_h, sizeof(float) * size));
  checkCudaErrors(hipMalloc(&inputVec_d, sizeof(float) * size));
  checkCudaErrors(hipMalloc(&outputVec_d, sizeof(double) * maxBlocks));
  checkCudaErrors(hipMalloc(&result_d, sizeof(double)));

  init_input(inputVec_h, size);

  cudaGraphsManual(inputVec_h, inputVec_d, outputVec_d, result_d, size,
                   maxBlocks);
  cudaGraphsUsingStreamCapture(inputVec_h, inputVec_d, outputVec_d, result_d,
                               size, maxBlocks);

  checkCudaErrors(hipFree(inputVec_d));
  checkCudaErrors(hipFree(outputVec_d));
  checkCudaErrors(hipFree(result_d));
  checkCudaErrors(hipHostFree(inputVec_h));
  return EXIT_SUCCESS;
}
