#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
  This sample has two kernels, one doing the rendering every frame, and
  another one used to generate the mip map levels at startup.

  For rendering we use a "virtual" texturing approach, where one 2d texture
  stores pointers to the actual textures used. This can be achieved by the
  new cudaTextureObject introduced in CUDA 5.0 and requiring sm3+ hardware.

  The mipmap generation kernel uses cudaSurfaceObject and cudaTextureObject
  passed as kernel arguments to compute the higher mip map level based on
  the lower.
*/

#ifndef _BINDLESSTEXTURE_KERNEL_CU_
#define _BINDLESSTEXTURE_KERNEL_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <vector>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "bindlessTexture.h"

// set this to just see the mipmap chain of first image
//#define SHOW_MIPMAPS

// local references to resources

Image atlasImage;
std::vector<Image> contentImages;
float highestLod = 1.0f;

#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif

//////////////////////////////////////////////////////////////////////////

__host__ __device__ __inline__ uint2 encodeTextureObject(
    hipTextureObject_t obj) {
  return make_uint2((uint)(obj & 0xFFFFFFFF), (uint)(obj >> 32));
}

__host__ __device__ __inline__ hipTextureObject_t decodeTextureObject(
    uint2 obj) {
  return (((hipTextureObject_t)obj.x) | ((hipTextureObject_t)obj.y) << 32);
}

__device__ __inline__ float4 to_float4(uchar4 vec) {
  return make_float4(vec.x, vec.y, vec.z, vec.w);
}

__device__ __inline__ uchar4 to_uchar4(float4 vec) {
  return make_uchar4((uchar)vec.x, (uchar)vec.y, (uchar)vec.z, (uchar)vec.w);
}

//////////////////////////////////////////////////////////////////////////
// Rendering

// the atlas texture stores the 64 bit cudaTextureObjects
// we use it for "virtual" texturing

__global__ void d_render(uchar4 *d_output, uint imageW, uint imageH, float lod,
                         hipTextureObject_t atlasTexture) {
  uint x = blockIdx.x * blockDim.x + threadIdx.x;
  uint y = blockIdx.y * blockDim.y + threadIdx.y;

  float u = x / (float)imageW;
  float v = y / (float)imageH;

  if ((x < imageW) && (y < imageH)) {
    // read from 2D atlas texture and decode texture object
    uint2 texCoded = tex2D<uint2>(atlasTexture, u, v);
    hipTextureObject_t tex = decodeTextureObject(texCoded);

    // read from cuda texture object, use template to specify what data will be
    // returned. tex2DLod allows us to pass the lod (mip map level) directly.
    // There is other functions with CUDA 5, e.g. tex2DGrad, that allow you
    // to pass derivatives to perform automatic mipmap/anisotropic filtering.
    float4 color = tex2DLod<float4>(tex, u, 1 - v, lod);
    // In our sample tex is always valid, but for something like your own
    // sparse texturing you would need to make sure to handle the zero case.

    // write output color
    uint i = y * imageW + x;
    d_output[i] = to_uchar4(color * 255.0);
  }
}

extern "C" void renderAtlasImage(dim3 gridSize, dim3 blockSize,
                                 uchar4 *d_output, uint imageW, uint imageH,
                                 float lod) {
  // psuedo animate lod
  lod = fmodf(lod, highestLod * 2);
  lod = highestLod - fabs(lod - highestLod);

#ifdef SHOW_MIPMAPS
  lod = 0.0f;
#endif

  d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, lod,
                                    atlasImage.textureObject);

  checkCudaErrors(hipGetLastError());
}

//////////////////////////////////////////////////////////////////////////
// MipMap Generation

//  A key benefit of using the new surface objects is that we don't need any
//  global binding points anymore. We can directly pass them as function
//  arguments.

__global__ void d_mipmap(hipSurfaceObject_t mipOutput,
                         hipTextureObject_t mipInput, uint imageW,
                         uint imageH) {
  uint x = blockIdx.x * blockDim.x + threadIdx.x;
  uint y = blockIdx.y * blockDim.y + threadIdx.y;

  float px = 1.0 / float(imageW);
  float py = 1.0 / float(imageH);

  if ((x < imageW) && (y < imageH)) {
    // take the average of 4 samples

    // we are using the normalized access to make sure non-power-of-two textures
    // behave well when downsized.
    float4 color = (tex2D<float4>(mipInput, (x + 0) * px, (y + 0) * py)) +
                   (tex2D<float4>(mipInput, (x + 1) * px, (y + 0) * py)) +
                   (tex2D<float4>(mipInput, (x + 1) * px, (y + 1) * py)) +
                   (tex2D<float4>(mipInput, (x + 0) * px, (y + 1) * py));

    color /= 4.0;
    color *= 255.0;
    color = fminf(color, make_float4(255.0));

    surf2Dwrite(to_uchar4(color), mipOutput, x * sizeof(uchar4), y);
  }
}

void generateMipMaps(hipMipmappedArray_t mipmapArray, hipExtent size) {
  size_t width = size.width;
  size_t height = size.height;

#ifdef SHOW_MIPMAPS
  hipArray_t levelFirst;
  checkCudaErrors(hipGetMipmappedArrayLevel(&levelFirst, mipmapArray, 0));
#endif

  uint level = 0;

  while (width != 1 || height != 1) {
    width /= 2;
    width = MAX((size_t)1, width);
    height /= 2;
    height = MAX((size_t)1, height);

    hipArray_t levelFrom;
    checkCudaErrors(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));
    hipArray_t levelTo;
    checkCudaErrors(
        hipGetMipmappedArrayLevel(&levelTo, mipmapArray, level + 1));

    hipExtent levelToSize;
    checkCudaErrors(hipArrayGetInfo(NULL, &levelToSize, NULL, levelTo));
    checkHost(levelToSize.width == width);
    checkHost(levelToSize.height == height);
    checkHost(levelToSize.depth == 0);

    // generate texture object for reading
    hipTextureObject_t texInput;
    hipResourceDesc texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));

    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = levelFrom;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = 1;
    texDescr.filterMode = hipFilterModeLinear;

    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;

    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(
        hipCreateTextureObject(&texInput, &texRes, &texDescr, NULL));

    // generate surface object for writing

    hipSurfaceObject_t surfOutput;
    hipResourceDesc surfRes;
    memset(&surfRes, 0, sizeof(hipResourceDesc));
    surfRes.resType = hipResourceTypeArray;
    surfRes.res.array.array = levelTo;

    checkCudaErrors(hipCreateSurfaceObject(&surfOutput, &surfRes));

    // run mipmap kernel
    dim3 blockSize(16, 16, 1);
    dim3 gridSize(((uint)width + blockSize.x - 1) / blockSize.x,
                  ((uint)height + blockSize.y - 1) / blockSize.y, 1);

    d_mipmap<<<gridSize, blockSize>>>(surfOutput, texInput, (uint)width,
                                      (uint)height);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipDestroySurfaceObject(surfOutput));

    checkCudaErrors(hipDestroyTextureObject(texInput));

#ifdef SHOW_MIPMAPS
    // we blit the current mipmap back into first level
    hipMemcpy3DParms copyParams = {0};
    copyParams.dstArray = levelFirst;
    copyParams.srcArray = levelTo;
    copyParams.extent = make_hipExtent(width, height, 1);
    copyParams.kind = hipMemcpyDeviceToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));
#endif

    level++;
  }
}

uint getMipMapLevels(hipExtent size) {
  size_t sz = MAX(MAX(size.width, size.height), size.depth);

  uint levels = 0;

  while (sz) {
    sz /= 2;
    levels++;
  }

  return levels;
}

//////////////////////////////////////////////////////////////////////////
// Initalization

extern "C" void randomizeAtlas() {
  uint2 *h_data = (uint2 *)atlasImage.h_data;

  // assign random texture object handles to our atlas image tiles
  for (size_t i = 0; i < atlasImage.size.width * atlasImage.size.height; i++) {
#ifdef SHOW_MIPMAPS
    h_data[i] = encodeTextureObject(contentImages[0].textureObject);
#else
    h_data[i] = encodeTextureObject(
        contentImages[rand() % contentImages.size()].textureObject);
#endif
  }

  // copy data to atlas array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr(
      atlasImage.h_data, atlasImage.size.width * sizeof(uint2),
      atlasImage.size.width, atlasImage.size.height);
  copyParams.dstArray = atlasImage.dataArray;
  copyParams.extent = atlasImage.size;
  copyParams.extent.depth = 1;
  copyParams.kind = hipMemcpyHostToDevice;
  checkCudaErrors(hipMemcpy3D(&copyParams));
};

extern "C" void deinitAtlasAndImages() {
  for (size_t i = 0; i < contentImages.size(); i++) {
    Image &image = contentImages[i];

    if (image.h_data) {
      free(image.h_data);
    }

    if (image.textureObject) {
      checkCudaErrors(hipDestroyTextureObject(image.textureObject));
    }

    if (image.mipmapArray) {
      checkCudaErrors(hipFreeMipmappedArray(image.mipmapArray));
    }
  }

  if (atlasImage.h_data) {
    free(atlasImage.h_data);
  }

  if (atlasImage.textureObject) {
    checkCudaErrors(hipDestroyTextureObject(atlasImage.textureObject));
  }

  if (atlasImage.dataArray) {
    checkCudaErrors(hipFreeArray(atlasImage.dataArray));
  }
}

extern "C" void initAtlasAndImages(const Image *images, size_t numImages,
                                   hipExtent atlasSize) {
  // create individual textures
  contentImages.resize(numImages);

  for (size_t i = 0; i < numImages; i++) {
    Image &image = contentImages[i];
    image.size = images[i].size;
    image.size.depth = 0;
    image.type = hipResourceTypeMipmappedArray;

    // how many mipmaps we need
    uint levels = getMipMapLevels(image.size);
    highestLod = MAX(highestLod, (float)levels - 1);

    hipChannelFormatDesc desc = hipCreateChannelDesc<uchar4>();
    checkCudaErrors(hipMallocMipmappedArray(&image.mipmapArray, &desc,
                                             image.size, levels));

    // upload level 0
    hipArray_t level0;
    checkCudaErrors(hipGetMipmappedArrayLevel(&level0, image.mipmapArray, 0));

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr =
        make_hipPitchedPtr(images[i].h_data, image.size.width * sizeof(uchar4),
                            image.size.width, image.size.height);
    copyParams.dstArray = level0;
    copyParams.extent = image.size;
    copyParams.extent.depth = 1;
    copyParams.kind = hipMemcpyHostToDevice;
    checkCudaErrors(hipMemcpy3D(&copyParams));

    // compute rest of mipmaps based on level 0
    generateMipMaps(image.mipmapArray, image.size);

    // generate bindless texture object

    hipResourceDesc resDescr;
    memset(&resDescr, 0, sizeof(hipResourceDesc));

    resDescr.resType = hipResourceTypeMipmappedArray;
    resDescr.res.mipmap.mipmap = image.mipmapArray;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = 1;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.mipmapFilterMode = hipFilterModeLinear;

    texDescr.addressMode[0] = hipAddressModeClamp;
    texDescr.addressMode[1] = hipAddressModeClamp;
    texDescr.addressMode[2] = hipAddressModeClamp;

    texDescr.maxMipmapLevelClamp = float(levels - 1);

    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&image.textureObject, &resDescr,
                                            &texDescr, NULL));
  }

  // create atlas array
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<uint2>();
  checkCudaErrors(hipMallocArray(&atlasImage.dataArray, &channelDesc,
                                  atlasSize.width, atlasSize.height));
  atlasImage.h_data =
      malloc(atlasSize.width * atlasSize.height * sizeof(uint2));
  atlasImage.type = hipResourceTypeArray;
  atlasImage.size = atlasSize;

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = atlasImage.dataArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.addressMode[1] = hipAddressModeClamp;
  texDescr.addressMode[1] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&atlasImage.textureObject, &texRes,
                                          &texDescr, NULL));

  randomizeAtlas();
}

#endif  // #ifndef _SIMPLETEXTURE3D_KERNEL_CU_
