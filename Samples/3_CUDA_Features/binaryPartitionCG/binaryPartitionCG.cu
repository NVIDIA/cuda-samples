#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample illustrates basic usage of binary partition cooperative groups
 * within the thread block tile when divergent path exists.
 * 1.) Each thread loads a value from random array.
 * 2.) then checks if it is odd or even.
 * 3.) create binary partition group based on the above predicate
 * 4.) we count the number of odd/even in the group based on size of the binary
       groups
 * 5.) write it global counter of odd.
 * 6.) sum the values loaded by individual threads(using reduce) and write it to
       global even & odd elements sum.
 *
 * **NOTE** :
 *    binary_partition results in splitting warp into divergent thread groups
 *    this is not good from performance perspective, but in cases where warp
 *    divergence is inevitable one can use binary_partition group.
*/

#include <stdio.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime_api.h>

namespace cg = cooperative_groups;

void initOddEvenArr(int *inputArr, unsigned int size) {
  for (int i = 0; i < size; i++) {
    inputArr[i] = rand() % 50;
  }
}

/**
 * CUDA kernel device code
 *
 * Creates cooperative groups and performs odd/even counting & summation.
 */
__global__ void oddEvenCountAndSumCG(int *inputArr, int *numOfOdds,
                                     int *sumOfOddAndEvens, unsigned int size) {
  cg::thread_block cta = cg::this_thread_block();
  cg::grid_group grid = cg::this_grid();
  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  for (int i = grid.thread_rank(); i < size; i += grid.size()) {
    int elem = inputArr[i];
    auto subTile = cg::binary_partition(tile32, elem & 1);
    if (elem & 1)  // Odd numbers group
    {
      int oddGroupSum = cg::reduce(subTile, elem, cg::plus<int>());

      if (subTile.thread_rank() == 0) {
        // Add number of odds present in this group of Odds.
        atomicAdd(numOfOdds, subTile.size());

        // Add local reduction of odds present in this group of Odds.
        atomicAdd(&sumOfOddAndEvens[0], oddGroupSum);
      }
    } else  // Even numbers group
    {
      int evenGroupSum = cg::reduce(subTile, elem, cg::plus<int>());

      if (subTile.thread_rank() == 0) {
        // Add local reduction of even present in this group of evens.
        atomicAdd(&sumOfOddAndEvens[1], evenGroupSum);
      }
    }
    // reconverge warp so for next loop iteration we ensure convergence of
    // above diverged threads to perform coalesced loads of inputArr.
    cg::sync(tile32);
  }
}

/**
 * Host main routine
 */
int main(int argc, const char **argv) {
  int deviceId = findCudaDevice(argc, argv);
  int *h_inputArr, *d_inputArr;
  int *h_numOfOdds, *d_numOfOdds;
  int *h_sumOfOddEvenElems, *d_sumOfOddEvenElems;
  unsigned int arrSize = 1024 * 100;

  checkCudaErrors(hipHostMalloc(&h_inputArr, sizeof(int) * arrSize));
  checkCudaErrors(hipHostMalloc(&h_numOfOdds, sizeof(int)));
  checkCudaErrors(hipHostMalloc(&h_sumOfOddEvenElems, sizeof(int) * 2));
  initOddEvenArr(h_inputArr, arrSize);

  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  checkCudaErrors(hipMalloc(&d_inputArr, sizeof(int) * arrSize));
  checkCudaErrors(hipMalloc(&d_numOfOdds, sizeof(int)));
  checkCudaErrors(hipMalloc(&d_sumOfOddEvenElems, sizeof(int) * 2));

  checkCudaErrors(hipMemcpyAsync(d_inputArr, h_inputArr, sizeof(int) * arrSize,
                                  hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipMemsetAsync(d_numOfOdds, 0, sizeof(int), stream));
  checkCudaErrors(
      hipMemsetAsync(d_sumOfOddEvenElems, 0, 2 * sizeof(int), stream));

  // Launch the kernel
  int threadsPerBlock = 0;
  int blocksPerGrid = 0;
  checkCudaErrors(hipOccupancyMaxPotentialBlockSize(
      &blocksPerGrid, &threadsPerBlock, oddEvenCountAndSumCG, 0, 0));

  printf("\nLaunching %d blocks with %d threads...\n\n", blocksPerGrid,
         threadsPerBlock);

  oddEvenCountAndSumCG<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(
      d_inputArr, d_numOfOdds, d_sumOfOddEvenElems, arrSize);

  checkCudaErrors(hipMemcpyAsync(h_numOfOdds, d_numOfOdds, sizeof(int),
                                  hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipMemcpyAsync(h_sumOfOddEvenElems, d_sumOfOddEvenElems,
                                  2 * sizeof(int), hipMemcpyDeviceToHost,
                                  stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  printf("Array size = %d Num of Odds = %d Sum of Odds = %d Sum of Evens %d\n",
         arrSize, h_numOfOdds[0], h_sumOfOddEvenElems[0],
         h_sumOfOddEvenElems[1]);
  printf("\n...Done.\n\n");

  checkCudaErrors(hipHostFree(h_inputArr));
  checkCudaErrors(hipHostFree(h_numOfOdds));
  checkCudaErrors(hipHostFree(h_sumOfOddEvenElems));

  checkCudaErrors(hipFree(d_inputArr));
  checkCudaErrors(hipFree(d_numOfOdds));
  checkCudaErrors(hipFree(d_sumOfOddEvenElems));

  return EXIT_SUCCESS;
}
