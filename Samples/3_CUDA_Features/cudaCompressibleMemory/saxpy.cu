#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//
// This sample uses the compressible memory allocation if device supports it
// and performs saxpy on it. 
// Compressible memory may give better performance if the data is amenable to 
// compression.

#include <stdio.h>
#include <hip/hip_runtime.h>
#define CUDA_DRIVER_API
#include "hip/hip_runtime_api.h"
#include "compMalloc.h"

__global__ void saxpy(const float a, const float4 *x, const float4 *y, float4 *z, const size_t n)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
    {
        const float4 x4 = x[i];
        const float4 y4 = y[i];
        z[i] = make_float4(a * x4.x + y4.x, a * x4.y + y4.y,
                            a * x4.z + y4.z, a * x4.w + y4.w);
    }
}

__global__ void init(float4 *x, float4 *y, const float val, const size_t n)
{
    const float4 val4 = make_float4(val, val, val, val);
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
    {
        x[i] = y[i] = val4;
    }
}

void launchSaxpy(const float a, float4 *x, float4 *y, float4 *z, const size_t n, const float init_val, const bool compressibleZbuf)
{
    hipEvent_t start, stop;
    float ms;
    int blockSize;
    int minGridSize;
    dim3 threads, blocks; 

    if (!compressibleZbuf)
    {
        // We are on config where compressible buffer can only be initialized through hipMemcpy
        // hence, x & y buffers are allocated as compressible and initialized via hipMemcpy
        // whereas z buffer is allocated as non-compressible.
        float4 *h_x = (float4 *) malloc(sizeof(float4) * n);
        float4 *h_y = (float4 *) malloc(sizeof(float4) * n);
        for (int i = 0; i < n; i++)
        {
            h_x[i].x = h_x[i].y = h_x[i].z = h_x[i].w = init_val;
            h_y[i].x = h_y[i].y = h_y[i].z = h_y[i].w = init_val;
        }
        checkCudaErrors(hipMemcpy(x, h_x, sizeof(float4) * n, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(y, h_y, sizeof(float4) * n, hipMemcpyHostToDevice));
        free(h_x);
        free(h_y);
    }
    else
    {
        checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)init));
        threads = dim3(blockSize, 1, 1);
        blocks  = dim3(minGridSize, 1, 1);
        init<<<blocks, threads>>>(x, y, init_val, n);
    }

    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, (void*)saxpy));
    threads = dim3(blockSize, 1, 1);
    blocks  = dim3(minGridSize, 1, 1);

    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    checkCudaErrors(hipEventRecord(start));
    saxpy<<<blocks, threads>>>(a, x, y, z, n);
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&ms, start, stop));

    const size_t size = n * sizeof(float4);
    printf("Running saxpy with %d blocks x %d threads = %.3f ms %.3f TB/s\n", blocks.x, threads.x, ms, (size*3)/ms/1e9);
}

int main(int argc, char **argv)
{
    const size_t n = 10485760;

    if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
            checkCmdLineFlag(argc, (const char **)argv, "?")) {
        printf("Usage -device=n (n >= 0 for deviceID)\n");
        exit(EXIT_SUCCESS);
    }

    findCudaDevice(argc, (const char**)argv);
    hipDevice_t currentDevice;
    checkCudaErrors(hipCtxGetDevice(&currentDevice));

    // Check that the selected device supports virtual memory management
    int vmm_supported = -1;
    checkCudaErrors(hipDeviceGetAttribute(&vmm_supported,
                          CU_DEVICE_ATTRIBUTE_VIRTUAL_ADDRESS_MANAGEMENT_SUPPORTED,
                          currentDevice));
    if (vmm_supported == 0) {
        printf("Device %d doesn't support Virtual Memory Management, waiving the execution.\n", currentDevice);
        exit(EXIT_WAIVED);
    }

    int isCompressionAvailable;
    checkCudaErrors(hipDeviceGetAttribute(&isCompressionAvailable,
                             CU_DEVICE_ATTRIBUTE_GENERIC_COMPRESSION_SUPPORTED,
                             currentDevice));
    if (isCompressionAvailable == 0)
    {
        printf("Device %d doesn't support Generic memory compression, waiving the execution.\n", currentDevice);
        exit(EXIT_WAIVED);
    }

    printf("Generic memory compression support is available\n");

    int major, minor;
    checkCudaErrors(hipDeviceGetAttribute(&major,
                          hipDeviceAttributeComputeCapabilityMajor,
                          currentDevice));
    checkCudaErrors(hipDeviceGetAttribute(&minor,
                          hipDeviceAttributeComputeCapabilityMinor,
                          currentDevice));
    float4 *x, *y, *z;
    const size_t size = n * sizeof(float4);

    // Allocating compressible memory
    checkCudaErrors(allocateCompressible((void **)&x, size, true));
    checkCudaErrors(allocateCompressible((void **)&y, size, true));
    bool compressibleZbuf = 0;
    if ((major == 8 && minor == 0) || (major == 8 && minor == 6))
    {
        // On SM 8.0 and 8.6 GPUs compressible buffer can only be initialized 
        // through hipMemcpy.
        printf("allocating non-compressible Z buffer\n");
        checkCudaErrors(allocateCompressible((void **)&z, size, false));
        compressibleZbuf = 0;
    }
    else
    {
        checkCudaErrors(allocateCompressible((void **)&z, size, true));
        compressibleZbuf = 1;
    }

    printf("Running saxpy on %zu bytes of Compressible memory\n", size);

    const float a = 1.0f;
    const float init_val = 1.0f;
    launchSaxpy(a, x, y, z, n, init_val, compressibleZbuf);
 
    checkCudaErrors(freeCompressible(x, size, true));
    checkCudaErrors(freeCompressible(y, size, true));
    checkCudaErrors(freeCompressible(z, size, true));

    printf("Running saxpy on %zu bytes of Non-Compressible memory\n", size);
    // Allocating non-compressible memory
    checkCudaErrors(allocateCompressible((void **)&x, size, false));
    checkCudaErrors(allocateCompressible((void **)&y, size, false));
    checkCudaErrors(allocateCompressible((void **)&z, size, false));

    launchSaxpy(a, x, y, z, n, init_val, compressibleZbuf);

    checkCudaErrors(freeCompressible(x, size, false));
    checkCudaErrors(freeCompressible(y, size, false));
    checkCudaErrors(freeCompressible(z, size, false));

    printf("\nNOTE: The CUDA Samples are not meant for performance measurements. "
      "Results may vary when GPU Boost is enabled.\n");
    return EXIT_SUCCESS;
}