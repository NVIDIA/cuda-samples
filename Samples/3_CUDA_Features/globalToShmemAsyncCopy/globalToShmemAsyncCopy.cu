#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * Matrix multiplication: C = A * B.
 *
 * This sample demonstrates implements matrix multiplication which makes use of
 * shared memory to ensure data reuse, the matrix multiplication is done using
 * tiling approach.
 * With compute capability 8.0 or higher the CUDA kernels involved uses
 * asynchronously copy data from global to shared memory; a.k.a., async-copy.
 * This sample has been written for clarity of exposition to illustrate various
 * CUDA programming principles, not with the goal of providing the most
 * performant generic kernel for matrix multiplication.
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <cuda/pipeline>

#if __CUDA_ARCH__ >= 700
#include <cuda/barrier>
#endif
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

enum kernels {
  AsyncCopyMultiStageLargeChunk = 0,
  AsyncCopyLargeChunk = 1,
  AsyncCopyLargeChunkAWBarrier = 2,
  AsyncCopyMultiStageSharedState = 3,
  AsyncCopyMultiStage = 4,
  AsyncCopySingleStage = 5,
  Naive = 6,
  NaiveLargeChunk = 7
};

const char *kernelNames[] = {"AsyncCopyMultiStageLargeChunk",
                             "AsyncCopyLargeChunk",
                             "AsyncCopyLargeChunkAWBarrier",
                             "AsyncCopyMultiStageSharedState",
                             "AsyncCopyMultiStage",
                             "AsyncCopySingleStage",
                             "Naive",
                             "NaiveLargeChunk"};

constexpr int blockSize = 16;

// Multi Stage memcpy_async pipeline with large chunk copy
template <int BLOCK_SIZE>
__global__ void MatrixMulAsyncCopyMultiStageLargeChunk(
    float *__restrict__ C, const float *__restrict__ A,
    const float *__restrict__ B, int wA, int wB) {
  // Requires BLOCK_SIZE % 4 == 0

  // Multi-stage pipeline version
  constexpr size_t maxPipelineStages = 4;

  // Declaration of the shared memory array As used to
  // store the sub-matrix of A for each stage
  __shared__ alignas(
      alignof(float4)) float As[maxPipelineStages][BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B for each stage
  __shared__ alignas(
      alignof(float4)) float Bs[maxPipelineStages][BLOCK_SIZE][BLOCK_SIZE];

  float Csub = 0.0;

  // Index of the first sub-matrix of A processed by the block
  const int aBegin = wA * (BLOCK_SIZE)*blockIdx.y;

  // Index of the last sub-matrix of A processed by the block
  const int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  const int bBegin = BLOCK_SIZE * blockIdx.x;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  const int t4x = threadIdx.x * 4;
  const auto shape4 = cuda::aligned_size_t<alignof(float4)>(sizeof(float4));

  cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin, i = 0, aStage = aBegin, bStage = bBegin,
           iStage = 0;
       a <= aEnd; a += aStep, b += bStep, ++i) {
    // Load the matrices from device memory to shared memory; each thread loads
    // one element of each matrix
    for (; aStage <= a + aStep * maxPipelineStages;
         aStage += aStep, bStage += bStep, ++iStage) {
      pipe.producer_acquire();
      if (aStage <= aEnd && t4x < BLOCK_SIZE) {
        // Rotating buffer
        const int j = iStage % maxPipelineStages;
        cuda::memcpy_async(&As[j][threadIdx.y][t4x],
                           &A[aStage + wA * threadIdx.y + t4x], shape4, pipe);
        cuda::memcpy_async(&Bs[j][threadIdx.y][t4x],
                           &B[aStage + wA * threadIdx.y + t4x], shape4, pipe);
      }
      pipe.producer_commit();
    }

    pipe.consumer_wait();
    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Rotating buffer
    const int j = i % maxPipelineStages;

// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[j][threadIdx.y][k] * Bs[j][k][threadIdx.x];
    }
    pipe.consumer_release();

    // Don't have to synchronize because maxPipelineStages is greater than one
    // therefore next iteration is loading to a different buffer.
  }

  // Write the block sub-matrix to device memory;
  // each thread writes four element
  int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
  C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

// Single Stage memcpy_async pipeline with Large copy chunk (float4)
template <int BLOCK_SIZE>
__global__ void MatrixMulAsyncCopyLargeChunk(float *__restrict__ C,
                                             const float *__restrict__ A,
                                             const float *__restrict__ B,
                                             int wA, int wB) {
  // Requires BLOCK_SIZE % 4 == 0

  // Declaration of the shared memory array As used to
  // store the sub-matrix of A
  __shared__ alignas(alignof(float4)) float As[BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B
  __shared__ alignas(alignof(float4)) float Bs[BLOCK_SIZE][BLOCK_SIZE];

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * blockIdx.y;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * blockIdx.x;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Single-stage pipeline version
  float Csub = 0.0;

  const int t4x = threadIdx.x * 4;
  const auto shape4 = cuda::aligned_size_t<alignof(float4)>(sizeof(float4));
  cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Load the matrices from device memory to shared memory;
    // a subset of threads loads a contiguous chunk of elements.

    // Previously, per-thread:
    // As[ty][tx] = A[a + wA * ty + tx];
    // Bs[ty][tx] = B[b + wB * ty + tx];

    // Now, one fourth of the threads load four elements of each matrix
    if (t4x < BLOCK_SIZE) {
      pipe.producer_acquire();

      cuda::memcpy_async(&As[threadIdx.y][t4x], &A[a + wA * threadIdx.y + t4x],
                         shape4, pipe);
      cuda::memcpy_async(&Bs[threadIdx.y][t4x], &B[a + wA * threadIdx.y + t4x],
                         shape4, pipe);

      pipe.producer_commit();
      pipe.consumer_wait();
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
    }

    pipe.consumer_release();

    // Synchronize to make sure that the preceding
    // computation is done before overwriting the
    // shared memory sub-matrix buffers As and Bs in the next iteration.
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes four element
  int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
  C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

// Single Stage memcpy_async pipeline with Large copy chunk (float4) using
// arrive-wait barrier
template <int BLOCK_SIZE>
__global__ void MatrixMulAsyncCopyLargeChunkAWBarrier(
    float *__restrict__ C, const float *__restrict__ A,
    const float *__restrict__ B, int wA, int wB) {
#if __CUDA_ARCH__ >= 700
#pragma diag_suppress static_var_with_dynamic_init
  // Requires BLOCK_SIZE % 4 == 0

  __shared__ cuda::barrier<cuda::thread_scope_block> bar;

  // Declaration of the shared memory array As used to
  // store the sub-matrix of A
  __shared__ alignas(alignof(float4)) float As[BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B
  __shared__ alignas(alignof(float4)) float Bs[BLOCK_SIZE][BLOCK_SIZE];

  if (threadIdx.x == 0) {
    init(&bar, blockDim.x * blockDim.y);
  }
  __syncthreads();

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * blockIdx.y;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * blockIdx.x;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  float Csub = 0.0;

  const int t4x = threadIdx.x * 4;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Load the matrices from device memory to shared memory;
    // a subset of threads loads a contiguous chunk of elements.

    // Now, one fourth of the threads load four elements of each matrix
    if (t4x < BLOCK_SIZE) {
      float4 *const A4s = reinterpret_cast<float4 *>(&As[threadIdx.y][t4x]);
      float4 *const B4s = reinterpret_cast<float4 *>(&Bs[threadIdx.y][t4x]);
      const float4 *const A4 =
          reinterpret_cast<const float4 *>(&A[a + wA * threadIdx.y + t4x]);
      const float4 *const B4 =
          reinterpret_cast<const float4 *>(&B[a + wA * threadIdx.y + t4x]);

      cuda::memcpy_async(A4s, A4, sizeof(float4), bar);
      cuda::memcpy_async(B4s, B4, sizeof(float4), bar);
    }

    // Synchronize to make sure the matrices are loaded
    bar.arrive_and_wait();

// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
    }

    // Synchronize to make sure that the preceding
    // computation is done before overwriting the
    // shared memory sub-matrix buffers As and Bs in the next iteration.
    bar.arrive_and_wait();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes four element
  int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
  C[c + wB * threadIdx.y + threadIdx.x] = Csub;
#endif
}

// Single Stage memcpy_async pipeline with float copy
template <int BLOCK_SIZE>
__global__ void MatrixMulAsyncCopySingleStage(float *C, const float *A,
                                              const float *B, int wA, int wB) {
  // Declaration of the shared memory array As used to
  // store the sub-matrix of A
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * blockIdx.y;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * blockIdx.x;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Single-stage pipeline version
  float Csub = 0.0;

  cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
  const auto shape1 = cuda::aligned_size_t<alignof(float)>(sizeof(float));

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Load the matrices from device memory to shared memory; each thread loads
    // one element of each matrix
    {
      pipe.producer_acquire();

      cuda::memcpy_async(&As[threadIdx.y][threadIdx.x],
                         &A[a + wA * threadIdx.y + threadIdx.x], shape1, pipe);
      cuda::memcpy_async(&Bs[threadIdx.y][threadIdx.x],
                         &B[b + wB * threadIdx.y + threadIdx.x], shape1, pipe);

      pipe.producer_commit();
    }

    pipe.consumer_wait();
    // Synchronize to make sure the matrices are loaded
    __syncthreads();

// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
    }

    // Synchronize to make sure that the preceding
    // computation is done before overwriting the
    // shared memory sub-matrix buffers As and Bs in the next iteration.
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes four element
  int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
  C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

// Multi Stage memcpy_async thread_scope_thread pipeline with single-element
// async-copy
template <int BLOCK_SIZE>
__global__ void MatrixMulAsyncCopyMultiStage(float *__restrict__ C,
                                             const float *__restrict__ A,
                                             const float *__restrict__ B,
                                             int wA, int wB) {
  // Multi-stage pipeline version
  constexpr size_t maxPipelineStages = 4;

  // Declaration of the shared memory array As used to
  // store the sub-matrix of A for each stage
  __shared__ float As[maxPipelineStages][BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B for each stage
  __shared__ float Bs[maxPipelineStages][BLOCK_SIZE][BLOCK_SIZE];

  float Csub = 0.0;

  // Index of the first sub-matrix of A processed by the block
  const int aBegin = wA * BLOCK_SIZE * blockIdx.y;

  // Index of the last sub-matrix of A processed by the block
  const int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  const int bBegin = BLOCK_SIZE * blockIdx.x;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  cuda::pipeline<cuda::thread_scope_thread> pipe = cuda::make_pipeline();
  const auto shape1 = cuda::aligned_size_t<alignof(float)>(sizeof(float));

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin, i = 0, aStage = aBegin, bStage = bBegin,
           iStage = 0;
       a <= aEnd; a += aStep, b += bStep, ++i) {
    // Load the matrices from device memory to shared memory; each thread loads
    // one element of each matrix

    for (; aStage <= a + aStep * maxPipelineStages;
         aStage += aStep, bStage += bStep, ++iStage) {
      if (aStage <= aEnd) {
        // Rotating buffer
        const int j = iStage % maxPipelineStages;

        pipe.producer_acquire();

        cuda::memcpy_async(&As[j][threadIdx.y][threadIdx.x],
                           &A[aStage + wA * threadIdx.y + threadIdx.x], shape1,
                           pipe);
        cuda::memcpy_async(&Bs[j][threadIdx.y][threadIdx.x],
                           &B[bStage + wB * threadIdx.y + threadIdx.x], shape1,
                           pipe);

        pipe.producer_commit();
      }
    }
    pipe.consumer_wait();

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    const int j = i % maxPipelineStages;

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[j][threadIdx.y][k] * Bs[j][k][threadIdx.x];
    }

    pipe.consumer_release();
    // Don't have to synchronize because maxPipelineStages is greater than one
    // therefore next iteration is loading to a different buffer.
  }

  // Write the block sub-matrix to device memory;
  // each thread writes four element
  int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
  C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

// Multi Stage shared state memcpy_async pipeline thread_scope_block
// with parititioned producer & consumer, here we've 1 warp as producer
// group which issues memcpy_async operations and rest all warps are part of
// consumer group which perform gemm computation on the loaded matrices by
// producer.
template <int BLOCK_SIZE_X>
__global__ void MatrixMulAsyncCopyMultiStageSharedState(
    float *__restrict__ C, const float *__restrict__ A,
    const float *__restrict__ B, int wA, int wB) {
  // Multi-stage pipeline version
  constexpr size_t maxPipelineStages = 4;

  // Declaration of the shared memory array As used to
  // store the sub-matrix of A for each stage
  __shared__ float As[maxPipelineStages][BLOCK_SIZE_X][BLOCK_SIZE_X];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B for each stage
  __shared__ float Bs[maxPipelineStages][BLOCK_SIZE_X][BLOCK_SIZE_X];

  float Csub = 0.0;

  // Index of the first sub-matrix of A processed by the block
  const int aBegin = wA * BLOCK_SIZE_X * blockIdx.y;

  // Index of the last sub-matrix of A processed by the block
  const int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  constexpr int aStep = BLOCK_SIZE_X;

  // Index of the first sub-matrix of B processed by the block
  const int bBegin = BLOCK_SIZE_X * blockIdx.x;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE_X * wB;

  auto cta = cg::this_thread_block();

  const auto shape1 = cuda::aligned_size_t<alignof(float)>(sizeof(float));
  __shared__ cuda::pipeline_shared_state<cuda::thread_scope_block,
                                         maxPipelineStages> shared_state;
  constexpr int consumer_row_count = BLOCK_SIZE_X;

  const auto thread_role = (cta.thread_index().y < consumer_row_count)
                               ? cuda::pipeline_role::consumer
                               : cuda::pipeline_role::producer;
  auto pipe = cuda::make_pipeline(cta, &shared_state, thread_role);

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin, i = 0, aStage = aBegin, bStage = bBegin,
           iStage = 0;
       a <= aEnd; a += aStep, b += bStep, ++i) {
    if (threadIdx.y >= consumer_row_count) {
      // this is a whole producer warp because threadIdx.y >= 16 where 16 ==
      // consumer_row_count,
      // which loads the matrices from device memory to shared memory;
      for (; aStage <= a + aStep * maxPipelineStages;
           aStage += aStep, bStage += bStep, ++iStage) {
        if (aStage <= aEnd) {
          // Rotating buffer
          const int j = iStage % maxPipelineStages;
          const int strideRows = (blockDim.y - consumer_row_count);
          pipe.producer_acquire();
          for (int rowId = threadIdx.y - consumer_row_count;
               rowId < BLOCK_SIZE_X; rowId += strideRows) {
            cuda::memcpy_async(&As[j][rowId][threadIdx.x],
                               &A[aStage + wA * rowId + threadIdx.x], shape1,
                               pipe);
            cuda::memcpy_async(&Bs[j][rowId][threadIdx.x],
                               &B[bStage + wB * rowId + threadIdx.x], shape1,
                               pipe);
          }
          pipe.producer_commit();
        }
      }
    } else {
      // this is a whole set of consumer group because threadIdx.y <
      // consumer_row_count where consumer_row_count == 16,
      // which computes gemm operation on matrices loaded in shared memory by
      // producer warp.
      const int j = i % maxPipelineStages;
      // Synchronize consumer group to make sure the matrices are loaded by
      // producer group.
      pipe.consumer_wait();
// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
#pragma unroll
      for (int k = 0; k < BLOCK_SIZE_X; ++k) {
        Csub += As[j][threadIdx.y][k] * Bs[j][k][threadIdx.x];
      }
      pipe.consumer_release();
    }
  }

  // Write the block sub-matrix to device memory;
  // each thread writes four element
  if (threadIdx.y < consumer_row_count) {
    const int c = wB * BLOCK_SIZE_X * blockIdx.y + BLOCK_SIZE_X * blockIdx.x;
    C[c + wB * threadIdx.y + threadIdx.x] = Csub;
  }
}

/**
 * Matrix multiplication (CUDA Kernel) on the device: C = A * B
 * wA is A's width and wB is B's width
 */
template <int BLOCK_SIZE>
__global__ void MatrixMulNaive(float *C, float *A, float *B, int wA, int wB) {
  // Declaration of the shared memory array As used to
  // store the sub-matrix of A
  __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B
  __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * blockIdx.y;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * blockIdx.x;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[threadIdx.y][threadIdx.x] = A[a + wA * threadIdx.y + threadIdx.x];
    Bs[threadIdx.y][threadIdx.x] = B[b + wB * threadIdx.y + threadIdx.x];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
  C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

template <int BLOCK_SIZE>
__global__ void MatrixMulNaiveLargeChunk(float *C, float *A, float *B, int wA,
                                         int wB) {
  // Declaration of the shared memory array As used to
  // store the sub-matrix of A
  __shared__ alignas(alignof(float4)) float As[BLOCK_SIZE][BLOCK_SIZE];

  // Declaration of the shared memory array Bs used to
  // store the sub-matrix of B
  __shared__ alignas(alignof(float4)) float Bs[BLOCK_SIZE][BLOCK_SIZE];

  int t4x = threadIdx.x * 4;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = wA * BLOCK_SIZE * blockIdx.y;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * blockIdx.x;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Load the matrices from device memory
    // to shared memory;

    // One fourth of the threads load four elements of each matrix
    if (t4x < BLOCK_SIZE) {
      float4 *const A4s = reinterpret_cast<float4 *>(&As[threadIdx.y][t4x]);
      float4 *const B4s = reinterpret_cast<float4 *>(&Bs[threadIdx.y][t4x]);
      const float4 *const A4 =
          reinterpret_cast<float4 *>(&A[a + wA * threadIdx.y + t4x]);
      const float4 *const B4 =
          reinterpret_cast<float4 *>(&B[a + wA * threadIdx.y + t4x]);
      *A4s = *A4;
      *B4s = *B4;
    }

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

// Multiply the two matrices together;
// each thread computes one element
// of the block sub-matrix
#pragma unroll
    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[threadIdx.y][k] * Bs[k][threadIdx.x];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x;
  C[c + wB * threadIdx.y + threadIdx.x] = Csub;
}

void ConstantInit(float *data, int size, float val) {
  for (int i = 0; i < size; ++i) {
    data[i] = val;
  }
}

/**
 * Run matrix multiplication using CUDA
 */
int MatrixMultiply(int argc, char **argv, const dim3 &dimsA, const dim3 &dimsB,
                   kernels kernel_number) {
  // Allocate host memory for matrices A and B
  unsigned int size_A = dimsA.x * dimsA.y;
  unsigned int mem_size_A = sizeof(float) * size_A;
  float *h_A;
  checkCudaErrors(hipHostMalloc(&h_A, mem_size_A));
  unsigned int size_B = dimsB.x * dimsB.y;
  unsigned int mem_size_B = sizeof(float) * size_B;
  float *h_B;
  checkCudaErrors(hipHostMalloc(&h_B, mem_size_B));
  hipStream_t stream;

  // Initialize host memory
  const float valB = 2.10f;
  ConstantInit(h_A, size_A, 1.0f);
  ConstantInit(h_B, size_B, valB);

  // Allocate device memory
  float *d_A, *d_B, *d_C;

  // Allocate host matrix C
  dim3 dimsC(dimsB.x, dimsA.y, 1);
  unsigned int mem_size_C = dimsC.x * dimsC.y * sizeof(float);
  float *h_C;
  checkCudaErrors(hipHostMalloc(&h_C, mem_size_C));

  if (h_C == NULL) {
    fprintf(stderr, "Failed to allocate host matrix C!\n");
    exit(EXIT_FAILURE);
  }

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_A), mem_size_A));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_B), mem_size_B));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_C), mem_size_C));
  // Allocate CUDA events that we'll use for timing
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  // copy host memory to device
  checkCudaErrors(
      hipMemcpyAsync(d_A, h_A, mem_size_A, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
      hipMemcpyAsync(d_B, h_B, mem_size_B, hipMemcpyHostToDevice, stream));
  checkCudaErrors(hipMemsetAsync(d_C, 0, mem_size_C, stream));

  // Setup execution parameters
  dim3 threads(blockSize, blockSize);
  dim3 grid(dimsB.x / threads.x, dimsA.y / threads.y);

  // Here the block size is 16x18, where first 16 rows are consumer thread group
  // and last 2 rows (1 warp) is producer thread group
  dim3 threadsSharedStateKernel(blockSize, blockSize + 2, 1);
  dim3 gridSharedStateKernel(dimsB.x / threadsSharedStateKernel.x,
                             dimsA.y / threadsSharedStateKernel.x);

  printf("Running kernel = %d - %s\n", kernel_number,
         kernelNames[kernel_number]);
  // Create and start timer
  printf("Computing result using CUDA Kernel...\n");

  // Performs warmup operation using matrixMul CUDA kernel
  switch (kernel_number) {
    case AsyncCopyMultiStageLargeChunk:
    default:
      MatrixMulAsyncCopyMultiStageLargeChunk<
          blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x,
                                                   dimsB.x);
      break;
    case AsyncCopyLargeChunk:
      MatrixMulAsyncCopyLargeChunk<blockSize><<<grid, threads, 0, stream>>>(
          d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case AsyncCopyLargeChunkAWBarrier:
      MatrixMulAsyncCopyLargeChunkAWBarrier<
          blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x,
                                                   dimsB.x);
      break;
    case AsyncCopyMultiStageSharedState:
      MatrixMulAsyncCopyMultiStageSharedState<blockSize><<<
          gridSharedStateKernel, threadsSharedStateKernel, 0, stream>>>(
          d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case AsyncCopyMultiStage:
      MatrixMulAsyncCopyMultiStage<blockSize><<<grid, threads, 0, stream>>>(
          d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case AsyncCopySingleStage:
      MatrixMulAsyncCopySingleStage<blockSize><<<grid, threads, 0, stream>>>(
          d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
    case Naive:
      MatrixMulNaive<blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B,
                                                              dimsA.x, dimsB.x);
      break;
    case NaiveLargeChunk:
      MatrixMulNaiveLargeChunk<blockSize><<<grid, threads, 0, stream>>>(
          d_C, d_A, d_B, dimsA.x, dimsB.x);
      break;
  }

  printf("done\n");
  checkCudaErrors(hipStreamSynchronize(stream));

  // Execute the kernel
  int nIter = 100;

  // Record the start event
  checkCudaErrors(hipEventRecord(start, stream));

  for (int j = 0; j < nIter; j++) {
    switch (kernel_number) {
      case AsyncCopyMultiStageLargeChunk:
      default:
        MatrixMulAsyncCopyMultiStageLargeChunk<
            blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x,
                                                     dimsB.x);
        break;
      case AsyncCopyLargeChunk:
        MatrixMulAsyncCopyLargeChunk<blockSize><<<grid, threads, 0, stream>>>(
            d_C, d_A, d_B, dimsA.x, dimsB.x);
        break;
      case AsyncCopyLargeChunkAWBarrier:
        MatrixMulAsyncCopyLargeChunkAWBarrier<
            blockSize><<<grid, threads, 0, stream>>>(d_C, d_A, d_B, dimsA.x,
                                                     dimsB.x);
        break;
      case AsyncCopyMultiStageSharedState:
        MatrixMulAsyncCopyMultiStageSharedState<blockSize><<<
            gridSharedStateKernel, threadsSharedStateKernel, 0, stream>>>(
            d_C, d_A, d_B, dimsA.x, dimsB.x);
        break;
      case AsyncCopyMultiStage:
        MatrixMulAsyncCopyMultiStage<blockSize><<<grid, threads, 0, stream>>>(
            d_C, d_A, d_B, dimsA.x, dimsB.x);
        break;
      case AsyncCopySingleStage:
        MatrixMulAsyncCopySingleStage<blockSize><<<grid, threads, 0, stream>>>(
            d_C, d_A, d_B, dimsA.x, dimsB.x);
        break;
      case Naive:
        MatrixMulNaive<blockSize><<<grid, threads, 0, stream>>>(
            d_C, d_A, d_B, dimsA.x, dimsB.x);
        break;
      case NaiveLargeChunk:
        MatrixMulNaiveLargeChunk<blockSize><<<grid, threads, 0, stream>>>(
            d_C, d_A, d_B, dimsA.x, dimsB.x);
        break;
    }
  }

  // Record the stop event
  checkCudaErrors(hipEventRecord(stop, stream));

  // Wait for the stop event to complete
  checkCudaErrors(hipEventSynchronize(stop));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, stop));

  // Compute and print the performance
  float msecPerMatrixMul = msecTotal / nIter;
  double flopsPerMatrixMul = 2.0 * static_cast<double>(dimsA.x) *
                             static_cast<double>(dimsA.y) *
                             static_cast<double>(dimsB.x);
  double gigaFlops =
      (flopsPerMatrixMul * 1.0e-9f) / (msecPerMatrixMul / 1000.0f);
  printf(
      "Performance= %.2f GFlop/s, Time= %.3f msec, Size= %.0f Ops,"
      " WorkgroupSize= %u threads/block\n",
      gigaFlops, msecPerMatrixMul, flopsPerMatrixMul, threads.x * threads.y);

  // Copy result from device to host
  checkCudaErrors(
      hipMemcpyAsync(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  printf("Checking computed result for correctness: ");
  bool correct = true;

  // test relative error by the formula
  // |<x, y>_cpu - <x,y>_gpu|/<|x|, |y|>  < eps
  double eps = 1.e-6;  // machine zero

  for (int i = 0; i < static_cast<int>(dimsC.x * dimsC.y); i++) {
    double abs_err = fabs(h_C[i] - (dimsA.x * valB));
    double dot_length = dimsA.x;
    double abs_val = fabs(h_C[i]);
    double rel_err = abs_err / abs_val / dot_length;

    if (rel_err > eps) {
      printf("Error! Matrix[%05d]=%.8f, ref=%.8f error term is > %E\n", i,
             h_C[i], dimsA.x * valB, eps);
      correct = false;
    }
  }

  printf("%s\n", correct ? "Result = PASS" : "Result = FAIL");

  // Clean up memory
  checkCudaErrors(hipHostFree(h_A));
  checkCudaErrors(hipHostFree(h_B));
  checkCudaErrors(hipHostFree(h_C));
  checkCudaErrors(hipFree(d_A));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  printf(
      "\nNOTE: The CUDA Samples are not meant for performance "
      "measurements. Results may vary when GPU Boost is enabled.\n");

  if (correct) {
    return EXIT_SUCCESS;
  } else {
    return EXIT_FAILURE;
  }
}

int main(int argc, char **argv) {
  printf("[globalToShmemAsyncCopy] - Starting...\n");

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("Usage -device=n (n >= 0 for deviceID)\n");
    printf("      -wA=WidthA -hA=HeightA (Width x Height of Matrix A)\n");
    printf("      -wB=WidthB -hB=HeightB (Width x Height of Matrix B)\n");
    printf(
        "      -kernel=kernel_number (0 - AsyncCopyMultiStageLargeChunk; 1 - "
        "AsyncCopyLargeChunk)\n");
    printf(
        "                            (2 - AsyncCopyLargeChunkAWBarrier; 3 - "
        "AsyncCopyMultiStageSharedState)\n");
    printf(
        "                            (4 - AsyncCopyMultiStage; 5 - "
        "AsyncCopySingleStage; 6 - Naive without memcpy_async)\n");
    printf(
        "                            (7 - NaiveLargeChunk without "
        "memcpy_async)\n");
    printf(
        "  Note: Outer matrix dimensions of A & B matrices must be equal.\n");

    exit(EXIT_SUCCESS);
  }

  // This will pick the best possible CUDA capable device, otherwise
  // override the device ID based on input provided at the command line
  int dev = findCudaDevice(argc, (const char **)argv);

  int matrixBlock = 32;
  dim3 dimsA(10 * 4 * matrixBlock, 10 * 4 * matrixBlock, 1);
  dim3 dimsB(10 * 4 * matrixBlock, 10 * 4 * matrixBlock, 1);

  // width of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "wA")) {
    dimsA.x = getCmdLineArgumentInt(argc, (const char **)argv, "wA");
  }

  // height of Matrix A
  if (checkCmdLineFlag(argc, (const char **)argv, "hA")) {
    dimsA.y = getCmdLineArgumentInt(argc, (const char **)argv, "hA");
  }

  // width of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "wB")) {
    dimsB.x = getCmdLineArgumentInt(argc, (const char **)argv, "wB");
  }

  // height of Matrix B
  if (checkCmdLineFlag(argc, (const char **)argv, "hB")) {
    dimsB.y = getCmdLineArgumentInt(argc, (const char **)argv, "hB");
  }

  if (dimsA.x != dimsB.y) {
    printf("Error: outer matrix dimensions must be equal. (%d != %d)\n",
           dimsA.x, dimsB.y);
    exit(EXIT_FAILURE);
  }

  kernels selected_kernel = AsyncCopyMultiStageLargeChunk;

  // kernel to run - default (AsyncCopyMultiStageLargeChunk == 0)
  if (checkCmdLineFlag(argc, (const char **)argv, "kernel")) {
    int kernel_number =
        getCmdLineArgumentInt(argc, (const char **)argv, "kernel");
    if (kernel_number < 8) {
      selected_kernel = (kernels)kernel_number;
    } else {
      printf(
          "Error: kernel number should be between 0 to 6, you have entered "
          "%d\n",
          kernel_number);
      exit(EXIT_FAILURE);
    }
  }

  int major = 0;
  checkCudaErrors(
      hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, dev));
  if (major < 7) {
    printf("globalToShmemAsyncCopy requires SM 7.0 or higher.  Exiting...\n");
    exit(EXIT_WAIVED);
  }

  printf("MatrixA(%d,%d), MatrixB(%d,%d)\n", dimsA.x, dimsA.y, dimsB.x,
         dimsB.y);

  int matrix_result = MatrixMultiply(argc, argv, dimsA, dimsB, selected_kernel);

  exit(matrix_result);
}
