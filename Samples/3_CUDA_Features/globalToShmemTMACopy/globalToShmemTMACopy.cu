#include "hip/hip_runtime.h"
/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


/**
 * This sample demonstrates how to:
 *
 * - Create a TensorMap (TMA descriptor)
 * - Load a 2D tile of data into shared memory
 *
 * Compile and run with:
 *
 * nvcc -arch sm_90 -run globalToShmemTMACopy.cu
 *
 * It can be that the compiler issues the following note. This can be safely ignored.
 *
 *   note: the ABI for passing parameters with 64-byte alignment has changed in
 *   GCC 4.6
 *
 */
#include <cstdio>               // printf
#include <vector>               // std::vector

#include <cudaTypedefs.h>       // PFN_cuTensorMapEncodeTiled
#include <hip/hip_runtime.h>               // CUtensormap

#include <cuda_awbarrier_primitives.h> // __mbarrier_*

#include "util.h"                      // CUDA_CHECK macro

/*
 * Constants.
 */
constexpr size_t W_global = 1024; // Width of tensor (in # elements)
constexpr size_t H_global = 1024; // Height of tensor (in # elements)

constexpr int SMEM_W = 32;     // Width of shared memory buffer (in # elements)
constexpr int SMEM_H = 8;      // Height of shared memory buffer (in # elements)

/*
 * CUDA Driver API
 */

PFN_cuTensorMapEncodeTiled get_cuTensorMapEncodeTiled() {
  void* driver_ptr = nullptr;
  hipDriverEntryPointQueryResult driver_status;
  CUDA_CHECK(hipGetDriverEntryPoint("cuTensorMapEncodeTiled", &driver_ptr, hipEnableDefault, &driver_status));
  return reinterpret_cast<PFN_cuTensorMapEncodeTiled>(driver_ptr);
}

/*
 * PTX wrappers
 */

inline __device__ __mbarrier_token_t barrier_arrive1_tx(
  __mbarrier_t *barrier, uint32_t expected_tx_count
)
{
  // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#parallel-synchronization-and-communication-instructions-mbarrier-arrive
  __mbarrier_token_t token;

  asm volatile("mbarrier.arrive.expect_tx.release.cta.shared::cta.b64 %0, [%1], %2;"
               : "=l"(token)
               : "r"(static_cast<unsigned int>(__cvta_generic_to_shared(barrier))), "r"(expected_tx_count)
               : "memory");
  return token;
}

inline __device__ bool barrier_try_wait_token(__mbarrier_t *barrier, __mbarrier_token_t token)
{
  // https://docs.nvidia.com/cuda/parallel-thread-execution/index.html#parallel-synchronization-and-communication-instructions-mbarrier-test-wait-try-wait
  //
  // This function returns a bool, so that software can retry.
  //
  //  The HW only provides best-effort waiting support. The wait time is limited
  //  by the HW capability, after which a fail occurs, in which case the SW is
  //  responsible for retrying.
  int __ready;
  asm volatile("{\n\t"
               ".reg .pred p;\n\t"
               "mbarrier.try_wait.acquire.cta.shared::cta.b64 p, [%1], %2;\n\t"
               "selp.b32 %0, 1, 0, p;\n\t"
               "}"
               : "=r"(__ready)
               : "r"(static_cast<unsigned int>(__cvta_generic_to_shared(barrier))),
                 "l"(token)
               : "memory");
  return __ready;
}

inline __device__ void cp_async_bulk_tensor_2d(
  __mbarrier_t *barrier, void *dst, int access_coord_x, int access_coord_y, const CUtensorMap *tensor_desc)
{
  unsigned smem_int_ptr = static_cast<unsigned int>(__cvta_generic_to_shared(dst));
  unsigned smem_barrier_int_ptr = static_cast<unsigned int>(__cvta_generic_to_shared(barrier));
  uint64_t tensor_desc_ptr = reinterpret_cast<uint64_t>(tensor_desc);

  asm volatile(
    "cp.async.bulk.tensor.2d.shared::cluster.global.tile.mbarrier::complete_tx::bytes "
    "[%0], [%1, {%2, %3}], [%4];\n"
    :
    : "r"(smem_int_ptr),
      "l"(tensor_desc_ptr),
      "r"(access_coord_x),
      "r"(access_coord_y),
      "r"(smem_barrier_int_ptr)
    : "memory");
}

// Layout of shared memory. It contains:
//
// - a buffer to hold a subset of a tensor,
// - a shared memory barrier.
template <int H, int W>
struct smem_t {

  // The destination shared memory buffer of a bulk tensor operation should be
  // 128 byte aligned.
  struct alignas(128) tensor_buffer {
    int data[H][W];

    __device__ constexpr int width() {return W;}
    __device__ constexpr int height() {return H;}
  };

  tensor_buffer buffer;

  // Put the barrier behind the tensor buffer to prevent 100+ bytes of padding.
  __mbarrier_t bar;

  __device__ constexpr int buffer_size_in_bytes() {
    return sizeof(tensor_buffer::data);
  }
};


/*
 * Main kernel: takes a TMA descriptor and two coordinates.
 *
 * Loads a tile into shared memory using TMA and prints the tile.
 *
 */
__global__ void kernel(const __grid_constant__ CUtensorMap tma_desc, int x_0, int y_0) {
  /*
   * ***NOTE***:

     A CUtensorMap can only be passed as a `const __grid_constant__`
     parameter. Passing a CUtensorMap in any other way from the host to
     device can result in difficult if not impossible to debug failures.

  */

  // Declare shared memory to hold tensor buffer and shared memory barrier.
  __shared__ smem_t<SMEM_H, SMEM_W> smem;

  // Utility variable to elect a leader thread.
  bool leader = threadIdx.x == 0;


  if (leader) {
    // Initialize barrier. We will participate in the barrier with `blockDim.x`
    // threads.
    __mbarrier_init(&smem.bar, blockDim.x);
  }
  // Syncthreads so initialized barrier is visible to all threads.
  __syncthreads();


  // This token is created when arriving on the shared memory barrier. It is
  // used again when waiting on the barrier.
  __mbarrier_token_t token;

  // Load first  batch
  if (leader) {
    // Initiate bulk tensor copy.
    cp_async_bulk_tensor_2d(&smem.bar, &smem.buffer.data, x_0, y_0, &tma_desc);
    // Arrive with arrival count of 1 and expected transaction count equal to
    // the number of bytes that are copied by cp_async_bulk_tensor_2d.
    token = barrier_arrive1_tx(&smem.bar, smem.buffer_size_in_bytes());
  } else {
    // Other threads arrive with arrival count of 1 and expected tx count of 0.
    token = barrier_arrive1_tx(&smem.bar, 0);
  }

  // The barrier will flip when the following two conditions have been met:
  //
  // - Its arrival count reaches blockDim.x (see __mbarrier_init above).
  //   Typically, each thread will arrive with an arrival count of one so this
  //   indicates that all threads have arrived.
  //
  // - Its expected transaction count reaches smem.buffer_size_in_bytes(). The
  //   bulk tensor operation will increment the transaction count as it copies
  //   bytes.

  // Wait for barrier to flip. Try_wait puts the thread to sleep while waiting.
  // It is woken up when the barrier flips or when a hardware-defined number of
  // clock cycles have passed. In the second case, we retry waiting.
  while(! barrier_try_wait_token(&smem.bar, token)) { };

  // From this point onwards, the data in smem.buffer is readable by all threads
  // participating the in the barrier.

  // Print the data:
  if (leader) {
    printf("\n\nPrinting tile at coordinates x0 = %d, y0 = %d\n", x_0, y_0);

    // Print global x coordinates
    printf("global->\t");
    for (int x = 0; x < smem.buffer.width(); ++x) {
      printf("[%4d] ", x_0 + x);
    }
    printf("\n");

    // Print local x coordinates
    printf("local ->\t");
    for (int x = 0; x < smem.buffer.width(); ++x) {
      printf("[%4d] ", x);
    }
    printf("\n");

    for (int y = 0; y < smem.buffer.height(); ++y) {
      // Print global and local y coordinates
      printf("[%4d] [%2d]\t", y_0 + y, y);
      for (int x = 0; x < smem.buffer.width(); ++x) {
        printf(" %4d  ", smem.buffer.data[y][x]);
      }
      printf("\n");
    }

    // Invalidate barrier. If further computations were to take place in the
    // kernel, this allows the memory location of the shared memory barrier to
    // be repurposed.
    __mbarrier_inval(&smem.bar);
  }
}

int main(int argc, char **argv) {

  // Create a 2D tensor in GPU global memory containing linear indices 0, 1, 2, ... .
  // The data layout is row-major.

  // First fill in a vector on the host.
  std::vector<int> tensor_host(H_global * W_global);
  for (int i = 0; i < H_global * W_global; ++i) {
    tensor_host[i] = i;
  }

  // Move it to device
  int * tensor = nullptr;
  CUDA_CHECK(hipMalloc(&tensor, H_global * W_global * sizeof(int)));
  CUDA_CHECK(hipMemcpy(tensor, tensor_host.data(), H_global * W_global * sizeof(int), hipMemcpyHostToDevice));

  // Set up parameters to create TMA descriptor.
  // https://docs.nvidia.com/cuda/cuda-driver-api/group__CUDA__TENSOR__MEMORY.html

  CUtensorMap tma_desc{};
  CUtensorMapDataType dtype = CUtensorMapDataType::CU_TENSOR_MAP_DATA_TYPE_INT32;
  auto rank = 2;
  uint64_t size[rank] = {W_global, H_global};
  // The stride is the number of bytes to traverse from the first element of one row to the next.
  // It must be a multiple of 16.
  uint64_t stride[rank - 1] = {W_global * sizeof(int)};
  // The box_size is the size of the shared memory buffer that is used as the destination of a TMA transfer.
  uint32_t box_size[rank] = {SMEM_W, SMEM_H};
  // The distance between elements in units of sizeof(element). A stride of 2
  // can be used to load only the real component of a complex-valued tensor, for instance.
  uint32_t elem_stride[rank] = {1, 1};
  // Interleave patterns are sometimes used to accelerate loading of values that
  // are less than 4 bytes long.
  CUtensorMapInterleave interleave = CUtensorMapInterleave::CU_TENSOR_MAP_INTERLEAVE_NONE;
  // Swizzling can be used to avoid shared memory bank conflicts.
  CUtensorMapSwizzle swizzle = CUtensorMapSwizzle::CU_TENSOR_MAP_SWIZZLE_NONE;
  CUtensorMapL2promotion l2_promotion = CUtensorMapL2promotion::CU_TENSOR_MAP_L2_PROMOTION_NONE;
  // Any element that is outside of bounds will be set to zero by the TMA transfer.
  CUtensorMapFloatOOBfill oob_fill = CUtensorMapFloatOOBfill::CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE;

  // Get a function pointer to the cuTensorMapEncodeTiled driver API.
  auto cuTensorMapEncodeTiled = get_cuTensorMapEncodeTiled();

  // Create the tensor descriptor.
  hipError_t res = cuTensorMapEncodeTiled(
      &tma_desc,    // CUtensorMap *tensorMap,
      dtype,        // CUtensorMapDataType tensorDataType,
      rank,         // cuuint32_t tensorRank,
      tensor,       // void *globalAddress,
      size,         // const cuuint64_t *globalDim,
      stride,       // const cuuint64_t *globalStrides,
      box_size,     // const cuuint32_t *boxDim,
      elem_stride,  // const cuuint32_t *elementStrides,
      interleave,   // CUtensorMapInterleave interleave,
      swizzle,      // CUtensorMapSwizzle swizzle,
      l2_promotion, // CUtensorMapL2promotion l2Promotion,
      oob_fill      // CUtensorMapFloatOOBfill oobFill);
    );
  // Print the result. Should be zero.
  printf("cuTensorMapEncodeTiled returned hipError_t: %d\n\n", res);

  CUDA_CHECK(hipDeviceSynchronize());

  dim3 grid(1);
  dim3 block(128);

  printf("Print the top right corner tile of the tensor:\n");
  kernel<<<grid, block>>>(tma_desc, 0, 0);
  CUDA_CHECK(hipDeviceSynchronize());

  printf("Negative indices work:\n");
  kernel<<<grid, block>>>(tma_desc, -4, 0);
  CUDA_CHECK(hipDeviceSynchronize());

  printf("When the indices are out of bounds, the shared memory buffer is filled with zeros:\n");
  kernel<<<grid, block>>>(tma_desc, W_global, H_global);
  CUDA_CHECK(hipDeviceSynchronize());

  printf(
    "\n**NOTE**: The following code will fail.\n "
    "\nCare must be taken to ensure that the coordinates result in a memory offset\n"
    "that is aligned to 16 bytes. With 32 bit integer elements, x coordinates\n"
    "that are not a multiple of 4 result in a non-recoverable error:\n\n"
  );
  kernel<<<grid, block>>>(tma_desc, 1, 0);
  CUDA_REPORT(hipDeviceSynchronize());
  kernel<<<grid, block>>>(tma_desc, 2, 0);
  CUDA_REPORT(hipDeviceSynchronize());
  kernel<<<grid, block>>>(tma_desc, 3, 0);
  CUDA_REPORT(hipDeviceSynchronize());

  CUDA_REPORT(hipFree(tensor));
  return 0;
}
