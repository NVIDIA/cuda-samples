#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// -----------------------------------------------------------------------------
// Transpose
//
// This file contains both device and host code for transposing a floating-point
// matrix.  It performs several transpose kernels, which incrementally improve
// performance through coalescing, removing shared memory bank conflicts, and
// eliminating partition camping.  Several of the kernels perform a copy, used
// to represent the best case performance that a transpose can achieve.
//
// Please see the whitepaper in the docs folder of the transpose project for a
// detailed description of this performance study.
// -----------------------------------------------------------------------------

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
// Utilities and system includes
#include <helper_string.h>    // helper for string parsing
#include <helper_image.h>     // helper for image and data comparison
#include <hip/hip_runtime_api.h>      // helper for cuda error checking functions

const char *sSDKsample = "Transpose";

// Each block transposes/copies a tile of TILE_DIM x TILE_DIM elements
// using TILE_DIM x BLOCK_ROWS threads, so that each thread transposes
// TILE_DIM/BLOCK_ROWS elements.  TILE_DIM must be an integral multiple of
// BLOCK_ROWS

#define TILE_DIM 16
#define BLOCK_ROWS 16

// This sample assumes that MATRIX_SIZE_X = MATRIX_SIZE_Y
int MATRIX_SIZE_X = 1024;
int MATRIX_SIZE_Y = 1024;
int MUL_FACTOR = TILE_DIM;

#define FLOOR(a, b) (a - (a % b))

// Compute the tile size necessary to illustrate performance cases for SM20+
// hardware
int MAX_TILES = (FLOOR(MATRIX_SIZE_X, 512) * FLOOR(MATRIX_SIZE_Y, 512)) /
                (TILE_DIM * TILE_DIM);

// Number of repetitions used for timing.  Two sets of repetitions are
// performed: 1) over kernel launches and 2) inside the kernel over just the
// loads and stores

#define NUM_REPS 100

// -------------------------------------------------------
// Copies
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void copy(float *odata, float *idata, int width, int height) {
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index = xIndex + width * yIndex;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index + i * width] = idata[index + i * width];
  }
}

__global__ void copySharedMem(float *odata, float *idata, int width,
                              int height) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index = xIndex + width * yIndex;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    if (xIndex < width && yIndex < height) {
      tile[threadIdx.y][threadIdx.x] = idata[index];
    }
  }

  cg::sync(cta);

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    if (xIndex < height && yIndex < width) {
      odata[index] = tile[threadIdx.y][threadIdx.x];
    }
  }
}

// -------------------------------------------------------
// Transposes
// width and height must be integral multiples of TILE_DIM
// -------------------------------------------------------

__global__ void transposeNaive(float *odata, float *idata, int width,
                               int height) {
  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

  int index_in = xIndex + width * yIndex;
  int index_out = yIndex + height * xIndex;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index_out + i] = idata[index_in + i * width];
  }
}

// coalesced transpose (with bank conflicts)

__global__ void transposeCoalesced(float *odata, float *idata, int width,
                                   int height) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    tile[threadIdx.y + i][threadIdx.x] = idata[index_in + i * width];
  }

  cg::sync(cta);

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
  }
}

// Coalesced transpose with no bank conflicts

__global__ void transposeNoBankConflicts(float *odata, float *idata, int width,
                                         int height) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float tile[TILE_DIM][TILE_DIM + 1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    tile[threadIdx.y + i][threadIdx.x] = idata[index_in + i * width];
  }

  cg::sync(cta);

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
  }
}

// Transpose that effectively reorders execution of thread blocks along
// diagonals of the matrix (also coalesced and has no bank conflicts)
//
// Here blockIdx.x is interpreted as the distance along a diagonal and
// blockIdx.y as corresponding to different diagonals
//
// blockIdx_x and blockIdx_y expressions map the diagonal coordinates to the
// more commonly used cartesian coordinates so that the only changes to the code
// from the coalesced version are the calculation of the blockIdx_x and
// blockIdx_y and replacement of blockIdx.x and bloclIdx.y with the subscripted
// versions in the remaining code

__global__ void transposeDiagonal(float *odata, float *idata, int width,
                                  int height) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float tile[TILE_DIM][TILE_DIM + 1];

  int blockIdx_x, blockIdx_y;

  // do diagonal reordering
  if (width == height) {
    blockIdx_y = blockIdx.x;
    blockIdx_x = (blockIdx.x + blockIdx.y) % gridDim.x;
  } else {
    int bid = blockIdx.x + gridDim.x * blockIdx.y;
    blockIdx_y = bid % gridDim.y;
    blockIdx_x = ((bid / gridDim.y) + blockIdx_y) % gridDim.x;
  }

  // from here on the code is same as previous kernel except blockIdx_x replaces
  // blockIdx.x and similarly for y

  int xIndex = blockIdx_x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx_y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx_y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx_x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    tile[threadIdx.y + i][threadIdx.x] = idata[index_in + i * width];
  }

  cg::sync(cta);

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index_out + i * height] = tile[threadIdx.x][threadIdx.y + i];
  }
}

// --------------------------------------------------------------------
// Partial transposes
// NB: the coarse- and fine-grained routines only perform part of a
//     transpose and will fail the test against the reference solution
//
//     They are used to assess performance characteristics of different
//     components of a full transpose
// --------------------------------------------------------------------

__global__ void transposeFineGrained(float *odata, float *idata, int width,
                                     int height) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float block[TILE_DIM][TILE_DIM + 1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index = xIndex + (yIndex)*width;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    block[threadIdx.y + i][threadIdx.x] = idata[index + i * width];
  }

  cg::sync(cta);

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index + i * height] = block[threadIdx.x][threadIdx.y + i];
  }
}

__global__ void transposeCoarseGrained(float *odata, float *idata, int width,
                                       int height) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float block[TILE_DIM][TILE_DIM + 1];

  int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
  int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;
  int index_in = xIndex + (yIndex)*width;

  xIndex = blockIdx.y * TILE_DIM + threadIdx.x;
  yIndex = blockIdx.x * TILE_DIM + threadIdx.y;
  int index_out = xIndex + (yIndex)*height;

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    block[threadIdx.y + i][threadIdx.x] = idata[index_in + i * width];
  }

  cg::sync(cta);

  for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
    odata[index_out + i * height] = block[threadIdx.y + i][threadIdx.x];
  }
}

// ---------------------
// host utility routines
// ---------------------

void computeTransposeGold(float *gold, float *idata, const int size_x,
                          const int size_y) {
  for (int y = 0; y < size_y; ++y) {
    for (int x = 0; x < size_x; ++x) {
      gold[(x * size_y) + y] = idata[(y * size_x) + x];
    }
  }
}

void getParams(int argc, char **argv, hipDeviceProp_t &deviceProp, int &size_x,
               int &size_y, int max_tile_dim) {
  // set matrix size (if (x,y) dim of matrix is not square, then this will have
  // to be modified
  if (checkCmdLineFlag(argc, (const char **)argv, "dimX")) {
    size_x = getCmdLineArgumentInt(argc, (const char **)argv, "dimX");

    if (size_x > max_tile_dim) {
      printf("> MatrixSize X = %d is greater than the recommended size = %d\n",
             size_x, max_tile_dim);
    } else {
      printf("> MatrixSize X = %d\n", size_x);
    }
  } else {
    size_x = max_tile_dim;
    size_x = FLOOR(size_x, 512);
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "dimY")) {
    size_y = getCmdLineArgumentInt(argc, (const char **)argv, "dimY");

    if (size_y > max_tile_dim) {
      printf("> MatrixSize Y = %d is greater than the recommended size = %d\n",
             size_y, max_tile_dim);
    } else {
      printf("> MatrixSize Y = %d\n", size_y);
    }
  } else {
    size_y = max_tile_dim;
    size_y = FLOOR(size_y, 512);
  }
}

void showHelp() {
  printf("\n%s : Command line options\n", sSDKsample);
  printf("\t-device=n          (where n=0,1,2.... for the GPU device)\n\n");
  printf("> The default matrix size can be overridden with these parameters\n");
  printf("\t-dimX=row_dim_size (matrix row    dimensions)\n");
  printf("\t-dimY=col_dim_size (matrix column dimensions)\n");
}

// ----
// main
// ----

int main(int argc, char **argv) {
  // Start logs
  printf("%s Starting...\n\n", sSDKsample);

  if (checkCmdLineFlag(argc, (const char **)argv, "help")) {
    showHelp();
    return 0;
  }

  int devID = findCudaDevice(argc, (const char **)argv);
  hipDeviceProp_t deviceProp;

  // get number of SMs on this GPU
  checkCudaErrors(hipGetDevice(&devID));
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

  // compute the scaling factor (for GPUs with fewer MPs)
  float scale_factor, total_tiles;
  scale_factor =
      max((192.0f / (_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
                     (float)deviceProp.multiProcessorCount)),
          1.0f);

  printf("> Device %d: \"%s\"\n", devID, deviceProp.name);
  printf("> SM Capability %d.%d detected:\n", deviceProp.major,
         deviceProp.minor);

  // Calculate number of tiles we will run for the Matrix Transpose performance
  // tests
  int size_x, size_y, max_matrix_dim, matrix_size_test;

  matrix_size_test = 512;  // we round down max_matrix_dim for this perf test
  total_tiles = (float)MAX_TILES / scale_factor;

  max_matrix_dim =
      FLOOR((int)(floor(sqrt(total_tiles)) * TILE_DIM), matrix_size_test);

  // This is the minimum size allowed
  if (max_matrix_dim == 0) {
    max_matrix_dim = matrix_size_test;
  }

  printf("> [%s] has %d MP(s) x %d (Cores/MP) = %d (Cores)\n", deviceProp.name,
         deviceProp.multiProcessorCount,
         _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor),
         _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) *
             deviceProp.multiProcessorCount);

  printf("> Compute performance scaling factor = %4.2f\n", scale_factor);

  // Extract parameters if there are any, command line -dimx and -dimy can
  // override any of these settings
  getParams(argc, argv, deviceProp, size_x, size_y, max_matrix_dim);

  if (size_x != size_y) {
    printf(
        "\n[%s] does not support non-square matrices (row_dim_size(%d) != "
        "col_dim_size(%d))\nExiting...\n\n",
        sSDKsample, size_x, size_y);
    exit(EXIT_FAILURE);
  }

  if (size_x % TILE_DIM != 0 || size_y % TILE_DIM != 0) {
    printf(
        "[%s] Matrix size must be integral multiple of tile "
        "size\nExiting...\n\n",
        sSDKsample);
    exit(EXIT_FAILURE);
  }

  // kernel pointer and descriptor
  void (*kernel)(float *, float *, int, int);
  const char *kernelName;

  // execution configuration parameters
  dim3 grid(size_x / TILE_DIM, size_y / TILE_DIM),
      threads(TILE_DIM, BLOCK_ROWS);

  if (grid.x < 1 || grid.y < 1) {
    printf("[%s] grid size computation incorrect in test \nExiting...\n\n",
           sSDKsample);
    exit(EXIT_FAILURE);
  }

  // CUDA events
  hipEvent_t start, stop;

  // size of memory required to store the matrix
  size_t mem_size = static_cast<size_t>(sizeof(float) * size_x * size_y);

  if (2 * mem_size > deviceProp.totalGlobalMem) {
    printf("Input matrix size is larger than the available device memory!\n");
    printf("Please choose a smaller size matrix\n");
    exit(EXIT_FAILURE);
  }

  // allocate host memory
  float *h_idata = (float *)malloc(mem_size);
  float *h_odata = (float *)malloc(mem_size);
  float *transposeGold = (float *)malloc(mem_size);
  float *gold;

  // allocate device memory
  float *d_idata, *d_odata;
  checkCudaErrors(hipMalloc((void **)&d_idata, mem_size));
  checkCudaErrors(hipMalloc((void **)&d_odata, mem_size));

  // initialize host data
  for (int i = 0; i < (size_x * size_y); ++i) {
    h_idata[i] = (float)i;
  }

  // copy host data to device
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice));

  // Compute reference transpose solution
  computeTransposeGold(transposeGold, h_idata, size_x, size_y);

  // print out common data for all kernels
  printf(
      "\nMatrix size: %dx%d (%dx%d tiles), tile size: %dx%d, block size: "
      "%dx%d\n\n",
      size_x, size_y, size_x / TILE_DIM, size_y / TILE_DIM, TILE_DIM, TILE_DIM,
      TILE_DIM, BLOCK_ROWS);

  // initialize events
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  //
  // loop over different kernels
  //

  bool success = true;

  for (int k = 0; k < 8; k++) {
    // set kernel pointer
    switch (k) {
      case 0:
        kernel = &copy;
        kernelName = "simple copy       ";
        break;

      case 1:
        kernel = &copySharedMem;
        kernelName = "shared memory copy";
        break;

      case 2:
        kernel = &transposeNaive;
        kernelName = "naive             ";
        break;

      case 3:
        kernel = &transposeCoalesced;
        kernelName = "coalesced         ";
        break;

      case 4:
        kernel = &transposeNoBankConflicts;
        kernelName = "optimized         ";
        break;

      case 5:
        kernel = &transposeCoarseGrained;
        kernelName = "coarse-grained    ";
        break;

      case 6:
        kernel = &transposeFineGrained;
        kernelName = "fine-grained      ";
        break;

      case 7:
        kernel = &transposeDiagonal;
        kernelName = "diagonal          ";
        break;
    }

    // set reference solution
    if (kernel == &copy || kernel == &copySharedMem) {
      gold = h_idata;
    } else if (kernel == &transposeCoarseGrained ||
               kernel == &transposeFineGrained) {
      gold = h_odata;  // fine- and coarse-grained kernels are not full
                       // transposes, so bypass check
    } else {
      gold = transposeGold;
    }

    // Clear error status
    checkCudaErrors(hipGetLastError());

    // warmup to avoid timing startup
    kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y);

    // take measurements for loop over kernel launches
    checkCudaErrors(hipEventRecord(start, 0));

    for (int i = 0; i < NUM_REPS; i++) {
      kernel<<<grid, threads>>>(d_odata, d_idata, size_x, size_y);
      // Ensure no launch failure
      checkCudaErrors(hipGetLastError());
    }

    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipEventSynchronize(stop));
    float kernelTime;
    checkCudaErrors(hipEventElapsedTime(&kernelTime, start, stop));

    checkCudaErrors(
        hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));
    bool res = compareData(gold, h_odata, size_x * size_y, 0.01f, 0.0f);

    if (res == false) {
      printf("*** %s kernel FAILED ***\n", kernelName);
      success = false;
    }

    // take measurements for loop inside kernel
    checkCudaErrors(
        hipMemcpy(h_odata, d_odata, mem_size, hipMemcpyDeviceToHost));
    res = compareData(gold, h_odata, size_x * size_y, 0.01f, 0.0f);

    if (res == false) {
      printf("*** %s kernel FAILED ***\n", kernelName);
      success = false;
    }

    // report effective bandwidths
    float kernelBandwidth = 2.0f * 1000.0f * mem_size / (1024 * 1024 * 1024) /
                            (kernelTime / NUM_REPS);
    printf(
        "transpose %s, Throughput = %.4f GB/s, Time = %.5f ms, Size = %u fp32 "
        "elements, NumDevsUsed = %u, Workgroup = %u\n",
        kernelName, kernelBandwidth, kernelTime / NUM_REPS, (size_x * size_y),
        1, TILE_DIM * BLOCK_ROWS);
  }

  // cleanup
  free(h_idata);
  free(h_odata);
  free(transposeGold);
  hipFree(d_idata);
  hipFree(d_odata);

  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));

  if (!success) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
