#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>
#include "commonDefs.hpp"
#include "commonKernels.hpp"

#define VERIFY_GPU_CORRECTNESS 0

size_t maxSampleSizeInMb = 64;
int numKernelRuns = 20;
int verboseResults = 0;

const char *memAllocTypeStr[MEMALLOC_TYPE_COUNT] = {
    "Managed_Memory_With_Hints",
    "Managed_Memory_With_Hints_FullyAsync",
    "Managed_Memory_NoHints",
    "Zero_Copy",
    "Memcpy_HostMalloc_DeviceCudaMalloc",
    "MemcpyAsync_HostMalloc_DeviceCudaMalloc",
    "Memcpy_HostCudaHostAlloc_DeviceCudaMalloc",
    "MemcpyAsync_HostCudaHostAlloc_DeviceCudaMalloc"};

const char *memAllocTypeShortStr[MEMALLOC_TYPE_COUNT] = {
    "UMhint",   // Managed Memory With Hints
    "UMhntAs",  // Managed Memory With_Hints Async
    "UMeasy",   // Managed_Memory with No Hints
    "0Copy",    // Zero Copy
    "MemCopy",  // USE HOST PAGEABLE AND DEVICE_MEMORY
    "CpAsync",  // USE HOST PAGEABLE AND DEVICE_MEMORY ASYNC
    "CpHpglk",  // USE HOST PAGELOCKED AND DEVICE MEMORY
    "CpPglAs"   // USE HOST PAGELOCKED AND DEVICE MEMORY ASYNC
};

static float RandFloat(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}

void fillMatrixWithRandomValues(float *matrix, unsigned int matrixDim) {
  unsigned int i, j;
  for (i = 0; i < matrixDim; ++i) {
    for (j = 0; j < matrixDim; ++j) {
      matrix[j + i * matrixDim] = RandFloat(0.0f, 10.0f);
    }
  }
}

#if VERIFY_GPU_CORRECTNESS
void verifyMatrixMultiplyCorrectness(float *C, float *A, float *B,
                                     unsigned int matrixDim) {
  unsigned int i, j, k, numErrors = 0;
  for (i = 0; i < matrixDim; ++i) {
    for (j = 0; j < matrixDim; ++j) {
      float result = 0.0f;
      for (k = 0; k < matrixDim; ++k) {
        result += A[k + i * matrixDim] * B[j + k * matrixDim];
      }
      if (fabs(C[j + i * matrixDim] - result) > 0.001 * matrixDim) {
        printf("At [%u, %u]: Expected %f, Found %f\n", i, j, result,
               C[j + i * matrixDim]);
        ++numErrors;
      }
    }
  }
  if (numErrors != 0) {
    printf("%d value mismatches occured\n", numErrors);
    fflush(stdout);
    exit(EXIT_FAILURE);  // exit since value mismatches occured
  }
}
#endif

void copyMatrix(float *dstMatrix, float *srcMatrix, unsigned int matrixDim) {
  size_t size = matrixDim * matrixDim * sizeof(float);
  memcpy(dstMatrix, srcMatrix, size);
}

void verifyMatrixData(float *expectedData, float *observedData,
                      unsigned int matrixDim) {
  unsigned int i, j, numErrors = 0;
  for (i = 0; i < matrixDim; ++i) {
    for (j = 0; j < matrixDim; ++j) {
      if (expectedData[j + i * matrixDim] != observedData[j + i * matrixDim]) {
        ++numErrors;
        if (verboseResults) {
          printf("At [%u, %u]: Expected %f, Found %f\n", i, j,
                 expectedData[j + i * matrixDim],
                 observedData[j + i * matrixDim]);
        }
      }
    }
  }
  if (numErrors != 0) {
    printf("%d value mismatches occured\n", numErrors);
    fflush(stdout);
    exit(EXIT_FAILURE);  // exit since value mismatches occured
  }
}

#define BLOCK_SIZE 32
__global__ void matrixMultiplyKernel(float *C, float *A, float *B,
                                     unsigned int matrixDim) {
  // Block index
  int bx = blockIdx.x;
  int by = blockIdx.y;

  // Thread index
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  unsigned int wA = matrixDim;
  unsigned int wB = matrixDim;

  // Index of the first sub-matrix of A processed by the block
  int aBegin = matrixDim * BLOCK_SIZE * by;

  // Index of the last sub-matrix of A processed by the block
  int aEnd = aBegin + wA - 1;

  // Step size used to iterate through the sub-matrices of A
  int aStep = BLOCK_SIZE;

  // Index of the first sub-matrix of B processed by the block
  int bBegin = BLOCK_SIZE * bx;

  // Step size used to iterate through the sub-matrices of B
  int bStep = BLOCK_SIZE * wB;

  // Csub is used to store the element of the block sub-matrix
  // that is computed by the thread
  float Csub = 0;

  // Loop over all the sub-matrices of A and B
  // required to compute the block sub-matrix
  for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
    // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Load the matrices from device memory
    // to shared memory; each thread loads
    // one element of each matrix
    As[ty][tx] = A[a + wA * ty + tx];
    Bs[ty][tx] = B[b + wB * ty + tx];

    // Synchronize to make sure the matrices are loaded
    __syncthreads();

    // Multiply the two matrices together;
    // each thread computes one element
    // of the block sub-matrix
#pragma unroll

    for (int k = 0; k < BLOCK_SIZE; ++k) {
      Csub += As[ty][k] * Bs[k][tx];
    }

    // Synchronize to make sure that the preceding
    // computation is done before loading two new
    // sub-matrices of A and B in the next iteration
    __syncthreads();
  }

  // Write the block sub-matrix to device memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}

void runMatrixMultiplyKernel(unsigned int matrixDim, int allocType,
                             unsigned int numLoops, double *gpuLaunchCallsTimes,
                             double *gpuTransferToCallsTimes,
                             double *gpuTransferFromCallsTimes,
                             double *gpuLaunchAndTransferCallsTimes,
                             double *gpuLaunchTransferSyncTimes,
                             double *cpuAccessTimes, double *overallTimes,
                             int device_id) {
  float *dptrA = NULL, *hptrA = NULL;
  float *dptrB = NULL, *hptrB = NULL;
  float *dptrC = NULL, *hptrC = NULL;
  float *randValuesX = NULL, *randValuesY = NULL;
  float *randValuesVerifyXmulY = NULL, *randValuesVerifyYmulX = NULL;
  bool copyRequired = false, hintsRequired = false;
  bool someTransferOpRequired;
  bool isAsync = false;
  hipStream_t streamToRunOn;
  unsigned int *latch;
  size_t size = matrixDim * matrixDim * sizeof(float);
  dim3 threads(32, 32);
  dim3 grid(matrixDim / threads.x, matrixDim / threads.y);
  StopWatchInterface *gpuLaunchCallsTimer = 0, *gpuTransferCallsTimer = 0;
  StopWatchInterface *gpuSyncTimer = 0, *cpuAccessTimer = 0;
  sdkCreateTimer(&gpuLaunchCallsTimer);
  sdkCreateTimer(&gpuTransferCallsTimer);
  sdkCreateTimer(&gpuSyncTimer);
  sdkCreateTimer(&cpuAccessTimer);
  unsigned int i;

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, device_id));
  checkCudaErrors(hipStreamCreate(&streamToRunOn));

  randValuesX = (float *)malloc(size);
  if (!randValuesX) {
    exit(EXIT_FAILURE);  // exit since memory allocation error
  }
  randValuesY = (float *)malloc(size);
  if (!randValuesY) {
    exit(EXIT_FAILURE);  // exit since memory allocation error
  }
  randValuesVerifyXmulY = (float *)malloc(size);
  if (!randValuesVerifyXmulY) {
    exit(EXIT_FAILURE);  // exit since memory allocation error
  }
  randValuesVerifyYmulX = (float *)malloc(size);
  if (!randValuesVerifyYmulX) {
    exit(EXIT_FAILURE);  // exit since memory allocation error
  }
  checkCudaErrors(hipMalloc(&dptrA, size));
  checkCudaErrors(hipMalloc(&dptrB, size));
  checkCudaErrors(hipMalloc(&dptrC, size));

  fillMatrixWithRandomValues(randValuesX, matrixDim);
  fillMatrixWithRandomValues(randValuesY, matrixDim);

  checkCudaErrors(
      hipMemcpyAsync(dptrA, randValuesX, size, hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpyAsync(dptrB, randValuesY, size, hipMemcpyHostToDevice));
  matrixMultiplyKernel<<<grid, threads>>>(dptrC, dptrA, dptrB, matrixDim);
  checkCudaErrors(hipMemcpyAsync(randValuesVerifyXmulY, dptrC, size,
                                  hipMemcpyDeviceToHost));
  checkCudaErrors(hipStreamSynchronize(NULL));
  matrixMultiplyKernel<<<grid, threads>>>(dptrC, dptrB, dptrA, matrixDim);
  checkCudaErrors(hipMemcpyAsync(randValuesVerifyYmulX, dptrC, size,
                                  hipMemcpyDeviceToHost));
  checkCudaErrors(hipStreamSynchronize(NULL));
#if VERIFY_GPU_CORRECTNESS
  verifyMatrixMultiplyCorrectness(randValuesVerifyXmulY, randValuesX,
                                  randValuesY, matrixDim);
  verifyMatrixMultiplyCorrectness(randValuesVerifyYmulX, randValuesY,
                                  randValuesX, matrixDim);
#endif
  checkCudaErrors(hipFree(dptrA));
  checkCudaErrors(hipFree(dptrB));
  checkCudaErrors(hipFree(dptrC));

  checkCudaErrors(hipHostMalloc(&latch, sizeof(unsigned int)));

  switch (allocType) {
    case USE_HOST_PAGEABLE_AND_DEVICE_MEMORY:
    case USE_HOST_PAGEABLE_AND_DEVICE_MEMORY_ASYNC:
      hptrA = (float *)malloc(size);
      if (!hptrA) {
        exit(EXIT_FAILURE);  // exit since memory allocation error
      }
      hptrB = (float *)malloc(size);
      if (!hptrB) {
        exit(EXIT_FAILURE);  // exit since memory allocation error
      }
      hptrC = (float *)malloc(size);
      if (!hptrC) {
        exit(EXIT_FAILURE);  // exit since memory allocation error
      }
      checkCudaErrors(hipMalloc(&dptrA, size));
      checkCudaErrors(hipMalloc(&dptrB, size));
      checkCudaErrors(hipMalloc(&dptrC, size));
      copyRequired = true;
      break;

    case USE_HOST_PAGELOCKED_AND_DEVICE_MEMORY:
    case USE_HOST_PAGELOCKED_AND_DEVICE_MEMORY_ASYNC:
      checkCudaErrors(hipHostMalloc(&hptrA, size));
      checkCudaErrors(hipHostMalloc(&hptrB, size));
      checkCudaErrors(hipHostMalloc(&hptrC, size));
      checkCudaErrors(hipMalloc(&dptrA, size));
      checkCudaErrors(hipMalloc(&dptrB, size));
      checkCudaErrors(hipMalloc(&dptrC, size));
      copyRequired = true;
      break;

    case USE_ZERO_COPY:
      checkCudaErrors(hipHostMalloc(&hptrA, size));
      checkCudaErrors(hipHostMalloc(&hptrB, size));
      checkCudaErrors(hipHostMalloc(&hptrC, size));
      checkCudaErrors(hipHostGetDevicePointer(&dptrA, hptrA, 0));
      checkCudaErrors(hipHostGetDevicePointer(&dptrB, hptrB, 0));
      checkCudaErrors(hipHostGetDevicePointer(&dptrC, hptrC, 0));
      break;

    case USE_MANAGED_MEMORY:
      checkCudaErrors(hipMallocManaged(&dptrA, size));
      checkCudaErrors(hipMallocManaged(&dptrB, size));
      checkCudaErrors(hipMallocManaged(&dptrC, size));
      hptrA = dptrA;
      hptrB = dptrB;
      hptrC = dptrC;
      break;

    case USE_MANAGED_MEMORY_WITH_HINTS:
    case USE_MANAGED_MEMORY_WITH_HINTS_ASYNC:
      if (deviceProp.concurrentManagedAccess) {
        checkCudaErrors(hipMallocManaged(&dptrA, size));
        checkCudaErrors(hipMallocManaged(&dptrB, size));
        checkCudaErrors(hipMallocManaged(&dptrC, size));
        checkCudaErrors(hipMemPrefetchAsync(dptrA, size, hipCpuDeviceId));
        checkCudaErrors(hipMemPrefetchAsync(dptrB, size, hipCpuDeviceId));
        checkCudaErrors(hipMemPrefetchAsync(dptrC, size, hipCpuDeviceId));
      } else {
        checkCudaErrors(hipMallocManaged(&dptrA, size, hipMemAttachHost));
        checkCudaErrors(hipMallocManaged(&dptrB, size, hipMemAttachHost));
        checkCudaErrors(hipMallocManaged(&dptrC, size, hipMemAttachHost));
      }
      hptrA = dptrA;
      hptrB = dptrB;
      hptrC = dptrC;
      hintsRequired = true;
      break;

    default:
      exit(EXIT_FAILURE);  // exit with error
  }

  if (allocType == USE_HOST_PAGEABLE_AND_DEVICE_MEMORY_ASYNC ||
      allocType == USE_HOST_PAGELOCKED_AND_DEVICE_MEMORY_ASYNC ||
      allocType == USE_MANAGED_MEMORY_WITH_HINTS_ASYNC) {
    isAsync = true;
  }

  someTransferOpRequired = copyRequired || hintsRequired;

  // fill buffers with 0 to avoid any first access page-fault overheads.
  memset(hptrA, 0, size);
  memset(hptrB, 0, size);
  memset(hptrC, 0, size);

  for (i = 0; i < numLoops; i++) {
    cpuAccessTimes[i] = 0.0;
    gpuLaunchCallsTimes[i] = 0.0;
    gpuTransferToCallsTimes[i] = 0.0;
    gpuTransferFromCallsTimes[i] = 0.0;

    sdkStartTimer(&cpuAccessTimer);
    {
      copyMatrix(hptrA, (i & 0x1 == 0) ? randValuesX : randValuesY, matrixDim);
      copyMatrix(hptrB, (i & 0x1 == 0) ? randValuesY : randValuesX, matrixDim);
    }
    sdkStopTimer(&cpuAccessTimer);
    cpuAccessTimes[i] += sdkGetAverageTimerValue(&cpuAccessTimer);
    sdkResetTimer(&cpuAccessTimer);

    if (isAsync && hintsRequired) {
      *latch = 0;
      // Prevent any work on stream from starting until all work is pushed
      spinWhileLessThanOne<<<1, 1, 0, streamToRunOn>>>(latch);
    }

    if (someTransferOpRequired) {
      sdkStartTimer(&gpuTransferCallsTimer);
      if (copyRequired) {
        if (isAsync) {
          checkCudaErrors(hipMemcpyAsync(
              dptrA, hptrA, size, hipMemcpyHostToDevice, streamToRunOn));
          checkCudaErrors(hipMemcpyAsync(
              dptrB, hptrB, size, hipMemcpyHostToDevice, streamToRunOn));
        } else {
          checkCudaErrors(
              hipMemcpy(dptrA, hptrA, size, hipMemcpyHostToDevice));
          checkCudaErrors(
              hipMemcpy(dptrB, hptrB, size, hipMemcpyHostToDevice));
        }
      }
      if (hintsRequired) {
        if (deviceProp.concurrentManagedAccess) {
          checkCudaErrors(
              hipMemPrefetchAsync(dptrA, size, device_id, streamToRunOn));
          checkCudaErrors(
              hipMemPrefetchAsync(dptrB, size, device_id, streamToRunOn));
          checkCudaErrors(
              hipMemPrefetchAsync(dptrC, size, device_id, streamToRunOn));
        } else {
          checkCudaErrors(hipStreamAttachMemAsync(streamToRunOn, dptrA, 0,
                                                   hipMemAttachGlobal));
          checkCudaErrors(hipStreamAttachMemAsync(streamToRunOn, dptrB, 0,
                                                   hipMemAttachGlobal));
          checkCudaErrors(hipStreamAttachMemAsync(streamToRunOn, dptrC, 0,
                                                   hipMemAttachGlobal));
        }
        if (!isAsync) {
          checkCudaErrors(hipStreamSynchronize(streamToRunOn));
        }
      }

      sdkStopTimer(&gpuTransferCallsTimer);
      gpuTransferToCallsTimes[i] +=
          sdkGetAverageTimerValue(&gpuTransferCallsTimer);
      sdkResetTimer(&gpuTransferCallsTimer);
    }

    sdkStartTimer(&gpuLaunchCallsTimer);
    {
      matrixMultiplyKernel<<<grid, threads, 0, streamToRunOn>>>(
          dptrC, dptrA, dptrB, matrixDim);
      if (!isAsync) {
        checkCudaErrors(hipStreamSynchronize(streamToRunOn));
      }
    }
    sdkStopTimer(&gpuLaunchCallsTimer);

    gpuLaunchCallsTimes[i] += sdkGetAverageTimerValue(&gpuLaunchCallsTimer);
    sdkResetTimer(&gpuLaunchCallsTimer);

    if (someTransferOpRequired) {
      sdkStartTimer(&gpuTransferCallsTimer);
      if (hintsRequired) {
        if (deviceProp.concurrentManagedAccess) {
          checkCudaErrors(hipMemPrefetchAsync(dptrA, size, hipCpuDeviceId));
          checkCudaErrors(hipMemPrefetchAsync(dptrB, size, hipCpuDeviceId));
          checkCudaErrors(hipMemPrefetchAsync(dptrC, size, hipCpuDeviceId));
        } else {
          checkCudaErrors(hipStreamAttachMemAsync(streamToRunOn, dptrA, 0,
                                                   hipMemAttachHost));
          checkCudaErrors(hipStreamAttachMemAsync(streamToRunOn, dptrB, 0,
                                                   hipMemAttachHost));
          checkCudaErrors(hipStreamAttachMemAsync(streamToRunOn, dptrC, 0,
                                                   hipMemAttachHost));
        }
        if (!isAsync) {
          checkCudaErrors(hipStreamSynchronize(streamToRunOn));
        }
      }
      if (copyRequired) {
        if (isAsync) {
          checkCudaErrors(hipMemcpyAsync(
              hptrC, dptrC, size, hipMemcpyDeviceToHost, streamToRunOn));
        } else {
          checkCudaErrors(
              hipMemcpy(hptrC, dptrC, size, hipMemcpyDeviceToHost));
        }
      }
      sdkStopTimer(&gpuTransferCallsTimer);
      gpuTransferFromCallsTimes[i] +=
          sdkGetAverageTimerValue(&gpuTransferCallsTimer);
      sdkResetTimer(&gpuTransferCallsTimer);
    }
    gpuLaunchAndTransferCallsTimes[i] = gpuLaunchCallsTimes[i] +
                                        gpuTransferToCallsTimes[i] +
                                        gpuTransferFromCallsTimes[i];
    gpuLaunchTransferSyncTimes[i] = gpuLaunchAndTransferCallsTimes[i];
    if (isAsync) {
      sdkStartTimer(&gpuSyncTimer);
      {
        if (hintsRequired) {
          *latch = 1;
        }
        checkCudaErrors(hipStreamSynchronize(streamToRunOn));
      }
      sdkStopTimer(&gpuSyncTimer);
      gpuLaunchTransferSyncTimes[i] += sdkGetAverageTimerValue(&gpuSyncTimer);
      sdkResetTimer(&gpuSyncTimer);
    }

    sdkStartTimer(&cpuAccessTimer);
    {
      verifyMatrixData(
          (i & 0x1 == 0) ? randValuesVerifyXmulY : randValuesVerifyYmulX, hptrC,
          matrixDim);
    }
    sdkStopTimer(&cpuAccessTimer);
    cpuAccessTimes[i] += sdkGetAverageTimerValue(&cpuAccessTimer);
    sdkResetTimer(&cpuAccessTimer);
    overallTimes[i] = cpuAccessTimes[i] + gpuLaunchTransferSyncTimes[i];
  }

  switch (allocType) {
    case USE_HOST_PAGEABLE_AND_DEVICE_MEMORY:
    case USE_HOST_PAGEABLE_AND_DEVICE_MEMORY_ASYNC:
      free(hptrA);
      free(hptrB);
      free(hptrC);
      checkCudaErrors(hipFree(dptrA));
      checkCudaErrors(hipFree(dptrB));
      checkCudaErrors(hipFree(dptrC));
      break;

    case USE_HOST_PAGELOCKED_AND_DEVICE_MEMORY:
    case USE_HOST_PAGELOCKED_AND_DEVICE_MEMORY_ASYNC:
      checkCudaErrors(hipHostFree(hptrA));
      checkCudaErrors(hipHostFree(hptrB));
      checkCudaErrors(hipHostFree(hptrC));
      checkCudaErrors(hipFree(dptrA));
      checkCudaErrors(hipFree(dptrB));
      checkCudaErrors(hipFree(dptrC));
      break;

    case USE_ZERO_COPY:
      checkCudaErrors(hipHostFree(hptrA));
      checkCudaErrors(hipHostFree(hptrB));
      checkCudaErrors(hipHostFree(hptrC));
      break;

    case USE_MANAGED_MEMORY:
    case USE_MANAGED_MEMORY_WITH_HINTS:
    case USE_MANAGED_MEMORY_WITH_HINTS_ASYNC:
      checkCudaErrors(hipFree(dptrA));
      checkCudaErrors(hipFree(dptrB));
      checkCudaErrors(hipFree(dptrC));
      break;

    default:
      exit(EXIT_FAILURE);  // exit due to error
  }

  checkCudaErrors(hipStreamDestroy(streamToRunOn));
  checkCudaErrors(hipHostFree(latch));
  free(randValuesX);
  free(randValuesY);
  free(randValuesVerifyXmulY);
  free(randValuesVerifyYmulX);
  sdkDeleteTimer(&gpuLaunchCallsTimer);
  sdkDeleteTimer(&gpuTransferCallsTimer);
  sdkDeleteTimer(&gpuSyncTimer);
  sdkDeleteTimer(&cpuAccessTimer);
}

void matrixMultiplyPerfRunner(bool reportAsBandwidth,
                              bool print_launch_transfer_results,
                              bool print_std_deviation, int device_id) {
  int i;
  unsigned int minMatrixDim = 32;
  unsigned int multiplierDim = 2;
  unsigned int matrixDim;
  unsigned int minSize = minMatrixDim * minMatrixDim * sizeof(float);
  unsigned int maxSize =
      (maxSampleSizeInMb * ONE_MB) /
      4;  // 3 buffers are used, but dividing by 4 (power of 2)
  unsigned int multiplier = multiplierDim * multiplierDim;
  unsigned int numSizesToTest;

  struct testResults *results;
  struct resultsData *gpuLaunchCallsTimes;
  struct resultsData *gpuTransferToCallsTimes;
  struct resultsData *gpuTransferFromCallsTimes;
  struct resultsData *gpuLaunchAndTransferCallsTimes;
  struct resultsData *gpuLaunchTransferSyncTimes;
  struct resultsData *cpuAccessTimes;
  struct resultsData *overallTimes;
  unsigned long *sizesToTest;
  unsigned int j;

  numSizesToTest = findNumSizesToTest(minSize, maxSize, multiplier);

  createAndInitTestResults(&results, "matrixMultiplyPerf", numKernelRuns,
                           numSizesToTest);

  sizesToTest = getPtrSizesToTest(results);

  createResultDataAndAddToTestResults(&gpuLaunchCallsTimes, results,
                                      "GPU Kernel Launch Call Time", false,
                                      reportAsBandwidth);
  createResultDataAndAddToTestResults(&gpuTransferToCallsTimes, results,
                                      "CPU to GPU Transfer Calls Time", false,
                                      reportAsBandwidth);
  createResultDataAndAddToTestResults(&gpuTransferFromCallsTimes, results,
                                      "GPU to CPU Transfer Calls Time", false,
                                      reportAsBandwidth);
  createResultDataAndAddToTestResults(&gpuLaunchAndTransferCallsTimes, results,
                                      "GPU Launch and Transfer Calls Time",
                                      false, reportAsBandwidth);
  createResultDataAndAddToTestResults(&gpuLaunchTransferSyncTimes, results,
                                      "GPU Launch Transfer and Sync Time",
                                      false, reportAsBandwidth);
  createResultDataAndAddToTestResults(
      &cpuAccessTimes, results, "CPU Access Time", false, reportAsBandwidth);
  createResultDataAndAddToTestResults(&overallTimes, results, "Overall Time",
                                      false, reportAsBandwidth);

  printf("Running ");
  for (matrixDim = minMatrixDim, j = 0;
       matrixDim * matrixDim <= maxSize / sizeof(float);
       matrixDim *= multiplierDim, ++j) {
    sizesToTest[j] = matrixDim * matrixDim * sizeof(float);
    for (i = MEMALLOC_TYPE_START; i <= MEMALLOC_TYPE_END; i++) {
      printf(".");
      fflush(stdout);
      runMatrixMultiplyKernel(
          matrixDim, i, numKernelRuns,
          getPtrRunTimesInMs(gpuLaunchCallsTimes, i, j),
          getPtrRunTimesInMs(gpuTransferToCallsTimes, i, j),
          getPtrRunTimesInMs(gpuTransferFromCallsTimes, i, j),
          getPtrRunTimesInMs(gpuLaunchAndTransferCallsTimes, i, j),
          getPtrRunTimesInMs(gpuLaunchTransferSyncTimes, i, j),
          getPtrRunTimesInMs(cpuAccessTimes, i, j),
          getPtrRunTimesInMs(overallTimes, i, j), device_id);
    }
  }
  printf("\n");
  printResults(results, print_launch_transfer_results, print_std_deviation);
  freeTestResultsAndAllResultsData(results);
}

static void usage() {
  printf(
      "./cudaMemoryTypesPerf [-device=<device_id>] [-reportAsBandwidth] "
      "[-print-launch-transfer-results] [-print-std-deviation] [-verbose]\n");
  printf("Options:\n");
  printf(
      "-reportAsBandwidth:             By default time taken is printed, this "
      "option allows to instead print bandwidth.\n");
  printf(
      "-print-launch-transfer-results: By default overall results are printed, "
      "this option allows to print data transfers and kernel time as well.\n");
  printf(
      "-print-std-deviation:           Prints std deviation of the results.\n");
  printf(
      "-kernel-iterations=<num>:       Number of times the kernel tests should "
      "be run[default is 100 iterations].\n");
  printf(
      "-device=<device_id>:            Allows to pass GPU Device ID on which "
      "the tests will be run.\n");
  printf("-verbose:                       Prints highly verbose output.\n");
}

int main(int argc, char **argv) {
  bool reportAsBandwidth = false;
  bool print_launch_transfer_results = false;
  bool print_std_deviation = false;

  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "h")) {
    usage();
    printf("&&&& %s WAIVED\n", argv[0]);
    exit(EXIT_WAIVED);
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "reportAsBandwidth")) {
    reportAsBandwidth = true;
  }

  if (checkCmdLineFlag(argc, (const char **)argv,
                       "print-launch-transfer-results")) {
    print_launch_transfer_results = true;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "print-std-deviation")) {
    print_std_deviation = true;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "kernel-iterations")) {
    numKernelRuns =
        getCmdLineArgumentInt(argc, (const char **)argv, "kernel-iterations");
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "verbose")) {
    verboseResults = 1;
  }

  int device_id = findCudaDevice(argc, (const char **)argv);

  matrixMultiplyPerfRunner(reportAsBandwidth, print_launch_transfer_results,
                           print_std_deviation, device_id);

  printf(
      "\nNOTE: The CUDA Samples are not meant for performance measurements. "
      "Results may vary when GPU Boost is enabled.\n");
  exit(EXIT_SUCCESS);
}
