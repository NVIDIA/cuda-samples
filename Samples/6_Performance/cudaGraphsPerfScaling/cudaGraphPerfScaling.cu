/* Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This is a simple application showing the performance characteristics of cudaGraphs.
 */

#define USE_NVTX

#include <cstdio>
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>

typedef volatile int LatchType;

std::chrono::time_point<std::chrono::high_resolution_clock> getCpuTime()
{
    return std::chrono::high_resolution_clock::now();
}

template <typename T>
float getMicroSecondDuration(T start, T end)
{
    return std::chrono::duration_cast<std::chrono::nanoseconds>(end-start).count() *.001f;
}

float getAsyncMicroSecondDuration(hipEvent_t start, hipEvent_t end)
{
    float ms;
    hipEventElapsedTime(&ms, start, end);
    return ms*1000;
}

#ifdef USE_NVTX
#include <nvtx3/nvToolsExt.h>

class Tracer {
public:
    Tracer(const char* name) {
        nvtxRangePushA(name);
    }
    ~Tracer() {
        nvtxRangePop();
    }
};
#define RANGE(name) Tracer uniq_name_using_macros(name);
#define RANGE_PUSH(name) nvtxRangePushA(name)
#define RANGE_POP() nvtxRangePop();
#else
#define RANGE(name)
#endif

std::vector<hipStream_t> stream;
hipEvent_t event[1];
hipEvent_t timingEvent[2];

struct hostData {
    long long timeElapsed;
    bool timeoutDetected;
    long long timeElapsed2;
    bool timeoutDetected2;
    LatchType latch;
    LatchType latch2;
};

struct hostData *hostData;

__global__ void empty()
{
}

// Function to read the GPU nanosecond timer in a kernel
__device__ __forceinline__ unsigned long long   __globaltimer() { 
    unsigned long long globaltimer;   
    asm volatile ("mov.u64 %0, %globaltimer;"   : "=l"(globaltimer));   
    return globaltimer; 
}

__global__ void delay(long long ticks)
{
    long long endTime = clock64() + ticks;
    while (clock64() < endTime);
}

__global__ void waitWithTimeout(long long nanoseconds, bool* timeoutDetected, long long *timeElapsed, LatchType* latch)
{
    long long startTime = __globaltimer();
    long long endTime = startTime + nanoseconds;
    long long time = 0;
    do {
        time = __globaltimer();
    } while (time < endTime && (latch == NULL || *latch == 0));
    if (timeElapsed != NULL) {
        *timeElapsed = time - startTime;
    }
    if (timeoutDetected) {
        // report timeout if latch not detected
        *timeoutDetected = (latch == NULL || *latch == 0);
    }
}

__global__ void preUploadAnnotation()
{
}

__global__ void postUploadAnnotation()
{
}

hipGraph_t createParallelChain(int length, int width, bool singleEntry = false)
{
    RANGE_PUSH(__func__);
    RANGE("capture");
    hipGraph_t graph;
    hipStreamBeginCapture(stream[0], hipStreamCaptureModeGlobal);
    int streamIdx = 0; 
    if (singleEntry) {
       empty<<<1,1,0,stream[streamIdx]>>>();
    }

    hipEventRecord(event[0], stream[0]);
    for (int i = 1; i < width; i++) {
        hipStreamWaitEvent(stream[i], event[0]);
    }

    for (int i = 0; i < width; i++) {
        streamIdx = i;
        for (int j = 0; j < length; j++) {
            empty<<<1,1,0,stream[streamIdx]>>>();
        }
    }

    for (int i = 1; i < width; i++) {
        hipEventRecord(event[0], stream[i]);
        hipStreamWaitEvent(stream[0], event[0]);
    }

    hipStreamEndCapture(stream[0], &graph);
    return graph;
}

std::vector<const char*> metricName;
std::vector<float> metricValue;

int counter2 = 0;
void runDemo(hipGraph_t graph, int length, int width)
{
    hipGraphExec_t graphExec;
    {
        auto start = getCpuTime();
        hipGraphInstantiateWithFlags(&graphExec, graph, 0);
        auto end = getCpuTime();
        metricName.push_back("instantiation");
        metricValue.push_back(getMicroSecondDuration(start, end));
    }
    {
        RANGE("launch including upload");
        auto start = getCpuTime();
        hipGraphLaunch(graphExec, stream[0]);
        auto apiReturn = getCpuTime();
        hipStreamSynchronize(stream[0]);
        auto streamSync = getCpuTime();
        metricName.push_back("first_launch_api");
        metricValue.push_back(getMicroSecondDuration(start, apiReturn));
        metricName.push_back("first_launch_total");
        metricValue.push_back(getMicroSecondDuration(start, streamSync));
    }
    {
        RANGE("repeat lauch in empty stream");
        auto start = getCpuTime();
        hipGraphLaunch(graphExec, stream[0]);
        auto apiReturn = getCpuTime();
        hipStreamSynchronize(stream[0]);
        auto streamSync = getCpuTime();
        metricName.push_back("repeat_launch_api");
        metricValue.push_back(getMicroSecondDuration(start, apiReturn));
        metricName.push_back("repeat_launch_total");
        metricValue.push_back(getMicroSecondDuration(start, streamSync));
    }
    {
        // re-instantiating the exec to simulate first launch into a busy stream. 
        hipGraphExecDestroy(graphExec);
        hipGraphInstantiateWithFlags(&graphExec, graph, 0);

        long long maxTimeoutNanoSeconds = 4000 + 500*length*width;
        waitWithTimeout<<<1,1,0,stream[0]>>>(maxTimeoutNanoSeconds, &hostData->timeoutDetected, &hostData->timeElapsed, &hostData->latch);

        RANGE("launch including upload in busy stream");
        hipEventRecord(timingEvent[0], stream[0]);
        hipGraphLaunch(graphExec, stream[0]);
        hipEventRecord(timingEvent[1], stream[0]);

        hostData->latch = 1;
        hipStreamSynchronize(stream[0]);

        metricName.push_back("first_launch_device");
        metricValue.push_back(getAsyncMicroSecondDuration(timingEvent[0], timingEvent[1]));
        metricName.push_back("blockingKernelTimeoutDetected");
        metricValue.push_back(hostData->timeoutDetected);
        hostData->latch = 0;
        hostData->timeoutDetected = 0;
    }
    {
        RANGE("repeat lauch in busy stream");
        long long maxTimeoutNanoSeconds = 4000 + 500*length*width;
        waitWithTimeout<<<1,1,0,stream[0]>>>(maxTimeoutNanoSeconds, &hostData->timeoutDetected, &hostData->timeElapsed, &hostData->latch);
        hipEventRecord(timingEvent[0], stream[0]);
        hipGraphLaunch(graphExec, stream[0]);
        hipEventRecord(timingEvent[1], stream[0]);

        hostData->latch = 1;
        hipStreamSynchronize(stream[0]);

        metricName.push_back("repeat_launch_device");
        metricValue.push_back(getAsyncMicroSecondDuration(timingEvent[0], timingEvent[1]));
        metricName.push_back("blockingKernelTimeoutDetected");
        metricValue.push_back(hostData->timeoutDetected);
        hostData->latch = 0;
        hostData->timeoutDetected = 0;
    }
    {
        // re-instantiating the exec to provide upload with work to do.
        hipGraphExecDestroy(graphExec);
        hipGraphInstantiateWithFlags(&graphExec, graph, 0);
        long long maxTimeoutNanoSeconds = 4000 + 1000*length*width;
        waitWithTimeout<<<1,1,0,stream[0]>>>(maxTimeoutNanoSeconds, &hostData->timeoutDetected2, &hostData->timeElapsed2, &hostData->latch2);
        maxTimeoutNanoSeconds = 2000 + 500*length*width;
        waitWithTimeout<<<1,1,0,stream[1]>>>(maxTimeoutNanoSeconds, &hostData->timeoutDetected, &hostData->timeElapsed, &hostData->latch);

        RANGE("uploading a graph off of the critical path");
        preUploadAnnotation<<<1,1,0,stream[1]>>>();
        hipEventRecord(timingEvent[0], stream[0]);
        auto start = getCpuTime();
        hipGraphUpload(graphExec, stream[1]);
        auto apiReturn = getCpuTime();
        hipEventRecord(event[0],stream[1]);
        hipEventRecord(timingEvent[1], stream[0]);
        postUploadAnnotation<<<1,1,0,stream[1]>>>();

        hostData->latch = 1; // release the blocking kernel for the upload
        hipStreamWaitEvent(stream[0],event[0]);
        hipGraphLaunch(graphExec, stream[0]);
        hipEventSynchronize(event[0]); // upload done, similuate critical path being ready for the graph to run by the release of the second latch

        hostData->latch2 = 1; // release the work 
        hipStreamSynchronize(stream[0]);

        metricName.push_back("upload_api_time");
        metricValue.push_back(getMicroSecondDuration(start, apiReturn));
        metricName.push_back("updoad_device_time");
        metricValue.push_back(getAsyncMicroSecondDuration(timingEvent[0], timingEvent[1]));
        metricName.push_back("blockingKernelTimeoutDetected");
        metricValue.push_back(hostData->timeoutDetected);

        hostData->latch = 0;
        hostData->latch2 = 0;
        hostData->timeoutDetected = 0;
        hostData->timeoutDetected2 = 0;
    }
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    RANGE_POP();
}

void usage() {
    printf("programName [outputFmt] [numTrials] [length] [width] [pattern] [stride] [maxLength] \n");
    printf("\toutputFmt - program output, default=3 (see below)\n");
    printf("\tnumTrials (per length)\n");
    printf("\tstarting length of the topology\n");
    printf("\twidth - width of the graph topology\n");
    printf("\tpattern - Structure of graph, default=0 (see below)\n");
    printf("\tstride - how to grow the length between each set of trials \n");
    printf("\tmaxLength - maximum lenght to try \n");
    printf("\n");
    printf("outputFmt can be:\n");
    printf("\t0: this help message\n");
    printf("\t1: csv data headers\n");
    printf("\t2: per trial csv data\n");
    printf("\t3: csv data & headers\n");
    printf("\t4: csv data is printed and trials are averaged for each length\n");
    printf("\t5: csv data is printed and trials are averaged for each length and headers are printed\n");
    printf("\n");
    printf("Pattern can be:\n");
    printf("\t0: No interconnect between branches\n");
    printf("\t1: Adds an extra root node before the initial fork\n");
}

int main(int argc, char **argv)
{
    if(argc < 1) {
        usage();
        return 0;
    }

    int numTrials=1, length=20, width=1, outputFmt=3, pattern=0, stride = 1;
    if(argc > 1) outputFmt = atoi(argv[1]);
    if(argc > 2) numTrials = atoi(argv[2]);
    if(argc > 3) length= atoi(argv[3]);
    if(argc > 4) width= atoi(argv[4]);
    if(argc > 5) pattern = atoi(argv[5]);
    if(argc > 6) stride = atoi(argv[6]);
    int maxLength = length;
    if(argc > 7) maxLength = atoi(argv[7]);
    if (maxLength < length) {
        maxLength = length;
    }

    if((outputFmt & 4) && (outputFmt & 2)) {
        printf("printing average and all samples doesn't make sense\n");
    }

    if(length == 0 ||
       width == 0 ||
       outputFmt == 0 ||
       outputFmt > 5 ||
       pattern > 1)
    {
        usage();
        return 0;
    }

    bool singleEntry = (pattern == 1);

    hipGraph_t graph;

    hipFree(0);
    hipHostMalloc(&hostData, sizeof(*hostData), hipHostMallocDefault);
    int numStreams = width;
    if (numStreams == 1) numStreams = 2; // demo needs two streams even if capture only needs 1.
    stream.resize(numStreams);
    for (int i = 0; i < numStreams; i++)
    {
        hipStreamCreate(&stream[i]);
    }

    hipEventCreateWithFlags(&event[0], hipEventDisableTiming);
    hipEventCreateWithFlags(&timingEvent[0], 0);
    hipEventCreateWithFlags(&timingEvent[1], 0);

    {
        RANGE("warmup");
        for (int i = 0; i < width; i++)
        {
            empty<<<1,1,0,stream[i]>>>();
        }
        hipStreamSynchronize(stream[0]);

        auto start = getCpuTime();
        graph = createParallelChain(length, width, singleEntry);
        auto end = getCpuTime();
        metricValue.push_back(getMicroSecondDuration(start, end));
        metricName.push_back("capture");
        runDemo(graph, length, width);
    }

    if (outputFmt & 1) {
        printf("length, width, pattern, ");
        for (int i = 0; i < metricName.size(); i++) {
            printf("%s, ", metricName[i]);
        } 
        printf("\r\n");
    }

    if (!(outputFmt & 6)) {
        printf("skipping trials since no output is expected\n");
        return 1;
    }
    
    std::vector<double> metricTotal;
    metricTotal.resize(metricValue.size());

    while (length <= maxLength) {
        for (int i = 0; i < numTrials; i++) {
            metricName.clear();
            metricValue.clear();
            auto start = getCpuTime();
            graph = createParallelChain(length, width, singleEntry);
            auto end = getCpuTime();
            metricValue.push_back(getMicroSecondDuration(start, end));

            runDemo(graph, length, width);

            if (outputFmt & 2) {
                printf("%d, %d, %d, ",length, width, pattern);
                for (int i = 0; i < metricValue.size(); i++) {
                    printf("%0.3f, ", metricValue[i]);
                } 
                printf("\r\n");
            }
            if (outputFmt & 4) {
                for (int i = 0; i < metricTotal.size(); i++) {
                    metricTotal[i] += metricValue[i];
                } 
            }
        }

        if (outputFmt & 4) {
            printf("%d, %d, %d, ",length, width, pattern);
            for (int i = 0; i < metricTotal.size(); i++) {
                printf("%0.3f, ", metricTotal[i]/numTrials);
                metricTotal[i] = 0;
            } 
            printf("\r\n");
        }

        length += stride;
    }

    hipHostFree(hostData);

    printf("\n");
    printf("Test passed\n");
    return 0;
}
