#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This is a simple test program to measure the memcopy bandwidth of the GPU.
 * It can measure device to device copy bandwidth, host to device copy bandwidth
 * for pageable and pinned memory, and device to host copy bandwidth for
 * pageable and pinned memory.
 *
 * Usage:
 * ./bandwidthTest [option]...
 */

// CUDA runtime
#include <hip/hip_runtime.h>

// includes
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_runtime.h>

#include <cassert>
#include <iostream>
#include <memory>

static const char *sSDKsample = "CUDA Bandwidth Test";

// defines, project
#define MEMCOPY_ITERATIONS 100
#define DEFAULT_SIZE (32 * (1e6))      // 32 M
#define DEFAULT_INCREMENT (4 * (1e6))  // 4 M
#define CACHE_CLEAR_SIZE (16 * (1e6))  // 16 M

// shmoo mode defines
#define SHMOO_MEMSIZE_MAX (64 * (1e6))       // 64 M
#define SHMOO_MEMSIZE_START (1e3)            // 1 KB
#define SHMOO_INCREMENT_1KB (1e3)            // 1 KB
#define SHMOO_INCREMENT_2KB (2 * 1e3)        // 2 KB
#define SHMOO_INCREMENT_10KB (10 * (1e3))    // 10KB
#define SHMOO_INCREMENT_100KB (100 * (1e3))  // 100 KB
#define SHMOO_INCREMENT_1MB (1e6)            // 1 MB
#define SHMOO_INCREMENT_2MB (2 * 1e6)        // 2 MB
#define SHMOO_INCREMENT_4MB (4 * 1e6)        // 4 MB
#define SHMOO_LIMIT_20KB (20 * (1e3))        // 20 KB
#define SHMOO_LIMIT_50KB (50 * (1e3))        // 50 KB
#define SHMOO_LIMIT_100KB (100 * (1e3))      // 100 KB
#define SHMOO_LIMIT_1MB (1e6)                // 1 MB
#define SHMOO_LIMIT_16MB (16 * 1e6)          // 16 MB
#define SHMOO_LIMIT_32MB (32 * 1e6)          // 32 MB

// CPU cache flush
#define FLUSH_SIZE (256 * 1024 * 1024)
char *flush_buf;

// enums, project
enum testMode { QUICK_MODE, RANGE_MODE, SHMOO_MODE };
enum memcpyKind { DEVICE_TO_HOST, HOST_TO_DEVICE, DEVICE_TO_DEVICE };
enum printMode { USER_READABLE, CSV };
enum memoryMode { PINNED, PAGEABLE };

const char *sMemoryCopyKind[] = {"Device to Host", "Host to Device",
                                 "Device to Device", NULL};

const char *sMemoryMode[] = {"PINNED", "PAGEABLE", NULL};

// if true, use CPU based timing for everything
static bool bDontUseGPUTiming;

int *pArgc = NULL;
char **pArgv = NULL;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
int runTest(const int argc, const char **argv);
void testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
                   testMode mode, memcpyKind kind, printMode printmode,
                   memoryMode memMode, int startDevice, int endDevice, bool wc);
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc);
void testBandwidthRange(unsigned int start, unsigned int end,
                        unsigned int increment, memcpyKind kind,
                        printMode printmode, memoryMode memMode,
                        int startDevice, int endDevice, bool wc);
void testBandwidthShmoo(memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc);
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc);
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc);
float testDeviceToDeviceTransfer(unsigned int memSize);
void printResultsReadable(unsigned int *memSizes, double *bandwidths,
                          unsigned int count, memcpyKind kind,
                          memoryMode memMode, int iNumDevs, bool wc);
void printResultsCSV(unsigned int *memSizes, double *bandwidths,
                     unsigned int count, memcpyKind kind, memoryMode memMode,
                     int iNumDevs, bool wc);
void printHelp(void);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  pArgc = &argc;
  pArgv = argv;

  flush_buf = (char *)malloc(FLUSH_SIZE);

  // set logfile name and start logs
  printf("[%s] - Starting...\n", sSDKsample);

  int iRetVal = runTest(argc, (const char **)argv);

  if (iRetVal < 0) {
    checkCudaErrors(hipSetDevice(0));
  }

  // finish
  printf("%s\n", (iRetVal == 0) ? "Result = PASS" : "Result = FAIL");

  printf(
      "\nNOTE: The CUDA Samples are not meant for performance measurements. "
      "Results may vary when GPU Boost is enabled.\n");

  free(flush_buf);

  exit((iRetVal == 0) ? EXIT_SUCCESS : EXIT_FAILURE);
}

///////////////////////////////////////////////////////////////////////////////
// Parse args, run the appropriate tests
///////////////////////////////////////////////////////////////////////////////
int runTest(const int argc, const char **argv) {
  int start = DEFAULT_SIZE;
  int end = DEFAULT_SIZE;
  int startDevice = 0;
  int endDevice = 0;
  int increment = DEFAULT_INCREMENT;
  testMode mode = QUICK_MODE;
  bool htod = false;
  bool dtoh = false;
  bool dtod = false;
  bool wc = false;
  char *modeStr;
  char *device = NULL;
  printMode printmode = USER_READABLE;
  char *memModeStr = NULL;
  memoryMode memMode = PINNED;

  // process command line args
  if (checkCmdLineFlag(argc, argv, "help")) {
    printHelp();
    return 0;
  }

  if (checkCmdLineFlag(argc, argv, "csv")) {
    printmode = CSV;
  }

  if (getCmdLineArgumentString(argc, argv, "memory", &memModeStr)) {
    if (strcmp(memModeStr, "pageable") == 0) {
      memMode = PAGEABLE;
    } else if (strcmp(memModeStr, "pinned") == 0) {
      memMode = PINNED;
    } else {
      printf("Invalid memory mode - valid modes are pageable or pinned\n");
      printf("See --help for more information\n");
      return -1000;
    }
  } else {
    // default - pinned memory
    memMode = PINNED;
  }

  if (getCmdLineArgumentString(argc, argv, "device", &device)) {
    int deviceCount;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if (error_id != hipSuccess) {
      printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id,
             hipGetErrorString(error_id));
      exit(EXIT_FAILURE);
    }

    if (deviceCount == 0) {
      printf("!!!!!No devices found!!!!!\n");
      return -2000;
    }

    if (strcmp(device, "all") == 0) {
      printf(
          "\n!!!!!Cumulative Bandwidth to be computed from all the devices "
          "!!!!!!\n\n");
      startDevice = 0;
      endDevice = deviceCount - 1;
    } else {
      startDevice = endDevice = atoi(device);

      if (startDevice >= deviceCount || startDevice < 0) {
        printf(
            "\n!!!!!Invalid GPU number %d given hence default gpu %d will be "
            "used !!!!!\n",
            startDevice, 0);
        startDevice = endDevice = 0;
      }
    }
  }

  printf("Running on...\n\n");

  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++) {
    hipDeviceProp_t deviceProp;
    hipError_t error_id = hipGetDeviceProperties(&deviceProp, currentDevice);

    if (error_id == hipSuccess) {
      printf(" Device %d: %s\n", currentDevice, deviceProp.name);

      if (deviceProp.computeMode == hipComputeModeProhibited) {
        fprintf(stderr,
                "Error: device is running in <Compute Mode Prohibited>, no "
                "threads can use ::hipSetDevice().\n");
        checkCudaErrors(hipSetDevice(currentDevice));

        exit(EXIT_FAILURE);
      }
    } else {
      printf("hipGetDeviceProperties returned %d\n-> %s\n", (int)error_id,
             hipGetErrorString(error_id));
      checkCudaErrors(hipSetDevice(currentDevice));

      exit(EXIT_FAILURE);
    }
  }

  if (getCmdLineArgumentString(argc, argv, "mode", &modeStr)) {
    // figure out the mode
    if (strcmp(modeStr, "quick") == 0) {
      printf(" Quick Mode\n\n");
      mode = QUICK_MODE;
    } else if (strcmp(modeStr, "shmoo") == 0) {
      printf(" Shmoo Mode\n\n");
      mode = SHMOO_MODE;
    } else if (strcmp(modeStr, "range") == 0) {
      printf(" Range Mode\n\n");
      mode = RANGE_MODE;
    } else {
      printf("Invalid mode - valid modes are quick, range, or shmoo\n");
      printf("See --help for more information\n");
      return -3000;
    }
  } else {
    // default mode - quick
    printf(" Quick Mode\n\n");
    mode = QUICK_MODE;
  }

  if (checkCmdLineFlag(argc, argv, "htod")) {
    htod = true;
  }

  if (checkCmdLineFlag(argc, argv, "dtoh")) {
    dtoh = true;
  }

  if (checkCmdLineFlag(argc, argv, "dtod")) {
    dtod = true;
  }

#if CUDART_VERSION >= 2020

  if (checkCmdLineFlag(argc, argv, "wc")) {
    wc = true;
  }

#endif

  if (checkCmdLineFlag(argc, argv, "cputiming")) {
    bDontUseGPUTiming = true;
  }

  if (!htod && !dtoh && !dtod) {
    // default:  All
    htod = true;
    dtoh = true;
    dtod = true;
  }

  if (RANGE_MODE == mode) {
    if (checkCmdLineFlag(argc, (const char **)argv, "start")) {
      start = getCmdLineArgumentInt(argc, argv, "start");

      if (start <= 0) {
        printf("Illegal argument - start must be greater than zero\n");
        return -4000;
      }
    } else {
      printf("Must specify a starting size in range mode\n");
      printf("See --help for more information\n");
      return -5000;
    }

    if (checkCmdLineFlag(argc, (const char **)argv, "end")) {
      end = getCmdLineArgumentInt(argc, argv, "end");

      if (end <= 0) {
        printf("Illegal argument - end must be greater than zero\n");
        return -6000;
      }

      if (start > end) {
        printf("Illegal argument - start is greater than end\n");
        return -7000;
      }
    } else {
      printf("Must specify an end size in range mode.\n");
      printf("See --help for more information\n");
      return -8000;
    }

    if (checkCmdLineFlag(argc, argv, "increment")) {
      increment = getCmdLineArgumentInt(argc, argv, "increment");

      if (increment <= 0) {
        printf("Illegal argument - increment must be greater than zero\n");
        return -9000;
      }
    } else {
      printf("Must specify an increment in user mode\n");
      printf("See --help for more information\n");
      return -10000;
    }
  }

  if (htod) {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, HOST_TO_DEVICE, printmode,
                  memMode, startDevice, endDevice, wc);
  }

  if (dtoh) {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, DEVICE_TO_HOST, printmode,
                  memMode, startDevice, endDevice, wc);
  }

  if (dtod) {
    testBandwidth((unsigned int)start, (unsigned int)end,
                  (unsigned int)increment, mode, DEVICE_TO_DEVICE, printmode,
                  memMode, startDevice, endDevice, wc);
  }

  // Ensure that we reset all CUDA Devices in question
  for (int nDevice = startDevice; nDevice <= endDevice; nDevice++) {
    hipSetDevice(nDevice);
  }

  return 0;
}

///////////////////////////////////////////////////////////////////////////////
//  Run a bandwidth test
///////////////////////////////////////////////////////////////////////////////
void testBandwidth(unsigned int start, unsigned int end, unsigned int increment,
                   testMode mode, memcpyKind kind, printMode printmode,
                   memoryMode memMode, int startDevice, int endDevice,
                   bool wc) {
  switch (mode) {
    case QUICK_MODE:
      testBandwidthQuick(DEFAULT_SIZE, kind, printmode, memMode, startDevice,
                         endDevice, wc);
      break;

    case RANGE_MODE:
      testBandwidthRange(start, end, increment, kind, printmode, memMode,
                         startDevice, endDevice, wc);
      break;

    case SHMOO_MODE:
      testBandwidthShmoo(kind, printmode, memMode, startDevice, endDevice, wc);
      break;

    default:
      break;
  }
}

//////////////////////////////////////////////////////////////////////
//  Run a quick mode bandwidth test
//////////////////////////////////////////////////////////////////////
void testBandwidthQuick(unsigned int size, memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc) {
  testBandwidthRange(size, size, DEFAULT_INCREMENT, kind, printmode, memMode,
                     startDevice, endDevice, wc);
}

///////////////////////////////////////////////////////////////////////
//  Run a range mode bandwidth test
//////////////////////////////////////////////////////////////////////
void testBandwidthRange(unsigned int start, unsigned int end,
                        unsigned int increment, memcpyKind kind,
                        printMode printmode, memoryMode memMode,
                        int startDevice, int endDevice, bool wc) {
  // count the number of copies we're going to run
  unsigned int count = 1 + ((end - start) / increment);

  unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
  double *bandwidths = (double *)malloc(count * sizeof(double));

  // Before calculating the cumulative bandwidth, initialize bandwidths array to
  // NULL
  for (unsigned int i = 0; i < count; i++) {
    bandwidths[i] = 0.0;
  }

  // Use the device asked by the user
  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++) {
    hipSetDevice(currentDevice);

    // run each of the copies
    for (unsigned int i = 0; i < count; i++) {
      memSizes[i] = start + i * increment;

      switch (kind) {
        case DEVICE_TO_HOST:
          bandwidths[i] += testDeviceToHostTransfer(memSizes[i], memMode, wc);
          break;

        case HOST_TO_DEVICE:
          bandwidths[i] += testHostToDeviceTransfer(memSizes[i], memMode, wc);
          break;

        case DEVICE_TO_DEVICE:
          bandwidths[i] += testDeviceToDeviceTransfer(memSizes[i]);
          break;
      }
    }
  }  // Complete the bandwidth computation on all the devices

  // print results
  if (printmode == CSV) {
    printResultsCSV(memSizes, bandwidths, count, kind, memMode,
                    (1 + endDevice - startDevice), wc);
  } else {
    printResultsReadable(memSizes, bandwidths, count, kind, memMode,
                         (1 + endDevice - startDevice), wc);
  }

  // clean up
  free(memSizes);
  free(bandwidths);
}

//////////////////////////////////////////////////////////////////////////////
// Intense shmoo mode - covers a large range of values with varying increments
//////////////////////////////////////////////////////////////////////////////
void testBandwidthShmoo(memcpyKind kind, printMode printmode,
                        memoryMode memMode, int startDevice, int endDevice,
                        bool wc) {
  // count the number of copies to make
  unsigned int count =
      1 + (SHMOO_LIMIT_20KB / SHMOO_INCREMENT_1KB) +
      ((SHMOO_LIMIT_50KB - SHMOO_LIMIT_20KB) / SHMOO_INCREMENT_2KB) +
      ((SHMOO_LIMIT_100KB - SHMOO_LIMIT_50KB) / SHMOO_INCREMENT_10KB) +
      ((SHMOO_LIMIT_1MB - SHMOO_LIMIT_100KB) / SHMOO_INCREMENT_100KB) +
      ((SHMOO_LIMIT_16MB - SHMOO_LIMIT_1MB) / SHMOO_INCREMENT_1MB) +
      ((SHMOO_LIMIT_32MB - SHMOO_LIMIT_16MB) / SHMOO_INCREMENT_2MB) +
      ((SHMOO_MEMSIZE_MAX - SHMOO_LIMIT_32MB) / SHMOO_INCREMENT_4MB);

  unsigned int *memSizes = (unsigned int *)malloc(count * sizeof(unsigned int));
  double *bandwidths = (double *)malloc(count * sizeof(double));

  // Before calculating the cumulative bandwidth, initialize bandwidths array to
  // NULL
  for (unsigned int i = 0; i < count; i++) {
    bandwidths[i] = 0.0;
  }

  // Use the device asked by the user
  for (int currentDevice = startDevice; currentDevice <= endDevice;
       currentDevice++) {
    hipSetDevice(currentDevice);
    // Run the shmoo
    int iteration = 0;
    unsigned int memSize = 0;

    while (memSize <= SHMOO_MEMSIZE_MAX) {
      if (memSize < SHMOO_LIMIT_20KB) {
        memSize += SHMOO_INCREMENT_1KB;
      } else if (memSize < SHMOO_LIMIT_50KB) {
        memSize += SHMOO_INCREMENT_2KB;
      } else if (memSize < SHMOO_LIMIT_100KB) {
        memSize += SHMOO_INCREMENT_10KB;
      } else if (memSize < SHMOO_LIMIT_1MB) {
        memSize += SHMOO_INCREMENT_100KB;
      } else if (memSize < SHMOO_LIMIT_16MB) {
        memSize += SHMOO_INCREMENT_1MB;
      } else if (memSize < SHMOO_LIMIT_32MB) {
        memSize += SHMOO_INCREMENT_2MB;
      } else {
        memSize += SHMOO_INCREMENT_4MB;
      }

      memSizes[iteration] = memSize;

      switch (kind) {
        case DEVICE_TO_HOST:
          bandwidths[iteration] +=
              testDeviceToHostTransfer(memSizes[iteration], memMode, wc);
          break;

        case HOST_TO_DEVICE:
          bandwidths[iteration] +=
              testHostToDeviceTransfer(memSizes[iteration], memMode, wc);
          break;

        case DEVICE_TO_DEVICE:
          bandwidths[iteration] +=
              testDeviceToDeviceTransfer(memSizes[iteration]);
          break;
      }

      iteration++;
      printf(".");
      fflush(0);
    }
  }  // Complete the bandwidth computation on all the devices

  // print results
  printf("\n");

  if (CSV == printmode) {
    printResultsCSV(memSizes, bandwidths, count, kind, memMode,
                    (1 + endDevice - startDevice), wc);
  } else {
    printResultsReadable(memSizes, bandwidths, count, kind, memMode,
                         (1 + endDevice - startDevice), wc);
  }

  // clean up
  free(memSizes);
  free(bandwidths);
}

///////////////////////////////////////////////////////////////////////////////
//  test the bandwidth of a device to host memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float testDeviceToHostTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc) {
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  unsigned char *h_idata = NULL;
  unsigned char *h_odata = NULL;
  hipEvent_t start, stop;

  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  if (PINNED == memMode) {
  // pinned memory mode - use special function to get OS-pinned memory
#if CUDART_VERSION >= 2020
    checkCudaErrors(hipHostAlloc((void **)&h_idata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
    checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
#else
    checkCudaErrors(hipHostMalloc((void **)&h_idata, memSize));
    checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
  } else {
    // pageable memory mode - use malloc
    h_idata = (unsigned char *)malloc(memSize);
    h_odata = (unsigned char *)malloc(memSize);

    if (h_idata == 0 || h_odata == 0) {
      fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
      exit(EXIT_FAILURE);
    }
  }

  // initialize the memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++) {
    h_idata[i] = (unsigned char)(i & 0xff);
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));

  // initialize the device memory
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice));

  // copy data from GPU to Host
  if (PINNED == memMode) {
    if (bDontUseGPUTiming) sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
      checkCudaErrors(hipMemcpyAsync(h_odata, d_idata, memSize,
                                      hipMemcpyDeviceToHost, 0));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    if (bDontUseGPUTiming) {
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }
  } else {
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
      sdkStartTimer(&timer);
      checkCudaErrors(
          hipMemcpy(h_odata, d_idata, memSize, hipMemcpyDeviceToHost));
      sdkStopTimer(&timer);
      elapsedTimeInMs += sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      memset(flush_buf, i, FLUSH_SIZE);
    }
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;
  // clean up memory
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  sdkDeleteTimer(&timer);

  if (PINNED == memMode) {
    checkCudaErrors(hipHostFree(h_idata));
    checkCudaErrors(hipHostFree(h_odata));
  } else {
    free(h_idata);
    free(h_odata);
  }

  checkCudaErrors(hipFree(d_idata));

  return bandwidthInGBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a host to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float testHostToDeviceTransfer(unsigned int memSize, memoryMode memMode,
                               bool wc) {
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  hipEvent_t start, stop;
  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  unsigned char *h_odata = NULL;

  if (PINNED == memMode) {
#if CUDART_VERSION >= 2020
    // pinned memory mode - use special function to get OS-pinned memory
    checkCudaErrors(hipHostAlloc((void **)&h_odata, memSize,
                                  (wc) ? hipHostMallocWriteCombined : 0));
#else
    // pinned memory mode - use special function to get OS-pinned memory
    checkCudaErrors(hipHostMalloc((void **)&h_odata, memSize));
#endif
  } else {
    // pageable memory mode - use malloc
    h_odata = (unsigned char *)malloc(memSize);

    if (h_odata == 0) {
      fprintf(stderr, "Not enough memory available on host to run test!\n");
      exit(EXIT_FAILURE);
    }
  }

  unsigned char *h_cacheClear1 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);
  unsigned char *h_cacheClear2 = (unsigned char *)malloc(CACHE_CLEAR_SIZE);

  if (h_cacheClear1 == 0 || h_cacheClear2 == 0) {
    fprintf(stderr, "Not enough memory available on host to run test!\n");
    exit(EXIT_FAILURE);
  }

  // initialize the memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++) {
    h_odata[i] = (unsigned char)(i & 0xff);
  }

  for (unsigned int i = 0; i < CACHE_CLEAR_SIZE / sizeof(unsigned char); i++) {
    h_cacheClear1[i] = (unsigned char)(i & 0xff);
    h_cacheClear2[i] = (unsigned char)(0xff - (i & 0xff));
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));

  // copy host memory to device memory
  if (PINNED == memMode) {
    if (bDontUseGPUTiming) sdkStartTimer(&timer);
    checkCudaErrors(hipEventRecord(start, 0));
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
      checkCudaErrors(hipMemcpyAsync(d_idata, h_odata, memSize,
                                      hipMemcpyHostToDevice, 0));
    }
    checkCudaErrors(hipEventRecord(stop, 0));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));
    if (bDontUseGPUTiming) {
      sdkStopTimer(&timer);
      elapsedTimeInMs = sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
    }
  } else {
    elapsedTimeInMs = 0;
    for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
      sdkStartTimer(&timer);
      checkCudaErrors(
          hipMemcpy(d_idata, h_odata, memSize, hipMemcpyHostToDevice));
      sdkStopTimer(&timer);
      elapsedTimeInMs += sdkGetTimerValue(&timer);
      sdkResetTimer(&timer);
      memset(flush_buf, i, FLUSH_SIZE);
    }
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;
  // clean up memory
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  sdkDeleteTimer(&timer);

  if (PINNED == memMode) {
    checkCudaErrors(hipHostFree(h_odata));
  } else {
    free(h_odata);
  }

  free(h_cacheClear1);
  free(h_cacheClear2);
  checkCudaErrors(hipFree(d_idata));

  return bandwidthInGBs;
}

///////////////////////////////////////////////////////////////////////////////
//! test the bandwidth of a device to device memcopy of a specific size
///////////////////////////////////////////////////////////////////////////////
float testDeviceToDeviceTransfer(unsigned int memSize) {
  StopWatchInterface *timer = NULL;
  float elapsedTimeInMs = 0.0f;
  float bandwidthInGBs = 0.0f;
  hipEvent_t start, stop;

  sdkCreateTimer(&timer);
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));

  // allocate host memory
  unsigned char *h_idata = (unsigned char *)malloc(memSize);

  if (h_idata == 0) {
    fprintf(stderr, "Not enough memory avaialable on host to run test!\n");
    exit(EXIT_FAILURE);
  }

  // initialize the host memory
  for (unsigned int i = 0; i < memSize / sizeof(unsigned char); i++) {
    h_idata[i] = (unsigned char)(i & 0xff);
  }

  // allocate device memory
  unsigned char *d_idata;
  checkCudaErrors(hipMalloc((void **)&d_idata, memSize));
  unsigned char *d_odata;
  checkCudaErrors(hipMalloc((void **)&d_odata, memSize));

  // initialize memory
  checkCudaErrors(
      hipMemcpy(d_idata, h_idata, memSize, hipMemcpyHostToDevice));

  // run the memcopy
  sdkStartTimer(&timer);
  checkCudaErrors(hipEventRecord(start, 0));

  for (unsigned int i = 0; i < MEMCOPY_ITERATIONS; i++) {
    checkCudaErrors(
        hipMemcpy(d_odata, d_idata, memSize, hipMemcpyDeviceToDevice));
  }

  checkCudaErrors(hipEventRecord(stop, 0));

  // Since device to device memory copies are non-blocking,
  // hipDeviceSynchronize() is required in order to get
  // proper timing.
  checkCudaErrors(hipDeviceSynchronize());

  // get the total elapsed time in ms
  sdkStopTimer(&timer);
  checkCudaErrors(hipEventElapsedTime(&elapsedTimeInMs, start, stop));

  if (bDontUseGPUTiming) {
    elapsedTimeInMs = sdkGetTimerValue(&timer);
  }

  // calculate bandwidth in GB/s
  double time_s = elapsedTimeInMs / 1e3;
  bandwidthInGBs = (2.0f * memSize * (float)MEMCOPY_ITERATIONS) / (double)1e9;
  bandwidthInGBs = bandwidthInGBs / time_s;

  // clean up memory
  sdkDeleteTimer(&timer);
  free(h_idata);
  checkCudaErrors(hipEventDestroy(stop));
  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(d_odata));

  return bandwidthInGBs;
}

/////////////////////////////////////////////////////////
// print results in an easily read format
////////////////////////////////////////////////////////
void printResultsReadable(unsigned int *memSizes, double *bandwidths,
                          unsigned int count, memcpyKind kind,
                          memoryMode memMode, int iNumDevs, bool wc) {
  printf(" %s Bandwidth, %i Device(s)\n", sMemoryCopyKind[kind], iNumDevs);
  printf(" %s Memory Transfers\n", sMemoryMode[memMode]);

  if (wc) {
    printf(" Write-Combined Memory Writes are Enabled");
  }

  printf("   Transfer Size (Bytes)\tBandwidth(GB/s)\n");
  unsigned int i;

  for (i = 0; i < (count - 1); i++) {
    printf("   %u\t\t\t%s%.1f\n", memSizes[i],
           (memSizes[i] < 10000) ? "\t" : "", bandwidths[i]);
  }

  printf("   %u\t\t\t%s%.1f\n\n", memSizes[i],
         (memSizes[i] < 10000) ? "\t" : "", bandwidths[i]);
}

///////////////////////////////////////////////////////////////////////////
// print results in a database format
///////////////////////////////////////////////////////////////////////////
void printResultsCSV(unsigned int *memSizes, double *bandwidths,
                     unsigned int count, memcpyKind kind, memoryMode memMode,
                     int iNumDevs, bool wc) {
  std::string sConfig;

  // log config information
  if (kind == DEVICE_TO_DEVICE) {
    sConfig += "D2D";
  } else {
    if (kind == DEVICE_TO_HOST) {
      sConfig += "D2H";
    } else if (kind == HOST_TO_DEVICE) {
      sConfig += "H2D";
    }

    if (memMode == PAGEABLE) {
      sConfig += "-Paged";
    } else if (memMode == PINNED) {
      sConfig += "-Pinned";

      if (wc) {
        sConfig += "-WriteCombined";
      }
    }
  }

  unsigned int i;
  double dSeconds = 0.0;

  for (i = 0; i < count; i++) {
    dSeconds = (double)memSizes[i] / (bandwidths[i] * (double)(1e9));
    printf(
        "bandwidthTest-%s, Bandwidth = %.1f GB/s, Time = %.5f s, Size = %u "
        "bytes, NumDevsUsed = %d\n",
        sConfig.c_str(), bandwidths[i], dSeconds, memSizes[i], iNumDevs);
  }
}

///////////////////////////////////////////////////////////////////////////
// Print help screen
///////////////////////////////////////////////////////////////////////////
void printHelp(void) {
  printf("Usage:  bandwidthTest [OPTION]...\n");
  printf(
      "Test the bandwidth for device to host, host to device, and device to "
      "device transfers\n");
  printf("\n");
  printf(
      "Example:  measure the bandwidth of device to host pinned memory copies "
      "in the range 1024 Bytes to 102400 Bytes in 1024 Byte increments\n");
  printf(
      "./bandwidthTest --memory=pinned --mode=range --start=1024 --end=102400 "
      "--increment=1024 --dtoh\n");

  printf("\n");
  printf("Options:\n");
  printf("--help\tDisplay this help menu\n");
  printf("--csv\tPrint results as a CSV\n");
  printf("--device=[deviceno]\tSpecify the device device to be used\n");
  printf("  all - compute cumulative bandwidth on all the devices\n");
  printf("  0,1,2,...,n - Specify any particular device to be used\n");
  printf("--memory=[MEMMODE]\tSpecify which memory mode to use\n");
  printf("  pageable - pageable memory\n");
  printf("  pinned   - non-pageable system memory\n");
  printf("--mode=[MODE]\tSpecify the mode to use\n");
  printf("  quick - performs a quick measurement\n");
  printf("  range - measures a user-specified range of values\n");
  printf("  shmoo - performs an intense shmoo of a large range of values\n");

  printf("--htod\tMeasure host to device transfers\n");
  printf("--dtoh\tMeasure device to host transfers\n");
  printf("--dtod\tMeasure device to device transfers\n");
#if CUDART_VERSION >= 2020
  printf("--wc\tAllocate pinned memory as write-combined\n");
#endif
  printf("--cputiming\tForce CPU-based timing always\n");

  printf("Range mode options\n");
  printf("--start=[SIZE]\tStarting transfer size in bytes\n");
  printf("--end=[SIZE]\tEnding transfer size in bytes\n");
  printf("--increment=[SIZE]\tIncrement size in bytes\n");
}
