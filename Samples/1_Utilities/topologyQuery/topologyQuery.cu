/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample demonstrates how to use query information on the current system
 * topology using a SDK 8.0 API.
 */

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>  // helper for shared that are common to CUDA Samples

int main(int argc, char **argv) {
  int deviceCount = 0;
  checkCudaErrors(hipGetDeviceCount(&deviceCount));

  // Enumerates Device <-> Device links
  for (int device1 = 0; device1 < deviceCount; device1++) {
    for (int device2 = 0; device2 < deviceCount; device2++) {
      if (device1 == device2) continue;

      int perfRank = 0;
      int atomicSupported = 0;
      int accessSupported = 0;

      checkCudaErrors(hipDeviceGetP2PAttribute(
          &accessSupported, hipDevP2PAttrAccessSupported, device1, device2));
      checkCudaErrors(hipDeviceGetP2PAttribute(
          &perfRank, hipDevP2PAttrPerformanceRank, device1, device2));
      checkCudaErrors(hipDeviceGetP2PAttribute(
          &atomicSupported, hipDevP2PAttrNativeAtomicSupported, device1,
          device2));

      if (accessSupported) {
        std::cout << "GPU" << device1 << " <-> GPU" << device2 << ":"
                  << std::endl;
        std::cout << "  * Atomic Supported: "
                  << (atomicSupported ? "yes" : "no") << std::endl;
        std::cout << "  * Perf Rank: " << perfRank << std::endl;
      }
    }
  }

  // Enumerates Device <-> Host links
  for (int device = 0; device < deviceCount; device++) {
    int atomicSupported = 0;
    checkCudaErrors(hipDeviceGetAttribute(
        &atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, device));
    std::cout << "GPU" << device << " <-> CPU:" << std::endl;
    std::cout << "  * Atomic Supported: " << (atomicSupported ? "yes" : "no")
              << std::endl;
  }

  return 0;
}
