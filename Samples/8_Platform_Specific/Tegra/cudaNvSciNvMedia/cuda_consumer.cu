#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <iostream>

#include <hip/hip_runtime.h>
#include "cuda_consumer.h"
#include <helper_image.h>
#include "nvmedia_image_nvscibuf.h"
#include "nvmedia_utils/cmdline.h"

// Enable this to 1 if require cuda processed output to ppm file.
#define WRITE_OUTPUT_IMAGE 0

#define checkNvSciErrors(call)                              \
  do {                                                      \
    NvSciError _status = call;                              \
    if (NvSciError_Success != _status) {                    \
      printf(                                               \
          "NVSCI call in file '%s' in line %i returned"     \
          " %d, expected %d\n",                             \
          __FILE__, __LINE__, _status, NvSciError_Success); \
      fflush(stdout);                                       \
      exit(EXIT_FAILURE);                                   \
    }                                                       \
  } while (0)

__global__ static void yuvToGrayscale(hipSurfaceObject_t surfaceObject,
                                      unsigned int *dstImage,
                                      int32_t imageWidth, int32_t imageHeight) {
  size_t x = blockIdx.x * blockDim.x + threadIdx.x;
  size_t y = blockIdx.y * blockDim.y + threadIdx.y;

  uchar4 *dstImageUchar4 = (uchar4 *)dstImage;
  for (; x < imageWidth && y < imageHeight;
       x += gridDim.x * blockDim.x, y += gridDim.y * blockDim.y) {
    int colInBytes = x * sizeof(unsigned char);
    unsigned char luma =
        surf2Dread<unsigned char>(surfaceObject, colInBytes, y);
    uchar4 grayscalePix = make_uchar4(luma, luma, luma, 0);

    dstImageUchar4[y * imageWidth + x] = grayscalePix;
  }
}

static void cudaImportNvSciSync(hipExternalSemaphore_t &extSem,
                                NvSciSyncObj &syncObj) {
  hipExternalSemaphoreHandleDesc extSemDesc;
  memset(&extSemDesc, 0, sizeof(extSemDesc));
  extSemDesc.type = cudaExternalSemaphoreHandleTypeNvSciSync;
  extSemDesc.handle.nvSciSyncObj = (void *)syncObj;

  checkCudaErrors(hipImportExternalSemaphore(&extSem, &extSemDesc));
}

static void waitExternalSemaphore(hipExternalSemaphore_t &waitSem,
                                  NvSciSyncFence *fence, hipStream_t stream) {
  hipExternalSemaphoreWaitParams waitParams;
  memset(&waitParams, 0, sizeof(waitParams));
  // For cross-process signaler-waiter applications need to use NvSciIpc
  // and NvSciSync[Export|Import] utilities to share the NvSciSyncFence
  // across process. This step is optional in single-process.
  waitParams.params.nvSciSync.fence = (void *)fence;
  waitParams.flags = 0;

  checkCudaErrors(
      hipWaitExternalSemaphoresAsync(&waitSem, &waitParams, 1, stream));
}

static void signalExternalSemaphore(hipExternalSemaphore_t &signalSem,
                                    NvSciSyncFence *fence,
                                    hipStream_t stream) {
  hipExternalSemaphoreSignalParams signalParams;
  memset(&signalParams, 0, sizeof(signalParams));
  // For cross-process signaler-waiter applications need to use NvSciIpc
  // and NvSciSync[Export|Import] utilities to share the NvSciSyncFence
  // across process. This step is optional in single-process.
  signalParams.params.nvSciSync.fence = (void *)fence;
  signalParams.flags = 0;

  checkCudaErrors(
      hipSignalExternalSemaphoresAsync(&signalSem, &signalParams, 1, stream));
}

static void yuvToGrayscaleCudaKernel(cudaExternalResInterop &cudaExtResObj,
                                     int32_t imageWidth, int32_t imageHeight) {
#if WRITE_OUTPUT_IMAGE
  unsigned int *h_dstImage;
  checkCudaErrors(hipHostMalloc(
      &h_dstImage, sizeof(unsigned int) * imageHeight * imageWidth));
#endif
  dim3 block(16, 16, 1);
  dim3 grid((imageWidth / block.x) + 1, (imageHeight / block.y) + 1, 1);

  yuvToGrayscale<<<grid, block, 0, cudaExtResObj.stream>>>(
      cudaExtResObj.cudaSurfaceNvmediaBuf[0], cudaExtResObj.d_outputImage,
      imageWidth, imageHeight);

#if WRITE_OUTPUT_IMAGE
  checkCudaErrors(
      hipMemcpyAsync(h_dstImage, cudaExtResObj.d_outputImage,
                      sizeof(unsigned int) * imageHeight * imageWidth,
                      hipMemcpyDeviceToHost, cudaExtResObj.stream));
  checkCudaErrors(hipStreamSynchronize(cudaExtResObj.stream));
  char outputFilename[1024];
  std::string image_filename = "Grayscale";
  strcpy(outputFilename, image_filename.c_str());
  strcpy(outputFilename + image_filename.length(), "_nvsci_out.ppm");
  sdkSavePPM4ub(outputFilename, (unsigned char *)h_dstImage, imageWidth,
                imageHeight);
  printf("Wrote '%s'\n", outputFilename);
  checkCudaErrors(hipHostFree(h_dstImage));
#endif
}

static void cudaImportNvSciImage(cudaExternalResInterop &cudaExtResObj,
                                 NvSciBufObj &inputBufObj) {
  NvSciBufModule module = NULL;
  NvSciBufAttrList attrlist = NULL;
  NvSciBufAttrKeyValuePair pairArrayOut[10];

  checkNvSciErrors(NvSciBufModuleOpen(&module));
  checkNvSciErrors(NvSciBufAttrListCreate(module, &attrlist));
  checkNvSciErrors(NvSciBufObjGetAttrList(inputBufObj, &attrlist));

  memset(pairArrayOut, 0, sizeof(NvSciBufAttrKeyValuePair) * 10);

  int numAttrs = 0;
  pairArrayOut[numAttrs++].key = NvSciBufImageAttrKey_Size;
  pairArrayOut[numAttrs++].key = NvSciBufImageAttrKey_PlaneChannelCount;
  pairArrayOut[numAttrs++].key = NvSciBufImageAttrKey_PlaneCount;
  pairArrayOut[numAttrs++].key = NvSciBufImageAttrKey_PlaneWidth;
  pairArrayOut[numAttrs++].key = NvSciBufImageAttrKey_PlaneHeight;
  pairArrayOut[numAttrs++].key = NvSciBufImageAttrKey_Layout;
  pairArrayOut[numAttrs++].key = NvSciBufImageAttrKey_PlaneBitsPerPixel;
  pairArrayOut[numAttrs++].key = NvSciBufImageAttrKey_PlaneOffset;

  checkNvSciErrors(NvSciBufAttrListGetAttrs(attrlist, pairArrayOut, numAttrs));

  uint64_t size = *(uint64_t *)pairArrayOut[0].value;
  uint8_t channelCount = *(uint8_t *)pairArrayOut[1].value;
  cudaExtResObj.planeCount = *(int32_t *)pairArrayOut[2].value;
  cudaExtResObj.imageWidth =
      (int32_t *)malloc(sizeof(int32_t) * cudaExtResObj.planeCount);
  cudaExtResObj.imageHeight =
      (int32_t *)malloc(sizeof(int32_t) * cudaExtResObj.planeCount);
  cudaExtResObj.planeOffset =
      (uint64_t *)malloc(sizeof(uint64_t) * cudaExtResObj.planeCount);

  memcpy(cudaExtResObj.imageWidth, (int32_t *)pairArrayOut[3].value,
         cudaExtResObj.planeCount * sizeof(int32_t));
  memcpy(cudaExtResObj.imageHeight, (int32_t *)pairArrayOut[4].value,
         cudaExtResObj.planeCount * sizeof(int32_t));
  memcpy(cudaExtResObj.planeOffset, (uint64_t *)pairArrayOut[7].value,
         cudaExtResObj.planeCount * sizeof(uint64_t));

  NvSciBufAttrValImageLayoutType layout =
      *(NvSciBufAttrValImageLayoutType *)pairArrayOut[5].value;
  uint32_t bitsPerPixel = *(uint32_t *)pairArrayOut[6].value;

  if (layout != NvSciBufImage_BlockLinearType) {
    printf("Image layout is not block linear.. waiving execution\n");
    exit(EXIT_WAIVED);
  }

  hipExternalMemoryHandleDesc memHandleDesc;
  memset(&memHandleDesc, 0, sizeof(memHandleDesc));
  memHandleDesc.type = cudaExternalMemoryHandleTypeNvSciBuf;
  memHandleDesc.handle.nvSciBufObject = inputBufObj;
  memHandleDesc.size = size;
  checkCudaErrors(
      hipImportExternalMemory(&cudaExtResObj.extMemImageBuf, &memHandleDesc));

  cudaExtResObj.d_mipmapArray = (hipMipmappedArray_t *)malloc(
      sizeof(hipMipmappedArray_t) * cudaExtResObj.planeCount);

  for (int i = 0; i < cudaExtResObj.planeCount; i++) {
    hipExtent extent = {};
    memset(&extent, 0, sizeof(extent));
    extent.width = cudaExtResObj.imageWidth[i];
    extent.height = cudaExtResObj.imageHeight[i];
    extent.depth = 0;
    hipChannelFormatDesc desc;
    switch (channelCount) {
      case 1:
      default:
        desc = hipCreateChannelDesc(bitsPerPixel, 0, 0, 0,
                                     hipChannelFormatKindUnsigned);
        break;
      case 2:
        desc = hipCreateChannelDesc(bitsPerPixel, bitsPerPixel, 0, 0,
                                     hipChannelFormatKindUnsigned);
        break;
      case 3:
        desc = hipCreateChannelDesc(bitsPerPixel, bitsPerPixel, bitsPerPixel,
                                     0, hipChannelFormatKindUnsigned);
        break;
      case 4:
        desc =
            hipCreateChannelDesc(bitsPerPixel, bitsPerPixel, bitsPerPixel,
                                  bitsPerPixel, hipChannelFormatKindUnsigned);
        break;
    }

    cudaExternalMemoryMipmappedArrayDesc mipmapDesc = {0};
    mipmapDesc.offset = cudaExtResObj.planeOffset[i];
    mipmapDesc.formatDesc = desc;
    mipmapDesc.extent = extent;
    mipmapDesc.flags = 0;
    mipmapDesc.numLevels = 1;
    checkCudaErrors(cudaExternalMemoryGetMappedMipmappedArray(
        &cudaExtResObj.d_mipmapArray[i], cudaExtResObj.extMemImageBuf,
        &mipmapDesc));
  }
}

static hipSurfaceObject_t createCudaSurface(hipArray_t &d_mipLevelArray) {
  hipResourceDesc resourceDesc;
  memset(&resourceDesc, 0, sizeof(resourceDesc));
  resourceDesc.resType = hipResourceTypeArray;
  resourceDesc.res.array.array = d_mipLevelArray;

  hipSurfaceObject_t surfaceObject;
  checkCudaErrors(hipCreateSurfaceObject(&surfaceObject, &resourceDesc));
  return surfaceObject;
}

static hipStream_t createCudaStream(int deviceId) {
  checkCudaErrors(hipSetDevice(deviceId));
  hipStream_t stream;
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  return stream;
}

// CUDA setup buffers/synchronization objects for interop via NvSci API.
void setupCuda(cudaExternalResInterop &cudaExtResObj, NvSciBufObj &inputBufObj,
               NvSciSyncObj &syncObj, NvSciSyncObj &cudaSignalerSyncObj,
               int deviceId) {
  checkCudaErrors(hipSetDevice(deviceId));
  cudaImportNvSciSync(cudaExtResObj.waitSem, syncObj);
  cudaImportNvSciSync(cudaExtResObj.signalSem, cudaSignalerSyncObj);

  cudaImportNvSciImage(cudaExtResObj, inputBufObj);
  cudaExtResObj.d_mipLevelArray =
      (hipArray_t *)malloc(sizeof(hipArray_t) * cudaExtResObj.planeCount);
  cudaExtResObj.cudaSurfaceNvmediaBuf = (hipSurfaceObject_t *)malloc(
      sizeof(hipSurfaceObject_t) * cudaExtResObj.planeCount);

  for (int i = 0; i < cudaExtResObj.planeCount; ++i) {
    uint32_t mipLevelId = 0;
    checkCudaErrors(
        hipGetMipmappedArrayLevel(&cudaExtResObj.d_mipLevelArray[i],
                                   cudaExtResObj.d_mipmapArray[i], mipLevelId));
    cudaExtResObj.cudaSurfaceNvmediaBuf[i] =
        createCudaSurface(cudaExtResObj.d_mipLevelArray[i]);
  }

  cudaExtResObj.stream = createCudaStream(deviceId);
  checkCudaErrors(hipMalloc(&cudaExtResObj.d_outputImage,
                             sizeof(unsigned int) *
                                 cudaExtResObj.imageWidth[0] *
                                 cudaExtResObj.imageHeight[0]));
}

// CUDA clean up buffers used **with** NvSci API.
void cleanupCuda(cudaExternalResInterop &cudaExtResObj) {
  for (int i = 0; i < cudaExtResObj.planeCount; i++) {
    checkCudaErrors(
        hipDestroySurfaceObject(cudaExtResObj.cudaSurfaceNvmediaBuf[i]));
    checkCudaErrors(hipFreeMipmappedArray(cudaExtResObj.d_mipmapArray[i]));
  }
  free(cudaExtResObj.d_mipmapArray);
  free(cudaExtResObj.d_mipLevelArray);
  free(cudaExtResObj.cudaSurfaceNvmediaBuf);
  free(cudaExtResObj.imageWidth);
  free(cudaExtResObj.imageHeight);
  checkCudaErrors(hipDestroyExternalSemaphore(cudaExtResObj.waitSem));
  checkCudaErrors(hipDestroyExternalSemaphore(cudaExtResObj.signalSem));
  checkCudaErrors(hipDestroyExternalMemory(cudaExtResObj.extMemImageBuf));
  checkCudaErrors(hipStreamDestroy(cudaExtResObj.stream));
  checkCudaErrors(hipFree(cudaExtResObj.d_outputImage));
}

void runCudaOperation(cudaExternalResInterop &cudaExtResObj,
                      NvSciSyncFence *cudaWaitFence,
                      NvSciSyncFence *cudaSignalFence, int deviceId,
                      int iterations) {
  checkCudaErrors(hipSetDevice(deviceId));
  static int64_t launch = 0;

  waitExternalSemaphore(cudaExtResObj.waitSem, cudaWaitFence,
                        cudaExtResObj.stream);

  // run cuda kernel over surface object of the LUMA surface part to extract
  // grayscale.
  yuvToGrayscaleCudaKernel(cudaExtResObj, cudaExtResObj.imageWidth[0],
                           cudaExtResObj.imageHeight[0]);

  // signal fence till the second last iterations for NvMedia2DBlit to wait for
  // cuda signal and for final iteration as there is no corresponding NvMedia
  // operation pending therefore we end with hipStreamSynchronize()
  if (launch < iterations - 1) {
    signalExternalSemaphore(cudaExtResObj.signalSem, cudaSignalFence,
                            cudaExtResObj.stream);
  } else {
    checkCudaErrors(hipStreamSynchronize(cudaExtResObj.stream));
  }
  launch++;
}

// CUDA imports and operates on NvSci buffer/synchronization objects
void setupCuda(Blit2DTest *ctx, cudaResources &cudaResObj, int deviceId) {
  checkCudaErrors(hipSetDevice(deviceId));
  cudaResObj.d_yuvArray =
      (hipArray_t *)malloc(sizeof(hipArray_t) * ctx->numSurfaces);
  cudaResObj.cudaSurfaceNvmediaBuf = (hipSurfaceObject_t *)malloc(
      sizeof(hipSurfaceObject_t) * ctx->numSurfaces);
  hipChannelFormatDesc channelDesc;
  switch (ctx->bytesPerPixel) {
    case 1:
    default:
      channelDesc =
          hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
      break;
  }

  for (int k = 0; k < ctx->numSurfaces; k++) {
    checkCudaErrors(hipMallocArray(
        &cudaResObj.d_yuvArray[k], &channelDesc,
        ctx->widthSurface * ctx->xScalePtr[k] * ctx->bytesPerPixel,
        ctx->heightSurface * ctx->yScalePtr[k]));
    cudaResObj.cudaSurfaceNvmediaBuf[k] =
        createCudaSurface(cudaResObj.d_yuvArray[k]);
  }
  checkCudaErrors(hipMalloc(
      &cudaResObj.d_outputImage,
      sizeof(unsigned int) * ctx->widthSurface * ctx->heightSurface));

  cudaResObj.stream = createCudaStream(deviceId);
}

// CUDA clean up buffers used **without** NvSci API.
void cleanupCuda(Blit2DTest *ctx, cudaResources &cudaResObj) {
  for (int k = 0; k < ctx->numSurfaces; k++) {
    checkCudaErrors(
        hipDestroySurfaceObject(cudaResObj.cudaSurfaceNvmediaBuf[k]));
    checkCudaErrors(hipFreeArray(cudaResObj.d_yuvArray[k]));
  }

  free(cudaResObj.cudaSurfaceNvmediaBuf);

  checkCudaErrors(hipStreamDestroy(cudaResObj.stream));
  checkCudaErrors(hipFree(cudaResObj.d_outputImage));
}

static void yuvToGrayscaleCudaKernelNonNvSci(cudaResources &cudaResObj,
                                             int deviceId, int32_t imageWidth,
                                             int32_t imageHeight) {
#if WRITE_OUTPUT_IMAGE
  unsigned int *h_dstImage;
  checkCudaErrors(hipHostMalloc(
      &h_dstImage, sizeof(unsigned int) * imageHeight * imageWidth));
#endif
  dim3 block(16, 16, 1);
  dim3 grid((imageWidth / block.x) + 1, (imageHeight / block.y) + 1, 1);

  yuvToGrayscale<<<grid, block, 0, cudaResObj.stream>>>(
      cudaResObj.cudaSurfaceNvmediaBuf[0], cudaResObj.d_outputImage, imageWidth,
      imageHeight);

#if WRITE_OUTPUT_IMAGE
  checkCudaErrors(
      hipMemcpyAsync(h_dstImage, cudaResObj.d_outputImage,
                      sizeof(unsigned int) * imageHeight * imageWidth,
                      hipMemcpyDeviceToHost, cudaResObj.stream));
  checkCudaErrors(hipStreamSynchronize(cudaResObj.stream));
  char outputFilename[1024];
  std::string image_filename = "Grayscale";
  strcpy(outputFilename, image_filename.c_str());
  strcpy(outputFilename + image_filename.length(), "_non-nvsci_out.ppm");
  sdkSavePPM4ub(outputFilename, (unsigned char *)h_dstImage, imageWidth,
                imageHeight);
  printf("Wrote '%s'\n", outputFilename);
  checkCudaErrors(hipHostFree(h_dstImage));
#else
  checkCudaErrors(hipStreamSynchronize(cudaResObj.stream));
#endif
}

// CUDA operates **without** NvSci APIs buffer/synchronization objects.
void runCudaOperation(Blit2DTest *ctx, cudaResources &cudaResObj,
                      int deviceId) {
  for (int k = 0; k < ctx->numSurfaces; k++) {
    checkCudaErrors(hipMemcpy2DToArray(
        cudaResObj.d_yuvArray[k], 0, 0, ctx->dstBuff[k],
        ctx->widthSurface * ctx->xScalePtr[k] * ctx->bytesPerPixel,
        ctx->widthSurface * ctx->xScalePtr[k] * ctx->bytesPerPixel,
        ctx->heightSurface * ctx->yScalePtr[k], hipMemcpyHostToDevice));
  }
  // run cuda kernel over surface object of the LUMA surface part to extract
  // grayscale.
  yuvToGrayscaleCudaKernelNonNvSci(cudaResObj, deviceId, ctx->widthSurface,
                                   ctx->heightSurface);
}
