#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Simple interop app demonstrating EGLImage + EGLSync interop with CUDA.
// Using EGLSync - CUDA Event interop one can achieve synchronization on GPU
// itself for GL-EGL-CUDA operations instead of blocking CPU for
// synchronization. This app requires GLES 3.2 or higher

//---------------------------INCLUDES---------------------------------//
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include "graphics_interface.h"
#include <hip/hip_runtime.h>
#include <helper_cuda_drvapi.h>
#include <cudaEGL.h>
#include <EGL/egl.h>
#include <EGL/eglext.h>
#include <GLES3/gl32.h>
#include "egl_common.h"

//---------------------------DEFINES---------------------------------//
#define MAX_ITR 100

#define FAILURE 0
#define SUCCESS 1
#define WAIVED 2

#define BLOCK_SIZE 16

#define GL_READ 0
#define GL_WRITE 1
//---------------------------MACROS---------------------------------//

// Error-checking wrapper around GL calls
#define GL_SAFE_CALL(call)                                              \
  {                                                                     \
    GLenum err;                                                         \
    call;                                                               \
    err = glGetError();                                                 \
    if (err != GL_NO_ERROR) {                                           \
      fprintf(stderr, "%s:%d GL error: %d\n", __FILE__, __LINE__, err); \
      cleanup(FAILURE);                                                 \
    }                                                                   \
  }

#define GL_SAFE_CALL_NO_CLEANUP(call, err)                                 \
  {                                                                        \
    GLenum status;                                                         \
    call;                                                                  \
    status = glGetError();                                                 \
    if (status != GL_NO_ERROR) {                                           \
      fprintf(stderr, "%s:%d GL error: %d\n", __FILE__, __LINE__, status); \
      err = status;                                                        \
    }                                                                      \
  }

// Error-checking wrapper around CUDA calls (taken from cutil.h)
#define CUDA_SAFE_CALL(call)                                                  \
  do {                                                                        \
    hipError_t err = call;                                                     \
    if (hipSuccess != err) {                                                 \
      fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, \
              __LINE__, hipGetErrorString(err));                             \
      cleanup(FAILURE);                                                       \
    }                                                                         \
  } while (0)

#define CUDA_SAFE_CALL_NO_CLEANUP(call, err)                                  \
  do {                                                                        \
    hipError_t status = call;                                                  \
    if (hipSuccess != status) {                                              \
      fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", __FILE__, \
              __LINE__, hipGetErrorString(status));                          \
      err = status;                                                           \
    }                                                                         \
  } while (0)

#if defined(EXTENSION_LIST)
EXTENSION_LIST(EXTLST_DECL)
typedef void (*extlst_fnptr_t)(void);
static struct {
  extlst_fnptr_t *fnptr;
  char const *name;
} extensionList[] = {EXTENSION_LIST(EXTLST_ENTRY)};

int eglSetupExtensions(void) {
  unsigned int i;

  for (i = 0; i < (sizeof(extensionList) / sizeof(*extensionList)); i++) {
    *extensionList[i].fnptr = eglGetProcAddress(extensionList[i].name);
    if (*extensionList[i].fnptr == NULL) {
      printf("Couldn't get address of %s()\n", extensionList[i].name);
      return 0;
    }
  }

  return 1;
}
#endif

#if defined(EXTENSION_LIST)
EXTENSION_LIST(EXTLST_EXTERN)
#endif

//------------------------GLOBAL VARIABLES--------------------------//

// GL texture
GLuint tex[2] = {0};

// Used to catch unexpected termination from GLUT
int cleanExit = 0;

// Use CPU Sync or GPU sync; Default GPU
int useGpu = 1;

// CUDA Resource
hipGraphicsResource_t writeResource = NULL;
hipGraphicsResource_t readResource = NULL;
hipArray_t writeArray, readArray;
hipDevice_t device;
hipCtx_t context;

// Which device to run on
unsigned int dev = 0;

// Default width, height, and iterations value
int width = 2048;
int height = 2048;
int itr = MAX_ITR;

// Error check variable
__device__ static unsigned int numErrors = 0;

//-----------------------FUNCTION PROTOTYPES------------------------//

void checkSync(int argc, char **argv);
int parseCmdLine(int argc, char **argv);
void printUsage(void);
void cleanup(int status);
void exitHandler(void);
void printStatus(int status);
void checkSyncOnCPU(void);
void checkSyncOnGPU(EGLDisplay dpy);

__global__ void verify_and_update_kernel(hipSurfaceObject_t write, hipSurfaceObject_t read,
                                         char expected, char newval, int width,
                                         int height);
extern "C" hipError_t cudaGetValueMismatch();

//-----------------------FUNCTION DEFINITIONS------------------------//

int main(int argc, char *argv[]) {
#if defined(__linux__)
  setenv("DISPLAY", ":0", 0);
#endif

  parseCmdLine(argc, argv);
  atexit(exitHandler);

  checkSync(argc, argv);
  return 0;
}

int parseCmdLine(int argc, char **argv) {
  int i;
  for (i = 1; i < argc; i++) {
    if (strcmp(argv[i], "-cpu") == 0) {
      useGpu = 0;
    }

    if (strcmp(argv[i], "-h") == 0) {
      printUsage();
      cleanup(SUCCESS);
    }

    if (strcmp(argv[i], "-width") == 0) {
      ++i;
      if (i == argc) {
        printf("width option must be followed by value\n");
        return FAILURE;
      }
      if (sscanf(argv[i], "%d", &width) != 1) {
        printf("Error: invalid width value\n");
        return FAILURE;
      }
    }

    if (strcmp(argv[i], "-height") == 0) {
      ++i;
      if (i == argc) {
        printf("height option must be followed by value\n");
        return FAILURE;
      }
      if (sscanf(argv[i], "%d", &height) != 1) {
        printf("Error: invalid height value\n");
        return FAILURE;
      }
    }
    if (strcmp(argv[i], "-itr") == 0) {
      ++i;
      if (i == argc) {
        printf("itr option must be followed by iteration value\n");
        return FAILURE;
      }
      if (sscanf(argv[i], "%d", &itr) != 1) {
        printf("Error: invalid iteration value\n");
        return FAILURE;
      }
    }
  }

  return SUCCESS;
}

void printUsage(void) {
  printf("Usage:\n");
  printf("\t-h\tPrint command line options\n");
  printf("\t-cpu\tSync on the CPU instead of the GPU\n");
  printf("\t-width w\tSet the width to w\n");
  printf("\t-height h\tSet the height to h\n");
  printf("\t-itr i\tSet number of iterations to i\n");
}

void checkSync(int argc, char **argv) {
  int x, y;
  int bufferSize = width * height * 4;
  unsigned char *pSurf_read = NULL, *pSurf_write = NULL;
  int integrated;

  hipError_t status = hipSuccess;

  // Init values for variables
  x = y = 0;

  if (hipSuccess != (status = hipInit(0))) {
    printf("Failed to initialize CUDA\n");
  }
  device = findCudaDeviceDRV(argc, (const char **)argv);

  if (hipSuccess != (status = hipCtxCreate(&context, 0, device))) {
    printf("failed to create CUDA context\n");
  }
  hipCtxPushCurrent(context);

  status =
      hipDeviceGetAttribute(&integrated, hipDeviceAttributeIntegrated, device);
  if (status != hipSuccess) {
    printf("Failed to get device attribute hipDeviceAttributeIntegrated\n");
    cleanup(FAILURE);
  }

  if (integrated != 1) {
    printf(
        "EGLSync_CUDAEvent_Interop does not support dGPU. Waiving sample.\n");
    cleanup(WAIVED);
  }

#if (defined(__arm__) || defined(__aarch64__)) && defined(__linux__)
  graphics_setup_window(0, 0, width, height, "EGLSync_CUDA_Interop");
#endif

  pSurf_read = (unsigned char *)malloc(bufferSize);
  pSurf_write = (unsigned char *)malloc(bufferSize);
  if (pSurf_read == NULL || pSurf_write == NULL) {
    printf("malloc failed\n");
    cleanup(FAILURE);
  }

  for (x = 0; x < width; x++) {
    for (y = 0; y < height; y++) {
      pSurf_read[(y * width + x) * 4] = 1;
      pSurf_read[(y * width + x) * 4 + 1] = 1;
      pSurf_read[(y * width + x) * 4 + 2] = 1;
      pSurf_read[(y * width + x) * 4 + 3] = 1;
      pSurf_write[(y * width + x) * 4] = 0;
      pSurf_write[(y * width + x) * 4 + 1] = 0;
      pSurf_write[(y * width + x) * 4 + 2] = 0;
      pSurf_write[(y * width + x) * 4 + 3] = 0;
    }
  }

  // NOP call to error-check the above glut calls
  GL_SAFE_CALL({});

  // Init texture
  GL_SAFE_CALL(glGenTextures(2, tex));

  GL_SAFE_CALL(glBindTexture(GL_TEXTURE_2D, tex[GL_READ]));
  GL_SAFE_CALL(
      glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST));
  GL_SAFE_CALL(
      glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST));
  GL_SAFE_CALL(glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0,
                            GL_RGBA, GL_UNSIGNED_BYTE, pSurf_read));
  GL_SAFE_CALL(glBindTexture(GL_TEXTURE_2D, 0));

  GL_SAFE_CALL(glBindTexture(GL_TEXTURE_2D, tex[GL_WRITE]));
  GL_SAFE_CALL(
      glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST));
  GL_SAFE_CALL(
      glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST));
  GL_SAFE_CALL(glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0,
                            GL_RGBA, GL_UNSIGNED_BYTE, pSurf_write));
  GL_SAFE_CALL(glBindTexture(GL_TEXTURE_2D, 0));

  glFinish();

  EGLDisplay eglDisplayHandle = eglGetCurrentDisplay();
  if (eglDisplayHandle == EGL_NO_DISPLAY) {
    printf("eglDisplayHandle failed \n");
    cleanup(FAILURE);
  } else {
    printf("eglDisplay Handle created \n");
  }

  if (!eglSetupExtensions()) {
    printf("SetupExtentions failed \n");
    cleanup(FAILURE);
  }

  EGLContext eglCtx = eglGetCurrentContext();
  if (eglCtx == EGL_NO_CONTEXT) {
    printf("Context1 create failed with error %d\n", eglGetError());
    cleanup(FAILURE);
  }

  // Create the EGL_Image
  EGLint eglImgAttrs[] = {EGL_IMAGE_PRESERVED_KHR, EGL_TRUE, EGL_NONE,
                          EGL_NONE};

  EGLImageKHR eglImage1 =
      eglCreateImageKHR(eglDisplayHandle, eglCtx, EGL_GL_TEXTURE_2D_KHR,
                        (EGLClientBuffer)(intptr_t)tex[GL_READ], eglImgAttrs);
  if (eglImage1 == EGL_NO_IMAGE_KHR) {
    printf("EGLImage create failed for read texture with error %d\n",
           eglGetError());
    cleanup(FAILURE);
  } else {
    printf("EGLImage1 created \n");
  }

  EGLImageKHR eglImage2 =
      eglCreateImageKHR(eglDisplayHandle, eglCtx, EGL_GL_TEXTURE_2D_KHR,
                        (EGLClientBuffer)(intptr_t)tex[GL_WRITE], eglImgAttrs);
  if (eglImage2 == EGL_NO_IMAGE_KHR) {
    printf("EGLImage create failed for write texture with error %d\n",
           eglGetError());
    cleanup(FAILURE);
  } else {
    printf("EGLImage2 created \n");
  }

  glFinish();

  status = cuGraphicsEGLRegisterImage(&writeResource, eglImage1,
                                      hipGraphicsRegisterFlagsNone);
  if (status != hipSuccess) {
    printf("cuGraphicsEGLRegisterImage failed with Texture 1\n");
    cleanup(FAILURE);
  } else {
    printf(
        "cuGraphicsEGLRegisterImage Passed, writeResource created with texture "
        "1\n");
  }

  status =
      hipGraphicsSubResourceGetMappedArray(&writeArray, writeResource, 0, 0);
  if (status != hipSuccess) {
    printf(
        "hipGraphicsSubResourceGetMappedArray failed for writeResource with "
        "texture 1\n");
    cleanup(FAILURE);
  }

  status = cuGraphicsEGLRegisterImage(&readResource, eglImage2,
                                      hipGraphicsRegisterFlagsNone);
  if (status != hipSuccess) {
    printf(
        "cuGraphicsEGLRegisterImage failed for readResource with Texture 2\n");
    cleanup(FAILURE);
  } else {
    printf(
        "cuGraphicsEGLRegisterImage Passed, readResource created with texture "
        "2\n");
  }

  status = hipGraphicsSubResourceGetMappedArray(&readArray, readResource, 0, 0);
  if (status != hipSuccess) {
    printf("hipGraphicsSubResourceGetMappedArray failed for texture 2\n");
    cleanup(FAILURE);
  }

  if (useGpu) {
    printf("Using GPU Sync path\n");
    checkSyncOnGPU(eglDisplayHandle);
  } else {
    printf("Using CPU Sync path\n");
    checkSyncOnCPU();
  }

  free(pSurf_read);
  free(pSurf_write);
  cleanup(SUCCESS);
}

void checkSyncOnCPU(void) {
  int z = 0;
  unsigned char expectedData, newData;
  hipError_t status = hipSuccess;
  HIP_RESOURCE_DESC wdsc, rdsc;
  memset(&wdsc, 0, sizeof(wdsc));
  memset(&rdsc, 0, sizeof(rdsc));

  expectedData = 0;
  newData = 1;

  wdsc.resType = HIP_RESOURCE_TYPE_ARRAY;
  wdsc.res.array.hArray = writeArray;
  hipSurfaceObject_t writeSurface;
  rdsc.resType = HIP_RESOURCE_TYPE_ARRAY;
  rdsc.res.array.hArray = readArray;
  hipSurfaceObject_t readSurface;

  status = cuSurfObjectCreate(&writeSurface, &wdsc);
  if (status != hipSuccess) {
    printf("Surface bounding failed with status %d\n", status);
    cleanup(FAILURE);
  }
  status = cuSurfObjectCreate(&readSurface, &rdsc);
  if (status != hipSuccess) {
    printf("Surface bounding failed\n");
    cleanup(FAILURE);
  }

  for (z = 0; z < itr; z++) {
    // GL call to copy from read texture to write texture
    GL_SAFE_CALL(glCopyImageSubData(tex[GL_READ], GL_TEXTURE_2D, 0, 0, 0, 0,
                                    tex[GL_WRITE], GL_TEXTURE_2D, 0, 0, 0, 0,
                                    width, height, 1));

    glFinish();

    newData++;
    expectedData++;

    verify_and_update_kernel<<<(width * height) / 256, 256>>>(
        writeSurface, readSurface, expectedData, newData, width, height);

    status = hipCtxSynchronize();
    if (status != hipSuccess) {
      printf("hipCtxSynchronize failed \n");
    }
  }

  hipError_t err = cudaGetValueMismatch();
  if (err != hipSuccess) {
    printf("Value mismatch seen when using CPU sync\n");
    cleanup(FAILURE);
  }

  // Clean up CUDA writeResource
  status = hipGraphicsUnregisterResource(writeResource);
  if (status != hipSuccess) {
    printf("Failed to unregister %d", status);
    cleanup(FAILURE);
  } else {
    printf("Unregistered writeResource. \n");
  }

  // Clean up CUDA readResource
  status = hipGraphicsUnregisterResource(readResource);
  if (status != hipSuccess) {
    printf("Failed to unregister %d", status);
    cleanup(FAILURE);
  } else {
    printf("Unregistered readResource. \n");
  }
}

/*
    Performs same function as checkSyncOnCPU
    Here instead of glFinish() and hipCtxSynchronize like in checkSyncOnCPU,
    we make use of EGLSync, CUDA Event and hipStreamWaitEvent, eglWaitSyncKHR to
   achieve the synchronization due to this CPU is not blocked for any
   synchronization needed between GL-EGL & CUDA operations all synchronizations
   happens on the GPU only.
*/
void checkSyncOnGPU(EGLDisplay dpy) {
  int z = 0;
  unsigned char expectedData, newData;
  hipError_t err;
  hipError_t status = hipSuccess;
  hipStream_t stream;
  hipEvent_t timingDisabledEvent;
  HIP_RESOURCE_DESC wdsc, rdsc;
  memset(&wdsc, 0, sizeof(wdsc));
  memset(&rdsc, 0, sizeof(rdsc));

  expectedData = 0;
  newData = 1;

  wdsc.resType = HIP_RESOURCE_TYPE_ARRAY;
  wdsc.res.array.hArray = writeArray;
  hipSurfaceObject_t writeSurface;
  rdsc.resType = HIP_RESOURCE_TYPE_ARRAY;
  rdsc.res.array.hArray = readArray;
  hipSurfaceObject_t readSurface;

  status = cuSurfObjectCreate(&writeSurface, &wdsc);
  if (status != hipSuccess) {
    printf("Surface bounding failed with status %d\n", status);
    cleanup(FAILURE);
  }
  status = cuSurfObjectCreate(&readSurface, &rdsc);
  if (status != hipSuccess) {
    printf("Surface bounding failed\n");
    cleanup(FAILURE);
  }

  status = hipStreamCreateWithFlags(&stream, hipStreamDefault);
  if (status != hipSuccess) {
    printf("Stream creation failed\n");
    cleanup(FAILURE);
  }

  // Creates timing disabled event which uses non-blocking synchronization
  status = hipEventCreateWithFlags(&timingDisabledEvent, hipEventDisableTiming);
  if (status != hipSuccess) {
    printf("Default event creation failed\n");
    cleanup(FAILURE);
  }

  /*
      1. We perform texture-to-texture copy in GLES which is async function
      2. Followed by creating EGLSync and a CUDA Event from that EGLSync object
      3. Using hipStreamWaitEvent() we wait in GPU for the GLES to finish texture
     copy.
      4. CUDA kernel verfiy_and_update_kernel verifies if the copied data by
     GLES is correct, and it updates the buffer with new values.
      5. This is followed by eglWaitSyncKHR() which waits for the cuda kernel to
     finish, so that in the next iteration GLES can perform the copying of the
     updated buffer to write texture,
  */
  for (z = 0; z < itr; z++) {
    // GL call to copy from read texture to write texture
    GL_SAFE_CALL(glCopyImageSubData(tex[GL_READ], GL_TEXTURE_2D, 0, 0, 0, 0,
                                    tex[GL_WRITE], GL_TEXTURE_2D, 0, 0, 0, 0,
                                    width, height, 1));

    EGLSyncKHR eglSyncForGL, eglSyncForCuda;
    EGLBoolean egl_status = EGL_TRUE;
    EGLAttribKHR eglattrib[] = {EGL_CUDA_EVENT_HANDLE_NV,
                                (EGLAttrib)timingDisabledEvent, EGL_NONE};

    hipEvent_t cudaEGLSyncEvent;

    eglSyncForGL = eglCreateSyncKHR(dpy, EGL_SYNC_FENCE_KHR, NULL);

    if (eglSyncForGL == EGL_NO_SYNC_KHR) {
      printf(" EGL Sync creation failed\n");
      cleanup(FAILURE);
    }

    status = cuEventCreateFromEGLSync(&cudaEGLSyncEvent, eglSyncForGL,
                                      hipEventDefault);
    if (status != hipSuccess) {
      printf("CUDA event creation from EGLSync failed\n");
      cleanup(FAILURE);
    }

    // We wait from CUDA in GPU for GL-EGL operation completion
    status = hipStreamWaitEvent(stream, cudaEGLSyncEvent, 0);
    if (status != hipSuccess) {
      printf("Stream wait for event created from EGLSync failed\n");
      cleanup(FAILURE);
    }

    egl_status = eglDestroySyncKHR(dpy, eglSyncForGL);
    if (egl_status != EGL_TRUE) {
      printf("EGL sync object destruction failed\n");
      cleanup(FAILURE);
    }

    newData++;
    expectedData++;

    // Verifies the values in readSurface which is copied by
    // glCopyImageSubData() And writes value of newData into writeSurface
    verify_and_update_kernel<<<(width * height) / 256, 256, 0, stream>>>(
        writeSurface, readSurface, expectedData, newData, width, height);

    status = hipEventDestroy(cudaEGLSyncEvent);
    if (status != hipSuccess) {
      printf("Event Destroy failed\n");
      cleanup(FAILURE);
    }

    status = hipEventRecord(timingDisabledEvent, stream);
    if (status != hipSuccess) {
      printf("Event Record failed\n");
      cleanup(FAILURE);
    }

    // creating an EGL sync object linked to a CUDA event object
    eglSyncForCuda = eglCreateSync64KHR(dpy, EGL_SYNC_CUDA_EVENT_NV, eglattrib);

    // We wait from EGL for CUDA operation completion
    egl_status = eglWaitSyncKHR(dpy, eglSyncForCuda, 0);
    if (egl_status != EGL_TRUE) {
      printf("eglWaitSyncKHR failed\n");
      cleanup(FAILURE);
    }
    egl_status = eglDestroySyncKHR(dpy, eglSyncForCuda);
    if (egl_status != EGL_TRUE) {
      printf("EGL sync object destruction failed\n");
      cleanup(FAILURE);
    }
  }

  err = cudaGetValueMismatch();
  if (err != hipSuccess) {
    printf("Value mismatch seen when using GPU sync\n");
    cleanup(FAILURE);
  }

  // Clean up CUDA writeResource
  status = hipGraphicsUnregisterResource(writeResource);
  if (status != hipSuccess) {
    printf("Failed to unregister %d", status);
    cleanup(FAILURE);
  } else {
    printf("Unregistered writeResource. \n");
  }

  // Clean up CUDA readResource
  status = hipGraphicsUnregisterResource(readResource);
  if (status != hipSuccess) {
    printf("Failed to unregister %d", status);
    cleanup(FAILURE);
  } else {
    printf("Unregistered readResource. \n");
  }
}

// Verifies the values in readSurface whether they are expected ones
// And writes value of newData into writeSurface
__global__ void verify_and_update_kernel(hipSurfaceObject_t write, hipSurfaceObject_t read,
                                         char expected, char newval, int width,
                                         int height) {
  unsigned int x = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int y = blockDim.y * blockIdx.y + threadIdx.y;

  if (x < width && y < height) {
    uchar4 check;
    surf2Dread(&check, read, x * 4, y);
    if (check.x != expected || check.y != expected || check.z != expected ||
        check.w != expected) {
      printf(
          "Mismatch found in values read[0]= %u read[1]= %u read[2]= %u "
          "read[3]= %u expected is %u\n",
          check.x, check.y, check.z, check.w, expected);
      numErrors++;
      return;
    }
    uchar4 data = make_uchar4(newval, newval, newval, newval);
    surf2Dwrite(data, write, x * 4, y);
  }
}

__global__ void getNumErrors(int *numErr) { *numErr = numErrors; }

extern "C" hipError_t cudaGetValueMismatch() {
  int numErr_h;
  int *numErr_d = NULL;
  hipError_t err = hipSuccess;

  err = hipMalloc(&numErr_d, sizeof(int));
  if (err != hipSuccess) {
    printf("Cuda Main: hipMemcpy failed with %s\n", hipGetErrorString(err));
    hipFree(numErr_d);
    return err;
  }

  getNumErrors<<<1, 1>>>(numErr_d);
  err = hipDeviceSynchronize();
  if (err != hipSuccess) {
    printf("Cuda Main: hipDeviceSynchronize failed with %s\n",
           hipGetErrorString(err));
  }
  err = hipMemcpy(&numErr_h, numErr_d, sizeof(int), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    printf("Cuda Main: hipMemcpy failed with %s\n", hipGetErrorString(err));
    hipFree(numErr_d);
    return err;
  }
  err = hipFree(numErr_d);
  if (err != hipSuccess) {
    printf("Cuda Main: hipFree failed with %s\n", hipGetErrorString(err));
    return err;
  }
  if (numErr_h > 0) {
    return hipErrorUnknown;
  }
  return hipSuccess;
}

// Clean up state and exit. If status is SUCCESS, regression success is printed
// to stdout. This will happen if the glut timer is triggered. If status is
// anything else, the regression failure message is printed.
void cleanup(int status) {
  GLenum glErr = GL_NO_ERROR;
  hipError_t cudaErr = hipSuccess;
  int exitStatus = status;

  // Clean up GL
  if (*tex) {
    GL_SAFE_CALL_NO_CLEANUP(glDeleteTextures(2, tex), glErr);
  }

  // Print test status and exit
  if (glErr != GL_NO_ERROR || cudaErr != hipSuccess) exitStatus = FAILURE;

  printStatus(exitStatus);

  cleanExit = 1;

  graphics_close_window();

  if (exitStatus == FAILURE) exit(EXIT_FAILURE);

  if (exitStatus == WAIVED) exit(EXIT_WAIVED);

  exit(0);
}

void exitHandler(void) {
  if (!cleanExit) {
    printf("&&&& EGLSync_CUDAEvent_Interop unexpected failure \n");
    printStatus(FAILURE);
  }
}

// Print test success or fail for regression testing
void printStatus(int status) {
  switch (status) {
    case SUCCESS:
      printf("&&&& EGLSync_CUDAEvent_Interop PASSED\n");
      break;
    case WAIVED:
      printf("&&&& EGLSync_CUDAEvent_Interop WAIVED\n");
      break;
    default:
      printf("&&&& EGLSync_CUDAEvent_Interop FAILED\n");
      break;
  }
  fflush(stdout);
}
