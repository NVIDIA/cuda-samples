#include "hip/hip_runtime.h"
/* Copyright (c) 2024, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

static __global__ void flipSurfaceBits(hipSurfaceObject_t surfObj, int width, int height) {
    char data;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        // Read from input surface
        surf2Dread(&data,  surfObj, x, y);
        // Write to output surface
        data = ~data;
        surf2Dwrite(data, surfObj, x, y);
    }
}

// Copy hipArray to surface memory and launch the CUDA kernel
void launchFlipSurfaceBitsKernel(
    hipArray_t *levelArray, 
    int32_t *multiPlanarWidth, 
    int32_t *multiPlanarHeight, 
    int numPlanes) {

    hipSurfaceObject_t surfObject[numPlanes] = {0};
    hipResourceDesc resDesc;
    
    for (int i = 0; i < numPlanes; i++) { 
        memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = levelArray[i];
        checkCudaErrors(hipCreateSurfaceObject(&surfObject[i], &resDesc));
        dim3 threadsperBlock(16, 16);
        dim3 numBlocks((multiPlanarWidth[i] + threadsperBlock.x - 1) / threadsperBlock.x,
                (multiPlanarHeight[i] + threadsperBlock.y - 1) / threadsperBlock.y);
        flipSurfaceBits<<<numBlocks, threadsperBlock>>>(surfObject[i], multiPlanarWidth[i], multiPlanarHeight[i]);
    }
}

