/* Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "cudla.h"
#include "hip/hip_runtime.h"
#include "cudlaExternalEtbl.hpp"

#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/stat.h>
#include <fstream>
#include <sstream>

#define MAX_FILENAME_LEN 200
#define RESERVED_SUFFIX_LEN 10

#define DPRINTF(...) printf(__VA_ARGS__)

static void printTensorDesc(cudlaModuleTensorDescriptor* tensorDesc) {
    DPRINTF("\tTENSOR NAME : %s\n", tensorDesc->name);
    DPRINTF("\tsize: %lu\n", tensorDesc->size);

    DPRINTF("\tdims: [%lu, %lu, %lu, %lu]\n", tensorDesc->n, tensorDesc->c,
            tensorDesc->h, tensorDesc->w);

    DPRINTF("\tdata fmt: %d\n", tensorDesc->dataFormat);
    DPRINTF("\tdata type: %d\n", tensorDesc->dataType);
    DPRINTF("\tdata category: %d\n", tensorDesc->dataCategory);
    DPRINTF("\tpixel fmt: %d\n", tensorDesc->pixelFormat);
    DPRINTF("\tpixel mapping: %d\n", tensorDesc->pixelMapping);
    DPRINTF("\tstride[0]: %d\n", tensorDesc->stride[0]);
    DPRINTF("\tstride[1]: %d\n", tensorDesc->stride[1]);
    DPRINTF("\tstride[2]: %d\n", tensorDesc->stride[2]);
    DPRINTF("\tstride[3]: %d\n", tensorDesc->stride[3]);
}

typedef struct {
    cudlaDevHandle devHandle;
    cudlaModule moduleHandle;
    unsigned char* loadableData;
    hipStream_t stream;
    uint32_t numInputTensors;
    uint32_t numOutputTensors;
    uint32_t numOutputTaskStatistics;
    unsigned char** inputBuffer;
    unsigned char** outputBuffer;
    unsigned char** statisticsOutputBuffer;
    void** inputBufferGPU;
    void** outputBufferGPU;
    void** outputTaskStatisticsGPU;
    void **csv;
    cudlaModuleTensorDescriptor* inputTensorDesc;
    cudlaModuleTensorDescriptor* outputTensorDesc;
    cudlaModuleTensorDescriptor* outputTaskStatisticsDesc;
    uint64_t** inputBufferRegisteredPtr;
    uint64_t** outputBufferRegisteredPtr;
    uint64_t** outputTaskStatisticsRegisteredPtr;
    uint64_t** outputStatisticsBufferRegisteredPtr;
} ResourceList;

void cleanUp(ResourceList* resourceList);

void cleanUp(ResourceList* resourceList) {
    uint32_t ii = 0;
    if (resourceList->inputTensorDesc != NULL) {
        free(resourceList->inputTensorDesc);
        resourceList->inputTensorDesc = NULL;
    }

    if (resourceList->outputTensorDesc != NULL) {
        free(resourceList->outputTensorDesc);
        resourceList->outputTensorDesc = NULL;
    }

    if (resourceList->outputTaskStatisticsDesc != NULL) {
        free(resourceList->outputTaskStatisticsDesc);
        resourceList->outputTaskStatisticsDesc = NULL;
    }

    if (resourceList->loadableData != NULL) {
        free(resourceList->loadableData);
        resourceList->loadableData = NULL;
    }

    if (resourceList->moduleHandle != NULL) {
        cudlaModuleUnload(resourceList->moduleHandle, 0);
        resourceList->moduleHandle = NULL;
    }

    if (resourceList->devHandle != NULL) {
        cudlaDestroyDevice(resourceList->devHandle);
        resourceList->devHandle = NULL;
    }

    if (resourceList->inputBufferGPU != NULL) {
        for (ii = 0; ii < resourceList->numInputTensors; ii++) {
            if ((resourceList->inputBufferGPU)[ii] != NULL) {
                hipFree((resourceList->inputBufferGPU)[ii]);
                (resourceList->inputBufferGPU)[ii] = NULL;
            }
        }
        free(resourceList->inputBufferGPU);
        resourceList->inputBufferGPU = NULL;
    }

    if (resourceList->outputBufferGPU != NULL) {
        for (ii = 0; ii < resourceList->numOutputTensors; ii++) {
            if ((resourceList->outputBufferGPU)[ii] != NULL) {
                hipFree((resourceList->outputBufferGPU)[ii]);
                (resourceList->outputBufferGPU)[ii] = NULL;
            }
        }
        free(resourceList->outputBufferGPU);
        resourceList->outputBufferGPU = NULL;
    }

    if (resourceList->outputTaskStatisticsGPU != NULL) {
        for (ii = 0; ii < resourceList->numOutputTaskStatistics; ii++) {
            if ((resourceList->outputTaskStatisticsGPU)[ii] != NULL) {
                hipFree((resourceList->outputTaskStatisticsGPU)[ii]);
                (resourceList->outputTaskStatisticsGPU)[ii] = NULL;
            }
        }
        free(resourceList->outputTaskStatisticsGPU);
        resourceList->outputTaskStatisticsGPU = NULL;
    }

    if (resourceList->csv != NULL) {
        for (ii = 0; ii < resourceList->numOutputTaskStatistics; ii++) {
            if ((resourceList->csv)[ii] != NULL)
            {
                free((resourceList->csv)[ii]);
                (resourceList->csv)[ii] = NULL;
            }
        }
        free(resourceList->csv);
        resourceList->csv = NULL;
    }

    if (resourceList->inputBuffer != NULL) {
        for (ii = 0; ii < resourceList->numInputTensors; ii++) {
            if ((resourceList->inputBuffer)[ii] != NULL) {
                free((resourceList->inputBuffer)[ii]);
                (resourceList->inputBuffer)[ii] = NULL;
            }
        }
        free(resourceList->inputBuffer);
        resourceList->inputBuffer = NULL;
    }

    if (resourceList->outputBuffer != NULL) {
        for (ii = 0; ii < resourceList->numOutputTensors; ii++) {
            if ((resourceList->outputBuffer)[ii] != NULL)
            {
                free((resourceList->outputBuffer)[ii]);
                (resourceList->outputBuffer)[ii] = NULL;
            }
        }
        free(resourceList->outputBuffer);
        resourceList->outputBuffer = NULL;
    }

    if (resourceList->statisticsOutputBuffer != NULL) {
        for (ii = 0; ii < resourceList->numOutputTaskStatistics; ii++) {
            if ((resourceList->statisticsOutputBuffer)[ii] != NULL) {
                free((resourceList->statisticsOutputBuffer)[ii]);
                (resourceList->statisticsOutputBuffer)[ii] = NULL;
            }
        }
        free(resourceList->statisticsOutputBuffer);
        resourceList->statisticsOutputBuffer = NULL;
    }

    if (resourceList->stream != NULL) {
        hipStreamDestroy(resourceList->stream);
        resourceList->stream = NULL;
    }

    if (resourceList->inputBufferRegisteredPtr != NULL) {
        free(resourceList->inputBufferRegisteredPtr);
        resourceList->inputBufferRegisteredPtr = NULL;
    }

    if (resourceList->outputBufferRegisteredPtr != NULL) {
        free(resourceList->outputBufferRegisteredPtr);
        resourceList->outputBufferRegisteredPtr = NULL;
    }

    if (resourceList->outputTaskStatisticsRegisteredPtr != NULL) {
        free(resourceList->outputTaskStatisticsRegisteredPtr);
        resourceList->outputTaskStatisticsRegisteredPtr = NULL;
    }

    if (resourceList->outputStatisticsBufferRegisteredPtr != NULL) {
        free(resourceList->outputStatisticsBufferRegisteredPtr);
        resourceList->outputStatisticsBufferRegisteredPtr = NULL;
    }

    resourceList->numInputTensors = 0;
    resourceList->numOutputTensors = 0;
    resourceList->numOutputTaskStatistics = 0;
}

int main(int argc, char** argv) {
    cudlaDevHandle devHandle;
    cudlaModule moduleHandle;
    cudlaStatus err;
    uint32_t statSupport = 0;
    uint32_t dlaFreqInMHz = 0;
    FILE* fp = NULL;
    struct stat st;
    size_t file_size;
    size_t actually_read = 0;
    unsigned char *loadableData = NULL;
    char filename[MAX_FILENAME_LEN];
    const char* suffix = ".csv";

    hipStream_t stream;
    hipError_t result;
    const char* errPtr = NULL;

    ResourceList resourceList;

    memset(&resourceList, 0x00, sizeof(ResourceList));

    if ((argc != 4) && (argc != 5)) {
        DPRINTF("Usage : ./test_cudla_layerwise_stats_L0_hybrid_test1 <loadable> <freqMHZ> <statSupport> <filename prefix>\n");
        return 1;
    }

    if (argc == 5) {
        if((strlen(argv[4])) > (MAX_FILENAME_LEN - RESERVED_SUFFIX_LEN))
        {
            DPRINTF("Filename prefix length is too big, greater than maximum permissible prefix length of %u \n",(MAX_FILENAME_LEN - RESERVED_SUFFIX_LEN));
            return 1;
        }
    }

    // Read loadable into buffer.
    fp = fopen(argv[1], "rb");
    if (fp == NULL) {
        DPRINTF("Cannot open file %s\n", argv[1]);
        return 1;
    }

    if (stat(argv[1], &st) != 0) {
        DPRINTF("Cannot stat file\n");
        return 1;
    }

    file_size = st.st_size;
    DPRINTF("The file size = %ld\n", file_size);

    dlaFreqInMHz = atoi(argv[2]);
    statSupport = atoi(argv[3]);

    loadableData = (unsigned char *)malloc(file_size);
    if (loadableData == NULL) {
        DPRINTF("Cannot Allocate memory for loadable\n");
        return 1;
    }

    actually_read = fread(loadableData, 1, file_size, fp);
    if ( actually_read != file_size ) {
        free(loadableData);
        DPRINTF("Read wrong size\n");
        return 1;
    }
    fclose(fp);

    resourceList.loadableData = loadableData;

    // Initialize CUDA.
    result = hipFree(0);
    if (result != hipSuccess) {
        errPtr = hipGetErrorName(result);
        DPRINTF("Error in creating hipFree = %s\n", errPtr);
        cleanUp(&resourceList);
        return 1;
    }

    result = hipSetDevice(0);
    if (result != hipSuccess) {
        errPtr = hipGetErrorName(result);
        DPRINTF("Error in creating hipSetDevice = %s\n", errPtr);
        cleanUp(&resourceList);
        return 1;
    }

    err = cudlaCreateDevice(0, &devHandle, CUDLA_CUDA_DLA);
    if (err != cudlaSuccess) {
        DPRINTF("Error in cuDLA create device = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    }

    DPRINTF("Device created successfully\n");
    resourceList.devHandle = devHandle;

    err = cudlaModuleLoadFromMemory(devHandle, loadableData, file_size, &moduleHandle, 0);
    if (err != cudlaSuccess) {
        DPRINTF("Error in cudlaModuleLoadFromMemory = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    } else {
      DPRINTF("Successfully loaded module\n");
    }

    resourceList.moduleHandle = moduleHandle;

    // Create CUDA stream.
    result = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    if (result != hipSuccess) {
        errPtr = hipGetErrorName(result);
        DPRINTF("Error in creating cuda stream = %s\n", errPtr);
        cleanUp(&resourceList);
        return 1;
    }

    resourceList.stream = stream;

    // Get tensor attributes.
    uint32_t numInputTensors = 0;
    uint32_t numOutputTensors = 0;
    uint32_t numOutputTaskStatistics = 0;

    cudlaModuleAttribute attribute;

    err = cudlaModuleGetAttributes(moduleHandle, CUDLA_NUM_INPUT_TENSORS, &attribute);
    if (err != cudlaSuccess) {
        DPRINTF("Error in getting numInputTensors = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    }
    numInputTensors = attribute.numInputTensors;
    DPRINTF("numInputTensors = %d\n", numInputTensors);

    err = cudlaModuleGetAttributes(moduleHandle, CUDLA_NUM_OUTPUT_TENSORS, &attribute);
    if (err != cudlaSuccess) {
        DPRINTF("Error in getting numOutputTensors = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    }
    numOutputTensors = attribute.numOutputTensors;
    DPRINTF("numOutputTensors = %d\n", numOutputTensors);

    // using the same attributes to get num_output_task_statistics_tensors
    attribute.numOutputTensors = 0;

    err = cudlaModuleGetAttributes(moduleHandle, CUDLA_NUM_OUTPUT_TASK_STATISTICS, &attribute);
    if (err != cudlaSuccess) {
        DPRINTF("Error in getting numOutputTensors = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    }
    numOutputTaskStatistics = attribute.numOutputTensors;
    DPRINTF("numOutputTaskStatistics = %d\n", numOutputTaskStatistics);

    if(numOutputTaskStatistics == 0) {
        DPRINTF("Layerwise stats is not supported for this Loadable \n");
        cleanUp(&resourceList);
        return 1;
    }

    resourceList.numInputTensors = numInputTensors;
    resourceList.numOutputTensors = numOutputTensors;
    resourceList.numOutputTaskStatistics = numOutputTaskStatistics;

    cudlaModuleTensorDescriptor* inputTensorDesc =
        (cudlaModuleTensorDescriptor*)malloc(sizeof(cudlaModuleTensorDescriptor)*numInputTensors);
    cudlaModuleTensorDescriptor* outputTensorDesc =
        (cudlaModuleTensorDescriptor*)malloc(sizeof(cudlaModuleTensorDescriptor)*numOutputTensors);

    if ((inputTensorDesc == NULL) || (outputTensorDesc == NULL)) {
        if (inputTensorDesc != NULL) {
            free(inputTensorDesc);
            inputTensorDesc = NULL;
        }

        if (outputTensorDesc != NULL) {
            free(outputTensorDesc);
            outputTensorDesc = NULL;
        }

        cleanUp(&resourceList);
        return 1;
    }

    resourceList.inputTensorDesc = inputTensorDesc;
    resourceList.outputTensorDesc = outputTensorDesc;

    cudlaModuleTensorDescriptor* outputTaskStatisticsDesc =
    (cudlaModuleTensorDescriptor*)malloc(sizeof(cudlaModuleTensorDescriptor)*numOutputTaskStatistics);
    if (outputTaskStatisticsDesc == NULL) {
        free(outputTaskStatisticsDesc);
        outputTaskStatisticsDesc = NULL;
        cleanUp(&resourceList);
        return 1;
    }

    resourceList.outputTaskStatisticsDesc = outputTaskStatisticsDesc;

    attribute.inputTensorDesc = inputTensorDesc;
    err = cudlaModuleGetAttributes(moduleHandle,
                                   CUDLA_INPUT_TENSOR_DESCRIPTORS,
                                   &attribute);
    if (err != cudlaSuccess) {
        DPRINTF("Error in getting input tensor descriptor = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    }
    DPRINTF("Printing input tensor descriptor\n");
    printTensorDesc(inputTensorDesc);

    attribute.outputTensorDesc = outputTensorDesc;
    err = cudlaModuleGetAttributes(moduleHandle,
                                   CUDLA_OUTPUT_TENSOR_DESCRIPTORS,
                                   &attribute);
    if (err != cudlaSuccess) {
        DPRINTF("Error in getting output tensor descriptor = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    }
    DPRINTF("Printing output tensor descriptor\n");
    printTensorDesc(outputTensorDesc);

    attribute.outputTensorDesc = outputTaskStatisticsDesc;
    err = cudlaModuleGetAttributes(moduleHandle,
                                   CUDLA_OUTPUT_TASK_STATISTICS_DESCRIPTORS,
                                   &attribute);
    if (err != cudlaSuccess) {
        DPRINTF("Error in getting task statistics descriptor = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    }

    DPRINTF("Printing output task statistics descriptor size\n");
    for (uint32_t ii = 0; ii < numOutputTaskStatistics; ii++) {
        DPRINTF("The size of %u descriptor is %lu\n", ii,outputTaskStatisticsDesc[ii].size);
    }

    // Setup the input and output buffers which will be used as an input to CUDA.
    unsigned char** inputBuffer = (unsigned char **)malloc(sizeof(unsigned char *)*numInputTensors);
    if (inputBuffer == NULL) {
        DPRINTF("Error in allocating memory for input buffer array\n");
        cleanUp(&resourceList);
        return 1;
    }
    memset(inputBuffer, 0x00, sizeof(unsigned char *)*numInputTensors);
    resourceList.inputBuffer = inputBuffer;
    for (uint32_t ii = 0; ii < numInputTensors; ii++) {
        inputBuffer[ii] = (unsigned char* )malloc(inputTensorDesc[ii].size);
        if (inputBuffer[ii] == NULL) {
            DPRINTF("Error in allocating input memory\n");
            cleanUp(&resourceList);
            return 1;
        }
        memset(inputBuffer[ii], 0x01, inputTensorDesc[ii].size);
    }

    unsigned char** outputBuffer = (unsigned char **)malloc(sizeof(unsigned char *)*numOutputTensors);
    if (outputBuffer == NULL) {
        DPRINTF("Error in allocating memory for output buffer array\n");
        cleanUp(&resourceList);
        return 1;
    }
    memset(outputBuffer, 0x00, sizeof(unsigned char *)*numOutputTensors);
    resourceList.outputBuffer = outputBuffer;

    for (uint32_t ii = 0; ii < numOutputTensors; ii++) {
        outputBuffer[ii] = (unsigned char* )malloc(outputTensorDesc[ii].size);
        if (outputBuffer[ii] == NULL) {
            DPRINTF("Error in allocating output memory\n");
            cleanUp(&resourceList);
            return 1;
        }
        memset(outputBuffer[ii], 0x00, outputTensorDesc[ii].size);
    }

    unsigned char** statisticsOutputBuffer = (unsigned char **)malloc(sizeof(unsigned char *)*numOutputTaskStatistics);
    if (statisticsOutputBuffer == NULL) {
        DPRINTF("Error in allocating memory for output buffer array\n");
        cleanUp(&resourceList);
        return 1;
    }
    memset(statisticsOutputBuffer, 0x00, sizeof(unsigned char *)*numOutputTaskStatistics);
    resourceList.statisticsOutputBuffer = statisticsOutputBuffer;

    for (uint32_t ii = 0; ii < numOutputTaskStatistics; ii++) {
        statisticsOutputBuffer[ii] = (unsigned char* )malloc(outputTaskStatisticsDesc[ii].size);
        if (outputBuffer[ii] == NULL) {
            DPRINTF("Error in allocating output memory\n");
            cleanUp(&resourceList);
            return 1;
        }
        memset(statisticsOutputBuffer[ii], 0x00, outputTaskStatisticsDesc[ii].size);
    }

    // Allocate memory on GPU.
    void** inputBufferGPU = (void **)malloc(sizeof(void *)*numInputTensors);
    if (inputBufferGPU == NULL) {
        DPRINTF("Error in allocating memory for input buffer GPU array\n");
        cleanUp(&resourceList);
        return 1;
    }
    memset(inputBufferGPU, 0x00, sizeof(void *)*numInputTensors);
    resourceList.inputBufferGPU = inputBufferGPU;

    for (uint32_t ii = 0; ii < numInputTensors; ii++) {
        result = hipMalloc(&(inputBufferGPU[ii]), inputTensorDesc[ii].size);
        if (result != hipSuccess)
        {
            DPRINTF("Error in allocating input memory on GPU\n");
            cleanUp(&resourceList);
            return 1;
        }
    }

    void** outputBufferGPU = (void **)malloc(sizeof(void *)*numOutputTensors);
    if (outputBufferGPU == NULL) {
        DPRINTF("Error in allocating memory for output buffer GPU array\n");
        cleanUp(&resourceList);
        return 1;
    }
    memset(outputBufferGPU, 0x00, sizeof(void *)*numOutputTensors);
    resourceList.outputBufferGPU = outputBufferGPU;

    for (uint32_t ii = 0; ii < numOutputTensors; ii++) {
        result = hipMalloc(&(outputBufferGPU[ii]), outputTensorDesc[ii].size);
        if (result != hipSuccess) {
            DPRINTF("Error in allocating output memory on GPU\n");
            cleanUp(&resourceList);
            return 1;
        }
    }

    void** outputTaskStatisticsGPU = (void **)malloc(sizeof(void *)*numOutputTaskStatistics);
    if (outputTaskStatisticsGPU == NULL) {
        DPRINTF("Error in allocating memory for output task statistics GPU array\n");
        cleanUp(&resourceList);
        return 1;
    }
    memset(outputTaskStatisticsGPU, 0x00, sizeof(void *)*numOutputTaskStatistics);
    resourceList.outputTaskStatisticsGPU = outputTaskStatisticsGPU;

    for (uint32_t ii = 0; ii < numOutputTaskStatistics; ii++) {
        result = hipMalloc(&(outputTaskStatisticsGPU[ii]), outputTaskStatisticsDesc[ii].size);
        if (result != hipSuccess) {
            DPRINTF("Error in allocating task statistics memory on GPU\n");
            cleanUp(&resourceList);
            return 1;
        }
    }

    uint64_t** inputBufferRegisteredPtr = (uint64_t **)malloc(sizeof(uint64_t*)*numInputTensors);
    uint64_t** outputBufferRegisteredPtr = (uint64_t **)malloc(sizeof(uint64_t*)*numOutputTensors);
    uint64_t** outputTaskStatisticsRegisteredPtr = (uint64_t **)malloc(sizeof(uint64_t*)*numOutputTaskStatistics);

    if ((inputBufferRegisteredPtr == NULL) || (outputBufferRegisteredPtr == NULL) || (outputTaskStatisticsRegisteredPtr == NULL)) {
        if (inputBufferRegisteredPtr != NULL) {
            free(inputBufferRegisteredPtr);
            inputBufferRegisteredPtr = NULL;
        }

        if (outputBufferRegisteredPtr != NULL) {
            free(outputBufferRegisteredPtr);
            outputBufferRegisteredPtr = NULL;
        }

        if (outputTaskStatisticsRegisteredPtr != NULL) {
            free(outputTaskStatisticsRegisteredPtr);
            outputTaskStatisticsRegisteredPtr = NULL;
        }

        cleanUp(&resourceList);
        return 1;
    }

    resourceList.inputBufferRegisteredPtr = inputBufferRegisteredPtr;
    resourceList.outputBufferRegisteredPtr = outputBufferRegisteredPtr;
    resourceList.outputTaskStatisticsRegisteredPtr = outputTaskStatisticsRegisteredPtr;

    // Register the CUDA-allocated buffers.
    for (uint32_t ii = 0; ii < numInputTensors; ii++) {
        err = cudlaMemRegister(devHandle,
                               (uint64_t* )(inputBufferGPU[ii]),
                               inputTensorDesc[ii].size,
                               &(inputBufferRegisteredPtr[ii]),
                               0);
        if (err != cudlaSuccess) {
            DPRINTF("Error in registering input memory = %d\n", err);
            cleanUp(&resourceList);
            return 1;
        }
    }

    for (uint32_t ii = 0; ii < numOutputTensors; ii++) {
        err = cudlaMemRegister(devHandle,
                               (uint64_t* )(outputBufferGPU[ii]),
                               outputTensorDesc[ii].size,
                               &(outputBufferRegisteredPtr[ii]),
                               0);
        if (err != cudlaSuccess) {
            DPRINTF("Error in registering output memory = %d\n", err);
            cleanUp(&resourceList);
            return 1;
        }
    }

    for (uint32_t ii = 0; ii < numOutputTaskStatistics; ii++) {
        err = cudlaMemRegister(devHandle,
                               (uint64_t* )(outputTaskStatisticsGPU[ii]),
                               outputTaskStatisticsDesc[ii].size,
                               &(outputTaskStatisticsRegisteredPtr[ii]),
                               CUDLA_TASK_STATISTICS);
        if (err != cudlaSuccess) {
            DPRINTF("Error in registering statistics output memory = %d\n", err);
            cleanUp(&resourceList);
            return 1;
        }
    }

    DPRINTF("ALL MEMORY REGISTERED SUCCESSFULLY\n");

    // Copy data from CPU buffers to GPU buffers.
    for (uint32_t ii = 0; ii < numInputTensors; ii++) {
        result = hipMemcpyAsync(inputBufferGPU[ii], inputBuffer[ii], inputTensorDesc[ii].size, hipMemcpyHostToDevice, stream);
        if (result != hipSuccess) {
            DPRINTF("Error in enqueueing memcpy for input\n");
            cleanUp(&resourceList);
            return 1;
        }
    }

    for (uint32_t ii = 0; ii < numOutputTensors; ii++) {
        result = hipMemsetAsync(outputBufferGPU[ii], 0, outputTensorDesc[ii].size, stream);
        if (result != hipSuccess) {
            DPRINTF("Error in enqueueing memset for output\n");
            cleanUp(&resourceList);
            return 1;
        }
    }

    for (uint32_t ii = 0; ii < numOutputTaskStatistics; ii++) {
        result = hipMemsetAsync(outputTaskStatisticsGPU[ii], 0, outputTaskStatisticsDesc[ii].size, stream);
        if (result != hipSuccess) {
            DPRINTF("Error in enqueueing memset for statistics output\n");
            cleanUp(&resourceList);
            return 1;
        }
    }

    uint64_t *outputStatisticsBufferRegisteredPtr[numOutputTensors + numOutputTaskStatistics] = {0};
    uint32_t index = 0;
    for (; index < numOutputTensors ; index++) {
        outputStatisticsBufferRegisteredPtr[index] = ((outputBufferRegisteredPtr[index]));
    }

    for (uint32_t jj=0; jj < numOutputTaskStatistics ; jj++) {
        outputStatisticsBufferRegisteredPtr[index++] = ((outputTaskStatisticsRegisteredPtr[jj]));
    }

    // Enqueue a cuDLA task.
    cudlaTask task;
    task.moduleHandle = moduleHandle;
    task.outputTensor = (uint64_t * const*)&outputStatisticsBufferRegisteredPtr;

    if(statSupport == 1) {
        task.numOutputTensors = (numOutputTensors + numOutputTaskStatistics);
        DPRINTF("Layerwise profiling is requested \n");
    } else {
      task.numOutputTensors = numOutputTensors;
      DPRINTF("Layerwise profiling is not requested \n");
    }

    task.numInputTensors = numInputTensors;
    task.inputTensor = inputBufferRegisteredPtr;
    task.waitEvents = NULL;
    task.signalEvents = NULL;

    err = cudlaSubmitTask(devHandle, &task, 1, stream, 0);
    if (err != cudlaSuccess) {
        DPRINTF("no of output tensor %u \n",(task.numOutputTensors));
        DPRINTF("Error in submitting task\n");
        cleanUp(&resourceList);
        return 1;
    }
    DPRINTF("SUBMIT IS DONE !!!\n");

    result = hipStreamSynchronize(stream);
    if (result != hipSuccess) {
        DPRINTF("Error in synchronizing stream = %s\n", hipGetErrorName(result));
        cleanUp(&resourceList);
        return 1;
    }

    // Wait for stream operations to finish and bring output buffer to CPU.
    for (uint32_t ii = 0; ii < numOutputTensors; ii++) {
        result = hipMemcpyAsync(outputBuffer[ii], outputBufferGPU[ii],
                                 outputTensorDesc[ii].size, hipMemcpyDeviceToHost, stream);
        if (result != hipSuccess) {
            DPRINTF("Error in bringing result back to CPU\n");
            cleanUp(&resourceList);
            return 1;
        }
    }

    result = hipStreamSynchronize(stream);
    if (result != hipSuccess) {
        DPRINTF("Error in synchronizing stream\n");
        cleanUp(&resourceList);
        return 1;
    }

    if(statSupport == 1) {
        // copy statistics data to cpu
        for (uint32_t ii = 0; ii < numOutputTaskStatistics; ii++) {
            result = hipMemcpyAsync(statisticsOutputBuffer[ii], outputTaskStatisticsGPU[ii],
                                     outputTaskStatisticsDesc[ii].size, hipMemcpyDeviceToHost, stream);
            if (result != hipSuccess) {
                DPRINTF("Error in bringing result back to CPU\n");
                cleanUp(&resourceList);
                return 1;
            }
        }

        result = hipStreamSynchronize(stream);
        if (result != hipSuccess) {
            DPRINTF("Error in synchronizing stream\n");
            cleanUp(&resourceList);
            return 1;
        }

        // To get the last index of the filename prefix in which statistics will be dumped
        uint32_t index = 0;
        if (argc == 5) {
            while(argv[4][index]!='\0') {
                index++;
            }
        }

        const cudlaExternalEtbl* etbl = NULL;
        if (cudlaGetExternalExportTable(&etbl,0) != cudlaSuccess) {
            DPRINTF("Error in getting export table\n");
            cleanUp(&resourceList);
            return 1;
        }

        void** csv = (void **)malloc(sizeof(void *)*numOutputTaskStatistics);
        if (csv == NULL) {
            DPRINTF("Error in allocating memory for csv stream\n");
            cleanUp(&resourceList);
            return 1;
        }
        memset(csv, 0x00, sizeof(void *)*numOutputTaskStatistics);
        resourceList.csv = csv;

        for (uint32_t ii = 0; ii < numOutputTaskStatistics; ii++) {
            cudlaTranslateCsvAttribute csvAttribute;
            uint64_t csvStreamLength = 0;

            err = etbl->etiTranslateStats(devHandle,statisticsOutputBuffer[ii],dlaFreqInMHz,ii,CUDLA_GET_CSV_LENGTH,&csvAttribute);
            csv[ii] = (void* )malloc(csvAttribute.csvStreamLength);
            csvStreamLength = csvAttribute.csvStreamLength;
            DPRINTF("size for statistics buffer %u is %lu \n",ii,csvStreamLength);

            if (csv[ii] == NULL) {
                DPRINTF("Error in allocating memory for csv stream\n");
                cleanUp(&resourceList);
                return 1;
            }
            memset(csv[ii], 0x00, csvAttribute.csvStreamLength);

            csvAttribute.csvStreamStats = csv[ii];
            err = etbl->etiTranslateStats(devHandle,statisticsOutputBuffer[ii],dlaFreqInMHz,ii,CUDLA_GET_CSV_STATS,&csvAttribute);
            if (err != cudlaSuccess) {
                DPRINTF("Error in translating stats\n");
                cleanUp(&resourceList);
                return 1;
            }

            if (argc == 5) {
                sprintf(filename,"%s%u%s", argv[4],(ii+1),suffix);
                fp = fopen(filename, "w+");
                if (fp == NULL) {
                    DPRINTF("Cannot open file %s\n", filename);
                    cleanUp(&resourceList);
                    return 1;
                }

                uint32_t ret_val = fwrite(csv[ii],sizeof(char),csvStreamLength,fp);
                if(ret_val != csvStreamLength) {
                    DPRINTF("number of elements written to file is %u \n", ret_val);
                    cleanUp(&resourceList);
                    return 1;
                }
                fclose(fp);
            } else {
              DPRINTF("%s \n",(char *)csv[ii]);
            }
        }
    }

    // unregister the CUDA-allocated buffers.
    for (uint32_t ii = 0; ii < numInputTensors; ii++) {
        err = cudlaMemUnregister(devHandle,
                                 (inputBufferRegisteredPtr[ii]));
        if (err != cudlaSuccess) {
            DPRINTF("Error in registering input memory = %d\n", err);
            cleanUp(&resourceList);
            return 1;
        }
    }

    for (uint32_t ii = 0; ii < numOutputTensors; ii++) {
        err = cudlaMemUnregister(devHandle,
                                 (outputBufferRegisteredPtr[ii]));
        if (err != cudlaSuccess) {
            DPRINTF("Error in registering output memory = %d\n", err);
            cleanUp(&resourceList);
            return 1;
        }
    }

    for (uint32_t ii = 0; ii < numOutputTaskStatistics; ii++) {
        err = cudlaMemUnregister(devHandle,
                                 (outputTaskStatisticsRegisteredPtr[ii]));
        if (err != cudlaSuccess) {
            DPRINTF("Error in registering output memory = %d\n", err);
            cleanUp(&resourceList);
            return 1;
        }
    }

    DPRINTF("ALL MEMORY UNREGISTERED SUCCESSFULLY\n");

    result = hipStreamDestroy(stream);
    if (result != hipSuccess) {
        errPtr = hipGetErrorName(result);
        DPRINTF("Error in destroying cuda stream = %s\n", errPtr);
        cleanUp(&resourceList);
        return 1;
    }

    resourceList.stream = NULL;

    err = cudlaModuleUnload(moduleHandle, 0);
    if (err != cudlaSuccess) {
        DPRINTF("Error in cudlaModuleUnload = %d\n", err);
        cleanUp(&resourceList);
        return 1;
    } else {
        DPRINTF("Successfully unloaded module\n");
    }

    resourceList.moduleHandle = NULL;

    err = cudlaDestroyDevice(devHandle);
    if (err != cudlaSuccess) {
        DPRINTF("Error in cuDLA destroy device = %d\n", err);
        return 1;
    }
    DPRINTF("Device destroyed successfully\n");

    resourceList.devHandle = NULL;

    cleanUp(&resourceList);

    DPRINTF("cuDLALayerwiseStatsHybrid DONE !!!\n");

    return 0;
}
