/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* This example demonstrates how to use the CUDA Direct3D bindings with the
 * runtime API.
 * Device code.
 */

#ifndef _SIMPLED3D_KERNEL_CU_
#define _SIMPLED3D_KERNEL_CU_

// includes, C string library
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include <hip/hip_runtime_api.h>

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param pos  pos in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void kernel(float4 *pos, unsigned int width, unsigned int height,
                       float time) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  // calculate uv coordinates
  float u = x / (float)width;
  float v = y / (float)height;
  u = u * 2.0f - 1.0f;
  v = v * 2.0f - 1.0f;

  // calculate simple sine wave pattern
  float freq = 4.0f;
  float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

  // write output vertex
  pos[y * width + x] = make_float4(u, w, v, __int_as_float(0xff00ff00));
}

extern "C" void simpleD3DKernel(float4 *pos, unsigned int width,
                                unsigned int height, float time) {
  hipError_t error = hipSuccess;

  dim3 block(8, 8, 1);
  dim3 grid(width / block.x, height / block.y, 1);

  kernel<<<grid, block>>>(pos, width, height, time);

  error = hipGetLastError();

  if (error != hipSuccess) {
    printf("kernel() failed to launch error = %d\n", error);
  }
}

#endif  // #ifndef _SIMPLED3D_KERNEL_CU_
