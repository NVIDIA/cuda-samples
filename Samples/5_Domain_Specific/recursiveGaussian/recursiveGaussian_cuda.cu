#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
  Recursive Gaussian filter
  sgreen 8/1/08

  This code sample implements a Gaussian blur using Deriche's recursive method:
  http://citeseer.ist.psu.edu/deriche93recursively.html

  This is similar to the box filter sample in the SDK, but it uses the previous
  outputs of the filter as well as the previous inputs. This is also known as an
  IIR (infinite impulse response) filter, since its response to an input impulse
  can last forever.

  The main advantage of this method is that the execution time is independent of
  the filter width.

  The GPU processes columns of the image in parallel. To avoid uncoalesced reads
  for the row pass we transpose the image and then transpose it back again
  afterwards.

  The implementation is based on code from the CImg library:
  http://cimg.sourceforge.net/
  Thanks to David Tschumperl� and all the CImg contributors!
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

#include "recursiveGaussian_kernel.cuh"

#define USE_SIMPLE_FILTER 0

// Round a / b to nearest higher integer value
int iDivUp(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

/*
  Transpose a 2D array (see SDK transpose example)
*/
extern "C" void transpose(uint *d_src, uint *d_dest, uint width, int height) {
  dim3 grid(iDivUp(width, BLOCK_DIM), iDivUp(height, BLOCK_DIM), 1);
  dim3 threads(BLOCK_DIM, BLOCK_DIM, 1);
  d_transpose<<<grid, threads>>>(d_dest, d_src, width, height);
  getLastCudaError("Kernel execution failed");
}

/*
  Perform Gaussian filter on a 2D image using CUDA

  Parameters:
  d_src  - pointer to input image in device memory
  d_dest - pointer to destination image in device memory
  d_temp - pointer to temporary storage in device memory
  width  - image width
  height - image height
  sigma  - sigma of Gaussian
  order  - filter order (0, 1 or 2)
*/

// 8-bit RGBA version
extern "C" void gaussianFilterRGBA(uint *d_src, uint *d_dest, uint *d_temp,
                                   int width, int height, float sigma,
                                   int order, int nthreads) {
  // compute filter coefficients
  const float nsigma = sigma < 0.1f ? 0.1f : sigma, alpha = 1.695f / nsigma,
              ema = (float)std::exp(-alpha), ema2 = (float)std::exp(-2 * alpha),
              b1 = -2 * ema, b2 = ema2;

  float a0 = 0, a1 = 0, a2 = 0, a3 = 0, coefp = 0, coefn = 0;

  switch (order) {
    case 0: {
      const float k = (1 - ema) * (1 - ema) / (1 + 2 * alpha * ema - ema2);
      a0 = k;
      a1 = k * (alpha - 1) * ema;
      a2 = k * (alpha + 1) * ema;
      a3 = -k * ema2;
    } break;

    case 1: {
      const float k = (1 - ema) * (1 - ema) / ema;
      a0 = k * ema;
      a1 = a3 = 0;
      a2 = -a0;
    } break;

    case 2: {
      const float ea = (float)std::exp(-alpha),
                  k = -(ema2 - 1) / (2 * alpha * ema),
                  kn = (-2 * (-1 + 3 * ea - 3 * ea * ea + ea * ea * ea) /
                        (3 * ea + 1 + 3 * ea * ea + ea * ea * ea));
      a0 = kn;
      a1 = -kn * (1 + k * alpha) * ema;
      a2 = kn * (1 - k * alpha) * ema;
      a3 = -kn * ema2;
    } break;

    default:
      fprintf(stderr, "gaussianFilter: invalid order parameter!\n");
      return;
  }

  coefp = (a0 + a1) / (1 + b1 + b2);
  coefn = (a2 + a3) / (1 + b1 + b2);

// process columns
#if USE_SIMPLE_FILTER
  d_simpleRecursive_rgba<<<iDivUp(width, nthreads), nthreads>>>(
      d_src, d_temp, width, height, ema);
#else
  d_recursiveGaussian_rgba<<<iDivUp(width, nthreads), nthreads>>>(
      d_src, d_temp, width, height, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
  getLastCudaError("Kernel execution failed");

  transpose(d_temp, d_dest, width, height);
  getLastCudaError("transpose: Kernel execution failed");

// process rows
#if USE_SIMPLE_FILTER
  d_simpleRecursive_rgba<<<iDivUp(height, nthreads), nthreads>>>(
      d_dest, d_temp, height, width, ema);
#else
  d_recursiveGaussian_rgba<<<iDivUp(height, nthreads), nthreads>>>(
      d_dest, d_temp, height, width, a0, a1, a2, a3, b1, b2, coefp, coefn);
#endif
  getLastCudaError("Kernel execution failed");

  transpose(d_temp, d_dest, height, width);
}
