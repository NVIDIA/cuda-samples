#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

////////////////////////////////////////////////////////////////////////////////
// Global types
////////////////////////////////////////////////////////////////////////////////
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include "MonteCarlo_common.h"

////////////////////////////////////////////////////////////////////////////////
// Helper reduction template
// Please see the "reduction" CUDA Sample for more information
////////////////////////////////////////////////////////////////////////////////
#include "MonteCarlo_reduction.cuh"

////////////////////////////////////////////////////////////////////////////////
// Internal GPU-side data structures
////////////////////////////////////////////////////////////////////////////////
#define MAX_OPTIONS (1024 * 1024)

// Preprocessed input option data
typedef struct {
  real S;
  real X;
  real MuByT;
  real VBySqrtT;
} __TOptionData;

////////////////////////////////////////////////////////////////////////////////
// Overloaded shortcut payoff functions for different precision modes
////////////////////////////////////////////////////////////////////////////////
__device__ inline float endCallValue(float S, float X, float r, float MuByT,
                                     float VBySqrtT) {
  float callValue = S * __expf(MuByT + VBySqrtT * r) - X;
  return (callValue > 0.0F) ? callValue : 0.0F;
}

__device__ inline double endCallValue(double S, double X, double r,
                                      double MuByT, double VBySqrtT) {
  double callValue = S * exp(MuByT + VBySqrtT * r) - X;
  return (callValue > 0.0) ? callValue : 0.0;
}

#define THREAD_N 256

////////////////////////////////////////////////////////////////////////////////
// This kernel computes the integral over all paths using a single thread block
// per option. It is fastest when the number of thread blocks times the work per
// block is high enough to keep the GPU busy.
////////////////////////////////////////////////////////////////////////////////
static __global__ void MonteCarloOneBlockPerOption(
    hiprandState *__restrict rngStates,
    const __TOptionData *__restrict d_OptionData,
    __TOptionValue *__restrict d_CallValue, int pathN, int optionN) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  const int SUM_N = THREAD_N;
  __shared__ real s_SumCall[SUM_N];
  __shared__ real s_Sum2Call[SUM_N];

  // determine global thread id
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // Copy random number state to local memory for efficiency
  hiprandState localState = rngStates[tid];
  for (int optionIndex = blockIdx.x; optionIndex < optionN;
       optionIndex += gridDim.x) {
    const real S = d_OptionData[optionIndex].S;
    const real X = d_OptionData[optionIndex].X;
    const real MuByT = d_OptionData[optionIndex].MuByT;
    const real VBySqrtT = d_OptionData[optionIndex].VBySqrtT;

    // Cycle through the entire samples array:
    // derive end stock price for each path
    // accumulate partial integrals into intermediate shared memory buffer
    for (int iSum = threadIdx.x; iSum < SUM_N; iSum += blockDim.x) {
      __TOptionValue sumCall = {0, 0};

#pragma unroll 8
      for (int i = iSum; i < pathN; i += SUM_N) {
        real r = hiprand_normal(&localState);
        real callValue = endCallValue(S, X, r, MuByT, VBySqrtT);
        sumCall.Expected += callValue;
        sumCall.Confidence += callValue * callValue;
      }

      s_SumCall[iSum] = sumCall.Expected;
      s_Sum2Call[iSum] = sumCall.Confidence;
    }

    // Reduce shared memory accumulators
    // and write final result to global memory
    cg::sync(cta);
    sumReduce<real, SUM_N, THREAD_N>(s_SumCall, s_Sum2Call, cta, tile32,
                                     &d_CallValue[optionIndex]);
  }
}

static __global__ void rngSetupStates(hiprandState *rngState, int device_id) {
  // determine global thread id
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  // Each threadblock gets different seed,
  // Threads within a threadblock get different sequence numbers
  hiprand_init(blockIdx.x + gridDim.x * device_id, threadIdx.x, 0,
              &rngState[tid]);
}

////////////////////////////////////////////////////////////////////////////////
// Host-side interface to GPU Monte Carlo
////////////////////////////////////////////////////////////////////////////////

extern "C" void initMonteCarloGPU(TOptionPlan *plan) {
  checkCudaErrors(hipMalloc(&plan->d_OptionData,
                             sizeof(__TOptionData) * (plan->optionCount)));
  checkCudaErrors(hipMalloc(&plan->d_CallValue,
                             sizeof(__TOptionValue) * (plan->optionCount)));
  checkCudaErrors(hipHostMalloc(&plan->h_OptionData,
                                 sizeof(__TOptionData) * (plan->optionCount)));
  // Allocate internal device memory
  checkCudaErrors(hipHostMalloc(&plan->h_CallValue,
                                 sizeof(__TOptionValue) * (plan->optionCount)));
  // Allocate states for pseudo random number generators
  checkCudaErrors(hipMalloc((void **)&plan->rngStates,
                             plan->gridSize * THREAD_N * sizeof(hiprandState)));
  checkCudaErrors(hipMemset(plan->rngStates, 0,
                             plan->gridSize * THREAD_N * sizeof(hiprandState)));

  // place each device pathN random numbers apart on the random number sequence
  rngSetupStates<<<plan->gridSize, THREAD_N>>>(plan->rngStates, plan->device);
  getLastCudaError("rngSetupStates kernel failed.\n");
}

// Compute statistics and deallocate internal device memory
extern "C" void closeMonteCarloGPU(TOptionPlan *plan) {
  for (int i = 0; i < plan->optionCount; i++) {
    const double RT = plan->optionData[i].R * plan->optionData[i].T;
    const double sum = plan->h_CallValue[i].Expected;
    const double sum2 = plan->h_CallValue[i].Confidence;
    const double pathN = plan->pathN;
    // Derive average from the total sum and discount by riskfree rate
    plan->callValue[i].Expected = (float)(exp(-RT) * sum / pathN);
    // Standard deviation
    double stdDev = sqrt((pathN * sum2 - sum * sum) / (pathN * (pathN - 1)));
    // Confidence width; in 95% of all cases theoretical value lies within these
    // borders
    plan->callValue[i].Confidence =
        (float)(exp(-RT) * 1.96 * stdDev / sqrt(pathN));
  }

  checkCudaErrors(hipFree(plan->rngStates));
  checkCudaErrors(hipHostFree(plan->h_CallValue));
  checkCudaErrors(hipHostFree(plan->h_OptionData));
  checkCudaErrors(hipFree(plan->d_CallValue));
  checkCudaErrors(hipFree(plan->d_OptionData));
}

// Main computations
extern "C" void MonteCarloGPU(TOptionPlan *plan, hipStream_t stream) {
  __TOptionValue *h_CallValue = plan->h_CallValue;

  if (plan->optionCount <= 0 || plan->optionCount > MAX_OPTIONS) {
    printf("MonteCarloGPU(): bad option count.\n");
    return;
  }

  __TOptionData *h_OptionData = (__TOptionData *)plan->h_OptionData;

  for (int i = 0; i < plan->optionCount; i++) {
    const double T = plan->optionData[i].T;
    const double R = plan->optionData[i].R;
    const double V = plan->optionData[i].V;
    const double MuByT = (R - 0.5 * V * V) * T;
    const double VBySqrtT = V * sqrt(T);
    h_OptionData[i].S = (real)plan->optionData[i].S;
    h_OptionData[i].X = (real)plan->optionData[i].X;
    h_OptionData[i].MuByT = (real)MuByT;
    h_OptionData[i].VBySqrtT = (real)VBySqrtT;
  }

  checkCudaErrors(hipMemcpyAsync(plan->d_OptionData, h_OptionData,
                                  plan->optionCount * sizeof(__TOptionData),
                                  hipMemcpyHostToDevice, stream));

  MonteCarloOneBlockPerOption<<<plan->gridSize, THREAD_N, 0, stream>>>(
      plan->rngStates, (__TOptionData *)(plan->d_OptionData),
      (__TOptionValue *)(plan->d_CallValue), plan->pathN, plan->optionCount);
  getLastCudaError("MonteCarloOneBlockPerOption() execution failed\n");

  checkCudaErrors(hipMemcpyAsync(h_CallValue, plan->d_CallValue,
                                  plan->optionCount * sizeof(__TOptionValue),
                                  hipMemcpyDeviceToHost, stream));

  // hipDeviceSynchronize();
}
