#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "SineWaveSimulation.h"
#include <algorithm>
#include <hip/hip_runtime_api.h>

__global__ void sinewave(float *heightMap, unsigned int width,
                         unsigned int height, float time) {
  const float freq = 4.0f;
  const size_t stride = gridDim.x * blockDim.x;

  // Iterate through the entire array in a way that is
  // independent of the grid configuration
  for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < width * height;
       tid += stride) {
    // Calculate the x, y coordinates
    const size_t y = tid / width;
    const size_t x = tid - y * width;
    // Normalize x, y to [0,1]
    const float u = ((2.0f * x) / width) - 1.0f;
    const float v = ((2.0f * y) / height) - 1.0f;
    // Calculate the new height value
    const float w = 0.5f * sinf(u * freq + time) * cosf(v * freq + time);
    // Store this new height value
    heightMap[tid] = w;
  }
}

SineWaveSimulation::SineWaveSimulation(size_t width, size_t height)
    : m_heightMap(nullptr), m_width(width), m_height(height) {}

void SineWaveSimulation::initCudaLaunchConfig(int device) {
  hipDeviceProp_t prop = {};
  checkCudaErrors(hipSetDevice(device));
  checkCudaErrors(hipGetDeviceProperties(&prop, device));

  // We don't need large block sizes, since there's not much inter-thread
  // communication
  m_threads = prop.warpSize;

  // Use the occupancy calculator and fill the gpu as best as we can
  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &m_blocks, sinewave, prop.warpSize, 0));
  m_blocks *= prop.multiProcessorCount;

  // Go ahead and the clamp the blocks to the minimum needed for this
  // height/width
  m_blocks = std::min(m_blocks,
                      (int)((m_width * m_height + m_threads - 1) / m_threads));
}

int SineWaveSimulation::initCuda(uint8_t *vkDeviceUUID, size_t UUID_SIZE) {
  int current_device = 0;
  int device_count = 0;
  int devices_prohibited = 0;

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDeviceCount(&device_count));

  if (device_count == 0) {
    fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
    exit(EXIT_FAILURE);
  }

  // Find the GPU which is selected by Vulkan
  while (current_device < device_count) {
    hipGetDeviceProperties(&deviceProp, current_device);

    if ((deviceProp.computeMode != hipComputeModeProhibited)) {
      // Compare the cuda device UUID with vulkan UUID
      int ret = memcmp((void *)&deviceProp.uuid, vkDeviceUUID, UUID_SIZE);
      if (ret == 0) {
        checkCudaErrors(hipSetDevice(current_device));
        checkCudaErrors(hipGetDeviceProperties(&deviceProp, current_device));
        printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
               current_device, deviceProp.name, deviceProp.major,
               deviceProp.minor);

        return current_device;
      }

    } else {
      devices_prohibited++;
    }

    current_device++;
  }

  if (devices_prohibited == device_count) {
    fprintf(stderr,
            "CUDA error:"
            " No Vulkan-CUDA Interop capable GPU found.\n");
    exit(EXIT_FAILURE);
  }

  return -1;
}

SineWaveSimulation::~SineWaveSimulation() { m_heightMap = NULL; }

void SineWaveSimulation::initSimulation(float *heights) {
  m_heightMap = heights;
}

void SineWaveSimulation::stepSimulation(float time, hipStream_t stream) {
  sinewave<<<m_blocks, m_threads, 0, stream>>>(m_heightMap, m_width, m_height,
                                               time);
  getLastCudaError("Failed to launch CUDA simulation");
}
