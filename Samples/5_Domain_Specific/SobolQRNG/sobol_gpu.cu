#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Portions Copyright (c) 2009 Mike Giles, Oxford University.  All rights
 * reserved.
 * Portions Copyright (c) 2008 Frances Y. Kuo and Stephen Joe.  All rights
 * reserved.
 *
 * Sobol Quasi-random Number Generator example
 *
 * Based on CUDA code submitted by Mike Giles, Oxford University, United Kingdom
 * http://people.maths.ox.ac.uk/~gilesm/
 *
 * and C code developed by Stephen Joe, University of Waikato, New Zealand
 * and Frances Kuo, University of New South Wales, Australia
 * http://web.maths.unsw.edu.au/~fkuo/sobol/
 *
 * For theoretical background see:
 *
 * P. Bratley and B.L. Fox.
 * Implementing Sobol's quasirandom sequence generator
 * http://portal.acm.org/citation.cfm?id=42288
 * ACM Trans. on Math. Software, 14(1):88-100, 1988
 *
 * S. Joe and F. Kuo.
 * Remark on algorithm 659: implementing Sobol's quasirandom sequence generator.
 * http://portal.acm.org/citation.cfm?id=641879
 * ACM Trans. on Math. Software, 29(1):49-57, 2003
 *
 */

#include "sobol.h"
#include "sobol_gpu.h"
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>

#define k_2powneg32 2.3283064E-10F

__global__ void sobolGPU_kernel(unsigned n_vectors, unsigned n_dimensions,
                                unsigned *d_directions, float *d_output) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ unsigned int v[n_directions];

  // Offset into the correct dimension as specified by the
  // block y coordinate
  d_directions = d_directions + n_directions * blockIdx.y;
  d_output = d_output + n_vectors * blockIdx.y;

  // Copy the direction numbers for this dimension into shared
  // memory - there are only 32 direction numbers so only the
  // first 32 (n_directions) threads need participate.
  if (threadIdx.x < n_directions) {
    v[threadIdx.x] = d_directions[threadIdx.x];
  }

  cg::sync(cta);

  // Set initial index (i.e. which vector this thread is
  // computing first) and stride (i.e. step to the next vector
  // for this thread)
  int i0 = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = gridDim.x * blockDim.x;

  // Get the gray code of the index
  // c.f. Numerical Recipes in C, chapter 20
  // http://www.nrbook.com/a/bookcpdf/c20-2.pdf
  unsigned int g = i0 ^ (i0 >> 1);

  // Initialisation for first point x[i0]
  // In the Bratley and Fox paper this is equation (*), where
  // we are computing the value for x[n] without knowing the
  // value of x[n-1].
  unsigned int X = 0;
  unsigned int mask;

  for (unsigned int k = 0; k < __ffs(stride) - 1; k++) {
    // We want X ^= g_k * v[k], where g_k is one or zero.
    // We do this by setting a mask with all bits equal to
    // g_k. In reality we keep shifting g so that g_k is the
    // LSB of g. This way we avoid multiplication.
    mask = -(g & 1);
    X ^= mask & v[k];
    g = g >> 1;
  }

  if (i0 < n_vectors) {
    d_output[i0] = (float)X * k_2powneg32;
  }

  // Now do rest of points, using the stride
  // Here we want to generate x[i] from x[i-stride] where we
  // don't have any of the x in between, therefore we have to
  // revisit the equation (**), this is easiest with an example
  // so assume stride is 16.
  // From x[n] to x[n+16] there will be:
  //   8 changes in the first bit
  //   4 changes in the second bit
  //   2 changes in the third bit
  //   1 change in the fourth
  //   1 change in one of the remaining bits
  //
  // What this means is that in the equation:
  //   x[n+1] = x[n] ^ v[p]
  //   x[n+2] = x[n+1] ^ v[q] = x[n] ^ v[p] ^ v[q]
  //   ...
  // We will apply xor with v[1] eight times, v[2] four times,
  // v[3] twice, v[4] once and one other direction number once.
  // Since two xors cancel out, we can skip even applications
  // and just apply xor with v[4] (i.e. log2(16)) and with
  // the current applicable direction number.
  // Note that all these indices count from 1, so we need to
  // subtract 1 from them all to account for C arrays counting
  // from zero.
  unsigned int v_log2stridem1 = v[__ffs(stride) - 2];
  unsigned int v_stridemask = stride - 1;

  for (unsigned int i = i0 + stride; i < n_vectors; i += stride) {
    // x[i] = x[i-stride] ^ v[b] ^ v[c]
    //  where b is log2(stride) minus 1 for C array indexing
    //  where c is the index of the rightmost zero bit in i,
    //  not including the bottom log2(stride) bits, minus 1
    //  for C array indexing
    // In the Bratley and Fox paper this is equation (**)
    X ^= v_log2stridem1 ^ v[__ffs(~((i - stride) | v_stridemask)) - 1];
    d_output[i] = (float)X * k_2powneg32;
  }
}

extern "C" void sobolGPU(int n_vectors, int n_dimensions,
                         unsigned int *d_directions, float *d_output) {
  const int threadsperblock = 64;

  // Set up the execution configuration
  dim3 dimGrid;
  dim3 dimBlock;

  int device;
  hipDeviceProp_t prop;
  checkCudaErrors(hipGetDevice(&device));
  checkCudaErrors(hipGetDeviceProperties(&prop, device));

  // This implementation of the generator outputs all the draws for
  // one dimension in a contiguous region of memory, followed by the
  // next dimension and so on.
  // Therefore all threads within a block will be processing different
  // vectors from the same dimension. As a result we want the total
  // number of blocks to be a multiple of the number of dimensions.
  dimGrid.y = n_dimensions;

  // If the number of dimensions is large then we will set the number
  // of blocks to equal the number of dimensions (i.e. dimGrid.x = 1)
  // but if the number of dimensions is small (e.g. less than four per
  // multiprocessor) then we'll partition the vectors across blocks
  // (as well as threads).
  if (n_dimensions < (4 * prop.multiProcessorCount)) {
    dimGrid.x = 4 * prop.multiProcessorCount;
  } else {
    dimGrid.x = 1;
  }

  // Cap the dimGrid.x if the number of vectors is small
  if (dimGrid.x > (unsigned int)(n_vectors / threadsperblock)) {
    dimGrid.x = (n_vectors + threadsperblock - 1) / threadsperblock;
  }

  // Round up to a power of two, required for the algorithm so that
  // stride is a power of two.
  unsigned int targetDimGridX = dimGrid.x;

  for (dimGrid.x = 1; dimGrid.x < targetDimGridX; dimGrid.x *= 2)
    ;

  // Fix the number of threads
  dimBlock.x = threadsperblock;

  // Execute GPU kernel
  sobolGPU_kernel<<<dimGrid, dimBlock>>>(n_vectors, n_dimensions, d_directions,
                                         d_output);
}
