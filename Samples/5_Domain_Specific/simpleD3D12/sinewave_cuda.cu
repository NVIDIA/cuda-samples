#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "ShaderStructs.h"

__global__ void sinewave_gen_kernel(Vertex *vertices, unsigned int width,
                                    unsigned int height, float time) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  // calculate uv coordinates
  float u = x / (float)width;
  float v = y / (float)height;
  u = u * 2.0f - 1.0f;
  v = v * 2.0f - 1.0f;

  // calculate simple sine wave pattern
  float freq = 4.0f;
  float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

  if (y < height && x < width) {
    // write output vertex
    vertices[y * width + x].position.x = u;
    vertices[y * width + x].position.y = w;
    vertices[y * width + x].position.z = v;
    // vertices[y*width+x].position[3] = 1.0f;
    vertices[y * width + x].color.x = 1.0f;
    vertices[y * width + x].color.y = 0.0f;
    vertices[y * width + x].color.z = 0.0f;
    vertices[y * width + x].color.w = 0.0f;
  }
}

// The host CPU Sinewave thread spawner
void RunSineWaveKernel(size_t mesh_width, size_t mesh_height,
                       Vertex *cudaDevVertptr, hipStream_t streamToRun,
                       float AnimTime) {
  dim3 block(16, 16, 1);
  dim3 grid(mesh_width / 16, mesh_height / 16, 1);
  Vertex *vertices = (Vertex *)cudaDevVertptr;
  sinewave_gen_kernel<<<grid, block, 0, streamToRun>>>(vertices, mesh_width,
                                                       mesh_height, AnimTime);

  getLastCudaError("sinewave_gen_kernel execution failed.\n");
}
