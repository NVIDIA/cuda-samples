#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include "ShaderStructs.h"
#include "hip/hip_runtime_api.h"
#include "sinewave_cuda.h"

__global__ void sinewave_gen_kernel(Vertex *vertices, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float) width;
    float v = y / (float) height;
    u = u*2.0f - 1.0f;
    v = v*2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

    if (y < height && x < width)
    {
        // write output vertex
        vertices[y*width+x].position.x = u;
        vertices[y*width+x].position.y = w;
        vertices[y*width+x].position.z = v;
        vertices[y*width+x].color.x = 1.0f;
        vertices[y*width+x].color.y = 0.0f;
        vertices[y*width+x].color.z = 0.0f;
        vertices[y*width + x].color.w = 0.0f;
    }
}

Vertex* cudaImportVertexBuffer(void*sharedHandle, hipExternalMemory_t &externalMemory, int meshWidth, int meshHeight)
{
    hipExternalMemoryHandleDesc externalMemoryHandleDesc;
    memset(&externalMemoryHandleDesc, 0, sizeof(externalMemoryHandleDesc));

    externalMemoryHandleDesc.type = hipExternalMemoryHandleTypeD3D11ResourceKmt;
    externalMemoryHandleDesc.size = sizeof(Vertex) * meshHeight * meshWidth;
    externalMemoryHandleDesc.flags = hipExternalMemoryDedicated;
    externalMemoryHandleDesc.handle.win32.handle = sharedHandle;

    checkCudaErrors(hipImportExternalMemory(&externalMemory, &externalMemoryHandleDesc));

    hipExternalMemoryBufferDesc externalMemoryBufferDesc;
    memset(&externalMemoryBufferDesc, 0, sizeof(externalMemoryBufferDesc));
    externalMemoryBufferDesc.offset = 0;
    externalMemoryBufferDesc.size = sizeof(Vertex) * meshHeight * meshWidth;
    externalMemoryBufferDesc.flags = 0;

    Vertex* cudaDevVertptr = NULL;
    checkCudaErrors(hipExternalMemoryGetMappedBuffer((void**)&cudaDevVertptr, externalMemory, &externalMemoryBufferDesc));

    return cudaDevVertptr;
}

void cudaImportKeyedMutex(void*sharedHandle, hipExternalSemaphore_t &extSemaphore)
{
    hipExternalSemaphoreHandleDesc extSemaDesc;
    memset(&extSemaDesc, 0, sizeof(extSemaDesc));
    extSemaDesc.type = cudaExternalSemaphoreHandleTypeKeyedMutexKmt;
    extSemaDesc.handle.win32.handle = sharedHandle;
    extSemaDesc.flags = 0;

    checkCudaErrors(hipImportExternalSemaphore(&extSemaphore, &extSemaDesc));
}

void cudaAcquireSync(hipExternalSemaphore_t &extSemaphore, uint64_t key, unsigned int timeoutMs, hipStream_t streamToRun)
{
    hipExternalSemaphoreWaitParams extSemWaitParams;
    memset(&extSemWaitParams, 0, sizeof(extSemWaitParams));
    extSemWaitParams.params.keyedMutex.key = key;
    extSemWaitParams.params.keyedMutex.timeoutMs = timeoutMs;

    checkCudaErrors(hipWaitExternalSemaphoresAsync(&extSemaphore, &extSemWaitParams, 1, streamToRun));
}

void cudaReleaseSync(hipExternalSemaphore_t &extSemaphore, uint64_t key, hipStream_t streamToRun)
{
    hipExternalSemaphoreSignalParams extSemSigParams;
    memset(&extSemSigParams, 0, sizeof(extSemSigParams));
    extSemSigParams.params.keyedMutex.key = key;

    checkCudaErrors(hipSignalExternalSemaphoresAsync(&extSemaphore, &extSemSigParams, 1, streamToRun));
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void RunSineWaveKernel(hipExternalSemaphore_t &extSemaphore, uint64_t &key, unsigned int timeoutMs, 
                        size_t mesh_width, size_t mesh_height, Vertex *cudaDevVertptr, hipStream_t streamToRun)
{
    static float t = 0.0f;
    cudaAcquireSync(extSemaphore, key++, timeoutMs, streamToRun);

    dim3 block(16, 16, 1);
    dim3 grid(mesh_width / 16, mesh_height / 16, 1);
    sinewave_gen_kernel<<< grid, block, 0, streamToRun >>>(cudaDevVertptr, mesh_width, mesh_height, t);
    getLastCudaError("sinewave_gen_kernel execution failed.\n");

    cudaReleaseSync(extSemaphore, key, streamToRun);
    t += 0.01f;
}

