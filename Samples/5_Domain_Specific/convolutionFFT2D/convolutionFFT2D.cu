/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include "convolutionFFT2D_common.h"
#include "convolutionFFT2D.cuh"

////////////////////////////////////////////////////////////////////////////////
/// Position convolution kernel center at (0, 0) in the image
////////////////////////////////////////////////////////////////////////////////
extern "C" void padKernel(float *d_Dst, float *d_Src, int fftH, int fftW,
                          int kernelH, int kernelW, int kernelY, int kernelX) {
  assert(d_Src != d_Dst);
  dim3 threads(32, 8);
  dim3 grid(iDivUp(kernelW, threads.x), iDivUp(kernelH, threads.y));

  SET_FLOAT_BASE;
#if (USE_TEXTURE)
  hipTextureObject_t texFloat;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_Src;
  texRes.res.linear.sizeInBytes = sizeof(float) * kernelH * kernelW;
  texRes.res.linear.desc = hipCreateChannelDesc<float>();

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&texFloat, &texRes, &texDescr, NULL));
#endif

  padKernel_kernel<<<grid, threads>>>(d_Dst, d_Src, fftH, fftW, kernelH,
                                      kernelW, kernelY, kernelX
#if (USE_TEXTURE)
                                      ,
                                      texFloat
#endif
                                      );
  getLastCudaError("padKernel_kernel<<<>>> execution failed\n");

#if (USE_TEXTURE)
  checkCudaErrors(hipDestroyTextureObject(texFloat));
#endif
}

////////////////////////////////////////////////////////////////////////////////
// Prepare data for "pad to border" addressing mode
////////////////////////////////////////////////////////////////////////////////
extern "C" void padDataClampToBorder(float *d_Dst, float *d_Src, int fftH,
                                     int fftW, int dataH, int dataW,
                                     int kernelW, int kernelH, int kernelY,
                                     int kernelX) {
  assert(d_Src != d_Dst);
  dim3 threads(32, 8);
  dim3 grid(iDivUp(fftW, threads.x), iDivUp(fftH, threads.y));

#if (USE_TEXTURE)
  hipTextureObject_t texFloat;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_Src;
  texRes.res.linear.sizeInBytes = sizeof(float) * dataH * dataW;
  texRes.res.linear.desc = hipCreateChannelDesc<float>();

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&texFloat, &texRes, &texDescr, NULL));
#endif

  padDataClampToBorder_kernel<<<grid, threads>>>(
      d_Dst, d_Src, fftH, fftW, dataH, dataW, kernelH, kernelW, kernelY, kernelX
#if (USE_TEXTURE)
      ,
      texFloat
#endif
      );
  getLastCudaError("padDataClampToBorder_kernel<<<>>> execution failed\n");

#if (USE_TEXTURE)
  checkCudaErrors(hipDestroyTextureObject(texFloat));
#endif
}

////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
extern "C" void modulateAndNormalize(fComplex *d_Dst, fComplex *d_Src, int fftH,
                                     int fftW, int padding) {
  assert(fftW % 2 == 0);
  const int dataSize = fftH * (fftW / 2 + padding);

  modulateAndNormalize_kernel<<<iDivUp(dataSize, 256), 256>>>(
      d_Dst, d_Src, dataSize, 1.0f / (float)(fftW * fftH));
  getLastCudaError("modulateAndNormalize() execution failed\n");
}

////////////////////////////////////////////////////////////////////////////////
// 2D R2C / C2R post/preprocessing kernels
////////////////////////////////////////////////////////////////////////////////
static const double PI = 3.1415926535897932384626433832795;
static const uint BLOCKDIM = 256;

extern "C" void spPostprocess2D(void *d_Dst, void *d_Src, uint DY, uint DX,
                                uint padding, int dir) {
  assert(d_Src != d_Dst);
  assert(DX % 2 == 0);

#if (POWER_OF_TWO)
  uint log2DX, log2DY;
  uint factorizationRemX = factorRadix2(log2DX, DX);
  uint factorizationRemY = factorRadix2(log2DY, DY);
  assert(factorizationRemX == 1 && factorizationRemY == 1);
#endif

  const uint threadCount = DY * (DX / 2);
  const double phaseBase = dir * PI / (double)DX;

#if (USE_TEXTURE)
  hipTextureObject_t texComplex;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_Src;
  texRes.res.linear.sizeInBytes = sizeof(fComplex) * DY * (DX + padding);
  texRes.res.linear.desc = hipCreateChannelDesc<fComplex>();

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&texComplex, &texRes, &texDescr, NULL));
#endif

  spPostprocess2D_kernel<<<iDivUp(threadCount, BLOCKDIM), BLOCKDIM>>>(
      (fComplex *)d_Dst, (fComplex *)d_Src, DY, DX, threadCount, padding,
      (float)phaseBase
#if (USE_TEXTURE)
      ,
      texComplex
#endif
      );
  getLastCudaError("spPostprocess2D_kernel<<<>>> execution failed\n");

#if (USE_TEXTURE)
  checkCudaErrors(hipDestroyTextureObject(texComplex));
#endif
}

extern "C" void spPreprocess2D(void *d_Dst, void *d_Src, uint DY, uint DX,
                               uint padding, int dir) {
  assert(d_Src != d_Dst);
  assert(DX % 2 == 0);

#if (POWER_OF_TWO)
  uint log2DX, log2DY;
  uint factorizationRemX = factorRadix2(log2DX, DX);
  uint factorizationRemY = factorRadix2(log2DY, DY);
  assert(factorizationRemX == 1 && factorizationRemY == 1);
#endif

  const uint threadCount = DY * (DX / 2);
  const double phaseBase = -dir * PI / (double)DX;

#if (USE_TEXTURE)
  hipTextureObject_t texComplex;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_Src;
  texRes.res.linear.sizeInBytes = sizeof(fComplex) * DY * (DX + padding);
  texRes.res.linear.desc = hipCreateChannelDesc<fComplex>();

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&texComplex, &texRes, &texDescr, NULL));
#endif
  spPreprocess2D_kernel<<<iDivUp(threadCount, BLOCKDIM), BLOCKDIM>>>(
      (fComplex *)d_Dst, (fComplex *)d_Src, DY, DX, threadCount, padding,
      (float)phaseBase
#if (USE_TEXTURE)
      ,
      texComplex
#endif
      );
  getLastCudaError("spPreprocess2D_kernel<<<>>> execution failed\n");

#if (USE_TEXTURE)
  checkCudaErrors(hipDestroyTextureObject(texComplex));
#endif
}

////////////////////////////////////////////////////////////////////////////////
// Combined spPostprocess2D + modulateAndNormalize + spPreprocess2D
////////////////////////////////////////////////////////////////////////////////
extern "C" void spProcess2D(void *d_Dst, void *d_SrcA, void *d_SrcB, uint DY,
                            uint DX, int dir) {
  assert(DY % 2 == 0);

#if (POWER_OF_TWO)
  uint log2DX, log2DY;
  uint factorizationRemX = factorRadix2(log2DX, DX);
  uint factorizationRemY = factorRadix2(log2DY, DY);
  assert(factorizationRemX == 1 && factorizationRemY == 1);
#endif

  const uint threadCount = (DY / 2) * DX;
  const double phaseBase = dir * PI / (double)DX;

#if (USE_TEXTURE)
  hipTextureObject_t texComplexA, texComplexB;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_SrcA;
  texRes.res.linear.sizeInBytes = sizeof(fComplex) * DY * DX;
  texRes.res.linear.desc = hipCreateChannelDesc<fComplex>();

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&texComplexA, &texRes, &texDescr, NULL));

  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_SrcB;
  texRes.res.linear.sizeInBytes = sizeof(fComplex) * DY * DX;
  texRes.res.linear.desc = hipCreateChannelDesc<fComplex>();

  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&texComplexB, &texRes, &texDescr, NULL));
#endif
  spProcess2D_kernel<<<iDivUp(threadCount, BLOCKDIM), BLOCKDIM>>>(
      (fComplex *)d_Dst, (fComplex *)d_SrcA, (fComplex *)d_SrcB, DY, DX,
      threadCount, (float)phaseBase, 0.5f / (float)(DY * DX)
#if (USE_TEXTURE)
                                         ,
      texComplexA, texComplexB
#endif
      );
  getLastCudaError("spProcess2D_kernel<<<>>> execution failed\n");

#if (USE_TEXTURE)
  checkCudaErrors(hipDestroyTextureObject(texComplexA));
  checkCudaErrors(hipDestroyTextureObject(texComplexB));
#endif
}
