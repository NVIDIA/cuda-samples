#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <cstdio>
#include <vector>

#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

using namespace std;

const char *sSampleName = "P2P (Peer-to-Peer) GPU Bandwidth Latency Test";

typedef enum {
  P2P_WRITE = 0,
  P2P_READ = 1,
} P2PDataTransfer;

typedef enum {
  CE = 0,
  SM = 1,
} P2PEngine;

P2PEngine p2p_mechanism = CE;  // By default use Copy Engine

// Macro for checking cuda errors following a cuda launch or api call
#define cudaCheckError()                                       \
  {                                                            \
    hipError_t e = hipGetLastError();                        \
    if (e != hipSuccess) {                                    \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, \
             hipGetErrorString(e));                           \
      exit(EXIT_FAILURE);                                      \
    }                                                          \
  }
__global__ void delay(volatile int *flag,
                      unsigned long long timeout_clocks = 10000000) {
  // Wait until the application notifies us that it has completed queuing up the
  // experiment, or timeout and exit, allowing the application to make progress
  long long int start_clock, sample_clock;
  start_clock = clock64();

  while (!*flag) {
    sample_clock = clock64();

    if (sample_clock - start_clock > timeout_clocks) {
      break;
    }
  }
}

// This kernel is for demonstration purposes only, not a performant kernel for
// p2p transfers.
__global__ void copyp2p(int4 *__restrict__ dest, int4 const *__restrict__ src,
                        size_t num_elems) {
  size_t globalId = blockIdx.x * blockDim.x + threadIdx.x;
  size_t gridSize = blockDim.x * gridDim.x;

#pragma unroll(5)
  for (size_t i = globalId; i < num_elems; i += gridSize) {
    dest[i] = src[i];
  }
}

///////////////////////////////////////////////////////////////////////////
// Print help screen
///////////////////////////////////////////////////////////////////////////
void printHelp(void) {
  printf("Usage:  p2pBandwidthLatencyTest [OPTION]...\n");
  printf("Tests bandwidth/latency of GPU pairs using P2P and without P2P\n");
  printf("\n");

  printf("Options:\n");
  printf("--help\t\tDisplay this help menu\n");
  printf(
      "--p2p_read\tUse P2P reads for data transfers between GPU pairs and show "
      "corresponding results.\n \t\tDefault used is P2P write operation.\n");
  printf("--sm_copy                      Use SM intiated p2p transfers instead of Copy Engine\n");
  printf("--numElems=<NUM_OF_INT_ELEMS>  Number of integer elements to be used in p2p copy.\n");
}

void checkP2Paccess(int numGPUs) {
  for (int i = 0; i < numGPUs; i++) {
    hipSetDevice(i);
    cudaCheckError();

    for (int j = 0; j < numGPUs; j++) {
      int access;
      if (i != j) {
        hipDeviceCanAccessPeer(&access, i, j);
        cudaCheckError();
        printf("Device=%d %s Access Peer Device=%d\n", i,
               access ? "CAN" : "CANNOT", j);
      }
    }
  }
  printf(
      "\n***NOTE: In case a device doesn't have P2P access to other one, it "
      "falls back to normal memcopy procedure.\nSo you can see lesser "
      "Bandwidth (GB/s) and unstable Latency (us) in those cases.\n\n");
}

void performP2PCopy(int *dest, int destDevice, int *src, int srcDevice,
                    int num_elems, int repeat, bool p2paccess,
                    hipStream_t streamToRun) {
  int blockSize = 0;
  int numBlocks = 0;

  hipOccupancyMaxPotentialBlockSize(&numBlocks, &blockSize, copyp2p);
  cudaCheckError();

  if (p2p_mechanism == SM && p2paccess) {
    for (int r = 0; r < repeat; r++) {
      copyp2p<<<numBlocks, blockSize, 0, streamToRun>>>(
          (int4 *)dest, (int4 *)src, num_elems / 4);
    }
  } else {
    for (int r = 0; r < repeat; r++) {
      hipMemcpyPeerAsync(dest, destDevice, src, srcDevice,
                          sizeof(int) * num_elems, streamToRun);
    }
  }
}

void outputBandwidthMatrix(int numElems, int numGPUs, bool p2p, P2PDataTransfer p2p_method) {
  int repeat = 5;
  volatile int *flag = NULL;
  vector<int *> buffers(numGPUs);
  vector<int *> buffersD2D(numGPUs);  // buffer for D2D, that is, intra-GPU copy
  vector<hipEvent_t> start(numGPUs);
  vector<hipEvent_t> stop(numGPUs);
  vector<hipStream_t> stream(numGPUs);

  hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable);
  cudaCheckError();

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking);
    hipMalloc(&buffers[d], numElems * sizeof(int));
    cudaCheckError();
    hipMemset(buffers[d], 0, numElems * sizeof(int));
    cudaCheckError();
    hipMalloc(&buffersD2D[d], numElems * sizeof(int));
    cudaCheckError();
    hipMemset(buffersD2D[d], 0, numElems * sizeof(int));
    cudaCheckError();
    hipEventCreate(&start[d]);
    cudaCheckError();
    hipEventCreate(&stop[d]);
    cudaCheckError();
  }

  vector<double> bandwidthMatrix(numGPUs * numGPUs);

  for (int i = 0; i < numGPUs; i++) {
    hipSetDevice(i);

    for (int j = 0; j < numGPUs; j++) {
      int access = 0;
      if (p2p) {
        hipDeviceCanAccessPeer(&access, i, j);
        if (access) {
          hipDeviceEnablePeerAccess(j, 0);
          cudaCheckError();
          hipSetDevice(j);
          cudaCheckError();
          hipDeviceEnablePeerAccess(i, 0);
          cudaCheckError();
          hipSetDevice(i);
          cudaCheckError();
        }
      }

      hipStreamSynchronize(stream[i]);
      cudaCheckError();

      // Block the stream until all the work is queued up
      // DANGER! - hipMemcpy*Async may infinitely block waiting for
      // room to push the operation, so keep the number of repeatitions
      // relatively low.  Higher repeatitions will cause the delay kernel
      // to timeout and lead to unstable results.
      *flag = 0;
      delay<<<1, 1, 0, stream[i]>>>(flag);
      cudaCheckError();
      hipEventRecord(start[i], stream[i]);
      cudaCheckError();

      if (i == j) {
        // Perform intra-GPU, D2D copies
        performP2PCopy(buffers[i], i, buffersD2D[i], i, numElems, repeat,
                       access, stream[i]);

      } else {
        if (p2p_method == P2P_WRITE) {
          performP2PCopy(buffers[j], j, buffers[i], i, numElems, repeat, access,
                         stream[i]);
        } else {
          performP2PCopy(buffers[i], i, buffers[j], j, numElems, repeat, access,
                         stream[i]);
        }
      }

      hipEventRecord(stop[i], stream[i]);
      cudaCheckError();

      // Release the queued events
      *flag = 1;
      hipStreamSynchronize(stream[i]);
      cudaCheckError();

      float time_ms;
      hipEventElapsedTime(&time_ms, start[i], stop[i]);
      double time_s = time_ms / 1e3;

      double gb = numElems * sizeof(int) * repeat / (double)1e9;
      if (i == j) {
        gb *= 2;  // must count both the read and the write here
      }
      bandwidthMatrix[i * numGPUs + j] = gb / time_s;
      if (p2p && access) {
        hipDeviceDisablePeerAccess(j);
        hipSetDevice(j);
        hipDeviceDisablePeerAccess(i);
        hipSetDevice(i);
        cudaCheckError();
      }
    }
  }

  printf("   D\\D");

  for (int j = 0; j < numGPUs; j++) {
    printf("%6d ", j);
  }

  printf("\n");

  for (int i = 0; i < numGPUs; i++) {
    printf("%6d ", i);

    for (int j = 0; j < numGPUs; j++) {
      printf("%6.02f ", bandwidthMatrix[i * numGPUs + j]);
    }

    printf("\n");
  }

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipFree(buffers[d]);
    hipFree(buffersD2D[d]);
    cudaCheckError();
    hipEventDestroy(start[d]);
    cudaCheckError();
    hipEventDestroy(stop[d]);
    cudaCheckError();
    hipStreamDestroy(stream[d]);
    cudaCheckError();
  }

  hipHostFree((void *)flag);
  cudaCheckError();
}

void outputBidirectionalBandwidthMatrix(int numElems, int numGPUs, bool p2p) {
  int repeat = 5;
  volatile int *flag = NULL;
  vector<int *> buffers(numGPUs);
  vector<int *> buffersD2D(numGPUs);
  vector<hipEvent_t> start(numGPUs);
  vector<hipEvent_t> stop(numGPUs);
  vector<hipStream_t> stream0(numGPUs);
  vector<hipStream_t> stream1(numGPUs);

  hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable);
  cudaCheckError();

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipMalloc(&buffers[d], numElems * sizeof(int));
    hipMemset(buffers[d], 0, numElems * sizeof(int));
    hipMalloc(&buffersD2D[d], numElems * sizeof(int));
    hipMemset(buffersD2D[d], 0, numElems * sizeof(int));
    cudaCheckError();
    hipEventCreate(&start[d]);
    cudaCheckError();
    hipEventCreate(&stop[d]);
    cudaCheckError();
    hipStreamCreateWithFlags(&stream0[d], hipStreamNonBlocking);
    cudaCheckError();
    hipStreamCreateWithFlags(&stream1[d], hipStreamNonBlocking);
    cudaCheckError();
  }

  vector<double> bandwidthMatrix(numGPUs * numGPUs);

  for (int i = 0; i < numGPUs; i++) {
    hipSetDevice(i);

    for (int j = 0; j < numGPUs; j++) {
      int access = 0;
      if (p2p) {
        hipDeviceCanAccessPeer(&access, i, j);
        if (access) {
          hipSetDevice(i);
          hipDeviceEnablePeerAccess(j, 0);
          cudaCheckError();
          hipSetDevice(j);
          hipDeviceEnablePeerAccess(i, 0);
          cudaCheckError();
        }
      }

      hipSetDevice(i);
      hipStreamSynchronize(stream0[i]);
      hipStreamSynchronize(stream1[j]);
      cudaCheckError();

      // Block the stream until all the work is queued up
      // DANGER! - hipMemcpy*Async may infinitely block waiting for
      // room to push the operation, so keep the number of repeatitions
      // relatively low.  Higher repeatitions will cause the delay kernel
      // to timeout and lead to unstable results.
      *flag = 0;
      hipSetDevice(i);
      // No need to block stream1 since it'll be blocked on stream0's event
      delay<<<1, 1, 0, stream0[i]>>>(flag);
      cudaCheckError();

      // Force stream1 not to start until stream0 does, in order to ensure
      // the events on stream0 fully encompass the time needed for all
      // operations
      hipEventRecord(start[i], stream0[i]);
      hipStreamWaitEvent(stream1[j], start[i], 0);

      if (i == j) {
        // For intra-GPU perform 2 memcopies buffersD2D <-> buffers
        performP2PCopy(buffers[i], i, buffersD2D[i], i, numElems, repeat,
                       access, stream0[i]);
        performP2PCopy(buffersD2D[i], i, buffers[i], i, numElems, repeat,
                       access, stream1[i]);
      } else {
        if (access && p2p_mechanism == SM) {
          hipSetDevice(j);
        }
        performP2PCopy(buffers[i], i, buffers[j], j, numElems, repeat, access,
                       stream1[j]);
        if (access && p2p_mechanism == SM) {
          hipSetDevice(i);
        }
        performP2PCopy(buffers[j], j, buffers[i], i, numElems, repeat, access,
                       stream0[i]);
      }

      // Notify stream0 that stream1 is complete and record the time of
      // the total transaction
      hipEventRecord(stop[j], stream1[j]);
      hipStreamWaitEvent(stream0[i], stop[j], 0);
      hipEventRecord(stop[i], stream0[i]);

      // Release the queued operations
      *flag = 1;
      hipStreamSynchronize(stream0[i]);
      hipStreamSynchronize(stream1[j]);
      cudaCheckError();

      float time_ms;
      hipEventElapsedTime(&time_ms, start[i], stop[i]);
      double time_s = time_ms / 1e3;

      double gb = 2.0 * numElems * sizeof(int) * repeat / (double)1e9;
      if (i == j) {
        gb *= 2;  // must count both the read and the write here
      }
      bandwidthMatrix[i * numGPUs + j] = gb / time_s;
      if (p2p && access) {
        hipSetDevice(i);
        hipDeviceDisablePeerAccess(j);
        hipSetDevice(j);
        hipDeviceDisablePeerAccess(i);
      }
    }
  }

  printf("   D\\D");

  for (int j = 0; j < numGPUs; j++) {
    printf("%6d ", j);
  }

  printf("\n");

  for (int i = 0; i < numGPUs; i++) {
    printf("%6d ", i);

    for (int j = 0; j < numGPUs; j++) {
      printf("%6.02f ", bandwidthMatrix[i * numGPUs + j]);
    }

    printf("\n");
  }

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipFree(buffers[d]);
    hipFree(buffersD2D[d]);
    cudaCheckError();
    hipEventDestroy(start[d]);
    cudaCheckError();
    hipEventDestroy(stop[d]);
    cudaCheckError();
    hipStreamDestroy(stream0[d]);
    cudaCheckError();
    hipStreamDestroy(stream1[d]);
    cudaCheckError();
  }

  hipHostFree((void *)flag);
  cudaCheckError();
}

void outputLatencyMatrix(int numGPUs, bool p2p, P2PDataTransfer p2p_method) {
  int repeat = 100;
  int numElems = 4;  // perform 1-int4 transfer.
  volatile int *flag = NULL;
  StopWatchInterface *stopWatch = NULL;
  vector<int *> buffers(numGPUs);
  vector<int *> buffersD2D(numGPUs);  // buffer for D2D, that is, intra-GPU copy
  vector<hipStream_t> stream(numGPUs);
  vector<hipEvent_t> start(numGPUs);
  vector<hipEvent_t> stop(numGPUs);

  hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable);
  cudaCheckError();

  if (!sdkCreateTimer(&stopWatch)) {
    printf("Failed to create stop watch\n");
    exit(EXIT_FAILURE);
  }
  sdkStartTimer(&stopWatch);

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipStreamCreateWithFlags(&stream[d], hipStreamNonBlocking);
    hipMalloc(&buffers[d], sizeof(int) * numElems);
    hipMemset(buffers[d], 0, sizeof(int) * numElems);
    hipMalloc(&buffersD2D[d], sizeof(int) * numElems);
    hipMemset(buffersD2D[d], 0, sizeof(int) * numElems);
    cudaCheckError();
    hipEventCreate(&start[d]);
    cudaCheckError();
    hipEventCreate(&stop[d]);
    cudaCheckError();
  }

  vector<double> gpuLatencyMatrix(numGPUs * numGPUs);
  vector<double> cpuLatencyMatrix(numGPUs * numGPUs);

  for (int i = 0; i < numGPUs; i++) {
    hipSetDevice(i);

    for (int j = 0; j < numGPUs; j++) {
      int access = 0;
      if (p2p) {
        hipDeviceCanAccessPeer(&access, i, j);
        if (access) {
          hipDeviceEnablePeerAccess(j, 0);
          cudaCheckError();
          hipSetDevice(j);
          hipDeviceEnablePeerAccess(i, 0);
          hipSetDevice(i);
          cudaCheckError();
        }
      }
      hipStreamSynchronize(stream[i]);
      cudaCheckError();

      // Block the stream until all the work is queued up
      // DANGER! - hipMemcpy*Async may infinitely block waiting for
      // room to push the operation, so keep the number of repeatitions
      // relatively low.  Higher repeatitions will cause the delay kernel
      // to timeout and lead to unstable results.
      *flag = 0;
      delay<<<1, 1, 0, stream[i]>>>(flag);
      cudaCheckError();
      hipEventRecord(start[i], stream[i]);

      sdkResetTimer(&stopWatch);
      if (i == j) {
        // Perform intra-GPU, D2D copies
        performP2PCopy(buffers[i], i, buffersD2D[i], i, numElems, repeat,
                       access, stream[i]);
      } else {
        if (p2p_method == P2P_WRITE) {
          performP2PCopy(buffers[j], j, buffers[i], i, numElems, repeat, access,
                         stream[i]);
        } else {
          performP2PCopy(buffers[i], i, buffers[j], j, numElems, repeat, access,
                         stream[i]);
        }
      }
      float cpu_time_ms = sdkGetTimerValue(&stopWatch);

      hipEventRecord(stop[i], stream[i]);
      // Now that the work has been queued up, release the stream
      *flag = 1;
      hipStreamSynchronize(stream[i]);
      cudaCheckError();

      float gpu_time_ms;
      hipEventElapsedTime(&gpu_time_ms, start[i], stop[i]);

      gpuLatencyMatrix[i * numGPUs + j] = gpu_time_ms * 1e3 / repeat;
      cpuLatencyMatrix[i * numGPUs + j] = cpu_time_ms * 1e3 / repeat;
      if (p2p && access) {
        hipDeviceDisablePeerAccess(j);
        hipSetDevice(j);
        hipDeviceDisablePeerAccess(i);
        hipSetDevice(i);
        cudaCheckError();
      }
    }
  }

  printf("   GPU");

  for (int j = 0; j < numGPUs; j++) {
    printf("%6d ", j);
  }

  printf("\n");

  for (int i = 0; i < numGPUs; i++) {
    printf("%6d ", i);

    for (int j = 0; j < numGPUs; j++) {
      printf("%6.02f ", gpuLatencyMatrix[i * numGPUs + j]);
    }

    printf("\n");
  }

  printf("\n   CPU");

  for (int j = 0; j < numGPUs; j++) {
    printf("%6d ", j);
  }

  printf("\n");

  for (int i = 0; i < numGPUs; i++) {
    printf("%6d ", i);

    for (int j = 0; j < numGPUs; j++) {
      printf("%6.02f ", cpuLatencyMatrix[i * numGPUs + j]);
    }

    printf("\n");
  }

  for (int d = 0; d < numGPUs; d++) {
    hipSetDevice(d);
    hipFree(buffers[d]);
    hipFree(buffersD2D[d]);
    cudaCheckError();
    hipEventDestroy(start[d]);
    cudaCheckError();
    hipEventDestroy(stop[d]);
    cudaCheckError();
    hipStreamDestroy(stream[d]);
    cudaCheckError();
  }

  sdkDeleteTimer(&stopWatch);

  hipHostFree((void *)flag);
  cudaCheckError();
}

int main(int argc, char **argv) {
  int numGPUs, numElems = 40000000;
  P2PDataTransfer p2p_method = P2P_WRITE;

  hipGetDeviceCount(&numGPUs);
  cudaCheckError();

  // process command line args
  if (checkCmdLineFlag(argc, (const char **)argv, "help")) {
    printHelp();
    return 0;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "p2p_read")) {
    p2p_method = P2P_READ;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "sm_copy")) {
    p2p_mechanism = SM;
  }

  // number of elements of int to be used in copy.
  if (checkCmdLineFlag(argc, (const char **)argv, "numElems")) {
    numElems = getCmdLineArgumentInt(argc, (const char **)argv, "numElems");
  }

  printf("[%s]\n", sSampleName);

  // output devices
  for (int i = 0; i < numGPUs; i++) {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, i);
    cudaCheckError();
    printf("Device: %d, %s, pciBusID: %x, pciDeviceID: %x, pciDomainID:%x\n", i,
           prop.name, prop.pciBusID, prop.pciDeviceID, prop.pciDomainID);
  }

  checkP2Paccess(numGPUs);

  // Check peer-to-peer connectivity
  printf("P2P Connectivity Matrix\n");
  printf("     D\\D");

  for (int j = 0; j < numGPUs; j++) {
    printf("%6d", j);
  }
  printf("\n");

  for (int i = 0; i < numGPUs; i++) {
    printf("%6d\t", i);
    for (int j = 0; j < numGPUs; j++) {
      if (i != j) {
        int access;
        hipDeviceCanAccessPeer(&access, i, j);
        cudaCheckError();
        printf("%6d", (access) ? 1 : 0);
      } else {
        printf("%6d", 1);
      }
    }
    printf("\n");
  }

  printf("Unidirectional P2P=Disabled Bandwidth Matrix (GB/s)\n");
  outputBandwidthMatrix(numElems, numGPUs, false, P2P_WRITE);
  printf("Unidirectional P2P=Enabled Bandwidth (P2P Writes) Matrix (GB/s)\n");
  outputBandwidthMatrix(numElems, numGPUs, true, P2P_WRITE);
  if (p2p_method == P2P_READ) {
    printf("Unidirectional P2P=Enabled Bandwidth (P2P Reads) Matrix (GB/s)\n");
    outputBandwidthMatrix(numElems, numGPUs, true, p2p_method);
  }
  printf("Bidirectional P2P=Disabled Bandwidth Matrix (GB/s)\n");
  outputBidirectionalBandwidthMatrix(numElems, numGPUs, false);
  printf("Bidirectional P2P=Enabled Bandwidth Matrix (GB/s)\n");
  outputBidirectionalBandwidthMatrix(numElems, numGPUs, true);

  printf("P2P=Disabled Latency Matrix (us)\n");
  outputLatencyMatrix(numGPUs, false, P2P_WRITE);
  printf("P2P=Enabled Latency (P2P Writes) Matrix (us)\n");
  outputLatencyMatrix(numGPUs, true, P2P_WRITE);
  if (p2p_method == P2P_READ) {
    printf("P2P=Enabled Latency (P2P Reads) Matrix (us)\n");
    outputLatencyMatrix(numGPUs, true, p2p_method);
  }

  printf(
      "\nNOTE: The CUDA Samples are not meant for performance measurements. "
      "Results may vary when GPU Boost is enabled.\n");

  exit(EXIT_SUCCESS);
}
