/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
* 1D DWT for Haar wavelet and signals with a length which is a power of 2.
* The code reduces bank conflicts and non-coalesced reads / writes as
* appropriate but does not fully remove them because the computational
* overhead to achieve this would outweighs the benefit (see inline comments
* for more details).
* Large signals are subdivided into sub-signals with 512 elements and the
* wavelet transform for these is computed with one block over 10 decomposition
* levels. The resulting signal consisting of the approximation coefficients at
* level X is then processed in a subsequent step on the device. This requires
* interblock synchronization which is only possible on host side.
* Detail coefficients which have been computed are not further referenced
* during the decomposition so that they can be stored directly in their final
* position in global memory. The transform and its storing scheme preserve
* locality in the coefficients so that these writes are coalesced.
* Approximation coefficients are stored in shared memory because they are
* needed to compute the subsequent decomposition step. The top most
* approximation coefficient for a sub-signal processed by one block is stored
* in a special global memory location to simplify the processing after the
* interblock synchronization.
* Most books on wavelets explain the Haar wavelet decomposition. A good freely
* available resource is the Wavelet primer by Stollnitz et al.
* http://grail.cs.washington.edu/projects/wavelets/article/wavelet1.pdf
* http://grail.cs.washington.edu/projects/wavelets/article/wavelet2.pdf
* The basic of all Wavelet transforms is to decompose a signal into
* approximation (a) and detail (d) coefficients where the detail tends to be
* small or zero which allows / simplifies compression. The following "graphs"
* demonstrate the transform for a signal
* of length eight. The index always describes the decomposition level where
* a coefficient arises. The input signal is interpreted as approximation signal
* at level 0. The coefficients computed on the device are stored in the same
* scheme as in the example. This data structure is particularly well suited for
* compression and also preserves the hierarchical structure of the
decomposition.

-------------------------------------------------
| a_0 | a_0 | a_0 | a_0 | a_0 | a_0 | a_0 | a_0 |
-------------------------------------------------

-------------------------------------------------
| a_1 | a_1 | a_1 | a_1 | d_1 | d_1 | d_1 | d_1 |
-------------------------------------------------

-------------------------------------------------
| a_2 | a_2 | d_2 | d_2 | d_1 | d_1 | d_1 | d_1 |
-------------------------------------------------

-------------------------------------------------
| a_3 | d_3 | d_2 | d_2 | d_1 | d_1 | d_1 | d_1 |
-------------------------------------------------

* Host code.
*/

#ifdef _WIN32
#define NOMINMAX
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>

// includes, project
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

// constants which are used in host and device code
#define INV_SQRT_2 0.70710678118654752440f;
const unsigned int LOG_NUM_BANKS = 4;
const unsigned int NUM_BANKS = 16;

////////////////////////////////////////////////////////////////////////////////
// includes, kernels
#include "dwtHaar1D_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char **argv);
bool getLevels(unsigned int len, unsigned int *levels);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  // run test
  runTest(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Perform the wavelet decomposition
////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char **argv) {
  bool bResult = false;  // flag for final validation of the results

  char *s_fname = NULL, *r_gold_fname = NULL;
  char r_fname[256];
  const char usage[] = {
      "\nUsage:\n"
      "  dwtHaar1D --signal=<signal_file> --result=<result_file> "
      "--gold=<gold_file>\n\n"
      "  <signal_file> Input file containing the signal\n"
      "  <result_file> Output file storing the result of the wavelet "
      "decomposition\n"
      "  <gold_file>   Input file containing the reference result of the "
      "wavelet decomposition\n"
      "\nExample:\n"
      "  ./dwtHaar1D\n"
      "       --signal=signal.dat\n"
      "       --result=result.dat\n"
      "       --gold=regression.gold.dat\n"};

  printf("%s Starting...\n\n", argv[0]);

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  findCudaDevice(argc, (const char **)argv);

  // file names, either specified as cmd line args or use default
  if (argc == 4) {
    char *tmp_sfname, *tmp_rfname, *tmp_goldfname;

    if ((getCmdLineArgumentString(argc, (const char **)argv, "signal",
                                  &tmp_sfname) != true) ||
        (getCmdLineArgumentString(argc, (const char **)argv, "result",
                                  &tmp_rfname) != true) ||
        (getCmdLineArgumentString(argc, (const char **)argv, "gold",
                                  &tmp_goldfname) != true)) {
      fprintf(stderr, "Invalid input syntax.\n%s", usage);
      exit(EXIT_FAILURE);
    }

    s_fname = sdkFindFilePath(tmp_sfname, argv[0]);
    r_gold_fname = sdkFindFilePath(tmp_goldfname, argv[0]);
    strcpy(r_fname, tmp_rfname);
  } else {
    s_fname = sdkFindFilePath("signal.dat", argv[0]);
    r_gold_fname = sdkFindFilePath("regression.gold.dat", argv[0]);
    strcpy(r_fname, "result.dat");
  }

  printf("source file    = \"%s\"\n", s_fname);
  printf("reference file = \"%s\"\n", r_fname);
  printf("gold file      = \"%s\"\n", r_gold_fname);

  // read in signal
  unsigned int slength = 0;
  float *signal = NULL;

  if (s_fname == NULL) {
    fprintf(stderr, "Cannot find the file containing the signal.\n%s", usage);

    exit(EXIT_FAILURE);
  }

  if (sdkReadFile(s_fname, &signal, &slength, false) == true) {
    printf("Reading signal from \"%s\"\n", s_fname);
  } else {
    exit(EXIT_FAILURE);
  }

  // get the number of decompositions necessary to perform a full decomposition
  unsigned int dlevels_complete = 0;

  if (true != getLevels(slength, &dlevels_complete)) {
    // error message
    fprintf(stderr, "Signal length not supported.\n");
    // cleanup and abort
    free(signal);
    exit(EXIT_FAILURE);
  }

  // device in data
  float *d_idata = NULL;
  // device out data
  float *d_odata = NULL;
  // device approx_final data
  float *approx_final = NULL;
  // The very final approximation coefficient has to be written to the output
  // data, all others are reused as input data in the next global step and
  // therefore have to be written to the input data again.
  // The following flag indicates where to copy approx_final data
  //   - 0 is input, 1 is output
  int approx_is_input;

  // allocate device mem
  const unsigned int smem_size = sizeof(float) * slength;
  checkCudaErrors(hipMalloc((void **)&d_idata, smem_size));
  checkCudaErrors(hipMalloc((void **)&d_odata, smem_size));
  checkCudaErrors(hipMalloc((void **)&approx_final, smem_size));
  // copy input data to device
  checkCudaErrors(
      hipMemcpy(d_idata, signal, smem_size, hipMemcpyHostToDevice));

  // total number of threads
  // in the first decomposition step always one thread computes the average and
  // detail signal for one pair of adjacent values
  unsigned int num_threads_total_left = slength / 2;
  // decomposition levels performed in the current / next step
  unsigned int dlevels_step = dlevels_complete;

  // 1D signal so the arrangement of elements is also 1D
  dim3 block_size;
  dim3 grid_size;

  // number of decomposition levels left after one iteration on the device
  unsigned int dlevels_left = dlevels_complete;

  // if less or equal 1k elements, then the data can be processed in one block,
  // this avoids the Wait-For-Idle (WFI) on host side which is necessary if the
  // computation is split across multiple SM's if enough input data
  if (dlevels_complete <= 10) {
    // decomposition can be performed at once
    block_size.x = num_threads_total_left;
    approx_is_input = 0;
  } else {
    // 512 threads per block
    grid_size.x = (num_threads_total_left / 512);
    block_size.x = 512;

    // 512 threads corresponds to 10 decomposition steps
    dlevels_step = 10;
    dlevels_left -= 10;

    approx_is_input = 1;
  }

  // Initialize d_odata to 0.0f
  initValue<<<grid_size, block_size>>>(d_odata, 0.0f);

  // do until full decomposition is accomplished
  while (0 != num_threads_total_left) {
    // double the number of threads as bytes
    unsigned int mem_shared = (2 * block_size.x) * sizeof(float);
    // extra memory requirements to avoid bank conflicts
    mem_shared += ((2 * block_size.x) / NUM_BANKS) * sizeof(float);

    // run kernel
    dwtHaar1D<<<grid_size, block_size, mem_shared>>>(
        d_idata, d_odata, approx_final, dlevels_step, num_threads_total_left,
        block_size.x);

    // Copy approx_final to appropriate location
    if (approx_is_input) {
      checkCudaErrors(hipMemcpy(d_idata, approx_final, grid_size.x * 4,
                                 hipMemcpyDeviceToDevice));
    } else {
      checkCudaErrors(hipMemcpy(d_odata, approx_final, grid_size.x * 4,
                                 hipMemcpyDeviceToDevice));
    }

    // update level variables
    if (dlevels_left < 10) {
      // approx_final = d_odata;
      approx_is_input = 0;
    }

    // more global steps necessary
    dlevels_step = (dlevels_left > 10) ? dlevels_left - 10 : dlevels_left;
    dlevels_left -= 10;

    // after each step only half the threads are used any longer
    // therefore after 10 steps 2^10 less threads
    num_threads_total_left = num_threads_total_left >> 10;

    // update block and grid size
    grid_size.x =
        (num_threads_total_left / 512) + (0 != (num_threads_total_left % 512))
            ? 1
            : 0;

    if (grid_size.x <= 1) {
      block_size.x = num_threads_total_left;
    }
  }

  // get the result back from the server
  // allocate mem for the result
  float *odata = (float *)malloc(smem_size);
  checkCudaErrors(
      hipMemcpy(odata, d_odata, smem_size, hipMemcpyDeviceToHost));

  // post processing
  // write file for regression test
  if (r_fname == NULL) {
    fprintf(stderr,
            "Cannot write the output file storing the result of the wavelet "
            "decomposition.\n%s",
            usage);
    exit(EXIT_FAILURE);
  }

  if (sdkWriteFile(r_fname, odata, slength, 0.001f, false) == true) {
    printf("Writing result to \"%s\"\n", r_fname);
  } else {
    exit(EXIT_FAILURE);
  }

  // load the reference solution
  unsigned int len_reference = 0;
  float *reference = NULL;

  if (r_gold_fname == NULL) {
    fprintf(stderr,
            "Cannot read the file containing the reference result of the "
            "wavelet decomposition.\n%s",
            usage);

    exit(EXIT_FAILURE);
  }

  if (sdkReadFile(r_gold_fname, &reference, &len_reference, false) == true) {
    printf("Reading reference result from \"%s\"\n", r_gold_fname);
  } else {
    exit(EXIT_FAILURE);
  }

  assert(slength == len_reference);

  // compare the computed solution and the reference
  bResult = (bool)sdkCompareL2fe(reference, odata, slength, 0.001f);
  free(reference);

  // free allocated host and device memory
  checkCudaErrors(hipFree(d_odata));
  checkCudaErrors(hipFree(d_idata));
  checkCudaErrors(hipFree(approx_final));

  free(signal);
  free(odata);
  free(s_fname);
  free(r_gold_fname);

  printf(bResult ? "Test success!\n" : "Test failure!\n");
}

////////////////////////////////////////////////////////////////////////////////
//! Get number of decomposition levels to perform a full decomposition
//! Also check if the input signal size is suitable
//! @return  true if the number of decomposition levels could be determined
//!          and the signal length is supported by the implementation,
//!          otherwise false
//! @param   len  length of input signal
//! @param   levels  number of decomposition levels necessary to perform a full
//!           decomposition
////////////////////////////////////////////////////////////////////////////////
bool getLevels(unsigned int len, unsigned int *levels) {
  bool retval = false;

  // currently signals up to a length of 2^20 supported
  for (unsigned int i = 0; i < 20; ++i) {
    if (len == (1 << i)) {
      *levels = i;
      retval = true;
      break;
    }
  }

  return retval;
}
