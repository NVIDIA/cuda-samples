#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Simple 3D volume renderer

#ifndef _VOLUMERENDER_KERNEL_CU_
#define _VOLUMERENDER_KERNEL_CU_

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>

typedef unsigned int uint;
typedef unsigned char uchar;

hipArray *d_volumeArray = 0;
hipArray *d_transferFuncArray;

typedef unsigned char VolumeType;
// typedef unsigned short VolumeType;

hipTextureObject_t texObject;    // For 3D texture
hipTextureObject_t transferTex;  // For 1D transfer function texture

typedef struct { float4 m[3]; } float3x4;

__constant__ float3x4 c_invViewMatrix;  // inverse view matrix

struct Ray {
  float3 o;  // origin
  float3 d;  // direction
};

// intersect ray with a box
// http://www.siggraph.org/education/materials/HyperGraph/raytrace/rtinter3.htm

__device__ int intersectBox(Ray r, float3 boxmin, float3 boxmax, float *tnear,
                            float *tfar) {
  // compute intersection of ray with all six bbox planes
  float3 invR = make_float3(1.0f) / r.d;
  float3 tbot = invR * (boxmin - r.o);
  float3 ttop = invR * (boxmax - r.o);

  // re-order intersections to find smallest and largest on each axis
  float3 tmin = fminf(ttop, tbot);
  float3 tmax = fmaxf(ttop, tbot);

  // find the largest tmin and the smallest tmax
  float largest_tmin = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
  float smallest_tmax = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));

  *tnear = largest_tmin;
  *tfar = smallest_tmax;

  return smallest_tmax > largest_tmin;
}

// transform vector by matrix (no translation)
__device__ float3 mul(const float3x4 &M, const float3 &v) {
  float3 r;
  r.x = dot(v, make_float3(M.m[0]));
  r.y = dot(v, make_float3(M.m[1]));
  r.z = dot(v, make_float3(M.m[2]));
  return r;
}

// transform vector by matrix with translation
__device__ float4 mul(const float3x4 &M, const float4 &v) {
  float4 r;
  r.x = dot(v, M.m[0]);
  r.y = dot(v, M.m[1]);
  r.z = dot(v, M.m[2]);
  r.w = 1.0f;
  return r;
}

__device__ uint rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return (uint(rgba.w * 255) << 24) | (uint(rgba.z * 255) << 16) |
         (uint(rgba.y * 255) << 8) | uint(rgba.x * 255);
}

__global__ void d_render(uint *d_output, uint imageW, uint imageH,
                         float density, float brightness, float transferOffset,
                         float transferScale, hipTextureObject_t tex,
                         hipTextureObject_t transferTex) {
  const int maxSteps = 500;
  const float tstep = 0.01f;
  const float opacityThreshold = 0.95f;
  const float3 boxMin = make_float3(-1.0f, -1.0f, -1.0f);
  const float3 boxMax = make_float3(1.0f, 1.0f, 1.0f);

  uint x = blockIdx.x * blockDim.x + threadIdx.x;
  uint y = blockIdx.y * blockDim.y + threadIdx.y;

  if ((x >= imageW) || (y >= imageH)) return;

  float u = (x / (float)imageW) * 2.0f - 1.0f;
  float v = (y / (float)imageH) * 2.0f - 1.0f;

  // calculate eye ray in world space
  Ray eyeRay;
  eyeRay.o =
      make_float3(mul(c_invViewMatrix, make_float4(0.0f, 0.0f, 0.0f, 1.0f)));
  eyeRay.d = normalize(make_float3(u, v, -2.0f));
  eyeRay.d = mul(c_invViewMatrix, eyeRay.d);

  // find intersection with box
  float tnear, tfar;
  int hit = intersectBox(eyeRay, boxMin, boxMax, &tnear, &tfar);

  if (!hit) return;

  if (tnear < 0.0f) tnear = 0.0f;  // clamp to near plane

  // march along ray from front to back, accumulating color
  float4 sum = make_float4(0.0f);
  float t = tnear;
  float3 pos = eyeRay.o + eyeRay.d * tnear;
  float3 step = eyeRay.d * tstep;

  for (int i = 0; i < maxSteps; i++) {
    // read from 3D texture
    // remap position to [0, 1] coordinates
    float sample = tex3D<float>(tex, pos.x * 0.5f + 0.5f, pos.y * 0.5f + 0.5f,
                                pos.z * 0.5f + 0.5f);
    // sample *= 64.0f;    // scale for 10-bit data

    // lookup in transfer function texture
    float4 col =
        tex1D<float4>(transferTex, (sample - transferOffset) * transferScale);
    col.w *= density;

    // "under" operator for back-to-front blending
    // sum = lerp(sum, col, col.w);

    // pre-multiply alpha
    col.x *= col.w;
    col.y *= col.w;
    col.z *= col.w;
    // "over" operator for front-to-back blending
    sum = sum + col * (1.0f - sum.w);

    // exit early if opaque
    if (sum.w > opacityThreshold) break;

    t += tstep;

    if (t > tfar) break;

    pos += step;
  }

  sum *= brightness;

  // write output color
  d_output[y * imageW + x] = rgbaFloatToInt(sum);
}

extern "C" void setTextureFilterMode(bool bLinearFilter) {
  if (texObject) {
    checkCudaErrors(hipDestroyTextureObject(texObject));
  }
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_volumeArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode =
      bLinearFilter ? hipFilterModeLinear : hipFilterModePoint;

  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.addressMode[2] = hipAddressModeWrap;

  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(
      hipCreateTextureObject(&texObject, &texRes, &texDescr, NULL));
}

extern "C" void initCuda(void *h_volume, hipExtent volumeSize) {
  // create 3D array
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
  checkCudaErrors(hipMalloc3DArray(&d_volumeArray, &channelDesc, volumeSize));

  // copy data to 3D array
  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr =
      make_hipPitchedPtr(h_volume, volumeSize.width * sizeof(VolumeType),
                          volumeSize.width, volumeSize.height);
  copyParams.dstArray = d_volumeArray;
  copyParams.extent = volumeSize;
  copyParams.kind = hipMemcpyHostToDevice;
  checkCudaErrors(hipMemcpy3D(&copyParams));

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_volumeArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords =
      true;  // access with normalized texture coordinates
  texDescr.filterMode = hipFilterModeLinear;  // linear interpolation

  texDescr.addressMode[0] = hipAddressModeClamp;  // clamp texture coordinates
  texDescr.addressMode[1] = hipAddressModeClamp;
  texDescr.addressMode[2] = hipAddressModeClamp;

  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(
      hipCreateTextureObject(&texObject, &texRes, &texDescr, NULL));

  // create transfer function texture
  float4 transferFunc[] = {
    {  0.0, 0.0, 0.0, 0.0, },
    {  1.0, 0.0, 0.0, 1.0, },
    {  1.0, 0.5, 0.0, 1.0, },
    {  1.0, 1.0, 0.0, 1.0, },
    {  0.0, 1.0, 0.0, 1.0, },
    {  0.0, 1.0, 1.0, 1.0, },
    {  0.0, 0.0, 1.0, 1.0, },
    {  1.0, 0.0, 1.0, 1.0, },
    {  0.0, 0.0, 0.0, 0.0, },
  };

  hipChannelFormatDesc channelDesc2 = hipCreateChannelDesc<float4>();
  hipArray *d_transferFuncArray;
  checkCudaErrors(hipMallocArray(&d_transferFuncArray, &channelDesc2,
                                  sizeof(transferFunc) / sizeof(float4), 1));
  checkCudaErrors(hipMemcpy2DToArray(d_transferFuncArray, 0, 0, transferFunc,
                                      0, sizeof(transferFunc), 1,
                                      hipMemcpyHostToDevice));

  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_transferFuncArray;

  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords =
      true;  // access with normalized texture coordinates
  texDescr.filterMode = hipFilterModeLinear;

  texDescr.addressMode[0] = hipAddressModeClamp;  // wrap texture coordinates

  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&transferTex, &texRes, &texDescr, NULL));
}

extern "C" void freeCudaBuffers() {
  checkCudaErrors(hipDestroyTextureObject(texObject));
  checkCudaErrors(hipDestroyTextureObject(transferTex));
  checkCudaErrors(hipFreeArray(d_volumeArray));
  checkCudaErrors(hipFreeArray(d_transferFuncArray));
}

extern "C" void render_kernel(dim3 gridSize, dim3 blockSize, uint *d_output,
                              uint imageW, uint imageH, float density,
                              float brightness, float transferOffset,
                              float transferScale) {
  d_render<<<gridSize, blockSize>>>(d_output, imageW, imageH, density,
                                    brightness, transferOffset, transferScale,
                                    texObject, transferTex);
}

extern "C" void copyInvViewMatrix(float *invViewMatrix, size_t sizeofMatrix) {
  checkCudaErrors(
      hipMemcpyToSymbol(HIP_SYMBOL(c_invViewMatrix), invViewMatrix, sizeofMatrix));
}

#endif  // #ifndef _VOLUMERENDER_KERNEL_CU_
