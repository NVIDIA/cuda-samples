#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
This file contains simple wrapper functions that call the CUDA kernels
*/
#define HELPERGL_EXTERN_GL_FUNC_IMPLEMENTATION
#include <helper_gl.h>
#include <hip/hip_runtime_api.h>
#include <cstdlib>
#include <cstdio>
#include <string.h>
#include <cuda_gl_interop.h>

#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_device.cuh"
#include "ParticleSystem.cuh"

extern "C" {

hipArray *noiseArray;

void setParameters(SimParams *hostParams) {
  // copy parameters to constant memory
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
}

// Round a / b to nearest higher integer value
int iDivUp(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// compute grid and thread block size for a given number of elements
void computeGridSize(int n, int blockSize, int &numBlocks, int &numThreads) {
  numThreads = min(blockSize, n);
  numBlocks = iDivUp(n, numThreads);
}

inline float frand() { return rand() / (float)RAND_MAX; }

// create 3D texture containing random values
void createNoiseTexture(int w, int h, int d) {
  hipExtent size = make_hipExtent(w, h, d);
  size_t elements = size.width * size.height * size.depth;

  float *volumeData = (float *)malloc(elements * 4 * sizeof(float));
  float *ptr = volumeData;

  for (size_t i = 0; i < elements; i++) {
    *ptr++ = frand() * 2.0f - 1.0f;
    *ptr++ = frand() * 2.0f - 1.0f;
    *ptr++ = frand() * 2.0f - 1.0f;
    *ptr++ = frand() * 2.0f - 1.0f;
  }

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float4>();
  checkCudaErrors(hipMalloc3DArray(&noiseArray, &channelDesc, size));

  hipMemcpy3DParms copyParams = {0};
  copyParams.srcPtr = make_hipPitchedPtr(
      (void *)volumeData, size.width * sizeof(float4), size.width, size.height);
  copyParams.dstArray = noiseArray;
  copyParams.extent = size;
  copyParams.kind = hipMemcpyHostToDevice;
  checkCudaErrors(hipMemcpy3D(&copyParams));

  free(volumeData);

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = noiseArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.addressMode[2] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&noiseTex, &texRes, &texDescr, NULL));
}

void integrateSystem(float4 *oldPos, float4 *newPos, float4 *oldVel,
                     float4 *newVel, float deltaTime, int numParticles) {
  thrust::device_ptr<float4> d_newPos(newPos);
  thrust::device_ptr<float4> d_newVel(newVel);
  thrust::device_ptr<float4> d_oldPos(oldPos);
  thrust::device_ptr<float4> d_oldVel(oldVel);

  thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(
                       d_newPos, d_newVel, d_oldPos, d_oldVel)),
                   thrust::make_zip_iterator(thrust::make_tuple(
                       d_newPos + numParticles, d_newVel + numParticles,
                       d_oldPos + numParticles, d_oldVel + numParticles)),
                   integrate_functor(deltaTime, noiseTex));
}

void calcDepth(float4 *pos,
               float *keys,    // output
               uint *indices,  // output
               float3 sortVector, int numParticles) {
  thrust::device_ptr<float4> d_pos(pos);
  thrust::device_ptr<float> d_keys(keys);
  thrust::device_ptr<uint> d_indices(indices);

  thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(d_pos, d_keys)),
                   thrust::make_zip_iterator(thrust::make_tuple(
                       d_pos + numParticles, d_keys + numParticles)),
                   calcDepth_functor(sortVector));

  thrust::sequence(d_indices, d_indices + numParticles);
}

void sortParticles(float *sortKeys, uint *indices, uint numParticles) {
  thrust::sort_by_key(thrust::device_ptr<float>(sortKeys),
                      thrust::device_ptr<float>(sortKeys + numParticles),
                      thrust::device_ptr<uint>(indices));
}

}  // extern "C"
