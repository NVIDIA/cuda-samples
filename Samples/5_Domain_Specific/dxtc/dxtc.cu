#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Utilities and system includes
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <hip/hip_vector_types.h>
#include <float.h>  // for FLT_MAX

#include "CudaMath.h"
#include "dds.h"
#include "permutations.h"

// Definitions
#define INPUT_IMAGE "teapot512_std.ppm"
#define REFERENCE_IMAGE "teapot512_ref.dds"

#define ERROR_THRESHOLD 0.02f

#define NUM_THREADS 64  // Number of threads per block.

#define __debugsync()

template <class T>
__device__ inline void swap(T &a, T &b) {
  T tmp = a;
  a = b;
  b = tmp;
}

//__constant__ float3 kColorMetric = { 0.2126f, 0.7152f, 0.0722f };
__constant__ float3 kColorMetric = {1.0f, 1.0f, 1.0f};

////////////////////////////////////////////////////////////////////////////////
// Sort colors
////////////////////////////////////////////////////////////////////////////////
__device__ void sortColors(const float *values, int *ranks,
                           cg::thread_group tile) {
  const int tid = threadIdx.x;

  int rank = 0;

#pragma unroll

  for (int i = 0; i < 16; i++) {
    rank += (values[i] < values[tid]);
  }

  ranks[tid] = rank;

  cg::sync(tile);

  // Resolve elements with the same index.
  for (int i = 0; i < 15; i++) {
    if (tid > i && ranks[tid] == ranks[i]) {
      ++ranks[tid];
    }
    cg::sync(tile);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Load color block to shared mem
////////////////////////////////////////////////////////////////////////////////
__device__ void loadColorBlock(const uint *image, float3 colors[16],
                               float3 sums[16], int xrefs[16], int blockOffset,
                               cg::thread_block cta) {
  const int bid = blockIdx.x + blockOffset;
  const int idx = threadIdx.x;

  __shared__ float dps[16];

  float3 tmp;

  cg::thread_group tile = cg::tiled_partition(cta, 16);

  if (idx < 16) {
    // Read color and copy to shared mem.
    uint c = image[(bid)*16 + idx];

    colors[idx].x = ((c >> 0) & 0xFF) * (1.0f / 255.0f);
    colors[idx].y = ((c >> 8) & 0xFF) * (1.0f / 255.0f);
    colors[idx].z = ((c >> 16) & 0xFF) * (1.0f / 255.0f);

    cg::sync(tile);
    // Sort colors along the best fit line.
    colorSums(colors, sums, tile);

    cg::sync(tile);

    float3 axis = bestFitLine(colors, sums[0], tile);

    cg::sync(tile);

    dps[idx] = dot(colors[idx], axis);

    cg::sync(tile);

    sortColors(dps, xrefs, tile);

    cg::sync(tile);

    tmp = colors[idx];

    cg::sync(tile);

    colors[xrefs[idx]] = tmp;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Round color to RGB565 and expand
////////////////////////////////////////////////////////////////////////////////
inline __device__ float3 roundAndExpand(float3 v, ushort *w) {
  v.x = rintf(__saturatef(v.x) * 31.0f);
  v.y = rintf(__saturatef(v.y) * 63.0f);
  v.z = rintf(__saturatef(v.z) * 31.0f);

  *w = ((ushort)v.x << 11) | ((ushort)v.y << 5) | (ushort)v.z;
  v.x *= 0.03227752766457f;  // approximate integer bit expansion.
  v.y *= 0.01583151765563f;
  v.z *= 0.03227752766457f;
  return v;
}

__constant__ float alphaTable4[4] = {9.0f, 0.0f, 6.0f, 3.0f};
__constant__ float alphaTable3[4] = {4.0f, 0.0f, 2.0f, 2.0f};
__constant__ const int prods4[4] = {0x090000, 0x000900, 0x040102, 0x010402};
__constant__ const int prods3[4] = {0x040000, 0x000400, 0x040101, 0x010401};

#define USE_TABLES 1

////////////////////////////////////////////////////////////////////////////////
// Evaluate permutations
////////////////////////////////////////////////////////////////////////////////
static __device__ float evalPermutation4(const float3 *colors, uint permutation,
                                         ushort *start, ushort *end,
                                         float3 color_sum) {
// Compute endpoints using least squares.
#if USE_TABLES
  float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

  int akku = 0;

  // Compute alpha & beta for this permutation.
  for (int i = 0; i < 16; i++) {
    const uint bits = permutation >> (2 * i);

    alphax_sum += alphaTable4[bits & 3] * colors[i];
    akku += prods4[bits & 3];
  }

  float alpha2_sum = float(akku >> 16);
  float beta2_sum = float((akku >> 8) & 0xff);
  float alphabeta_sum = float((akku >> 0) & 0xff);
  float3 betax_sum = (9.0f * color_sum) - alphax_sum;
#else
  float alpha2_sum = 0.0f;
  float beta2_sum = 0.0f;
  float alphabeta_sum = 0.0f;
  float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

  // Compute alpha & beta for this permutation.
  for (int i = 0; i < 16; i++) {
    const uint bits = permutation >> (2 * i);

    float beta = (bits & 1);

    if (bits & 2) {
      beta = (1 + beta) * (1.0f / 3.0f);
    }

    float alpha = 1.0f - beta;

    alpha2_sum += alpha * alpha;
    beta2_sum += beta * beta;
    alphabeta_sum += alpha * beta;
    alphax_sum += alpha * colors[i];
  }

  float3 betax_sum = color_sum - alphax_sum;
#endif

  // alpha2, beta2, alphabeta and factor could be precomputed for each
  // permutation, but it's faster to recompute them.
  const float factor =
      1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

  float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
  float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

  // Round a, b to the closest 5-6-5 color and expand...
  a = roundAndExpand(a, start);
  b = roundAndExpand(b, end);

  // compute the error
  float3 e = a * a * alpha2_sum + b * b * beta2_sum +
             2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

  return (0.111111111111f) * dot(e, kColorMetric);
}

static __device__ float evalPermutation3(const float3 *colors, uint permutation,
                                         ushort *start, ushort *end,
                                         float3 color_sum) {
// Compute endpoints using least squares.
#if USE_TABLES
  float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

  int akku = 0;

  // Compute alpha & beta for this permutation.
  for (int i = 0; i < 16; i++) {
    const uint bits = permutation >> (2 * i);

    alphax_sum += alphaTable3[bits & 3] * colors[i];
    akku += prods3[bits & 3];
  }

  float alpha2_sum = float(akku >> 16);
  float beta2_sum = float((akku >> 8) & 0xff);
  float alphabeta_sum = float((akku >> 0) & 0xff);
  float3 betax_sum = (4.0f * color_sum) - alphax_sum;
#else
  float alpha2_sum = 0.0f;
  float beta2_sum = 0.0f;
  float alphabeta_sum = 0.0f;
  float3 alphax_sum = make_float3(0.0f, 0.0f, 0.0f);

  // Compute alpha & beta for this permutation.
  for (int i = 0; i < 16; i++) {
    const uint bits = permutation >> (2 * i);

    float beta = (bits & 1);

    if (bits & 2) {
      beta = 0.5f;
    }

    float alpha = 1.0f - beta;

    alpha2_sum += alpha * alpha;
    beta2_sum += beta * beta;
    alphabeta_sum += alpha * beta;
    alphax_sum += alpha * colors[i];
  }

  float3 betax_sum = color_sum - alphax_sum;
#endif

  const float factor =
      1.0f / (alpha2_sum * beta2_sum - alphabeta_sum * alphabeta_sum);

  float3 a = (alphax_sum * beta2_sum - betax_sum * alphabeta_sum) * factor;
  float3 b = (betax_sum * alpha2_sum - alphax_sum * alphabeta_sum) * factor;

  // Round a, b to the closest 5-6-5 color and expand...
  a = roundAndExpand(a, start);
  b = roundAndExpand(b, end);

  // compute the error
  float3 e = a * a * alpha2_sum + b * b * beta2_sum +
             2.0f * (a * b * alphabeta_sum - a * alphax_sum - b * betax_sum);

  return (0.25f) * dot(e, kColorMetric);
}

__device__ void evalAllPermutations(const float3 *colors,
                                    const uint *permutations, ushort &bestStart,
                                    ushort &bestEnd, uint &bestPermutation,
                                    float *errors, float3 color_sum,
                                    cg::thread_block cta) {
  const int idx = threadIdx.x;

  float bestError = FLT_MAX;

  __shared__ uint s_permutations[160];

  for (int i = 0; i < 16; i++) {
    int pidx = idx + NUM_THREADS * i;

    if (pidx >= 992) {
      break;
    }

    ushort start, end;
    uint permutation = permutations[pidx];

    if (pidx < 160) {
      s_permutations[pidx] = permutation;
    }

    float error =
        evalPermutation4(colors, permutation, &start, &end, color_sum);

    if (error < bestError) {
      bestError = error;
      bestPermutation = permutation;
      bestStart = start;
      bestEnd = end;
    }
  }

  if (bestStart < bestEnd) {
    swap(bestEnd, bestStart);
    bestPermutation ^= 0x55555555;  // Flip indices.
  }

  cg::sync(cta);  // Sync here to ensure s_permutations is valid going forward

  for (int i = 0; i < 3; i++) {
    int pidx = idx + NUM_THREADS * i;

    if (pidx >= 160) {
      break;
    }

    ushort start, end;
    uint permutation = s_permutations[pidx];
    float error =
        evalPermutation3(colors, permutation, &start, &end, color_sum);

    if (error < bestError) {
      bestError = error;
      bestPermutation = permutation;
      bestStart = start;
      bestEnd = end;

      if (bestStart > bestEnd) {
        swap(bestEnd, bestStart);
        bestPermutation ^=
            (~bestPermutation >> 1) & 0x55555555;  // Flip indices.
      }
    }
  }

  errors[idx] = bestError;
}

////////////////////////////////////////////////////////////////////////////////
// Find index with minimum error
////////////////////////////////////////////////////////////////////////////////
__device__ int findMinError(float *errors, cg::thread_block cta) {
  const int idx = threadIdx.x;
  __shared__ int indices[NUM_THREADS];
  indices[idx] = idx;

  cg::sync(cta);

  for (int d = NUM_THREADS / 2; d > 0; d >>= 1) {
    float err0 = errors[idx];
    float err1 = (idx + d) < NUM_THREADS ? errors[idx + d] : FLT_MAX;
    int index1 = (idx + d) < NUM_THREADS ? indices[idx + d] : 0;

    cg::sync(cta);

    if (err1 < err0) {
      errors[idx] = err1;
      indices[idx] = index1;
    }

    cg::sync(cta);
  }

  return indices[0];
}

////////////////////////////////////////////////////////////////////////////////
// Save DXT block
////////////////////////////////////////////////////////////////////////////////
__device__ void saveBlockDXT1(ushort start, ushort end, uint permutation,
                              int xrefs[16], uint2 *result, int blockOffset) {
  const int bid = blockIdx.x + blockOffset;

  if (start == end) {
    permutation = 0;
  }

  // Reorder permutation.
  uint indices = 0;

  for (int i = 0; i < 16; i++) {
    int ref = xrefs[i];
    indices |= ((permutation >> (2 * ref)) & 3) << (2 * i);
  }

  // Write endpoints.
  result[bid].x = (end << 16) | start;

  // Write palette indices.
  result[bid].y = indices;
}

////////////////////////////////////////////////////////////////////////////////
// Compress color block
////////////////////////////////////////////////////////////////////////////////
__global__ void compress(const uint *permutations, const uint *image,
                         uint2 *result, int blockOffset) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();

  const int idx = threadIdx.x;

  __shared__ float3 colors[16];
  __shared__ float3 sums[16];
  __shared__ int xrefs[16];

  loadColorBlock(image, colors, sums, xrefs, blockOffset, cta);

  cg::sync(cta);

  ushort bestStart, bestEnd;
  uint bestPermutation;

  __shared__ float errors[NUM_THREADS];

  evalAllPermutations(colors, permutations, bestStart, bestEnd, bestPermutation,
                      errors, sums[0], cta);

  // Use a parallel reduction to find minimum error.
  const int minIdx = findMinError(errors, cta);

  cg::sync(cta);

  // Only write the result of the winner thread.
  if (idx == minIdx) {
    saveBlockDXT1(bestStart, bestEnd, bestPermutation, xrefs, result,
                  blockOffset);
  }
}

// Helper structs and functions to validate the output of the compressor.
// We cannot simply do a bitwise compare, because different compilers produce
// different
// results for different targets due to floating point arithmetic.

union Color32 {
  struct {
    unsigned char b, g, r, a;
  };
  unsigned int u;
};

union Color16 {
  struct {
    unsigned short b : 5;
    unsigned short g : 6;
    unsigned short r : 5;
  };
  unsigned short u;
};

struct BlockDXT1 {
  Color16 col0;
  Color16 col1;
  union {
    unsigned char row[4];
    unsigned int indices;
  };

  void decompress(Color32 colors[16]) const;
};

void BlockDXT1::decompress(Color32 *colors) const {
  Color32 palette[4];

  // Does bit expansion before interpolation.
  palette[0].b = (col0.b << 3) | (col0.b >> 2);
  palette[0].g = (col0.g << 2) | (col0.g >> 4);
  palette[0].r = (col0.r << 3) | (col0.r >> 2);
  palette[0].a = 0xFF;

  palette[1].r = (col1.r << 3) | (col1.r >> 2);
  palette[1].g = (col1.g << 2) | (col1.g >> 4);
  palette[1].b = (col1.b << 3) | (col1.b >> 2);
  palette[1].a = 0xFF;

  if (col0.u > col1.u) {
    // Four-color block: derive the other two colors.
    palette[2].r = (2 * palette[0].r + palette[1].r) / 3;
    palette[2].g = (2 * palette[0].g + palette[1].g) / 3;
    palette[2].b = (2 * palette[0].b + palette[1].b) / 3;
    palette[2].a = 0xFF;

    palette[3].r = (2 * palette[1].r + palette[0].r) / 3;
    palette[3].g = (2 * palette[1].g + palette[0].g) / 3;
    palette[3].b = (2 * palette[1].b + palette[0].b) / 3;
    palette[3].a = 0xFF;
  } else {
    // Three-color block: derive the other color.
    palette[2].r = (palette[0].r + palette[1].r) / 2;
    palette[2].g = (palette[0].g + palette[1].g) / 2;
    palette[2].b = (palette[0].b + palette[1].b) / 2;
    palette[2].a = 0xFF;

    palette[3].r = 0x00;
    palette[3].g = 0x00;
    palette[3].b = 0x00;
    palette[3].a = 0x00;
  }

  for (int i = 0; i < 16; i++) {
    colors[i] = palette[(indices >> (2 * i)) & 0x3];
  }
}

static int compareColors(const Color32 *b0, const Color32 *b1) {
  int sum = 0;

  for (int i = 0; i < 16; i++) {
    int r = (b0[i].r - b1[i].r);
    int g = (b0[i].g - b1[i].g);
    int b = (b0[i].b - b1[i].b);
    sum += r * r + g * g + b * b;
  }

  return sum;
}

static int compareBlock(const BlockDXT1 *b0, const BlockDXT1 *b1) {
  Color32 colors0[16];
  Color32 colors1[16];

  if (memcmp(b0, b1, sizeof(BlockDXT1)) == 0) {
    return 0;
  } else {
    b0->decompress(colors0);
    b1->decompress(colors1);

    return compareColors(colors0, colors1);
  }
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  printf("%s Starting...\n\n", argv[0]);

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  findCudaDevice(argc, (const char **)argv);

  // Load input image.
  unsigned char *data = NULL;
  uint W, H;

  char *image_path = sdkFindFilePath(INPUT_IMAGE, argv[0]);

  if (image_path == 0) {
    printf("Error, unable to find source image  <%s>\n", image_path);
    exit(EXIT_FAILURE);
  }

  if (!sdkLoadPPM4ub(image_path, &data, &W, &H)) {
    printf("Error, unable to open source image file <%s>\n", image_path);

    exit(EXIT_FAILURE);
  }

  uint w = W, h = H;

  printf("Image Loaded '%s', %d x %d pixels\n\n", image_path, w, h);

  // Allocate input image.
  const uint memSize = w * h * 4;
  assert(0 != memSize);
  uint *block_image = (uint *)malloc(memSize);

  // Convert linear image to block linear.
  for (uint by = 0; by < h / 4; by++) {
    for (uint bx = 0; bx < w / 4; bx++) {
      for (int i = 0; i < 16; i++) {
        const int x = i & 3;
        const int y = i / 4;
        block_image[(by * w / 4 + bx) * 16 + i] =
            ((uint *)data)[(by * 4 + y) * 4 * (W / 4) + bx * 4 + x];
      }
    }
  }

  // copy into global mem
  uint *d_data = NULL;
  checkCudaErrors(hipMalloc((void **)&d_data, memSize));

  // Result
  uint *d_result = NULL;
  const uint compressedSize = (w / 4) * (h / 4) * 8;
  checkCudaErrors(hipMalloc((void **)&d_result, compressedSize));
  uint *h_result = (uint *)malloc(compressedSize);

  // Compute permutations.
  uint permutations[1024];
  computePermutations(permutations);

  // Copy permutations host to devie.
  uint *d_permutations = NULL;
  checkCudaErrors(hipMalloc((void **)&d_permutations, 1024 * sizeof(uint)));
  checkCudaErrors(hipMemcpy(d_permutations, permutations, 1024 * sizeof(uint),
                             hipMemcpyHostToDevice));

  // create a timer
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);

  // Copy image from host to device
  checkCudaErrors(
      hipMemcpy(d_data, block_image, memSize, hipMemcpyHostToDevice));

  // Determine launch configuration and run timed computation numIterations
  // times
  uint blocks = ((w + 3) / 4) *
                ((h + 3) / 4);  // rounds up by 1 block in each dim if %4 != 0

  int devID;
  hipDeviceProp_t deviceProp;

  // get number of SMs on this GPU
  checkCudaErrors(hipGetDevice(&devID));
  checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

  // Restrict the numbers of blocks to launch on low end GPUs to avoid kernel
  // timeout
  int blocksPerLaunch = min(blocks, 768 * deviceProp.multiProcessorCount);

  printf("Running DXT Compression on %u x %u image...\n", w, h);
  printf("\n%u Blocks, %u Threads per Block, %u Threads in Grid...\n\n", blocks,
         NUM_THREADS, blocks * NUM_THREADS);
  int numIterations = 1;

  for (int i = -1; i < numIterations; ++i) {
    if (i == 0) {
      checkCudaErrors(hipDeviceSynchronize());
      sdkStartTimer(&timer);
    }

    for (int j = 0; j < (int)blocks; j += blocksPerLaunch) {
      compress<<<min(blocksPerLaunch, blocks - j), NUM_THREADS>>>(
          d_permutations, d_data, (uint2 *)d_result, j);
    }
  }

  getLastCudaError("compress");

  // sync to host, stop timer, record perf
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  double dAvgTime = 1.0e-3 * sdkGetTimerValue(&timer) / (double)numIterations;
  printf(
      "dxtc, Throughput = %.4f MPixels/s, Time = %.5f s, Size = %u Pixels, "
      "NumDevsUsed = %i, Workgroup = %d\n",
      (1.0e-6 * (double)(W * H) / dAvgTime), dAvgTime, (W * H), 1, NUM_THREADS);

  // copy result data from device to host
  checkCudaErrors(
      hipMemcpy(h_result, d_result, compressedSize, hipMemcpyDeviceToHost));

  // Write out result data to DDS file
  char output_filename[1024];
  strcpy(output_filename, image_path);
  strcpy(output_filename + strlen(image_path) - 3, "dds");
  FILE *fp = fopen(output_filename, "wb");

  if (fp == 0) {
    printf("Error, unable to open output image <%s>\n", output_filename);
    exit(EXIT_FAILURE);
  }

  DDSHeader header;
  header.fourcc = FOURCC_DDS;
  header.size = 124;
  header.flags = (DDSD_WIDTH | DDSD_HEIGHT | DDSD_CAPS | DDSD_PIXELFORMAT |
                  DDSD_LINEARSIZE);
  header.height = h;
  header.width = w;
  header.pitch = compressedSize;
  header.depth = 0;
  header.mipmapcount = 0;
  memset(header.reserved, 0, sizeof(header.reserved));
  header.pf.size = 32;
  header.pf.flags = DDPF_FOURCC;
  header.pf.fourcc = FOURCC_DXT1;
  header.pf.bitcount = 0;
  header.pf.rmask = 0;
  header.pf.gmask = 0;
  header.pf.bmask = 0;
  header.pf.amask = 0;
  header.caps.caps1 = DDSCAPS_TEXTURE;
  header.caps.caps2 = 0;
  header.caps.caps3 = 0;
  header.caps.caps4 = 0;
  header.notused = 0;
  fwrite(&header, sizeof(DDSHeader), 1, fp);
  fwrite(h_result, compressedSize, 1, fp);
  fclose(fp);

  // Make sure the generated image is correct.
  const char *reference_image_path = sdkFindFilePath(REFERENCE_IMAGE, argv[0]);

  if (reference_image_path == 0) {
    printf("Error, unable to find reference image\n");

    exit(EXIT_FAILURE);
  }

  fp = fopen(reference_image_path, "rb");

  if (fp == 0) {
    printf("Error, unable to open reference image\n");

    exit(EXIT_FAILURE);
  }

  fseek(fp, sizeof(DDSHeader), SEEK_SET);
  uint referenceSize = (W / 4) * (H / 4) * 8;
  uint *reference = (uint *)malloc(referenceSize);
  fread(reference, referenceSize, 1, fp);
  fclose(fp);

  printf("\nChecking accuracy...\n");
  float rms = 0;

  for (uint y = 0; y < h; y += 4) {
    for (uint x = 0; x < w; x += 4) {
      uint referenceBlockIdx = ((y / 4) * (W / 4) + (x / 4));
      uint resultBlockIdx = ((y / 4) * (w / 4) + (x / 4));

      int cmp = compareBlock(((BlockDXT1 *)h_result) + resultBlockIdx,
                             ((BlockDXT1 *)reference) + referenceBlockIdx);

      if (cmp != 0.0f) {
        printf("Deviation at (%4d,%4d):\t%f rms\n", x / 4, y / 4,
               float(cmp) / 16 / 3);
      }

      rms += cmp;
    }
  }

  rms /= w * h * 3;

  // Free allocated resources and exit
  checkCudaErrors(hipFree(d_permutations));
  checkCudaErrors(hipFree(d_data));
  checkCudaErrors(hipFree(d_result));
  free(image_path);
  free(data);
  free(block_image);
  free(h_result);
  free(reference);
  sdkDeleteTimer(&timer);

  printf("RMS(reference, result) = %f\n\n", rms);
  printf(rms <= ERROR_THRESHOLD ? "Test passed\n" : "Test failed!\n");
  /* Return zero if test passed, one otherwise */
  return rms > ERROR_THRESHOLD;
}
