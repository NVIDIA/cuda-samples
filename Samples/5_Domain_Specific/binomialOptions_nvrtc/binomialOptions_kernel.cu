#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "common_gpu_header.h"
#include "binomialOptions_common.h"
#include "realtype.h"

// Preprocessed input option data
typedef struct {
  real S;
  real X;
  real vDt;
  real puByDf;
  real pdByDf;
} __TOptionData;
static __constant__ __TOptionData d_OptionData[MAX_OPTIONS];
__device__ real d_CallValue[MAX_OPTIONS];

#define THREADBLOCK_SIZE 128
#define ELEMS_PER_THREAD (NUM_STEPS / THREADBLOCK_SIZE)
#if NUM_STEPS % THREADBLOCK_SIZE
#error Bad constants
#endif

////////////////////////////////////////////////////////////////////////////////
// Overloaded shortcut functions for different precision modes
////////////////////////////////////////////////////////////////////////////////

#ifndef DOUBLE_PRECISION
__device__ inline float expiryCallValue(float S, float X, float vDt, int i) {
  float d = S * __expf(vDt * (2.0f * i - NUM_STEPS)) - X;
  return (d > 0.0F) ? d : 0.0F;
}

#else
__device__ inline double expiryCallValue(double S, double X, double vDt,
                                         int i) {
  double d = S * exp(vDt * (2.0 * i - NUM_STEPS)) - X;
  return (d > 0.0) ? d : 0.0;
}
#endif

////////////////////////////////////////////////////////////////////////////////
// GPU kernel
////////////////////////////////////////////////////////////////////////////////
extern "C" __global__ void binomialOptionsKernel() {
  __shared__ real call_exchange[THREADBLOCK_SIZE + 1];

  const int tid = threadIdx.x;
  const real S = d_OptionData[blockIdx.x].S;
  const real X = d_OptionData[blockIdx.x].X;
  const real vDt = d_OptionData[blockIdx.x].vDt;
  const real puByDf = d_OptionData[blockIdx.x].puByDf;
  const real pdByDf = d_OptionData[blockIdx.x].pdByDf;

  real call[ELEMS_PER_THREAD + 1];
#pragma unroll
  for (int i = 0; i < ELEMS_PER_THREAD; ++i)
    call[i] = expiryCallValue(S, X, vDt, tid * ELEMS_PER_THREAD + i);

  if (tid == 0)
    call_exchange[THREADBLOCK_SIZE] = expiryCallValue(S, X, vDt, NUM_STEPS);

  int final_it = max(0, tid * ELEMS_PER_THREAD - 1);

#pragma unroll 16
  for (int i = NUM_STEPS; i > 0; --i) {
    call_exchange[tid] = call[0];
    __syncthreads();
    call[ELEMS_PER_THREAD] = call_exchange[tid + 1];
    __syncthreads();

    if (i > final_it) {
#pragma unroll
      for (int j = 0; j < ELEMS_PER_THREAD; ++j)
        call[j] = puByDf * call[j + 1] + pdByDf * call[j];
    }
  }

  if (tid == 0) {
    d_CallValue[blockIdx.x] = call[0];
  }
}
