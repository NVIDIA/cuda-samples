/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/*
 * Paint a 3D texture with a gradient in X (blue) and Z (green), and have every
 * other Z slice have full red.
 */
__global__ void cuda_kernel_texture_3d(unsigned char *surface, int width,
                                       int height, int depth, size_t pitch,
                                       size_t pitchSlice, float t) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // in the case where, due to quantization into grids, we have
  // more threads than pixels, skip the threads which don't
  // correspond to valid pixels
  if (x >= width || y >= height) return;

  // walk across the Z slices of this texture.  it should be noted that
  // this is far from optimal data access.
  for (int z = 0; z < depth; ++z) {
    // get a pointer to this pixel
    unsigned char *pixel = surface + z * pitchSlice + y * pitch + 4 * x;
    pixel[0] = (unsigned char)(255.f * (0.5f + 0.5f * 
        cos(t + (x * x + y * y + z * z) * 0.0001f * 3.14f)));  // red
    pixel[1] = (unsigned char)(255.f * (0.5f + 0.5f * 
        sin(t + (x * x + y * y + z * z) * 0.0001f * 3.14f)));  // green
    pixel[2] = (unsigned char)0;                               // blue
    pixel[3] = 255;                                            // alpha
  }
}

extern "C" void cuda_texture_3d(void *surface, int width, int height, int depth,
                                size_t pitch, size_t pitchSlice, float t) {
  hipError_t error = hipSuccess;

  dim3 Db = dim3(16, 16);  // block dimensions are fixed to be 256 threads
  dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);

  cuda_kernel_texture_3d<<<Dg, Db>>>((unsigned char *)surface, width, height,
                                     depth, pitch, pitchSlice, t);

  error = hipGetLastError();

  if (error != hipSuccess) {
    printf("cuda_kernel_texture_3d() failed to launch error = %d\n", error);
  }
}
