#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * See: https://www.piday.org/million/
 */

#include "MonteCarloPi.h"
#include <algorithm>
#define CUDA_DRIVER_API
#include <hip/hip_runtime_api.h>
#include <iostream>

#define ROUND_UP_TO_GRANULARITY(x, n) (((x + n - 1) / n) * n)

// `ipcHandleTypeFlag` specifies the platform specific handle type this sample
// uses for importing and exporting memory allocation. On Linux this sample
// specifies the type as hipMemHandleTypePosixFileDescriptor meaning that
// file descriptors will be used. On Windows this sample specifies the type as
// hipMemHandleTypeWin32 meaning that NT HANDLEs will be used. The
// ipcHandleTypeFlag variable is a convenience variable and is passed by value
// to individual requests.
#if defined(__linux__)
hipMemAllocationHandleType ipcHandleTypeFlag =
    hipMemHandleTypePosixFileDescriptor;
#else
hipMemAllocationHandleType ipcHandleTypeFlag = hipMemHandleTypeWin32;
#endif

// Windows-specific LPSECURITYATTRIBUTES
void getDefaultSecurityDescriptor(hipMemAllocationProp *prop) {
#if defined(__linux__)
  return;
#elif defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
  static const char sddl[] = "D:P(OA;;GARCSDWDWOCCDCLCSWLODTWPRPCRFA;;;WD)";
  static OBJECT_ATTRIBUTES objAttributes;
  static bool objAttributesConfigured = false;

  if (!objAttributesConfigured) {
    PSECURITY_DESCRIPTOR secDesc;
    BOOL result = ConvertStringSecurityDescriptorToSecurityDescriptorA(
        sddl, SDDL_REVISION_1, &secDesc, NULL);
    if (result == 0) {
      printf("IPC failure: getDefaultSecurityDescriptor Failed! (%d)\n",
             GetLastError());
    }

    InitializeObjectAttributes(&objAttributes, NULL, 0, NULL, secDesc);

    objAttributesConfigured = true;
  }

  prop->win32HandleMetaData = &objAttributes;
  return;
#endif
}

__global__ void monte_carlo_kernel(vec2 *xyVector, float *pointsInsideCircle,
                                   float *numPointsInCircle,
                                   unsigned int numPoints, float time) {
  const size_t stride = gridDim.x * blockDim.x;
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  float count = 0.0f;

  hiprandState rgnState;
  hiprand_init((unsigned long long)time, tid, 0, &rgnState);

  for (; tid < numPoints; tid += stride) {
    float x = hiprand_uniform(&rgnState);
    float y = hiprand_uniform(&rgnState);
    x = (2.0f * x) - 1.0f;
    y = (2.0f * y) - 1.0f;
    xyVector[tid][0] = x;
    xyVector[tid][1] = y;

    // Compute the distance of this point form the center(0, 0)
    float dist = sqrtf((x * x) + (y * y));

    // If distance is less than the radius of the unit circle, the point lies in
    // the circle.
    pointsInsideCircle[tid] = (dist <= 1.0f);
    count += (dist <= 1.0f);
  }
  atomicAdd(numPointsInCircle, count);
}

MonteCarloPiSimulation::MonteCarloPiSimulation(size_t num_points)
    : m_xyVector(nullptr),
      m_pointsInsideCircle(nullptr),
      m_totalPointsInsideCircle(0),
      m_totalPointsSimulated(0),
      m_numPoints(num_points) {}

MonteCarloPiSimulation::~MonteCarloPiSimulation() {
  if (m_numPointsInCircle) {
    checkCudaErrors(hipFree(m_numPointsInCircle));
    m_numPointsInCircle = nullptr;
  }
  if (m_hostNumPointsInCircle) {
    checkCudaErrors(hipHostFree(m_hostNumPointsInCircle));
    m_hostNumPointsInCircle = nullptr;
  }

  cleanupSimulationAllocations();
}

void MonteCarloPiSimulation::initSimulation(int cudaDevice,
                                            hipStream_t stream) {
  m_cudaDevice = cudaDevice;
  getIdealExecutionConfiguration();

  // Allocate a position buffer that contains random location of the points in
  // XY cartesian plane.
  // Allocate a bitmap buffer which holds information of whether a point in the
  // position buffer is inside the unit circle or not.
  setupSimulationAllocations();

  checkCudaErrors(
      hipMalloc((float **)&m_numPointsInCircle, sizeof(*m_numPointsInCircle)));
  checkCudaErrors(hipHostMalloc((float **)&m_hostNumPointsInCircle,
                                 sizeof(*m_hostNumPointsInCircle)));
}

void MonteCarloPiSimulation::stepSimulation(float time, hipStream_t stream) {
  checkCudaErrors(hipMemsetAsync(m_numPointsInCircle, 0,
                                  sizeof(*m_numPointsInCircle), stream));

  monte_carlo_kernel<<<m_blocks, m_threads, 0, stream>>>(
      m_xyVector, m_pointsInsideCircle, m_numPointsInCircle, m_numPoints, time);
  getLastCudaError("Failed to launch CUDA simulation");

  checkCudaErrors(hipMemcpyAsync(m_hostNumPointsInCircle, m_numPointsInCircle,
                                  sizeof(*m_numPointsInCircle),
                                  hipMemcpyDeviceToHost, stream));

  // Queue up a stream callback to compute and print the PI value.
  checkCudaErrors(
      hipLaunchHostFunc(stream, this->computePiCallback, (void *)this));
}

void MonteCarloPiSimulation::computePiCallback(void *args) {
  MonteCarloPiSimulation *cbData = (MonteCarloPiSimulation *)args;
  cbData->m_totalPointsInsideCircle += *(cbData->m_hostNumPointsInCircle);
  cbData->m_totalPointsSimulated += cbData->m_numPoints;
  double piValue = 4.0 * ((double)cbData->m_totalPointsInsideCircle /
                          (double)cbData->m_totalPointsSimulated);
  printf("Approximate Pi value for %zd data points: %lf \n",
         cbData->m_totalPointsSimulated, piValue);
}

void MonteCarloPiSimulation::getIdealExecutionConfiguration() {
  int warpSize = 0;
  int multiProcessorCount = 0;

  checkCudaErrors(hipSetDevice(m_cudaDevice));
  checkCudaErrors(
      hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, m_cudaDevice));

  // We don't need large block sizes, since there's not much inter-thread
  // communication
  m_threads = warpSize;

  // Use the occupancy calculator and fill the gpu as best as we can
  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &m_blocks, monte_carlo_kernel, warpSize, 0));

  checkCudaErrors(hipDeviceGetAttribute(
      &multiProcessorCount, hipDeviceAttributeMultiprocessorCount, m_cudaDevice));
  m_blocks *= multiProcessorCount;

  // Go ahead and the clamp the blocks to the minimum needed for this
  // height/width
  m_blocks =
      std::min(m_blocks, (int)((m_numPoints + m_threads - 1) / m_threads));
}

void MonteCarloPiSimulation::setupSimulationAllocations() {
  hipDeviceptr_t d_ptr = 0U;
  size_t granularity = 0;
  hipMemGenericAllocationHandle_t cudaPositionHandle, cudaInCircleHandle;

  hipMemAllocationProp allocProp = {};
  allocProp.type = hipMemAllocationTypePinned;
  allocProp.location.type = hipMemLocationTypeDevice;
  allocProp.location.id = m_cudaDevice;
  allocProp.win32HandleMetaData = NULL;
  allocProp.requestedHandleTypes = ipcHandleTypeFlag;

  // Windows-specific LPSECURITYATTRIBUTES is required when
  // hipMemHandleTypeWin32 is used. The security attribute defines the scope
  // of which exported allocations may be tranferred to other processes. For all
  // other handle types, pass NULL.
  getDefaultSecurityDescriptor(&allocProp);

  // Get the recommended granularity for m_cudaDevice.
  checkCudaErrors(hipMemGetAllocationGranularity(
      &granularity, &allocProp, hipMemAllocationGranularityRecommended));

  size_t xyPositionVecSize = m_numPoints * sizeof(*m_xyVector);
  size_t inCircleVecSize = m_numPoints * sizeof(*m_pointsInsideCircle);

  size_t xyPositionSize =
      ROUND_UP_TO_GRANULARITY(xyPositionVecSize, granularity);
  size_t inCircleSize = ROUND_UP_TO_GRANULARITY(inCircleVecSize, granularity);
  m_totalAllocationSize = (xyPositionSize + inCircleSize);

  // Reserve the required contiguous VA space for the allocations
  checkCudaErrors(
      hipMemAddressReserve(&d_ptr, m_totalAllocationSize, granularity, 0U, 0));

  // Create the allocations as a pinned allocation on this device.
  // Create an allocation to store all the positions of points on the xy plane
  // and a second allocation which stores information if the corresponding
  // position is inside the unit circle or not.
  checkCudaErrors(
      hipMemCreate(&cudaPositionHandle, xyPositionSize, &allocProp, 0));
  checkCudaErrors(
      hipMemCreate(&cudaInCircleHandle, inCircleSize, &allocProp, 0));

  // Export the allocation to a platform-specific handle. The type of handle
  // requested here must match the requestedHandleTypes field in the prop
  // structure passed to hipMemCreate. The handle obtained here will be passed to
  // vulkan to import the allocation.
  checkCudaErrors(hipMemExportToShareableHandle(
      (void *)&m_posShareableHandle, cudaPositionHandle, ipcHandleTypeFlag, 0));
  checkCudaErrors(
      hipMemExportToShareableHandle((void *)&m_inCircleShareableHandle,
                                   cudaInCircleHandle, ipcHandleTypeFlag, 0));

  hipDeviceptr_t va_position = d_ptr;
  hipDeviceptr_t va_InCircle = va_position + xyPositionSize;
  m_pointsInsideCircle = (float *)va_InCircle;
  m_xyVector = (vec2 *)va_position;

  // Assign the chunk to the appropriate VA range
  checkCudaErrors(
      hipMemMap(va_position, xyPositionSize, 0, cudaPositionHandle, 0));
  checkCudaErrors(
      hipMemMap(va_InCircle, inCircleSize, 0, cudaInCircleHandle, 0));

  // Release the handles for the allocation. Since the allocation is currently
  // mapped to a VA range with a previous call to hipMemMap the actual freeing of
  // memory allocation will happen on an eventual call to hipMemUnmap. Thus the
  // allocation will be kept live until it is unmapped.
  checkCudaErrors(hipMemRelease(cudaPositionHandle));
  checkCudaErrors(hipMemRelease(cudaInCircleHandle));

  hipMemAccessDesc accessDescriptor = {};
  accessDescriptor.location.id = m_cudaDevice;
  accessDescriptor.location.type = hipMemLocationTypeDevice;
  accessDescriptor.flags = hipMemAccessFlagsProtReadWrite;

  // Apply the access descriptor to the whole VA range. Essentially enables
  // Read-Write access to the range.
  checkCudaErrors(
      hipMemSetAccess(d_ptr, m_totalAllocationSize, &accessDescriptor, 1));
}

void MonteCarloPiSimulation::cleanupSimulationAllocations() {
  if (m_xyVector && m_pointsInsideCircle) {
    // Unmap the mapped virtual memory region
    // Since the handles to the mapped backing stores have already been released
    // by hipMemRelease, and these are the only/last mappings referencing them,
    // The backing stores will be freed.
    checkCudaErrors(hipMemUnmap((hipDeviceptr_t)m_xyVector, m_totalAllocationSize));

    checkIpcErrors(ipcCloseShareableHandle(m_posShareableHandle));
    checkIpcErrors(ipcCloseShareableHandle(m_inCircleShareableHandle));

    // Free the virtual address region.
    checkCudaErrors(
        hipMemAddressFree((hipDeviceptr_t)m_xyVector, m_totalAllocationSize));

    m_xyVector = nullptr;
    m_pointsInsideCircle = nullptr;
  }
}
