#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef QUASIRANDOMGENERATOR_KERNEL_CUH
#define QUASIRANDOMGENERATOR_KERNEL_CUH

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>
#include "quasirandomGenerator_common.h"

// Fast integer multiplication
#define MUL(a, b) __umul24(a, b)

////////////////////////////////////////////////////////////////////////////////
// Niederreiter quasirandom number generation kernel
////////////////////////////////////////////////////////////////////////////////
static __constant__ unsigned int c_Table[QRNG_DIMENSIONS][QRNG_RESOLUTION];

static __global__ void quasirandomGeneratorKernel(float *d_Output,
                                                  unsigned int seed,
                                                  unsigned int N) {
  unsigned int *dimBase = &c_Table[threadIdx.y][0];
  unsigned int tid = MUL(blockDim.x, blockIdx.x) + threadIdx.x;
  unsigned int threadN = MUL(blockDim.x, gridDim.x);

  for (unsigned int pos = tid; pos < N; pos += threadN) {
    unsigned int result = 0;
    unsigned int data = seed + pos;

    for (int bit = 0; bit < QRNG_RESOLUTION; bit++, data >>= 1)
      if (data & 1) {
        result ^= dimBase[bit];
      }

    d_Output[MUL(threadIdx.y, N) + pos] = (float)(result + 1) * INT_SCALE;
  }
}

// Table initialization routine
extern "C" void initTableGPU(
    unsigned int tableCPU[QRNG_DIMENSIONS][QRNG_RESOLUTION]) {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(
      c_Table), tableCPU,
      QRNG_DIMENSIONS * QRNG_RESOLUTION * sizeof(unsigned int)));
}

// Host-side interface
extern "C" void quasirandomGeneratorGPU(float *d_Output, unsigned int seed,
                                        unsigned int N) {
  dim3 threads(128, QRNG_DIMENSIONS);
  quasirandomGeneratorKernel<<<128, threads>>>(d_Output, seed, N);
  getLastCudaError("quasirandomGeneratorKernel() execution failed.\n");
}

////////////////////////////////////////////////////////////////////////////////
// Moro's Inverse Cumulative Normal Distribution function approximation
////////////////////////////////////////////////////////////////////////////////
__device__ inline float MoroInvCNDgpu(unsigned int x) {
  const float a1 = 2.50662823884f;
  const float a2 = -18.61500062529f;
  const float a3 = 41.39119773534f;
  const float a4 = -25.44106049637f;
  const float b1 = -8.4735109309f;
  const float b2 = 23.08336743743f;
  const float b3 = -21.06224101826f;
  const float b4 = 3.13082909833f;
  const float c1 = 0.337475482272615f;
  const float c2 = 0.976169019091719f;
  const float c3 = 0.160797971491821f;
  const float c4 = 2.76438810333863E-02f;
  const float c5 = 3.8405729373609E-03f;
  const float c6 = 3.951896511919E-04f;
  const float c7 = 3.21767881768E-05f;
  const float c8 = 2.888167364E-07f;
  const float c9 = 3.960315187E-07f;

  float z;

  bool negate = false;

  // Ensure the conversion to floating point will give a value in the
  // range (0,0.5] by restricting the input to the bottom half of the
  // input domain. We will later reflect the result if the input was
  // originally in the top half of the input domain
  if (x >= 0x80000000UL) {
    x = 0xffffffffUL - x;
    negate = true;
  }

  // x is now in the range [0,0x80000000) (i.e. [0,0x7fffffff])
  // Convert to floating point in (0,0.5]
  const float x1 = 1.0f / static_cast<float>(0xffffffffUL);
  const float x2 = x1 / 2.0f;
  float p1 = x * x1 + x2;
  // Convert to floating point in (-0.5,0]
  float p2 = p1 - 0.5f;

  // The input to the Moro inversion is p2 which is in the range
  // (-0.5,0]. This means that our output will be the negative side
  // of the bell curve (which we will reflect if "negate" is true).

  // Main body of the bell curve for |p| < 0.42
  if (p2 > -0.42f) {
    z = p2 * p2;
    z = p2 * (((a4 * z + a3) * z + a2) * z + a1) /
        ((((b4 * z + b3) * z + b2) * z + b1) * z + 1.0f);
  }
  // Special case (Chebychev) for tail
  else {
    z = __logf(-__logf(p1));
    z = -(c1 + z * (c2 + z * (c3 + z * (c4 + z * (c5 + z * (c6 + z * (c7 + z 
        * (c8 + z * c9))))))));
  }

  // If the original input (x) was in the top half of the range, reflect
  // to get the positive side of the bell curve
  return negate ? -z : z;
}

////////////////////////////////////////////////////////////////////////////////
// Main kernel. Choose between transforming
// input sequence and uniform ascending (0, 1) sequence
////////////////////////////////////////////////////////////////////////////////
static __global__ void inverseCNDKernel(float *d_Output, unsigned int *d_Input,
                                        unsigned int pathN) {
  unsigned int distance = ((unsigned int)-1) / (pathN + 1);
  unsigned int tid = MUL(blockDim.x, blockIdx.x) + threadIdx.x;
  unsigned int threadN = MUL(blockDim.x, gridDim.x);

  // Transform input number sequence if it's supplied
  if (d_Input) {
    for (unsigned int pos = tid; pos < pathN; pos += threadN) {
      unsigned int d = d_Input[pos];
      d_Output[pos] = (float)MoroInvCNDgpu(d);
    }
  }
  // Else generate input uniformly placed samples on the fly
  // and write to destination
  else {
    for (unsigned int pos = tid; pos < pathN; pos += threadN) {
      unsigned int d = (pos + 1) * distance;
      d_Output[pos] = (float)MoroInvCNDgpu(d);
    }
  }
}

extern "C" void inverseCNDgpu(float *d_Output, unsigned int *d_Input,
                              unsigned int N) {
  inverseCNDKernel<<<128, 128>>>(d_Output, d_Input, N);
  getLastCudaError("inverseCNDKernel() execution failed.\n");
}

#endif
