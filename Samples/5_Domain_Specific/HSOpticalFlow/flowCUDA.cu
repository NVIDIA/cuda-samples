/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "common.h"

// include kernels
#include "downscaleKernel.cuh"
#include "upscaleKernel.cuh"
#include "warpingKernel.cuh"
#include "derivativesKernel.cuh"
#include "solverKernel.cuh"
#include "addKernel.cuh"

///////////////////////////////////////////////////////////////////////////////
/// \brief method logic
///
/// handles memory allocations, control flow
/// \param[in]  I0           source image
/// \param[in]  I1           tracked image
/// \param[in]  width        images width
/// \param[in]  height       images height
/// \param[in]  stride       images stride
/// \param[in]  alpha        degree of displacement field smoothness
/// \param[in]  nLevels      number of levels in a pyramid
/// \param[in]  nWarpIters   number of warping iterations per pyramid level
/// \param[in]  nSolverIters number of solver iterations (Jacobi iterations)
/// \param[out] u            horizontal displacement
/// \param[out] v            vertical displacement
///////////////////////////////////////////////////////////////////////////////
void ComputeFlowCUDA(const float *I0, const float *I1, int width, int height,
                     int stride, float alpha, int nLevels, int nWarpIters,
                     int nSolverIters, float *u, float *v) {
  printf("Computing optical flow on GPU...\n");

  // pI0 and pI1 will hold device pointers
  const float **pI0 = new const float *[nLevels];
  const float **pI1 = new const float *[nLevels];

  int *pW = new int[nLevels];
  int *pH = new int[nLevels];
  int *pS = new int[nLevels];

  // device memory pointers
  float *d_tmp;
  float *d_du0;
  float *d_dv0;
  float *d_du1;
  float *d_dv1;

  float *d_Ix;
  float *d_Iy;
  float *d_Iz;

  float *d_u;
  float *d_v;
  float *d_nu;
  float *d_nv;

  const int dataSize = stride * height * sizeof(float);

  checkCudaErrors(hipMalloc(&d_tmp, dataSize));
  checkCudaErrors(hipMalloc(&d_du0, dataSize));
  checkCudaErrors(hipMalloc(&d_dv0, dataSize));
  checkCudaErrors(hipMalloc(&d_du1, dataSize));
  checkCudaErrors(hipMalloc(&d_dv1, dataSize));

  checkCudaErrors(hipMalloc(&d_Ix, dataSize));
  checkCudaErrors(hipMalloc(&d_Iy, dataSize));
  checkCudaErrors(hipMalloc(&d_Iz, dataSize));

  checkCudaErrors(hipMalloc(&d_u, dataSize));
  checkCudaErrors(hipMalloc(&d_v, dataSize));
  checkCudaErrors(hipMalloc(&d_nu, dataSize));
  checkCudaErrors(hipMalloc(&d_nv, dataSize));

  // prepare pyramid

  int currentLevel = nLevels - 1;
  // allocate GPU memory for input images
  checkCudaErrors(hipMalloc(pI0 + currentLevel, dataSize));
  checkCudaErrors(hipMalloc(pI1 + currentLevel, dataSize));

  checkCudaErrors(hipMemcpy((void *)pI0[currentLevel], I0, dataSize,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy((void *)pI1[currentLevel], I1, dataSize,
                             hipMemcpyHostToDevice));

  pW[currentLevel] = width;
  pH[currentLevel] = height;
  pS[currentLevel] = stride;

  for (; currentLevel > 0; --currentLevel) {
    int nw = pW[currentLevel] / 2;
    int nh = pH[currentLevel] / 2;
    int ns = iAlignUp(nw);

    checkCudaErrors(
        hipMalloc(pI0 + currentLevel - 1, ns * nh * sizeof(float)));
    checkCudaErrors(
        hipMalloc(pI1 + currentLevel - 1, ns * nh * sizeof(float)));

    Downscale(pI0[currentLevel], pW[currentLevel], pH[currentLevel],
              pS[currentLevel], nw, nh, ns, (float *)pI0[currentLevel - 1]);

    Downscale(pI1[currentLevel], pW[currentLevel], pH[currentLevel],
              pS[currentLevel], nw, nh, ns, (float *)pI1[currentLevel - 1]);

    pW[currentLevel - 1] = nw;
    pH[currentLevel - 1] = nh;
    pS[currentLevel - 1] = ns;
  }

  checkCudaErrors(hipMemset(d_u, 0, stride * height * sizeof(float)));
  checkCudaErrors(hipMemset(d_v, 0, stride * height * sizeof(float)));

  // compute flow
  for (; currentLevel < nLevels; ++currentLevel) {
    for (int warpIter = 0; warpIter < nWarpIters; ++warpIter) {
      checkCudaErrors(hipMemset(d_du0, 0, dataSize));
      checkCudaErrors(hipMemset(d_dv0, 0, dataSize));

      checkCudaErrors(hipMemset(d_du1, 0, dataSize));
      checkCudaErrors(hipMemset(d_dv1, 0, dataSize));

      // on current level we compute optical flow
      // between frame 0 and warped frame 1
      WarpImage(pI1[currentLevel], pW[currentLevel], pH[currentLevel],
                pS[currentLevel], d_u, d_v, d_tmp);

      ComputeDerivatives(pI0[currentLevel], d_tmp, pW[currentLevel],
                         pH[currentLevel], pS[currentLevel], d_Ix, d_Iy, d_Iz);

      for (int iter = 0; iter < nSolverIters; ++iter) {
        SolveForUpdate(d_du0, d_dv0, d_Ix, d_Iy, d_Iz, pW[currentLevel],
                       pH[currentLevel], pS[currentLevel], alpha, d_du1, d_dv1);

        Swap(d_du0, d_du1);
        Swap(d_dv0, d_dv1);
      }

      // update u, v
      Add(d_u, d_du0, pH[currentLevel] * pS[currentLevel], d_u);
      Add(d_v, d_dv0, pH[currentLevel] * pS[currentLevel], d_v);
    }

    if (currentLevel != nLevels - 1) {
      // prolongate solution
      float scaleX = (float)pW[currentLevel + 1] / (float)pW[currentLevel];

      Upscale(d_u, pW[currentLevel], pH[currentLevel], pS[currentLevel],
              pW[currentLevel + 1], pH[currentLevel + 1], pS[currentLevel + 1],
              scaleX, d_nu);

      float scaleY = (float)pH[currentLevel + 1] / (float)pH[currentLevel];

      Upscale(d_v, pW[currentLevel], pH[currentLevel], pS[currentLevel],
              pW[currentLevel + 1], pH[currentLevel + 1], pS[currentLevel + 1],
              scaleY, d_nv);

      Swap(d_u, d_nu);
      Swap(d_v, d_nv);
    }
  }

  checkCudaErrors(hipMemcpy(u, d_u, dataSize, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(v, d_v, dataSize, hipMemcpyDeviceToHost));

  // cleanup
  for (int i = 0; i < nLevels; ++i) {
    checkCudaErrors(hipFree((void *)pI0[i]));
    checkCudaErrors(hipFree((void *)pI1[i]));
  }

  delete[] pI0;
  delete[] pI1;
  delete[] pW;
  delete[] pH;
  delete[] pS;

  checkCudaErrors(hipFree(d_tmp));
  checkCudaErrors(hipFree(d_du0));
  checkCudaErrors(hipFree(d_dv0));
  checkCudaErrors(hipFree(d_du1));
  checkCudaErrors(hipFree(d_dv1));
  checkCudaErrors(hipFree(d_Ix));
  checkCudaErrors(hipFree(d_Iy));
  checkCudaErrors(hipFree(d_Iz));
  checkCudaErrors(hipFree(d_nu));
  checkCudaErrors(hipFree(d_nv));
  checkCudaErrors(hipFree(d_u));
  checkCudaErrors(hipFree(d_v));
}
