#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include "fluidsD3D9_kernels.h"

// Texture object for reading velocity field
hipTextureObject_t texObj;
static hipArray *array = NULL;

void setupTexture(int x, int y) {
  hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();

  hipMallocArray(&array, &desc, y, x);
  getLastCudaError("hipMalloc failed");

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = array;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&texObj, &texRes, &texDescr, NULL));
}

void updateTexture(cData *data, size_t wib, size_t h, size_t pitch) {
  checkCudaErrors(hipMemcpy2DToArray(array, 0, 0, data, pitch, wib, h,
                                      hipMemcpyDeviceToDevice));
}

void deleteTexture(void) {
  checkCudaErrors(hipDestroyTextureObject(texObj));
  checkCudaErrors(hipFreeArray(array));
}

// Note that these kernels are designed to work with arbitrary
// domain sizes, not just domains that are multiples of the tile
// size. Therefore, we have extra code that checks to make sure
// a given thread location falls within the domain boundaries in
// both X and Y. Also, the domain is covered by looping over
// multiple elements in the Y direction, while there is a one-to-one
// mapping between threads in X and the tile size in X.
// Nolan Goodnight 9/22/06

// This method adds constant force vectors to the velocity field
// stored in 'v' according to v(x,t+1) = v(x,t) + dt * f.
__global__ void addForces_k(cData *v, int dx, int dy, int spx, int spy,
                            float fx, float fy, int r, size_t pitch) {
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  cData *fj = (cData *)((char *)v + (ty + spy) * pitch) + tx + spx;

  cData vterm = *fj;
  tx -= r;
  ty -= r;
  float s = 1.f / (1.f + tx * tx * tx * tx + ty * ty * ty * ty);
  vterm.x += s * fx;
  vterm.y += s * fy;
  *fj = vterm;
}

// This method performs the velocity advection step, where we
// trace velocity vectors back in time to update each grid cell.
// That is, v(x,t+1) = v(p(x,-dt),t). Here we perform bilinear
// interpolation in the velocity space.
__global__ void advectVelocity_k(cData *v, float *vx, float *vy, int dx,
                                 int pdx, int dy, float dt, int lb,
                                 hipTextureObject_t texObject) {
  int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
  int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
  int p;

  cData vterm, ploc;
  float vxterm, vyterm;

  // gtidx is the domain location in x for this thread
  if (gtidx < dx) {
    for (p = 0; p < lb; p++) {
      // fi is the domain location in y for this thread
      int fi = gtidy + p;

      if (fi < dy) {
        int fj = fi * pdx + gtidx;
        vterm = tex2D<cData>(texObject, (float)gtidx, (float)fi);
        ploc.x = (gtidx + 0.5f) - (dt * vterm.x * dx);
        ploc.y = (fi + 0.5f) - (dt * vterm.y * dy);
        vterm = tex2D<cData>(texObject, ploc.x, ploc.y);
        vxterm = vterm.x;
        vyterm = vterm.y;
        vx[fj] = vxterm;
        vy[fj] = vyterm;
      }
    }
  }
}

// This method performs velocity diffusion and forces mass conservation
// in the frequency domain. The inputs 'vx' and 'vy' are complex-valued
// arrays holding the Fourier coefficients of the velocity field in
// X and Y. Diffusion in this space takes a simple form described as:
// v(k,t) = v(k,t) / (1 + visc * dt * k^2), where visc is the viscosity,
// and k is the wavenumber. The projection step forces the Fourier
// velocity vectors to be orthogonal to the vectors for each
// wavenumber: v(k,t) = v(k,t) - ((k dot v(k,t) * k) / k^2.
__global__ void diffuseProject_k(cData *vx, cData *vy, int dx, int dy, float dt,
                                 float visc, int lb) {
  int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
  int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
  int p;

  cData xterm, yterm;

  // gtidx is the domain location in x for this thread
  if (gtidx < dx) {
    for (p = 0; p < lb; p++) {
      // fi is the domain location in y for this thread
      int fi = gtidy + p;

      if (fi < dy) {
        int fj = fi * dx + gtidx;
        xterm = vx[fj];
        yterm = vy[fj];

        // Compute the index of the wavenumber based on the
        // data order produced by a standard NN FFT.
        int iix = gtidx;
        int iiy = (fi > dy / 2) ? (fi - (dy)) : fi;

        // Velocity diffusion
        float kk = (float)(iix * iix + iiy * iiy);  // k^2
        float diff = 1.f / (1.f + visc * dt * kk);
        xterm.x *= diff;
        xterm.y *= diff;
        yterm.x *= diff;
        yterm.y *= diff;

        // Velocity projection
        if (kk > 0.f) {
          float rkk = 1.f / kk;
          // Real portion of velocity projection
          float rkp = (iix * xterm.x + iiy * yterm.x);
          // Imaginary portion of velocity projection
          float ikp = (iix * xterm.y + iiy * yterm.y);
          xterm.x -= rkk * rkp * iix;
          xterm.y -= rkk * ikp * iix;
          yterm.x -= rkk * rkp * iiy;
          yterm.y -= rkk * ikp * iiy;
        }

        vx[fj] = xterm;
        vy[fj] = yterm;
      }
    }
  }
}

// This method updates the velocity field 'v' using the two complex
// arrays from the previous step: 'vx' and 'vy'. Here we scale the
// real components by 1/(dx*dy) to account for an unnormalized FFT.
__global__ void updateVelocity_k(cData *v, float *vx, float *vy, int dx,
                                 int pdx, int dy, int lb, size_t pitch) {
  int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
  int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
  int p;

  float vxterm, vyterm;
  cData nvterm;

  // gtidx is the domain location in x for this thread
  if (gtidx < dx) {
    for (p = 0; p < lb; p++) {
      // fi is the domain location in y for this thread
      int fi = gtidy + p;

      if (fi < dy) {
        int fjr = fi * pdx + gtidx;
        vxterm = vx[fjr];
        vyterm = vy[fjr];

        // Normalize the result of the inverse FFT
        float scale = 1.f / (dx * dy);
        nvterm.x = vxterm * scale;
        nvterm.y = vyterm * scale;

        cData *fj = (cData *)((char *)v + fi * pitch) + gtidx;
        *fj = nvterm;
      }
    }  // If this thread is inside the domain in Y
  }    // If this thread is inside the domain in X
}

// This method updates the particles by moving particle positions
// according to the velocity field and time step. That is, for each
// particle: p(t+1) = p(t) + dt * v(p(t)).
__global__ void advectParticles_k(Vertex *part, cData *v, int dx, int dy,
                                  float dt, int lb, size_t pitch) {
  int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
  int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
  int p;

  // gtidx is the domain location in x for this thread
  cData vterm;
  Vertex pterm;

  if (gtidx < dx) {
    for (p = 0; p < lb; p++) {
      // fi is the domain location in y for this thread
      int fi = gtidy + p;

      if (fi < dy) {
        int fj = fi * dx + gtidx;
        pterm = part[fj];

        int xvi = ((int)(pterm.x * dx));
        int yvi = ((int)(pterm.y * dy));
        vterm = *((cData *)((char *)v + yvi * pitch) + xvi);

        pterm.x += dt * vterm.x;
        pterm.x = pterm.x - (int)pterm.x;
        pterm.x += 1.f;
        pterm.x = pterm.x - (int)pterm.x;
        pterm.y += dt * vterm.y;
        pterm.y = pterm.y - (int)pterm.y;
        pterm.y += 1.f;
        pterm.y = pterm.y - (int)pterm.y;

        part[fj] = pterm;
      }
    }  // If this thread is inside the domain in Y
  }    // If this thread is inside the domain in X
}

extern "C" void addForces(cData *v, int dx, int dy, int spx, int spy, float fx,
                          float fy, int r, size_t tPitch) {
  dim3 tids(2 * r + 1, 2 * r + 1);

  addForces_k<<<1, tids>>>(v, dx, dy, spx, spy, fx, fy, r, tPitch);
  getLastCudaError("addForces_k failed.");
}

extern "C" void advectVelocity(cData *v, float *vx, float *vy, int dx, int pdx,
                               int dy, float dt, size_t tPitch) {
  dim3 grid((dx / TILEX) + (!(dx % TILEX) ? 0 : 1),
            (dy / TILEY) + (!(dy % TILEY) ? 0 : 1));

  dim3 tids(TIDSX, TIDSY);

  updateTexture(v, DIM * sizeof(cData), DIM, tPitch);
  advectVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, dt, TILEY / TIDSY,
                                   texObj);

  getLastCudaError("advectVelocity_k failed.");
}

extern "C" void diffuseProject(cData *vx, cData *vy, int dx, int dy, float dt,
                               float visc, size_t tPitch) {
  // Forward FFT
  //    hipfftExecR2C(planr2c, (hipfftReal*)vx, (hipfftComplex*)vx);
  //    hipfftExecR2C(planr2c, (hipfftReal*)vy, (hipfftComplex*)vy);

  uint3 grid = make_uint3((dx / TILEX) + (!(dx % TILEX) ? 0 : 1),
                          (dy / TILEY) + (!(dy % TILEY) ? 0 : 1), 1);

  uint3 tids = make_uint3(TIDSX, TIDSY, 1);

  diffuseProject_k<<<grid, tids>>>(vx, vy, dx, dy, dt, visc, TILEY / TIDSY);
  getLastCudaError("diffuseProject_k failed.");

  // Inverse FFT
  //    hipfftExecC2R(planc2r, (hipfftComplex*)vx, (hipfftReal*)vx);
  //    hipfftExecC2R(planc2r, (hipfftComplex*)vy, (hipfftReal*)vy);
}

extern "C" void updateVelocity(cData *v, float *vx, float *vy, int dx, int pdx,
                               int dy, size_t tPitch) {
  dim3 grid((dx / TILEX) + (!(dx % TILEX) ? 0 : 1),
            (dy / TILEY) + (!(dy % TILEY) ? 0 : 1));

  dim3 tids(TIDSX, TIDSY);

  updateVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, TILEY / TIDSY,
                                   tPitch);
  getLastCudaError("updateVelocity_k failed.");
}

extern "C" void advectParticles(Vertex *p, cData *v, int dx, int dy, float dt,
                                size_t tPitch) {
  dim3 grid((dx / TILEX) + (!(dx % TILEX) ? 0 : 1),
            (dy / TILEY) + (!(dy % TILEY) ? 0 : 1));

  dim3 tids(TIDSX, TIDSY);

  advectParticles_k<<<grid, tids>>>(p, v, dx, dy, dt, TILEY / TIDSY, tPitch);
  getLastCudaError("advectParticles_k failed.");
}
