/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

//
// Paint a 2D texture with a moving red/green hatch pattern on a
// strobing blue background.  Note that this kernel reads to and
// writes from the texture, hence why this texture was not mapped
// as WriteDiscard.
//
__global__ void cuda_kernel_texture_2d(unsigned char *surface, int width,
                                       int height, size_t pitch, float t) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  float *pixel;

  // in the case where, due to quantization into grids, we have
  // more threads than pixels, skip the threads which don't
  // correspond to valid pixels
  if (x >= width || y >= height) return;

  // get a pointer to the pixel at (x,y)
  pixel = (float *)(surface + y * pitch) + 4 * x;

  // populate it
  float value_x = 0.5f + 0.5f * cos(t + 10.0f * ((2.0f * x) / width - 1.0f));
  float value_y = 0.5f + 0.5f * cos(t + 10.0f * ((2.0f * y) / height - 1.0f));
  pixel[0] = value_x > 0.5 ? 1 : 0;
  pixel[1] = value_y > 0.5 ? 1 : 0;
  pixel[2] = 0.5f + 0.5f * cos(t);
  pixel[3] = 1;  // alpha

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 4; ++j) {
      pixel[j] = sqrt(pixel[j]);
    }
  }

  for (int i = 0; i < 6; ++i) {
    for (int j = 0; j < 4; ++j) {
      pixel[j] *= pixel[j];
    }
  }
}

extern "C" void cuda_texture_2d(void *surface, int width, int height,
                                size_t pitch, float t) {
  hipError_t error = hipSuccess;

  dim3 Db = dim3(16, 16);  // block dimensions are fixed to be 256 threads
  dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);

  cuda_kernel_texture_2d<<<Dg, Db>>>((unsigned char *)surface, width, height,
                                     pitch, t);

  error = hipGetLastError();

  if (error != hipSuccess) {
    printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
  }
}
