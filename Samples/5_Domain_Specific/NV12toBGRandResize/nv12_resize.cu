#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Implements interlace NV12 frames batch resize

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "resize_convert.h"

__global__ static void resizeNV12BatchKernel(hipTextureObject_t texSrcLuma,
                                             hipTextureObject_t texSrcChroma,
                                             uint8_t *pDstNv12, int nSrcWidth,
                                             int nSrcHeight, int nDstPitch,
                                             int nDstWidth, int nDstHeight,
                                             int nBatchSize) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  int px = x * 2, py = y * 2;

  if ((px + 1) >= nDstWidth || (py + 1) >= nDstHeight) return;

  float fxScale = 1.0f * nSrcWidth / nDstWidth;
  float fyScale = 1.0f * nSrcHeight / nDstHeight;

  uint8_t *p = pDstNv12 + px + py * nDstPitch;
  int hh = nDstHeight * 3 / 2;
  int nByte = nDstPitch * hh;
  int px_fxScale = px * fxScale;
  int px_fxScale_1 = (px + 1) * fxScale;
  int py_fyScale = py * fyScale;
  int py_fyScale_1 = (py + 1) * fyScale;

  for (int i = blockIdx.z; i < nBatchSize; i+=gridDim.z) {
    *(uchar2 *)p = make_uchar2(tex2D<uint8_t>(texSrcLuma, px_fxScale, py_fyScale),
                          tex2D<uint8_t>(texSrcLuma, px_fxScale_1, py_fyScale));
    *(uchar2 *)(p + nDstPitch) =
        make_uchar2(tex2D<uint8_t>(texSrcLuma, px_fxScale, py_fyScale_1),
               tex2D<uint8_t>(texSrcLuma, px_fxScale_1, py_fyScale_1));
    *(uchar2 *)(p + (nDstHeight - y) * nDstPitch) = tex2D<uchar2>(
        texSrcChroma, x * fxScale, (hh * i + nDstHeight + y) * fyScale);
    p += nByte;
    py += hh;
  }
}

void resizeNV12Batch(uint8_t *dpSrc, int nSrcPitch, int nSrcWidth,
                     int nSrcHeight, uint8_t *dpDst, int nDstPitch,
                     int nDstWidth, int nDstHeight, int nBatchSize,
                     hipStream_t stream) {
  int hhSrc = ceilf(nSrcHeight * 3.0f / 2.0f);
  hipResourceDesc resDesc = {};
  resDesc.resType = hipResourceTypePitch2D;
  resDesc.res.pitch2D.devPtr = dpSrc;
  resDesc.res.pitch2D.desc = hipCreateChannelDesc<uint8_t>();
  resDesc.res.pitch2D.width = nSrcWidth;
  resDesc.res.pitch2D.height = hhSrc * nBatchSize;
  resDesc.res.pitch2D.pitchInBytes = nSrcPitch;

  hipTextureDesc texDesc = {};
  texDesc.filterMode = hipFilterModePoint;
  texDesc.readMode = hipReadModeElementType;

  hipTextureObject_t texLuma = 0;
  checkCudaErrors(hipCreateTextureObject(&texLuma, &resDesc, &texDesc, NULL));

  resDesc.res.pitch2D.desc = hipCreateChannelDesc<uchar2>();
  resDesc.res.pitch2D.width /= 2;

  hipTextureObject_t texChroma = 0;
  checkCudaErrors(hipCreateTextureObject(&texChroma, &resDesc, &texDesc, NULL));

  dim3 block(32, 32, 1);

  size_t blockDimZ = nBatchSize;

  // Restricting blocks in Z-dim till 32 to not launch too many blocks
  blockDimZ = (blockDimZ > 32) ? 32 : blockDimZ;

  dim3 grid((nDstWidth / 2 + block.x) / block.x,
            (nDstHeight / 2 + block.y) / block.y, blockDimZ);
  resizeNV12BatchKernel<<<grid, block, 0, stream>>>(
      texLuma, texChroma, dpDst, nSrcWidth, nSrcHeight, nDstPitch, nDstWidth,
      nDstHeight, nBatchSize);

  checkCudaErrors(hipDestroyTextureObject(texLuma));
  checkCudaErrors(hipDestroyTextureObject(texChroma));
}
