#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


// Implements NV12 to BGR batch conversion

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "resize_convert.h"

#define CONV_THREADS_X 64
#define CONV_THREADS_Y 10

__forceinline__ __device__ static float clampF(float x, float lower,
                                               float upper) {
  return x < lower ? lower : (x > upper ? upper : x);
}

__global__ static void nv12ToBGRplanarBatchKernel(const uint8_t *pNv12,
                                                  int nNv12Pitch, float *pBgr,
                                                  int nRgbPitch, int nWidth,
                                                  int nHeight, int nBatchSize) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;

  if ((x << 2) + 1 > nWidth || (y << 1) + 1 > nHeight) return;

  const uint8_t *__restrict__ pSrc = pNv12;

  for (int i = blockIdx.z; i < nBatchSize; i += gridDim.z) {
    pSrc = pNv12 + i * ((nHeight * nNv12Pitch * 3) >> 1) + (x << 2) +
           (y << 1) * nNv12Pitch;
    uchar4 luma2x01, luma2x23, uv2;
    *(uint32_t *)&luma2x01 = *(uint32_t *)pSrc;
    *(uint32_t *)&luma2x23 = *(uint32_t *)(pSrc + nNv12Pitch);
    *(uint32_t *)&uv2 = *(uint32_t *)(pSrc + (nHeight - y) * nNv12Pitch);

    float *pDstBlock = (pBgr + i * ((nHeight * nRgbPitch * 3) >> 2) +
                        ((blockIdx.x * blockDim.x) << 2) +
                        ((blockIdx.y * blockDim.y) << 1) * (nRgbPitch >> 2));

    float2 add1;
    float2 add2;
    float2 add3;
    float2 add00, add01, add02, add03;
    float2 d, e;

    add00.x = 1.1644f * luma2x01.x;
    add01.x = 1.1644f * luma2x01.y;
    add00.y = 1.1644f * luma2x01.z;
    add01.y = 1.1644f * luma2x01.w;

    add02.x = 1.1644f * luma2x23.x;
    add03.x = 1.1644f * luma2x23.y;
    add02.y = 1.1644f * luma2x23.z;
    add03.y = 1.1644f * luma2x23.w;

    d.x = uv2.x - 128.0f;
    e.x = uv2.y - 128.0f;
    d.y = uv2.z - 128.0f;
    e.y = uv2.w - 128.0f;

    add1.x = 2.0172f * d.x;
    add1.y = 2.0172f * d.y;

    add2.x = (-0.3918f) * d.x + (-0.8130f) * e.x;
    add2.y = (-0.3918f) * d.y + (-0.8130f) * e.y;

    add3.x = 1.5960f * e.x;
    add3.y = 1.5960f * e.y;

    int rowStride = (threadIdx.y << 1) * (nRgbPitch >> 2);
    int nextRowStride = ((threadIdx.y << 1) + 1) * (nRgbPitch >> 2);
    // B
    *((float4 *)&pDstBlock[rowStride + (threadIdx.x << 2)]) =
        make_float4(clampF(add00.x + add1.x, 0.0f, 255.0f),
                    clampF(add01.x + add1.x, 0.0f, 255.0f),
                    clampF(add00.y + add1.y, 0.0f, 255.0f),
                    clampF(add01.y + add1.y, 0.0f, 255.0f));
    *((float4 *)&pDstBlock[nextRowStride + (threadIdx.x << 2)]) =
        make_float4(clampF(add02.x + add1.x, 0.0f, 255.0f),
                    clampF(add03.x + add1.x, 0.0f, 255.0f),
                    clampF(add02.y + add1.y, 0.0f, 255.0f),
                    clampF(add03.y + add1.y, 0.0f, 255.0f));

    int planeStride = nHeight * nRgbPitch >> 2;
    // G
    *((float4 *)&pDstBlock[planeStride + rowStride + (threadIdx.x << 2)]) =
        make_float4(clampF(add00.x + add2.x, 0.0f, 255.0f),
                    clampF(add01.x + add2.x, 0.0f, 255.0f),
                    clampF(add00.y + add2.y, 0.0f, 255.0f),
                    clampF(add01.y + add2.y, 0.0f, 255.0f));
    *((float4 *)&pDstBlock[planeStride + nextRowStride + (threadIdx.x << 2)]) =
        make_float4(clampF(add02.x + add2.x, 0.0f, 255.0f),
                    clampF(add03.x + add2.x, 0.0f, 255.0f),
                    clampF(add02.y + add2.y, 0.0f, 255.0f),
                    clampF(add03.y + add2.y, 0.0f, 255.0f));

    // R
    *((float4
           *)&pDstBlock[(planeStride << 1) + rowStride + (threadIdx.x << 2)]) =
        make_float4(clampF(add00.x + add3.x, 0.0f, 255.0f),
                    clampF(add01.x + add3.x, 0.0f, 255.0f),
                    clampF(add00.y + add3.y, 0.0f, 255.0f),
                    clampF(add01.y + add3.y, 0.0f, 255.0f));
    *((float4 *)&pDstBlock[(planeStride << 1) + nextRowStride +
                           (threadIdx.x << 2)]) =
        make_float4(clampF(add02.x + add3.x, 0.0f, 255.0f),
                    clampF(add03.x + add3.x, 0.0f, 255.0f),
                    clampF(add02.y + add3.y, 0.0f, 255.0f),
                    clampF(add03.y + add3.y, 0.0f, 255.0f));
  }
}

void nv12ToBGRplanarBatch(uint8_t *pNv12, int nNv12Pitch, float *pBgr,
                          int nRgbPitch, int nWidth, int nHeight,
                          int nBatchSize, hipStream_t stream) {
  dim3 threads(CONV_THREADS_X, CONV_THREADS_Y);

  size_t blockDimZ = nBatchSize;

  // Restricting blocks in Z-dim till 32 to not launch too many blocks
  blockDimZ = (blockDimZ > 32) ? 32 : blockDimZ;

  dim3 blocks((nWidth / 4 - 1) / threads.x + 1,
              (nHeight / 2 - 1) / threads.y + 1, blockDimZ);
  nv12ToBGRplanarBatchKernel<<<blocks, threads, 0, stream>>>(
      pNv12, nNv12Pitch, pBgr, nRgbPitch, nWidth, nHeight, nBatchSize);
}
