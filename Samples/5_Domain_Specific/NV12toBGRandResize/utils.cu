#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdlib.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <fstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "resize_convert.h"
#include "utils.h"

__global__ void floatToChar(float *src, unsigned char *dst, int height,
                            int width, int batchSize) {
  int x = threadIdx.x + blockIdx.x * blockDim.x;

  if (x >= height * width) return;

  int offset = height * width * 3;

  for (int j = 0; j < batchSize; j++) {
    // b
    *(dst + j * offset + x * 3 + 0) =
        (unsigned char)*(src + j * offset + height * width * 0 + x);
    // g
    *(dst + j * offset + x * 3 + 1) =
        (unsigned char)*(src + j * offset + height * width * 1 + x);
    // r
    *(dst + j * offset + x * 3 + 2) =
        (unsigned char)*(src + j * offset + height * width * 2 + x);
  }
}

void floatPlanarToChar(float *src, unsigned char *dst, int height, int width,
                       int batchSize) {
  floatToChar<<<(height * width - 1) / 1024 + 1, 1024, 0, NULL>>>(
      src, dst, height, width, batchSize);
}

void dumpRawBGR(float *d_srcBGR, int pitch, int width, int height,
                int batchSize, char *folder, char *tag) {
  float *bgr, *d_bgr;
  int frameSize;
  char directory[120];
  char mkdir_cmd[256];
#if !defined(_WIN32)
  sprintf(directory, "output/%s", folder);
  sprintf(mkdir_cmd, "mkdir -p %s 2> /dev/null", directory);
#else
  sprintf(directory, "output\\%s", folder);
  sprintf(mkdir_cmd, "mkdir %s 2> nul", directory);
#endif

  int ret = system(mkdir_cmd);

  frameSize = width * height * 3 * sizeof(float);
  bgr = (float *)malloc(frameSize);
  if (bgr == NULL) {
    std::cerr << "Failed malloc for bgr\n";
    return;
  }

  d_bgr = d_srcBGR;
  for (int i = 0; i < batchSize; i++) {
    char filename[120];
    std::ofstream *outputFile;

    checkCudaErrors(hipMemcpy((void *)bgr, (void *)d_bgr, frameSize,
                               hipMemcpyDeviceToHost));
    sprintf(filename, "%s/%s_%d.raw", directory, tag, (i + 1));

    outputFile = new std::ofstream(filename);
    if (outputFile) {
      outputFile->write((char *)bgr, frameSize);
      delete outputFile;
    }

    d_bgr += pitch * height * 3;
  }

  free(bgr);
}

void dumpBGR(float *d_srcBGR, int pitch, int width, int height, int batchSize,
             char *folder, char *tag) {
  dumpRawBGR(d_srcBGR, pitch, width, height, batchSize, folder, tag);
}

void dumpYUV(unsigned char *d_nv12, int size, char *folder, char *tag) {
  unsigned char *nv12Data;
  std::ofstream *nv12File;
  char filename[120];
  char directory[60];
  char mkdir_cmd[256];
#if !defined(_WIN32)
  sprintf(directory, "output/%s", folder);
  sprintf(mkdir_cmd, "mkdir -p %s 2> /dev/null", directory);
#else
  sprintf(directory, "output\\%s", folder);
  sprintf(mkdir_cmd, "mkdir %s 2> nul", directory);
#endif

  int ret = system(mkdir_cmd);

  sprintf(filename, "%s/%s.nv12", directory, tag);

  nv12File = new std::ofstream(filename);
  if (nv12File == NULL) {
    std::cerr << "Failed to new " << filename;
    return;
  }

  nv12Data = (unsigned char *)malloc(size * (sizeof(char)));
  if (nv12Data == NULL) {
    std::cerr << "Failed to allcoate memory\n";
    return;
  }

  hipMemcpy((void *)nv12Data, (void *)d_nv12, size, hipMemcpyDeviceToHost);

  nv12File->write((const char *)nv12Data, size);

  free(nv12Data);
  delete nv12File;
}
