#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


// Implements BGR 3 progressive planars frames batch resize

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "resize_convert.h"

__global__ void resizeBGRplanarBatchKernel(hipTextureObject_t texSrc,
    float *pDst, int nDstPitch, int nDstHeight, int nSrcHeight,
    int batch, float scaleX, float scaleY,
    int cropX, int cropY, int cropW, int cropH) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if (x >= (int)(cropW/scaleX) || y >= (int)(cropH/scaleY))
        return;

    int frameSize = nDstPitch*nDstHeight;
    float *p = NULL;
    for (int i = blockIdx.z; i < batch; i += gridDim.z) {
        #pragma unroll
        for (int channel=0; channel < 3; channel++){
            p = pDst + i * 3 * frameSize + y * nDstPitch + x + channel * frameSize;
            *p = tex2D<float>(texSrc, x * scaleX + cropX,
                                ((3 * i + channel) * nSrcHeight + y * scaleY + cropY));
        }
    }
}


static void resizeBGRplanarBatchCore(
        float *dpSrc, int nSrcPitch, int nSrcWidth, int nSrcHeight,
        float *dpDst, int nDstPitch, int nDstWidth, int nDstHeight,
        int nBatchSize, hipStream_t stream, bool whSameResizeRatio,
        int cropX, int cropY, int cropW, int cropH) {
    hipTextureObject_t texSrc[2];
    int nTiles = 1, h, iTile;

    h = nSrcHeight * 3 * nBatchSize;
    while ((h + nTiles - 1) / nTiles > 65536)
        nTiles++;

    if (nTiles > 2)
        return;

    int batchTile = nBatchSize / nTiles;
    int batchTileLast = nBatchSize - batchTile * (nTiles-1);

    for (iTile = 0; iTile < nTiles; ++iTile) {
        int bs = (iTile == nTiles - 1) ? batchTileLast : batchTile;
        float *dpSrcNew = dpSrc +
            iTile * (batchTile * 3 * nSrcHeight * nSrcPitch);

        hipResourceDesc resDesc = {};
        resDesc.resType = hipResourceTypePitch2D;
        resDesc.res.pitch2D.devPtr = dpSrcNew;
        resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
        resDesc.res.pitch2D.width = nSrcWidth;
        resDesc.res.pitch2D.height = bs * 3 * nSrcHeight;
        resDesc.res.pitch2D.pitchInBytes = nSrcPitch * sizeof(float);
        hipTextureDesc texDesc = {};
        texDesc.filterMode = hipFilterModeLinear;
        texDesc.readMode = hipReadModeElementType;

        checkCudaErrors(hipCreateTextureObject(&texSrc[iTile], &resDesc, &texDesc, NULL));
        float *dpDstNew = dpDst +
            iTile * (batchTile * 3 * nDstHeight * nDstPitch);

        if(cropW == 0 || cropH == 0) {
            cropX = 0;
            cropY = 0;
            cropW = nSrcWidth;
            cropH = nSrcHeight;
        }

        float scaleX = (cropW*1.0f / nDstWidth);
        float scaleY = (cropH*1.0f / nDstHeight);

        if(whSameResizeRatio == true)
            scaleX = scaleY = scaleX > scaleY ? scaleX : scaleY;
        dim3 block(32, 32, 1);

        size_t blockDimZ = bs;
        // Restricting blocks in Z-dim till 32 to not launch too many blocks
        blockDimZ = (blockDimZ > 32) ? 32 : blockDimZ;
        dim3 grid((cropW*1.0f/scaleX + block.x - 1) / block.x,
                  (cropH*1.0f/scaleY + block.y - 1) / block.y, blockDimZ);

        resizeBGRplanarBatchKernel<<<grid, block, 0, stream>>>
                (texSrc[iTile], dpDstNew, nDstPitch, nDstHeight, nSrcHeight,
                bs, scaleX, scaleY, cropX, cropY, cropW, cropH);

    }

    for (iTile = 0; iTile < nTiles; ++iTile)
        checkCudaErrors(hipDestroyTextureObject(texSrc[iTile]));
}

void resizeBGRplanarBatch(
        float *dpSrc, int nSrcPitch, int nSrcWidth, int nSrcHeight,
        float *dpDst, int nDstPitch, int nDstWidth, int nDstHeight,
        int nBatchSize, hipStream_t stream,
        int cropX, int cropY, int cropW, int cropH, bool whSameResizeRatio) {
    resizeBGRplanarBatchCore(dpSrc, nSrcPitch, nSrcWidth, nSrcHeight,
        dpDst, nDstPitch, nDstWidth, nDstHeight, nBatchSize, stream,
        whSameResizeRatio, cropX, cropY, cropW, cropH);
}
