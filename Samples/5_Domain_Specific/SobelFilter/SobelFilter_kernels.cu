#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <helper_string.h>

#include "SobelFilter_kernels.h"

// Texture object for reading image
hipTextureObject_t texObject;
extern __shared__ unsigned char LocalBlock[];
static hipArray *array = NULL;

#define RADIUS 1

#ifdef FIXED_BLOCKWIDTH
#define BlockWidth 80
#define SharedPitch 384
#endif

// This will output the proper CUDA error strings in the event that a CUDA host
// call returns an error
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n", file, line,
            (int)err, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

__device__ unsigned char ComputeSobel(unsigned char ul,  // upper left
                                      unsigned char um,  // upper middle
                                      unsigned char ur,  // upper right
                                      unsigned char ml,  // middle left
                                      unsigned char mm,  // middle (unused)
                                      unsigned char mr,  // middle right
                                      unsigned char ll,  // lower left
                                      unsigned char lm,  // lower middle
                                      unsigned char lr,  // lower right
                                      float fScale) {
  short Horz = ur + 2 * mr + lr - ul - 2 * ml - ll;
  short Vert = ul + 2 * um + ur - ll - 2 * lm - lr;
  short Sum = (short)(fScale * (abs((int)Horz) + abs((int)Vert)));

  if (Sum < 0) {
    return 0;
  } else if (Sum > 0xff) {
    return 0xff;
  }

  return (unsigned char)Sum;
}

__global__ void SobelShared(uchar4 *pSobelOriginal, unsigned short SobelPitch,
#ifndef FIXED_BLOCKWIDTH
                            short BlockWidth, short SharedPitch,
#endif
                            short w, short h, float fScale,
                            hipTextureObject_t tex) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  short u = 4 * blockIdx.x * BlockWidth;
  short v = blockIdx.y * blockDim.y + threadIdx.y;
  short ib;

  int SharedIdx = threadIdx.y * SharedPitch;

  for (ib = threadIdx.x; ib < BlockWidth + 2 * RADIUS; ib += blockDim.x) {
    LocalBlock[SharedIdx + 4 * ib + 0] = tex2D<unsigned char>(
        tex, (float)(u + 4 * ib - RADIUS + 0), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 1] = tex2D<unsigned char>(
        tex, (float)(u + 4 * ib - RADIUS + 1), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 2] = tex2D<unsigned char>(
        tex, (float)(u + 4 * ib - RADIUS + 2), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 3] = tex2D<unsigned char>(
        tex, (float)(u + 4 * ib - RADIUS + 3), (float)(v - RADIUS));
  }

  if (threadIdx.y < RADIUS * 2) {
    //
    // copy trailing RADIUS*2 rows of pixels into shared
    //
    SharedIdx = (blockDim.y + threadIdx.y) * SharedPitch;

    for (ib = threadIdx.x; ib < BlockWidth + 2 * RADIUS; ib += blockDim.x) {
      LocalBlock[SharedIdx + 4 * ib + 0] =
          tex2D<unsigned char>(tex, (float)(u + 4 * ib - RADIUS + 0),
                               (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 1] =
          tex2D<unsigned char>(tex, (float)(u + 4 * ib - RADIUS + 1),
                               (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 2] =
          tex2D<unsigned char>(tex, (float)(u + 4 * ib - RADIUS + 2),
                               (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 3] =
          tex2D<unsigned char>(tex, (float)(u + 4 * ib - RADIUS + 3),
                               (float)(v + blockDim.y - RADIUS));
    }
  }

  cg::sync(cta);

  u >>= 2;  // index as uchar4 from here
  uchar4 *pSobel = (uchar4 *)(((char *)pSobelOriginal) + v * SobelPitch);
  SharedIdx = threadIdx.y * SharedPitch;

  for (ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x) {
    unsigned char pix00 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 0];
    unsigned char pix01 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 1];
    unsigned char pix02 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 2];
    unsigned char pix10 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 0];
    unsigned char pix11 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 1];
    unsigned char pix12 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 2];
    unsigned char pix20 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 0];
    unsigned char pix21 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 1];
    unsigned char pix22 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 2];

    uchar4 out;

    out.x = ComputeSobel(pix00, pix01, pix02, pix10, pix11, pix12, pix20, pix21,
                         pix22, fScale);

    pix00 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 3];
    pix10 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 3];
    pix20 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 3];
    out.y = ComputeSobel(pix01, pix02, pix00, pix11, pix12, pix10, pix21, pix22,
                         pix20, fScale);

    pix01 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 4];
    pix11 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 4];
    pix21 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 4];
    out.z = ComputeSobel(pix02, pix00, pix01, pix12, pix10, pix11, pix22, pix20,
                         pix21, fScale);

    pix02 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 5];
    pix12 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 5];
    pix22 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 5];
    out.w = ComputeSobel(pix00, pix01, pix02, pix10, pix11, pix12, pix20, pix21,
                         pix22, fScale);

    if (u + ib < w / 4 && v < h) {
      pSobel[u + ib] = out;
    }
  }

  cg::sync(cta);
}

__global__ void SobelCopyImage(Pixel *pSobelOriginal, unsigned int Pitch, int w,
                               int h, float fscale, hipTextureObject_t tex) {
  unsigned char *pSobel =
      (unsigned char *)(((char *)pSobelOriginal) + blockIdx.x * Pitch);

  for (int i = threadIdx.x; i < w; i += blockDim.x) {
    pSobel[i] = min(
        max((tex2D<unsigned char>(tex, (float)i, (float)blockIdx.x) * fscale),
            0.f),
        255.f);
  }
}

__global__ void SobelTex(Pixel *pSobelOriginal, unsigned int Pitch, int w,
                         int h, float fScale, hipTextureObject_t tex) {
  unsigned char *pSobel =
      (unsigned char *)(((char *)pSobelOriginal) + blockIdx.x * Pitch);

  for (int i = threadIdx.x; i < w; i += blockDim.x) {
    unsigned char pix00 =
        tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x - 1);
    unsigned char pix01 =
        tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x - 1);
    unsigned char pix02 =
        tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x - 1);
    unsigned char pix10 =
        tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x + 0);
    unsigned char pix11 =
        tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x + 0);
    unsigned char pix12 =
        tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x + 0);
    unsigned char pix20 =
        tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x + 1);
    unsigned char pix21 =
        tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x + 1);
    unsigned char pix22 =
        tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x + 1);
    pSobel[i] = ComputeSobel(pix00, pix01, pix02, pix10, pix11, pix12, pix20,
                             pix21, pix22, fScale);
  }
}

extern "C" void setupTexture(int iw, int ih, Pixel *data, int Bpp) {
  hipChannelFormatDesc desc;

  if (Bpp == 1) {
    desc = hipCreateChannelDesc<unsigned char>();
  } else {
    desc = hipCreateChannelDesc<uchar4>();
  }

  checkCudaErrors(hipMallocArray(&array, &desc, iw, ih));
  checkCudaErrors(hipMemcpy2DToArray(
      array, 0, 0, data, iw * Bpp * sizeof(Pixel), iw * Bpp * sizeof(Pixel), ih,
      hipMemcpyHostToDevice));

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = array;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&texObject, &texRes, &texDescr, NULL));
}

extern "C" void deleteTexture(void) {
  checkCudaErrors(hipFreeArray(array));
  checkCudaErrors(hipDestroyTextureObject(texObject));
}

// Wrapper for the __global__ call that sets up the texture and threads
extern "C" void sobelFilter(Pixel *odata, int iw, int ih,
                            enum SobelDisplayMode mode, float fScale) {
  switch (mode) {
    case SOBELDISPLAY_IMAGE:
      SobelCopyImage<<<ih, 384>>>(odata, iw, iw, ih, fScale, texObject);
      break;

    case SOBELDISPLAY_SOBELTEX:
      SobelTex<<<ih, 384>>>(odata, iw, iw, ih, fScale, texObject);
      break;

    case SOBELDISPLAY_SOBELSHARED: {
      dim3 threads(16, 4);
#ifndef FIXED_BLOCKWIDTH
      int BlockWidth = 80;  // must be divisible by 16 for coalescing
#endif
      dim3 blocks = dim3(iw / (4 * BlockWidth) + (0 != iw % (4 * BlockWidth)),
                         ih / threads.y + (0 != ih % threads.y));
      int SharedPitch = ~0x3f & (4 * (BlockWidth + 2 * RADIUS) + 0x3f);
      int sharedMem = SharedPitch * (threads.y + 2 * RADIUS);

      // for the shared kernel, width must be divisible by 4
      iw &= ~3;

      SobelShared<<<blocks, threads, sharedMem>>>((uchar4 *)odata, iw,
#ifndef FIXED_BLOCKWIDTH
                                                  BlockWidth, SharedPitch,
#endif
                                                  iw, ih, fScale, texObject);
    } break;
  }
}
