#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

////////////////////////////////////////////////////////////////////////////////
// Global types and parameters
////////////////////////////////////////////////////////////////////////////////
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <hip/hip_runtime_api.h>
#include "binomialOptions_common.h"
#include "realtype.h"

// Preprocessed input option data
typedef struct {
  real S;
  real X;
  real vDt;
  real puByDf;
  real pdByDf;
} __TOptionData;
static __constant__ __TOptionData d_OptionData[MAX_OPTIONS];
static __device__ real d_CallValue[MAX_OPTIONS];

////////////////////////////////////////////////////////////////////////////////
// Overloaded shortcut functions for different precision modes
////////////////////////////////////////////////////////////////////////////////
#ifndef DOUBLE_PRECISION
__device__ inline float expiryCallValue(float S, float X, float vDt, int i) {
  float d = S * __expf(vDt * (2.0f * i - NUM_STEPS)) - X;
  return (d > 0.0F) ? d : 0.0F;
}
#else
__device__ inline double expiryCallValue(double S, double X, double vDt,
                                         int i) {
  double d = S * exp(vDt * (2.0 * i - NUM_STEPS)) - X;
  return (d > 0.0) ? d : 0.0;
}
#endif

////////////////////////////////////////////////////////////////////////////////
// GPU kernel
////////////////////////////////////////////////////////////////////////////////
#define THREADBLOCK_SIZE 128
#define ELEMS_PER_THREAD (NUM_STEPS / THREADBLOCK_SIZE)
#if NUM_STEPS % THREADBLOCK_SIZE
#error Bad constants
#endif

__global__ void binomialOptionsKernel() {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ real call_exchange[THREADBLOCK_SIZE + 1];

  const int tid = threadIdx.x;
  const real S = d_OptionData[blockIdx.x].S;
  const real X = d_OptionData[blockIdx.x].X;
  const real vDt = d_OptionData[blockIdx.x].vDt;
  const real puByDf = d_OptionData[blockIdx.x].puByDf;
  const real pdByDf = d_OptionData[blockIdx.x].pdByDf;

  real call[ELEMS_PER_THREAD + 1];
#pragma unroll
  for (int i = 0; i < ELEMS_PER_THREAD; ++i)
    call[i] = expiryCallValue(S, X, vDt, tid * ELEMS_PER_THREAD + i);

  if (tid == 0)
    call_exchange[THREADBLOCK_SIZE] = expiryCallValue(S, X, vDt, NUM_STEPS);

  int final_it = max(0, tid * ELEMS_PER_THREAD - 1);

#pragma unroll 16
  for (int i = NUM_STEPS; i > 0; --i) {
    call_exchange[tid] = call[0];
    cg::sync(cta);
    call[ELEMS_PER_THREAD] = call_exchange[tid + 1];
    cg::sync(cta);

    if (i > final_it) {
#pragma unroll
      for (int j = 0; j < ELEMS_PER_THREAD; ++j)
        call[j] = puByDf * call[j + 1] + pdByDf * call[j];
    }
  }

  if (tid == 0) {
    d_CallValue[blockIdx.x] = call[0];
  }
}

////////////////////////////////////////////////////////////////////////////////
// Host-side interface to GPU binomialOptions
////////////////////////////////////////////////////////////////////////////////
extern "C" void binomialOptionsGPU(real *callValue, TOptionData *optionData,
                                   int optN) {
  __TOptionData h_OptionData[MAX_OPTIONS];

  for (int i = 0; i < optN; i++) {
    const real T = optionData[i].T;
    const real R = optionData[i].R;
    const real V = optionData[i].V;

    const real dt = T / (real)NUM_STEPS;
    const real vDt = V * sqrt(dt);
    const real rDt = R * dt;
    // Per-step interest and discount factors
    const real If = exp(rDt);
    const real Df = exp(-rDt);
    // Values and pseudoprobabilities of upward and downward moves
    const real u = exp(vDt);
    const real d = exp(-vDt);
    const real pu = (If - d) / (u - d);
    const real pd = (real)1.0 - pu;
    const real puByDf = pu * Df;
    const real pdByDf = pd * Df;

    h_OptionData[i].S = (real)optionData[i].S;
    h_OptionData[i].X = (real)optionData[i].X;
    h_OptionData[i].vDt = (real)vDt;
    h_OptionData[i].puByDf = (real)puByDf;
    h_OptionData[i].pdByDf = (real)pdByDf;
  }

  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_OptionData), h_OptionData,
                                     optN * sizeof(__TOptionData)));
  binomialOptionsKernel<<<optN, THREADBLOCK_SIZE>>>();
  getLastCudaError("binomialOptionsKernel() execution failed.\n");
  checkCudaErrors(
      hipMemcpyFromSymbol(callValue, HIP_SYMBOL(d_CallValue), optN * sizeof(real)));
}
