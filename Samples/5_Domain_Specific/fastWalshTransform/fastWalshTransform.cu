#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Walsh transforms belong to a class of generalized Fourier transformations.
 * They have applications in various fields of electrical engineering
 * and numeric theory. In this sample we demonstrate efficient implementation
 * of naturally-ordered Walsh transform
 * (also known as Walsh-Hadamard or Hadamard transform) in CUDA and its
 * particular application to dyadic convolution computation.
 * Refer to excellent Jorg Arndt's "Algorithms for Programmers" textbook
 * http://www.jjj.de/fxt/fxtbook.pdf (Chapter 22)
 *
 * Victor Podlozhnyuk (vpodlozhnyuk@nvidia.com)
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////////////
// Reference CPU FWT
////////////////////////////////////////////////////////////////////////////////
extern "C" void fwtCPU(float *h_Output, float *h_Input, int log2N);
extern "C" void slowWTcpu(float *h_Output, float *h_Input, int log2N);
extern "C" void dyadicConvolutionCPU(float *h_Result, float *h_Data,
                                     float *h_Kernel, int log2dataN,
                                     int log2kernelN);

////////////////////////////////////////////////////////////////////////////////
// GPU FWT
////////////////////////////////////////////////////////////////////////////////
#include "fastWalshTransform_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int log2Kernel = 7;
const int log2Data = 23;

const int dataN = 1 << log2Data;
const int kernelN = 1 << log2Kernel;

const int DATA_SIZE = dataN * sizeof(float);
const int KERNEL_SIZE = kernelN * sizeof(float);

const double NOPS = 3.0 * (double)dataN * (double)log2Data / 2.0;

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char *argv[]) {
  float *h_Data, *h_Kernel, *h_ResultCPU, *h_ResultGPU;

  float *d_Data, *d_Kernel;

  double delta, ref, sum_delta2, sum_ref2, L2norm, gpuTime;

  StopWatchInterface *hTimer = NULL;
  int i;

  printf("%s Starting...\n\n", argv[0]);

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  findCudaDevice(argc, (const char **)argv);

  sdkCreateTimer(&hTimer);

  printf("Initializing data...\n");
  printf("...allocating CPU memory\n");
  h_Kernel = (float *)malloc(KERNEL_SIZE);
  h_Data = (float *)malloc(DATA_SIZE);
  h_ResultCPU = (float *)malloc(DATA_SIZE);
  h_ResultGPU = (float *)malloc(DATA_SIZE);
  printf("...allocating GPU memory\n");
  checkCudaErrors(hipMalloc((void **)&d_Kernel, DATA_SIZE));
  checkCudaErrors(hipMalloc((void **)&d_Data, DATA_SIZE));

  printf("...generating data\n");
  printf("Data length: %i; kernel length: %i\n", dataN, kernelN);
  srand(2007);

  for (i = 0; i < kernelN; i++) {
    h_Kernel[i] = (float)rand() / (float)RAND_MAX;
  }

  for (i = 0; i < dataN; i++) {
    h_Data[i] = (float)rand() / (float)RAND_MAX;
  }

  checkCudaErrors(hipMemset(d_Kernel, 0, DATA_SIZE));
  checkCudaErrors(
      hipMemcpy(d_Kernel, h_Kernel, KERNEL_SIZE, hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpy(d_Data, h_Data, DATA_SIZE, hipMemcpyHostToDevice));

  printf("Running GPU dyadic convolution using Fast Walsh Transform...\n");
  checkCudaErrors(hipDeviceSynchronize());
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);
  fwtBatchGPU(d_Data, 1, log2Data);
  fwtBatchGPU(d_Kernel, 1, log2Data);
  modulateGPU(d_Data, d_Kernel, dataN);
  fwtBatchGPU(d_Data, 1, log2Data);
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&hTimer);
  gpuTime = sdkGetTimerValue(&hTimer);
  printf("GPU time: %f ms; GOP/s: %f\n", gpuTime,
         NOPS / (gpuTime * 0.001 * 1E+9));

  printf("Reading back GPU results...\n");
  checkCudaErrors(
      hipMemcpy(h_ResultGPU, d_Data, DATA_SIZE, hipMemcpyDeviceToHost));

  printf("Running straightforward CPU dyadic convolution...\n");
  dyadicConvolutionCPU(h_ResultCPU, h_Data, h_Kernel, log2Data, log2Kernel);

  printf("Comparing the results...\n");
  sum_delta2 = 0;
  sum_ref2 = 0;

  for (i = 0; i < dataN; i++) {
    delta = h_ResultCPU[i] - h_ResultGPU[i];
    ref = h_ResultCPU[i];
    sum_delta2 += delta * delta;
    sum_ref2 += ref * ref;
  }

  L2norm = sqrt(sum_delta2 / sum_ref2);

  printf("Shutting down...\n");
  sdkDeleteTimer(&hTimer);
  checkCudaErrors(hipFree(d_Data));
  checkCudaErrors(hipFree(d_Kernel));
  free(h_ResultGPU);
  free(h_ResultCPU);
  free(h_Data);
  free(h_Kernel);

  printf("L2 norm: %E\n", L2norm);
  printf(L2norm < 1e-6 ? "Test passed\n" : "Test failed!\n");
}
