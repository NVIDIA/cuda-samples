#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BICUBICTEXTURE_CU_
#define _BICUBICTEXTURE_CU_

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include <hip/hip_vector_types.h>

// includes, cuda
#include <hip/hip_runtime_api.h>

typedef unsigned int uint;
typedef unsigned char uchar;

#include "bicubicTexture_kernel.cuh"

hipArray *d_imageArray = 0;

extern "C" void initTexture(int imageWidth, int imageHeight, uchar *h_data) {
  // allocate array and copy image data
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
  checkCudaErrors(
      hipMallocArray(&d_imageArray, &channelDesc, imageWidth, imageHeight));
  checkCudaErrors(hipMemcpy2DToArray(
      d_imageArray, 0, 0, h_data, imageWidth * sizeof(uchar),
      imageWidth * sizeof(uchar), imageHeight, hipMemcpyHostToDevice));
  free(h_data);

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_imageArray;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.addressMode[1] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(
      hipCreateTextureObject(&texObjLinear, &texRes, &texDescr, NULL));

  memset(&texDescr, 0, sizeof(hipTextureDesc));
  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.addressMode[1] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(
      hipCreateTextureObject(&texObjPoint, &texRes, &texDescr, NULL));
}

extern "C" void freeTexture() {
  checkCudaErrors(hipDestroyTextureObject(texObjPoint));
  checkCudaErrors(hipDestroyTextureObject(texObjLinear));
  checkCudaErrors(hipFreeArray(d_imageArray));
}

// render image using CUDA
extern "C" void render(int width, int height, float tx, float ty, float scale,
                       float cx, float cy, dim3 blockSize, dim3 gridSize,
                       int filter_mode, uchar4 *output) {
  // call CUDA kernel, writing results to PBO memory
  switch (filter_mode) {
    case MODE_NEAREST:
      d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale,
                                        cx, cy, texObjPoint);
      break;

    case MODE_BILINEAR:
      d_render<<<gridSize, blockSize>>>(output, width, height, tx, ty, scale,
                                        cx, cy, texObjLinear);
      break;

    case MODE_BICUBIC:
      d_renderBicubic<<<gridSize, blockSize>>>(output, width, height, tx, ty,
                                               scale, cx, cy, texObjPoint);
      break;

    case MODE_FAST_BICUBIC:
      d_renderFastBicubic<<<gridSize, blockSize>>>(
          output, width, height, tx, ty, scale, cx, cy, texObjLinear);
      break;

    case MODE_CATROM:
      d_renderCatRom<<<gridSize, blockSize>>>(output, width, height, tx, ty,
                                              scale, cx, cy, texObjPoint);
      break;
  }

  getLastCudaError("kernel failed");
}

#endif
