#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample evaluates fair call and put prices for a
 * given set of European options by Black-Scholes formula.
 * See supplied whitepaper for more explanations.
 */

#include <helper_functions.h>  // helper functions for string parsing
#include <hip/hip_runtime_api.h>  // helper functions CUDA error checking and initialization

////////////////////////////////////////////////////////////////////////////////
// Process an array of optN options on CPU
////////////////////////////////////////////////////////////////////////////////
extern "C" void BlackScholesCPU(float *h_CallResult, float *h_PutResult,
                                float *h_StockPrice, float *h_OptionStrike,
                                float *h_OptionYears, float Riskfree,
                                float Volatility, int optN);

////////////////////////////////////////////////////////////////////////////////
// Process an array of OptN options on GPU
////////////////////////////////////////////////////////////////////////////////
#include "BlackScholes_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}

////////////////////////////////////////////////////////////////////////////////
// Data configuration
////////////////////////////////////////////////////////////////////////////////
const int OPT_N = 4000000;
const int NUM_ITERATIONS = 512;

const int OPT_SZ = OPT_N * sizeof(float);
const float RISKFREE = 0.02f;
const float VOLATILITY = 0.30f;

#define DIV_UP(a, b) (((a) + (b)-1) / (b))

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  // Start logs
  printf("[%s] - Starting...\n", argv[0]);

  //'h_' prefix - CPU (host) memory space
  float
      // Results calculated by CPU for reference
      *h_CallResultCPU,
      *h_PutResultCPU,
      // CPU copy of GPU results
      *h_CallResultGPU, *h_PutResultGPU,
      // CPU instance of input data
      *h_StockPrice, *h_OptionStrike, *h_OptionYears;

  //'d_' prefix - GPU (device) memory space
  float
      // Results calculated by GPU
      *d_CallResult,
      *d_PutResult,
      // GPU instance of input data
      *d_StockPrice, *d_OptionStrike, *d_OptionYears;

  double delta, ref, sum_delta, sum_ref, max_delta, L1norm, gpuTime;

  StopWatchInterface *hTimer = NULL;
  int i;

  findCudaDevice(argc, (const char **)argv);

  sdkCreateTimer(&hTimer);

  printf("Initializing data...\n");
  printf("...allocating CPU memory for options.\n");
  h_CallResultCPU = (float *)malloc(OPT_SZ);
  h_PutResultCPU = (float *)malloc(OPT_SZ);
  h_CallResultGPU = (float *)malloc(OPT_SZ);
  h_PutResultGPU = (float *)malloc(OPT_SZ);
  h_StockPrice = (float *)malloc(OPT_SZ);
  h_OptionStrike = (float *)malloc(OPT_SZ);
  h_OptionYears = (float *)malloc(OPT_SZ);

  printf("...allocating GPU memory for options.\n");
  checkCudaErrors(hipMalloc((void **)&d_CallResult, OPT_SZ));
  checkCudaErrors(hipMalloc((void **)&d_PutResult, OPT_SZ));
  checkCudaErrors(hipMalloc((void **)&d_StockPrice, OPT_SZ));
  checkCudaErrors(hipMalloc((void **)&d_OptionStrike, OPT_SZ));
  checkCudaErrors(hipMalloc((void **)&d_OptionYears, OPT_SZ));

  printf("...generating input data in CPU mem.\n");
  srand(5347);

  // Generate options set
  for (i = 0; i < OPT_N; i++) {
    h_CallResultCPU[i] = 0.0f;
    h_PutResultCPU[i] = -1.0f;
    h_StockPrice[i] = RandFloat(5.0f, 30.0f);
    h_OptionStrike[i] = RandFloat(1.0f, 100.0f);
    h_OptionYears[i] = RandFloat(0.25f, 10.0f);
  }

  printf("...copying input data to GPU mem.\n");
  // Copy options data to GPU memory for further processing
  checkCudaErrors(
      hipMemcpy(d_StockPrice, h_StockPrice, OPT_SZ, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_OptionStrike, h_OptionStrike, OPT_SZ,
                             hipMemcpyHostToDevice));
  checkCudaErrors(
      hipMemcpy(d_OptionYears, h_OptionYears, OPT_SZ, hipMemcpyHostToDevice));
  printf("Data init done.\n\n");

  printf("Executing Black-Scholes GPU kernel (%i iterations)...\n",
         NUM_ITERATIONS);
  checkCudaErrors(hipDeviceSynchronize());
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);

  for (i = 0; i < NUM_ITERATIONS; i++) {
    BlackScholesGPU<<<DIV_UP((OPT_N / 2), 128), 128 /*480, 128*/>>>(
        (float2 *)d_CallResult, (float2 *)d_PutResult, (float2 *)d_StockPrice,
        (float2 *)d_OptionStrike, (float2 *)d_OptionYears, RISKFREE, VOLATILITY,
        OPT_N);
    getLastCudaError("BlackScholesGPU() execution failed\n");
  }

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&hTimer);
  gpuTime = sdkGetTimerValue(&hTimer) / NUM_ITERATIONS;

  // Both call and put is calculated
  printf("Options count             : %i     \n", 2 * OPT_N);
  printf("BlackScholesGPU() time    : %f msec\n", gpuTime);
  printf("Effective memory bandwidth: %f GB/s\n",
         ((double)(5 * OPT_N * sizeof(float)) * 1E-9) / (gpuTime * 1E-3));
  printf("Gigaoptions per second    : %f     \n\n",
         ((double)(2 * OPT_N) * 1E-9) / (gpuTime * 1E-3));

  printf(
      "BlackScholes, Throughput = %.4f GOptions/s, Time = %.5f s, Size = %u "
      "options, NumDevsUsed = %u, Workgroup = %u\n",
      (((double)(2.0 * OPT_N) * 1.0E-9) / (gpuTime * 1.0E-3)), gpuTime * 1e-3,
      (2 * OPT_N), 1, 128);

  printf("\nReading back GPU results...\n");
  // Read back GPU results to compare them to CPU results
  checkCudaErrors(hipMemcpy(h_CallResultGPU, d_CallResult, OPT_SZ,
                             hipMemcpyDeviceToHost));
  checkCudaErrors(
      hipMemcpy(h_PutResultGPU, d_PutResult, OPT_SZ, hipMemcpyDeviceToHost));

  printf("Checking the results...\n");
  printf("...running CPU calculations.\n\n");
  // Calculate options values on CPU
  BlackScholesCPU(h_CallResultCPU, h_PutResultCPU, h_StockPrice, h_OptionStrike,
                  h_OptionYears, RISKFREE, VOLATILITY, OPT_N);

  printf("Comparing the results...\n");
  // Calculate max absolute difference and L1 distance
  // between CPU and GPU results
  sum_delta = 0;
  sum_ref = 0;
  max_delta = 0;

  for (i = 0; i < OPT_N; i++) {
    ref = h_CallResultCPU[i];
    delta = fabs(h_CallResultCPU[i] - h_CallResultGPU[i]);

    if (delta > max_delta) {
      max_delta = delta;
    }

    sum_delta += delta;
    sum_ref += fabs(ref);
  }

  L1norm = sum_delta / sum_ref;
  printf("L1 norm: %E\n", L1norm);
  printf("Max absolute error: %E\n\n", max_delta);

  printf("Shutting down...\n");
  printf("...releasing GPU memory.\n");
  checkCudaErrors(hipFree(d_OptionYears));
  checkCudaErrors(hipFree(d_OptionStrike));
  checkCudaErrors(hipFree(d_StockPrice));
  checkCudaErrors(hipFree(d_PutResult));
  checkCudaErrors(hipFree(d_CallResult));

  printf("...releasing CPU memory.\n");
  free(h_OptionYears);
  free(h_OptionStrike);
  free(h_StockPrice);
  free(h_PutResultGPU);
  free(h_CallResultGPU);
  free(h_PutResultCPU);
  free(h_CallResultCPU);
  sdkDeleteTimer(&hTimer);
  printf("Shutdown done.\n");

  printf("\n[BlackScholes] - Test Summary\n");

  if (L1norm > 1e-6) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf(
      "\nNOTE: The CUDA Samples are not meant for performance measurements. "
      "Results may vary when GPU Boost is enabled.\n\n");
  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
