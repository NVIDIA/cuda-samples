#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#define GLFW_INCLUDE_VULKAN
#ifdef _WIN64
#include <aclapi.h>
#include <dxgi1_2.h>
#include <windows.h>
#include <VersionHelpers.h>
#define _USE_MATH_DEFINES
#endif

#include <GLFW/glfw3.h>
#include <vulkan/vulkan.h>
#ifdef _WIN64
#include <vulkan/vulkan_win32.h>
#endif

#include <algorithm>
#include <array>
#include <chrono>
#include <cstdlib>
#include <cstring>
#include <fstream>
#include <iostream>
#include <set>
#include <stdexcept>
#include <thread>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_image.h>
#include <hip/hip_vector_types.h>

#include "linmath.h"

#define WIDTH 800
#define HEIGHT 600

const int MAX_FRAMES = 4;

const std::vector<const char*> validationLayers = {
    "VK_LAYER_KHRONOS_validation"};

#ifdef NDEBUG
const bool enableValidationLayers = true;
#else
const bool enableValidationLayers = false;
#endif

std::string execution_path;

VkResult CreateDebugUtilsMessengerEXT(
    VkInstance instance, const VkDebugUtilsMessengerCreateInfoEXT* pCreateInfo,
    const VkAllocationCallbacks* pAllocator,
    VkDebugUtilsMessengerEXT* pDebugMessenger) {
  auto func = (PFN_vkCreateDebugUtilsMessengerEXT)vkGetInstanceProcAddr(
      instance, "vkCreateDebugUtilsMessengerEXT");
  if (func != nullptr) {
    return func(instance, pCreateInfo, pAllocator, pDebugMessenger);
  } else {
    return VK_ERROR_EXTENSION_NOT_PRESENT;
  }
};

const std::vector<const char*> deviceExtensions = {
    VK_KHR_SWAPCHAIN_EXTENSION_NAME,
    VK_KHR_EXTERNAL_MEMORY_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_EXTENSION_NAME,
#ifdef _WIN64
    VK_KHR_EXTERNAL_MEMORY_WIN32_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_WIN32_EXTENSION_NAME,
#else
    VK_KHR_EXTERNAL_MEMORY_FD_EXTENSION_NAME,
    VK_KHR_EXTERNAL_SEMAPHORE_FD_EXTENSION_NAME,
#endif
};

#ifdef _WIN64
class WindowsSecurityAttributes {
 protected:
  SECURITY_ATTRIBUTES m_winSecurityAttributes;
  PSECURITY_DESCRIPTOR m_winPSecurityDescriptor;

 public:
  WindowsSecurityAttributes();
  SECURITY_ATTRIBUTES* operator&();
  ~WindowsSecurityAttributes();
};

WindowsSecurityAttributes::WindowsSecurityAttributes() {
  m_winPSecurityDescriptor = (PSECURITY_DESCRIPTOR)calloc(
      1, SECURITY_DESCRIPTOR_MIN_LENGTH + 2 * sizeof(void**));

  PSID* ppSID =
      (PSID*)((PBYTE)m_winPSecurityDescriptor + SECURITY_DESCRIPTOR_MIN_LENGTH);
  PACL* ppACL = (PACL*)((PBYTE)ppSID + sizeof(PSID*));

  InitializeSecurityDescriptor(m_winPSecurityDescriptor,
                               SECURITY_DESCRIPTOR_REVISION);

  SID_IDENTIFIER_AUTHORITY sidIdentifierAuthority =
      SECURITY_WORLD_SID_AUTHORITY;
  AllocateAndInitializeSid(&sidIdentifierAuthority, 1, SECURITY_WORLD_RID, 0, 0,
                           0, 0, 0, 0, 0, ppSID);

  EXPLICIT_ACCESS explicitAccess;
  ZeroMemory(&explicitAccess, sizeof(EXPLICIT_ACCESS));
  explicitAccess.grfAccessPermissions =
      STANDARD_RIGHTS_ALL | SPECIFIC_RIGHTS_ALL;
  explicitAccess.grfAccessMode = SET_ACCESS;
  explicitAccess.grfInheritance = INHERIT_ONLY;
  explicitAccess.Trustee.TrusteeForm = TRUSTEE_IS_SID;
  explicitAccess.Trustee.TrusteeType = TRUSTEE_IS_WELL_KNOWN_GROUP;
  explicitAccess.Trustee.ptstrName = (LPTSTR)*ppSID;

  SetEntriesInAcl(1, &explicitAccess, NULL, ppACL);

  SetSecurityDescriptorDacl(m_winPSecurityDescriptor, TRUE, *ppACL, FALSE);

  m_winSecurityAttributes.nLength = sizeof(m_winSecurityAttributes);
  m_winSecurityAttributes.lpSecurityDescriptor = m_winPSecurityDescriptor;
  m_winSecurityAttributes.bInheritHandle = TRUE;
}

SECURITY_ATTRIBUTES* WindowsSecurityAttributes::operator&() {
  return &m_winSecurityAttributes;
}

WindowsSecurityAttributes::~WindowsSecurityAttributes() {
  PSID* ppSID =
      (PSID*)((PBYTE)m_winPSecurityDescriptor + SECURITY_DESCRIPTOR_MIN_LENGTH);
  PACL* ppACL = (PACL*)((PBYTE)ppSID + sizeof(PSID*));

  if (*ppSID) {
    FreeSid(*ppSID);
  }
  if (*ppACL) {
    LocalFree(*ppACL);
  }
  free(m_winPSecurityDescriptor);
}
#endif

void DestroyDebugUtilsMessengerEXT(VkInstance instance,
                                   VkDebugUtilsMessengerEXT debugMessenger,
                                   const VkAllocationCallbacks* pAllocator) {
  auto func = (PFN_vkDestroyDebugUtilsMessengerEXT)vkGetInstanceProcAddr(
      instance, "vkDestroyDebugUtilsMessengerEXT");
  if (func != nullptr) {
    func(instance, debugMessenger, pAllocator);
  }
}

struct QueueFamilyIndices {
  int graphicsFamily = -1;
  int presentFamily = -1;

  bool isComplete() { return graphicsFamily >= 0 && presentFamily >= 0; }
};

struct SwapChainSupportDetails {
  VkSurfaceCapabilitiesKHR capabilities;
  std::vector<VkSurfaceFormatKHR> formats;
  std::vector<VkPresentModeKHR> presentModes;
};

typedef float vec2[2];

struct Vertex {
  vec4 pos;
  vec3 color;
  vec2 texCoord;

  static VkVertexInputBindingDescription getBindingDescription() {
    VkVertexInputBindingDescription bindingDescription = {};
    bindingDescription.binding = 0;
    bindingDescription.stride = sizeof(Vertex);
    bindingDescription.inputRate = VK_VERTEX_INPUT_RATE_VERTEX;

    return bindingDescription;
  }

  static std::array<VkVertexInputAttributeDescription, 3>
  getAttributeDescriptions() {
    std::array<VkVertexInputAttributeDescription, 3> attributeDescriptions = {};

    attributeDescriptions[0].binding = 0;
    attributeDescriptions[0].location = 0;
    attributeDescriptions[0].format = VK_FORMAT_R32G32B32A32_SFLOAT;
    attributeDescriptions[0].offset = offsetof(Vertex, pos);

    attributeDescriptions[1].binding = 0;
    attributeDescriptions[1].location = 1;
    attributeDescriptions[1].format = VK_FORMAT_R32G32B32_SFLOAT;
    attributeDescriptions[1].offset = offsetof(Vertex, color);

    attributeDescriptions[2].binding = 0;
    attributeDescriptions[2].location = 2;
    attributeDescriptions[2].format = VK_FORMAT_R32G32_SFLOAT;
    attributeDescriptions[2].offset = offsetof(Vertex, texCoord);

    return attributeDescriptions;
  }
};

struct UniformBufferObject {
  alignas(16) mat4x4 model;
  alignas(16) mat4x4 view;
  alignas(16) mat4x4 proj;
};

const std::vector<Vertex> vertices = {
    {{-1.0f, -1.0f, 0.0f, 1.0f}, {1.0f, 0.0f, 0.0f}, {0.0f, 0.0f}},
    {{1.0f, -1.0f, 0.0f, 1.0f}, {0.0f, 1.0f, 0.0f}, {1.0f, 0.0f}},
    {{1.0f, 1.0f, 0.0f, 1.0f}, {0.0f, 0.0f, 1.0f}, {1.0f, 1.0f}},
    {{-1.0f, 1.0f, 0.0f, 1.0f}, {1.0f, 1.0f, 1.0f}, {0.0f, 1.0f}}};

const std::vector<uint16_t> indices = {0, 1, 2, 2, 3, 0};

// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return ((unsigned int)(rgba.w * 255.0f) << 24) |
         ((unsigned int)(rgba.z * 255.0f) << 16) |
         ((unsigned int)(rgba.y * 255.0f) << 8) |
         ((unsigned int)(rgba.x * 255.0f));
}

__device__ float4 rgbaIntToFloat(unsigned int c) {
  float4 rgba;
  rgba.x = (c & 0xff) * 0.003921568627f;          //  /255.0f;
  rgba.y = ((c >> 8) & 0xff) * 0.003921568627f;   //  /255.0f;
  rgba.z = ((c >> 16) & 0xff) * 0.003921568627f;  //  /255.0f;
  rgba.w = ((c >> 24) & 0xff) * 0.003921568627f;  //  /255.0f;
  return rgba;
}

int filter_radius = 14;
int g_nFilterSign = 1;

// This varies the filter radius, so we can see automatic animation
void varySigma() {
  filter_radius += g_nFilterSign;

  if (filter_radius > 64) {
    filter_radius = 64;  // clamp to 64 and then negate sign
    g_nFilterSign = -1;
  } else if (filter_radius < 0) {
    filter_radius = 0;
    g_nFilterSign = 1;
  }
}

// row pass using texture lookups
__global__ void d_boxfilter_rgba_x(hipSurfaceObject_t* dstSurfMipMapArray,
                                   hipTextureObject_t textureMipMapInput,
                                   size_t baseWidth, size_t baseHeight,
                                   size_t mipLevels, int filter_radius) {
  float scale = 1.0f / (float)((filter_radius << 1) + 1);
  unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;

  if (y < baseHeight) {
    for (uint32_t mipLevelIdx = 0; mipLevelIdx < mipLevels; mipLevelIdx++) {
      uint32_t width =
          (baseWidth >> mipLevelIdx) ? (baseWidth >> mipLevelIdx) : 1;
      uint32_t height =
          (baseHeight >> mipLevelIdx) ? (baseHeight >> mipLevelIdx) : 1;
      if (y < height && filter_radius < width) {
        float px = 1.0 / width;
        float py = 1.0 / height;
        float4 t = make_float4(0.0f);
        for (int x = -filter_radius; x <= filter_radius; x++) {
          t += tex2DLod<float4>(textureMipMapInput, x * px, y * py,
                                (float)mipLevelIdx);
        }

        unsigned int dataB = rgbaFloatToInt(t * scale);
        surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], 0, y);

        for (int x = 1; x < width; x++) {
          t += tex2DLod<float4>(textureMipMapInput, (x + filter_radius) * px,
                                y * py, (float)mipLevelIdx);
          t -=
              tex2DLod<float4>(textureMipMapInput, (x - filter_radius - 1) * px,
                               y * py, (float)mipLevelIdx);
          unsigned int dataB = rgbaFloatToInt(t * scale);
          surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx],
                      x * sizeof(uchar4), y);
        }
      }
    }
  }
}

// column pass using coalesced global memory reads
__global__ void d_boxfilter_rgba_y(hipSurfaceObject_t* dstSurfMipMapArray,
                                   hipSurfaceObject_t* srcSurfMipMapArray,
                                   size_t baseWidth, size_t baseHeight,
                                   size_t mipLevels, int filter_radius) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  float scale = 1.0f / (float)((filter_radius << 1) + 1);

  for (uint32_t mipLevelIdx = 0; mipLevelIdx < mipLevels; mipLevelIdx++) {
    uint32_t width =
        (baseWidth >> mipLevelIdx) ? (baseWidth >> mipLevelIdx) : 1;
    uint32_t height =
        (baseHeight >> mipLevelIdx) ? (baseHeight >> mipLevelIdx) : 1;

    if (x < width && height > filter_radius) {
      float4 t;
      // do left edge
      int colInBytes = x * sizeof(uchar4);
      unsigned int pixFirst = surf2Dread<unsigned int>(
          srcSurfMipMapArray[mipLevelIdx], colInBytes, 0);
      t = rgbaIntToFloat(pixFirst) * filter_radius;

      for (int y = 0; (y < (filter_radius + 1)) && (y < height); y++) {
        unsigned int pix = surf2Dread<unsigned int>(
            srcSurfMipMapArray[mipLevelIdx], colInBytes, y);
        t += rgbaIntToFloat(pix);
      }

      unsigned int dataB = rgbaFloatToInt(t * scale);
      surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], colInBytes, 0);

      for (int y = 1; (y < filter_radius + 1) && ((y + filter_radius) < height);
           y++) {
        unsigned int pix = surf2Dread<unsigned int>(
            srcSurfMipMapArray[mipLevelIdx], colInBytes, y + filter_radius);
        t += rgbaIntToFloat(pix);
        t -= rgbaIntToFloat(pixFirst);

        dataB = rgbaFloatToInt(t * scale);
        surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], colInBytes, y);
      }

      // main loop
      for (int y = (filter_radius + 1); y < (height - filter_radius); y++) {
        unsigned int pix = surf2Dread<unsigned int>(
            srcSurfMipMapArray[mipLevelIdx], colInBytes, y + filter_radius);
        t += rgbaIntToFloat(pix);

        pix = surf2Dread<unsigned int>(srcSurfMipMapArray[mipLevelIdx],
                                       colInBytes, y - filter_radius - 1);
        t -= rgbaIntToFloat(pix);

        dataB = rgbaFloatToInt(t * scale);
        surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], colInBytes, y);
      }

      // do right edge
      unsigned int pixLast = surf2Dread<unsigned int>(
          srcSurfMipMapArray[mipLevelIdx], colInBytes, height - 1);
      for (int y = height - filter_radius;
           (y < height) && ((y - filter_radius - 1) > 1); y++) {
        t += rgbaIntToFloat(pixLast);
        unsigned int pix = surf2Dread<unsigned int>(
            srcSurfMipMapArray[mipLevelIdx], colInBytes, y - filter_radius - 1);
        t -= rgbaIntToFloat(pix);
        dataB = rgbaFloatToInt(t * scale);
        surf2Dwrite(dataB, dstSurfMipMapArray[mipLevelIdx], colInBytes, y);
      }
    }
  }
}

class vulkanImageCUDA {
 public:
  void loadImageData(const std::string& filename) {
    // load image (needed so we can get the width and height before we create
    // the window
    char* image_path =
        sdkFindFilePath(filename.c_str(), execution_path.c_str());

    if (image_path == 0) {
      printf("Error finding image file '%s'\n", filename.c_str());
      exit(EXIT_FAILURE);
    }

    sdkLoadPPM4(image_path, (unsigned char**)&image_data, &imageWidth,
                &imageHeight);

    if (!image_data) {
      printf("Error opening file '%s'\n", image_path);
      exit(EXIT_FAILURE);
    }

    printf("Loaded '%s', %d x %d pixels\n", image_path, imageWidth,
           imageHeight);
  }

  void run() {
    initWindow();
    initVulkan();
    initCuda();
    mainLoop();
    cleanup();
  }

 private:
  GLFWwindow* window;

  VkInstance instance;
  VkDebugUtilsMessengerEXT debugMessenger;
  VkSurfaceKHR surface;

  VkPhysicalDevice physicalDevice = VK_NULL_HANDLE;
  VkDevice device;
  uint8_t vkDeviceUUID[VK_UUID_SIZE];

  VkQueue graphicsQueue;
  VkQueue presentQueue;

  VkSwapchainKHR swapChain;
  std::vector<VkImage> swapChainImages;
  VkFormat swapChainImageFormat;
  VkExtent2D swapChainExtent;
  std::vector<VkImageView> swapChainImageViews;
  std::vector<VkFramebuffer> swapChainFramebuffers;

  VkRenderPass renderPass;
  VkDescriptorSetLayout descriptorSetLayout;
  VkPipelineLayout pipelineLayout;
  VkPipeline graphicsPipeline;

  VkCommandPool commandPool;

  VkImage textureImage;
  VkDeviceMemory textureImageMemory;
  VkImageView textureImageView;
  VkSampler textureSampler;

  VkBuffer vertexBuffer;
  VkDeviceMemory vertexBufferMemory;
  VkBuffer indexBuffer;
  VkDeviceMemory indexBufferMemory;

  std::vector<VkBuffer> uniformBuffers;
  std::vector<VkDeviceMemory> uniformBuffersMemory;

  VkDescriptorPool descriptorPool;
  std::vector<VkDescriptorSet> descriptorSets;

  std::vector<VkCommandBuffer> commandBuffers;

  std::vector<VkSemaphore> imageAvailableSemaphores;
  std::vector<VkSemaphore> renderFinishedSemaphores;
  VkSemaphore cudaUpdateVkSemaphore, vkUpdateCudaSemaphore;
  std::vector<VkFence> inFlightFences;

  size_t currentFrame = 0;

  bool framebufferResized = false;

#ifdef _WIN64
  PFN_vkGetMemoryWin32HandleKHR fpGetMemoryWin32HandleKHR;
  PFN_vkGetSemaphoreWin32HandleKHR fpGetSemaphoreWin32HandleKHR;
#else
  PFN_vkGetMemoryFdKHR fpGetMemoryFdKHR = NULL;
  PFN_vkGetSemaphoreFdKHR fpGetSemaphoreFdKHR = NULL;
#endif

  PFN_vkGetPhysicalDeviceProperties2 fpGetPhysicalDeviceProperties2;

  unsigned int* image_data = NULL;
  unsigned int imageWidth, imageHeight;
  unsigned int mipLevels = 1;
  size_t totalImageMemSize;

  // CUDA objects
  hipExternalMemory_t cudaExtMemImageBuffer;
  hipMipmappedArray_t cudaMipmappedImageArray, cudaMipmappedImageArrayTemp,
      cudaMipmappedImageArrayOrig;
  std::vector<hipSurfaceObject_t> surfaceObjectList, surfaceObjectListTemp;
  hipSurfaceObject_t *d_surfaceObjectList, *d_surfaceObjectListTemp;
  hipTextureObject_t textureObjMipMapInput;

  hipExternalSemaphore_t cudaExtCudaUpdateVkSemaphore;
  hipExternalSemaphore_t cudaExtVkUpdateCudaSemaphore;
  hipStream_t streamToRun;

  void initWindow() {
    glfwInit();

    glfwWindowHint(GLFW_CLIENT_API, GLFW_NO_API);

    window = glfwCreateWindow(WIDTH, HEIGHT, "Vulkan Image CUDA Box Filter",
                              nullptr, nullptr);
    glfwSetWindowUserPointer(window, this);
    glfwSetFramebufferSizeCallback(window, framebufferResizeCallback);
  }

  static void framebufferResizeCallback(GLFWwindow* window, int width,
                                        int height) {
    auto app =
        reinterpret_cast<vulkanImageCUDA*>(glfwGetWindowUserPointer(window));
    app->framebufferResized = true;
  }

  void initVulkan() {
    createInstance();
    setupDebugMessenger();
    createSurface();
    pickPhysicalDevice();
    createLogicalDevice();
    getKhrExtensionsFn();
    createSwapChain();
    createImageViews();
    createRenderPass();
    createDescriptorSetLayout();
    createGraphicsPipeline();
    createFramebuffers();
    createCommandPool();
    createTextureImage();
    createTextureImageView();
    createTextureSampler();
    createVertexBuffer();
    createIndexBuffer();
    createUniformBuffers();
    createDescriptorPool();
    createDescriptorSets();
    createCommandBuffers();
    createSyncObjects();
    createSyncObjectsExt();
  }

  void initCuda() {
    setCudaVkDevice();
    checkCudaErrors(hipStreamCreate(&streamToRun));
    cudaVkImportImageMem();
    cudaVkImportSemaphore();
  }

  void mainLoop() {
    updateUniformBuffer();
    while (!glfwWindowShouldClose(window)) {
      glfwPollEvents();
      drawFrame();
    }

    vkDeviceWaitIdle(device);
  }

  void cleanupSwapChain() {
    for (auto framebuffer : swapChainFramebuffers) {
      vkDestroyFramebuffer(device, framebuffer, nullptr);
    }

    vkFreeCommandBuffers(device, commandPool,
                         static_cast<uint32_t>(commandBuffers.size()),
                         commandBuffers.data());

    vkDestroyPipeline(device, graphicsPipeline, nullptr);
    vkDestroyPipelineLayout(device, pipelineLayout, nullptr);
    vkDestroyRenderPass(device, renderPass, nullptr);

    for (auto imageView : swapChainImageViews) {
      vkDestroyImageView(device, imageView, nullptr);
    }

    vkDestroySwapchainKHR(device, swapChain, nullptr);

    for (size_t i = 0; i < swapChainImages.size(); i++) {
      vkDestroyBuffer(device, uniformBuffers[i], nullptr);
      vkFreeMemory(device, uniformBuffersMemory[i], nullptr);
    }

    vkDestroyDescriptorPool(device, descriptorPool, nullptr);
  }

  void cleanup() {
    cleanupSwapChain();

    vkDestroySampler(device, textureSampler, nullptr);
    vkDestroyImageView(device, textureImageView, nullptr);

    for (int i = 0; i < mipLevels; i++) {
      checkCudaErrors(hipDestroySurfaceObject(surfaceObjectList[i]));
      checkCudaErrors(hipDestroySurfaceObject(surfaceObjectListTemp[i]));
    }

    checkCudaErrors(hipFree(d_surfaceObjectList));
    checkCudaErrors(hipFree(d_surfaceObjectListTemp));
    checkCudaErrors(hipFreeMipmappedArray(cudaMipmappedImageArrayTemp));
    checkCudaErrors(hipFreeMipmappedArray(cudaMipmappedImageArrayOrig));
    checkCudaErrors(hipFreeMipmappedArray(cudaMipmappedImageArray));
    checkCudaErrors(hipDestroyTextureObject(textureObjMipMapInput));
    checkCudaErrors(hipDestroyExternalMemory(cudaExtMemImageBuffer));
    checkCudaErrors(hipDestroyExternalSemaphore(cudaExtCudaUpdateVkSemaphore));
    checkCudaErrors(hipDestroyExternalSemaphore(cudaExtVkUpdateCudaSemaphore));

    vkDestroyImage(device, textureImage, nullptr);
    vkFreeMemory(device, textureImageMemory, nullptr);

    vkDestroyDescriptorSetLayout(device, descriptorSetLayout, nullptr);

    vkDestroyBuffer(device, indexBuffer, nullptr);
    vkFreeMemory(device, indexBufferMemory, nullptr);

    vkDestroyBuffer(device, vertexBuffer, nullptr);
    vkFreeMemory(device, vertexBufferMemory, nullptr);

    vkDestroySemaphore(device, cudaUpdateVkSemaphore, nullptr);
    vkDestroySemaphore(device, vkUpdateCudaSemaphore, nullptr);

    for (size_t i = 0; i < MAX_FRAMES; i++) {
      vkDestroySemaphore(device, renderFinishedSemaphores[i], nullptr);
      vkDestroySemaphore(device, imageAvailableSemaphores[i], nullptr);
      vkDestroyFence(device, inFlightFences[i], nullptr);
    }

    vkDestroyCommandPool(device, commandPool, nullptr);

    vkDestroyDevice(device, nullptr);

    if (enableValidationLayers) {
      DestroyDebugUtilsMessengerEXT(instance, debugMessenger, nullptr);
    }

    vkDestroySurfaceKHR(instance, surface, nullptr);
    vkDestroyInstance(instance, nullptr);

    glfwDestroyWindow(window);

    glfwTerminate();
  }

  void recreateSwapChain() {
    int width = 0, height = 0;
    while (width == 0 || height == 0) {
      glfwGetFramebufferSize(window, &width, &height);
      glfwWaitEvents();
    }

    vkDeviceWaitIdle(device);

    cleanupSwapChain();

    createSwapChain();
    createImageViews();
    createRenderPass();
    createGraphicsPipeline();
    createFramebuffers();
    createUniformBuffers();
    createDescriptorPool();
    createDescriptorSets();
    createCommandBuffers();
  }

  void createInstance() {
    if (enableValidationLayers && !checkValidationLayerSupport()) {
      throw std::runtime_error(
          "validation layers requested, but not available!");
    }

    VkApplicationInfo appInfo = {};
    appInfo.sType = VK_STRUCTURE_TYPE_APPLICATION_INFO;
    appInfo.pApplicationName = "Vulkan Image CUDA Interop";
    appInfo.applicationVersion = VK_MAKE_VERSION(1, 0, 0);
    appInfo.pEngineName = "No Engine";
    appInfo.engineVersion = VK_MAKE_VERSION(1, 0, 0);
    appInfo.apiVersion = VK_API_VERSION_1_1;

    VkInstanceCreateInfo createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_INSTANCE_CREATE_INFO;
    createInfo.pApplicationInfo = &appInfo;

    auto extensions = getRequiredExtensions();
    createInfo.enabledExtensionCount = static_cast<uint32_t>(extensions.size());
    createInfo.ppEnabledExtensionNames = extensions.data();

    VkDebugUtilsMessengerCreateInfoEXT debugCreateInfo;
    if (enableValidationLayers) {
      createInfo.enabledLayerCount =
          static_cast<uint32_t>(validationLayers.size());
      createInfo.ppEnabledLayerNames = validationLayers.data();

      populateDebugMessengerCreateInfo(debugCreateInfo);
      createInfo.pNext = (VkDebugUtilsMessengerCreateInfoEXT*)&debugCreateInfo;
    } else {
      createInfo.enabledLayerCount = 0;

      createInfo.pNext = nullptr;
    }

    if (vkCreateInstance(&createInfo, nullptr, &instance) != VK_SUCCESS) {
      throw std::runtime_error("failed to create instance!");
    }

    fpGetPhysicalDeviceProperties2 =
        (PFN_vkGetPhysicalDeviceProperties2)vkGetInstanceProcAddr(
            instance, "vkGetPhysicalDeviceProperties2");
    if (fpGetPhysicalDeviceProperties2 == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetPhysicalDeviceProperties2KHR\" not "
          "found.\n");
    }

#ifdef _WIN64
    fpGetMemoryWin32HandleKHR =
        (PFN_vkGetMemoryWin32HandleKHR)vkGetInstanceProcAddr(
            instance, "vkGetMemoryWin32HandleKHR");
    if (fpGetMemoryWin32HandleKHR == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetMemoryWin32HandleKHR\" not "
          "found.\n");
    }
#else
    fpGetMemoryFdKHR = (PFN_vkGetMemoryFdKHR)vkGetInstanceProcAddr(
        instance, "vkGetMemoryFdKHR");
    if (fpGetMemoryFdKHR == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetMemoryFdKHR\" not found.\n");
    } else {
      std::cout << "Vulkan proc address for vkGetMemoryFdKHR - "
                << fpGetMemoryFdKHR << std::endl;
    }
#endif
  }

  void populateDebugMessengerCreateInfo(
      VkDebugUtilsMessengerCreateInfoEXT& createInfo) {
    createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_DEBUG_UTILS_MESSENGER_CREATE_INFO_EXT;
    createInfo.messageSeverity =
        VK_DEBUG_UTILS_MESSAGE_SEVERITY_VERBOSE_BIT_EXT |
        VK_DEBUG_UTILS_MESSAGE_SEVERITY_WARNING_BIT_EXT |
        VK_DEBUG_UTILS_MESSAGE_SEVERITY_ERROR_BIT_EXT;
    createInfo.messageType = VK_DEBUG_UTILS_MESSAGE_TYPE_GENERAL_BIT_EXT |
                             VK_DEBUG_UTILS_MESSAGE_TYPE_VALIDATION_BIT_EXT |
                             VK_DEBUG_UTILS_MESSAGE_TYPE_PERFORMANCE_BIT_EXT;
    createInfo.pfnUserCallback = debugCallback;
  }

  void setupDebugMessenger() {
    if (!enableValidationLayers) return;

    VkDebugUtilsMessengerCreateInfoEXT createInfo;
    populateDebugMessengerCreateInfo(createInfo);

    if (CreateDebugUtilsMessengerEXT(instance, &createInfo, nullptr,
                                     &debugMessenger) != VK_SUCCESS) {
      throw std::runtime_error("failed to set up debug messenger!");
    }
  }

  void createSurface() {
    if (glfwCreateWindowSurface(instance, window, nullptr, &surface) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create window surface!");
    }
  }

  void pickPhysicalDevice() {
    uint32_t deviceCount = 0;
    vkEnumeratePhysicalDevices(instance, &deviceCount, nullptr);

    if (deviceCount == 0) {
      throw std::runtime_error("failed to find GPUs with Vulkan support!");
    }

    std::vector<VkPhysicalDevice> devices(deviceCount);
    vkEnumeratePhysicalDevices(instance, &deviceCount, devices.data());

    for (const auto& device : devices) {
      if (isDeviceSuitable(device)) {
        physicalDevice = device;
        break;
      }
    }

    if (physicalDevice == VK_NULL_HANDLE) {
      throw std::runtime_error("failed to find a suitable GPU!");
    }

    std::cout << "Selected physical device = " << physicalDevice << std::endl;

    VkPhysicalDeviceIDProperties vkPhysicalDeviceIDProperties = {};
    vkPhysicalDeviceIDProperties.sType =
        VK_STRUCTURE_TYPE_PHYSICAL_DEVICE_ID_PROPERTIES;
    vkPhysicalDeviceIDProperties.pNext = NULL;

    VkPhysicalDeviceProperties2 vkPhysicalDeviceProperties2 = {};
    vkPhysicalDeviceProperties2.sType =
        VK_STRUCTURE_TYPE_PHYSICAL_DEVICE_PROPERTIES_2;
    vkPhysicalDeviceProperties2.pNext = &vkPhysicalDeviceIDProperties;

    fpGetPhysicalDeviceProperties2(physicalDevice,
                                   &vkPhysicalDeviceProperties2);

    memcpy(vkDeviceUUID, vkPhysicalDeviceIDProperties.deviceUUID,
           sizeof(vkDeviceUUID));
  }

  void getKhrExtensionsFn() {
#ifdef _WIN64

    fpGetSemaphoreWin32HandleKHR =
        (PFN_vkGetSemaphoreWin32HandleKHR)vkGetDeviceProcAddr(
            device, "vkGetSemaphoreWin32HandleKHR");
    if (fpGetSemaphoreWin32HandleKHR == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetSemaphoreWin32HandleKHR\" not "
          "found.\n");
    }
#else
    fpGetSemaphoreFdKHR = (PFN_vkGetSemaphoreFdKHR)vkGetDeviceProcAddr(
        device, "vkGetSemaphoreFdKHR");
    if (fpGetSemaphoreFdKHR == NULL) {
      throw std::runtime_error(
          "Vulkan: Proc address for \"vkGetSemaphoreFdKHR\" not found.\n");
    }
#endif
  }

  int setCudaVkDevice() {
    int current_device = 0;
    int device_count = 0;
    int devices_prohibited = 0;

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceCount(&device_count));

    if (device_count == 0) {
      fprintf(stderr, "CUDA error: no devices supporting CUDA.\n");
      exit(EXIT_FAILURE);
    }

    // Find the GPU which is selected by Vulkan
    while (current_device < device_count) {
      hipGetDeviceProperties(&deviceProp, current_device);

      if ((deviceProp.computeMode != hipComputeModeProhibited)) {
        // Compare the cuda device UUID with vulkan UUID
        int ret = memcmp(&deviceProp.uuid, &vkDeviceUUID, VK_UUID_SIZE);
        if (ret == 0) {
          checkCudaErrors(hipSetDevice(current_device));
          checkCudaErrors(hipGetDeviceProperties(&deviceProp, current_device));
          printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
                 current_device, deviceProp.name, deviceProp.major,
                 deviceProp.minor);

          return current_device;
        }

      } else {
        devices_prohibited++;
      }

      current_device++;
    }

    if (devices_prohibited == device_count) {
      fprintf(stderr,
              "CUDA error:"
              " No Vulkan-CUDA Interop capable GPU found.\n");
      exit(EXIT_FAILURE);
    }

    return -1;
  }

  void createLogicalDevice() {
    QueueFamilyIndices indices = findQueueFamilies(physicalDevice);

    std::vector<VkDeviceQueueCreateInfo> queueCreateInfos;
    std::set<int> uniqueQueueFamilies = {indices.graphicsFamily,
                                         indices.presentFamily};

    float queuePriority = 1.0f;
    for (int queueFamily : uniqueQueueFamilies) {
      VkDeviceQueueCreateInfo queueCreateInfo = {};
      queueCreateInfo.sType = VK_STRUCTURE_TYPE_DEVICE_QUEUE_CREATE_INFO;
      queueCreateInfo.queueFamilyIndex = queueFamily;
      queueCreateInfo.queueCount = 1;
      queueCreateInfo.pQueuePriorities = &queuePriority;
      queueCreateInfos.push_back(queueCreateInfo);
    }

    VkPhysicalDeviceFeatures deviceFeatures = {};
    deviceFeatures.samplerAnisotropy = VK_TRUE;

    VkDeviceCreateInfo createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_DEVICE_CREATE_INFO;

    createInfo.pQueueCreateInfos = queueCreateInfos.data();
    createInfo.queueCreateInfoCount = queueCreateInfos.size();

    createInfo.pEnabledFeatures = &deviceFeatures;
    std::vector<const char*> enabledExtensionNameList;

    for (int i = 0; i < deviceExtensions.size(); i++) {
      enabledExtensionNameList.push_back(deviceExtensions[i]);
    }
    if (enableValidationLayers) {
      createInfo.enabledLayerCount =
          static_cast<uint32_t>(validationLayers.size());
      createInfo.ppEnabledLayerNames = validationLayers.data();
    } else {
      createInfo.enabledLayerCount = 0;
    }
    createInfo.enabledExtensionCount =
        static_cast<uint32_t>(enabledExtensionNameList.size());
    createInfo.ppEnabledExtensionNames = enabledExtensionNameList.data();

    if (vkCreateDevice(physicalDevice, &createInfo, nullptr, &device) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create logical device!");
    }
    vkGetDeviceQueue(device, indices.graphicsFamily, 0, &graphicsQueue);
    vkGetDeviceQueue(device, indices.presentFamily, 0, &presentQueue);
  }

  void createSwapChain() {
    SwapChainSupportDetails swapChainSupport =
        querySwapChainSupport(physicalDevice);

    VkSurfaceFormatKHR surfaceFormat =
        chooseSwapSurfaceFormat(swapChainSupport.formats);
    VkPresentModeKHR presentMode =
        chooseSwapPresentMode(swapChainSupport.presentModes);
    VkExtent2D extent = chooseSwapExtent(swapChainSupport.capabilities);

    uint32_t imageCount = swapChainSupport.capabilities.minImageCount + 1;
    if (swapChainSupport.capabilities.maxImageCount > 0 &&
        imageCount > swapChainSupport.capabilities.maxImageCount) {
      imageCount = swapChainSupport.capabilities.maxImageCount;
    }

    VkSwapchainCreateInfoKHR createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_SWAPCHAIN_CREATE_INFO_KHR;
    createInfo.surface = surface;

    createInfo.minImageCount = imageCount;
    createInfo.imageFormat = surfaceFormat.format;
    createInfo.imageColorSpace = surfaceFormat.colorSpace;
    createInfo.imageExtent = extent;
    createInfo.imageArrayLayers = 1;
    createInfo.imageUsage = VK_IMAGE_USAGE_COLOR_ATTACHMENT_BIT;

    QueueFamilyIndices indices = findQueueFamilies(physicalDevice);
    uint32_t queueFamilyIndices[] = {(uint32_t)indices.graphicsFamily,
                                     (uint32_t)indices.presentFamily};

    if (indices.graphicsFamily != indices.presentFamily) {
      createInfo.imageSharingMode = VK_SHARING_MODE_CONCURRENT;
      createInfo.queueFamilyIndexCount = 2;
      createInfo.pQueueFamilyIndices = queueFamilyIndices;
    } else {
      createInfo.imageSharingMode = VK_SHARING_MODE_EXCLUSIVE;
    }

    createInfo.preTransform = swapChainSupport.capabilities.currentTransform;
    createInfo.compositeAlpha = VK_COMPOSITE_ALPHA_OPAQUE_BIT_KHR;
    createInfo.presentMode = presentMode;
    createInfo.clipped = VK_TRUE;

    if (vkCreateSwapchainKHR(device, &createInfo, nullptr, &swapChain) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create swap chain!");
    }

    vkGetSwapchainImagesKHR(device, swapChain, &imageCount, nullptr);
    swapChainImages.resize(imageCount);
    vkGetSwapchainImagesKHR(device, swapChain, &imageCount,
                            swapChainImages.data());

    swapChainImageFormat = surfaceFormat.format;
    swapChainExtent = extent;
  }

  void createImageViews() {
    swapChainImageViews.resize(swapChainImages.size());

    for (size_t i = 0; i < swapChainImages.size(); i++) {
      swapChainImageViews[i] =
          createImageView(swapChainImages[i], swapChainImageFormat);
    }
  }

  void createRenderPass() {
    VkAttachmentDescription colorAttachment = {};
    colorAttachment.format = swapChainImageFormat;
    colorAttachment.samples = VK_SAMPLE_COUNT_1_BIT;
    colorAttachment.loadOp = VK_ATTACHMENT_LOAD_OP_CLEAR;
    colorAttachment.storeOp = VK_ATTACHMENT_STORE_OP_STORE;
    colorAttachment.stencilLoadOp = VK_ATTACHMENT_LOAD_OP_DONT_CARE;
    colorAttachment.stencilStoreOp = VK_ATTACHMENT_STORE_OP_DONT_CARE;
    colorAttachment.initialLayout = VK_IMAGE_LAYOUT_UNDEFINED;
    colorAttachment.finalLayout = VK_IMAGE_LAYOUT_PRESENT_SRC_KHR;

    VkAttachmentReference colorAttachmentRef = {};
    colorAttachmentRef.attachment = 0;
    colorAttachmentRef.layout = VK_IMAGE_LAYOUT_COLOR_ATTACHMENT_OPTIMAL;

    VkSubpassDescription subpass = {};
    subpass.pipelineBindPoint = VK_PIPELINE_BIND_POINT_GRAPHICS;
    subpass.colorAttachmentCount = 1;
    subpass.pColorAttachments = &colorAttachmentRef;

    VkSubpassDependency dependency = {};
    dependency.srcSubpass = VK_SUBPASS_EXTERNAL;
    dependency.dstSubpass = 0;
    dependency.srcStageMask = VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT;
    dependency.srcAccessMask = 0;
    dependency.dstStageMask = VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT;
    dependency.dstAccessMask = VK_ACCESS_COLOR_ATTACHMENT_READ_BIT |
                               VK_ACCESS_COLOR_ATTACHMENT_WRITE_BIT;

    VkRenderPassCreateInfo renderPassInfo = {};
    renderPassInfo.sType = VK_STRUCTURE_TYPE_RENDER_PASS_CREATE_INFO;
    renderPassInfo.attachmentCount = 1;
    renderPassInfo.pAttachments = &colorAttachment;
    renderPassInfo.subpassCount = 1;
    renderPassInfo.pSubpasses = &subpass;
    renderPassInfo.dependencyCount = 1;
    renderPassInfo.pDependencies = &dependency;

    if (vkCreateRenderPass(device, &renderPassInfo, nullptr, &renderPass) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create render pass!");
    }
  }

  void createDescriptorSetLayout() {
    VkDescriptorSetLayoutBinding uboLayoutBinding = {};
    uboLayoutBinding.binding = 0;
    uboLayoutBinding.descriptorCount = 1;
    uboLayoutBinding.descriptorType = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
    uboLayoutBinding.pImmutableSamplers = nullptr;
    uboLayoutBinding.stageFlags = VK_SHADER_STAGE_VERTEX_BIT;

    VkDescriptorSetLayoutBinding samplerLayoutBinding = {};
    samplerLayoutBinding.binding = 1;
    samplerLayoutBinding.descriptorCount = 1;
    samplerLayoutBinding.descriptorType =
        VK_DESCRIPTOR_TYPE_COMBINED_IMAGE_SAMPLER;
    samplerLayoutBinding.pImmutableSamplers = nullptr;
    samplerLayoutBinding.stageFlags = VK_SHADER_STAGE_FRAGMENT_BIT;

    std::array<VkDescriptorSetLayoutBinding, 2> bindings = {
        uboLayoutBinding, samplerLayoutBinding};
    VkDescriptorSetLayoutCreateInfo layoutInfo = {};
    layoutInfo.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_LAYOUT_CREATE_INFO;
    layoutInfo.bindingCount = static_cast<uint32_t>(bindings.size());
    layoutInfo.pBindings = bindings.data();

    if (vkCreateDescriptorSetLayout(device, &layoutInfo, nullptr,
                                    &descriptorSetLayout) != VK_SUCCESS) {
      throw std::runtime_error("failed to create descriptor set layout!");
    }
  }

  void createGraphicsPipeline() {
    auto vertShaderCode = readFile("vert.spv");
    auto fragShaderCode = readFile("frag.spv");

    VkShaderModule vertShaderModule = createShaderModule(vertShaderCode);
    VkShaderModule fragShaderModule = createShaderModule(fragShaderCode);

    VkPipelineShaderStageCreateInfo vertShaderStageInfo = {};
    vertShaderStageInfo.sType =
        VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
    vertShaderStageInfo.stage = VK_SHADER_STAGE_VERTEX_BIT;
    vertShaderStageInfo.module = vertShaderModule;
    vertShaderStageInfo.pName = "main";

    VkPipelineShaderStageCreateInfo fragShaderStageInfo = {};
    fragShaderStageInfo.sType =
        VK_STRUCTURE_TYPE_PIPELINE_SHADER_STAGE_CREATE_INFO;
    fragShaderStageInfo.stage = VK_SHADER_STAGE_FRAGMENT_BIT;
    fragShaderStageInfo.module = fragShaderModule;
    fragShaderStageInfo.pName = "main";

    VkPipelineShaderStageCreateInfo shaderStages[] = {vertShaderStageInfo,
                                                      fragShaderStageInfo};

    VkPipelineVertexInputStateCreateInfo vertexInputInfo = {};
    vertexInputInfo.sType =
        VK_STRUCTURE_TYPE_PIPELINE_VERTEX_INPUT_STATE_CREATE_INFO;

    auto bindingDescription = Vertex::getBindingDescription();
    auto attributeDescriptions = Vertex::getAttributeDescriptions();

    vertexInputInfo.vertexBindingDescriptionCount = 1;
    vertexInputInfo.vertexAttributeDescriptionCount =
        static_cast<uint32_t>(attributeDescriptions.size());
    vertexInputInfo.pVertexBindingDescriptions = &bindingDescription;
    vertexInputInfo.pVertexAttributeDescriptions = attributeDescriptions.data();

    VkPipelineInputAssemblyStateCreateInfo inputAssembly = {};
    inputAssembly.sType =
        VK_STRUCTURE_TYPE_PIPELINE_INPUT_ASSEMBLY_STATE_CREATE_INFO;
    inputAssembly.topology = VK_PRIMITIVE_TOPOLOGY_TRIANGLE_LIST;
    inputAssembly.primitiveRestartEnable = VK_FALSE;

    VkViewport viewport = {};
    viewport.x = 0.0f;
    viewport.y = 0.0f;
    viewport.width = (float)swapChainExtent.width;
    viewport.height = (float)swapChainExtent.height;
    viewport.minDepth = 0.0f;
    viewport.maxDepth = 1.0f;

    VkRect2D scissor = {};
    scissor.offset = {0, 0};
    scissor.extent = swapChainExtent;

    VkPipelineViewportStateCreateInfo viewportState = {};
    viewportState.sType = VK_STRUCTURE_TYPE_PIPELINE_VIEWPORT_STATE_CREATE_INFO;
    viewportState.viewportCount = 1;
    viewportState.pViewports = &viewport;
    viewportState.scissorCount = 1;
    viewportState.pScissors = &scissor;

    VkPipelineRasterizationStateCreateInfo rasterizer = {};
    rasterizer.sType =
        VK_STRUCTURE_TYPE_PIPELINE_RASTERIZATION_STATE_CREATE_INFO;
    rasterizer.depthClampEnable = VK_FALSE;
    rasterizer.rasterizerDiscardEnable = VK_FALSE;
    rasterizer.polygonMode = VK_POLYGON_MODE_FILL;
    rasterizer.lineWidth = 1.0f;
    rasterizer.cullMode = VK_CULL_MODE_BACK_BIT;
    rasterizer.frontFace = VK_FRONT_FACE_COUNTER_CLOCKWISE;
    rasterizer.depthBiasEnable = VK_FALSE;

    VkPipelineMultisampleStateCreateInfo multisampling = {};
    multisampling.sType =
        VK_STRUCTURE_TYPE_PIPELINE_MULTISAMPLE_STATE_CREATE_INFO;
    multisampling.sampleShadingEnable = VK_FALSE;
    multisampling.rasterizationSamples = VK_SAMPLE_COUNT_1_BIT;

    VkPipelineColorBlendAttachmentState colorBlendAttachment = {};
    colorBlendAttachment.colorWriteMask =
        VK_COLOR_COMPONENT_R_BIT | VK_COLOR_COMPONENT_G_BIT |
        VK_COLOR_COMPONENT_B_BIT | VK_COLOR_COMPONENT_A_BIT;
    colorBlendAttachment.blendEnable = VK_FALSE;

    VkPipelineColorBlendStateCreateInfo colorBlending = {};
    colorBlending.sType =
        VK_STRUCTURE_TYPE_PIPELINE_COLOR_BLEND_STATE_CREATE_INFO;
    colorBlending.logicOpEnable = VK_FALSE;
    colorBlending.logicOp = VK_LOGIC_OP_COPY;
    colorBlending.attachmentCount = 1;
    colorBlending.pAttachments = &colorBlendAttachment;
    colorBlending.blendConstants[0] = 0.0f;
    colorBlending.blendConstants[1] = 0.0f;
    colorBlending.blendConstants[2] = 0.0f;
    colorBlending.blendConstants[3] = 0.0f;

    VkPipelineLayoutCreateInfo pipelineLayoutInfo = {};
    pipelineLayoutInfo.sType = VK_STRUCTURE_TYPE_PIPELINE_LAYOUT_CREATE_INFO;
    pipelineLayoutInfo.setLayoutCount = 1;
    pipelineLayoutInfo.pSetLayouts = &descriptorSetLayout;

    if (vkCreatePipelineLayout(device, &pipelineLayoutInfo, nullptr,
                               &pipelineLayout) != VK_SUCCESS) {
      throw std::runtime_error("failed to create pipeline layout!");
    }

    VkGraphicsPipelineCreateInfo pipelineInfo = {};
    pipelineInfo.sType = VK_STRUCTURE_TYPE_GRAPHICS_PIPELINE_CREATE_INFO;
    pipelineInfo.stageCount = 2;
    pipelineInfo.pStages = shaderStages;
    pipelineInfo.pVertexInputState = &vertexInputInfo;
    pipelineInfo.pInputAssemblyState = &inputAssembly;
    pipelineInfo.pViewportState = &viewportState;
    pipelineInfo.pRasterizationState = &rasterizer;
    pipelineInfo.pMultisampleState = &multisampling;
    pipelineInfo.pColorBlendState = &colorBlending;
    pipelineInfo.layout = pipelineLayout;
    pipelineInfo.renderPass = renderPass;
    pipelineInfo.subpass = 0;
    pipelineInfo.basePipelineHandle = VK_NULL_HANDLE;

    if (vkCreateGraphicsPipelines(device, VK_NULL_HANDLE, 1, &pipelineInfo,
                                  nullptr, &graphicsPipeline) != VK_SUCCESS) {
      throw std::runtime_error("failed to create graphics pipeline!");
    }

    vkDestroyShaderModule(device, fragShaderModule, nullptr);
    vkDestroyShaderModule(device, vertShaderModule, nullptr);
  }

  void createFramebuffers() {
    swapChainFramebuffers.resize(swapChainImageViews.size());

    for (size_t i = 0; i < swapChainImageViews.size(); i++) {
      VkImageView attachments[] = {swapChainImageViews[i]};

      VkFramebufferCreateInfo framebufferInfo = {};
      framebufferInfo.sType = VK_STRUCTURE_TYPE_FRAMEBUFFER_CREATE_INFO;
      framebufferInfo.renderPass = renderPass;
      framebufferInfo.attachmentCount = 1;
      framebufferInfo.pAttachments = attachments;
      framebufferInfo.width = swapChainExtent.width;
      framebufferInfo.height = swapChainExtent.height;
      framebufferInfo.layers = 1;

      if (vkCreateFramebuffer(device, &framebufferInfo, nullptr,
                              &swapChainFramebuffers[i]) != VK_SUCCESS) {
        throw std::runtime_error("failed to create framebuffer!");
      }
    }
  }

  void createCommandPool() {
    QueueFamilyIndices queueFamilyIndices = findQueueFamilies(physicalDevice);

    VkCommandPoolCreateInfo poolInfo = {};
    poolInfo.sType = VK_STRUCTURE_TYPE_COMMAND_POOL_CREATE_INFO;
    poolInfo.queueFamilyIndex = queueFamilyIndices.graphicsFamily;

    if (vkCreateCommandPool(device, &poolInfo, nullptr, &commandPool) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create graphics command pool!");
    }
  }

  void createTextureImage() {
    VkDeviceSize imageSize = imageWidth * imageHeight * 4;
    mipLevels = static_cast<uint32_t>(
                    std::floor(std::log2(std::max(imageWidth, imageHeight)))) +
                1;
    printf("mipLevels = %d\n", mipLevels);

    if (!image_data) {
      throw std::runtime_error("failed to load texture image!");
    }

    VkBuffer stagingBuffer;
    VkDeviceMemory stagingBufferMemory;
    createBuffer(imageSize, VK_BUFFER_USAGE_TRANSFER_SRC_BIT,
                 VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT |
                     VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                 stagingBuffer, stagingBufferMemory);

    void* data;
    vkMapMemory(device, stagingBufferMemory, 0, imageSize, 0, &data);
    memcpy(data, image_data, static_cast<size_t>(imageSize));
    vkUnmapMemory(device, stagingBufferMemory);

    // VK_FORMAT_R8G8B8A8_UNORM changed to VK_FORMAT_R8G8B8A8_UINT
    createImage(
        imageWidth, imageHeight, VK_FORMAT_R8G8B8A8_UNORM,
        VK_IMAGE_TILING_OPTIMAL,
        VK_IMAGE_USAGE_STORAGE_BIT | VK_IMAGE_USAGE_TRANSFER_SRC_BIT |
            VK_IMAGE_USAGE_TRANSFER_DST_BIT | VK_IMAGE_USAGE_SAMPLED_BIT,
        VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT, textureImage, textureImageMemory);

    transitionImageLayout(textureImage, VK_FORMAT_R8G8B8A8_UINT,
                          VK_IMAGE_LAYOUT_UNDEFINED,
                          VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL);
    copyBufferToImage(stagingBuffer, textureImage,
                      static_cast<uint32_t>(imageWidth),
                      static_cast<uint32_t>(imageHeight));

    vkDestroyBuffer(device, stagingBuffer, nullptr);
    vkFreeMemory(device, stagingBufferMemory, nullptr);

    generateMipmaps(textureImage, VK_FORMAT_R8G8B8A8_UNORM);
  }

  void generateMipmaps(VkImage image, VkFormat imageFormat) {
    VkFormatProperties formatProperties;
    vkGetPhysicalDeviceFormatProperties(physicalDevice, imageFormat,
                                        &formatProperties);

    if (!(formatProperties.optimalTilingFeatures &
          VK_FORMAT_FEATURE_SAMPLED_IMAGE_FILTER_LINEAR_BIT)) {
      throw std::runtime_error(
          "texture image format does not support linear blitting!");
    }

    VkCommandBuffer commandBuffer = beginSingleTimeCommands();

    VkImageMemoryBarrier barrier = {};
    barrier.sType = VK_STRUCTURE_TYPE_IMAGE_MEMORY_BARRIER;
    barrier.image = image;
    barrier.srcQueueFamilyIndex = VK_QUEUE_FAMILY_IGNORED;
    barrier.dstQueueFamilyIndex = VK_QUEUE_FAMILY_IGNORED;
    barrier.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
    barrier.subresourceRange.baseArrayLayer = 0;
    barrier.subresourceRange.layerCount = 1;
    barrier.subresourceRange.levelCount = 1;

    int32_t mipWidth = imageWidth;
    int32_t mipHeight = imageHeight;

    for (uint32_t i = 1; i < mipLevels; i++) {
      barrier.subresourceRange.baseMipLevel = i - 1;
      barrier.oldLayout = VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL;
      barrier.newLayout = VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL;
      barrier.srcAccessMask = VK_ACCESS_TRANSFER_WRITE_BIT;
      barrier.dstAccessMask = VK_ACCESS_TRANSFER_READ_BIT;

      vkCmdPipelineBarrier(commandBuffer, VK_PIPELINE_STAGE_TRANSFER_BIT,
                           VK_PIPELINE_STAGE_TRANSFER_BIT, 0, 0, nullptr, 0,
                           nullptr, 1, &barrier);

      VkImageBlit blit = {};
      blit.srcOffsets[0] = {0, 0, 0};
      blit.srcOffsets[1] = {mipWidth, mipHeight, 1};
      blit.srcSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
      blit.srcSubresource.mipLevel = i - 1;
      blit.srcSubresource.baseArrayLayer = 0;
      blit.srcSubresource.layerCount = 1;
      blit.dstOffsets[0] = {0, 0, 0};
      blit.dstOffsets[1] = {mipWidth > 1 ? mipWidth / 2 : 1,
                            mipHeight > 1 ? mipHeight / 2 : 1, 1};
      blit.dstSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
      blit.dstSubresource.mipLevel = i;
      blit.dstSubresource.baseArrayLayer = 0;
      blit.dstSubresource.layerCount = 1;

      vkCmdBlitImage(commandBuffer, image, VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL,
                     image, VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL, 1, &blit,
                     VK_FILTER_LINEAR);

      barrier.oldLayout = VK_IMAGE_LAYOUT_TRANSFER_SRC_OPTIMAL;
      barrier.newLayout = VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL;
      barrier.srcAccessMask = VK_ACCESS_TRANSFER_READ_BIT;
      barrier.dstAccessMask = VK_ACCESS_SHADER_READ_BIT;

      vkCmdPipelineBarrier(commandBuffer, VK_PIPELINE_STAGE_TRANSFER_BIT,
                           VK_PIPELINE_STAGE_FRAGMENT_SHADER_BIT, 0, 0, nullptr,
                           0, nullptr, 1, &barrier);

      if (mipWidth > 1) mipWidth /= 2;
      if (mipHeight > 1) mipHeight /= 2;
    }

    barrier.subresourceRange.baseMipLevel = mipLevels - 1;
    barrier.oldLayout = VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL;
    barrier.newLayout = VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL;
    barrier.srcAccessMask = VK_ACCESS_TRANSFER_WRITE_BIT;
    barrier.dstAccessMask = VK_ACCESS_SHADER_READ_BIT;

    vkCmdPipelineBarrier(commandBuffer, VK_PIPELINE_STAGE_TRANSFER_BIT,
                         VK_PIPELINE_STAGE_FRAGMENT_SHADER_BIT, 0, 0, nullptr,
                         0, nullptr, 1, &barrier);

    endSingleTimeCommands(commandBuffer);
  }

#ifdef _WIN64  // For windows
  HANDLE getVkImageMemHandle(
      VkExternalMemoryHandleTypeFlagsKHR externalMemoryHandleType) {
    HANDLE handle;

    VkMemoryGetWin32HandleInfoKHR vkMemoryGetWin32HandleInfoKHR = {};
    vkMemoryGetWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_MEMORY_GET_WIN32_HANDLE_INFO_KHR;
    vkMemoryGetWin32HandleInfoKHR.pNext = NULL;
    vkMemoryGetWin32HandleInfoKHR.memory = textureImageMemory;
    vkMemoryGetWin32HandleInfoKHR.handleType =
        (VkExternalMemoryHandleTypeFlagBitsKHR)externalMemoryHandleType;

    fpGetMemoryWin32HandleKHR(device, &vkMemoryGetWin32HandleInfoKHR, &handle);
    return handle;
  }
  HANDLE getVkSemaphoreHandle(
      VkExternalSemaphoreHandleTypeFlagBitsKHR externalSemaphoreHandleType,
      VkSemaphore& semVkCuda) {
    HANDLE handle;

    VkSemaphoreGetWin32HandleInfoKHR vulkanSemaphoreGetWin32HandleInfoKHR = {};
    vulkanSemaphoreGetWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_SEMAPHORE_GET_WIN32_HANDLE_INFO_KHR;
    vulkanSemaphoreGetWin32HandleInfoKHR.pNext = NULL;
    vulkanSemaphoreGetWin32HandleInfoKHR.semaphore = semVkCuda;
    vulkanSemaphoreGetWin32HandleInfoKHR.handleType =
        externalSemaphoreHandleType;

    fpGetSemaphoreWin32HandleKHR(device, &vulkanSemaphoreGetWin32HandleInfoKHR,
                                 &handle);

    return handle;
  }
#else
  int getVkImageMemHandle(
      VkExternalMemoryHandleTypeFlagsKHR externalMemoryHandleType) {
    if (externalMemoryHandleType ==
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT_KHR) {
      int fd;

      VkMemoryGetFdInfoKHR vkMemoryGetFdInfoKHR = {};
      vkMemoryGetFdInfoKHR.sType = VK_STRUCTURE_TYPE_MEMORY_GET_FD_INFO_KHR;
      vkMemoryGetFdInfoKHR.pNext = NULL;
      vkMemoryGetFdInfoKHR.memory = textureImageMemory;
      vkMemoryGetFdInfoKHR.handleType =
          VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT_KHR;

      fpGetMemoryFdKHR(device, &vkMemoryGetFdInfoKHR, &fd);

      return fd;
    }
    return -1;
  }

  int getVkSemaphoreHandle(
      VkExternalSemaphoreHandleTypeFlagBitsKHR externalSemaphoreHandleType,
      VkSemaphore& semVkCuda) {
    if (externalSemaphoreHandleType ==
        VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT) {
      int fd;

      VkSemaphoreGetFdInfoKHR vulkanSemaphoreGetFdInfoKHR = {};
      vulkanSemaphoreGetFdInfoKHR.sType =
          VK_STRUCTURE_TYPE_SEMAPHORE_GET_FD_INFO_KHR;
      vulkanSemaphoreGetFdInfoKHR.pNext = NULL;
      vulkanSemaphoreGetFdInfoKHR.semaphore = semVkCuda;
      vulkanSemaphoreGetFdInfoKHR.handleType =
          VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT_KHR;

      fpGetSemaphoreFdKHR(device, &vulkanSemaphoreGetFdInfoKHR, &fd);

      return fd;
    }
    return -1;
  }
#endif

  void createTextureImageView() {
    textureImageView = createImageView(textureImage, VK_FORMAT_R8G8B8A8_UNORM);
  }

  void createTextureSampler() {
    VkSamplerCreateInfo samplerInfo = {};
    samplerInfo.sType = VK_STRUCTURE_TYPE_SAMPLER_CREATE_INFO;
    samplerInfo.magFilter = VK_FILTER_LINEAR;
    samplerInfo.minFilter = VK_FILTER_LINEAR;
    samplerInfo.addressModeU = VK_SAMPLER_ADDRESS_MODE_REPEAT;
    samplerInfo.addressModeV = VK_SAMPLER_ADDRESS_MODE_REPEAT;
    samplerInfo.addressModeW = VK_SAMPLER_ADDRESS_MODE_REPEAT;
    samplerInfo.anisotropyEnable = VK_TRUE;
    samplerInfo.maxAnisotropy = 16;
    samplerInfo.borderColor = VK_BORDER_COLOR_INT_OPAQUE_BLACK;
    samplerInfo.unnormalizedCoordinates = VK_FALSE;
    samplerInfo.compareEnable = VK_FALSE;
    samplerInfo.compareOp = VK_COMPARE_OP_ALWAYS;
    samplerInfo.mipmapMode = VK_SAMPLER_MIPMAP_MODE_LINEAR;
    samplerInfo.minLod = 0;  // Optional
    samplerInfo.maxLod = static_cast<float>(mipLevels);
    samplerInfo.mipLodBias = 0;  // Optional

    if (vkCreateSampler(device, &samplerInfo, nullptr, &textureSampler) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create texture sampler!");
    }
  }

  VkImageView createImageView(VkImage image, VkFormat format) {
    VkImageViewCreateInfo viewInfo = {};
    viewInfo.sType = VK_STRUCTURE_TYPE_IMAGE_VIEW_CREATE_INFO;
    viewInfo.image = image;
    viewInfo.viewType = VK_IMAGE_VIEW_TYPE_2D;
    viewInfo.format = format;
    viewInfo.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
    viewInfo.subresourceRange.baseMipLevel = 0;
    viewInfo.subresourceRange.levelCount = mipLevels;
    viewInfo.subresourceRange.baseArrayLayer = 0;
    viewInfo.subresourceRange.layerCount = 1;

    VkImageView imageView;
    if (vkCreateImageView(device, &viewInfo, nullptr, &imageView) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create texture image view!");
    }

    return imageView;
  }

  void createImage(uint32_t width, uint32_t height, VkFormat format,
                   VkImageTiling tiling, VkImageUsageFlags usage,
                   VkMemoryPropertyFlags properties, VkImage& image,
                   VkDeviceMemory& imageMemory) {
    VkImageCreateInfo imageInfo = {};
    imageInfo.sType = VK_STRUCTURE_TYPE_IMAGE_CREATE_INFO;
    imageInfo.imageType = VK_IMAGE_TYPE_2D;
    imageInfo.extent.width = width;
    imageInfo.extent.height = height;
    imageInfo.extent.depth = 1;
    imageInfo.mipLevels = mipLevels;
    imageInfo.arrayLayers = 1;
    imageInfo.format = format;
    imageInfo.tiling = tiling;
    imageInfo.initialLayout = VK_IMAGE_LAYOUT_UNDEFINED;
    imageInfo.usage = usage;
    imageInfo.samples = VK_SAMPLE_COUNT_1_BIT;
    imageInfo.sharingMode = VK_SHARING_MODE_EXCLUSIVE;

    VkExternalMemoryImageCreateInfo vkExternalMemImageCreateInfo = {};
    vkExternalMemImageCreateInfo.sType =
        VK_STRUCTURE_TYPE_EXTERNAL_MEMORY_IMAGE_CREATE_INFO;
    vkExternalMemImageCreateInfo.pNext = NULL;
#ifdef _WIN64
    vkExternalMemImageCreateInfo.handleTypes =
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT;
#else
    vkExternalMemImageCreateInfo.handleTypes =
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT_KHR;
#endif

    imageInfo.pNext = &vkExternalMemImageCreateInfo;

    if (vkCreateImage(device, &imageInfo, nullptr, &image) != VK_SUCCESS) {
      throw std::runtime_error("failed to create image!");
    }

    VkMemoryRequirements memRequirements;
    vkGetImageMemoryRequirements(device, image, &memRequirements);

#ifdef _WIN64
    WindowsSecurityAttributes winSecurityAttributes;

    VkExportMemoryWin32HandleInfoKHR vulkanExportMemoryWin32HandleInfoKHR = {};
    vulkanExportMemoryWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_EXPORT_MEMORY_WIN32_HANDLE_INFO_KHR;
    vulkanExportMemoryWin32HandleInfoKHR.pNext = NULL;
    vulkanExportMemoryWin32HandleInfoKHR.pAttributes = &winSecurityAttributes;
    vulkanExportMemoryWin32HandleInfoKHR.dwAccess =
        DXGI_SHARED_RESOURCE_READ | DXGI_SHARED_RESOURCE_WRITE;
    vulkanExportMemoryWin32HandleInfoKHR.name = (LPCWSTR)NULL;
#endif
    VkExportMemoryAllocateInfoKHR vulkanExportMemoryAllocateInfoKHR = {};
    vulkanExportMemoryAllocateInfoKHR.sType =
        VK_STRUCTURE_TYPE_EXPORT_MEMORY_ALLOCATE_INFO_KHR;
#ifdef _WIN64
    vulkanExportMemoryAllocateInfoKHR.pNext =
        IsWindows8OrGreater() ? &vulkanExportMemoryWin32HandleInfoKHR : NULL;
    vulkanExportMemoryAllocateInfoKHR.handleTypes =
        IsWindows8OrGreater()
            ? VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT;
#else
    vulkanExportMemoryAllocateInfoKHR.pNext = NULL;
    vulkanExportMemoryAllocateInfoKHR.handleTypes =
        VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT_KHR;
#endif

    VkMemoryAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
    allocInfo.allocationSize = memRequirements.size;
    allocInfo.pNext = &vulkanExportMemoryAllocateInfoKHR;
    allocInfo.memoryTypeIndex =
        findMemoryType(memRequirements.memoryTypeBits, properties);

    VkMemoryRequirements vkMemoryRequirements = {};
    vkGetImageMemoryRequirements(device, image, &vkMemoryRequirements);
    totalImageMemSize = vkMemoryRequirements.size;

    if (vkAllocateMemory(device, &allocInfo, nullptr, &textureImageMemory) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to allocate image memory!");
    }

    vkBindImageMemory(device, image, textureImageMemory, 0);
  }

  void cudaVkImportSemaphore() {
    hipExternalSemaphoreHandleDesc externalSemaphoreHandleDesc;
    memset(&externalSemaphoreHandleDesc, 0,
           sizeof(externalSemaphoreHandleDesc));
#ifdef _WIN64
    externalSemaphoreHandleDesc.type =
        IsWindows8OrGreater() ? hipExternalSemaphoreHandleTypeOpaqueWin32
                              : hipExternalSemaphoreHandleTypeOpaqueWin32Kmt;
    externalSemaphoreHandleDesc.handle.win32.handle = getVkSemaphoreHandle(
        IsWindows8OrGreater()
            ? VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT,
        cudaUpdateVkSemaphore);
#else
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd = getVkSemaphoreHandle(
        VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT, cudaUpdateVkSemaphore);
#endif
    externalSemaphoreHandleDesc.flags = 0;

    checkCudaErrors(hipImportExternalSemaphore(&cudaExtCudaUpdateVkSemaphore,
                                                &externalSemaphoreHandleDesc));

    memset(&externalSemaphoreHandleDesc, 0,
           sizeof(externalSemaphoreHandleDesc));
#ifdef _WIN64
    externalSemaphoreHandleDesc.type =
        IsWindows8OrGreater() ? hipExternalSemaphoreHandleTypeOpaqueWin32
                              : hipExternalSemaphoreHandleTypeOpaqueWin32Kmt;
    ;
    externalSemaphoreHandleDesc.handle.win32.handle = getVkSemaphoreHandle(
        IsWindows8OrGreater()
            ? VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT,
        vkUpdateCudaSemaphore);
#else
    externalSemaphoreHandleDesc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    externalSemaphoreHandleDesc.handle.fd = getVkSemaphoreHandle(
        VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT, vkUpdateCudaSemaphore);
#endif
    externalSemaphoreHandleDesc.flags = 0;
    checkCudaErrors(hipImportExternalSemaphore(&cudaExtVkUpdateCudaSemaphore,
                                                &externalSemaphoreHandleDesc));
    printf("CUDA Imported Vulkan semaphore\n");
  }

  void cudaVkImportImageMem() {
    hipExternalMemoryHandleDesc cudaExtMemHandleDesc;
    memset(&cudaExtMemHandleDesc, 0, sizeof(cudaExtMemHandleDesc));
#ifdef _WIN64
    cudaExtMemHandleDesc.type =
        IsWindows8OrGreater() ? hipExternalMemoryHandleTypeOpaqueWin32
                              : hipExternalMemoryHandleTypeOpaqueWin32Kmt;
    cudaExtMemHandleDesc.handle.win32.handle = getVkImageMemHandle(
        IsWindows8OrGreater()
            ? VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT);
#else
    cudaExtMemHandleDesc.type = hipExternalMemoryHandleTypeOpaqueFd;

    cudaExtMemHandleDesc.handle.fd =
        getVkImageMemHandle(VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT_KHR);
#endif
    cudaExtMemHandleDesc.size = totalImageMemSize;

    checkCudaErrors(hipImportExternalMemory(&cudaExtMemImageBuffer,
                                             &cudaExtMemHandleDesc));

    cudaExternalMemoryMipmappedArrayDesc externalMemoryMipmappedArrayDesc;

    memset(&externalMemoryMipmappedArrayDesc, 0,
           sizeof(externalMemoryMipmappedArrayDesc));

    hipExtent extent = make_hipExtent(imageWidth, imageHeight, 0);
    hipChannelFormatDesc formatDesc;
    formatDesc.x = 8;
    formatDesc.y = 8;
    formatDesc.z = 8;
    formatDesc.w = 8;
    formatDesc.f = hipChannelFormatKindUnsigned;

    externalMemoryMipmappedArrayDesc.offset = 0;
    externalMemoryMipmappedArrayDesc.formatDesc = formatDesc;
    externalMemoryMipmappedArrayDesc.extent = extent;
    externalMemoryMipmappedArrayDesc.flags = 0;
    externalMemoryMipmappedArrayDesc.numLevels = mipLevels;

    checkCudaErrors(cudaExternalMemoryGetMappedMipmappedArray(
        &cudaMipmappedImageArray, cudaExtMemImageBuffer,
        &externalMemoryMipmappedArrayDesc));

    checkCudaErrors(hipMallocMipmappedArray(&cudaMipmappedImageArrayTemp,
                                             &formatDesc, extent, mipLevels));
    checkCudaErrors(hipMallocMipmappedArray(&cudaMipmappedImageArrayOrig,
                                             &formatDesc, extent, mipLevels));

    for (int mipLevelIdx = 0; mipLevelIdx < mipLevels; mipLevelIdx++) {
      hipArray_t cudaMipLevelArray, cudaMipLevelArrayTemp,
          cudaMipLevelArrayOrig;
      hipResourceDesc resourceDesc;

      checkCudaErrors(hipGetMipmappedArrayLevel(
          &cudaMipLevelArray, cudaMipmappedImageArray, mipLevelIdx));
      checkCudaErrors(hipGetMipmappedArrayLevel(
          &cudaMipLevelArrayTemp, cudaMipmappedImageArrayTemp, mipLevelIdx));
      checkCudaErrors(hipGetMipmappedArrayLevel(
          &cudaMipLevelArrayOrig, cudaMipmappedImageArrayOrig, mipLevelIdx));

      uint32_t width =
          (imageWidth >> mipLevelIdx) ? (imageWidth >> mipLevelIdx) : 1;
      uint32_t height =
          (imageHeight >> mipLevelIdx) ? (imageHeight >> mipLevelIdx) : 1;
      checkCudaErrors(hipMemcpy2DArrayToArray(
          cudaMipLevelArrayOrig, 0, 0, cudaMipLevelArray, 0, 0,
          width * sizeof(uchar4), height, hipMemcpyDeviceToDevice));

      memset(&resourceDesc, 0, sizeof(resourceDesc));
      resourceDesc.resType = hipResourceTypeArray;
      resourceDesc.res.array.array = cudaMipLevelArray;

      hipSurfaceObject_t surfaceObject;
      checkCudaErrors(hipCreateSurfaceObject(&surfaceObject, &resourceDesc));

      surfaceObjectList.push_back(surfaceObject);

      memset(&resourceDesc, 0, sizeof(resourceDesc));
      resourceDesc.resType = hipResourceTypeArray;
      resourceDesc.res.array.array = cudaMipLevelArrayTemp;

      hipSurfaceObject_t surfaceObjectTemp;
      checkCudaErrors(
          hipCreateSurfaceObject(&surfaceObjectTemp, &resourceDesc));
      surfaceObjectListTemp.push_back(surfaceObjectTemp);
    }

    hipResourceDesc resDescr;
    memset(&resDescr, 0, sizeof(hipResourceDesc));

    resDescr.resType = hipResourceTypeMipmappedArray;
    resDescr.res.mipmap.mipmap = cudaMipmappedImageArrayOrig;

    hipTextureDesc texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));

    texDescr.normalizedCoords = true;
    texDescr.filterMode = hipFilterModeLinear;
    texDescr.mipmapFilterMode = hipFilterModeLinear;

    texDescr.addressMode[0] = hipAddressModeWrap;
    texDescr.addressMode[1] = hipAddressModeWrap;

    texDescr.maxMipmapLevelClamp = float(mipLevels - 1);

    texDescr.readMode = hipReadModeNormalizedFloat;

    checkCudaErrors(hipCreateTextureObject(&textureObjMipMapInput, &resDescr,
                                            &texDescr, NULL));

    checkCudaErrors(hipMalloc((void**)&d_surfaceObjectList,
                               sizeof(hipSurfaceObject_t) * mipLevels));
    checkCudaErrors(hipMalloc((void**)&d_surfaceObjectListTemp,
                               sizeof(hipSurfaceObject_t) * mipLevels));

    checkCudaErrors(hipMemcpy(d_surfaceObjectList, surfaceObjectList.data(),
                               sizeof(hipSurfaceObject_t) * mipLevels,
                               hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(
        d_surfaceObjectListTemp, surfaceObjectListTemp.data(),
        sizeof(hipSurfaceObject_t) * mipLevels, hipMemcpyHostToDevice));

    printf("CUDA Kernel Vulkan image buffer\n");
  }

  void cudaUpdateVkImage() {
    cudaVkSemaphoreWait(cudaExtVkUpdateCudaSemaphore);

    int nthreads = 128;

    /*Perform 2D box filter on image using CUDA */
    d_boxfilter_rgba_x<<<imageHeight / nthreads, nthreads, 0, streamToRun>>>(
        d_surfaceObjectListTemp, textureObjMipMapInput, imageWidth, imageHeight,
        mipLevels, filter_radius);

    d_boxfilter_rgba_y<<<imageWidth / nthreads, nthreads, 0, streamToRun>>>(
        d_surfaceObjectList, d_surfaceObjectListTemp, imageWidth, imageHeight,
        mipLevels, filter_radius);

    varySigma();

    cudaVkSemaphoreSignal(cudaExtCudaUpdateVkSemaphore);
  }

  void transitionImageLayout(VkImage image, VkFormat format,
                             VkImageLayout oldLayout, VkImageLayout newLayout) {
    VkCommandBuffer commandBuffer = beginSingleTimeCommands();

    VkImageMemoryBarrier barrier = {};
    barrier.sType = VK_STRUCTURE_TYPE_IMAGE_MEMORY_BARRIER;
    barrier.oldLayout = oldLayout;
    barrier.newLayout = newLayout;
    barrier.srcQueueFamilyIndex = VK_QUEUE_FAMILY_IGNORED;
    barrier.dstQueueFamilyIndex = VK_QUEUE_FAMILY_IGNORED;
    barrier.image = image;
    barrier.subresourceRange.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
    barrier.subresourceRange.baseMipLevel = 0;
    barrier.subresourceRange.levelCount = mipLevels;
    barrier.subresourceRange.baseArrayLayer = 0;
    barrier.subresourceRange.layerCount = 1;

    VkPipelineStageFlags sourceStage;
    VkPipelineStageFlags destinationStage;

    if (oldLayout == VK_IMAGE_LAYOUT_UNDEFINED &&
        newLayout == VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL) {
      barrier.srcAccessMask = 0;
      barrier.dstAccessMask = VK_ACCESS_TRANSFER_WRITE_BIT;

      sourceStage = VK_PIPELINE_STAGE_TOP_OF_PIPE_BIT;
      destinationStage = VK_PIPELINE_STAGE_TRANSFER_BIT;
    } else if (oldLayout == VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL &&
               newLayout == VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL) {
      barrier.srcAccessMask = VK_ACCESS_TRANSFER_WRITE_BIT;
      barrier.dstAccessMask = VK_ACCESS_SHADER_READ_BIT;

      sourceStage = VK_PIPELINE_STAGE_TRANSFER_BIT;
      destinationStage = VK_PIPELINE_STAGE_FRAGMENT_SHADER_BIT;
    } else {
      throw std::invalid_argument("unsupported layout transition!");
    }

    vkCmdPipelineBarrier(commandBuffer, sourceStage, destinationStage, 0, 0,
                         nullptr, 0, nullptr, 1, &barrier);

    endSingleTimeCommands(commandBuffer);
  }

  void copyBufferToImage(VkBuffer buffer, VkImage image, uint32_t width,
                         uint32_t height) {
    VkCommandBuffer commandBuffer = beginSingleTimeCommands();

    VkBufferImageCopy region = {};
    region.bufferOffset = 0;
    region.bufferRowLength = 0;
    region.bufferImageHeight = 0;
    region.imageSubresource.aspectMask = VK_IMAGE_ASPECT_COLOR_BIT;
    region.imageSubresource.mipLevel = 0;
    region.imageSubresource.baseArrayLayer = 0;
    region.imageSubresource.layerCount = 1;
    region.imageOffset = {0, 0, 0};
    region.imageExtent = {width, height, 1};

    vkCmdCopyBufferToImage(commandBuffer, buffer, image,
                           VK_IMAGE_LAYOUT_TRANSFER_DST_OPTIMAL, 1, &region);

    endSingleTimeCommands(commandBuffer);
  }

  void createVertexBuffer() {
    VkDeviceSize bufferSize = sizeof(vertices[0]) * vertices.size();

    VkBuffer stagingBuffer;
    VkDeviceMemory stagingBufferMemory;
    createBuffer(bufferSize, VK_BUFFER_USAGE_TRANSFER_SRC_BIT,
                 VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT |
                     VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                 stagingBuffer, stagingBufferMemory);

    void* data;
    vkMapMemory(device, stagingBufferMemory, 0, bufferSize, 0, &data);
    memcpy(data, vertices.data(), (size_t)bufferSize);
    vkUnmapMemory(device, stagingBufferMemory);

    createBuffer(
        bufferSize,
        VK_BUFFER_USAGE_TRANSFER_DST_BIT | VK_BUFFER_USAGE_VERTEX_BUFFER_BIT,
        VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT, vertexBuffer, vertexBufferMemory);

    copyBuffer(stagingBuffer, vertexBuffer, bufferSize);

    vkDestroyBuffer(device, stagingBuffer, nullptr);
    vkFreeMemory(device, stagingBufferMemory, nullptr);
  }

  void createIndexBuffer() {
    VkDeviceSize bufferSize = sizeof(indices[0]) * indices.size();

    VkBuffer stagingBuffer;
    VkDeviceMemory stagingBufferMemory;
    createBuffer(bufferSize, VK_BUFFER_USAGE_TRANSFER_SRC_BIT,
                 VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT |
                     VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                 stagingBuffer, stagingBufferMemory);

    void* data;
    vkMapMemory(device, stagingBufferMemory, 0, bufferSize, 0, &data);
    memcpy(data, indices.data(), (size_t)bufferSize);
    vkUnmapMemory(device, stagingBufferMemory);

    createBuffer(
        bufferSize,
        VK_BUFFER_USAGE_TRANSFER_DST_BIT | VK_BUFFER_USAGE_INDEX_BUFFER_BIT,
        VK_MEMORY_PROPERTY_DEVICE_LOCAL_BIT, indexBuffer, indexBufferMemory);

    copyBuffer(stagingBuffer, indexBuffer, bufferSize);

    vkDestroyBuffer(device, stagingBuffer, nullptr);
    vkFreeMemory(device, stagingBufferMemory, nullptr);
  }

  void createUniformBuffers() {
    VkDeviceSize bufferSize = sizeof(UniformBufferObject);

    uniformBuffers.resize(swapChainImages.size());
    uniformBuffersMemory.resize(swapChainImages.size());

    for (size_t i = 0; i < swapChainImages.size(); i++) {
      createBuffer(bufferSize, VK_BUFFER_USAGE_UNIFORM_BUFFER_BIT,
                   VK_MEMORY_PROPERTY_HOST_VISIBLE_BIT |
                       VK_MEMORY_PROPERTY_HOST_COHERENT_BIT,
                   uniformBuffers[i], uniformBuffersMemory[i]);
    }
  }

  void createDescriptorPool() {
    std::array<VkDescriptorPoolSize, 2> poolSizes = {};
    poolSizes[0].type = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
    poolSizes[0].descriptorCount =
        static_cast<uint32_t>(swapChainImages.size());
    poolSizes[1].type = VK_DESCRIPTOR_TYPE_COMBINED_IMAGE_SAMPLER;
    poolSizes[1].descriptorCount =
        static_cast<uint32_t>(swapChainImages.size());

    VkDescriptorPoolCreateInfo poolInfo = {};
    poolInfo.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_POOL_CREATE_INFO;
    poolInfo.poolSizeCount = static_cast<uint32_t>(poolSizes.size());
    poolInfo.pPoolSizes = poolSizes.data();
    poolInfo.maxSets = static_cast<uint32_t>(swapChainImages.size());

    if (vkCreateDescriptorPool(device, &poolInfo, nullptr, &descriptorPool) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create descriptor pool!");
    }
  }

  void createDescriptorSets() {
    std::vector<VkDescriptorSetLayout> layouts(swapChainImages.size(),
                                               descriptorSetLayout);
    VkDescriptorSetAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_DESCRIPTOR_SET_ALLOCATE_INFO;
    allocInfo.descriptorPool = descriptorPool;
    allocInfo.descriptorSetCount =
        static_cast<uint32_t>(swapChainImages.size());
    allocInfo.pSetLayouts = layouts.data();

    descriptorSets.resize(swapChainImages.size());
    if (vkAllocateDescriptorSets(device, &allocInfo, descriptorSets.data()) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to allocate descriptor sets!");
    }

    for (size_t i = 0; i < swapChainImages.size(); i++) {
      VkDescriptorBufferInfo bufferInfo = {};
      bufferInfo.buffer = uniformBuffers[i];
      bufferInfo.offset = 0;
      bufferInfo.range = sizeof(UniformBufferObject);

      VkDescriptorImageInfo imageInfo = {};
      imageInfo.imageLayout = VK_IMAGE_LAYOUT_SHADER_READ_ONLY_OPTIMAL;
      imageInfo.imageView = textureImageView;
      imageInfo.sampler = textureSampler;

      std::array<VkWriteDescriptorSet, 2> descriptorWrites = {};

      descriptorWrites[0].sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
      descriptorWrites[0].dstSet = descriptorSets[i];
      descriptorWrites[0].dstBinding = 0;
      descriptorWrites[0].dstArrayElement = 0;
      descriptorWrites[0].descriptorType = VK_DESCRIPTOR_TYPE_UNIFORM_BUFFER;
      descriptorWrites[0].descriptorCount = 1;
      descriptorWrites[0].pBufferInfo = &bufferInfo;

      descriptorWrites[1].sType = VK_STRUCTURE_TYPE_WRITE_DESCRIPTOR_SET;
      descriptorWrites[1].dstSet = descriptorSets[i];
      descriptorWrites[1].dstBinding = 1;
      descriptorWrites[1].dstArrayElement = 0;
      descriptorWrites[1].descriptorType =
          VK_DESCRIPTOR_TYPE_COMBINED_IMAGE_SAMPLER;
      descriptorWrites[1].descriptorCount = 1;
      descriptorWrites[1].pImageInfo = &imageInfo;

      vkUpdateDescriptorSets(device,
                             static_cast<uint32_t>(descriptorWrites.size()),
                             descriptorWrites.data(), 0, nullptr);
    }
  }

  void createBuffer(VkDeviceSize size, VkBufferUsageFlags usage,
                    VkMemoryPropertyFlags properties, VkBuffer& buffer,
                    VkDeviceMemory& bufferMemory) {
    VkBufferCreateInfo bufferInfo = {};
    bufferInfo.sType = VK_STRUCTURE_TYPE_BUFFER_CREATE_INFO;
    bufferInfo.size = size;
    bufferInfo.usage = usage;
    bufferInfo.sharingMode = VK_SHARING_MODE_EXCLUSIVE;

    if (vkCreateBuffer(device, &bufferInfo, nullptr, &buffer) != VK_SUCCESS) {
      throw std::runtime_error("failed to create buffer!");
    }

    VkMemoryRequirements memRequirements;
    vkGetBufferMemoryRequirements(device, buffer, &memRequirements);

    VkMemoryAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_MEMORY_ALLOCATE_INFO;
    allocInfo.allocationSize = memRequirements.size;
    allocInfo.memoryTypeIndex =
        findMemoryType(memRequirements.memoryTypeBits, properties);

    if (vkAllocateMemory(device, &allocInfo, nullptr, &bufferMemory) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to allocate buffer memory!");
    }

    vkBindBufferMemory(device, buffer, bufferMemory, 0);
  }

  VkCommandBuffer beginSingleTimeCommands() {
    VkCommandBufferAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_ALLOCATE_INFO;
    allocInfo.level = VK_COMMAND_BUFFER_LEVEL_PRIMARY;
    allocInfo.commandPool = commandPool;
    allocInfo.commandBufferCount = 1;

    VkCommandBuffer commandBuffer;
    vkAllocateCommandBuffers(device, &allocInfo, &commandBuffer);

    VkCommandBufferBeginInfo beginInfo = {};
    beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
    beginInfo.flags = VK_COMMAND_BUFFER_USAGE_ONE_TIME_SUBMIT_BIT;

    vkBeginCommandBuffer(commandBuffer, &beginInfo);

    return commandBuffer;
  }

  void endSingleTimeCommands(VkCommandBuffer commandBuffer) {
    vkEndCommandBuffer(commandBuffer);

    VkSubmitInfo submitInfo = {};
    submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;
    submitInfo.commandBufferCount = 1;
    submitInfo.pCommandBuffers = &commandBuffer;

    vkQueueSubmit(graphicsQueue, 1, &submitInfo, VK_NULL_HANDLE);
    vkQueueWaitIdle(graphicsQueue);

    vkFreeCommandBuffers(device, commandPool, 1, &commandBuffer);
  }

  void copyBuffer(VkBuffer srcBuffer, VkBuffer dstBuffer, VkDeviceSize size) {
    VkCommandBuffer commandBuffer = beginSingleTimeCommands();

    VkBufferCopy copyRegion = {};
    copyRegion.size = size;
    vkCmdCopyBuffer(commandBuffer, srcBuffer, dstBuffer, 1, &copyRegion);

    endSingleTimeCommands(commandBuffer);
  }

  uint32_t findMemoryType(uint32_t typeFilter,
                          VkMemoryPropertyFlags properties) {
    VkPhysicalDeviceMemoryProperties memProperties;
    vkGetPhysicalDeviceMemoryProperties(physicalDevice, &memProperties);

    for (uint32_t i = 0; i < memProperties.memoryTypeCount; i++) {
      if ((typeFilter & (1 << i)) &&
          (memProperties.memoryTypes[i].propertyFlags & properties) ==
              properties) {
        return i;
      }
    }

    throw std::runtime_error("failed to find suitable memory type!");
  }

  void createCommandBuffers() {
    commandBuffers.resize(swapChainFramebuffers.size());

    VkCommandBufferAllocateInfo allocInfo = {};
    allocInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_ALLOCATE_INFO;
    allocInfo.commandPool = commandPool;
    allocInfo.level = VK_COMMAND_BUFFER_LEVEL_PRIMARY;
    allocInfo.commandBufferCount = (uint32_t)commandBuffers.size();

    if (vkAllocateCommandBuffers(device, &allocInfo, commandBuffers.data()) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to allocate command buffers!");
    }

    for (size_t i = 0; i < commandBuffers.size(); i++) {
      VkCommandBufferBeginInfo beginInfo = {};
      beginInfo.sType = VK_STRUCTURE_TYPE_COMMAND_BUFFER_BEGIN_INFO;
      beginInfo.flags = VK_COMMAND_BUFFER_USAGE_SIMULTANEOUS_USE_BIT;

      if (vkBeginCommandBuffer(commandBuffers[i], &beginInfo) != VK_SUCCESS) {
        throw std::runtime_error("failed to begin recording command buffer!");
      }

      VkRenderPassBeginInfo renderPassInfo = {};
      renderPassInfo.sType = VK_STRUCTURE_TYPE_RENDER_PASS_BEGIN_INFO;
      renderPassInfo.renderPass = renderPass;
      renderPassInfo.framebuffer = swapChainFramebuffers[i];
      renderPassInfo.renderArea.offset = {0, 0};
      renderPassInfo.renderArea.extent = swapChainExtent;

      VkClearValue clearColor = {0.0f, 0.0f, 0.0f, 1.0f};
      renderPassInfo.clearValueCount = 1;
      renderPassInfo.pClearValues = &clearColor;

      vkCmdBeginRenderPass(commandBuffers[i], &renderPassInfo,
                           VK_SUBPASS_CONTENTS_INLINE);

      vkCmdBindPipeline(commandBuffers[i], VK_PIPELINE_BIND_POINT_GRAPHICS,
                        graphicsPipeline);

      VkBuffer vertexBuffers[] = {vertexBuffer};
      VkDeviceSize offsets[] = {0};
      vkCmdBindVertexBuffers(commandBuffers[i], 0, 1, vertexBuffers, offsets);

      vkCmdBindIndexBuffer(commandBuffers[i], indexBuffer, 0,
                           VK_INDEX_TYPE_UINT16);

      vkCmdBindDescriptorSets(commandBuffers[i],
                              VK_PIPELINE_BIND_POINT_GRAPHICS, pipelineLayout,
                              0, 1, &descriptorSets[i], 0, nullptr);

      vkCmdDrawIndexed(commandBuffers[i], static_cast<uint32_t>(indices.size()),
                       1, 0, 0, 0);
      // vkCmdDraw(commandBuffers[i], static_cast<uint32_t>(vertices.size()), 1,
      // 0, 0);

      vkCmdEndRenderPass(commandBuffers[i]);

      if (vkEndCommandBuffer(commandBuffers[i]) != VK_SUCCESS) {
        throw std::runtime_error("failed to record command buffer!");
      }
    }
  }

  void createSyncObjects() {
    imageAvailableSemaphores.resize(MAX_FRAMES);
    renderFinishedSemaphores.resize(MAX_FRAMES);
    inFlightFences.resize(MAX_FRAMES);

    VkSemaphoreCreateInfo semaphoreInfo = {};
    semaphoreInfo.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;

    VkFenceCreateInfo fenceInfo = {};
    fenceInfo.sType = VK_STRUCTURE_TYPE_FENCE_CREATE_INFO;
    fenceInfo.flags = VK_FENCE_CREATE_SIGNALED_BIT;

    for (size_t i = 0; i < MAX_FRAMES; i++) {
      if (vkCreateSemaphore(device, &semaphoreInfo, nullptr,
                            &imageAvailableSemaphores[i]) != VK_SUCCESS ||
          vkCreateSemaphore(device, &semaphoreInfo, nullptr,
                            &renderFinishedSemaphores[i]) != VK_SUCCESS ||
          vkCreateFence(device, &fenceInfo, nullptr, &inFlightFences[i]) !=
              VK_SUCCESS) {
        throw std::runtime_error(
            "failed to create synchronization objects for a frame!");
      }
    }
  }

  void createSyncObjectsExt() {
    VkSemaphoreCreateInfo semaphoreInfo = {};
    semaphoreInfo.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;

    memset(&semaphoreInfo, 0, sizeof(semaphoreInfo));
    semaphoreInfo.sType = VK_STRUCTURE_TYPE_SEMAPHORE_CREATE_INFO;

#ifdef _WIN64
    WindowsSecurityAttributes winSecurityAttributes;

    VkExportSemaphoreWin32HandleInfoKHR
        vulkanExportSemaphoreWin32HandleInfoKHR = {};
    vulkanExportSemaphoreWin32HandleInfoKHR.sType =
        VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_WIN32_HANDLE_INFO_KHR;
    vulkanExportSemaphoreWin32HandleInfoKHR.pNext = NULL;
    vulkanExportSemaphoreWin32HandleInfoKHR.pAttributes =
        &winSecurityAttributes;
    vulkanExportSemaphoreWin32HandleInfoKHR.dwAccess =
        DXGI_SHARED_RESOURCE_READ | DXGI_SHARED_RESOURCE_WRITE;
    vulkanExportSemaphoreWin32HandleInfoKHR.name = (LPCWSTR)NULL;
#endif
    VkExportSemaphoreCreateInfoKHR vulkanExportSemaphoreCreateInfo = {};
    vulkanExportSemaphoreCreateInfo.sType =
        VK_STRUCTURE_TYPE_EXPORT_SEMAPHORE_CREATE_INFO_KHR;
#ifdef _WIN64
    vulkanExportSemaphoreCreateInfo.pNext =
        IsWindows8OrGreater() ? &vulkanExportSemaphoreWin32HandleInfoKHR : NULL;
    vulkanExportSemaphoreCreateInfo.handleTypes =
        IsWindows8OrGreater()
            ? VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT
            : VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT;
#else
    vulkanExportSemaphoreCreateInfo.pNext = NULL;
    vulkanExportSemaphoreCreateInfo.handleTypes =
        VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BIT;
#endif
    semaphoreInfo.pNext = &vulkanExportSemaphoreCreateInfo;

    if (vkCreateSemaphore(device, &semaphoreInfo, nullptr,
                          &cudaUpdateVkSemaphore) != VK_SUCCESS ||
        vkCreateSemaphore(device, &semaphoreInfo, nullptr,
                          &vkUpdateCudaSemaphore) != VK_SUCCESS) {
      throw std::runtime_error(
          "failed to create synchronization objects for a CUDA-Vulkan!");
    }
  }

  void updateUniformBuffer() {
    UniformBufferObject ubo = {};

    mat4x4_identity(ubo.model);
    mat4x4 Model;
    mat4x4_dup(Model, ubo.model);
    mat4x4_rotate(ubo.model, Model, 0.0f, 0.0f, 1.0f, degreesToRadians(135.0f));

    vec3 eye = {2.0f, 2.0f, 2.0f};
    vec3 center = {0.0f, 0.0f, 0.0f};
    vec3 up = {0.0f, 0.0f, 1.0f};
    mat4x4_look_at(ubo.view, eye, center, up);

    mat4x4_perspective(ubo.proj, degreesToRadians(45.0f),
                       swapChainExtent.width / (float)swapChainExtent.height,
                       0.1f, 10.0f);
    ubo.proj[1][1] *= -1;

    for (size_t i = 0; i < swapChainImages.size(); i++) {
      void* data;
      vkMapMemory(device, uniformBuffersMemory[i], 0, sizeof(ubo), 0, &data);
      memcpy(data, &ubo, sizeof(ubo));
      vkUnmapMemory(device, uniformBuffersMemory[i]);
    }
  }

  void drawFrame() {
    static int startSubmit = 0;

    vkWaitForFences(device, 1, &inFlightFences[currentFrame], VK_TRUE,
                    std::numeric_limits<uint64_t>::max());

    uint32_t imageIndex;
    VkResult result = vkAcquireNextImageKHR(
        device, swapChain, std::numeric_limits<uint64_t>::max(),
        imageAvailableSemaphores[currentFrame], VK_NULL_HANDLE, &imageIndex);

    if (result == VK_ERROR_OUT_OF_DATE_KHR) {
      recreateSwapChain();
      return;
    } else if (result != VK_SUCCESS && result != VK_SUBOPTIMAL_KHR) {
      throw std::runtime_error("failed to acquire swap chain image!");
    }

    vkResetFences(device, 1, &inFlightFences[currentFrame]);

    if (!startSubmit) {
      submitVulkan(imageIndex);
      startSubmit = 1;
    } else {
      submitVulkanCuda(imageIndex);
    }

    VkPresentInfoKHR presentInfo = {};
    presentInfo.sType = VK_STRUCTURE_TYPE_PRESENT_INFO_KHR;

    VkSemaphore signalSemaphores[] = {renderFinishedSemaphores[currentFrame]};

    presentInfo.waitSemaphoreCount = 1;
    presentInfo.pWaitSemaphores = signalSemaphores;

    VkSwapchainKHR swapChains[] = {swapChain};
    presentInfo.swapchainCount = 1;
    presentInfo.pSwapchains = swapChains;
    presentInfo.pImageIndices = &imageIndex;
    presentInfo.pResults = nullptr;  // Optional

    result = vkQueuePresentKHR(presentQueue, &presentInfo);

    if (result == VK_ERROR_OUT_OF_DATE_KHR || result == VK_SUBOPTIMAL_KHR ||
        framebufferResized) {
      framebufferResized = false;
      recreateSwapChain();
    } else if (result != VK_SUCCESS) {
      throw std::runtime_error("failed to present swap chain image!");
    }

    cudaUpdateVkImage();

    currentFrame = (currentFrame + 1) % MAX_FRAMES;
    // Added sleep of 10 millisecs so that CPU does not submit too much work to
    // GPU
    std::this_thread::sleep_for(std::chrono::microseconds(10000));
    char title[256];
    sprintf(title, "Vulkan Image CUDA Box Filter (radius=%d)", filter_radius);
    glfwSetWindowTitle(window, title);
  }

  void cudaVkSemaphoreSignal(hipExternalSemaphore_t& extSemaphore) {
    hipExternalSemaphoreSignalParams extSemaphoreSignalParams;
    memset(&extSemaphoreSignalParams, 0, sizeof(extSemaphoreSignalParams));

    extSemaphoreSignalParams.params.fence.value = 0;
    extSemaphoreSignalParams.flags = 0;
    checkCudaErrors(hipSignalExternalSemaphoresAsync(
        &extSemaphore, &extSemaphoreSignalParams, 1, streamToRun));
  }

  void cudaVkSemaphoreWait(hipExternalSemaphore_t& extSemaphore) {
    hipExternalSemaphoreWaitParams extSemaphoreWaitParams;

    memset(&extSemaphoreWaitParams, 0, sizeof(extSemaphoreWaitParams));

    extSemaphoreWaitParams.params.fence.value = 0;
    extSemaphoreWaitParams.flags = 0;

    checkCudaErrors(hipWaitExternalSemaphoresAsync(
        &extSemaphore, &extSemaphoreWaitParams, 1, streamToRun));
  }

  void submitVulkan(uint32_t imageIndex) {
    VkSubmitInfo submitInfo = {};
    submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;

    VkSemaphore waitSemaphores[] = {imageAvailableSemaphores[currentFrame]};
    VkPipelineStageFlags waitStages[] = {
        VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT};
    submitInfo.waitSemaphoreCount = 1;
    submitInfo.pWaitSemaphores = waitSemaphores;
    submitInfo.pWaitDstStageMask = waitStages;
    submitInfo.commandBufferCount = 1;
    submitInfo.pCommandBuffers = &commandBuffers[imageIndex];

    VkSemaphore signalSemaphores[] = {renderFinishedSemaphores[currentFrame],
                                      vkUpdateCudaSemaphore};

    submitInfo.signalSemaphoreCount = 2;
    submitInfo.pSignalSemaphores = signalSemaphores;

    if (vkQueueSubmit(graphicsQueue, 1, &submitInfo,
                      inFlightFences[currentFrame]) != VK_SUCCESS) {
      throw std::runtime_error("failed to submit draw command buffer!");
    }
  }

  void submitVulkanCuda(uint32_t imageIndex) {
    VkSubmitInfo submitInfo = {};
    submitInfo.sType = VK_STRUCTURE_TYPE_SUBMIT_INFO;

    VkSemaphore waitSemaphores[] = {imageAvailableSemaphores[currentFrame],
                                    cudaUpdateVkSemaphore};
    VkPipelineStageFlags waitStages[] = {
        VK_PIPELINE_STAGE_COLOR_ATTACHMENT_OUTPUT_BIT,
        VK_PIPELINE_STAGE_ALL_COMMANDS_BIT};
    submitInfo.waitSemaphoreCount = 2;
    submitInfo.pWaitSemaphores = waitSemaphores;
    submitInfo.pWaitDstStageMask = waitStages;
    submitInfo.commandBufferCount = 1;
    submitInfo.pCommandBuffers = &commandBuffers[imageIndex];

    VkSemaphore signalSemaphores[] = {renderFinishedSemaphores[currentFrame],
                                      vkUpdateCudaSemaphore};

    submitInfo.signalSemaphoreCount = 2;
    submitInfo.pSignalSemaphores = signalSemaphores;

    if (vkQueueSubmit(graphicsQueue, 1, &submitInfo,
                      inFlightFences[currentFrame]) != VK_SUCCESS) {
      throw std::runtime_error("failed to submit draw command buffer!");
    }
  }

  VkShaderModule createShaderModule(const std::vector<char>& code) {
    VkShaderModuleCreateInfo createInfo = {};
    createInfo.sType = VK_STRUCTURE_TYPE_SHADER_MODULE_CREATE_INFO;
    createInfo.codeSize = code.size();
    createInfo.pCode = reinterpret_cast<const uint32_t*>(code.data());

    VkShaderModule shaderModule;
    if (vkCreateShaderModule(device, &createInfo, nullptr, &shaderModule) !=
        VK_SUCCESS) {
      throw std::runtime_error("failed to create shader module!");
    }

    return shaderModule;
  }

  VkSurfaceFormatKHR chooseSwapSurfaceFormat(
      const std::vector<VkSurfaceFormatKHR>& availableFormats) {
    if (availableFormats.size() == 1 &&
        availableFormats[0].format == VK_FORMAT_UNDEFINED) {
      return {VK_FORMAT_B8G8R8A8_UNORM, VK_COLOR_SPACE_SRGB_NONLINEAR_KHR};
    }

    for (const auto& availableFormat : availableFormats) {
      if (availableFormat.format == VK_FORMAT_B8G8R8A8_UNORM &&
          availableFormat.colorSpace == VK_COLOR_SPACE_SRGB_NONLINEAR_KHR) {
        return availableFormat;
      }
    }

    return availableFormats[0];
  }

  VkPresentModeKHR chooseSwapPresentMode(
      const std::vector<VkPresentModeKHR>& availablePresentModes) {
    VkPresentModeKHR bestMode = VK_PRESENT_MODE_FIFO_KHR;

    for (const auto& availablePresentMode : availablePresentModes) {
      if (availablePresentMode == VK_PRESENT_MODE_MAILBOX_KHR) {
        return availablePresentMode;
      } else if (availablePresentMode == VK_PRESENT_MODE_IMMEDIATE_KHR) {
        bestMode = availablePresentMode;
      }
    }

    return bestMode;
  }

  VkExtent2D chooseSwapExtent(const VkSurfaceCapabilitiesKHR& capabilities) {
    if (capabilities.currentExtent.width !=
        std::numeric_limits<uint32_t>::max()) {
      return capabilities.currentExtent;
    } else {
      int width, height;
      glfwGetFramebufferSize(window, &width, &height);

      VkExtent2D actualExtent = {static_cast<uint32_t>(width),
                                 static_cast<uint32_t>(height)};

      actualExtent.width = std::max(
          capabilities.minImageExtent.width,
          std::min(capabilities.maxImageExtent.width, actualExtent.width));
      actualExtent.height = std::max(
          capabilities.minImageExtent.height,
          std::min(capabilities.maxImageExtent.height, actualExtent.height));

      return actualExtent;
    }
  }

  SwapChainSupportDetails querySwapChainSupport(VkPhysicalDevice device) {
    SwapChainSupportDetails details;

    vkGetPhysicalDeviceSurfaceCapabilitiesKHR(device, surface,
                                              &details.capabilities);

    uint32_t formatCount;
    vkGetPhysicalDeviceSurfaceFormatsKHR(device, surface, &formatCount,
                                         nullptr);

    if (formatCount != 0) {
      details.formats.resize(formatCount);
      vkGetPhysicalDeviceSurfaceFormatsKHR(device, surface, &formatCount,
                                           details.formats.data());
    }

    uint32_t presentModeCount;
    vkGetPhysicalDeviceSurfacePresentModesKHR(device, surface,
                                              &presentModeCount, nullptr);

    if (presentModeCount != 0) {
      details.presentModes.resize(presentModeCount);
      vkGetPhysicalDeviceSurfacePresentModesKHR(
          device, surface, &presentModeCount, details.presentModes.data());
    }

    return details;
  }

  bool isDeviceSuitable(VkPhysicalDevice device) {
    QueueFamilyIndices indices = findQueueFamilies(device);

    bool extensionsSupported = checkDeviceExtensionSupport(device);

    bool swapChainAdequate = false;
    if (extensionsSupported) {
      SwapChainSupportDetails swapChainSupport = querySwapChainSupport(device);
      swapChainAdequate = !swapChainSupport.formats.empty() &&
                          !swapChainSupport.presentModes.empty();
    }

    VkPhysicalDeviceFeatures supportedFeatures;
    vkGetPhysicalDeviceFeatures(device, &supportedFeatures);

    return indices.isComplete() && extensionsSupported && swapChainAdequate &&
           supportedFeatures.samplerAnisotropy;
  }

  bool checkDeviceExtensionSupport(VkPhysicalDevice device) {
    uint32_t extensionCount;
    vkEnumerateDeviceExtensionProperties(device, nullptr, &extensionCount,
                                         nullptr);

    std::vector<VkExtensionProperties> availableExtensions(extensionCount);
    vkEnumerateDeviceExtensionProperties(device, nullptr, &extensionCount,
                                         availableExtensions.data());

    std::set<std::string> requiredExtensions(deviceExtensions.begin(),
                                             deviceExtensions.end());

    for (const auto& extension : availableExtensions) {
      requiredExtensions.erase(extension.extensionName);
    }

    return requiredExtensions.empty();
  }

  QueueFamilyIndices findQueueFamilies(VkPhysicalDevice device) {
    QueueFamilyIndices indices;

    uint32_t queueFamilyCount = 0;
    vkGetPhysicalDeviceQueueFamilyProperties(device, &queueFamilyCount,
                                             nullptr);

    std::vector<VkQueueFamilyProperties> queueFamilies(queueFamilyCount);
    vkGetPhysicalDeviceQueueFamilyProperties(device, &queueFamilyCount,
                                             queueFamilies.data());

    int i = 0;
    for (const auto& queueFamily : queueFamilies) {
      if (queueFamily.queueCount > 0 &&
          queueFamily.queueFlags & VK_QUEUE_GRAPHICS_BIT) {
        indices.graphicsFamily = i;
      }

      VkBool32 presentSupport = false;
      vkGetPhysicalDeviceSurfaceSupportKHR(device, i, surface, &presentSupport);

      if (queueFamily.queueCount > 0 && presentSupport) {
        indices.presentFamily = i;
      }

      if (indices.isComplete()) {
        break;
      }

      i++;
    }

    return indices;
  }

  std::vector<const char*> getRequiredExtensions() {
    uint32_t glfwExtensionCount = 0;
    const char** glfwExtensions;
    glfwExtensions = glfwGetRequiredInstanceExtensions(&glfwExtensionCount);

    std::vector<const char*> extensions(glfwExtensions,
                                        glfwExtensions + glfwExtensionCount);

    if (enableValidationLayers) {
      extensions.push_back(VK_EXT_DEBUG_UTILS_EXTENSION_NAME);
    }

    return extensions;
  }

  bool checkValidationLayerSupport() {
    uint32_t layerCount;
    vkEnumerateInstanceLayerProperties(&layerCount, nullptr);

    std::vector<VkLayerProperties> availableLayers(layerCount);
    vkEnumerateInstanceLayerProperties(&layerCount, availableLayers.data());

    for (const char* layerName : validationLayers) {
      bool layerFound = false;

      for (const auto& layerProperties : availableLayers) {
        if (strcmp(layerName, layerProperties.layerName) == 0) {
          layerFound = true;
          break;
        }
      }

      if (!layerFound) {
        return false;
      }
    }

    return true;
  }

  static std::vector<char> readFile(const std::string& filename) {
    char* file_path = sdkFindFilePath(filename.c_str(), execution_path.c_str());
    std::ifstream file(file_path, std::ios::ate | std::ios::binary);

    if (!file.is_open()) {
      throw std::runtime_error("failed to open file!");
    }

    size_t fileSize = (size_t)file.tellg();
    std::vector<char> buffer(fileSize);

    file.seekg(0);
    file.read(buffer.data(), fileSize);

    file.close();

    return buffer;
  }

  static VKAPI_ATTR VkBool32 VKAPI_CALL
  debugCallback(VkDebugUtilsMessageSeverityFlagBitsEXT messageSeverity,
                VkDebugUtilsMessageTypeFlagsEXT messageType,
                const VkDebugUtilsMessengerCallbackDataEXT* pCallbackData,
                void* pUserData) {
    std::cerr << "validation layer: " << pCallbackData->pMessage << std::endl;

    return VK_FALSE;
  }
};

int main(int argc, char** argv) {
  execution_path = argv[0];
  std::string image_filename = "teapot1024.ppm";

  if (checkCmdLineFlag(argc, (const char**)argv, "file")) {
    getCmdLineArgumentString(argc, (const char**)argv, "file",
                             (char**)&image_filename);
  }

  vulkanImageCUDA app;

  try {
    // This app only works on ppm images
    app.loadImageData(image_filename);
    app.run();
  } catch (const std::exception& e) {
    std::cerr << e.what() << std::endl;
    return EXIT_FAILURE;
  }

  return EXIT_SUCCESS;
}
