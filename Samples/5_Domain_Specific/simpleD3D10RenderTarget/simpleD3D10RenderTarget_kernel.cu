#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* This example demonstrates how to use the CUDA Direct3D bindings with the
 * runtime API.
 * Device code.
 */

#ifndef SIMPLED3D10RENDERTARGET_KERNEL_CU
#define SIMPLED3D10RENDERTARGET_KERNEL_CU

// includes, C string library
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// includes, cuda
#include <hip/hip_runtime.h>
#include <builtin_types.h>
#include <hip/hip_runtime_api.h>

// includes, project
#include <hip/hip_runtime_api.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h
//#include "checkCudaErrors"

#define BIN_COUNT 256
#define HISTOGRAM_SIZE (BIN_COUNT * sizeof(unsigned int))

texture<uchar4, 2, hipReadModeElementType> colorTex;

////////////////////////////////////////////////////////////////////////////////
// GPU-specific definitions
////////////////////////////////////////////////////////////////////////////////
// Fast mul on G8x / G9x / G100
#define IMUL(a, b) __mul24(a, b)

// Machine warp size
// G80's warp size is 32 threads
#define WARP_LOG2SIZE 5

// Warps in thread block for histogram256Kernel()
#define WARP_N 6

// Corresponding thread block size in threads for histogram256Kernel()
#define THREAD_N (WARP_N << WARP_LOG2SIZE)

// Total histogram size (in counters) per thread block for histogram256Kernel()
#define BLOCK_MEMORY (WARP_N * BIN_COUNT)

// Thread block count for histogram256Kernel()
#define BLOCK_N 64

////////////////////////////////////////////////////////////////////////////////
// If threadPos == threadIdx.x, there are always  4-way bank conflicts,
// since each group of 16 threads (half-warp) accesses different bytes,
// but only within 4 shared memory banks. Having shuffled bits of threadIdx.x
// as in histogram64GPU(), each half-warp accesses different shared memory banks
// avoiding any bank conflicts at all.
// Refer to the supplied whitepaper for detailed explanations.
////////////////////////////////////////////////////////////////////////////////
__device__ inline void addData256(volatile unsigned int *s_WarpHist,
                                  unsigned int data, unsigned int threadTag) {
  unsigned int count;

  do {
    count = s_WarpHist[data] & 0x07FFFFFFU;
    count = threadTag | (count + 1);
    s_WarpHist[data] = count;
  } while (s_WarpHist[data] != count);
}

////////////////////////////////////////////////////////////////////////////////
// Main histogram calculation kernel
////////////////////////////////////////////////////////////////////////////////
static __global__ void histogramTex256Kernel(unsigned int *d_Result,
                                             unsigned int width,
                                             unsigned int height, int dataN) {
  // Current global thread index
  const int globalTid = IMUL(blockIdx.x, blockDim.x) + threadIdx.x;
  // Total number of threads in the compute grid
  const int numThreads = IMUL(blockDim.x, gridDim.x);

  // Thread tag for addData256()
  // WARP_LOG2SIZE higher bits of counter values are tagged
  // by lower WARP_LOG2SIZE threadID bits
  const unsigned int threadTag = threadIdx.x << (32 - WARP_LOG2SIZE);

  // Shared memory storage for each warp
  volatile __shared__ unsigned int s_Hist[BLOCK_MEMORY];

  // Current warp shared memory base
  const int warpBase = (threadIdx.x >> WARP_LOG2SIZE) * BIN_COUNT;

  // Clear shared memory buffer for current thread block before processing
  for (int pos = threadIdx.x; pos < BLOCK_MEMORY; pos += blockDim.x)
    s_Hist[pos] = 0;

  // Cycle through the entire data set, update subhistograms for each warp
  __syncthreads();

  for (int pos = globalTid; pos < dataN; pos += numThreads) {
    // NOTE: check this... Not sure this is what needs to be done
    int py = pos / width;
    int px = pos - (py * width);
    uchar4 data4 = tex2D(colorTex, px, py);

    addData256(s_Hist + warpBase, (data4.x), threadTag);
    addData256(s_Hist + warpBase, (data4.y), threadTag);
    addData256(s_Hist + warpBase, (data4.z), threadTag);
    addData256(s_Hist + warpBase, (data4.w), threadTag);
  }

  __syncthreads();

  // Merge per-warp histograms into per-block and write to global memory
  for (int pos = threadIdx.x; pos < BIN_COUNT; pos += blockDim.x) {
    unsigned int sum = 0;

    for (int base = 0; base < BLOCK_MEMORY; base += BIN_COUNT)
      sum += s_Hist[base + pos] & 0x07FFFFFFU;

    d_Result[blockIdx.x * BIN_COUNT + pos] = sum;
  }
}

///////////////////////////////////////////////////////////////////////////////
// Merge BLOCK_N subhistograms of BIN_COUNT bins into final histogram
///////////////////////////////////////////////////////////////////////////////
// gridDim.x   == BIN_COUNT
// blockDim.x  == BLOCK_N
// blockIdx.x  == bin counter processed by current block
// threadIdx.x == subhistogram index
static __global__ void mergeHistogramTex256Kernel(unsigned int *d_Result) {
  __shared__ unsigned int data[BLOCK_N];

  // Reads are uncoalesced, but this final stage takes
  // only a fraction of total processing time
  data[threadIdx.x] = d_Result[threadIdx.x * BIN_COUNT + blockIdx.x];

  for (int stride = BLOCK_N / 2; stride > 0; stride >>= 1) {
    __syncthreads();

    if (threadIdx.x < stride) data[threadIdx.x] += data[threadIdx.x + stride];
  }

  if (threadIdx.x == 0) d_Result[blockIdx.x] = data[0];
}

////////////////////////////////////////////////////////////////////////////////
// Host interface to GPU histogram
////////////////////////////////////////////////////////////////////////////////

extern "C" void checkCudaError() {
  hipError_t err = hipGetLastError();

  if (hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s.\n", hipGetErrorString(err));
    exit(2);
  }
}

// Maximum block count for histogram64kernel()
// Limits input data size to 756MB
// const int MAX_BLOCK_N = 16384;

// Internal memory allocation
// const int BLOCK_N2 = 32;

extern "C" void createHistogramTex(unsigned int *h_Result, unsigned int width,
                                   unsigned int height, hipArray *colorArray) {
  hipBindTextureToArray(colorTex, colorArray);
  checkCudaError();

  histogramTex256Kernel<<<BLOCK_N, THREAD_N>>>(h_Result, width, height,
                                               width * height / 4);
  checkCudaError();

  mergeHistogramTex256Kernel<<<BIN_COUNT, BLOCK_N>>>(h_Result);
  checkCudaError();

  hipUnbindTexture(colorTex);
  checkCudaError();

#if 0
    // Dummy fill test
    unsigned int toto[256];

    for (int i=0; i<256; i++)
    {
        toto[i] = i * 100;
    }
    hipMemcpy(h_Result, toto, HISTOGRAM_SIZE, hipMemcpyHostToDevice);
#endif
  checkCudaError();
}

extern "C" void bindArrayToTexture(hipArray *pArray) {}

#endif  // #ifndef SIMPLED3D10RENDERTARGET_KERNEL_CU
