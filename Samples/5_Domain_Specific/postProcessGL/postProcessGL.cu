#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Utilities and system includes

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <hip/hip_runtime_api.h>

hipTextureObject_t inTexObject;

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b) { return max(a, min(b, x)); }

__device__ int clamp(int x, int a, int b) { return max(a, min(b, x)); }

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b) {
  r = clamp(r, 0.0f, 255.0f);
  g = clamp(g, 0.0f, 255.0f);
  b = clamp(b, 0.0f, 255.0f);
  return (int(b) << 16) | (int(g) << 8) | int(r);
}

// get pixel from 2D image, with clamping to border
__device__ uchar4 getPixel(int x, int y, hipTextureObject_t inTex) {
#ifndef USE_TEXTURE_RGBA8UI
  float4 res = tex2D<float4>(inTex, x, y);
  uchar4 ucres = make_uchar4(res.x * 255.0f, res.y * 255.0f, res.z * 255.0f,
                             res.w * 255.0f);
#else
  uchar4 ucres = tex2D<uchar4>(inTex, x, y);
#endif
  return ucres;
}

// macros to make indexing shared memory easier
#define SMEM(X, Y) sdata[(Y)*tilew + (X)]

/*
    2D convolution using shared memory
    - operates on 8-bit RGB data stored in 32-bit int
    - assumes kernel radius is less than or equal to block size
    - not optimized for performance
     _____________
    |   :     :   |
    |_ _:_____:_ _|
    |   |     |   |
    |   |     |   |
    |_ _|_____|_ _|
  r |   :     :   |
    |___:_____:___|
      r    bw   r
    <----tilew---->
*/

__global__ void cudaProcess(unsigned int *g_odata, int imgw, int imgh,
                            int tilew, int r, float threshold, float highlight,
                            hipTextureObject_t inTex) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  extern __shared__ uchar4 sdata[];

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int bw = blockDim.x;
  int bh = blockDim.y;
  int x = blockIdx.x * bw + tx;
  int y = blockIdx.y * bh + ty;

#if 0
    uchar4 c4 = getPixel(x, y);
    g_odata[y*imgw+x] = rgbToInt(c4.z, c4.y, c4.x);
#else
  // copy tile to shared memory
  // center region
  SMEM(r + tx, r + ty) = getPixel(x, y, inTex);

  // borders
  if (threadIdx.x < r) {
    // left
    SMEM(tx, r + ty) = getPixel(x - r, y, inTex);
    // right
    SMEM(r + bw + tx, r + ty) = getPixel(x + bw, y, inTex);
  }

  if (threadIdx.y < r) {
    // top
    SMEM(r + tx, ty) = getPixel(x, y - r, inTex);
    // bottom
    SMEM(r + tx, r + bh + ty) = getPixel(x, y + bh, inTex);
  }

  // load corners
  if ((threadIdx.x < r) && (threadIdx.y < r)) {
    // tl
    SMEM(tx, ty) = getPixel(x - r, y - r, inTex);
    // bl
    SMEM(tx, r + bh + ty) = getPixel(x - r, y + bh, inTex);
    // tr
    SMEM(r + bw + tx, ty) = getPixel(x + bh, y - r, inTex);
    // br
    SMEM(r + bw + tx, r + bh + ty) = getPixel(x + bw, y + bh, inTex);
  }

  // wait for loads to complete
  cg::sync(cta);

  // perform convolution
  float rsum = 0.0f;
  float gsum = 0.0f;
  float bsum = 0.0f;
  float samples = 0.0f;

  for (int dy = -r; dy <= r; dy++) {
    for (int dx = -r; dx <= r; dx++) {
#if 0
            // try this to see the benefit of using shared memory
            uchar4 pixel = getPixel(x+dx, y+dy);
#else
      uchar4 pixel = SMEM(r + tx + dx, r + ty + dy);
#endif

      // only sum pixels within disc-shaped kernel
      float l = dx * dx + dy * dy;

      if (l <= r * r) {
        float r = float(pixel.x);
        float g = float(pixel.y);
        float b = float(pixel.z);
#if 1
        // brighten highlights
        float lum = (r + g + b) / (255 * 3);

        if (lum > threshold) {
          r *= highlight;
          g *= highlight;
          b *= highlight;
        }

#endif
        rsum += r;
        gsum += g;
        bsum += b;
        samples += 1.0f;
      }
    }
  }

  rsum /= samples;
  gsum /= samples;
  bsum /= samples;
  // ABGR
  g_odata[y * imgw + x] = rgbToInt(rsum, gsum, bsum);
// g_odata[y*imgw+x] = rgbToInt(x,y,0);
#endif
}

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                                   hipArray *g_data_array,
                                   unsigned int *g_odata, int imgw, int imgh,
                                   int tilew, int radius, float threshold,
                                   float highlight) {
  struct hipChannelFormatDesc desc;
  checkCudaErrors(hipGetChannelDesc(&desc, g_data_array));

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = g_data_array;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&inTexObject, &texRes, &texDescr, NULL));

#if 0
    printf("CUDA Array channel descriptor, bits per component:\n");
    printf("X %d Y %d Z %d W %d, kind %d\n",
           desc.x,desc.y,desc.z,desc.w,desc.f);

    printf("Possible values for channel format kind: i %d, u%d, f%d:\n",
           hipChannelFormatKindSigned, hipChannelFormatKindUnsigned,
           hipChannelFormatKindFloat);
#endif

// printf("\n");
#ifdef GPU_PROFILING
  StopWatchInterface *timer = 0;
  sdkCreateTimer(&timer);

  int nIter = 30;

  for (int i = -1; i < nIter; ++i) {
    if (i == 0) {
      sdkStartTimer(&timer);
    }

#endif

    cudaProcess<<<grid, block, sbytes>>>(g_odata, imgw, imgh,
                                         block.x + (2 * radius), radius, 0.8f,
                                         4.0f, inTexObject);

#ifdef GPU_PROFILING
  }

  hipDeviceSynchronize();
  sdkStopTimer(&timer);
  double dSeconds = sdkGetTimerValue(&timer) / ((double)nIter * 1000.0);
  double dNumTexels = (double)imgw * (double)imgh;
  double mtexps = 1.0e-6 * dNumTexels / dSeconds;

  if (radius == 4) {
    printf("\n");
    printf(
        "postprocessGL, Throughput = %.4f MTexels/s, Time = %.5f s, Size = "
        "%.0f Texels, NumDevsUsed = %d, Workgroup = %u\n",
        mtexps, dSeconds, dNumTexels, 1, block.x * block.y);
  }

#endif
}
