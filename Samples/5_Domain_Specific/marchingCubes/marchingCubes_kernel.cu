#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _MARCHING_CUBES_KERNEL_CU_
#define _MARCHING_CUBES_KERNEL_CU_

#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime_api.h>  // includes for helper CUDA functions
#include <hip/hip_vector_types.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>

#include "defines.h"
#include "tables.h"

// textures containing look-up tables
hipTextureObject_t triTex;
hipTextureObject_t numVertsTex;

// volume data
hipTextureObject_t volumeTex;

extern "C" void allocateTextures(uint **d_edgeTable, uint **d_triTable,
                                 uint **d_numVertsTable) {
  checkCudaErrors(hipMalloc((void **)d_edgeTable, 256 * sizeof(uint)));
  checkCudaErrors(hipMemcpy((void *)*d_edgeTable, (void *)edgeTable,
                             256 * sizeof(uint), hipMemcpyHostToDevice));
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

  checkCudaErrors(hipMalloc((void **)d_triTable, 256 * 16 * sizeof(uint)));
  checkCudaErrors(hipMemcpy((void *)*d_triTable, (void *)triTable,
                             256 * 16 * sizeof(uint), hipMemcpyHostToDevice));

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = *d_triTable;
  texRes.res.linear.sizeInBytes = 256 * 16 * sizeof(uint);
  texRes.res.linear.desc = channelDesc;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&triTex, &texRes, &texDescr, NULL));

  checkCudaErrors(hipMalloc((void **)d_numVertsTable, 256 * sizeof(uint)));
  checkCudaErrors(hipMemcpy((void *)*d_numVertsTable, (void *)numVertsTable,
                             256 * sizeof(uint), hipMemcpyHostToDevice));

  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = *d_numVertsTable;
  texRes.res.linear.sizeInBytes = 256 * sizeof(uint);
  texRes.res.linear.desc = channelDesc;

  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&numVertsTex, &texRes, &texDescr, NULL));
}

extern "C" void createVolumeTexture(uchar *d_volume, size_t buffSize) {
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeLinear;
  texRes.res.linear.devPtr = d_volume;
  texRes.res.linear.sizeInBytes = buffSize;
  texRes.res.linear.desc =
      hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(
      hipCreateTextureObject(&volumeTex, &texRes, &texDescr, NULL));
}

extern "C" void destroyAllTextureObjects() {
  checkCudaErrors(hipDestroyTextureObject(triTex));
  checkCudaErrors(hipDestroyTextureObject(numVertsTex));
  checkCudaErrors(hipDestroyTextureObject(volumeTex));
}

// an interesting field function
__device__ float tangle(float x, float y, float z) {
  x *= 3.0f;
  y *= 3.0f;
  z *= 3.0f;
  return (x * x * x * x - 5.0f * x * x + y * y * y * y - 5.0f * y * y +
          z * z * z * z - 5.0f * z * z + 11.8f) * 0.2f + 0.5f;
}

// evaluate field function at point
__device__ float fieldFunc(float3 p) { return tangle(p.x, p.y, p.z); }

// evaluate field function at a point
// returns value and gradient in float4
__device__ float4 fieldFunc4(float3 p) {
  float v = tangle(p.x, p.y, p.z);
  const float d = 0.001f;
  float dx = tangle(p.x + d, p.y, p.z) - v;
  float dy = tangle(p.x, p.y + d, p.z) - v;
  float dz = tangle(p.x, p.y, p.z + d) - v;
  return make_float4(dx, dy, dz, v);
}

// sample volume data set at a point
__device__ float sampleVolume(hipTextureObject_t volumeTex, uchar *data,
                              uint3 p, uint3 gridSize) {
  p.x = min(p.x, gridSize.x - 1);
  p.y = min(p.y, gridSize.y - 1);
  p.z = min(p.z, gridSize.z - 1);
  uint i = (p.z * gridSize.x * gridSize.y) + (p.y * gridSize.x) + p.x;
  //    return (float) data[i] / 255.0f;
  return tex1Dfetch<float>(volumeTex, i);
}

// compute position in 3d grid from 1d index
// only works for power of 2 sizes
__device__ uint3 calcGridPos(uint i, uint3 gridSizeShift, uint3 gridSizeMask) {
  uint3 gridPos;
  gridPos.x = i & gridSizeMask.x;
  gridPos.y = (i >> gridSizeShift.y) & gridSizeMask.y;
  gridPos.z = (i >> gridSizeShift.z) & gridSizeMask.z;
  return gridPos;
}

// classify voxel based on number of vertices it will generate
// one thread per voxel
__global__ void classifyVoxel(uint *voxelVerts, uint *voxelOccupied,
                              uchar *volume, uint3 gridSize,
                              uint3 gridSizeShift, uint3 gridSizeMask,
                              uint numVoxels, float3 voxelSize, float isoValue,
                              hipTextureObject_t numVertsTex,
                              hipTextureObject_t volumeTex) {
  uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
  uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

  uint3 gridPos = calcGridPos(i, gridSizeShift, gridSizeMask);

// read field values at neighbouring grid vertices
#if SAMPLE_VOLUME
  float field[8];
  field[0] = sampleVolume(volumeTex, volume, gridPos, gridSize);
  field[1] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(1, 0, 0), gridSize);
  field[2] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(1, 1, 0), gridSize);
  field[3] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(0, 1, 0), gridSize);
  field[4] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(0, 0, 1), gridSize);
  field[5] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(1, 0, 1), gridSize);
  field[6] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(1, 1, 1), gridSize);
  field[7] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(0, 1, 1), gridSize);
#else
  float3 p;
  p.x = -1.0f + (gridPos.x * voxelSize.x);
  p.y = -1.0f + (gridPos.y * voxelSize.y);
  p.z = -1.0f + (gridPos.z * voxelSize.z);

  float field[8];
  field[0] = fieldFunc(p);
  field[1] = fieldFunc(p + make_float3(voxelSize.x, 0, 0));
  field[2] = fieldFunc(p + make_float3(voxelSize.x, voxelSize.y, 0));
  field[3] = fieldFunc(p + make_float3(0, voxelSize.y, 0));
  field[4] = fieldFunc(p + make_float3(0, 0, voxelSize.z));
  field[5] = fieldFunc(p + make_float3(voxelSize.x, 0, voxelSize.z));
  field[6] = fieldFunc(p + make_float3(voxelSize.x, voxelSize.y, voxelSize.z));
  field[7] = fieldFunc(p + make_float3(0, voxelSize.y, voxelSize.z));
#endif

  // calculate flag indicating if each vertex is inside or outside isosurface
  uint cubeindex;
  cubeindex = uint(field[0] < isoValue);
  cubeindex += uint(field[1] < isoValue) * 2;
  cubeindex += uint(field[2] < isoValue) * 4;
  cubeindex += uint(field[3] < isoValue) * 8;
  cubeindex += uint(field[4] < isoValue) * 16;
  cubeindex += uint(field[5] < isoValue) * 32;
  cubeindex += uint(field[6] < isoValue) * 64;
  cubeindex += uint(field[7] < isoValue) * 128;

  // read number of vertices from texture
  uint numVerts = tex1Dfetch<uint>(numVertsTex, cubeindex);

  if (i < numVoxels) {
    voxelVerts[i] = numVerts;
    voxelOccupied[i] = (numVerts > 0);
  }
}

extern "C" void launch_classifyVoxel(dim3 grid, dim3 threads, uint *voxelVerts,
                                     uint *voxelOccupied, uchar *volume,
                                     uint3 gridSize, uint3 gridSizeShift,
                                     uint3 gridSizeMask, uint numVoxels,
                                     float3 voxelSize, float isoValue) {
  // calculate number of vertices need per voxel
  classifyVoxel<<<grid, threads>>>(voxelVerts, voxelOccupied, volume, gridSize,
                                   gridSizeShift, gridSizeMask, numVoxels,
                                   voxelSize, isoValue, numVertsTex, volumeTex);
  getLastCudaError("classifyVoxel failed");
}

// compact voxel array
__global__ void compactVoxels(uint *compactedVoxelArray, uint *voxelOccupied,
                              uint *voxelOccupiedScan, uint numVoxels) {
  uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
  uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

  if (voxelOccupied[i] && (i < numVoxels)) {
    compactedVoxelArray[voxelOccupiedScan[i]] = i;
  }
}

extern "C" void launch_compactVoxels(dim3 grid, dim3 threads,
                                     uint *compactedVoxelArray,
                                     uint *voxelOccupied,
                                     uint *voxelOccupiedScan, uint numVoxels) {
  compactVoxels<<<grid, threads>>>(compactedVoxelArray, voxelOccupied,
                                   voxelOccupiedScan, numVoxels);
  getLastCudaError("compactVoxels failed");
}

// compute interpolated vertex along an edge
__device__ float3 vertexInterp(float isolevel, float3 p0, float3 p1, float f0,
                               float f1) {
  float t = (isolevel - f0) / (f1 - f0);
  return lerp(p0, p1, t);
}

// compute interpolated vertex position and normal along an edge
__device__ void vertexInterp2(float isolevel, float3 p0, float3 p1, float4 f0,
                              float4 f1, float3 &p, float3 &n) {
  float t = (isolevel - f0.w) / (f1.w - f0.w);
  p = lerp(p0, p1, t);
  n.x = lerp(f0.x, f1.x, t);
  n.y = lerp(f0.y, f1.y, t);
  n.z = lerp(f0.z, f1.z, t);
  //    n = normalize(n);
}

// generate triangles for each voxel using marching cubes
// interpolates normals from field function
__global__ void generateTriangles(
    float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned,
    uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask, float3 voxelSize,
    float isoValue, uint activeVoxels, uint maxVerts,
    hipTextureObject_t triTex, hipTextureObject_t numVertsTex) {
  uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
  uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

  if (i > activeVoxels - 1) {
    // can't return here because of syncthreads()
    i = activeVoxels - 1;
  }

#if SKIP_EMPTY_VOXELS
  uint voxel = compactedVoxelArray[i];
#else
  uint voxel = i;
#endif

  // compute position in 3d grid
  uint3 gridPos = calcGridPos(voxel, gridSizeShift, gridSizeMask);

  float3 p;
  p.x = -1.0f + (gridPos.x * voxelSize.x);
  p.y = -1.0f + (gridPos.y * voxelSize.y);
  p.z = -1.0f + (gridPos.z * voxelSize.z);

  // calculate cell vertex positions
  float3 v[8];
  v[0] = p;
  v[1] = p + make_float3(voxelSize.x, 0, 0);
  v[2] = p + make_float3(voxelSize.x, voxelSize.y, 0);
  v[3] = p + make_float3(0, voxelSize.y, 0);
  v[4] = p + make_float3(0, 0, voxelSize.z);
  v[5] = p + make_float3(voxelSize.x, 0, voxelSize.z);
  v[6] = p + make_float3(voxelSize.x, voxelSize.y, voxelSize.z);
  v[7] = p + make_float3(0, voxelSize.y, voxelSize.z);

  // evaluate field values
  float4 field[8];
  field[0] = fieldFunc4(v[0]);
  field[1] = fieldFunc4(v[1]);
  field[2] = fieldFunc4(v[2]);
  field[3] = fieldFunc4(v[3]);
  field[4] = fieldFunc4(v[4]);
  field[5] = fieldFunc4(v[5]);
  field[6] = fieldFunc4(v[6]);
  field[7] = fieldFunc4(v[7]);

  // recalculate flag
  // (this is faster than storing it in global memory)
  uint cubeindex;
  cubeindex = uint(field[0].w < isoValue);
  cubeindex += uint(field[1].w < isoValue) * 2;
  cubeindex += uint(field[2].w < isoValue) * 4;
  cubeindex += uint(field[3].w < isoValue) * 8;
  cubeindex += uint(field[4].w < isoValue) * 16;
  cubeindex += uint(field[5].w < isoValue) * 32;
  cubeindex += uint(field[6].w < isoValue) * 64;
  cubeindex += uint(field[7].w < isoValue) * 128;

// find the vertices where the surface intersects the cube

#if USE_SHARED
  // use partioned shared memory to avoid using local memory
  __shared__ float3 vertlist[12 * NTHREADS];
  __shared__ float3 normlist[12 * NTHREADS];

  vertexInterp2(isoValue, v[0], v[1], field[0], field[1], vertlist[threadIdx.x],
                normlist[threadIdx.x]);
  vertexInterp2(isoValue, v[1], v[2], field[1], field[2],
                vertlist[threadIdx.x + NTHREADS],
                normlist[threadIdx.x + NTHREADS]);
  vertexInterp2(isoValue, v[2], v[3], field[2], field[3],
                vertlist[threadIdx.x + (NTHREADS * 2)],
                normlist[threadIdx.x + (NTHREADS * 2)]);
  vertexInterp2(isoValue, v[3], v[0], field[3], field[0],
                vertlist[threadIdx.x + (NTHREADS * 3)],
                normlist[threadIdx.x + (NTHREADS * 3)]);
  vertexInterp2(isoValue, v[4], v[5], field[4], field[5],
                vertlist[threadIdx.x + (NTHREADS * 4)],
                normlist[threadIdx.x + (NTHREADS * 4)]);
  vertexInterp2(isoValue, v[5], v[6], field[5], field[6],
                vertlist[threadIdx.x + (NTHREADS * 5)],
                normlist[threadIdx.x + (NTHREADS * 5)]);
  vertexInterp2(isoValue, v[6], v[7], field[6], field[7],
                vertlist[threadIdx.x + (NTHREADS * 6)],
                normlist[threadIdx.x + (NTHREADS * 6)]);
  vertexInterp2(isoValue, v[7], v[4], field[7], field[4],
                vertlist[threadIdx.x + (NTHREADS * 7)],
                normlist[threadIdx.x + (NTHREADS * 7)]);
  vertexInterp2(isoValue, v[0], v[4], field[0], field[4],
                vertlist[threadIdx.x + (NTHREADS * 8)],
                normlist[threadIdx.x + (NTHREADS * 8)]);
  vertexInterp2(isoValue, v[1], v[5], field[1], field[5],
                vertlist[threadIdx.x + (NTHREADS * 9)],
                normlist[threadIdx.x + (NTHREADS * 9)]);
  vertexInterp2(isoValue, v[2], v[6], field[2], field[6],
                vertlist[threadIdx.x + (NTHREADS * 10)],
                normlist[threadIdx.x + (NTHREADS * 10)]);
  vertexInterp2(isoValue, v[3], v[7], field[3], field[7],
                vertlist[threadIdx.x + (NTHREADS * 11)],
                normlist[threadIdx.x + (NTHREADS * 11)]);
  __syncthreads();

#else
  float3 vertlist[12];
  float3 normlist[12];

  vertexInterp2(isoValue, v[0], v[1], field[0], field[1], vertlist[0],
                normlist[0]);
  vertexInterp2(isoValue, v[1], v[2], field[1], field[2], vertlist[1],
                normlist[1]);
  vertexInterp2(isoValue, v[2], v[3], field[2], field[3], vertlist[2],
                normlist[2]);
  vertexInterp2(isoValue, v[3], v[0], field[3], field[0], vertlist[3],
                normlist[3]);

  vertexInterp2(isoValue, v[4], v[5], field[4], field[5], vertlist[4],
                normlist[4]);
  vertexInterp2(isoValue, v[5], v[6], field[5], field[6], vertlist[5],
                normlist[5]);
  vertexInterp2(isoValue, v[6], v[7], field[6], field[7], vertlist[6],
                normlist[6]);
  vertexInterp2(isoValue, v[7], v[4], field[7], field[4], vertlist[7],
                normlist[7]);

  vertexInterp2(isoValue, v[0], v[4], field[0], field[4], vertlist[8],
                normlist[8]);
  vertexInterp2(isoValue, v[1], v[5], field[1], field[5], vertlist[9],
                normlist[9]);
  vertexInterp2(isoValue, v[2], v[6], field[2], field[6], vertlist[10],
                normlist[10]);
  vertexInterp2(isoValue, v[3], v[7], field[3], field[7], vertlist[11],
                normlist[11]);
#endif

  // output triangle vertices
  uint numVerts = tex1Dfetch<uint>(numVertsTex, cubeindex);

  for (int i = 0; i < numVerts; i++) {
    uint edge = tex1Dfetch<uint>(triTex, cubeindex * 16 + i);

    uint index = numVertsScanned[voxel] + i;

    if (index < maxVerts) {
#if USE_SHARED
      pos[index] = make_float4(vertlist[(edge * NTHREADS) + threadIdx.x], 1.0f);
      norm[index] =
          make_float4(normlist[(edge * NTHREADS) + threadIdx.x], 0.0f);
#else
      pos[index] = make_float4(vertlist[edge], 1.0f);
      norm[index] = make_float4(normlist[edge], 0.0f);
#endif
    }
  }
}

extern "C" void launch_generateTriangles(
    dim3 grid, dim3 threads, float4 *pos, float4 *norm,
    uint *compactedVoxelArray, uint *numVertsScanned, uint3 gridSize,
    uint3 gridSizeShift, uint3 gridSizeMask, float3 voxelSize, float isoValue,
    uint activeVoxels, uint maxVerts) {
  generateTriangles<<<grid, NTHREADS>>>(
      pos, norm, compactedVoxelArray, numVertsScanned, gridSize, gridSizeShift,
      gridSizeMask, voxelSize, isoValue, activeVoxels, maxVerts, triTex,
      numVertsTex);
  getLastCudaError("generateTriangles failed");
}

// calculate triangle normal
__device__ float3 calcNormal(float3 *v0, float3 *v1, float3 *v2) {
  float3 edge0 = *v1 - *v0;
  float3 edge1 = *v2 - *v0;
  // note - it's faster to perform normalization in vertex shader rather than
  // here
  return cross(edge0, edge1);
}

// version that calculates flat surface normal for each triangle
__global__ void generateTriangles2(
    float4 *pos, float4 *norm, uint *compactedVoxelArray, uint *numVertsScanned,
    uchar *volume, uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask,
    float3 voxelSize, float isoValue, uint activeVoxels, uint maxVerts,
    hipTextureObject_t triTex, hipTextureObject_t numVertsTex,
    hipTextureObject_t volumeTex) {
  uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
  uint i = __mul24(blockId, blockDim.x) + threadIdx.x;

  if (i > activeVoxels - 1) {
    i = activeVoxels - 1;
  }

#if SKIP_EMPTY_VOXELS
  uint voxel = compactedVoxelArray[i];
#else
  uint voxel = i;
#endif

  // compute position in 3d grid
  uint3 gridPos = calcGridPos(voxel, gridSizeShift, gridSizeMask);

  float3 p;
  p.x = -1.0f + (gridPos.x * voxelSize.x);
  p.y = -1.0f + (gridPos.y * voxelSize.y);
  p.z = -1.0f + (gridPos.z * voxelSize.z);

  // calculate cell vertex positions
  float3 v[8];
  v[0] = p;
  v[1] = p + make_float3(voxelSize.x, 0, 0);
  v[2] = p + make_float3(voxelSize.x, voxelSize.y, 0);
  v[3] = p + make_float3(0, voxelSize.y, 0);
  v[4] = p + make_float3(0, 0, voxelSize.z);
  v[5] = p + make_float3(voxelSize.x, 0, voxelSize.z);
  v[6] = p + make_float3(voxelSize.x, voxelSize.y, voxelSize.z);
  v[7] = p + make_float3(0, voxelSize.y, voxelSize.z);

#if SAMPLE_VOLUME
  float field[8];
  field[0] = sampleVolume(volumeTex, volume, gridPos, gridSize);
  field[1] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(1, 0, 0), gridSize);
  field[2] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(1, 1, 0), gridSize);
  field[3] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(0, 1, 0), gridSize);
  field[4] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(0, 0, 1), gridSize);
  field[5] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(1, 0, 1), gridSize);
  field[6] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(1, 1, 1), gridSize);
  field[7] =
      sampleVolume(volumeTex, volume, gridPos + make_uint3(0, 1, 1), gridSize);
#else
  // evaluate field values
  float field[8];
  field[0] = fieldFunc(v[0]);
  field[1] = fieldFunc(v[1]);
  field[2] = fieldFunc(v[2]);
  field[3] = fieldFunc(v[3]);
  field[4] = fieldFunc(v[4]);
  field[5] = fieldFunc(v[5]);
  field[6] = fieldFunc(v[6]);
  field[7] = fieldFunc(v[7]);
#endif

  // recalculate flag
  uint cubeindex;
  cubeindex = uint(field[0] < isoValue);
  cubeindex += uint(field[1] < isoValue) * 2;
  cubeindex += uint(field[2] < isoValue) * 4;
  cubeindex += uint(field[3] < isoValue) * 8;
  cubeindex += uint(field[4] < isoValue) * 16;
  cubeindex += uint(field[5] < isoValue) * 32;
  cubeindex += uint(field[6] < isoValue) * 64;
  cubeindex += uint(field[7] < isoValue) * 128;

// find the vertices where the surface intersects the cube

#if USE_SHARED
  // use shared memory to avoid using local
  __shared__ float3 vertlist[12 * NTHREADS];

  vertlist[threadIdx.x] =
      vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
  vertlist[NTHREADS + threadIdx.x] =
      vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
  vertlist[(NTHREADS * 2) + threadIdx.x] =
      vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
  vertlist[(NTHREADS * 3) + threadIdx.x] =
      vertexInterp(isoValue, v[3], v[0], field[3], field[0]);
  vertlist[(NTHREADS * 4) + threadIdx.x] =
      vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
  vertlist[(NTHREADS * 5) + threadIdx.x] =
      vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
  vertlist[(NTHREADS * 6) + threadIdx.x] =
      vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
  vertlist[(NTHREADS * 7) + threadIdx.x] =
      vertexInterp(isoValue, v[7], v[4], field[7], field[4]);
  vertlist[(NTHREADS * 8) + threadIdx.x] =
      vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
  vertlist[(NTHREADS * 9) + threadIdx.x] =
      vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
  vertlist[(NTHREADS * 10) + threadIdx.x] =
      vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
  vertlist[(NTHREADS * 11) + threadIdx.x] =
      vertexInterp(isoValue, v[3], v[7], field[3], field[7]);
  __syncthreads();
#else

  float3 vertlist[12];

  vertlist[0] = vertexInterp(isoValue, v[0], v[1], field[0], field[1]);
  vertlist[1] = vertexInterp(isoValue, v[1], v[2], field[1], field[2]);
  vertlist[2] = vertexInterp(isoValue, v[2], v[3], field[2], field[3]);
  vertlist[3] = vertexInterp(isoValue, v[3], v[0], field[3], field[0]);

  vertlist[4] = vertexInterp(isoValue, v[4], v[5], field[4], field[5]);
  vertlist[5] = vertexInterp(isoValue, v[5], v[6], field[5], field[6]);
  vertlist[6] = vertexInterp(isoValue, v[6], v[7], field[6], field[7]);
  vertlist[7] = vertexInterp(isoValue, v[7], v[4], field[7], field[4]);

  vertlist[8] = vertexInterp(isoValue, v[0], v[4], field[0], field[4]);
  vertlist[9] = vertexInterp(isoValue, v[1], v[5], field[1], field[5]);
  vertlist[10] = vertexInterp(isoValue, v[2], v[6], field[2], field[6]);
  vertlist[11] = vertexInterp(isoValue, v[3], v[7], field[3], field[7]);
#endif

  // output triangle vertices
  uint numVerts = tex1Dfetch<uint>(numVertsTex, cubeindex);

  for (int i = 0; i < numVerts; i += 3) {
    uint index = numVertsScanned[voxel] + i;

    float3 *v[3];
    uint edge;
    edge = tex1Dfetch<uint>(triTex, (cubeindex * 16) + i);
#if USE_SHARED
    v[0] = &vertlist[(edge * NTHREADS) + threadIdx.x];
#else
    v[0] = &vertlist[edge];
#endif

    edge = tex1Dfetch<uint>(triTex, (cubeindex * 16) + i + 1);
#if USE_SHARED
    v[1] = &vertlist[(edge * NTHREADS) + threadIdx.x];
#else
    v[1] = &vertlist[edge];
#endif

    edge = tex1Dfetch<uint>(triTex, (cubeindex * 16) + i + 2);
#if USE_SHARED
    v[2] = &vertlist[(edge * NTHREADS) + threadIdx.x];
#else
    v[2] = &vertlist[edge];
#endif

    // calculate triangle surface normal
    float3 n = calcNormal(v[0], v[1], v[2]);

    if (index < (maxVerts - 3)) {
      pos[index] = make_float4(*v[0], 1.0f);
      norm[index] = make_float4(n, 0.0f);

      pos[index + 1] = make_float4(*v[1], 1.0f);
      norm[index + 1] = make_float4(n, 0.0f);

      pos[index + 2] = make_float4(*v[2], 1.0f);
      norm[index + 2] = make_float4(n, 0.0f);
    }
  }
}

extern "C" void launch_generateTriangles2(
    dim3 grid, dim3 threads, float4 *pos, float4 *norm,
    uint *compactedVoxelArray, uint *numVertsScanned, uchar *volume,
    uint3 gridSize, uint3 gridSizeShift, uint3 gridSizeMask, float3 voxelSize,
    float isoValue, uint activeVoxels, uint maxVerts) {
  generateTriangles2<<<grid, NTHREADS>>>(
      pos, norm, compactedVoxelArray, numVertsScanned, volume, gridSize,
      gridSizeShift, gridSizeMask, voxelSize, isoValue, activeVoxels, maxVerts,
      triTex, numVertsTex, volumeTex);
  getLastCudaError("generateTriangles2 failed");
}

extern "C" void ThrustScanWrapper(unsigned int *output, unsigned int *input,
                                  unsigned int numElements) {
  thrust::exclusive_scan(thrust::device_ptr<unsigned int>(input),
                         thrust::device_ptr<unsigned int>(input + numElements),
                         thrust::device_ptr<unsigned int>(output));
}

#endif
