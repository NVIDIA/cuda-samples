#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
    This example demonstrates how to use the CUDA C bindings to OpenGL ES to
    dynamically modify a vertex buffer using a CUDA C kernel.

    The steps are:
    1. Create an empty vertex buffer object (VBO)
    2. Register the VBO with CUDA C
    3. Map the VBO for writing from CUDA C
    4. Run CUDA C kernel to modify the vertex positions
    5. Unmap the VBO
    6. Render the results using OpenGL ES

    Host code
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <stdarg.h>
#include <unistd.h>
#include <X11/Xlib.h>
#include <X11/Xutil.h>

void error_exit(const char *format, ...) {
  va_list args;
  va_start(args, format);
  vfprintf(stderr, format, args);
  va_end(args);
  exit(1);
}

#include "graphics_interface.c"

#ifdef _WIN32
#define WINDOWS_LEAN_AND_MEAN
#define NOMINMAX
#include <windows.h>
#endif

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>  // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>  // helper functions for CUDA error check
//#include <helper_cuda_gl.h>      // helper functions for CUDA/GL interop

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 0.0f
#define THRESHOLD 0.0f
#define REFRESH_DELAY 1  // ms

#define GUI_IDLE 0x100
#define GUI_ROTATE 0x101
#define GUI_TRANSLATE 0x102

int gui_mode;

////////////////////////////////////////////////////////////////////////////////
// constants
const unsigned int window_width = 512;
const unsigned int window_height = 512;

const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// OpenGL ES variables and interop with CUDA C
GLuint mesh_vao, mesh_vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// UI / mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Frame statistics
int frame;
int fpsCount = 0;  // FPS count for averaging
int fpsLimit = 1;  // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;

// Auto-Verification Code
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a, b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

// CUDA functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGLES (VBO)";

void computeFPS() {
  frameCount++;
  fpsCount++;

  if (fpsCount == fpsLimit) {
    avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
    fpsCount = 0;
    fpsLimit = (int)MAX(avgFPS, 1.f);

    sdkResetTimer(&timer);
  }

  char fps[256];
  sprintf(fps, "Cuda/OpenGL ES Interop (VBO): %3.1f fps (Max 1000 fps)",
          avgFPS);
  graphics_set_windowtitle(fps);
}

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, unsigned int width,
                                  unsigned int height, float time) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  // calculate uv coordinates
  float u = x / (float)width;
  float v = y / (float)height;
  u = u * 2.0f - 1.0f;
  v = v * 2.0f - 1.0f;

  // calculate simple sine wave pattern
  float freq = 4.0f;
  float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

  // write output vertex
  pos[y * width + x] = make_float4(u, w, v, 1.0f);
}

void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, float time) {
  // execute the kernel
  dim3 block(8, 8, 1);
  dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
  simple_vbo_kernel<<<grid, block>>>(pos, mesh_width, mesh_height, time);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource) {
  // map OpenGL buffer object for writing from CUDA
  float4 *dptr;
  hipGraphicsMapResources(1, vbo_resource, 0);
  size_t num_bytes;
  hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                       *vbo_resource);
  // printf("Sample CUDA mapped VBO: May access %ld bytes\n", num_bytes);

  // execute the kernel
  //    dim3 block(8, 8, 1);
  //    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
  //    kernel<<< grid, block>>>(dptr, mesh_width, mesh_height, g_fAnim);

  launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

  // unmap buffer object
  hipGraphicsUnmapResources(1, vbo_resource, 0);
}

#ifdef _WIN32
#ifndef FOPEN
#define FOPEN(fHandle, filename, mode) fopen_s(&fHandle, filename, mode)
#endif
#else
#ifndef FOPEN
#define FOPEN(fHandle, filename, mode) (fHandle = fopen(filename, mode))
#endif
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename) {
  printf("sdkDumpBin: <%s>\n", filename);
  FILE *fp;
  FOPEN(fp, filename, "wb");
  fwrite(data, bytes, 1, fp);
  fflush(fp);
  fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file) {
  char *reference_file = NULL;
  void *imageData = malloc(mesh_width * mesh_height * sizeof(float));

  // execute the kernel
  launch_kernel((float4 *)d_vbo_buffer, mesh_width, mesh_height, g_fAnim);

  hipDeviceSynchronize();
  getLastCudaError("launch_kernel failed");

  hipMemcpy(imageData, d_vbo_buffer, mesh_width * mesh_height * sizeof(float),
             hipMemcpyDeviceToHost);

  sdkDumpBin2(imageData, mesh_width * mesh_height * sizeof(float),
              "simpleGL.bin");
  reference_file = sdkFindFilePath(ref_file, argv[0]);

  if (reference_file &&
      !sdkCompareBin2BinFloat("simpleGL.bin", reference_file,
                              mesh_width * mesh_height * sizeof(float),
                              MAX_EPSILON_ERROR, THRESHOLD, pArgv[0])) {
    g_TotalErrors++;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display_thisframe(float time_delta) {
  sdkStartTimer(&timer);

  // run CUDA kernel to generate vertex positions
  runCuda(&cuda_vbo_resource);

  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  // GET_GLERROR(0);

  // set view matrix: broken, it doesn't work in OpenGL ES! Must put into shader
  // glMatrixMode(GL_MODELVIEW);
  // glLoadIdentity();
  // glTranslatef(0.0, 0.0, translate_z);
  // glRotatef(rotate_x, 1.0, 0.0, 0.0);
  // glRotatef(rotate_y, 0.0, 1.0, 0.0);

  glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);

  // GET_GLERROR(0);
  glFinish();
  // GET_GLERROR(0);

  g_fAnim += time_delta;

  sdkStopTimer(&timer);
  computeFPS();
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo) {
  if (!d_vbo_buffer) {
    printf("%s: Mapping result buffer from OpenGL ES\n", __FUNCTION__);

    hipGraphicsUnregisterResource(cuda_vbo_resource);

    // map buffer object
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    float *data = (float *)glMapBufferRange(
        GL_ARRAY_BUFFER, 0, mesh_width * mesh_height * 4 * sizeof(float),
        GL_READ_ONLY);

    // check result
    if (checkCmdLineFlag(argc, (const char **)argv, "regression")) {
      // write file for regression test
      sdkWriteFile<float>("./data/regression.dat", data,
                          mesh_width * mesh_height * 3, 0.0, false);
    }

    // unmap GL buffer object
    if (!glUnmapBuffer(GL_ARRAY_BUFFER)) {
      fprintf(stderr, "Unmap buffer failed.\n");
      fflush(stderr);
    }

    checkCudaErrors(hipGraphicsGLRegisterBuffer(
        &cuda_vbo_resource, vbo, cudaGraphicsMapFlagsWriteDiscard));

    GET_GLERROR(0);
  }
}

GLuint mesh_shader = 0;

void readAndCompileShaderFromGLSLFile(GLuint new_shaderprogram,
                                      const char *filename, GLenum shaderType) {
  FILE *file = fopen(filename, "rb");  // open shader text file
  if (!file) error_exit("Filename %s does not exist\n", filename);

  /* get the size of the file and read it */
  fseek(file, 0, SEEK_END);
  GLint size = ftell(file);
  char *data = (char *)malloc(sizeof(char) * (size + 1));
  memset(data, 0, sizeof(char) * (size + 1));
  fseek(file, 0, SEEK_SET);
  size_t res = fread(data, 1, size, file);
  fclose(file);

  GLuint shader = glCreateShader(shaderType);
  glShaderSource(shader, 1, (const GLchar **)&data, &size);
  glCompileShader(shader);

  GET_GLERROR(0);
  GLint compile_success = 0;
  glGetShaderiv(shader, GL_COMPILE_STATUS, &compile_success);
  GET_GLERROR(0);

  if (compile_success == GL_FALSE) {
    printf("Compilation of %s failed!\n Reason:\n", filename);

    GLint maxLength = 0;
    glGetShaderiv(shader, GL_INFO_LOG_LENGTH, &maxLength);

    char errorLog[maxLength];
    glGetShaderInfoLog(shader, maxLength, &maxLength, &errorLog[0]);

    printf("%s", errorLog);

    glDeleteShader(shader);
    exit(1);
  }

  glAttachShader(new_shaderprogram, shader);
  glDeleteShader(shader);  // good to do?

  free(data);
}

GLuint ShaderCreate(const char *vshader_filename,
                    const char *fshader_filename) {
  printf("Loading GLSL shaders %s %s\n", vshader_filename, fshader_filename);

  GLuint new_shaderprogram = glCreateProgram();

  GET_GLERROR(0);
  if (vshader_filename)
    readAndCompileShaderFromGLSLFile(new_shaderprogram, vshader_filename,
                                     GL_VERTEX_SHADER);

  GET_GLERROR(0);
  if (fshader_filename)
    readAndCompileShaderFromGLSLFile(new_shaderprogram, fshader_filename,
                                     GL_FRAGMENT_SHADER);

  GET_GLERROR(0);

  glLinkProgram(new_shaderprogram);

  GET_GLERROR(0);
  GLint link_success;
  glGetProgramiv(new_shaderprogram, GL_LINK_STATUS, &link_success);

  if (link_success == GL_FALSE) {
    printf("Linking of %s with %s failed!\n Reason:\n", vshader_filename,
           fshader_filename);

    GLint maxLength = 0;
    glGetShaderiv(new_shaderprogram, GL_INFO_LOG_LENGTH, &maxLength);

    char errorLog[maxLength];
    glGetShaderInfoLog(new_shaderprogram, maxLength, &maxLength, &errorLog[0]);

    printf("%s", errorLog);

    exit(EXIT_FAILURE);
  }

  return new_shaderprogram;
}

//===========================================================================
// InitGraphicsState() - initialize OpenGL
//===========================================================================
static void InitGraphicsState(void) {
  char *GL_version = (char *)glGetString(GL_VERSION);
  char *GL_vendor = (char *)glGetString(GL_VENDOR);
  char *GL_renderer = (char *)glGetString(GL_RENDERER);

  printf("Version: %s\n", GL_version);
  printf("Vendor: %s\n", GL_vendor);
  printf("Renderer: %s\n", GL_renderer);

  // RENDERING SETUP (OpenGL ES or OpenGL Core Profile!)
  glGenVertexArrays(1, &mesh_vao);  // Features' Vertex Array Object allocation
  glBindVertexArray(mesh_vao);      // bind VAO

  // initialize buffer object
  glGenBuffers(1, &mesh_vbo);
  glBindBuffer(GL_ARRAY_BUFFER, mesh_vbo);

  unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
  glBufferData(GL_ARRAY_BUFFER, size, NULL, GL_DYNAMIC_DRAW);
  glVertexAttribPointer((GLuint)0, 4, GL_FLOAT, GL_FALSE, 0, 0);
  glEnableVertexAttribArray(0);

  checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, mesh_vbo,
                                               cudaGraphicsMapFlagsNone));
  // glBindVertexArray(0); // keep above Vertex Array Object bound (it's the
  // only one throughout)

  // GLSL stuff
  char *vertex_shader_path = sdkFindFilePath("mesh.vert.glsl", pArgv[0]);
  char *fragment_shader_path = sdkFindFilePath("mesh.frag.glsl", pArgv[0]);

  if (vertex_shader_path == NULL || fragment_shader_path == NULL) {
    printf("Error finding shader file\n");
    exit(EXIT_FAILURE);
  }

  mesh_shader = ShaderCreate(vertex_shader_path, fragment_shader_path);
  GET_GLERROR(0);

  free(vertex_shader_path);
  free(fragment_shader_path);

  glUseProgram(mesh_shader);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file) {
  // Create the CUTIL timer
  sdkCreateTimer(&timer);

  int devID = 0;
#if defined(__aarch64__) || defined(__arm__)
  // find iGPU on the system which is compute capable which will perform
  // GLES-CUDA interop
  devID = findIntegratedGPU();
#else
  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  devID = findCudaDevice(argc, (const char **)argv);
#endif

  // command line mode only
  if (ref_file != NULL) {
    // create VBO
    checkCudaErrors(hipMalloc((void **)&d_vbo_buffer,
                               mesh_width * mesh_height * 4 * sizeof(float)));

    // run the cuda part
    runAutoTest(devID, argv, ref_file);

    // check result of Cuda step
    checkResultCuda(argc, argv, mesh_vbo);

    hipFree(d_vbo_buffer);
    d_vbo_buffer = NULL;
  } else {
    // this would use command-line specified CUDA device, note that CUDA
    // defaults to highest Gflops/s device
    if (checkCmdLineFlag(argc, (const char **)argv, "device"))
      error_exit("Device setting not yet implemented!\n");

    // create X11 window and set up associated OpenGL ES context
    graphics_setup_window(0, 0, window_width, window_height, sSDKsample);

    InitGraphicsState();  // set up GLES stuff

    glClearColor(0, 0.5, 1, 1);  // blue-ish background
    glClear(GL_COLOR_BUFFER_BIT);

    // printf("WP%d\n", __LINE__);
    graphics_swap_buffers();

    XEvent event;
    KeySym key;
    char text[255];

    int frame = 0;

    while (frame < 100000) {
      if (XPending(display)) {
        XNextEvent(display, &event);

        if (event.type == Expose && event.xexpose.count == 0) {
          printf("Redraw requested!\n");
        }
        if (event.type == KeyPress &&
            XLookupString(&event.xkey, text, 255, &key, 0) == 1) {
          if (text[0] == 27) goto label_stop_x;

          printf("You pressed the %c key!\n", text[0]);
        }

        if (event.type == ButtonPress) {
          printf("Mouse button %d press at (%d,%d)\n", event.xbutton.button,
                 event.xbutton.x, event.xbutton.y);

          if (event.xbutton.button == Button1) gui_mode = GUI_TRANSLATE;
          if (event.xbutton.button == Button3) gui_mode = GUI_ROTATE;
          mouse_old_x = event.xbutton.x;
          mouse_old_y = event.xbutton.y;
        }

        if (event.type == ButtonRelease) {
          printf("Mouse button %d released at (%d,%d)\n", event.xbutton.button,
                 event.xbutton.x, event.xbutton.y);

          gui_mode = GUI_IDLE;
          mouse_old_x = event.xbutton.x;
          mouse_old_y = event.xbutton.y;
        }

        if (event.type == MotionNotify) {
          // printf("Mouse motion towards %d %d, GUI mode is 0x%x\n",
          //	   event.xmotion.x, event.xmotion.y, gui_mode);
          float dx, dy;
          dx = (float)(event.xmotion.x - mouse_old_x);
          dy = (float)(event.xmotion.y - mouse_old_y);

          if (gui_mode == GUI_ROTATE) {
            rotate_x += dy * 0.2f;
            rotate_y += dx * 0.2f;
            printf("rot x %f y %f\n", rotate_x, rotate_y);
          }
          if (gui_mode == GUI_TRANSLATE) {
            translate_z += dy * 0.01f;
            printf("translate z %f\n", translate_z);
          }

          mouse_old_x = event.xmotion.x;
          mouse_old_y = event.xmotion.y;
        }
      }

      display_thisframe(0.010);
      usleep(1000);  // need not take full CPU and GPU

      graphics_swap_buffers();
      // printf("frame %d\n",frame++);
    }

  label_stop_x:
    // NOTE: Before destroying OpenGL ES context, must unregister all shared
    // resources from CUDA !
    hipGraphicsUnregisterResource(cuda_vbo_resource);

    graphics_close_window();  // close window and destroy OpenGL ES context

    sdkDeleteTimer(&timer);
  }

  return true;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  char *ref_file = NULL;

  pArgc = &argc;
  pArgv = argv;

#if defined(__linux__)
  setenv("DISPLAY", ":0", 0);
#endif

  printf("%s starting...\n", sSDKsample);

  if (argc > 1) {
    if (checkCmdLineFlag(argc, (const char **)argv, "file")) {
      // In this mode, we run without OpenGL and see if VBO is generated
      // correctly
      getCmdLineArgumentString(argc, (const char **)argv, "file",
                               (char **)&ref_file);
    }
  }

  printf("\n");

  runTest(argc, argv, ref_file);

  printf("%s completed, returned %s\n", sSDKsample,
         (g_TotalErrors == 0) ? "OK" : "ERROR!");

  exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}
