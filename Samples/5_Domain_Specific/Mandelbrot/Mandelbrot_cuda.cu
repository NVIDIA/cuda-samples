#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include "hip/hip_runtime_api.h"
#include "Mandelbrot_kernel.h"
#include "Mandelbrot_kernel.cuh"

// The Mandelbrot CUDA GPU thread function

template <class T>
__global__ void Mandelbrot0(uchar4 *dst, const int imageW, const int imageH,
                            const int crunch, const T xOff, const T yOff,
                            const T xJP, const T yJP, const T scale,
                            const uchar4 colors, const int frame,
                            const int animationFrame, const int gridWidth,
                            const int numBlocks, const bool isJ) {
  // loop until all blocks completed
  for (unsigned int blockIndex = blockIdx.x; blockIndex < numBlocks;
       blockIndex += gridDim.x) {
    unsigned int blockX = blockIndex % gridWidth;
    unsigned int blockY = blockIndex / gridWidth;

    // process this block
    const int ix = blockDim.x * blockX + threadIdx.x;
    const int iy = blockDim.y * blockY + threadIdx.y;

    if ((ix < imageW) && (iy < imageH)) {
      // Calculate the location
      const T xPos = (T)ix * scale + xOff;
      const T yPos = (T)iy * scale + yOff;

      // Calculate the Mandelbrot index for the current location
      int m = CalcMandelbrot<T>(xPos, yPos, xJP, yJP, crunch, isJ);
      //            int m = blockIdx.x;         // uncomment to see scheduling
      //            order
      m = m > 0 ? crunch - m : 0;

      // Convert the Mandelbrot index into a color
      uchar4 color;

      if (m) {
        m += animationFrame;
        color.x = m * colors.x;
        color.y = m * colors.y;
        color.z = m * colors.z;
      } else {
        color.x = 0;
        color.y = 0;
        color.z = 0;
      }

      // Output the pixel
      int pixel = imageW * iy + ix;

      if (frame == 0) {
        color.w = 0;
        dst[pixel] = color;
      } else {
        int frame1 = frame + 1;
        int frame2 = frame1 / 2;
        dst[pixel].x = (dst[pixel].x * frame + color.x + frame2) / frame1;
        dst[pixel].y = (dst[pixel].y * frame + color.y + frame2) / frame1;
        dst[pixel].z = (dst[pixel].z * frame + color.z + frame2) / frame1;
      }
    }
  }

}  // Mandelbrot0

// The Mandelbrot CUDA GPU thread function (double single version)
__global__ void MandelbrotDS0(uchar4 *dst, const int imageW, const int imageH,
                              const int crunch, const float xOff0,
                              const float xOff1, const float yOff0,
                              const float yOff1, const float xJP,
                              const float yJP, const float scale,
                              const uchar4 colors, const int frame,
                              const int animationFrame, const int gridWidth,
                              const int numBlocks, const bool isJ) {
  // loop until all blocks completed
  for (unsigned int blockIndex = blockIdx.x; blockIndex < numBlocks;
       blockIndex += gridDim.x) {
    unsigned int blockX = blockIndex % gridWidth;
    unsigned int blockY = blockIndex / gridWidth;

    // process this block
    const int ix = blockDim.x * blockX + threadIdx.x;
    const int iy = blockDim.y * blockY + threadIdx.y;

    if ((ix < imageW) && (iy < imageH)) {
      // Calculate the location
      float xPos0 = (float)ix * scale;
      float xPos1 = 0.0f;
      float yPos0 = (float)iy * scale;
      float yPos1 = 0.0f;
      dsadd(xPos0, xPos1, xPos0, xPos1, xOff0, xOff1);
      dsadd(yPos0, yPos1, yPos0, yPos1, yOff0, yOff1);

      // Calculate the Mandelbrot index for the current location
      int m =
          CalcMandelbrotDS(xPos0, xPos1, yPos0, yPos1, xJP, yJP, crunch, isJ);
      m = m > 0 ? crunch - m : 0;

      // Convert the Mandelbrot index into a color
      uchar4 color;

      if (m) {
        m += animationFrame;
        color.x = m * colors.x;
        color.y = m * colors.y;
        color.z = m * colors.z;
      } else {
        color.x = 0;
        color.y = 0;
        color.z = 0;
      }

      // Output the pixel
      int pixel = imageW * iy + ix;

      if (frame == 0) {
        color.w = 0;
        dst[pixel] = color;
      } else {
        int frame1 = frame + 1;
        int frame2 = frame1 / 2;
        dst[pixel].x = (dst[pixel].x * frame + color.x + frame2) / frame1;
        dst[pixel].y = (dst[pixel].y * frame + color.y + frame2) / frame1;
        dst[pixel].z = (dst[pixel].z * frame + color.z + frame2) / frame1;
      }
    }
  }
}  // MandelbrotDS0

// The Mandelbrot secondary AA pass CUDA GPU thread function
template <class T>
__global__ void Mandelbrot1(uchar4 *dst, const int imageW, const int imageH,
                            const int crunch, const T xOff, const T yOff,
                            const T xJP, const T yJP, const T scale,
                            const uchar4 colors, const int frame,
                            const int animationFrame, const int gridWidth,
                            const int numBlocks, const bool isJ) {
  // loop until all blocks completed
  for (unsigned int blockIndex = blockIdx.x; blockIndex < numBlocks;
       blockIndex += gridDim.x) {
    unsigned int blockX = blockIndex % gridWidth;
    unsigned int blockY = blockIndex / gridWidth;

    // process this block
    const int ix = blockDim.x * blockX + threadIdx.x;
    const int iy = blockDim.y * blockY + threadIdx.y;

    if ((ix < imageW) && (iy < imageH)) {
      // Get the current pixel color
      int pixel = imageW * iy + ix;
      uchar4 pixelColor = dst[pixel];
      int count = 0;

      // Search for pixels out of tolerance surrounding the current pixel
      if (ix > 0) {
        count += CheckColors(pixelColor, dst[pixel - 1]);
      }

      if (ix + 1 < imageW) {
        count += CheckColors(pixelColor, dst[pixel + 1]);
      }

      if (iy > 0) {
        count += CheckColors(pixelColor, dst[pixel - imageW]);
      }

      if (iy + 1 < imageH) {
        count += CheckColors(pixelColor, dst[pixel + imageW]);
      }

      if (count) {
        // Calculate the location
        const T xPos = (T)ix * scale + xOff;
        const T yPos = (T)iy * scale + yOff;

        // Calculate the Mandelbrot index for the current location
        int m = CalcMandelbrot(xPos, yPos, xJP, yJP, crunch, isJ);
        m = m > 0 ? crunch - m : 0;

        // Convert the Mandelbrot index into a color
        uchar4 color;

        if (m) {
          m += animationFrame;
          color.x = m * colors.x;
          color.y = m * colors.y;
          color.z = m * colors.z;
        } else {
          color.x = 0;
          color.y = 0;
          color.z = 0;
        }

        // Output the pixel
        int frame1 = frame + 1;
        int frame2 = frame1 / 2;
        dst[pixel].x = (pixelColor.x * frame + color.x + frame2) / frame1;
        dst[pixel].y = (pixelColor.y * frame + color.y + frame2) / frame1;
        dst[pixel].z = (pixelColor.z * frame + color.z + frame2) / frame1;
      }
    }
  }

}  // Mandelbrot1

// The Mandelbrot secondary AA pass CUDA GPU thread function (double single
// version)
__global__ void MandelbrotDS1(uchar4 *dst, const int imageW, const int imageH,
                              const int crunch, const float xOff0,
                              const float xOff1, const float yOff0,
                              const float yOff1, const float xJP,
                              const float yJP, const float scale,
                              const uchar4 colors, const int frame,
                              const int animationFrame, const int gridWidth,
                              const int numBlocks, const bool isJ) {
  // loop until all blocks completed
  for (unsigned int blockIndex = blockIdx.x; blockIndex < numBlocks;
       blockIndex += gridDim.x) {
    unsigned int blockX = blockIndex % gridWidth;
    unsigned int blockY = blockIndex / gridWidth;

    // process this block
    const int ix = blockDim.x * blockX + threadIdx.x;
    const int iy = blockDim.y * blockY + threadIdx.y;

    if ((ix < imageW) && (iy < imageH)) {
      // Get the current pixel color
      int pixel = imageW * iy + ix;
      uchar4 pixelColor = dst[pixel];
      int count = 0;

      // Search for pixels out of tolerance surrounding the current pixel
      if (ix > 0) {
        count += CheckColors(pixelColor, dst[pixel - 1]);
      }

      if (ix + 1 < imageW) {
        count += CheckColors(pixelColor, dst[pixel + 1]);
      }

      if (iy > 0) {
        count += CheckColors(pixelColor, dst[pixel - imageW]);
      }

      if (iy + 1 < imageH) {
        count += CheckColors(pixelColor, dst[pixel + imageW]);
      }

      if (count) {
        // Calculate the location
        float xPos0 = (float)ix * scale;
        float xPos1 = 0.0f;
        float yPos0 = (float)iy * scale;
        float yPos1 = 0.0f;
        dsadd(xPos0, xPos1, xPos0, xPos1, xOff0, xOff1);
        dsadd(yPos0, yPos1, yPos0, yPos1, yOff0, yOff1);

        // Calculate the Mandelbrot index for the current location
        int m =
            CalcMandelbrotDS(xPos0, xPos1, yPos0, yPos1, xJP, yJP, crunch, isJ);
        m = m > 0 ? crunch - m : 0;

        // Convert the Mandelbrot index into a color
        uchar4 color;

        if (m) {
          m += animationFrame;
          color.x = m * colors.x;
          color.y = m * colors.y;
          color.z = m * colors.z;
        } else {
          color.x = 0;
          color.y = 0;
          color.z = 0;
        }

        // Output the pixel
        int frame1 = frame + 1;
        int frame2 = frame1 / 2;
        dst[pixel].x = (pixelColor.x * frame + color.x + frame2) / frame1;
        dst[pixel].y = (pixelColor.y * frame + color.y + frame2) / frame1;
        dst[pixel].z = (pixelColor.z * frame + color.z + frame2) / frame1;
      }
    }
  }

}  // MandelbrotDS1

// The host CPU Mandelbrot thread spawner
void RunMandelbrot0(uchar4 *dst, const int imageW, const int imageH,
                    const int crunch, const double xOff, const double yOff,
                    const double xjp, const double yjp, const double scale,
                    const uchar4 colors, const int frame,
                    const int animationFrame, const int mode, const int numSMs,
                    const bool isJ, int version) {
  dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
  dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

  int numWorkerBlocks = numSMs;

  switch (mode) {
    default:
    case 0:
      Mandelbrot0<float><<<numWorkerBlocks, threads>>>(
          dst, imageW, imageH, crunch, (float)xOff, (float)yOff, (float)xjp,
          (float)yjp, (float)scale, colors, frame, animationFrame, grid.x,
          grid.x * grid.y, isJ);
      break;
    case 1:
      float x0, x1, y0, y1;
      dsdeq(x0, x1, xOff);
      dsdeq(y0, y1, yOff);
      MandelbrotDS0<<<numWorkerBlocks, threads>>>(
          dst, imageW, imageH, crunch, x0, x1, y0, y1, (float)xjp, (float)yjp,
          (float)scale, colors, frame, animationFrame, grid.x, grid.x * grid.y,
          isJ);
      break;
    case 2:
      Mandelbrot0<double><<<numWorkerBlocks, threads>>>(
          dst, imageW, imageH, crunch, xOff, yOff, xjp, yjp, scale, colors,
          frame, animationFrame, grid.x, grid.x * grid.y, isJ);
      break;
  }

  getLastCudaError("Mandelbrot0 kernel execution failed.\n");
}  // RunMandelbrot0

// The host CPU Mandelbrot thread spawner
void RunMandelbrot1(uchar4 *dst, const int imageW, const int imageH,
                    const int crunch, const double xOff, const double yOff,
                    const double xjp, const double yjp, const double scale,
                    const uchar4 colors, const int frame,
                    const int animationFrame, const int mode, const int numSMs,
                    const bool isJ, int version) {
  dim3 threads(BLOCKDIM_X, BLOCKDIM_Y);
  dim3 grid(iDivUp(imageW, BLOCKDIM_X), iDivUp(imageH, BLOCKDIM_Y));

  int numWorkerBlocks = numSMs;

  switch (mode) {
    default:
    case 0:
      Mandelbrot1<float><<<numWorkerBlocks, threads>>>(
          dst, imageW, imageH, crunch, (float)xOff, (float)yOff, (float)xjp,
          (float)yjp, (float)scale, colors, frame, animationFrame, grid.x,
          grid.x * grid.y, isJ);
      break;
    case 1:
      float x0, x1, y0, y1;
      dsdeq(x0, x1, xOff);
      dsdeq(y0, y1, yOff);
      MandelbrotDS1<<<numWorkerBlocks, threads>>>(
          dst, imageW, imageH, crunch, x0, x1, y0, y1, (float)xjp, (float)yjp,
          (float)scale, colors, frame, animationFrame, grid.x, grid.x * grid.y,
          isJ);
      break;
    case 2:
      Mandelbrot1<double><<<numWorkerBlocks, threads>>>(
          dst, imageW, imageH, crunch, xOff, yOff, xjp, yjp, scale, colors,
          frame, animationFrame, grid.x, grid.x * grid.y, isJ);
      break;
  }

  getLastCudaError("Mandelbrot1 kernel execution failed.\n");
}  // RunMandelbrot1
