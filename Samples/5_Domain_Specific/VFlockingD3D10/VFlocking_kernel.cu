#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_vector_types.h>
#include <VFlockingD3D10.h>

#define PI 3.1415926536f

typedef unsigned int uint;

__device__ bool isInsideQuad_D(float2 pos0, float2 pos1, float width,
                               float height) {
  if (fabs(pos0.x - pos1.x) < 0.5f * width &&
      fabs(pos0.y - pos1.y) < 0.5f * height) {
    return true;
  } else {
    return false;
  }
}

__device__ bool isInsideBird(float2 pixel, float2 pos, float width,
                             float height, float radius) {
  if (abs(pixel.x - pos.x) < 0.5f * width &&
          abs(pixel.y - pos.y) < 0.5f * height ||
      (pixel.x - pos.x) * (pixel.x - pos.x) +
              (pixel.y - pos.y) * (pixel.y - pos.y) <
          radius * radius) {
    return true;
  } else {
    return false;
  }
}

__global__ void cuda_kernel_update(float2 *newPos, float2 *curPos,
                                   uint numBirds, bool *hasproxy,
                                   bool *neighbors, bool *rightgoals,
                                   bool *leftgoals, Params *params) {
  uint i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= numBirds) {
    return;
  }

  float minDist = 50000.f;
  float2 dij = make_float2(0.f);

  if (!hasproxy[i]) {
    for (uint j = 0; j < numBirds; j++) {
      if (j == i) {
        continue;
      }

      if (leftgoals[i * numBirds + j]) {
        dij = params->dX * normalize(curPos[j] - curPos[i]);
        break;
      }
    }
  } else {
    bool collision = false;

    for (uint j = 0; j < numBirds; j++) {
      float d;

      if (leftgoals[i * numBirds + j]) {
        d = curPos[j].x - (params->wingspan + params->lambda) - curPos[i].x;

        if (fabs(d) < fabs(minDist)) {
          minDist = d;
        }
      }

      if (rightgoals[i * numBirds + j]) {
        d = curPos[j].x + (params->wingspan + params->lambda) - curPos[i].x;

        if (fabs(d) < fabs(minDist)) {
          minDist = d;
        }
      }

      if (neighbors[i * numBirds + j] && !collision) {
        if (curPos[j].y >= curPos[i].y &&
            curPos[j].y < curPos[i].y + params->epsilon) {
          dij.y = -params->dY;
          collision = true;
        }
      }
    }

    if (fabs(minDist) <= params->dX) {
      return;
    }

    dij.x = minDist > 0 ? params->dX : -params->dX;
  }

  newPos[i].x = curPos[i].x + dij.x;
  newPos[i].y = curPos[i].y + dij.y;
}

__global__ void cuda_kernel_checktriples(float2 *pos, uint numBirds,
                                         bool *hasproxy, bool *neighbors,
                                         bool *rightgoals, bool *leftgoals,
                                         uint3 *triples, Params *params) {
  uint ith = blockIdx.x * blockDim.x + threadIdx.x;

  if (ith >= numBirds * (numBirds - 1) * (numBirds - 2) / 6) {
    return;
  }

  uint a[3];
  a[0] = triples[ith].x;
  a[1] = triples[ith].y;
  a[2] = triples[ith].z;

  uint i, j, x;

  for (i = 0; i < 3; i++) {
    for (j = 2; j > i; j--) {
      if (pos[a[j - 1]].y > pos[a[j]].y) {
        x = a[j - 1];
        a[j - 1] = a[j];
        a[j] = x;
      }
    }
  }

  if (hasproxy[a[0]]) {
    float a2a1 = pos[a[2]].x - pos[a[1]].x;

    if (fabs(a2a1) < 2.f * (params->wingspan + params->lambda))
      if (a2a1 >= 0) {
        if (leftgoals[a[0] * numBirds + a[2]]) {
          leftgoals[a[0] * numBirds + a[2]] = false;
        }

        if (rightgoals[a[0] * numBirds + a[1]]) {
          rightgoals[a[0] * numBirds + a[1]] = false;
        }
      } else {
        if (leftgoals[a[0] * numBirds + a[1]]) {
          leftgoals[a[0] * numBirds + a[1]] = false;
        }

        if (rightgoals[a[0] * numBirds + a[2]]) {
          rightgoals[a[0] * numBirds + a[2]] = false;
        }
      }
  } else {
    if ((leftgoals[a[0] * numBirds + a[2]]) &&
        (leftgoals[a[0] * numBirds + a[1]]))
      if ((length(pos[a[1]] - pos[a[0]]) < length(pos[a[2]] - pos[a[0]]))) {
        leftgoals[a[0] * numBirds + a[2]] = false;
      } else {
        leftgoals[a[0] * numBirds + a[1]] = false;
      }
  }
}

__global__ void cuda_kernel_checkpairs(float2 *pos, uint numBirds,
                                       bool *hasproxy, bool *neighbors,
                                       bool *rightgoals, bool *leftgoals,
                                       uint2 *pairs, Params *params) {
  uint i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i >= numBirds * (numBirds - 1) / 2) {
    return;
  }

  uint front, back;

  if (pos[pairs[i].y].y > pos[pairs[i].x].y) {
    front = pairs[i].y;
    back = pairs[i].x;
  } else {
    front = pairs[i].x;
    back = pairs[i].y;
  }

  leftgoals[back * numBirds + front] = true;
  rightgoals[back * numBirds + front] = true;

  float2 stepback;
  stepback.x = pos[front].x;
  stepback.y = pos[front].y - 0.5f * params->upwashY;

  if (isInsideQuad_D(
          pos[back], stepback,
          2.f * (params->wingspan + params->lambda + params->upwashX),
          params->upwashY)) {
    neighbors[back * numBirds + front] = true;

    if (!hasproxy[back]) {
      hasproxy[back] = true;
    }
  }
}

extern "C" void cuda_simulate(float2 *newPos, float2 *curPos, uint numBirds,
                              bool *d_hasproxy, bool *d_neighbors,
                              bool *d_leftgoals, bool *d_rightgoals,
                              uint2 *d_pairs, uint3 *d_triples,
                              Params *d_params) {
  hipError_t error = hipSuccess;
  float tempms;
  static float ms = 0.f;
  static uint step = 0;
  int smallblockSize = 32, midblockSize = 128, bigblockSize = 32;

  hipEvent_t e_start, e_stop;
  hipEventCreate(&e_start);
  hipEventCreate(&e_stop);
  hipEventRecord(e_start, 0);

  hipMemset(d_leftgoals, 0, numBirds * numBirds * sizeof(bool));
  hipMemset(d_rightgoals, 0, numBirds * numBirds * sizeof(bool));
  hipMemset(d_hasproxy, 0, numBirds * sizeof(bool));
  hipMemset(d_neighbors, 0, numBirds * numBirds * sizeof(bool));

  dim3 Db = dim3(bigblockSize);
  dim3 Dg =
      dim3((numBirds * (numBirds - 1) / 2 + bigblockSize - 1) / bigblockSize);
  cuda_kernel_checkpairs<<<Dg, Db>>>(curPos, numBirds, d_hasproxy, d_neighbors,
                                     d_rightgoals, d_leftgoals, d_pairs,
                                     d_params);

  Db = dim3(midblockSize);
  Dg =
      dim3((numBirds * (numBirds - 1) * (numBirds - 2) / 6 + bigblockSize - 1) /
           bigblockSize);
  cuda_kernel_checktriples<<<Dg, Db>>>(curPos, numBirds, d_hasproxy,
                                       d_neighbors, d_rightgoals, d_leftgoals,
                                       d_triples, d_params);

  Db = dim3(smallblockSize);
  Dg = dim3((numBirds + smallblockSize - 1) / smallblockSize);
  cuda_kernel_update<<<Dg, Db>>>(newPos, curPos, numBirds, d_hasproxy,
                                 d_neighbors, d_rightgoals, d_leftgoals,
                                 d_params /*, d_pWingTips */);

  hipDeviceSynchronize();

  hipEventRecord(e_stop, 0);
  hipEventSynchronize(e_stop);
  hipEventElapsedTime(&tempms, e_start, e_stop);
  ms += tempms;

  if (!(step % 100) && step) {
    printf("GPU, step %d \ntime per step %6.3f ms \n", step, ms / 100.f);
    ms = 0.f;
  }

  step++;

  error = hipGetLastError();

  if (error != hipSuccess) {
    printf("one of the cuda kernels failed to launch, error = %d\n", error);
  }
}
