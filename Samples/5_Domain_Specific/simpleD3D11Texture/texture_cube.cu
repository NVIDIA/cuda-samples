/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define PI 3.1415926536f

/*
 * Paint a 2D surface with a moving bulls-eye pattern.  The "face" parameter
 * selects
 * between 6 different colors to use.  We will use a different color on each
 * face of a
 * cube map.
 */
__global__ void cuda_kernel_texture_cube(char *surface, int width, int height,
                                         size_t pitch, int face, float t) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned char *pixel;

  // in the case where, due to quantization into grids, we have
  // more threads than pixels, skip the threads which don't
  // correspond to valid pixels
  if (x >= width || y >= height) return;

  // get a pointer to this pixel
  pixel = (unsigned char *)(surface + y * pitch) + 4 * x;

  // populate it
  float theta_x = (2.0f * x) / width - 1.0f;
  float theta_y = (2.0f * y) / height - 1.0f;
  float theta = 2.0f * PI * sqrt(theta_x * theta_x + theta_y * theta_y);
  unsigned char value = 255 * (0.6f + 0.4f * cos(theta + t));

  pixel[3] = 255;  // alpha

  if (face % 2) {
    pixel[0] =           // blue
        pixel[1] =       // green
        pixel[2] = 0.5;  // red
    pixel[face / 2] = value;
  } else {
    pixel[0] =             // blue
        pixel[1] =         // green
        pixel[2] = value;  // red
    pixel[face / 2] = 0.5;
  }
}

extern "C" void cuda_texture_cube(void *surface, int width, int height,
                                  size_t pitch, int face, float t) {
  hipError_t error = hipSuccess;

  dim3 Db = dim3(16, 16);  // block dimensions are fixed to be 256 threads
  dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);

  cuda_kernel_texture_cube<<<Dg, Db>>>((char *)surface, width, height, pitch,
                                       face, t);

  error = hipGetLastError();

  if (error != hipSuccess) {
    printf("cuda_kernel_texture_cube() failed to launch error = %d\n", error);
  }
}
