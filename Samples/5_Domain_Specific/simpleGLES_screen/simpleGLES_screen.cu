#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
   This example demonstrates how to use the CUDA C bindings to OpenGL ES to
   dynamically modify a vertex buffer using a CUDA C kernel.

   The steps are:
   1. Create an empty vertex buffer object (VBO)
   2. Register the VBO with CUDA C
   3. Map the VBO for writing from CUDA C
   4. Run CUDA C kernel to modify the vertex positions
   5. Unmap the VBO
   6. Render the results using OpenGL ES

   Host code
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <stdarg.h>
#include <unistd.h>
#include <screen/screen.h>

#include "graphics_interface.c"

// includes, cuda
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

// Utilities and timing functions
#include <helper_functions.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h

// CUDA helper functions
#include <hip/hip_runtime_api.h>         // helper functions for CUDA error check

#include <hip/hip_vector_types.h>

#define MAX_EPSILON_ERROR 0.0f
#define THRESHOLD 0.0f
#define REFRESH_DELAY 1  // ms

#define GUI_IDLE 0x100
#define GUI_ROTATE 0x101
#define GUI_TRANSLATE 0x102

int gui_mode;

////////////////////////////////////////////////////////////////////////////////
// Default configuration
unsigned int window_width = 512;
unsigned int window_height = 512;
unsigned int dispno = 0;

// constants
const unsigned int mesh_width = 256;
const unsigned int mesh_height = 256;

// OpenGL ES variables and interop with CUDA C
GLuint mesh_vao, mesh_vbo;
struct hipGraphicsResource *cuda_vbo_resource;
void *d_vbo_buffer = NULL;

float g_fAnim = 0.0;

// UI / mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

StopWatchInterface *timer = NULL;

// Frame statistics
int frame;
int fpsCount = 0;  // FPS count for averaging
int fpsLimit = 1;  // FPS limit for sampling
int g_Index = 0;
float avgFPS = 0.0f;
unsigned int frameCount = 0;
unsigned int g_TotalErrors = 0;

// The default number of seconds after which the test will end.
#define TIME_LIMIT 10.0  // 10 secs

// Flag indicating it is time to shut down
static GLboolean shutdown = GL_FALSE;

// Callback to close window
static void closeCB_app(void) { shutdown = GL_TRUE; }

// Callback to handle key presses
static void keyCB_app(char key, int state) {
  // Ignoring releases
  if (!state) return;

  if ((key == 'q') || (key == 'Q') || (key == NvGlDemoKeyCode_Escape))
    shutdown = GL_TRUE;
}

// Auto-Verification Code
bool g_bQAReadback = false;

int *pArgc = NULL;
char **pArgv = NULL;

#define MAX(a, b) ((a > b) ? a : b)

////////////////////////////////////////////////////////////////////////////////
// declaration, forward

// CUDA functionality
void runCuda(struct hipGraphicsResource **vbo_resource);
void runAutoTest(int devID, char **argv, char *ref_file);
void checkResultCuda(int argc, char **argv, const GLuint &vbo);

const char *sSDKsample = "simpleGLES on Screen (VBO)";

void computeFPS() {
  frameCount++;
  fpsCount++;

  if (fpsCount == fpsLimit) {
    avgFPS = 1.f / (sdkGetAverageTimerValue(&timer) / 1000.f);
    fpsCount = 0;
    fpsLimit = (int)MAX(avgFPS, 1.f);

    sdkResetTimer(&timer);
  }

  char fps[256];
  sprintf(fps, "Cuda/OpenGL ES Interop (VBO): %3.1f fps (Max 1000 fps)",
          avgFPS);
  graphics_set_windowtitle(fps);
}

///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4 *pos, unsigned int width,
                                  unsigned int height, float time) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

  // calculate uv coordinates
  float u = x / (float)width;
  float v = y / (float)height;
  u = u * 2.0f - 1.0f;
  v = v * 2.0f - 1.0f;

  // calculate simple sine wave pattern
  float freq = 4.0f;
  float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

  // write output vertex
  pos[y * width + x] = make_float4(u, w, v, 1.0f);
}

void launch_kernel(float4 *pos, unsigned int mesh_width,
                   unsigned int mesh_height, float time) {
  // execute the kernel
  dim3 block(8, 8, 1);
  dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
  simple_vbo_kernel<<<grid, block>>>(pos, mesh_width, mesh_height, time);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runCuda(struct hipGraphicsResource **vbo_resource) {
  // map OpenGL buffer object for writing from CUDA
  float4 *dptr;
  hipGraphicsMapResources(1, vbo_resource, 0);
  size_t num_bytes;
  hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes,
                                       *vbo_resource);

  launch_kernel(dptr, mesh_width, mesh_height, g_fAnim);

  // unmap buffer object
  hipGraphicsUnmapResources(1, vbo_resource, 0);
}

#ifndef FOPEN
#define FOPEN(fHandle, filename, mode) (fHandle = fopen(filename, mode))
#endif

void sdkDumpBin2(void *data, unsigned int bytes, const char *filename) {
  printf("sdkDumpBin: <%s>\n", filename);
  FILE *fp;
  FOPEN(fp, filename, "wb");
  fwrite(data, bytes, 1, fp);
  fflush(fp);
  fclose(fp);
}

////////////////////////////////////////////////////////////////////////////////
//! Run the Cuda part of the computation
////////////////////////////////////////////////////////////////////////////////
void runAutoTest(int devID, char **argv, char *ref_file) {
  char *reference_file = NULL;
  void *imageData = malloc(mesh_width * mesh_height * sizeof(float));

  // execute the kernel
  launch_kernel((float4 *)d_vbo_buffer, mesh_width, mesh_height, g_fAnim);

  hipDeviceSynchronize();
  getLastCudaError("launch_kernel failed");

  hipMemcpy(imageData, d_vbo_buffer, mesh_width * mesh_height * sizeof(float),
             hipMemcpyDeviceToHost);

  sdkDumpBin2(imageData, mesh_width * mesh_height * sizeof(float),
              "simpleGLES_screen.bin");
  reference_file = sdkFindFilePath(ref_file, argv[0]);

  if (reference_file &&
      !sdkCompareBin2BinFloat("simpleGLES_screen.bin", reference_file,
                              mesh_width * mesh_height * sizeof(float),
                              MAX_EPSILON_ERROR, THRESHOLD, pArgv[0])) {
    g_TotalErrors++;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Display callback
////////////////////////////////////////////////////////////////////////////////
void display_thisframe(float time_delta) {
  sdkStartTimer(&timer);

  // run CUDA kernel to generate vertex positions
  runCuda(&cuda_vbo_resource);

  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

  glDrawArrays(GL_POINTS, 0, mesh_width * mesh_height);

  glFinish();

  g_fAnim += time_delta;

  sdkStopTimer(&timer);
  computeFPS();
}

////////////////////////////////////////////////////////////////////////////////
//! Check if the result is correct or write data to file for external
//! regression testing
////////////////////////////////////////////////////////////////////////////////
void checkResultCuda(int argc, char **argv, const GLuint &vbo) {
  if (!d_vbo_buffer) {
    printf("%s: Mapping result buffer from OpenGL ES\n", __FUNCTION__);

    hipGraphicsUnregisterResource(cuda_vbo_resource);

    // map buffer object
    glBindBuffer(GL_ARRAY_BUFFER, vbo);
    float *data = (float *)glMapBufferRange(
        GL_ARRAY_BUFFER, 0, mesh_width * mesh_height * 4 * sizeof(float),
        GL_READ_ONLY);

    // check result
    if (checkCmdLineFlag(argc, (const char **)argv, "regression")) {
      // write file for regression test
      sdkWriteFile<float>("./data/regression.dat", data,
                          mesh_width * mesh_height * 3, 0.0, false);
    }

    // unmap GL buffer object
    if (!glUnmapBuffer(GL_ARRAY_BUFFER)) {
      fprintf(stderr, "Unmap buffer failed.\n");
      fflush(stderr);
    }

    checkCudaErrors(hipGraphicsGLRegisterBuffer(
        &cuda_vbo_resource, vbo, cudaGraphicsMapFlagsWriteDiscard));

    CHECK_GLERROR();
  }
}

GLuint mesh_shader = 0;

void readAndCompileShaderFromGLSLFile(GLuint new_shaderprogram,
                                      const char *filename, GLenum shaderType) {
  FILE *file = fopen(filename, "rb");  // open shader text file
  if (!file) {
    error_exit("Filename %s does not exist\n", filename);
  }

  // get the size of the file and read it
  fseek(file, 0, SEEK_END);
  GLint size = ftell(file);
  char *data = (char *)malloc(sizeof(char) * (size + 1));
  memset(data, 0, sizeof(char) * (size + 1));
  fseek(file, 0, SEEK_SET);
  size_t res = fread(data, 1, size, file);
  fclose(file);

  GLuint shader = glCreateShader(shaderType);
  glShaderSource(shader, 1, (const GLchar **)&data, &size);
  glCompileShader(shader);

  CHECK_GLERROR();
  GLint compile_success = 0;
  glGetShaderiv(shader, GL_COMPILE_STATUS, &compile_success);
  CHECK_GLERROR();

  if (compile_success == GL_FALSE) {
    printf("Compilation of %s failed!\n Reason:\n", filename);

    GLint maxLength = 0;
    glGetShaderiv(shader, GL_INFO_LOG_LENGTH, &maxLength);

    char errorLog[maxLength];
    glGetShaderInfoLog(shader, maxLength, &maxLength, &errorLog[0]);

    printf("%s", errorLog);

    glDeleteShader(shader);
    exit(1);
  }

  glAttachShader(new_shaderprogram, shader);
  glDeleteShader(shader);

  free(data);
}

GLuint ShaderCreate(const char *vshader_filename,
                    const char *fshader_filename) {
  printf("Loading GLSL shaders %s %s\n", vshader_filename, fshader_filename);

  GLuint new_shaderprogram = glCreateProgram();

  CHECK_GLERROR();
  if (vshader_filename) {
    readAndCompileShaderFromGLSLFile(new_shaderprogram, vshader_filename,
                                     GL_VERTEX_SHADER);
  }

  CHECK_GLERROR();
  if (fshader_filename) {
    readAndCompileShaderFromGLSLFile(new_shaderprogram, fshader_filename,
                                     GL_FRAGMENT_SHADER);
  }

  CHECK_GLERROR();

  glLinkProgram(new_shaderprogram);

  CHECK_GLERROR();
  GLint link_success;
  glGetProgramiv(new_shaderprogram, GL_LINK_STATUS, &link_success);

  if (link_success == GL_FALSE) {
    printf("Linking of %s with %s failed!\n Reason:\n", vshader_filename,
           fshader_filename);

    GLint maxLength = 0;
    glGetShaderiv(new_shaderprogram, GL_INFO_LOG_LENGTH, &maxLength);

    char errorLog[maxLength];
    glGetShaderInfoLog(new_shaderprogram, maxLength, &maxLength, &errorLog[0]);

    printf("%s", errorLog);

    exit(EXIT_FAILURE);
  }

  return new_shaderprogram;
}

//===========================================================================
// InitGraphicsState() - initialize OpenGL
//===========================================================================
static void InitGraphicsState(void) {
  char *GL_version = (char *)glGetString(GL_VERSION);
  char *GL_vendor = (char *)glGetString(GL_VENDOR);
  char *GL_renderer = (char *)glGetString(GL_RENDERER);

  printf("Version: %s\n", GL_version);
  printf("Vendor: %s\n", GL_vendor);
  printf("Renderer: %s\n", GL_renderer);

  // RENDERING SETUP (OpenGL ES or OpenGL Core Profile!)
  glGenVertexArrays(1, &mesh_vao);  // Features' Vertex Array Object allocation
  glBindVertexArray(mesh_vao);      // bind VAO

  // initialize buffer object
  glGenBuffers(1, &mesh_vbo);
  glBindBuffer(GL_ARRAY_BUFFER, mesh_vbo);

  unsigned int size = mesh_width * mesh_height * 4 * sizeof(float);
  glBufferData(GL_ARRAY_BUFFER, size, NULL, GL_DYNAMIC_DRAW);
  glVertexAttribPointer((GLuint)0, 4, GL_FLOAT, GL_FALSE, 0, 0);
  glEnableVertexAttribArray(0);

  checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, mesh_vbo,
                                               cudaGraphicsMapFlagsNone));

  // GLSL stuff
  char *vertex_shader_path = sdkFindFilePath("mesh.vert.glsl", pArgv[0]);
  char *fragment_shader_path = sdkFindFilePath("mesh.frag.glsl", pArgv[0]);

  if (vertex_shader_path == NULL || fragment_shader_path == NULL) {
    printf("Error finding shader file\n");
    exit(EXIT_FAILURE);
  }

  mesh_shader = ShaderCreate(vertex_shader_path, fragment_shader_path);
  CHECK_GLERROR();

  free(vertex_shader_path);
  free(fragment_shader_path);

  glUseProgram(mesh_shader);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv, char *ref_file) {
  // command line mode only
  if (ref_file != NULL) {
    // This will pick the best possible CUDA capable device
    // int devID = findCudaDevice(argc, (const char **)argv);
#if defined(__aarch64__) || defined(__arm__)
    // find iGPU on the system which is compute capable which will perform
    // GLES-CUDA interop
    int devID = findIntegratedGPU();
#else
    // use command-line specified CUDA device, otherwise use device with highest
    // Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);
#endif

    // create VBO
    checkCudaErrors(hipMalloc((void **)&d_vbo_buffer,
                               mesh_width * mesh_height * 4 * sizeof(float)));

    // run the cuda part
    runAutoTest(devID, argv, ref_file);

    // check result of Cuda step
    checkResultCuda(argc, argv, mesh_vbo);

    hipFree(d_vbo_buffer);
    d_vbo_buffer = NULL;
  } else {
    double endTime = TIME_LIMIT;

    // this would use command-line specified CUDA device, note that CUDA
    // defaults to highest Gflops/s device
    if (checkCmdLineFlag(argc, (const char **)argv, "device")) {
      error_exit("Device setting not yet implemented!\n");
    }

    // display selection
    if (checkCmdLineFlag(argc, (const char **)argv, "dispno")) {
      dispno = getCmdLineArgumentInt(argc, (const char **)argv, "dispno");
    }

    // Window width
    if (checkCmdLineFlag(argc, (const char **)argv, "width")) {
      window_width = getCmdLineArgumentInt(argc, (const char **)argv, "width");
    }

    // Window Height
    if (checkCmdLineFlag(argc, (const char **)argv, "height")) {
      window_height =
          getCmdLineArgumentInt(argc, (const char **)argv, "height");
    }

    // Determine how long to run for in secs: default is 10s
    if (checkCmdLineFlag(argc, (const char **)argv, "runtime")) {
      endTime = getCmdLineArgumentInt(argc, (const char **)argv, "runtime");
    }

    SetCloseCB(closeCB_app);
    SetKeyCB(keyCB_app);

    // create QNX screen window and set up associated OpenGL ES context
    graphics_setup_window(0, 0, window_width, window_height, sSDKsample,
                          dispno);

#if defined(__aarch64__) || defined(__arm__)
    // find iGPU on the system which is compute capable which will perform
    // GLES-CUDA interop
    int devID = findIntegratedGPU();
#else
    // use command-line specified CUDA device, otherwise use device with highest
    // Gflops/s
    int devID = findCudaDevice(argc, (const char **)argv);
#endif
    InitGraphicsState();  // set up GLES stuff

    glClearColor(0, 0.5, 1, 1);  // blue-ish background
    glClear(GL_COLOR_BUFFER_BIT);

    graphics_swap_buffers();

    int frame = 0;

    struct timeval begin, end;
    gettimeofday(&begin, NULL);

    // Print runtime
    if (endTime < 0.0) {
      endTime = TIME_LIMIT;
      printf(" running forever...\n");
    } else {
      printf(" running for %f seconds...\n", endTime);
    }

    while (!shutdown) {
      frame++;
      display_thisframe(0.010);
      usleep(1000);
      graphics_swap_buffers();
      CheckEvents();

      gettimeofday(&end, 0);
      double elapsed = (end.tv_sec - begin.tv_sec) +
                       ((end.tv_usec - begin.tv_usec) / 1000000.0);

      // Check whether time limit has been exceeded
      if (!shutdown) shutdown = (endTime <= elapsed);
    }

    // NOTE: Before destroying OpenGL ES context, must unregister all shared
    // resources from CUDA !
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));

    graphics_close_window();  // close window and destroy OpenGL ES context
  }

  return true;
}

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  char *ref_file = NULL;

  pArgc = &argc;
  pArgv = argv;

#if defined(__linux__)
  setenv("DISPLAY", ":0", 0);
#endif

  printf("%s starting...\n", sSDKsample);

  if (argc > 1) {
    if (checkCmdLineFlag(argc, (const char **)argv, "file")) {
      // In this mode, we run without OpenGL and see if VBO is generated
      // correctly
      getCmdLineArgumentString(argc, (const char **)argv, "file",
                               (char **)&ref_file);
    }
  }

  printf("\n");

  runTest(argc, argv, ref_file);

  printf("%s completed, returned %s\n", sSDKsample,
         (g_TotalErrors == 0) ? "OK" : "ERROR!");

  exit(g_TotalErrors == 0 ? EXIT_SUCCESS : EXIT_FAILURE);
}
