#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime_api.h>
#include <math.h>

#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

// CUDA standard includes
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include "bodysystem.h"

__constant__ float softeningSquared;
__constant__ double softeningSquared_fp64;

hipError_t setSofteningSquared(float softeningSq) {
  return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared), &softeningSq, sizeof(float), 0,
                            hipMemcpyHostToDevice);
}

hipError_t setSofteningSquared(double softeningSq) {
  return hipMemcpyToSymbol(HIP_SYMBOL(softeningSquared_fp64), &softeningSq, sizeof(double),
                            0, hipMemcpyHostToDevice);
}

template <class T>
struct SharedMemory {
  __device__ inline operator T *() {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

template <typename T>
__device__ T rsqrt_T(T x) {
  return rsqrt(x);
}

template <>
__device__ float rsqrt_T<float>(float x) {
  return rsqrtf(x);
}

template <>
__device__ double rsqrt_T<double>(double x) {
  return rsqrt(x);
}

// Macros to simplify shared memory addressing
#define SX(i) sharedPos[i + blockDim.x * threadIdx.y]
// This macro is only used when multithreadBodies is true (below)
#define SX_SUM(i, j) sharedPos[i + blockDim.x * j]

template <typename T>
__device__ T getSofteningSquared() {
  return softeningSquared;
}
template <>
__device__ double getSofteningSquared<double>() {
  return softeningSquared_fp64;
}

template <typename T>
struct DeviceData {
  T *dPos[2];  // mapped host pointers
  T *dVel;
  hipEvent_t event;
  unsigned int offset;
  unsigned int numBodies;
};

template <typename T>
__device__ typename vec3<T>::Type bodyBodyInteraction(
    typename vec3<T>::Type ai, typename vec4<T>::Type bi,
    typename vec4<T>::Type bj) {
  typename vec3<T>::Type r;

  // r_ij  [3 FLOPS]
  r.x = bj.x - bi.x;
  r.y = bj.y - bi.y;
  r.z = bj.z - bi.z;

  // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPS]
  T distSqr = r.x * r.x + r.y * r.y + r.z * r.z;
  distSqr += getSofteningSquared<T>();

  // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
  T invDist = rsqrt_T(distSqr);
  T invDistCube = invDist * invDist * invDist;

  // s = m_j * invDistCube [1 FLOP]
  T s = bj.w * invDistCube;

  // a_i =  a_i + s * r_ij [6 FLOPS]
  ai.x += r.x * s;
  ai.y += r.y * s;
  ai.z += r.z * s;

  return ai;
}

template <typename T>
__device__ typename vec3<T>::Type computeBodyAccel(
    typename vec4<T>::Type bodyPos, typename vec4<T>::Type *positions,
    int numTiles, cg::thread_block cta) {
  typename vec4<T>::Type *sharedPos = SharedMemory<typename vec4<T>::Type>();

  typename vec3<T>::Type acc = {0.0f, 0.0f, 0.0f};

  for (int tile = 0; tile < numTiles; tile++) {
    sharedPos[threadIdx.x] = positions[tile * blockDim.x + threadIdx.x];

    cg::sync(cta);

// This is the "tile_calculation" from the GPUG3 article.
#pragma unroll 128

    for (unsigned int counter = 0; counter < blockDim.x; counter++) {
      acc = bodyBodyInteraction<T>(acc, bodyPos, sharedPos[counter]);
    }

    cg::sync(cta);
  }

  return acc;
}

template <typename T>
__global__ void integrateBodies(typename vec4<T>::Type *__restrict__ newPos,
                                typename vec4<T>::Type *__restrict__ oldPos,
                                typename vec4<T>::Type *vel,
                                unsigned int deviceOffset,
                                unsigned int deviceNumBodies, float deltaTime,
                                float damping, int numTiles) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index >= deviceNumBodies) {
    return;
  }

  typename vec4<T>::Type position = oldPos[deviceOffset + index];

  typename vec3<T>::Type accel =
      computeBodyAccel<T>(position, oldPos, numTiles, cta);

  // acceleration = force / mass;
  // new velocity = old velocity + acceleration * deltaTime
  // note we factor out the body's mass from the equation, here and in
  // bodyBodyInteraction
  // (because they cancel out).  Thus here force == acceleration
  typename vec4<T>::Type velocity = vel[deviceOffset + index];

  velocity.x += accel.x * deltaTime;
  velocity.y += accel.y * deltaTime;
  velocity.z += accel.z * deltaTime;

  velocity.x *= damping;
  velocity.y *= damping;
  velocity.z *= damping;

  // new position = old position + velocity * deltaTime
  position.x += velocity.x * deltaTime;
  position.y += velocity.y * deltaTime;
  position.z += velocity.z * deltaTime;

  // store new position and velocity
  newPos[deviceOffset + index] = position;
  vel[deviceOffset + index] = velocity;
}

template <typename T>
void integrateNbodySystem(DeviceData<T> *deviceData,
                          hipGraphicsResource **pgres,
                          unsigned int currentRead, float deltaTime,
                          float damping, unsigned int numBodies,
                          unsigned int numDevices, int blockSize,
                          bool bUsePBO) {
  if (bUsePBO) {
    checkCudaErrors(cudaGraphicsResourceSetMapFlags(
        pgres[currentRead], cudaGraphicsMapFlagsReadOnly));
    checkCudaErrors(cudaGraphicsResourceSetMapFlags(
        pgres[1 - currentRead], cudaGraphicsMapFlagsWriteDiscard));
    checkCudaErrors(hipGraphicsMapResources(2, pgres, 0));
    size_t bytes;
    checkCudaErrors(hipGraphicsResourceGetMappedPointer(
        (void **)&(deviceData[0].dPos[currentRead]), &bytes,
        pgres[currentRead]));
    checkCudaErrors(hipGraphicsResourceGetMappedPointer(
        (void **)&(deviceData[0].dPos[1 - currentRead]), &bytes,
        pgres[1 - currentRead]));
  }

  for (unsigned int dev = 0; dev != numDevices; dev++) {
    if (numDevices > 1) {
      hipSetDevice(dev);
    }

    int numBlocks = (deviceData[dev].numBodies + blockSize - 1) / blockSize;
    int numTiles = (numBodies + blockSize - 1) / blockSize;
    int sharedMemSize = blockSize * 4 * sizeof(T);  // 4 floats for pos

    integrateBodies<T><<<numBlocks, blockSize, sharedMemSize>>>(
        (typename vec4<T>::Type *)deviceData[dev].dPos[1 - currentRead],
        (typename vec4<T>::Type *)deviceData[dev].dPos[currentRead],
        (typename vec4<T>::Type *)deviceData[dev].dVel, deviceData[dev].offset,
        deviceData[dev].numBodies, deltaTime, damping, numTiles);

    if (numDevices > 1) {
      checkCudaErrors(hipEventRecord(deviceData[dev].event));
      // MJH: Hack on older driver versions to force kernel launches to flush!
      hipStreamQuery(0);
    }

    // check if kernel invocation generated an error
    getLastCudaError("Kernel execution failed");
  }

  if (numDevices > 1) {
    for (unsigned int dev = 0; dev < numDevices; dev++) {
      checkCudaErrors(hipEventSynchronize(deviceData[dev].event));
    }
  }

  if (bUsePBO) {
    checkCudaErrors(hipGraphicsUnmapResources(2, pgres, 0));
  }
}

// Explicit specializations needed to generate code
template void integrateNbodySystem<float>(DeviceData<float> *deviceData,
                                          hipGraphicsResource **pgres,
                                          unsigned int currentRead,
                                          float deltaTime, float damping,
                                          unsigned int numBodies,
                                          unsigned int numDevices,
                                          int blockSize, bool bUsePBO);

template void integrateNbodySystem<double>(DeviceData<double> *deviceData,
                                           hipGraphicsResource **pgres,
                                           unsigned int currentRead,
                                           float deltaTime, float damping,
                                           unsigned int numBodies,
                                           unsigned int numDevices,
                                           int blockSize, bool bUsePBO);
