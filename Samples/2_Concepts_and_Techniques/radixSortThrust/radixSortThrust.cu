#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/random.h>
#include <thrust/generate.h>
#include <thrust/detail/type_traits.h>

#include <hip/hip_runtime_api.h>

#include <algorithm>
#include <time.h>
#include <limits.h>

template <typename T, bool floatKeys>
bool testSort(int argc, char **argv) {
  int cmdVal;
  int keybits = 32;

  unsigned int numElements = 1048576;
  bool keysOnly = checkCmdLineFlag(argc, (const char **)argv, "keysonly");
  bool quiet = checkCmdLineFlag(argc, (const char **)argv, "quiet");

  if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
    cmdVal = getCmdLineArgumentInt(argc, (const char **)argv, "n");
    numElements = cmdVal;

    if (cmdVal < 0) {
      printf("Error: elements must be > 0, elements=%d is invalid\n", cmdVal);
      exit(EXIT_SUCCESS);
    }
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "keybits")) {
    cmdVal = getCmdLineArgumentInt(argc, (const char **)argv, "keybits");
    keybits = cmdVal;

    if (keybits <= 0) {
      printf("Error: keybits must be > 0, keybits=%d is invalid\n", keybits);
      exit(EXIT_SUCCESS);
    }
  }

  unsigned int numIterations = (numElements >= 16777216) ? 10 : 100;

  if (checkCmdLineFlag(argc, (const char **)argv, "iterations")) {
    cmdVal = getCmdLineArgumentInt(argc, (const char **)argv, "iterations");
    numIterations = cmdVal;
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "help")) {
    printf("Command line:\nradixSortThrust [-option]\n");
    printf("Valid options:\n");
    printf("-n=<N>        : number of elements to sort\n");
    printf("-keybits=bits : keybits must be > 0\n");
    printf(
        "-keysonly     : only sort an array of keys (default sorts key-value "
        "pairs)\n");
    printf(
        "-float        : use 32-bit float keys (default is 32-bit unsigned "
        "int)\n");
    printf(
        "-quiet        : Output only the number of elements and the time to "
        "sort\n");
    printf("-help         : Output a help message\n");
    exit(EXIT_SUCCESS);
  }

  if (!quiet)
    printf("\nSorting %d %d-bit %s keys %s\n\n", numElements, keybits,
           floatKeys ? "float" : "unsigned int",
           keysOnly ? "(only)" : "and values");

  int deviceID = -1;

  if (hipSuccess == hipGetDevice(&deviceID)) {
    hipDeviceProp_t devprop;
    hipGetDeviceProperties(&devprop, deviceID);
    unsigned int totalMem = (keysOnly ? 2 : 4) * numElements * sizeof(T);

    if (devprop.totalGlobalMem < totalMem) {
      printf("Error: insufficient amount of memory to sort %d elements.\n",
             numElements);
      printf("%d bytes needed, %d bytes available\n", (int)totalMem,
             (int)devprop.totalGlobalMem);
      exit(EXIT_SUCCESS);
    }
  }

  thrust::host_vector<T> h_keys(numElements);
  thrust::host_vector<T> h_keysSorted(numElements);
  thrust::host_vector<unsigned int> h_values;

  if (!keysOnly) h_values = thrust::host_vector<unsigned int>(numElements);

  // Fill up with some random data
  thrust::default_random_engine rng(clock());

  if (floatKeys) {
    thrust::uniform_real_distribution<float> u01(0, 1);

    for (int i = 0; i < (int)numElements; i++) h_keys[i] = u01(rng);
  } else {
    thrust::uniform_int_distribution<unsigned int> u(0, UINT_MAX);

    for (int i = 0; i < (int)numElements; i++) h_keys[i] = u(rng);
  }

  if (!keysOnly) thrust::sequence(h_values.begin(), h_values.end());

  // Copy data onto the GPU
  thrust::device_vector<T> d_keys;
  thrust::device_vector<unsigned int> d_values;

  // run multiple iterations to compute an average sort time
  hipEvent_t start_event, stop_event;
  checkCudaErrors(hipEventCreate(&start_event));
  checkCudaErrors(hipEventCreate(&stop_event));

  float totalTime = 0;

  for (unsigned int i = 0; i < numIterations; i++) {
    // reset data before sort
    d_keys = h_keys;

    if (!keysOnly) d_values = h_values;

    checkCudaErrors(hipEventRecord(start_event, 0));

    if (keysOnly)
      thrust::sort(d_keys.begin(), d_keys.end());
    else
      thrust::sort_by_key(d_keys.begin(), d_keys.end(), d_values.begin());

    checkCudaErrors(hipEventRecord(stop_event, 0));
    checkCudaErrors(hipEventSynchronize(stop_event));

    float time = 0;
    checkCudaErrors(hipEventElapsedTime(&time, start_event, stop_event));
    totalTime += time;
  }

  totalTime /= (1.0e3f * numIterations);
  printf(
      "radixSortThrust, Throughput = %.4f MElements/s, Time = %.5f s, Size = "
      "%u elements\n",
      1.0e-6f * numElements / totalTime, totalTime, numElements);

  getLastCudaError("after radixsort");

  // Get results back to host for correctness checking
  thrust::copy(d_keys.begin(), d_keys.end(), h_keysSorted.begin());

  if (!keysOnly)
    thrust::copy(d_values.begin(), d_values.end(), h_values.begin());

  getLastCudaError("copying results to host memory");

  // Check results
  bool bTestResult =
      thrust::is_sorted(h_keysSorted.begin(), h_keysSorted.end());

  checkCudaErrors(hipEventDestroy(start_event));
  checkCudaErrors(hipEventDestroy(stop_event));

  if (!bTestResult && !quiet) {
    return false;
  }

  return bTestResult;
}

int main(int argc, char **argv) {
  // Start logs
  printf("%s Starting...\n\n", argv[0]);

  findCudaDevice(argc, (const char **)argv);

  bool bTestResult = false;

  if (checkCmdLineFlag(argc, (const char **)argv, "float"))
    bTestResult = testSort<float, true>(argc, argv);
  else
    bTestResult = testSort<unsigned int, false>(argc, argv);

  printf(bTestResult ? "Test passed\n" : "Test failed!\n");
}
