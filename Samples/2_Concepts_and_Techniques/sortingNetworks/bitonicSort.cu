#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//Based on http://www.iti.fh-flensburg.de/lang/algorithmen/sortieren/bitonic/bitonicen.htm

#include <assert.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include "sortingNetworks_common.h"
#include "sortingNetworks_common.cuh"

////////////////////////////////////////////////////////////////////////////////
// Monolithic bitonic sort kernel for short arrays fitting into shared memory
////////////////////////////////////////////////////////////////////////////////
__global__ void bitonicSortShared(uint *d_DstKey, uint *d_DstVal,
                                  uint *d_SrcKey, uint *d_SrcVal,
                                  uint arrayLength, uint dir) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  // Shared memory storage for one or more short vectors
  __shared__ uint s_key[SHARED_SIZE_LIMIT];
  __shared__ uint s_val[SHARED_SIZE_LIMIT];

  // Offset to the beginning of subbatch and load data
  d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  s_key[threadIdx.x + 0] = d_SrcKey[0];
  s_val[threadIdx.x + 0] = d_SrcVal[0];
  s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
  s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

  for (uint size = 2; size < arrayLength; size <<= 1) {
    // Bitonic merge
    uint ddd = dir ^ ((threadIdx.x & (size / 2)) != 0);

    for (uint stride = size / 2; stride > 0; stride >>= 1) {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride],
                 s_val[pos + stride], ddd);
    }
  }

  // ddd == dir for the last bitonic merge step
  {
    for (uint stride = arrayLength / 2; stride > 0; stride >>= 1) {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride],
                 s_val[pos + stride], dir);
    }
  }

  cg::sync(cta);
  d_DstKey[0] = s_key[threadIdx.x + 0];
  d_DstVal[0] = s_val[threadIdx.x + 0];
  d_DstKey[(SHARED_SIZE_LIMIT / 2)] =
      s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
  d_DstVal[(SHARED_SIZE_LIMIT / 2)] =
      s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

////////////////////////////////////////////////////////////////////////////////
// Bitonic sort kernel for large arrays (not fitting into shared memory)
////////////////////////////////////////////////////////////////////////////////
// Bottom-level bitonic sort
// Almost the same as bitonicSortShared with the exception of
// even / odd subarrays being sorted in opposite directions
// Bitonic merge accepts both
// Ascending | descending or descending | ascending sorted pairs
__global__ void bitonicSortShared1(uint *d_DstKey, uint *d_DstVal,
                                   uint *d_SrcKey, uint *d_SrcVal) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  // Shared memory storage for current subarray
  __shared__ uint s_key[SHARED_SIZE_LIMIT];
  __shared__ uint s_val[SHARED_SIZE_LIMIT];

  // Offset to the beginning of subarray and load data
  d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  s_key[threadIdx.x + 0] = d_SrcKey[0];
  s_val[threadIdx.x + 0] = d_SrcVal[0];
  s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
  s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

  for (uint size = 2; size < SHARED_SIZE_LIMIT; size <<= 1) {
    // Bitonic merge
    uint ddd = (threadIdx.x & (size / 2)) != 0;

    for (uint stride = size / 2; stride > 0; stride >>= 1) {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride],
                 s_val[pos + stride], ddd);
    }
  }

  // Odd / even arrays of SHARED_SIZE_LIMIT elements
  // sorted in opposite directions
  uint ddd = blockIdx.x & 1;
  {
    for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1) {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride],
                 s_val[pos + stride], ddd);
    }
  }

  cg::sync(cta);
  d_DstKey[0] = s_key[threadIdx.x + 0];
  d_DstVal[0] = s_val[threadIdx.x + 0];
  d_DstKey[(SHARED_SIZE_LIMIT / 2)] =
      s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
  d_DstVal[(SHARED_SIZE_LIMIT / 2)] =
      s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

// Bitonic merge iteration for stride >= SHARED_SIZE_LIMIT
__global__ void bitonicMergeGlobal(uint *d_DstKey, uint *d_DstVal,
                                   uint *d_SrcKey, uint *d_SrcVal,
                                   uint arrayLength, uint size, uint stride,
                                   uint dir) {
  uint global_comparatorI = blockIdx.x * blockDim.x + threadIdx.x;
  uint comparatorI = global_comparatorI & (arrayLength / 2 - 1);

  // Bitonic merge
  uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);
  uint pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));

  uint keyA = d_SrcKey[pos + 0];
  uint valA = d_SrcVal[pos + 0];
  uint keyB = d_SrcKey[pos + stride];
  uint valB = d_SrcVal[pos + stride];

  Comparator(keyA, valA, keyB, valB, ddd);

  d_DstKey[pos + 0] = keyA;
  d_DstVal[pos + 0] = valA;
  d_DstKey[pos + stride] = keyB;
  d_DstVal[pos + stride] = valB;
}

// Combined bitonic merge steps for
// size > SHARED_SIZE_LIMIT and stride = [1 .. SHARED_SIZE_LIMIT / 2]
__global__ void bitonicMergeShared(uint *d_DstKey, uint *d_DstVal,
                                   uint *d_SrcKey, uint *d_SrcVal,
                                   uint arrayLength, uint size, uint dir) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  // Shared memory storage for current subarray
  __shared__ uint s_key[SHARED_SIZE_LIMIT];
  __shared__ uint s_val[SHARED_SIZE_LIMIT];

  d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  s_key[threadIdx.x + 0] = d_SrcKey[0];
  s_val[threadIdx.x + 0] = d_SrcVal[0];
  s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
  s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

  // Bitonic merge
  uint comparatorI =
      UMAD(blockIdx.x, blockDim.x, threadIdx.x) & ((arrayLength / 2) - 1);
  uint ddd = dir ^ ((comparatorI & (size / 2)) != 0);

  for (uint stride = SHARED_SIZE_LIMIT / 2; stride > 0; stride >>= 1) {
    cg::sync(cta);
    uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
    Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride],
               s_val[pos + stride], ddd);
  }

  cg::sync(cta);
  d_DstKey[0] = s_key[threadIdx.x + 0];
  d_DstVal[0] = s_val[threadIdx.x + 0];
  d_DstKey[(SHARED_SIZE_LIMIT / 2)] =
      s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
  d_DstVal[(SHARED_SIZE_LIMIT / 2)] =
      s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
// Helper function (also used by odd-even merge sort)
extern "C" uint factorRadix2(uint *log2L, uint L) {
  if (!L) {
    *log2L = 0;
    return 0;
  } else {
    for (*log2L = 0; (L & 1) == 0; L >>= 1, *log2L++)
      ;

    return L;
  }
}

extern "C" uint bitonicSort(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey,
                            uint *d_SrcVal, uint batchSize, uint arrayLength,
                            uint dir) {
  // Nothing to sort
  if (arrayLength < 2) return 0;

  // Only power-of-two array lengths are supported by this implementation
  uint log2L;
  uint factorizationRemainder = factorRadix2(&log2L, arrayLength);
  assert(factorizationRemainder == 1);

  dir = (dir != 0);

  uint blockCount = batchSize * arrayLength / SHARED_SIZE_LIMIT;
  uint threadCount = SHARED_SIZE_LIMIT / 2;

  if (arrayLength <= SHARED_SIZE_LIMIT) {
    assert((batchSize * arrayLength) % SHARED_SIZE_LIMIT == 0);
    bitonicSortShared<<<blockCount, threadCount>>>(d_DstKey, d_DstVal, d_SrcKey,
                                                   d_SrcVal, arrayLength, dir);
  } else {
    bitonicSortShared1<<<blockCount, threadCount>>>(d_DstKey, d_DstVal,
                                                    d_SrcKey, d_SrcVal);

    for (uint size = 2 * SHARED_SIZE_LIMIT; size <= arrayLength; size <<= 1)
      for (unsigned stride = size / 2; stride > 0; stride >>= 1)
        if (stride >= SHARED_SIZE_LIMIT) {
          bitonicMergeGlobal<<<(batchSize * arrayLength) / 512, 256>>>(
              d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, stride,
              dir);
        } else {
          bitonicMergeShared<<<blockCount, threadCount>>>(
              d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, dir);
          break;
        }
  }

  return threadCount;
}
