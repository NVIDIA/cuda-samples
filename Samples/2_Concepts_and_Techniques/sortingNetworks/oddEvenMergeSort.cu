#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

//Based on http://www.iti.fh-flensburg.de/lang/algorithmen/sortieren/networks/oemen.htm


#include <assert.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <hip/hip_runtime_api.h>
#include "sortingNetworks_common.h"
#include "sortingNetworks_common.cuh"

////////////////////////////////////////////////////////////////////////////////
// Monolithic Bacther's sort kernel for short arrays fitting into shared memory
////////////////////////////////////////////////////////////////////////////////
__global__ void oddEvenMergeSortShared(uint *d_DstKey, uint *d_DstVal,
                                       uint *d_SrcKey, uint *d_SrcVal,
                                       uint arrayLength, uint dir) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  // Shared memory storage for one or more small vectors
  __shared__ uint s_key[SHARED_SIZE_LIMIT];
  __shared__ uint s_val[SHARED_SIZE_LIMIT];

  // Offset to the beginning of subbatch and load data
  d_SrcKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_SrcVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstKey += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  d_DstVal += blockIdx.x * SHARED_SIZE_LIMIT + threadIdx.x;
  s_key[threadIdx.x + 0] = d_SrcKey[0];
  s_val[threadIdx.x + 0] = d_SrcVal[0];
  s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcKey[(SHARED_SIZE_LIMIT / 2)];
  s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)] =
      d_SrcVal[(SHARED_SIZE_LIMIT / 2)];

  for (uint size = 2; size <= arrayLength; size <<= 1) {
    uint stride = size / 2;
    uint offset = threadIdx.x & (stride - 1);

    {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));
      Comparator(s_key[pos + 0], s_val[pos + 0], s_key[pos + stride],
                 s_val[pos + stride], dir);
      stride >>= 1;
    }

    for (; stride > 0; stride >>= 1) {
      cg::sync(cta);
      uint pos = 2 * threadIdx.x - (threadIdx.x & (stride - 1));

      if (offset >= stride)
        Comparator(s_key[pos - stride], s_val[pos - stride], s_key[pos + 0],
                   s_val[pos + 0], dir);
    }
  }

  cg::sync(cta);
  d_DstKey[0] = s_key[threadIdx.x + 0];
  d_DstVal[0] = s_val[threadIdx.x + 0];
  d_DstKey[(SHARED_SIZE_LIMIT / 2)] =
      s_key[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
  d_DstVal[(SHARED_SIZE_LIMIT / 2)] =
      s_val[threadIdx.x + (SHARED_SIZE_LIMIT / 2)];
}

////////////////////////////////////////////////////////////////////////////////
// Odd-even merge sort iteration kernel
// for large arrays (not fitting into shared memory)
////////////////////////////////////////////////////////////////////////////////
__global__ void oddEvenMergeGlobal(uint *d_DstKey, uint *d_DstVal,
                                   uint *d_SrcKey, uint *d_SrcVal,
                                   uint arrayLength, uint size, uint stride,
                                   uint dir) {
  uint global_comparatorI = blockIdx.x * blockDim.x + threadIdx.x;

  // Odd-even merge
  uint pos = 2 * global_comparatorI - (global_comparatorI & (stride - 1));

  if (stride < size / 2) {
    uint offset = global_comparatorI & ((size / 2) - 1);

    if (offset >= stride) {
      uint keyA = d_SrcKey[pos - stride];
      uint valA = d_SrcVal[pos - stride];
      uint keyB = d_SrcKey[pos + 0];
      uint valB = d_SrcVal[pos + 0];

      Comparator(keyA, valA, keyB, valB, dir);

      d_DstKey[pos - stride] = keyA;
      d_DstVal[pos - stride] = valA;
      d_DstKey[pos + 0] = keyB;
      d_DstVal[pos + 0] = valB;
    }
  } else {
    uint keyA = d_SrcKey[pos + 0];
    uint valA = d_SrcVal[pos + 0];
    uint keyB = d_SrcKey[pos + stride];
    uint valB = d_SrcVal[pos + stride];

    Comparator(keyA, valA, keyB, valB, dir);

    d_DstKey[pos + 0] = keyA;
    d_DstVal[pos + 0] = valA;
    d_DstKey[pos + stride] = keyB;
    d_DstVal[pos + stride] = valB;
  }
}

////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
// Helper function
extern "C" uint factorRadix2(uint *log2L, uint L);

extern "C" void oddEvenMergeSort(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey,
                                 uint *d_SrcVal, uint batchSize,
                                 uint arrayLength, uint dir) {
  // Nothing to sort
  if (arrayLength < 2) return;

  // Only power-of-two array lengths are supported by this implementation
  uint log2L;
  uint factorizationRemainder = factorRadix2(&log2L, arrayLength);
  assert(factorizationRemainder == 1);

  dir = (dir != 0);

  uint blockCount = (batchSize * arrayLength) / SHARED_SIZE_LIMIT;
  uint threadCount = SHARED_SIZE_LIMIT / 2;

  if (arrayLength <= SHARED_SIZE_LIMIT) {
    assert(SHARED_SIZE_LIMIT % arrayLength == 0);
    oddEvenMergeSortShared<<<blockCount, threadCount>>>(
        d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, arrayLength, dir);
  } else {
    oddEvenMergeSortShared<<<blockCount, threadCount>>>(
        d_DstKey, d_DstVal, d_SrcKey, d_SrcVal, SHARED_SIZE_LIMIT, dir);

    for (uint size = 2 * SHARED_SIZE_LIMIT; size <= arrayLength; size <<= 1)
      for (unsigned stride = size / 2; stride > 0; stride >>= 1) {
        // Unlike with bitonic sort, combining bitonic merge steps with
        // stride = [SHARED_SIZE_LIMIT / 2 .. 1] seems to be impossible as there
        // are dependencies between data elements crossing the SHARED_SIZE_LIMIT
        // borders
        oddEvenMergeGlobal<<<(batchSize * arrayLength) / 512, 256>>>(
            d_DstKey, d_DstVal, d_DstKey, d_DstVal, arrayLength, size, stride,
            dir);
      }
  }
}
