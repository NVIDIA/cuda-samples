#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
    Parallel reduction kernels
*/

#ifndef _REDUCE_KERNEL_H_
#define _REDUCE_KERNEL_H_

#define _CG_ABI_EXPERIMENTAL
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <stdio.h>

namespace cg = cooperative_groups;

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
template <class T>
struct SharedMemory {
  __device__ inline operator T *() {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

// specialize for double to avoid unaligned memory
// access compile errors
template <>
struct SharedMemory<double> {
  __device__ inline operator double *() {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }

  __device__ inline operator const double *() const {
    extern __shared__ double __smem_d[];
    return (double *)__smem_d;
  }
};

template <class T>
__device__ __forceinline__ T warpReduceSum(unsigned int mask, T mySum) {
  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    mySum += __shfl_down_sync(mask, mySum, offset);
  }
  return mySum;
}

#if __CUDA_ARCH__ >= 800
// Specialize warpReduceFunc for int inputs to use __reduce_add_sync intrinsic
// when on SM 8.0 or higher
template <>
__device__ __forceinline__ int warpReduceSum<int>(unsigned int mask,
                                                  int mySum) {
  mySum = __reduce_add_sync(mask, mySum);
  return mySum;
}
#endif

/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved
   inactivity means that no whole warps are active, which is also very
   inefficient */
template <class T>
__global__ void reduce0(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? g_idata[i] : 0;

  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    // modulo arithmetic is slow!
    if ((tid % (2 * s)) == 0) {
      sdata[tid] += sdata[tid + s];
    }

    cg::sync(cta);
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/* This version uses contiguous threads, but its interleaved
   addressing results in many shared memory bank conflicts.
*/
template <class T>
__global__ void reduce1(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? g_idata[i] : 0;

  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = 1; s < blockDim.x; s *= 2) {
    int index = 2 * s * tid;

    if (index < blockDim.x) {
      sdata[index] += sdata[index + s];
    }

    cg::sync(cta);
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
    This version uses sequential addressing -- no divergence or bank conflicts.
*/
template <class T>
__global__ void reduce2(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  sdata[tid] = (i < n) ? g_idata[i] : 0;

  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }

    cg::sync(cta);
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

/*
    This version uses n/2 threads --
    it performs the first level of reduction when reading from global memory.
*/
template <class T>
__global__ void reduce3(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  T mySum = (i < n) ? g_idata[i] : 0;

  if (i + blockDim.x < n) mySum += g_idata[i + blockDim.x];

  sdata[tid] = mySum;
  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (tid < s) {
      sdata[tid] = mySum = mySum + sdata[tid + s];
    }

    cg::sync(cta);
  }

  // write result for this block to global mem
  if (tid == 0) g_odata[blockIdx.x] = mySum;
}

/*
    This version uses the warp shuffle operation if available to reduce
    warp synchronization. When shuffle is not available the final warp's
    worth of work is unrolled to reduce looping overhead.

    See
   http://devblogs.nvidia.com/parallelforall/faster-parallel-reductions-kepler/
    for additional information about using shuffle to perform a reduction
    within a warp.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize>
__global__ void reduce4(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

  T mySum = (i < n) ? g_idata[i] : 0;

  if (i + blockSize < n) mySum += g_idata[i + blockSize];

  sdata[tid] = mySum;
  cg::sync(cta);

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (tid < s) {
      sdata[tid] = mySum = mySum + sdata[tid + s];
    }

    cg::sync(cta);
  }

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockSize >= 64) mySum += sdata[tid + 32];
    // Reduce final warp using shuffle
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      mySum += tile32.shfl_down(mySum, offset);
    }
  }

  // write result for this block to global mem
  if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}

/*
    This version is completely unrolled, unless warp shuffle is available, then
    shuffle is used within a loop.  It uses a template parameter to achieve
    optimal code for any (power of 2) number of threads.  This requires a switch
    statement in the host code to handle all the different thread block sizes at
    compile time. When shuffle is available, it is used to reduce warp
   synchronization.

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize>
__global__ void reduce5(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockSize * 2) + threadIdx.x;

  T mySum = (i < n) ? g_idata[i] : 0;

  if (i + blockSize < n) mySum += g_idata[i + blockSize];

  sdata[tid] = mySum;
  cg::sync(cta);

  // do reduction in shared mem
  if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] = mySum = mySum + sdata[tid + 256];
  }

  cg::sync(cta);

  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] = mySum = mySum + sdata[tid + 128];
  }

  cg::sync(cta);

  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] = mySum = mySum + sdata[tid + 64];
  }

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockSize >= 64) mySum += sdata[tid + 32];
    // Reduce final warp using shuffle
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      mySum += tile32.shfl_down(mySum, offset);
    }
  }

  // write result for this block to global mem
  if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}

/*
    This version adds multiple elements per thread sequentially.  This reduces
   the overall cost of the algorithm while keeping the work complexity O(n) and
   the step complexity O(log n). (Brent's Theorem optimization)

    Note, this kernel needs a minimum of 64*sizeof(T) bytes of shared memory.
    In other words if blockSize <= 32, allocate 64*sizeof(T) bytes.
    If blockSize > 32, allocate blockSize*sizeof(T) bytes.
*/
template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void reduce6(T *g_idata, T *g_odata, unsigned int n) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  T *sdata = SharedMemory<T>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int gridSize = blockSize * gridDim.x;

  T mySum = 0;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  if (nIsPow2) {
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    gridSize = gridSize << 1;

    while (i < n) {
      mySum += g_idata[i];
      // ensure we don't read out of bounds -- this is optimized away for
      // powerOf2 sized arrays
      if ((i + blockSize) < n) {
        mySum += g_idata[i + blockSize];
      }
      i += gridSize;
    }
  } else {
    unsigned int i = blockIdx.x * blockSize + threadIdx.x;
    while (i < n) {
      mySum += g_idata[i];
      i += gridSize;
    }
  }

  // each thread puts its local sum into shared memory
  sdata[tid] = mySum;
  cg::sync(cta);

  // do reduction in shared mem
  if ((blockSize >= 512) && (tid < 256)) {
    sdata[tid] = mySum = mySum + sdata[tid + 256];
  }

  cg::sync(cta);

  if ((blockSize >= 256) && (tid < 128)) {
    sdata[tid] = mySum = mySum + sdata[tid + 128];
  }

  cg::sync(cta);

  if ((blockSize >= 128) && (tid < 64)) {
    sdata[tid] = mySum = mySum + sdata[tid + 64];
  }

  cg::sync(cta);

  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  if (cta.thread_rank() < 32) {
    // Fetch final intermediate sum from 2nd warp
    if (blockSize >= 64) mySum += sdata[tid + 32];
    // Reduce final warp using shuffle
    for (int offset = tile32.size() / 2; offset > 0; offset /= 2) {
      mySum += tile32.shfl_down(mySum, offset);
    }
  }

  // write result for this block to global mem
  if (cta.thread_rank() == 0) g_odata[blockIdx.x] = mySum;
}

template <typename T, unsigned int blockSize, bool nIsPow2>
__global__ void reduce7(const T *__restrict__ g_idata, T *__restrict__ g_odata,
                        unsigned int n) {
  T *sdata = SharedMemory<T>();

  // perform first level of reduction,
  // reading from global memory, writing to shared memory
  unsigned int tid = threadIdx.x;
  unsigned int gridSize = blockSize * gridDim.x;
  unsigned int maskLength = (blockSize & 31);  // 31 = warpSize-1
  maskLength = (maskLength > 0) ? (32 - maskLength) : maskLength;
  const unsigned int mask = (0xffffffff) >> maskLength;

  T mySum = 0;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  if (nIsPow2) {
    unsigned int i = blockIdx.x * blockSize * 2 + threadIdx.x;
    gridSize = gridSize << 1;

    while (i < n) {
      mySum += g_idata[i];
      // ensure we don't read out of bounds -- this is optimized away for
      // powerOf2 sized arrays
      if ((i + blockSize) < n) {
        mySum += g_idata[i + blockSize];
      }
      i += gridSize;
    }
  } else {
    unsigned int i = blockIdx.x * blockSize + threadIdx.x;
    while (i < n) {
      mySum += g_idata[i];
      i += gridSize;
    }
  }

  // Reduce within warp using shuffle or reduce_add if T==int & CUDA_ARCH ==
  // SM 8.0
  mySum = warpReduceSum<T>(mask, mySum);

  // each thread puts its local sum into shared memory
  if ((tid % warpSize) == 0) {
    sdata[tid / warpSize] = mySum;
  }

  __syncthreads();

  const unsigned int shmem_extent =
      (blockSize / warpSize) > 0 ? (blockSize / warpSize) : 1;
  const unsigned int ballot_result = __ballot_sync(mask, tid < shmem_extent);
  if (tid < shmem_extent) {
    mySum = sdata[tid];
    // Reduce final warp using shuffle or reduce_add if T==int & CUDA_ARCH ==
    // SM 8.0
    mySum = warpReduceSum<T>(ballot_result, mySum);
  }

  // write result for this block to global mem
  if (tid == 0) {
    g_odata[blockIdx.x] = mySum;
  }
}

// Performs a reduction step and updates numTotal with how many are remaining
template <typename T, typename Group>
__device__ T cg_reduce_n(T in, Group &threads) {
  return cg::reduce(threads, in, cg::plus<T>());
}

template <class T>
__global__ void cg_reduce(T *g_idata, T *g_odata, unsigned int n) {
  // Shared memory for intermediate steps
  T *sdata = SharedMemory<T>();
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  // Handle to tile in thread block
  cg::thread_block_tile<32> tile = cg::tiled_partition<32>(cta);

  unsigned int ctaSize = cta.size();
  unsigned int numCtas = gridDim.x;
  unsigned int threadRank = cta.thread_rank();
  unsigned int threadIndex = (blockIdx.x * ctaSize) + threadRank;

  T threadVal = 0;
  {
    unsigned int i = threadIndex;
    unsigned int indexStride = (numCtas * ctaSize);
    while (i < n) {
      threadVal += g_idata[i];
      i += indexStride;
    }
    sdata[threadRank] = threadVal;
  }

  // Wait for all tiles to finish and reduce within CTA
  {
    unsigned int ctaSteps = tile.meta_group_size();
    unsigned int ctaIndex = ctaSize >> 1;
    while (ctaIndex >= 32) {
      cta.sync();
      if (threadRank < ctaIndex) {
        threadVal += sdata[threadRank + ctaIndex];
        sdata[threadRank] = threadVal;
      }
      ctaSteps >>= 1;
      ctaIndex >>= 1;
    }
  }

  // Shuffle redux instead of smem redux
  {
    cta.sync();
    if (tile.meta_group_rank() == 0) {
      threadVal = cg_reduce_n(threadVal, tile);
    }
  }

  if (threadRank == 0) g_odata[blockIdx.x] = threadVal;
}

template <class T, size_t BlockSize, size_t MultiWarpGroupSize>
__global__ void multi_warp_cg_reduce(T *g_idata, T *g_odata, unsigned int n) {
  // Shared memory for intermediate steps
  T *sdata = SharedMemory<T>();
  __shared__ cg::experimental::block_tile_memory<sizeof(T), BlockSize> scratch;

  // Handle to thread block group
  auto cta = cg::experimental::this_thread_block(scratch);
  // Handle to multiWarpTile in thread block
  auto multiWarpTile = cg::experimental::tiled_partition<MultiWarpGroupSize>(cta);

  unsigned int gridSize = BlockSize * gridDim.x;
  T threadVal = 0;

  // we reduce multiple elements per thread.  The number is determined by the
  // number of active thread blocks (via gridDim).  More blocks will result
  // in a larger gridSize and therefore fewer elements per thread
  int nIsPow2 = !(n & n-1);
  if (nIsPow2) {
    unsigned int i = blockIdx.x * BlockSize * 2 + threadIdx.x;
    gridSize = gridSize << 1;

    while (i < n) {
      threadVal += g_idata[i];
      // ensure we don't read out of bounds -- this is optimized away for
      // powerOf2 sized arrays
      if ((i + BlockSize) < n) {
        threadVal += g_idata[i + blockDim.x];
      }
      i += gridSize;
    }
  } else {
    unsigned int i = blockIdx.x * BlockSize + threadIdx.x;
    while (i < n) {
      threadVal += g_idata[i];
      i += gridSize;
    }
  }

  threadVal = cg_reduce_n(threadVal, multiWarpTile);

  if (multiWarpTile.thread_rank() == 0) {
    sdata[multiWarpTile.meta_group_rank()] = threadVal;
  }
  cg::sync(cta);

  if (threadIdx.x == 0) {
    threadVal = 0;
    for (int i=0; i < multiWarpTile.meta_group_size(); i++) {
      threadVal += sdata[i];
    }
    g_odata[blockIdx.x] = threadVal;
  }
}

extern "C" bool isPow2(unsigned int x);

////////////////////////////////////////////////////////////////////////////////
// Wrapper function for kernel launch
////////////////////////////////////////////////////////////////////////////////
template <class T>
void reduce(int size, int threads, int blocks, int whichKernel, T *d_idata,
            T *d_odata) {
  dim3 dimBlock(threads, 1, 1);
  dim3 dimGrid(blocks, 1, 1);

  // when there is only one warp per block, we need to allocate two warps
  // worth of shared memory so that we don't index shared memory out of bounds
  int smemSize =
      (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);

  // as kernel 9 - multi_warp_cg_reduce cannot work for more than 64 threads
  // we choose to set kernel 7 for this purpose.
  if (threads < 64 && whichKernel == 9)
  {
    whichKernel = 7;
  }

  // choose which of the optimized versions of reduction to launch
  switch (whichKernel) {
    case 0:
      reduce0<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
      break;

    case 1:
      reduce1<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
      break;

    case 2:
      reduce2<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
      break;

    case 3:
      reduce3<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
      break;

    case 4:
      switch (threads) {
        case 512:
          reduce4<T, 512>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 256:
          reduce4<T, 256>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 128:
          reduce4<T, 128>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 64:
          reduce4<T, 64>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 32:
          reduce4<T, 32>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 16:
          reduce4<T, 16>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 8:
          reduce4<T, 8>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 4:
          reduce4<T, 4>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 2:
          reduce4<T, 2>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 1:
          reduce4<T, 1>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;
      }

      break;

    case 5:
      switch (threads) {
        case 512:
          reduce5<T, 512>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 256:
          reduce5<T, 256>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 128:
          reduce5<T, 128>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 64:
          reduce5<T, 64>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 32:
          reduce5<T, 32>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 16:
          reduce5<T, 16>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 8:
          reduce5<T, 8>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 4:
          reduce5<T, 4>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 2:
          reduce5<T, 2>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 1:
          reduce5<T, 1>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;
      }

      break;

    case 6:
      if (isPow2(size)) {
        switch (threads) {
          case 512:
            reduce6<T, 512, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 256:
            reduce6<T, 256, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 128:
            reduce6<T, 128, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 64:
            reduce6<T, 64, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 32:
            reduce6<T, 32, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 16:
            reduce6<T, 16, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 8:
            reduce6<T, 8, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 4:
            reduce6<T, 4, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 2:
            reduce6<T, 2, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 1:
            reduce6<T, 1, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;
        }
      } else {
        switch (threads) {
          case 512:
            reduce6<T, 512, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 256:
            reduce6<T, 256, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 128:
            reduce6<T, 128, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 64:
            reduce6<T, 64, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 32:
            reduce6<T, 32, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 16:
            reduce6<T, 16, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 8:
            reduce6<T, 8, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 4:
            reduce6<T, 4, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 2:
            reduce6<T, 2, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 1:
            reduce6<T, 1, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;
        }
      }

      break;

    case 7:
      // For reduce7 kernel we require only blockSize/warpSize
      // number of elements in shared memory
      smemSize = ((threads / 32) + 1) * sizeof(T);
      if (isPow2(size)) {
        switch (threads) {
          case 1024:
            reduce7<T, 1024, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;
          case 512:
            reduce7<T, 512, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 256:
            reduce7<T, 256, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 128:
            reduce7<T, 128, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 64:
            reduce7<T, 64, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 32:
            reduce7<T, 32, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 16:
            reduce7<T, 16, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 8:
            reduce7<T, 8, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 4:
            reduce7<T, 4, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 2:
            reduce7<T, 2, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 1:
            reduce7<T, 1, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;
        }
      } else {
        switch (threads) {
          case 1024:
            reduce7<T, 1024, true>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;
          case 512:
            reduce7<T, 512, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 256:
            reduce7<T, 256, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 128:
            reduce7<T, 128, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 64:
            reduce7<T, 64, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 32:
            reduce7<T, 32, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 16:
            reduce7<T, 16, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 8:
            reduce7<T, 8, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 4:
            reduce7<T, 4, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 2:
            reduce7<T, 2, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;

          case 1:
            reduce7<T, 1, false>
                <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
            break;
        }
      }

      break;
    case 8:
      cg_reduce<T><<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
      break;
    case 9:
      constexpr int numOfMultiWarpGroups = 2;
      smemSize = numOfMultiWarpGroups * sizeof(T);
      switch (threads) {
        case 1024:
          multi_warp_cg_reduce<T, 1024, 1024/numOfMultiWarpGroups>
            <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 512:
          multi_warp_cg_reduce<T, 512, 512/numOfMultiWarpGroups>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 256:
          multi_warp_cg_reduce<T, 256, 256/numOfMultiWarpGroups>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 128:
          multi_warp_cg_reduce<T, 128, 128/numOfMultiWarpGroups>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        case 64:
          multi_warp_cg_reduce<T, 64, 64/numOfMultiWarpGroups>
              <<<dimGrid, dimBlock, smemSize>>>(d_idata, d_odata, size);
          break;

        default:
          printf("thread block size of < 64 is not supported for this kernel\n");
          break;
      }
      break;
  }
}

// Instantiate the reduction function for 3 types
template void reduce<int>(int size, int threads, int blocks, int whichKernel,
                          int *d_idata, int *d_odata);

template void reduce<float>(int size, int threads, int blocks, int whichKernel,
                            float *d_idata, float *d_odata);

template void reduce<double>(int size, int threads, int blocks, int whichKernel,
                             double *d_idata, double *d_odata);

#endif  // #ifndef _REDUCE_KERNEL_H_
