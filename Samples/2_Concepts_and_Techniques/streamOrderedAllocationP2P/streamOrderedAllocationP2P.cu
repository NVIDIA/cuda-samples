#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample demonstrates peer-to-peer access of stream ordered memory
 * allocated with hipMallocAsync and cudaMemPool family of APIs through simple
 * kernel which does peer-to-peer to access & scales vector elements.
 */

// System includes
#include <assert.h>
#include <stdio.h>
#include <iostream>
#include <map>
#include <set>
#include <utility>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

// Simple kernel to demonstrate copying hipMallocAsync memory via P2P to peer
// device
__global__ void copyP2PAndScale(const int *src, int *dst, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    // scale & store src vector.
    dst[idx] = 2 * src[idx];
  }
}

// Map of device version to device number
std::multimap<std::pair<int, int>, int> getIdenticalGPUs() {
  int numGpus = 0;
  checkCudaErrors(hipGetDeviceCount(&numGpus));

  std::multimap<std::pair<int, int>, int> identicalGpus;

  for (int i = 0; i < numGpus; i++) {
    int isMemPoolSupported = 0;
    checkCudaErrors(hipDeviceGetAttribute(&isMemPoolSupported,
                                           hipDeviceAttributeMemoryPoolsSupported, i));

    // Filter unsupported devices
    if (isMemPoolSupported) {
      int major = 0, minor = 0;
      checkCudaErrors(
          hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, i));
      checkCudaErrors(
          hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, i));
      identicalGpus.emplace(std::make_pair(major, minor), i);
    }
  }

  return identicalGpus;
}

std::pair<int, int> getP2PCapableGpuPair() {
  constexpr size_t kNumGpusRequired = 2;

  auto gpusByArch = getIdenticalGPUs();

  auto it = gpusByArch.begin();
  auto end = gpusByArch.end();

  auto bestFit = std::make_pair(it, it);
  // use std::distance to find the largest number of GPUs amongst architectures
  auto distance = [](decltype(bestFit) p) {
    return std::distance(p.first, p.second);
  };

  // Read each unique key/pair element in order
  for (; it != end; it = gpusByArch.upper_bound(it->first)) {
    // first and second are iterators bounded within the architecture group
    auto testFit = gpusByArch.equal_range(it->first);
    // Always use devices with highest architecture version or whichever has the
    // most devices available
    if (distance(bestFit) <= distance(testFit)) bestFit = testFit;
  }

  if (distance(bestFit) < kNumGpusRequired) {
    printf(
        "No Two or more GPUs with same architecture capable of cuda Memory "
        "Pools found."
        "\nWaiving the sample\n");
    exit(EXIT_WAIVED);
  }

  std::set<int> bestFitDeviceIds;

  // check & select peer-to-peer access capable GPU devices.
  int devIds[2];
  for (auto itr = bestFit.first; itr != bestFit.second; itr++) {
    int deviceId = itr->second;
    checkCudaErrors(hipSetDevice(deviceId));

    std::for_each(itr, bestFit.second, [&deviceId, &bestFitDeviceIds,
                                        &kNumGpusRequired](
                                           decltype(*itr) mapPair) {
      if (deviceId != mapPair.second) {
        int access = 0;
        checkCudaErrors(
            hipDeviceCanAccessPeer(&access, deviceId, mapPair.second));
        printf("Device=%d %s Access Peer Device=%d\n", deviceId,
               access ? "CAN" : "CANNOT", mapPair.second);
        if (access && bestFitDeviceIds.size() < kNumGpusRequired) {
          bestFitDeviceIds.emplace(deviceId);
          bestFitDeviceIds.emplace(mapPair.second);
        } else {
          printf("Ignoring device %i (max devices exceeded)\n", mapPair.second);
        }
      }
    });

    if (bestFitDeviceIds.size() >= kNumGpusRequired) {
      printf("Selected p2p capable devices - ");
      int i = 0;
      for (auto devicesItr = bestFitDeviceIds.begin();
           devicesItr != bestFitDeviceIds.end(); devicesItr++) {
        devIds[i++] = *devicesItr;
        printf("deviceId = %d  ", *devicesItr);
      }
      printf("\n");
      break;
    }
  }

  // if bestFitDeviceIds.size() == 0 it means the GPUs in system are not p2p
  // capable, hence we add it without p2p capability check.
  if (!bestFitDeviceIds.size()) {
    printf("No Two or more Devices p2p capable found.. exiting..\n");
    exit(EXIT_WAIVED);
  }

  auto p2pGpuPair = std::make_pair(devIds[0], devIds[1]);

  return p2pGpuPair;
}

int memPoolP2PCopy() {
  int *dev0_srcVec, *dev1_dstVec;  // Device buffers
  hipStream_t stream1, stream2;
  hipMemPool_t memPool;
  hipEvent_t waitOnStream1;

  // Allocate CPU memory.
  size_t nelem = 1048576;
  size_t bytes = nelem * sizeof(int);

  int *a = (int *)malloc(bytes);
  int *output = (int *)malloc(bytes);

  /* Initialize the vectors. */
  for (int n = 0; n < nelem; n++) {
    a[n] = rand() / (int)RAND_MAX;
  }

  auto p2pDevices = getP2PCapableGpuPair();
  printf("selected devices = %d & %d\n", p2pDevices.first, p2pDevices.second);
  checkCudaErrors(hipSetDevice(p2pDevices.first));
  checkCudaErrors(hipEventCreate(&waitOnStream1));

  checkCudaErrors(hipStreamCreateWithFlags(&stream1, hipStreamNonBlocking));

  // Get the default mempool for device p2pDevices.first from the pair
  checkCudaErrors(hipDeviceGetDefaultMemPool(&memPool, p2pDevices.first));

  // Allocate memory in a stream from the pool set above.
  checkCudaErrors(hipMallocAsync(&dev0_srcVec, bytes, stream1));

  checkCudaErrors(
      hipMemcpyAsync(dev0_srcVec, a, bytes, hipMemcpyHostToDevice, stream1));
  checkCudaErrors(hipEventRecord(waitOnStream1, stream1));

  checkCudaErrors(hipSetDevice(p2pDevices.second));
  checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));

  // Allocate memory in p2pDevices.second device
  checkCudaErrors(hipMallocAsync(&dev1_dstVec, bytes, stream2));

  // Setup peer mappings for p2pDevices.second device
  hipMemAccessDesc desc;
  memset(&desc, 0, sizeof(hipMemAccessDesc));
  desc.location.type = hipMemLocationTypeDevice;
  desc.location.id = p2pDevices.second;
  desc.flags = hipMemAccessFlagsProtReadWrite;
  checkCudaErrors(hipMemPoolSetAccess(memPool, &desc, 1));

  printf("> copyP2PAndScale kernel running ...\n");
  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem / (int)block.x));
  checkCudaErrors(hipStreamWaitEvent(stream2, waitOnStream1));
  copyP2PAndScale<<<grid, block, 0, stream2>>>(dev0_srcVec, dev1_dstVec, nelem);

  checkCudaErrors(hipMemcpyAsync(output, dev1_dstVec, bytes,
                                  hipMemcpyDeviceToHost, stream2));
  checkCudaErrors(hipFreeAsync(dev0_srcVec, stream2));
  checkCudaErrors(hipFreeAsync(dev1_dstVec, stream2));
  checkCudaErrors(hipStreamSynchronize(stream2));

  /* Compare the results */
  printf("> Checking the results from copyP2PAndScale() ...\n");

  for (int n = 0; n < nelem; n++) {
    if ((2 * a[n]) != output[n]) {
      printf("mismatch i = %d expected = %d val = %d\n", n, 2 * a[n],
             output[n]);
      return EXIT_FAILURE;
    }
  }

  free(a);
  free(output);
  checkCudaErrors(hipStreamDestroy(stream1));
  checkCudaErrors(hipStreamDestroy(stream2));
  printf("PASSED\n");

  return EXIT_SUCCESS;
}

int main(int argc, char **argv) {
  int ret = memPoolP2PCopy();
  return ret;
}
