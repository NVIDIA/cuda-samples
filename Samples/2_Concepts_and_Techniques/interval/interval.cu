/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Example of program using the interval_gpu<T> template class and operators:
 * Search for roots of a function using an interval Newton method.
  *
 * Use the command-line argument "--n=<N>" to select which GPU implementation to
 * use,
 * otherwise the naive implementation will be used by default.
 * 0: the naive implementation
 * 1: the optimized implementation
 * 2: the recursive implementation
 *
 */

const static char *sSDKsample = "Interval Computing";

#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime_api.h"
#include "interval.h"
#include "cuda_interval.h"
#include "cpu_interval.h"

int main(int argc, char *argv[]) {
  int implementation_choice = 0;

  printf("[%s]  starting ...\n\n", sSDKsample);

  if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
    implementation_choice =
        getCmdLineArgumentInt(argc, (const char **)argv, "n");
  }

  // Pick the best GPU available, or if the developer selects one at the command
  // line
  int devID = findCudaDevice(argc, (const char **)argv);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, devID);
  printf("> GPU Device has Compute Capabilities SM %d.%d\n\n", deviceProp.major,
         deviceProp.minor);

  switch (implementation_choice) {
    case 0:
      printf("GPU naive implementation\n");
      break;

    case 1:
      printf("GPU optimized implementation\n");
      break;

    case 2:
      printf("GPU recursive implementation (requires Compute SM 2.0+)\n");
      break;

    default:
      printf("GPU naive implementation\n");
  }

  interval_gpu<T> *d_result;
  int *d_nresults;
  int *h_nresults = new int[THREADS];
  hipEvent_t start, stop;

  CHECKED_CALL(hipSetDevice(devID));
  CHECKED_CALL(hipMalloc((void **)&d_result,
                          THREADS * DEPTH_RESULT * sizeof(*d_result)));
  CHECKED_CALL(hipMalloc((void **)&d_nresults, THREADS * sizeof(*d_nresults)));
  CHECKED_CALL(hipEventCreate(&start));
  CHECKED_CALL(hipEventCreate(&stop));

  // We need L1 cache to store the stack (only applicable to sm_20 and higher)
  CHECKED_CALL(
      hipFuncSetCacheConfig(reinterpret_cast<const void*>(test_interval_newton<T>), hipFuncCachePreferL1));

  // Increase the stack size large enough for the non-inlined and recursive
  // function calls (only applicable to sm_20 and higher)
  CHECKED_CALL(hipDeviceSetLimit(hipLimitStackSize, 8192));

  interval_gpu<T> i(0.01f, 4.0f);
  std::cout << "Searching for roots in [" << i.lower() << ", " << i.upper()
            << "]...\n";

  CHECKED_CALL(hipEventRecord(start, 0));

  for (int it = 0; it < NUM_RUNS; ++it) {
    test_interval_newton<T><<<GRID_SIZE, BLOCK_SIZE>>>(d_result, d_nresults, i,
                                                       implementation_choice);
    CHECKED_CALL(hipGetLastError());
  }

  CHECKED_CALL(hipEventRecord(stop, 0));
  CHECKED_CALL(hipDeviceSynchronize());

  I_CPU *h_result = new I_CPU[THREADS * DEPTH_RESULT];
  CHECKED_CALL(hipMemcpy(h_result, d_result,
                          THREADS * DEPTH_RESULT * sizeof(*d_result),
                          hipMemcpyDeviceToHost));
  CHECKED_CALL(hipMemcpy(h_nresults, d_nresults, THREADS * sizeof(*d_nresults),
                          hipMemcpyDeviceToHost));

  std::cout << "Found " << h_nresults[0]
            << " intervals that may contain the root(s)\n";
  std::cout.precision(15);

  for (int i = 0; i != h_nresults[0]; ++i) {
    std::cout << " i[" << i << "] ="
              << " [" << h_result[THREADS * i + 0].lower() << ", "
              << h_result[THREADS * i + 0].upper() << "]\n";
  }

  float time;
  CHECKED_CALL(hipEventElapsedTime(&time, start, stop));
  std::cout << "Number of equations solved: " << THREADS << "\n";
  std::cout << "Time per equation: "
            << 1000000.0f * (time / (float)(THREADS)) / NUM_RUNS << " us\n";

  CHECKED_CALL(hipEventDestroy(start));
  CHECKED_CALL(hipEventDestroy(stop));
  CHECKED_CALL(hipFree(d_result));
  CHECKED_CALL(hipFree(d_nresults));

  // Compute the results using a CPU implementation based on the Boost library
  I_CPU i_cpu(0.01f, 4.0f);
  I_CPU *h_result_cpu = new I_CPU[THREADS * DEPTH_RESULT];
  int *h_nresults_cpu = new int[THREADS];
  test_interval_newton_cpu<I_CPU>(h_result_cpu, h_nresults_cpu, i_cpu);

  // Compare the CPU and GPU results
  bool bTestResult =
      checkAgainstHost(h_nresults, h_nresults_cpu, h_result, h_result_cpu);

  delete[] h_result_cpu;
  delete[] h_nresults_cpu;
  delete[] h_result;
  delete[] h_nresults;

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}
