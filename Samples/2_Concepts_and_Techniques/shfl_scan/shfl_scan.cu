#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// Shuffle intrinsics CUDA Sample
// This sample demonstrates the use of the shuffle intrinsic
// First, a simple example of a prefix sum using the shuffle to
// perform a scan operation is provided.
// Secondly, a more involved example of computing an integral image
// using the shuffle intrinsic is provided, where the shuffle
// scan operation and shuffle xor operations are used

#include <stdio.h>

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "shfl_integral_image.cuh"

// Scan using shfl - takes log2(n) steps
// This function demonstrates basic use of the shuffle intrinsic, __shfl_up,
// to perform a scan operation across a block.
// First, it performs a scan (prefix sum in this case) inside a warp
// Then to continue the scan operation across the block,
// each warp's sum is placed into shared memory.  A single warp
// then performs a shuffle scan on that shared memory.  The results
// are then uniformly added to each warp's threads.
// This pyramid type approach is continued by placing each block's
// final sum in global memory and prefix summing that via another kernel call,
// then uniformly adding across the input data via the uniform_add<<<>>> kernel.

__global__ void shfl_scan_test(int *data, int width, int *partial_sums = NULL) {
  extern __shared__ int sums[];
  int id = ((blockIdx.x * blockDim.x) + threadIdx.x);
  int lane_id = id % warpSize;
  // determine a warp_id within a block
  int warp_id = threadIdx.x / warpSize;

  // Below is the basic structure of using a shfl instruction
  // for a scan.
  // Record "value" as a variable - we accumulate it along the way
  int value = data[id];

  // Now accumulate in log steps up the chain
  // compute sums, with another thread's value who is
  // distance delta away (i).  Note
  // those threads where the thread 'i' away would have
  // been out of bounds of the warp are unaffected.  This
  // creates the scan sum.

#pragma unroll
  for (int i = 1; i <= width; i *= 2) {
    unsigned int mask = 0xffffffff;
    int n = __shfl_up_sync(mask, value, i, width);

    if (lane_id >= i) value += n;
  }

  // value now holds the scan value for the individual thread
  // next sum the largest values for each warp

  // write the sum of the warp to smem
  if (threadIdx.x % warpSize == warpSize - 1) {
    sums[warp_id] = value;
  }

  __syncthreads();

  //
  // scan sum the warp sums
  // the same shfl scan operation, but performed on warp sums
  //
  if (warp_id == 0 && lane_id < (blockDim.x / warpSize)) {
    int warp_sum = sums[lane_id];

    int mask = (1 << (blockDim.x / warpSize)) - 1;
    for (int i = 1; i <= (blockDim.x / warpSize); i *= 2) {
      int n = __shfl_up_sync(mask, warp_sum, i, (blockDim.x / warpSize));

      if (lane_id >= i) warp_sum += n;
    }

    sums[lane_id] = warp_sum;
  }

  __syncthreads();

  // perform a uniform add across warps in the block
  // read neighbouring warp's sum and add it to threads value
  int blockSum = 0;

  if (warp_id > 0) {
    blockSum = sums[warp_id - 1];
  }

  value += blockSum;

  // Now write out our result
  data[id] = value;

  // last thread has sum, write write out the block's sum
  if (partial_sums != NULL && threadIdx.x == blockDim.x - 1) {
    partial_sums[blockIdx.x] = value;
  }
}

// Uniform add: add partial sums array
__global__ void uniform_add(int *data, int *partial_sums, int len) {
  __shared__ int buf;
  int id = ((blockIdx.x * blockDim.x) + threadIdx.x);

  if (id > len) return;

  if (threadIdx.x == 0) {
    buf = partial_sums[blockIdx.x];
  }

  __syncthreads();
  data[id] += buf;
}

static unsigned int iDivUp(unsigned int dividend, unsigned int divisor) {
  return ((dividend % divisor) == 0) ? (dividend / divisor)
                                     : (dividend / divisor + 1);
}

// This function verifies the shuffle scan result, for the simple
// prefix sum case.
bool CPUverify(int *h_data, int *h_result, int n_elements) {
  // cpu verify
  for (int i = 0; i < n_elements - 1; i++) {
    h_data[i + 1] = h_data[i] + h_data[i + 1];
  }

  int diff = 0;

  for (int i = 0; i < n_elements; i++) {
    diff += h_data[i] - h_result[i];
  }

  printf("CPU verify result diff (GPUvsCPU) = %d\n", diff);
  bool bTestResult = false;

  if (diff == 0) bTestResult = true;

  StopWatchInterface *hTimer = NULL;
  sdkCreateTimer(&hTimer);
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);

  for (int j = 0; j < 100; j++)
    for (int i = 0; i < n_elements - 1; i++) {
      h_data[i + 1] = h_data[i] + h_data[i + 1];
    }

  sdkStopTimer(&hTimer);
  double cput = sdkGetTimerValue(&hTimer);
  printf("CPU sum (naive) took %f ms\n", cput / 100);
  return bTestResult;
}

// this verifies the row scan result for synthetic data of all 1's
unsigned int verifyDataRowSums(unsigned int *h_image, int w, int h) {
  unsigned int diff = 0;

  for (int j = 0; j < h; j++) {
    for (int i = 0; i < w; i++) {
      int gold = i + 1;
      diff +=
          abs(static_cast<int>(gold) - static_cast<int>(h_image[j * w + i]));
    }
  }

  return diff;
}

bool shuffle_simple_test(int argc, char **argv) {
  int *h_data, *h_partial_sums, *h_result;
  int *d_data, *d_partial_sums;
  const int n_elements = 65536;
  int sz = sizeof(int) * n_elements;
  int cuda_device = 0;

  printf("Starting shfl_scan\n");

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  cuda_device = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDevice(&cuda_device));

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

  // __shfl intrinsic needs SM 3.0 or higher
  if (deviceProp.major < 3) {
    printf("> __shfl() intrinsic requires device SM 3.0+\n");
    printf("> Waiving test.\n");
    exit(EXIT_WAIVED);
  }

  checkCudaErrors(hipHostMalloc(reinterpret_cast<void **>(&h_data),
                                 sizeof(int) * n_elements));
  checkCudaErrors(hipHostMalloc(reinterpret_cast<void **>(&h_result),
                                 sizeof(int) * n_elements));

  // initialize data:
  printf("Computing Simple Sum test\n");
  printf("---------------------------------------------------\n");

  printf("Initialize test data [1, 1, 1...]\n");

  for (int i = 0; i < n_elements; i++) {
    h_data[i] = 1;
  }

  int blockSize = 256;
  int gridSize = n_elements / blockSize;
  int nWarps = blockSize / 32;
  int shmem_sz = nWarps * sizeof(int);
  int n_partialSums = n_elements / blockSize;
  int partial_sz = n_partialSums * sizeof(int);

  printf("Scan summation for %d elements, %d partial sums\n", n_elements,
         n_elements / blockSize);

  int p_blockSize = min(n_partialSums, blockSize);
  int p_gridSize = iDivUp(n_partialSums, p_blockSize);
  printf("Partial summing %d elements with %d blocks of size %d\n",
         n_partialSums, p_gridSize, p_blockSize);

  // initialize a timer
  hipEvent_t start, stop;
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&stop));
  float et = 0;
  float inc = 0;

  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_data), sz));
  checkCudaErrors(
      hipMalloc(reinterpret_cast<void **>(&d_partial_sums), partial_sz));
  checkCudaErrors(hipMemset(d_partial_sums, 0, partial_sz));

  checkCudaErrors(
      hipHostMalloc(reinterpret_cast<void **>(&h_partial_sums), partial_sz));
  checkCudaErrors(hipMemcpy(d_data, h_data, sz, hipMemcpyHostToDevice));

  checkCudaErrors(hipEventRecord(start, 0));
  shfl_scan_test<<<gridSize, blockSize, shmem_sz>>>(d_data, 32, d_partial_sums);
  shfl_scan_test<<<p_gridSize, p_blockSize, shmem_sz>>>(d_partial_sums, 32);
  uniform_add<<<gridSize - 1, blockSize>>>(d_data + blockSize, d_partial_sums,
                                           n_elements);
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&inc, start, stop));
  et += inc;

  checkCudaErrors(hipMemcpy(h_result, d_data, sz, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_partial_sums, d_partial_sums, partial_sz,
                             hipMemcpyDeviceToHost));

  printf("Test Sum: %d\n", h_partial_sums[n_partialSums - 1]);
  printf("Time (ms): %f\n", et);
  printf("%d elements scanned in %f ms -> %f MegaElements/s\n", n_elements, et,
         n_elements / (et / 1000.0f) / 1000000.0f);

  bool bTestResult = CPUverify(h_data, h_result, n_elements);

  checkCudaErrors(hipHostFree(h_data));
  checkCudaErrors(hipHostFree(h_result));
  checkCudaErrors(hipHostFree(h_partial_sums));
  checkCudaErrors(hipFree(d_data));
  checkCudaErrors(hipFree(d_partial_sums));

  return bTestResult;
}

// This function tests creation of an integral image using
// synthetic data, of size 1920x1080 pixels greyscale.
bool shuffle_integral_image_test() {
  char *d_data;
  unsigned int *h_image;
  unsigned int *d_integral_image;
  int w = 1920;
  int h = 1080;
  int n_elements = w * h;
  int sz = sizeof(unsigned int) * n_elements;

  printf("\nComputing Integral Image Test on size %d x %d synthetic data\n", w,
         h);
  printf("---------------------------------------------------\n");
  checkCudaErrors(hipHostMalloc(reinterpret_cast<void **>(&h_image), sz));
  // fill test "image" with synthetic 1's data
  memset(h_image, 0, sz);

  // each thread handles 16 values, use 1 block/row
  int blockSize = iDivUp(w, 16);
  // launch 1 block / row
  int gridSize = h;

  // Create a synthetic image for testing
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_data), sz));
  checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_integral_image),
                             n_elements * sizeof(int) * 4));
  checkCudaErrors(hipMemset(d_data, 1, sz));
  checkCudaErrors(hipMemset(d_integral_image, 0, sz));

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float et = 0;
  unsigned int err;

  // Execute scan line prefix sum kernel, and time it
  hipEventRecord(start);
  shfl_intimage_rows<<<gridSize, blockSize>>>(
      reinterpret_cast<uint4 *>(d_data),
      reinterpret_cast<uint4 *>(d_integral_image));
  hipEventRecord(stop);
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&et, start, stop));
  printf("Method: Fast  Time (GPU Timer): %f ms ", et);

  // verify the scan line results
  checkCudaErrors(
      hipMemcpy(h_image, d_integral_image, sz, hipMemcpyDeviceToHost));
  err = verifyDataRowSums(h_image, w, h);
  printf("Diff = %d\n", err);

  // Execute column prefix sum kernel and time it
  dim3 blockSz(32, 8);
  dim3 testGrid(w / blockSz.x, 1);

  hipEventRecord(start);
  shfl_vertical_shfl<<<testGrid, blockSz>>>((unsigned int *)d_integral_image, w,
                                            h);
  hipEventRecord(stop);
  checkCudaErrors(hipEventSynchronize(stop));
  checkCudaErrors(hipEventElapsedTime(&et, start, stop));
  printf("Method: Vertical Scan  Time (GPU Timer): %f ms ", et);

  // Verify the column results
  checkCudaErrors(
      hipMemcpy(h_image, d_integral_image, sz, hipMemcpyDeviceToHost));
  printf("\n");

  int finalSum = h_image[w * h - 1];
  printf("CheckSum: %d, (expect %dx%d=%d)\n", finalSum, w, h, w * h);

  checkCudaErrors(hipFree(d_data));
  checkCudaErrors(hipFree(d_integral_image));
  checkCudaErrors(hipHostFree(h_image));
  // verify final sum: if the final value in the corner is the same as the size
  // of the buffer (all 1's) then the integral image was generated successfully
  return (finalSum == w * h) ? true : false;
}

int main(int argc, char *argv[]) {
  // Initialization.  The shuffle intrinsic is not available on SM < 3.0
  // so waive the test if the hardware is not present.
  int cuda_device = 0;

  printf("Starting shfl_scan\n");

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  cuda_device = findCudaDevice(argc, (const char **)argv);

  hipDeviceProp_t deviceProp;
  checkCudaErrors(hipGetDevice(&cuda_device));

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, cuda_device));

  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
         deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

  // __shfl intrinsic needs SM 3.0 or higher
  if (deviceProp.major < 3) {
    printf("> __shfl() intrinsic requires device SM 3.0+\n");
    printf("> Waiving test.\n");
    exit(EXIT_WAIVED);
  }

  bool bTestResult = true;
  bool simpleTest = shuffle_simple_test(argc, argv);
  bool intTest = shuffle_integral_image_test();

  bTestResult = simpleTest & intTest;

  exit((bTestResult) ? EXIT_SUCCESS : EXIT_FAILURE);
}
