#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Computation of eigenvalues of symmetric, tridiagonal matrix using
 * bisection.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <assert.h>

// includes, project
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "config.h"
#include "structs.h"
#include "matlab.h"
#include "util.h"
#include "gerschgorin.h"

#include "bisect_small.cuh"
#include "bisect_large.cuh"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  bool bQAResults = false;

  printf("Starting eigenvalues\n");

  bQAResults = runTest(argc, argv);
  printf("Test %s\n", bQAResults ? "Succeeded!" : "Failed!");

  exit(bQAResults ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize the input data to the algorithm
//! @param input  handles to the input data
//! @param exec_path  path where executable is run (argv[0])
//! @param mat_size  size of the matrix
//! @param user_defined  1 if the matrix size has been requested by the user,
//!                      0 if the default size
////////////////////////////////////////////////////////////////////////////////
void initInputData(InputData &input, char *exec_path,
                   const unsigned int mat_size,
                   const unsigned int user_defined) {
  // allocate memory
  input.a = (float *)malloc(sizeof(float) * mat_size);
  input.b = (float *)malloc(sizeof(float) * mat_size);

  if (1 == user_defined) {
    // initialize diagonal and superdiagonal entries with random values
    srand(278217421);

    // srand( clock());
    for (unsigned int i = 0; i < mat_size; ++i) {
      input.a[i] = (float)(2.0 * (((double)rand() / (double)RAND_MAX) - 0.5));
      input.b[i] = (float)(2.0 * (((double)rand() / (double)RAND_MAX) - 0.5));
    }

    // the first element of s is used as padding on the device (thus the
    // whole vector is copied to the device but the kernels are launched
    // with (s+1) as start address
    input.b[0] = 0.0f;
  } else {
    // read default matrix
    unsigned int input_data_size = mat_size;
    char *diag_path = sdkFindFilePath("diagonal.dat", exec_path);
    assert(NULL != diag_path);
    sdkReadFile(diag_path, &(input.a), &input_data_size, false);

    char *sdiag_path = sdkFindFilePath("superdiagonal.dat", exec_path);
    assert(NULL != sdiag_path);
    sdkReadFile(sdiag_path, &(input.b), &input_data_size, false);

    free(diag_path);
    free(sdiag_path);
  }

  // allocate device memory for input
  checkCudaErrors(hipMalloc((void **)&(input.g_a), sizeof(float) * mat_size));
  checkCudaErrors(
      hipMalloc((void **)&(input.g_b_raw), sizeof(float) * mat_size));

  // copy data to device
  checkCudaErrors(hipMemcpy(input.g_a, input.a, sizeof(float) * mat_size,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(input.g_b_raw, input.b, sizeof(float) * mat_size,
                             hipMemcpyHostToDevice));

  input.g_b = input.g_b_raw + 1;
}

////////////////////////////////////////////////////////////////////////////////
//! Clean up input data, in particular allocated memory
//! @param input  handles to the input data
////////////////////////////////////////////////////////////////////////////////
void cleanupInputData(InputData &input) {
  freePtr(input.a);
  freePtr(input.b);

  checkCudaErrors(hipFree(input.g_a));
  input.g_a = NULL;
  checkCudaErrors(hipFree(input.g_b_raw));
  input.g_b_raw = NULL;
  input.g_b = NULL;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a specific matrix size has to be used
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param matrix_size  size of matrix, updated if specific size specified on
//!                     command line
////////////////////////////////////////////////////////////////////////////////
void getMatrixSize(int argc, char **argv, unsigned int &mat_size,
                   unsigned int &user_defined) {
  int temp = -1;

  if (checkCmdLineFlag(argc, (const char **)argv, "matrix-size")) {
    temp = getCmdLineArgumentInt(argc, (const char **)argv, "matrix-size");
  }

  if (temp > 0) {
    mat_size = (unsigned int)temp;
    // data type short is used in the kernel
    assert(mat_size < (1 << 16));

    // mat_size should be large than 2
    assert(mat_size >= 2);

    user_defined = 1;
  }

  printf("Matrix size: %i x %i\n", mat_size, mat_size);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a specific precision of the eigenvalue has to be obtained
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param iters_timing  numbers of iterations for timing, updated if a
//!                      specific number is specified on the command line
//! @param user_defined  1 if the precision has been requested by the user,
//!                      0 if the default size
////////////////////////////////////////////////////////////////////////////////
void getPrecision(int argc, char **argv, float &precision,
                  unsigned int &user_defined) {
  float temp = -1.0f;

  if (checkCmdLineFlag(argc, (const char **)argv, "precision")) {
    temp = getCmdLineArgumentFloat(argc, (const char **)argv, "precision");
    printf("Precision is between [0.001, 0.000001]\n");
  }

  if (temp > 1e-6 && temp <= 0.001) {
    precision = temp;
    user_defined = 1;
  }

  printf("Precision: %f\n", precision);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a particular number of iterations for timings has to be used
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param  iters_timing  number of timing iterations, updated if user
//!                       specific value
////////////////////////////////////////////////////////////////////////////////
void getItersTiming(int argc, char **argv, unsigned int &iters_timing) {
  int temp = -1;

  if (checkCmdLineFlag(argc, (const char **)argv, "iters-timing")) {
    temp = getCmdLineArgumentInt(argc, (const char **)argv, "iters-timing");
  }

  if (temp > 0) {
    iters_timing = temp;
  }

  printf("Iterations to be timed: %i\n", iters_timing);
}

////////////////////////////////////////////////////////////////////////////////
//! Check if a particular filename has to be used for the file where the result
//! is stored
//! @param argc  number of command line arguments (from main(argc, argv)
//! @param argv  pointers to command line arguments (from main(argc, argv)
//! @param  filename  filename of result file, updated if user specified
//!                   filename
////////////////////////////////////////////////////////////////////////////////
void getResultFilename(int argc, char **argv, char *&filename) {
  char *temp = NULL;
  getCmdLineArgumentString(argc, (const char **)argv, "filename-result", &temp);

  if (NULL != temp) {
    filename = (char *)malloc(sizeof(char) * strlen(temp));
    strcpy(filename, temp);

    free(temp);
  }

  printf("Result filename: '%s'\n", filename);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv) {
  bool bCompareResult = false;

  findCudaDevice(argc, (const char **)argv);

  StopWatchInterface *timer = NULL;
  StopWatchInterface *timer_total = NULL;
  sdkCreateTimer(&timer);
  sdkCreateTimer(&timer_total);

  // default
  unsigned int mat_size = 2048;
  // flag if the matrix size is due to explicit user request
  unsigned int user_defined = 0;
  // desired precision of eigenvalues
  float precision = 0.00001f;
  unsigned int iters_timing = 100;
  char *result_file = (char *)"eigenvalues.dat";

  // check if there is a command line request for the matrix size
  getMatrixSize(argc, argv, mat_size, user_defined);

  // check if user requested specific precision
  getPrecision(argc, argv, precision, user_defined);

  // check if user requested specific number of iterations for timing
  getItersTiming(argc, argv, iters_timing);

  // file name for result file
  getResultFilename(argc, argv, result_file);

  // set up input
  InputData input;
  initInputData(input, argv[0], mat_size, user_defined);

  // compute Gerschgorin interval
  float lg = FLT_MAX;
  float ug = -FLT_MAX;
  computeGerschgorin(input.a, input.b + 1, mat_size, lg, ug);
  printf("Gerschgorin interval: %f / %f\n", lg, ug);

  // two kernels, for small matrices a lot of overhead can be avoided
  if (mat_size <= MAX_SMALL_MATRIX) {
    // initialize memory for result
    ResultDataSmall result;
    initResultSmallMatrix(result, mat_size);

    // run the kernel
    computeEigenvaluesSmallMatrix(input, result, mat_size, lg, ug, precision,
                                  iters_timing);

    // get the result from the device and do some sanity checks,
    // save the result
    processResultSmallMatrix(input, result, mat_size, result_file);

    // clean up
    cleanupResultSmallMatrix(result);

    printf("User requests non-default argument(s), skipping self-check!\n");
    bCompareResult = true;
  } else {
    // initialize memory for result
    ResultDataLarge result;
    initResultDataLargeMatrix(result, mat_size);

    // run the kernel
    computeEigenvaluesLargeMatrix(input, result, mat_size, precision, lg, ug,
                                  iters_timing);

    // get the result from the device and do some sanity checks
    // save the result if user specified matrix size
    bCompareResult = processResultDataLargeMatrix(
        input, result, mat_size, result_file, user_defined, argv[0]);

    // cleanup
    cleanupResultDataLargeMatrix(result);
  }

  cleanupInputData(input);

  return bCompareResult;
}
