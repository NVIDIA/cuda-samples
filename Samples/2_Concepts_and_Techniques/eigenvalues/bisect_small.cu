/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Computation of eigenvalues of a small symmetric, tridiagonal matrix */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// includes, project
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"
#include "config.h"
#include "structs.h"
#include "matlab.h"

// includes, kernels
#include "bisect_kernel_small.cuh"

// includes, file
#include "bisect_small.cuh"

////////////////////////////////////////////////////////////////////////////////
//! Determine eigenvalues for matrices smaller than MAX_SMALL_MATRIX
//! @param TimingIterations  number of iterations for timing
//! @param  input  handles to input data of kernel
//! @param  result handles to result of kernel
//! @param  mat_size  matrix size
//! @param  lg  lower limit of Gerschgorin interval
//! @param  ug  upper limit of Gerschgorin interval
//! @param  precision  desired precision of eigenvalues
//! @param  iterations  number of iterations for timing
////////////////////////////////////////////////////////////////////////////////
void computeEigenvaluesSmallMatrix(const InputData &input,
                                   ResultDataSmall &result,
                                   const unsigned int mat_size, const float lg,
                                   const float ug, const float precision,
                                   const unsigned int iterations) {
  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);
  sdkStartTimer(&timer);

  for (unsigned int i = 0; i < iterations; ++i) {
    dim3 blocks(1, 1, 1);
    dim3 threads(MAX_THREADS_BLOCK_SMALL_MATRIX, 1, 1);

    bisectKernel<<<blocks, threads>>>(input.g_a, input.g_b, mat_size,
                                      result.g_left, result.g_right,
                                      result.g_left_count, result.g_right_count,
                                      lg, ug, 0, mat_size, precision);
  }

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timer);
  getLastCudaError("Kernel launch failed");
  printf("Average time: %f ms (%i iterations)\n",
         sdkGetTimerValue(&timer) / (float)iterations, iterations);

  sdkDeleteTimer(&timer);
}

////////////////////////////////////////////////////////////////////////////////
//! Initialize variables and memory for the result for small matrices
//! @param result  handles to the necessary memory
//! @param  mat_size  matrix_size
////////////////////////////////////////////////////////////////////////////////
void initResultSmallMatrix(ResultDataSmall &result,
                           const unsigned int mat_size) {
  result.mat_size_f = sizeof(float) * mat_size;
  result.mat_size_ui = sizeof(unsigned int) * mat_size;

  result.eigenvalues = (float *)malloc(result.mat_size_f);

  // helper variables
  result.zero_f = (float *)malloc(result.mat_size_f);
  result.zero_ui = (unsigned int *)malloc(result.mat_size_ui);

  for (unsigned int i = 0; i < mat_size; ++i) {
    result.zero_f[i] = 0.0f;
    result.zero_ui[i] = 0;

    result.eigenvalues[i] = 0.0f;
  }

  checkCudaErrors(hipMalloc((void **)&result.g_left, result.mat_size_f));
  checkCudaErrors(hipMalloc((void **)&result.g_right, result.mat_size_f));

  checkCudaErrors(
      hipMalloc((void **)&result.g_left_count, result.mat_size_ui));
  checkCudaErrors(
      hipMalloc((void **)&result.g_right_count, result.mat_size_ui));

  // initialize result memory
  checkCudaErrors(hipMemcpy(result.g_left, result.zero_f, result.mat_size_f,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(result.g_right, result.zero_f, result.mat_size_f,
                             hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(result.g_right_count, result.zero_ui,
                             result.mat_size_ui, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(result.g_left_count, result.zero_ui,
                             result.mat_size_ui, hipMemcpyHostToDevice));
}

////////////////////////////////////////////////////////////////////////////////
//! Cleanup memory and variables for result for small matrices
//! @param  result  handle to variables
////////////////////////////////////////////////////////////////////////////////
void cleanupResultSmallMatrix(ResultDataSmall &result) {
  freePtr(result.eigenvalues);
  freePtr(result.zero_f);
  freePtr(result.zero_ui);

  checkCudaErrors(hipFree(result.g_left));
  checkCudaErrors(hipFree(result.g_right));
  checkCudaErrors(hipFree(result.g_left_count));
  checkCudaErrors(hipFree(result.g_right_count));
}

////////////////////////////////////////////////////////////////////////////////
//! Process the result obtained on the device, that is transfer to host and
//! perform basic sanity checking
//! @param  input  handles to input data
//! @param  result  handles to result data
//! @param  mat_size   matrix size
//! @param  filename  output filename
////////////////////////////////////////////////////////////////////////////////
void processResultSmallMatrix(const InputData &input,
                              const ResultDataSmall &result,
                              const unsigned int mat_size,
                              const char *filename) {
  const unsigned int mat_size_f = sizeof(float) * mat_size;
  const unsigned int mat_size_ui = sizeof(unsigned int) * mat_size;

  // copy data back to host
  float *left = (float *)malloc(mat_size_f);
  unsigned int *left_count = (unsigned int *)malloc(mat_size_ui);

  checkCudaErrors(
      hipMemcpy(left, result.g_left, mat_size_f, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(left_count, result.g_left_count, mat_size_ui,
                             hipMemcpyDeviceToHost));

  float *eigenvalues = (float *)malloc(mat_size_f);

  for (unsigned int i = 0; i < mat_size; ++i) {
    eigenvalues[left_count[i]] = left[i];
  }

  // save result in matlab format
  writeTridiagSymMatlab(filename, input.a, input.b + 1, eigenvalues, mat_size);

  freePtr(left);
  freePtr(left_count);
  freePtr(eigenvalues);
}
