#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/* Utility / shared functionality for bisection kernels */

#ifndef _BISECT_UTIL_H_
#define _BISECT_UTIL_H_

#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

// includes, project
#include "config.h"
#include "util.h"

////////////////////////////////////////////////////////////////////////////////
//! Compute the next lower power of two of n
//! @param  n  number for which next higher power of two is sought
////////////////////////////////////////////////////////////////////////////////
__device__ inline int floorPow2(int n) {
  // early out if already power of two
  if (0 == (n & (n - 1))) {
    return n;
  }

  int exp;
  frexp((float)n, &exp);
  return (1 << (exp - 1));
}

////////////////////////////////////////////////////////////////////////////////
//! Compute the next higher power of two of n
//! @param  n  number for which next higher power of two is sought
////////////////////////////////////////////////////////////////////////////////
__device__ inline int ceilPow2(int n) {
  // early out if already power of two
  if (0 == (n & (n - 1))) {
    return n;
  }

  int exp;
  frexp((float)n, &exp);
  return (1 << exp);
}

////////////////////////////////////////////////////////////////////////////////
//! Compute midpoint of interval [\a left, \a right] avoiding overflow if
//! possible
//! @param left   left / lower limit of interval
//! @param right  right / upper limit of interval
////////////////////////////////////////////////////////////////////////////////
__device__ inline float computeMidpoint(const float left, const float right) {
  float mid;

  if (sign_f(left) == sign_f(right)) {
    mid = left + (right - left) * 0.5f;
  } else {
    mid = (left + right) * 0.5f;
  }

  return mid;
}

////////////////////////////////////////////////////////////////////////////////
//! Check if interval converged and store appropriately
//! @param  addr    address where to store the information of the interval
//! @param  s_left  shared memory storage for left interval limits
//! @param  s_right  shared memory storage for right interval limits
//! @param  s_left_count  shared memory storage for number of eigenvalues less
//!                       than left interval limits
//! @param  s_right_count  shared memory storage for number of eigenvalues less
//!                       than right interval limits
//! @param  left   lower limit of interval
//! @param  right  upper limit of interval
//! @param  left_count  eigenvalues less than \a left
//! @param  right_count  eigenvalues less than \a right
//! @param  precision  desired precision for eigenvalues
////////////////////////////////////////////////////////////////////////////////
template <class S, class T>
__device__ void storeInterval(unsigned int addr, float *s_left, float *s_right,
                              T *s_left_count, T *s_right_count, float left,
                              float right, S left_count, S right_count,
                              float precision) {
  s_left_count[addr] = left_count;
  s_right_count[addr] = right_count;

  // check if interval converged
  float t0 = abs(right - left);
  float t1 = max(abs(left), abs(right)) * precision;

  if (t0 <= max(MIN_ABS_INTERVAL, t1)) {
    // compute mid point
    float lambda = computeMidpoint(left, right);

    // mark as converged
    s_left[addr] = lambda;
    s_right[addr] = lambda;
  } else {
    // store current limits
    s_left[addr] = left;
    s_right[addr] = right;
  }
}

////////////////////////////////////////////////////////////////////////////////
//! Compute number of eigenvalues that are smaller than x given a symmetric,
//! real, and tridiagonal matrix
//! @param  g_d  diagonal elements stored in global memory
//! @param  g_s  superdiagonal elements stored in global memory
//! @param  n    size of matrix
//! @param  x    value for which the number of eigenvalues that are smaller is
//!              seeked
//! @param  tid  thread identified (e.g. threadIdx.x or gtid)
//! @param  num_intervals_active  number of active intervals / threads that
//!                               currently process an interval
//! @param  s_d  scratch space to store diagonal entries of the tridiagonal
//!              matrix in shared memory
//! @param  s_s  scratch space to store superdiagonal entries of the tridiagonal
//!              matrix in shared memory
//! @param  converged  flag if the current thread is already converged (that
//!         is count does not have to be computed)
////////////////////////////////////////////////////////////////////////////////
__device__ inline unsigned int computeNumSmallerEigenvals(
    float *g_d, float *g_s, const unsigned int n, const float x,
    const unsigned int tid, const unsigned int num_intervals_active, float *s_d,
    float *s_s, unsigned int converged, cg::thread_block cta) {
  float delta = 1.0f;
  unsigned int count = 0;

  cg::sync(cta);

  // read data into shared memory
  if (threadIdx.x < n) {
    s_d[threadIdx.x] = *(g_d + threadIdx.x);
    s_s[threadIdx.x] = *(g_s + threadIdx.x - 1);
  }

  cg::sync(cta);

  // perform loop only for active threads
  if ((tid < num_intervals_active) && (0 == converged)) {
    // perform (optimized) Gaussian elimination to determine the number
    // of eigenvalues that are smaller than n
    for (unsigned int k = 0; k < n; ++k) {
      delta = s_d[k] - x - (s_s[k] * s_s[k]) / delta;
      count += (delta < 0) ? 1 : 0;
    }

  }  // end if thread currently processing an interval

  return count;
}

////////////////////////////////////////////////////////////////////////////////
//! Compute number of eigenvalues that are smaller than x given a symmetric,
//! real, and tridiagonal matrix
//! @param  g_d  diagonal elements stored in global memory
//! @param  g_s  superdiagonal elements stored in global memory
//! @param  n    size of matrix
//! @param  x    value for which the number of eigenvalues that are smaller is
//!              seeked
//! @param  tid  thread identified (e.g. threadIdx.x or gtid)
//! @param  num_intervals_active  number of active intervals / threads that
//!                               currently process an interval
//! @param  s_d  scratch space to store diagonal entries of the tridiagonal
//!              matrix in shared memory
//! @param  s_s  scratch space to store superdiagonal entries of the tridiagonal
//!              matrix in shared memory
//! @param  converged  flag if the current thread is already converged (that
//!         is count does not have to be computed)
////////////////////////////////////////////////////////////////////////////////
__device__ inline unsigned int computeNumSmallerEigenvalsLarge(
    float *g_d, float *g_s, const unsigned int n, const float x,
    const unsigned int tid, const unsigned int num_intervals_active, float *s_d,
    float *s_s, unsigned int converged, cg::thread_block cta) {
  float delta = 1.0f;
  unsigned int count = 0;

  unsigned int rem = n;

  // do until whole diagonal and superdiagonal has been loaded and processed
  for (unsigned int i = 0; i < n; i += blockDim.x) {
    cg::sync(cta);

    // read new chunk of data into shared memory
    if ((i + threadIdx.x) < n) {
      s_d[threadIdx.x] = *(g_d + i + threadIdx.x);
      s_s[threadIdx.x] = *(g_s + i + threadIdx.x - 1);
    }

    cg::sync(cta);

    if (tid < num_intervals_active) {
      // perform (optimized) Gaussian elimination to determine the number
      // of eigenvalues that are smaller than n
      for (unsigned int k = 0; k < min(rem, blockDim.x); ++k) {
        delta = s_d[k] - x - (s_s[k] * s_s[k]) / delta;
        // delta = (abs( delta) < (1.0e-10)) ? -(1.0e-10) : delta;
        count += (delta < 0) ? 1 : 0;
      }

    }  // end if thread currently processing an interval

    rem -= blockDim.x;
  }

  return count;
}

////////////////////////////////////////////////////////////////////////////////
//! Store all non-empty intervals resulting from the subdivision of the interval
//! currently processed by the thread
//! @param  addr  base address for storing intervals
//! @param  num_threads_active  number of threads / intervals in current sweep
//! @param  s_left  shared memory storage for left interval limits
//! @param  s_right  shared memory storage for right interval limits
//! @param  s_left_count  shared memory storage for number of eigenvalues less
//!                       than left interval limits
//! @param  s_right_count  shared memory storage for number of eigenvalues less
//!                       than right interval limits
//! @param  left   lower limit of interval
//! @param  mid    midpoint of interval
//! @param  right  upper limit of interval
//! @param  left_count  eigenvalues less than \a left
//! @param  mid_count  eigenvalues less than \a mid
//! @param  right_count  eigenvalues less than \a right
//! @param  precision  desired precision for eigenvalues
//! @param  compact_second_chunk  shared mem flag if second chunk is used and
//!                               ergo requires compaction
//! @param  s_compaction_list_exc  helper array for stream compaction,
//!                                s_compaction_list_exc[tid] = 1 when the
//!                                thread generated two child intervals
//! @is_active_interval  mark is thread has a second non-empty child interval
////////////////////////////////////////////////////////////////////////////////
template <class S, class T>
__device__ void storeNonEmptyIntervals(
    unsigned int addr, const unsigned int num_threads_active, float *s_left,
    float *s_right, T *s_left_count, T *s_right_count, float left, float mid,
    float right, const S left_count, const S mid_count, const S right_count,
    float precision, unsigned int &compact_second_chunk,
    T *s_compaction_list_exc, unsigned int &is_active_second) {
  // check if both child intervals are valid
  if ((left_count != mid_count) && (mid_count != right_count)) {
    // store the left interval
    storeInterval(addr, s_left, s_right, s_left_count, s_right_count, left, mid,
                  left_count, mid_count, precision);

    // mark that a second interval has been generated, only stored after
    // stream compaction of second chunk
    is_active_second = 1;
    s_compaction_list_exc[threadIdx.x] = 1;
    atomicExch(&compact_second_chunk, 1);
  } else {
    // only one non-empty child interval

    // mark that no second child
    is_active_second = 0;
    s_compaction_list_exc[threadIdx.x] = 0;

    // store the one valid child interval
    if (left_count != mid_count) {
      storeInterval(addr, s_left, s_right, s_left_count, s_right_count, left,
                    mid, left_count, mid_count, precision);
    } else {
      storeInterval(addr, s_left, s_right, s_left_count, s_right_count, mid,
                    right, mid_count, right_count, precision);
    }
  }
}
////////////////////////////////////////////////////////////////////////////////
//! Create indices for compaction, that is process \a s_compaction_list_exc
//! which is 1 for intervals that generated a second child and 0 otherwise
//! and create for each of the non-zero elements the index where the new
//! interval belongs to in a compact representation of all generated second
//! childs
//! @param   s_compaction_list_exc  list containing the flags which threads
//!                                 generated two children
//! @param   num_threads_compaction number of threads to employ for compaction
////////////////////////////////////////////////////////////////////////////////
template <class T>
__device__ void createIndicesCompaction(T *s_compaction_list_exc,
                                        unsigned int num_threads_compaction,
                                        cg::thread_block cta) {
  unsigned int offset = 1;
  const unsigned int tid = threadIdx.x;

  // higher levels of scan tree
  for (int d = (num_threads_compaction >> 1); d > 0; d >>= 1) {
    cg::sync(cta);

    if (tid < d) {
      unsigned int ai = offset * (2 * tid + 1) - 1;
      unsigned int bi = offset * (2 * tid + 2) - 1;

      s_compaction_list_exc[bi] =
          s_compaction_list_exc[bi] + s_compaction_list_exc[ai];
    }

    offset <<= 1;
  }

  // traverse down tree: first down to level 2 across
  for (int d = 2; d < num_threads_compaction; d <<= 1) {
    offset >>= 1;
    cg::sync(cta);

    if (tid < (d - 1)) {
      unsigned int ai = offset * (tid + 1) - 1;
      unsigned int bi = ai + (offset >> 1);

      s_compaction_list_exc[bi] =
          s_compaction_list_exc[bi] + s_compaction_list_exc[ai];
    }
  }

  cg::sync(cta);
}

///////////////////////////////////////////////////////////////////////////////
//! Perform stream compaction for second child intervals
//! @param  s_left  shared
//! @param  s_left  shared memory storage for left interval limits
//! @param  s_right  shared memory storage for right interval limits
//! @param  s_left_count  shared memory storage for number of eigenvalues less
//!                       than left interval limits
//! @param  s_right_count  shared memory storage for number of eigenvalues less
//!                       than right interval limits
//! @param  mid    midpoint of current interval (left of new interval)
//! @param  right  upper limit of interval
//! @param  mid_count  eigenvalues less than \a mid
//! @param  s_compaction_list  list containing the indices where the data has
//!         to be stored
//! @param  num_threads_active  number of active threads / intervals
//! @is_active_interval  mark is thread has a second non-empty child interval
///////////////////////////////////////////////////////////////////////////////
template <class T>
__device__ void compactIntervals(float *s_left, float *s_right, T *s_left_count,
                                 T *s_right_count, float mid, float right,
                                 unsigned int mid_count,
                                 unsigned int right_count, T *s_compaction_list,
                                 unsigned int num_threads_active,
                                 unsigned int is_active_second) {
  const unsigned int tid = threadIdx.x;

  // perform compaction / copy data for all threads where the second
  // child is not dead
  if ((tid < num_threads_active) && (1 == is_active_second)) {
    unsigned int addr_w = num_threads_active + s_compaction_list[tid];

    s_left[addr_w] = mid;
    s_right[addr_w] = right;
    s_left_count[addr_w] = mid_count;
    s_right_count[addr_w] = right_count;
  }
}

///////////////////////////////////////////////////////////////////////////////
//! Store intervals that have already converged (w.r.t. the desired precision),
//! duplicating intervals that contain multiple eigenvalues
//! @param  s_left  shared memory storage for left interval limits
//! @param  s_right  shared memory storage for right interval limits
//! @param  s_left_count  shared memory storage for number of eigenvalues less
//!                       than left interval limits
//! @param  s_right_count  shared memory storage for number of eigenvalues less
//!                       than right interval limits
//! @param  left   lower limit of interval
//! @param  mid    midpoint of interval (updated if split is necessary)
//! @param  right  upper limit of interval
//! @param  left_count  eigenvalues less than \a left
//! @param  mid_count  eigenvalues less than \a mid
//! @param  right_count  eigenvalues less than \a right
//! @param  s_compaction_list_exc  helper array for stream compaction, updated
//!                                at tid if split is necessary
//! @param  compact_second_chunk  shared mem flag if second chunk is used and
//!                               ergo requires compaction
//! @param  num_threads_active  number of active threads / intervals
///////////////////////////////////////////////////////////////////////////////
template <class T, class S>
__device__ void storeIntervalConverged(float *s_left, float *s_right,
                                       T *s_left_count, T *s_right_count,
                                       float &left, float &mid, float &right,
                                       S &left_count, S &mid_count,
                                       S &right_count, T *s_compaction_list_exc,
                                       unsigned int &compact_second_chunk,
                                       const unsigned int num_threads_active) {
  const unsigned int tid = threadIdx.x;
  const unsigned int multiplicity = right_count - left_count;

  // check multiplicity of eigenvalue
  if (1 == multiplicity) {
    // just re-store intervals, simple eigenvalue
    s_left[tid] = left;
    s_right[tid] = right;
    s_left_count[tid] = left_count;
    s_right_count[tid] = right_count;

    // mark that no second child / clear
    s_right_count[tid + num_threads_active] = 0;
    s_compaction_list_exc[tid] = 0;
  } else {
    // number of eigenvalues after the split less than mid
    mid_count = left_count + (multiplicity >> 1);

    // store left interval
    s_left[tid] = left;
    s_right[tid] = right;
    s_left_count[tid] = left_count;
    s_right_count[tid] = mid_count;

    mid = left;

    // mark that second child interval exists
    s_right_count[tid + num_threads_active] = right_count;
    s_compaction_list_exc[tid] = 1;
    compact_second_chunk = 1;
  }
}

template <class T, class S>
__device__ void storeIntervalConverged(float *s_left, float *s_right,
                                       T *s_left_count, T *s_right_count,
                                       float &left, float &mid, float &right,
                                       S &left_count, S &mid_count,
                                       S &right_count, T *s_compaction_list_exc,
                                       unsigned int &compact_second_chunk,
                                       const unsigned int num_threads_active,
                                       unsigned int &is_active_second) {
  const unsigned int tid = threadIdx.x;
  const unsigned int multiplicity = right_count - left_count;

  // check multiplicity of eigenvalue
  if (1 == multiplicity) {
    // just re-store intervals, simple eigenvalue
    s_left[tid] = left;
    s_right[tid] = right;
    s_left_count[tid] = left_count;
    s_right_count[tid] = right_count;

    // mark that no second child / clear
    is_active_second = 0;
    s_compaction_list_exc[tid] = 0;
  } else {
    // number of eigenvalues after the split less than mid
    mid_count = left_count + (multiplicity >> 1);

    // store left interval
    s_left[tid] = left;
    s_right[tid] = right;
    s_left_count[tid] = left_count;
    s_right_count[tid] = mid_count;

    mid = left;

    // mark that second child interval exists
    is_active_second = 1;
    s_compaction_list_exc[tid] = 1;
    compact_second_chunk = 1;
  }
}

///////////////////////////////////////////////////////////////////////////////
//! Subdivide interval if active and not already converged
//! @param tid  id of thread
//! @param  s_left  shared memory storage for left interval limits
//! @param  s_right  shared memory storage for right interval limits
//! @param  s_left_count  shared memory storage for number of eigenvalues less
//!                       than left interval limits
//! @param  s_right_count  shared memory storage for number of eigenvalues less
//!                       than right interval limits
//! @param  num_threads_active  number of active threads in warp
//! @param  left   lower limit of interval
//! @param  right  upper limit of interval
//! @param  left_count  eigenvalues less than \a left
//! @param  right_count  eigenvalues less than \a right
//! @param  all_threads_converged  shared memory flag if all threads are
//!                                 converged
///////////////////////////////////////////////////////////////////////////////
template <class T>
__device__ void subdivideActiveInterval(
    const unsigned int tid, float *s_left, float *s_right, T *s_left_count,
    T *s_right_count, const unsigned int num_threads_active, float &left,
    float &right, unsigned int &left_count, unsigned int &right_count,
    float &mid, unsigned int &all_threads_converged) {
  // for all active threads
  if (tid < num_threads_active) {
    left = s_left[tid];
    right = s_right[tid];
    left_count = s_left_count[tid];
    right_count = s_right_count[tid];

    // check if thread already converged
    if (left != right) {
      mid = computeMidpoint(left, right);
      atomicExch(&all_threads_converged, 0);
    } else if ((right_count - left_count) > 1) {
      // mark as not converged if multiple eigenvalues enclosed
      // duplicate interval in storeIntervalsConverged()
      atomicExch(&all_threads_converged, 0);
    }

  }  // end for all active threads
}

#endif  // #ifndef _BISECT_UTIL_H_
