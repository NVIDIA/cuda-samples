/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/**
**************************************************************************
* \file dct8x8.cu
* \brief Contains entry point, wrappers to host and device code and benchmark.
*
* This sample implements forward and inverse Discrete Cosine Transform to blocks
* of image pixels (of 8x8 size), as in JPEG standard. The typical work flow is
*as
* follows:
* 1. Run CPU version (Host code) and measure execution time;
* 2. Run CUDA version (Device code) and measure execution time;
* 3. Output execution timings and calculate CUDA speedup.
*/

#include "Common.h"
#include "DCT8x8_Gold.h"
#include "BmpUtil.h"

/**
*  The number of DCT kernel calls
*/
#define BENCHMARK_SIZE 10

/**
*  The PSNR values over this threshold indicate images equality
*/
#define PSNR_THRESHOLD_EQUAL 40

// includes kernels
#include "dct8x8_kernel1.cuh"
#include "dct8x8_kernel2.cuh"
#include "dct8x8_kernel_short.cuh"
#include "dct8x8_kernel_quantization.cuh"

/**
**************************************************************************
*  Wrapper function for 1st gold version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperGold1(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // allocate float buffers for DCT and other data
  int StrideF;
  float *ImgF1 = MallocPlaneFloat(Size.width, Size.height, &StrideF);
  float *ImgF2 = MallocPlaneFloat(Size.width, Size.height, &StrideF);

  // convert source image to float representation
  CopyByte2Float(ImgSrc, Stride, ImgF1, StrideF, Size);
  AddFloatPlane(-128.0f, ImgF1, StrideF, Size);

  // create and start CUDA timer
  StopWatchInterface *timerGold = 0;
  sdkCreateTimer(&timerGold);
  sdkResetTimer(&timerGold);

  // perform block-wise DCT processing and benchmarking
  for (int i = 0; i < BENCHMARK_SIZE; i++) {
    sdkStartTimer(&timerGold);
    computeDCT8x8Gold1(ImgF1, ImgF2, StrideF, Size);
    sdkStopTimer(&timerGold);
  }

  // stop and destroy CUDA timer
  float TimerGoldSpan = sdkGetAverageTimerValue(&timerGold);
  sdkDeleteTimer(&timerGold);

  // perform quantization
  quantizeGoldFloat(ImgF2, StrideF, Size);

  // perform block-wise IDCT processing
  computeIDCT8x8Gold1(ImgF2, ImgF1, StrideF, Size);

  // convert image back to byte representation
  AddFloatPlane(128.0f, ImgF1, StrideF, Size);
  CopyFloat2Byte(ImgF1, StrideF, ImgDst, Stride, Size);

  // free float buffers
  FreePlane(ImgF1);
  FreePlane(ImgF2);

  // return time taken by the operation
  return TimerGoldSpan;
}

/**
**************************************************************************
*  Wrapper function for 2nd gold version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperGold2(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // allocate float buffers for DCT and other data
  int StrideF;
  float *ImgF1 = MallocPlaneFloat(Size.width, Size.height, &StrideF);
  float *ImgF2 = MallocPlaneFloat(Size.width, Size.height, &StrideF);

  // convert source image to float representation
  CopyByte2Float(ImgSrc, Stride, ImgF1, StrideF, Size);
  AddFloatPlane(-128.0f, ImgF1, StrideF, Size);

  // create and start CUDA timer
  StopWatchInterface *timerGold = 0;
  sdkCreateTimer(&timerGold);
  sdkResetTimer(&timerGold);

  // perform block-wise DCT processing and benchmarking
  for (int i = 0; i < BENCHMARK_SIZE; i++) {
    sdkStartTimer(&timerGold);
    computeDCT8x8Gold2(ImgF1, ImgF2, StrideF, Size);
    sdkStopTimer(&timerGold);
  }

  // stop and destroy CUDA timer
  float TimerGoldSpan = sdkGetAverageTimerValue(&timerGold);
  sdkDeleteTimer(&timerGold);

  // perform quantization
  quantizeGoldFloat(ImgF2, StrideF, Size);

  // perform block-wise IDCT processing
  computeIDCT8x8Gold2(ImgF2, ImgF1, StrideF, Size);

  // convert image back to byte representation
  AddFloatPlane(128.0f, ImgF1, StrideF, Size);
  CopyFloat2Byte(ImgF1, StrideF, ImgDst, Stride, Size);

  // free float buffers
  FreePlane(ImgF1);
  FreePlane(ImgF2);

  // return time taken by the operation
  return TimerGoldSpan;
}

/**
**************************************************************************
*  Wrapper function for 1st CUDA version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperCUDA1(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // prepare channel format descriptor for passing texture into kernels
  hipChannelFormatDesc floattex = hipCreateChannelDesc<float>();

  // allocate device memory
  hipArray *Src;
  float *Dst;
  size_t DstStride;
  checkCudaErrors(hipMallocArray(&Src, &floattex, Size.width, Size.height));
  checkCudaErrors(hipMallocPitch((void **)(&Dst), &DstStride,
                                  Size.width * sizeof(float), Size.height));
  DstStride /= sizeof(float);

  // convert source image to float representation
  int ImgSrcFStride;
  float *ImgSrcF = MallocPlaneFloat(Size.width, Size.height, &ImgSrcFStride);
  CopyByte2Float(ImgSrc, Stride, ImgSrcF, ImgSrcFStride, Size);
  AddFloatPlane(-128.0f, ImgSrcF, ImgSrcFStride, Size);

  // copy from host memory to device
  checkCudaErrors(hipMemcpy2DToArray(
      Src, 0, 0, ImgSrcF, ImgSrcFStride * sizeof(float),
      Size.width * sizeof(float), Size.height, hipMemcpyHostToDevice));

  // setup execution parameters
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

  // create and start CUDA timer
  StopWatchInterface *timerCUDA = 0;
  sdkCreateTimer(&timerCUDA);
  sdkResetTimer(&timerCUDA);

  // execute DCT kernel and benchmark
  hipTextureObject_t TexSrc;
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = Src;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&TexSrc, &texRes, &texDescr, NULL));

  for (int i = 0; i < BENCHMARK_SIZE; i++) {
    sdkStartTimer(&timerCUDA);
    CUDAkernel1DCT<<<grid, threads>>>(Dst, (int)DstStride, 0, 0, TexSrc);
    checkCudaErrors(hipDeviceSynchronize());
    sdkStopTimer(&timerCUDA);
  }

  getLastCudaError("Kernel execution failed");

  // finalize CUDA timer
  float TimerCUDASpan = sdkGetAverageTimerValue(&timerCUDA);
  sdkDeleteTimer(&timerCUDA);

  // execute Quantization kernel
  CUDAkernelQuantizationFloat<<<grid, threads>>>(Dst, (int)DstStride);
  getLastCudaError("Kernel execution failed");

  // copy quantized coefficients from host memory to device array
  checkCudaErrors(hipMemcpy2DToArray(Src, 0, 0, Dst, DstStride * sizeof(float),
                                      Size.width * sizeof(float), Size.height,
                                      hipMemcpyDeviceToDevice));

  // execute IDCT kernel
  CUDAkernel1IDCT<<<grid, threads>>>(Dst, (int)DstStride, 0, 0, TexSrc);
  getLastCudaError("Kernel execution failed");

  // copy quantized image block to host
  checkCudaErrors(hipMemcpy2D(
      ImgSrcF, ImgSrcFStride * sizeof(float), Dst, DstStride * sizeof(float),
      Size.width * sizeof(float), Size.height, hipMemcpyDeviceToHost));

  // convert image back to byte representation
  AddFloatPlane(128.0f, ImgSrcF, ImgSrcFStride, Size);
  CopyFloat2Byte(ImgSrcF, ImgSrcFStride, ImgDst, Stride, Size);

  // clean up memory
  checkCudaErrors(hipDestroyTextureObject(TexSrc));
  checkCudaErrors(hipFreeArray(Src));
  checkCudaErrors(hipFree(Dst));
  FreePlane(ImgSrcF);

  // return time taken by the operation
  return TimerCUDASpan;
}

/**
**************************************************************************
*  Wrapper function for 2nd CUDA version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/

float WrapperCUDA2(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // allocate host buffers for DCT and other data
  int StrideF;
  float *ImgF1 = MallocPlaneFloat(Size.width, Size.height, &StrideF);

  // convert source image to float representation
  CopyByte2Float(ImgSrc, Stride, ImgF1, StrideF, Size);
  AddFloatPlane(-128.0f, ImgF1, StrideF, Size);

  // allocate device memory
  float *src, *dst;
  size_t DeviceStride;
  checkCudaErrors(hipMallocPitch((void **)&src, &DeviceStride,
                                  Size.width * sizeof(float), Size.height));
  checkCudaErrors(hipMallocPitch((void **)&dst, &DeviceStride,
                                  Size.width * sizeof(float), Size.height));
  DeviceStride /= sizeof(float);

  // copy from host memory to device
  checkCudaErrors(hipMemcpy2D(
      src, DeviceStride * sizeof(float), ImgF1, StrideF * sizeof(float),
      Size.width * sizeof(float), Size.height, hipMemcpyHostToDevice));

  // create and start CUDA timer
  StopWatchInterface *timerCUDA = 0;
  sdkCreateTimer(&timerCUDA);

  // setup execution parameters
  dim3 GridFullWarps(Size.width / KER2_BLOCK_WIDTH,
                     Size.height / KER2_BLOCK_HEIGHT, 1);
  dim3 ThreadsFullWarps(8, KER2_BLOCK_WIDTH / 8, KER2_BLOCK_HEIGHT / 8);

  // perform block-wise DCT processing and benchmarking
  const int numIterations = 100;

  for (int i = -1; i < numIterations; i++) {
    if (i == 0) {
      checkCudaErrors(hipDeviceSynchronize());
      sdkResetTimer(&timerCUDA);
      sdkStartTimer(&timerCUDA);
    }

    CUDAkernel2DCT<<<GridFullWarps, ThreadsFullWarps>>>(dst, src,
                                                        (int)DeviceStride);
    getLastCudaError("Kernel execution failed");
  }

  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timerCUDA);

  // finalize timing of CUDA Kernels
  float avgTime = (float)sdkGetTimerValue(&timerCUDA) / (float)numIterations;
  sdkDeleteTimer(&timerCUDA);
  printf("%f MPix/s //%f ms\n",
         (1E-6 * (float)Size.width * (float)Size.height) / (1E-3 * avgTime),
         avgTime);

  // setup execution parameters for quantization
  dim3 ThreadsSmallBlocks(BLOCK_SIZE, BLOCK_SIZE);
  dim3 GridSmallBlocks(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

  // execute Quantization kernel
  CUDAkernelQuantizationFloat<<<GridSmallBlocks, ThreadsSmallBlocks>>>(
      dst, (int)DeviceStride);
  getLastCudaError("Kernel execution failed");

  // perform block-wise IDCT processing
  CUDAkernel2IDCT<<<GridFullWarps, ThreadsFullWarps>>>(src, dst,
                                                       (int)DeviceStride);
  checkCudaErrors(hipDeviceSynchronize());
  getLastCudaError("Kernel execution failed");

  // copy quantized image block to host
  checkCudaErrors(hipMemcpy2D(
      ImgF1, StrideF * sizeof(float), src, DeviceStride * sizeof(float),
      Size.width * sizeof(float), Size.height, hipMemcpyDeviceToHost));

  // convert image back to byte representation
  AddFloatPlane(128.0f, ImgF1, StrideF, Size);
  CopyFloat2Byte(ImgF1, StrideF, ImgDst, Stride, Size);

  // clean up memory
  checkCudaErrors(hipFree(dst));
  checkCudaErrors(hipFree(src));
  FreePlane(ImgF1);

  // return time taken by the operation
  return avgTime;
}

/**
**************************************************************************
*  Wrapper function for short CUDA version of DCT, quantization and IDCT
*implementations
*
* \param ImgSrc         [IN] - Source byte image plane
* \param ImgDst         [IN] - Quantized result byte image plane
* \param Stride         [IN] - Stride for both source and result planes
* \param Size           [IN] - Size of both planes
*
* \return Execution time in milliseconds
*/
float WrapperCUDAshort(byte *ImgSrc, byte *ImgDst, int Stride, ROI Size) {
  // allocate host buffers for DCT and other data
  int StrideS;
  short *ImgS1 = MallocPlaneShort(Size.width, Size.height, &StrideS);

  // convert source image to short representation centered at 128
  for (int i = 0; i < Size.height; i++) {
    for (int j = 0; j < Size.width; j++) {
      ImgS1[i * StrideS + j] = (short)ImgSrc[i * Stride + j] - 128;
    }
  }

  // allocate device memory
  short *SrcDst;
  size_t DeviceStride;
  checkCudaErrors(hipMallocPitch((void **)(&SrcDst), &DeviceStride,
                                  Size.width * sizeof(short), Size.height));
  DeviceStride /= sizeof(short);

  // copy from host memory to device
  checkCudaErrors(hipMemcpy2D(
      SrcDst, DeviceStride * sizeof(short), ImgS1, StrideS * sizeof(short),
      Size.width * sizeof(short), Size.height, hipMemcpyHostToDevice));

  // create and start CUDA timer
  StopWatchInterface *timerLibJpeg = 0;
  sdkCreateTimer(&timerLibJpeg);
  sdkResetTimer(&timerLibJpeg);

  // setup execution parameters
  dim3 GridShort(Size.width / KERS_BLOCK_WIDTH, Size.height / KERS_BLOCK_HEIGHT,
                 1);
  dim3 ThreadsShort(8, KERS_BLOCK_WIDTH / 8, KERS_BLOCK_HEIGHT / 8);

  // perform block-wise DCT processing and benchmarking
  sdkStartTimer(&timerLibJpeg);
  CUDAkernelShortDCT<<<GridShort, ThreadsShort>>>(SrcDst, (int)DeviceStride);
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&timerLibJpeg);
  getLastCudaError("Kernel execution failed");

  // stop and destroy CUDA timer
  float TimerLibJpegSpan16b = sdkGetAverageTimerValue(&timerLibJpeg);
  sdkDeleteTimer(&timerLibJpeg);

  // setup execution parameters for quantization
  dim3 ThreadsSmallBlocks(BLOCK_SIZE, BLOCK_SIZE);
  dim3 GridSmallBlocks(Size.width / BLOCK_SIZE, Size.height / BLOCK_SIZE);

  // execute Quantization kernel
  CUDAkernelQuantizationShort<<<GridSmallBlocks, ThreadsSmallBlocks>>>(
      SrcDst, (int)DeviceStride);
  getLastCudaError("Kernel execution failed");

  // perform block-wise IDCT processing
  CUDAkernelShortIDCT<<<GridShort, ThreadsShort>>>(SrcDst, (int)DeviceStride);
  checkCudaErrors(hipDeviceSynchronize());
  getLastCudaError("Kernel execution failed");

  // copy quantized image block to host
  checkCudaErrors(hipMemcpy2D(
      ImgS1, StrideS * sizeof(short), SrcDst, DeviceStride * sizeof(short),
      Size.width * sizeof(short), Size.height, hipMemcpyDeviceToHost));

  // convert image back to byte representation
  for (int i = 0; i < Size.height; i++) {
    for (int j = 0; j < Size.width; j++) {
      ImgDst[i * Stride + j] = clamp_0_255(ImgS1[i * StrideS + j] + 128);
    }
  }

  // free float buffers
  checkCudaErrors(hipFree(SrcDst));
  FreePlane(ImgS1);

  // return time taken by the operation
  return TimerLibJpegSpan16b;
}

/**
**************************************************************************
*  Program entry point
*
* \param argc       [IN] - Number of command-line arguments
* \param argv       [IN] - Array of command-line arguments
*
* \return Status code
*/

int main(int argc, char **argv) {
  //
  // Sample initialization
  //
  printf("%s Starting...\n\n", argv[0]);

  // initialize CUDA
  findCudaDevice(argc, (const char **)argv);

  // source and results image filenames
  char SampleImageFname[] = "teapot512.bmp";
  char SampleImageFnameResGold1[] = "teapot512_gold1.bmp";
  char SampleImageFnameResGold2[] = "teapot512_gold2.bmp";
  char SampleImageFnameResCUDA1[] = "teapot512_cuda1.bmp";
  char SampleImageFnameResCUDA2[] = "teapot512_cuda2.bmp";
  char SampleImageFnameResCUDAshort[] = "teapot512_cuda_short.bmp";

  char *pSampleImageFpath = sdkFindFilePath(SampleImageFname, argv[0]);

  if (pSampleImageFpath == NULL) {
    printf("dct8x8 could not locate Sample Image <%s>\nExiting...\n",
           pSampleImageFpath);
    exit(EXIT_FAILURE);
  }

  // preload image (acquire dimensions)
  int ImgWidth, ImgHeight;
  ROI ImgSize;
  int res = PreLoadBmp(pSampleImageFpath, &ImgWidth, &ImgHeight);
  ImgSize.width = ImgWidth;
  ImgSize.height = ImgHeight;

  // CONSOLE INFORMATION: saying hello to user
  printf("CUDA sample DCT/IDCT implementation\n");
  printf("===================================\n");
  printf("Loading test image: %s... ", SampleImageFname);

  if (res) {
    printf("\nError: Image file not found or invalid!\n");
    exit(EXIT_FAILURE);
    return 1;
  }

  // check image dimensions are multiples of BLOCK_SIZE
  if (ImgWidth % BLOCK_SIZE != 0 || ImgHeight % BLOCK_SIZE != 0) {
    printf("\nError: Input image dimensions must be multiples of 8!\n");
    exit(EXIT_FAILURE);
    return 1;
  }

  printf("[%d x %d]... ", ImgWidth, ImgHeight);

  // allocate image buffers
  int ImgStride;
  byte *ImgSrc = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte *ImgDstGold1 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte *ImgDstGold2 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte *ImgDstCUDA1 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte *ImgDstCUDA2 = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);
  byte *ImgDstCUDAshort = MallocPlaneByte(ImgWidth, ImgHeight, &ImgStride);

  // load sample image
  LoadBmpAsGray(pSampleImageFpath, ImgStride, ImgSize, ImgSrc);

  //
  // RUNNING WRAPPERS
  //

  // compute Gold 1 version of DCT/quantization/IDCT
  printf("Success\nRunning Gold 1 (CPU) version... ");
  float TimeGold1 = WrapperGold1(ImgSrc, ImgDstGold1, ImgStride, ImgSize);

  // compute Gold 2 version of DCT/quantization/IDCT
  printf("Success\nRunning Gold 2 (CPU) version... ");
  float TimeGold2 = WrapperGold2(ImgSrc, ImgDstGold2, ImgStride, ImgSize);

  // compute CUDA 1 version of DCT/quantization/IDCT
  printf("Success\nRunning CUDA 1 (GPU) version... ");
  float TimeCUDA1 = WrapperCUDA1(ImgSrc, ImgDstCUDA1, ImgStride, ImgSize);

  // compute CUDA 2 version of DCT/quantization/IDCT
  printf("Success\nRunning CUDA 2 (GPU) version... ");
  float TimeCUDA2 = WrapperCUDA2(ImgSrc, ImgDstCUDA2, ImgStride, ImgSize);

  // compute CUDA short version of DCT/quantization/IDCT
  printf("Success\nRunning CUDA short (GPU) version... ");
  float TimeCUDAshort =
      WrapperCUDAshort(ImgSrc, ImgDstCUDAshort, ImgStride, ImgSize);
  //
  // Execution statistics, result saving and validation
  //

  // dump result of Gold 1 processing
  printf("Success\nDumping result to %s... ", SampleImageFnameResGold1);
  DumpBmpAsGray(SampleImageFnameResGold1, ImgDstGold1, ImgStride, ImgSize);

  // dump result of Gold 2 processing
  printf("Success\nDumping result to %s... ", SampleImageFnameResGold2);
  DumpBmpAsGray(SampleImageFnameResGold2, ImgDstGold2, ImgStride, ImgSize);

  // dump result of CUDA 1 processing
  printf("Success\nDumping result to %s... ", SampleImageFnameResCUDA1);
  DumpBmpAsGray(SampleImageFnameResCUDA1, ImgDstCUDA1, ImgStride, ImgSize);

  // dump result of CUDA 2 processing
  printf("Success\nDumping result to %s... ", SampleImageFnameResCUDA2);
  DumpBmpAsGray(SampleImageFnameResCUDA2, ImgDstCUDA2, ImgStride, ImgSize);

  // dump result of CUDA short processing
  printf("Success\nDumping result to %s... ", SampleImageFnameResCUDAshort);
  DumpBmpAsGray(SampleImageFnameResCUDAshort, ImgDstCUDAshort, ImgStride,
                ImgSize);
  // print speed info
  printf("Success\n");

  printf("Processing time (CUDA 1)    : %f ms \n", TimeCUDA1);
  printf("Processing time (CUDA 2)    : %f ms \n", TimeCUDA2);
  printf("Processing time (CUDA short): %f ms \n", TimeCUDAshort);

  // calculate PSNR between each pair of images
  float PSNR_Src_DstGold1 =
      CalculatePSNR(ImgSrc, ImgDstGold1, ImgStride, ImgSize);
  float PSNR_Src_DstGold2 =
      CalculatePSNR(ImgSrc, ImgDstGold2, ImgStride, ImgSize);
  float PSNR_Src_DstCUDA1 =
      CalculatePSNR(ImgSrc, ImgDstCUDA1, ImgStride, ImgSize);
  float PSNR_Src_DstCUDA2 =
      CalculatePSNR(ImgSrc, ImgDstCUDA2, ImgStride, ImgSize);
  float PSNR_Src_DstCUDAshort =
      CalculatePSNR(ImgSrc, ImgDstCUDAshort, ImgStride, ImgSize);
  float PSNR_DstGold1_DstCUDA1 =
      CalculatePSNR(ImgDstGold1, ImgDstCUDA1, ImgStride, ImgSize);
  float PSNR_DstGold2_DstCUDA2 =
      CalculatePSNR(ImgDstGold2, ImgDstCUDA2, ImgStride, ImgSize);
  float PSNR_DstGold2_DstCUDA16b =
      CalculatePSNR(ImgDstGold2, ImgDstCUDAshort, ImgStride, ImgSize);

  printf("PSNR Original    <---> CPU(Gold 1)    : %f\n", PSNR_Src_DstGold1);
  printf("PSNR Original    <---> CPU(Gold 2)    : %f\n", PSNR_Src_DstGold2);
  printf("PSNR Original    <---> GPU(CUDA 1)    : %f\n", PSNR_Src_DstCUDA1);
  printf("PSNR Original    <---> GPU(CUDA 2)    : %f\n", PSNR_Src_DstCUDA2);
  printf("PSNR Original    <---> GPU(CUDA short): %f\n", PSNR_Src_DstCUDAshort);
  printf("PSNR CPU(Gold 1) <---> GPU(CUDA 1)    : %f\n",
         PSNR_DstGold1_DstCUDA1);
  printf("PSNR CPU(Gold 2) <---> GPU(CUDA 2)    : %f\n",
         PSNR_DstGold2_DstCUDA2);
  printf("PSNR CPU(Gold 2) <---> GPU(CUDA short): %f\n",
         PSNR_DstGold2_DstCUDA16b);

  bool bTestResult = (PSNR_DstGold1_DstCUDA1 > PSNR_THRESHOLD_EQUAL &&
                      PSNR_DstGold2_DstCUDA2 > PSNR_THRESHOLD_EQUAL &&
                      PSNR_DstGold2_DstCUDA16b > PSNR_THRESHOLD_EQUAL);

  //
  // Finalization
  //

  // release byte planes
  FreePlane(ImgSrc);
  FreePlane(ImgDstGold1);
  FreePlane(ImgDstGold2);
  FreePlane(ImgDstCUDA1);
  FreePlane(ImgDstCUDA2);
  FreePlane(ImgDstCUDAshort);

  // finalize
  printf("\nTest Summary...\n");

  if (!bTestResult) {
    printf("Test failed!\n");
    exit(EXIT_FAILURE);
  }

  printf("Test passed\n");
  exit(EXIT_SUCCESS);
}
