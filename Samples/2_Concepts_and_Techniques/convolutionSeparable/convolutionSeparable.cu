#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include "convolutionSeparable_common.h"

////////////////////////////////////////////////////////////////////////////////
// Convolution kernel storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[KERNEL_LENGTH];

extern "C" void setConvolutionKernel(float *h_Kernel) {
  hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}

////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
#define ROWS_BLOCKDIM_X 16
#define ROWS_BLOCKDIM_Y 4
#define ROWS_RESULT_STEPS 8
#define ROWS_HALO_STEPS 1

__global__ void convolutionRowsKernel(float *d_Dst, float *d_Src, int imageW,
                                      int imageH, int pitch) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float
      s_Data[ROWS_BLOCKDIM_Y][(ROWS_RESULT_STEPS + 2 * ROWS_HALO_STEPS) *
                              ROWS_BLOCKDIM_X];

  // Offset to the left halo edge
  const int baseX =
      (blockIdx.x * ROWS_RESULT_STEPS - ROWS_HALO_STEPS) * ROWS_BLOCKDIM_X +
      threadIdx.x;
  const int baseY = blockIdx.y * ROWS_BLOCKDIM_Y + threadIdx.y;

  d_Src += baseY * pitch + baseX;
  d_Dst += baseY * pitch + baseX;

// Load main data
#pragma unroll

  for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++) {
    s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] =
        d_Src[i * ROWS_BLOCKDIM_X];
  }

// Load left halo
#pragma unroll

  for (int i = 0; i < ROWS_HALO_STEPS; i++) {
    s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] =
        (baseX >= -i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
  }

// Load right halo
#pragma unroll

  for (int i = ROWS_HALO_STEPS + ROWS_RESULT_STEPS;
       i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS + ROWS_HALO_STEPS; i++) {
    s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X] =
        (imageW - baseX > i * ROWS_BLOCKDIM_X) ? d_Src[i * ROWS_BLOCKDIM_X] : 0;
  }

  // Compute and store results
  cg::sync(cta);
#pragma unroll

  for (int i = ROWS_HALO_STEPS; i < ROWS_HALO_STEPS + ROWS_RESULT_STEPS; i++) {
    float sum = 0;

#pragma unroll

    for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++) {
      sum += c_Kernel[KERNEL_RADIUS - j] *
             s_Data[threadIdx.y][threadIdx.x + i * ROWS_BLOCKDIM_X + j];
    }

    d_Dst[i * ROWS_BLOCKDIM_X] = sum;
  }
}

extern "C" void convolutionRowsGPU(float *d_Dst, float *d_Src, int imageW,
                                   int imageH) {
  assert(ROWS_BLOCKDIM_X * ROWS_HALO_STEPS >= KERNEL_RADIUS);
  assert(imageW % (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X) == 0);
  assert(imageH % ROWS_BLOCKDIM_Y == 0);

  dim3 blocks(imageW / (ROWS_RESULT_STEPS * ROWS_BLOCKDIM_X),
              imageH / ROWS_BLOCKDIM_Y);
  dim3 threads(ROWS_BLOCKDIM_X, ROWS_BLOCKDIM_Y);

  convolutionRowsKernel<<<blocks, threads>>>(d_Dst, d_Src, imageW, imageH,
                                             imageW);
  getLastCudaError("convolutionRowsKernel() execution failed\n");
}

////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
#define COLUMNS_BLOCKDIM_X 16
#define COLUMNS_BLOCKDIM_Y 8
#define COLUMNS_RESULT_STEPS 8
#define COLUMNS_HALO_STEPS 1

__global__ void convolutionColumnsKernel(float *d_Dst, float *d_Src, int imageW,
                                         int imageH, int pitch) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ float s_Data[COLUMNS_BLOCKDIM_X][(COLUMNS_RESULT_STEPS +
                                               2 * COLUMNS_HALO_STEPS) *
                                                  COLUMNS_BLOCKDIM_Y +
                                              1];

  // Offset to the upper halo edge
  const int baseX = blockIdx.x * COLUMNS_BLOCKDIM_X + threadIdx.x;
  const int baseY = (blockIdx.y * COLUMNS_RESULT_STEPS - COLUMNS_HALO_STEPS) *
                        COLUMNS_BLOCKDIM_Y +
                    threadIdx.y;
  d_Src += baseY * pitch + baseX;
  d_Dst += baseY * pitch + baseX;

// Main data
#pragma unroll

  for (int i = COLUMNS_HALO_STEPS;
       i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++) {
    s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] =
        d_Src[i * COLUMNS_BLOCKDIM_Y * pitch];
  }

// Upper halo
#pragma unroll

  for (int i = 0; i < COLUMNS_HALO_STEPS; i++) {
    s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] =
        (baseY >= -i * COLUMNS_BLOCKDIM_Y)
            ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch]
            : 0;
  }

// Lower halo
#pragma unroll

  for (int i = COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS;
       i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS + COLUMNS_HALO_STEPS;
       i++) {
    s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y] =
        (imageH - baseY > i * COLUMNS_BLOCKDIM_Y)
            ? d_Src[i * COLUMNS_BLOCKDIM_Y * pitch]
            : 0;
  }

  // Compute and store results
  cg::sync(cta);
#pragma unroll

  for (int i = COLUMNS_HALO_STEPS;
       i < COLUMNS_HALO_STEPS + COLUMNS_RESULT_STEPS; i++) {
    float sum = 0;
#pragma unroll

    for (int j = -KERNEL_RADIUS; j <= KERNEL_RADIUS; j++) {
      sum += c_Kernel[KERNEL_RADIUS - j] *
             s_Data[threadIdx.x][threadIdx.y + i * COLUMNS_BLOCKDIM_Y + j];
    }

    d_Dst[i * COLUMNS_BLOCKDIM_Y * pitch] = sum;
  }
}

extern "C" void convolutionColumnsGPU(float *d_Dst, float *d_Src, int imageW,
                                      int imageH) {
  assert(COLUMNS_BLOCKDIM_Y * COLUMNS_HALO_STEPS >= KERNEL_RADIUS);
  assert(imageW % COLUMNS_BLOCKDIM_X == 0);
  assert(imageH % (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y) == 0);

  dim3 blocks(imageW / COLUMNS_BLOCKDIM_X,
              imageH / (COLUMNS_RESULT_STEPS * COLUMNS_BLOCKDIM_Y));
  dim3 threads(COLUMNS_BLOCKDIM_X, COLUMNS_BLOCKDIM_Y);

  convolutionColumnsKernel<<<blocks, threads>>>(d_Dst, d_Src, imageW, imageH,
                                                imageW);
  getLastCudaError("convolutionColumnsKernel() execution failed\n");
}
