#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * Demonstration of inline PTX (assembly language) usage in CUDA kernels
 */

// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

__global__ void sequence_gpu(int *d_ptr, int length)
{
    int elemID = blockIdx.x * blockDim.x + threadIdx.x;

    if (elemID < length)
    {
        unsigned int laneid;
        //This command gets the lane ID within the current warp
        asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
        d_ptr[elemID] = laneid;
    }
}


void sequence_cpu(int *h_ptr, int length)
{
    for (int elemID=0; elemID<length; elemID++)
    {
        h_ptr[elemID] = elemID % 32;
    }
}

int main(int argc, char **argv)
{
    printf("CUDA inline PTX assembler sample\n");

    const int N = 1000;

    int dev = findCudaDevice(argc, (const char **) argv);

    if (dev == -1)
    {
        return EXIT_FAILURE;
    }

    int *d_ptr;
    checkCudaErrors(hipMalloc(&d_ptr, N * sizeof(int)));

    int *h_ptr;
    checkCudaErrors(hipHostMalloc(&h_ptr, N * sizeof(int)));

    dim3 cudaBlockSize(256,1,1);
    dim3 cudaGridSize((N + cudaBlockSize.x - 1) / cudaBlockSize.x, 1, 1);
    sequence_gpu<<<cudaGridSize, cudaBlockSize>>>(d_ptr, N);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    sequence_cpu(h_ptr, N);

    int *h_d_ptr;
    checkCudaErrors(hipHostMalloc(&h_d_ptr, N *sizeof(int)));
    checkCudaErrors(hipMemcpy(h_d_ptr, d_ptr, N *sizeof(int), hipMemcpyDeviceToHost));

    bool bValid = true;

    for (int i=0; i<N && bValid; i++)
    {
        if (h_ptr[i] != h_d_ptr[i])
        {
            bValid = false;
        }
    }

    printf("Test %s.\n", bValid ? "Successful" : "Failed");

    checkCudaErrors(hipFree(d_ptr));
    checkCudaErrors(hipHostFree(h_ptr));
    checkCudaErrors(hipHostFree(h_d_ptr));

    return bValid ? EXIT_SUCCESS: EXIT_FAILURE;
}
