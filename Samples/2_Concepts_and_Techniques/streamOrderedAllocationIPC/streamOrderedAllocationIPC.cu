#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample demonstrates Inter Process Communication
 * using one process per GPU for computation.
 */

#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <hip/hip_runtime.h>
#define CUDA_DRIVER_API 1
#include "hip/hip_runtime_api.h"
#include "helper_cuda_drvapi.h"
#include "helper_multiprocess.h"

static const char shmName[] = "streamOrderedAllocationIPCshm";
static const char ipcName[] = "streamOrderedAllocationIPC_pipe";
// For direct NVLINK and PCI-E peers, at max 8 simultaneous peers are allowed
// For NVSWITCH connected peers like DGX-2, simultaneous peers are not limited
// in the same way.
#define MAX_DEVICES (32)
#define DATA_SIZE (64ULL << 20ULL)  // 64MB

#if defined(__linux__)
#define cpu_atomic_add32(a, x) __sync_add_and_fetch(a, x)
#elif defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
#define cpu_atomic_add32(a, x) InterlockedAdd((volatile LONG *)a, x)
#else
#error Unsupported system
#endif

typedef struct shmStruct_st {
  size_t nprocesses;
  int barrier;
  int sense;
  int devices[MAX_DEVICES];
  hipMemPoolPtrExportData exportPtrData[MAX_DEVICES];
} shmStruct;

__global__ void simpleKernel(char *ptr, int sz, char val) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for (; idx < sz; idx += (gridDim.x * blockDim.x)) {
    ptr[idx] = val;
  }
}

static void barrierWait(volatile int *barrier, volatile int *sense,
                        unsigned int n) {
  int count;

  // Check-in
  count = cpu_atomic_add32(barrier, 1);
  if (count == n)  // Last one in
    *sense = 1;
  while (!*sense)
    ;

  // Check-out
  count = cpu_atomic_add32(barrier, -1);
  if (count == 0)  // Last one out
    *sense = 0;
  while (*sense)
    ;
}

static void childProcess(int id) {
  volatile shmStruct *shm = NULL;
  hipStream_t stream;
  sharedMemoryInfo info;
  size_t procCount, i;
  int blocks = 0;
  int threads = 128;
  hipDeviceProp_t prop;
  std::vector<void *> ptrs;

  std::vector<char> verification_buffer(DATA_SIZE);

  ipcHandle *ipcChildHandle = NULL;
  checkIpcErrors(ipcOpenSocket(ipcChildHandle));

  if (sharedMemoryOpen(shmName, sizeof(shmStruct), &info) != 0) {
    printf("Failed to create shared memory slab\n");
    exit(EXIT_FAILURE);
  }
  shm = (volatile shmStruct *)info.addr;
  procCount = shm->nprocesses;

  barrierWait(&shm->barrier, &shm->sense, (unsigned int)(procCount + 1));

  // Receive all allocation handles shared by Parent.
  std::vector<ShareableHandle> shHandle(shm->nprocesses);
  checkIpcErrors(ipcRecvShareableHandles(ipcChildHandle, shHandle));

  checkCudaErrors(hipSetDevice(shm->devices[id]));
  checkCudaErrors(hipGetDeviceProperties(&prop, shm->devices[id]));
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      &blocks, simpleKernel, threads, 0));
  blocks *= prop.multiProcessorCount;

  std::vector<hipMemPool_t> pools(shm->nprocesses);

  hipMemAllocationHandleType handleType = hipMemHandleTypePosixFileDescriptor;

  // Import mem pools from all the devices created in the master
  // process using shareable handles received via socket
  // and import the pointer to the allocated buffer using
  // exportData filled in shared memory by the master process.
  for (i = 0; i < procCount; i++) {
    checkCudaErrors(hipMemPoolImportFromShareableHandle(
        &pools[i], (void *)shHandle[i], handleType, 0));

    hipMemAccessFlags accessFlags;
    hipMemLocation location;
    location.type = hipMemLocationTypeDevice;
    location.id = shm->devices[id];
    checkCudaErrors(hipMemPoolGetAccess(&accessFlags, pools[i], &location));
    if (accessFlags != hipMemAccessFlagsProtReadWrite) {
      hipMemAccessDesc desc;
      memset(&desc, 0, sizeof(hipMemAccessDesc));
      desc.location.type = hipMemLocationTypeDevice;
      desc.location.id = shm->devices[id];
      desc.flags = hipMemAccessFlagsProtReadWrite;
      checkCudaErrors(hipMemPoolSetAccess(pools[i], &desc, 1));
    }

    // Import the allocation from each memory pool by iterating over exportData
    // until import is success.
    for (int j = 0; j < procCount; j++) {
      void *ptr = NULL;
      // Import the allocation using the opaque export data retrieved through
      // the shared memory".
      hipError_t ret = hipMemPoolImportPointer(
          &ptr, pools[i], (hipMemPoolPtrExportData *)&shm->exportPtrData[j]);

      if (ret == hipSuccess) {
        // Pointer import is successful hence add it to the ptrs bag.
        ptrs.push_back(ptr);
        break;
      } else {
        // Reset failure error received from hipMemPoolImportPointer
        // for further try.
        hipGetLastError();
      }
    }
    // Since we have imported allocations shared by the parent with us, we can
    // close this ShareableHandle.
    checkIpcErrors(ipcCloseShareableHandle(shHandle[i]));
  }

  // Since we have imported allocations shared by the parent with us, we can
  // close the socket.
  checkIpcErrors(ipcCloseSocket(ipcChildHandle));

  // At each iteration of the loop, each sibling process will push work on
  // their respective devices accessing the next peer mapped buffer allocated
  // by the master process (these can come from other sibling processes as
  // well). To coordinate each process' access, we force the stream to wait for
  // the work already accessing this buffer.
  for (i = 0; i < procCount; i++) {
    size_t bufferId = (i + id) % procCount;

    // Push a simple kernel on it
    simpleKernel<<<blocks, threads, 0, stream>>>((char *)ptrs[bufferId],
                                                 DATA_SIZE, id);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipStreamSynchronize(stream));

    // Wait for all my sibling processes to push this stage of their work
    // before proceeding to the next. This prevents siblings from racing
    // ahead and clobbering the recorded event or waiting on the wrong
    // recorded event.
    barrierWait(&shm->barrier, &shm->sense, (unsigned int)procCount);
    if (id == 0) {
      printf("Step %lld done\n", (unsigned long long)i);
    }
  }

  // Now wait for my buffer to be ready so I can copy it locally and verify it
  checkCudaErrors(hipMemcpyAsync(&verification_buffer[0], ptrs[id], DATA_SIZE,
                                  hipMemcpyDeviceToHost, stream));

  // And wait for all the queued up work to complete
  checkCudaErrors(hipStreamSynchronize(stream));

  printf("Process %d: verifying...\n", id);

  // The contents should have the id of the sibling just after me
  char compareId = (char)((id + 1) % procCount);
  for (unsigned long long j = 0; j < DATA_SIZE; j++) {
    if (verification_buffer[j] != compareId) {
      printf("Process %d: Verification mismatch at %lld: %d != %d\n", id, j,
             (int)verification_buffer[j], (int)compareId);
    }
  }

  // Clean up!
  for (i = 0; i < procCount; i++) {
    // Free the memory before the exporter process frees it
    checkCudaErrors(hipFreeAsync(ptrs[i], stream));
  }

  // And wait for all the queued up work to complete
  checkCudaErrors(hipStreamSynchronize(stream));
  checkCudaErrors(hipStreamDestroy(stream));

  printf("Process %d complete!\n", id);
}

static void parentProcess(char *app) {
  sharedMemoryInfo info;
  int devCount, i;
  volatile shmStruct *shm = NULL;
  std::vector<void *> ptrs;
  std::vector<Process> processes;

  checkCudaErrors(hipGetDeviceCount(&devCount));
  std::vector<hipDevice_t> devices(devCount);
  for (i = 0; i < devCount; i++) {
    hipDeviceGet(&devices[i], i);
  }

  if (sharedMemoryCreate(shmName, sizeof(*shm), &info) != 0) {
    printf("Failed to create shared memory slab\n");
    exit(EXIT_FAILURE);
  }
  shm = (volatile shmStruct *)info.addr;
  memset((void *)shm, 0, sizeof(*shm));

  // Pick all the devices that can access each other's memory for this test
  // Keep in mind that CUDA has minimal support for fork() without a
  // corresponding exec() in the child process, but in this case our
  // spawnProcess will always exec, so no need to worry.
  for (i = 0; i < devCount; i++) {
    bool allPeers = true;
    hipDeviceProp_t prop;
    checkCudaErrors(hipGetDeviceProperties(&prop, i));

    int isMemPoolSupported = 0;
    checkCudaErrors(hipDeviceGetAttribute(&isMemPoolSupported,
                                           hipDeviceAttributeMemoryPoolsSupported, i));
    // CUDA IPC is only supported on devices with unified addressing
    if (!isMemPoolSupported) {
      printf("Device %d does not support cuda memory pools, skipping...\n", i);
      continue;
    }
    int deviceSupportsIpcHandle = 0;
#if defined(__linux__)
    checkCudaErrors(hipDeviceGetAttribute(
        &deviceSupportsIpcHandle,
        CU_DEVICE_ATTRIBUTE_HANDLE_TYPE_POSIX_FILE_DESCRIPTOR_SUPPORTED,
        devices[i]));
#else
    hipDeviceGetAttribute(&deviceSupportsIpcHandle,
                         CU_DEVICE_ATTRIBUTE_HANDLE_TYPE_WIN32_HANDLE_SUPPORTED,
                         devices[i]);
#endif

    if (!deviceSupportsIpcHandle) {
      printf("Device %d does not support CUDA IPC Handle, skipping...\n", i);
      continue;
    }
    // This sample requires two processes accessing each device, so we need
    // to ensure exclusive or prohibited mode is not set
    if (prop.computeMode != hipComputeModeDefault) {
      printf("Device %d is in an unsupported compute mode for this sample\n",
             i);
      continue;
    }
#if defined(WIN32) || defined(_WIN32) || defined(WIN64) || defined(_WIN64)
    // CUDA IPC on Windows is only supported on TCC
    if (!prop.tccDriver) {
      printf("Device %d is not in TCC mode\n", i);
      continue;
    }
#endif

    for (int j = 0; j < shm->nprocesses; j++) {
      int canAccessPeerIJ, canAccessPeerJI;
      checkCudaErrors(
          hipDeviceCanAccessPeer(&canAccessPeerJI, shm->devices[j], i));
      checkCudaErrors(
          hipDeviceCanAccessPeer(&canAccessPeerIJ, i, shm->devices[j]));
      if (!canAccessPeerIJ || !canAccessPeerJI) {
        allPeers = false;
        break;
      }
    }
    if (allPeers) {
      // Enable peers here.  This isn't necessary for IPC, but it will
      // setup the peers for the device.  For systems that only allow 8
      // peers per GPU at a time, this acts to remove devices from CanAccessPeer
      for (int j = 0; j < shm->nprocesses; j++) {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipDeviceEnablePeerAccess(shm->devices[j], 0));
        checkCudaErrors(hipSetDevice(shm->devices[j]));
        checkCudaErrors(hipDeviceEnablePeerAccess(i, 0));
      }
      shm->devices[shm->nprocesses++] = i;
      if (shm->nprocesses >= MAX_DEVICES) break;
    } else {
      printf(
          "Device %d is not peer capable with some other selected peers, "
          "skipping\n",
          i);
    }
  }

  if (shm->nprocesses == 0) {
    printf("No CUDA devices support IPC\n");
    exit(EXIT_WAIVED);
  }

  std::vector<ShareableHandle> shareableHandles(shm->nprocesses);
  std::vector<hipStream_t> streams(shm->nprocesses);
  std::vector<hipMemPool_t> pools(shm->nprocesses);

  // Now allocate memory for each process and fill the shared
  // memory buffer with the export data and get memPool handles to communicate
  for (i = 0; i < shm->nprocesses; i++) {
    void *ptr = NULL;
    checkCudaErrors(hipSetDevice(shm->devices[i]));
    checkCudaErrors(
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking));
    // Allocate an explicit pool with IPC capabilities
    hipMemPoolProps poolProps;
    memset(&poolProps, 0, sizeof(hipMemPoolProps));
    poolProps.allocType = hipMemAllocationTypePinned;
    poolProps.handleTypes = hipMemHandleTypePosixFileDescriptor;

    poolProps.location.type = hipMemLocationTypeDevice;
    poolProps.location.id = shm->devices[i];

    checkCudaErrors(hipMemPoolCreate(&pools[i], &poolProps));

    // Query the shareable handle for the pool
    hipMemAllocationHandleType handleType =
        hipMemHandleTypePosixFileDescriptor;
    // Allocate memory in a stream from the pool just created
    checkCudaErrors(hipMallocAsync(&ptr, DATA_SIZE, pools[i], streams[i]));

    checkCudaErrors(hipMemPoolExportToShareableHandle(
        &shareableHandles[i], pools[i], handleType, 0));

    // Get the opaque ‘bag-of-bits’ representing the allocation
    memset((void *)&shm->exportPtrData[i], 0, sizeof(hipMemPoolPtrExportData));
    checkCudaErrors(hipMemPoolExportPointer(
        (hipMemPoolPtrExportData *)&shm->exportPtrData[i], ptr));
    ptrs.push_back(ptr);
  }

  // Launch the child processes!
  for (i = 0; i < shm->nprocesses; i++) {
    char devIdx[10];
    char *const args[] = {app, devIdx, NULL};
    Process process;

    SPRINTF(devIdx, "%d", i);

    if (spawnProcess(&process, app, args)) {
      printf("Failed to create process\n");
      exit(EXIT_FAILURE);
    }

    processes.push_back(process);
  }

  barrierWait(&shm->barrier, &shm->sense, (unsigned int)(shm->nprocesses + 1));

  ipcHandle *ipcParentHandle = NULL;
  checkIpcErrors(ipcCreateSocket(ipcParentHandle, ipcName, processes));
  checkIpcErrors(
      ipcSendShareableHandles(ipcParentHandle, shareableHandles, processes));

  // Close the shareable handles as they are not needed anymore.
  for (int i = 0; i < shm->nprocesses; i++) {
    checkIpcErrors(ipcCloseShareableHandle(shareableHandles[i]));
  }
  checkIpcErrors(ipcCloseSocket(ipcParentHandle));

  // And wait for them to finish
  for (i = 0; i < processes.size(); i++) {
    if (waitProcess(&processes[i]) != EXIT_SUCCESS) {
      printf("Process %d failed!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  // Clean up!
  for (i = 0; i < shm->nprocesses; i++) {
    checkCudaErrors(hipSetDevice(shm->devices[i]));
    checkCudaErrors(hipFreeAsync(ptrs[i], streams[i]));
    checkCudaErrors(hipStreamSynchronize(streams[i]));
    checkCudaErrors(hipMemPoolDestroy(pools[i]));
  }

  sharedMemoryClose(&info);
}

// Host code
int main(int argc, char **argv) {
#if defined(__arm__) || defined(__aarch64__) || defined(WIN32) || \
    defined(_WIN32) || defined(WIN64) || defined(_WIN64)
  printf("Not supported on ARM\n");
  return EXIT_WAIVED;
#else
  if (argc == 1) {
    parentProcess(argv[0]);
  } else {
    childProcess(atoi(argv[1]));
  }
  return EXIT_SUCCESS;
#endif
}
