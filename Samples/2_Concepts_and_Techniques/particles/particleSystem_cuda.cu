#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

// This file contains C wrappers around the some of the CUDA API and the
// kernel functions so that they can be called from "particleSystem.cpp"

#if defined(__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#include <GLUT/glut.h>
#else
#include <GL/freeglut.h>
#endif

#include <cstdlib>
#include <cstdio>
#include <string.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <hip/hip_runtime_api.h>

#include <helper_functions.h>
#include "thrust/device_ptr.h"
#include "thrust/for_each.h"
#include "thrust/iterator/zip_iterator.h"
#include "thrust/sort.h"

#include "particles_kernel_impl.cuh"

extern "C" {

void cudaInit(int argc, char **argv) {
  int devID;

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  devID = findCudaDevice(argc, (const char **)argv);

  if (devID < 0) {
    printf("No CUDA Capable devices found, exiting...\n");
    exit(EXIT_SUCCESS);
  }
}

void allocateArray(void **devPtr, size_t size) {
  checkCudaErrors(hipMalloc(devPtr, size));
}

void freeArray(void *devPtr) { checkCudaErrors(hipFree(devPtr)); }

void threadSync() { checkCudaErrors(hipDeviceSynchronize()); }

void copyArrayToDevice(void *device, const void *host, int offset, int size) {
  checkCudaErrors(
      hipMemcpy((char *)device + offset, host, size, hipMemcpyHostToDevice));
}

void registerGLBufferObject(uint vbo,
                            struct hipGraphicsResource **cuda_vbo_resource) {
  checkCudaErrors(hipGraphicsGLRegisterBuffer(cuda_vbo_resource, vbo,
                                               cudaGraphicsMapFlagsNone));
}

void unregisterGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource) {
  checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource));
}

void *mapGLBufferObject(struct hipGraphicsResource **cuda_vbo_resource) {
  void *ptr;
  checkCudaErrors(hipGraphicsMapResources(1, cuda_vbo_resource, 0));
  size_t num_bytes;
  checkCudaErrors(hipGraphicsResourceGetMappedPointer(
      (void **)&ptr, &num_bytes, *cuda_vbo_resource));
  return ptr;
}

void unmapGLBufferObject(struct hipGraphicsResource *cuda_vbo_resource) {
  checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0));
}

void copyArrayFromDevice(void *host, const void *device,
                         struct hipGraphicsResource **cuda_vbo_resource,
                         int size) {
  if (cuda_vbo_resource) {
    device = mapGLBufferObject(cuda_vbo_resource);
  }

  checkCudaErrors(hipMemcpy(host, device, size, hipMemcpyDeviceToHost));

  if (cuda_vbo_resource) {
    unmapGLBufferObject(*cuda_vbo_resource);
  }
}

void setParameters(SimParams *hostParams) {
  // copy parameters to constant memory
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(params), hostParams, sizeof(SimParams)));
}

// Round a / b to nearest higher integer value
uint iDivUp(uint a, uint b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// compute grid and thread block size for a given number of elements
void computeGridSize(uint n, uint blockSize, uint &numBlocks,
                     uint &numThreads) {
  numThreads = min(blockSize, n);
  numBlocks = iDivUp(n, numThreads);
}

void integrateSystem(float *pos, float *vel, float deltaTime,
                     uint numParticles) {
  thrust::device_ptr<float4> d_pos4((float4 *)pos);
  thrust::device_ptr<float4> d_vel4((float4 *)vel);

  thrust::for_each(
      thrust::make_zip_iterator(thrust::make_tuple(d_pos4, d_vel4)),
      thrust::make_zip_iterator(
          thrust::make_tuple(d_pos4 + numParticles, d_vel4 + numParticles)),
      integrate_functor(deltaTime));
}

void calcHash(uint *gridParticleHash, uint *gridParticleIndex, float *pos,
              int numParticles) {
  uint numThreads, numBlocks;
  computeGridSize(numParticles, 256, numBlocks, numThreads);

  // execute the kernel
  calcHashD<<<numBlocks, numThreads>>>(gridParticleHash, gridParticleIndex,
                                       (float4 *)pos, numParticles);

  // check if kernel invocation generated an error
  getLastCudaError("Kernel execution failed");
}

void reorderDataAndFindCellStart(uint *cellStart, uint *cellEnd,
                                 float *sortedPos, float *sortedVel,
                                 uint *gridParticleHash,
                                 uint *gridParticleIndex, float *oldPos,
                                 float *oldVel, uint numParticles,
                                 uint numCells) {
  uint numThreads, numBlocks;
  computeGridSize(numParticles, 256, numBlocks, numThreads);

  // set all cells to empty
  checkCudaErrors(hipMemset(cellStart, 0xffffffff, numCells * sizeof(uint)));

  uint smemSize = sizeof(uint) * (numThreads + 1);
  reorderDataAndFindCellStartD<<<numBlocks, numThreads, smemSize>>>(
      cellStart, cellEnd, (float4 *)sortedPos, (float4 *)sortedVel,
      gridParticleHash, gridParticleIndex, (float4 *)oldPos, (float4 *)oldVel,
      numParticles);
  getLastCudaError("Kernel execution failed: reorderDataAndFindCellStartD");
}

void collide(float *newVel, float *sortedPos, float *sortedVel,
             uint *gridParticleIndex, uint *cellStart, uint *cellEnd,
             uint numParticles, uint numCells) {
  // thread per particle
  uint numThreads, numBlocks;
  computeGridSize(numParticles, 64, numBlocks, numThreads);

  // execute the kernel
  collideD<<<numBlocks, numThreads>>>((float4 *)newVel, (float4 *)sortedPos,
                                      (float4 *)sortedVel, gridParticleIndex,
                                      cellStart, cellEnd, numParticles);

  // check if kernel invocation generated an error
  getLastCudaError("Kernel execution failed");
}

void sortParticles(uint *dGridParticleHash, uint *dGridParticleIndex,
                   uint numParticles) {
  thrust::sort_by_key(
      thrust::device_ptr<uint>(dGridParticleHash),
      thrust::device_ptr<uint>(dGridParticleHash + numParticles),
      thrust::device_ptr<uint>(dGridParticleIndex));
}

}  // extern "C"
