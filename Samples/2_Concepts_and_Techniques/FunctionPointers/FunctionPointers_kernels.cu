#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;

#include <hip/hip_runtime_api.h>

#include "FunctionPointers_kernels.h"

// Texture object for reading image
hipTextureObject_t tex;
extern __shared__ unsigned char LocalBlock[];
static hipArray *array = NULL;

#define RADIUS 1

// pixel value used for thresholding function,
// works well with sample image 'teapot512'
#define THRESHOLD 150.0f

#ifdef FIXED_BLOCKWIDTH
#define BlockWidth 80
#define SharedPitch 384
#endif

// A function pointer can be declared explicitly like this line:
//__device__ unsigned char (*pointFunction)(unsigned char, float ) = NULL;
// or by using typedef's like below:

typedef unsigned char (*blockFunction_t)(unsigned char, unsigned char,
                                         unsigned char, unsigned char,
                                         unsigned char, unsigned char,
                                         unsigned char, unsigned char,
                                         unsigned char, float);

typedef unsigned char (*pointFunction_t)(unsigned char, float);

__device__ blockFunction_t blockFunction;

__device__ unsigned char ComputeSobel(unsigned char ul,  // upper left
                                      unsigned char um,  // upper middle
                                      unsigned char ur,  // upper right
                                      unsigned char ml,  // middle left
                                      unsigned char mm,  // middle (unused)
                                      unsigned char mr,  // middle right
                                      unsigned char ll,  // lower left
                                      unsigned char lm,  // lower middle
                                      unsigned char lr,  // lower right
                                      float fScale) {
  short Horz = ur + 2 * mr + lr - ul - 2 * ml - ll;
  short Vert = ul + 2 * um + ur - ll - 2 * lm - lr;
  short Sum = (short)(fScale * (abs((int)Horz) + abs((int)Vert)));
  return (unsigned char)((Sum < 0) ? 0 : ((Sum > 255) ? 255 : Sum));
}

// define a function pointer and initialize to NULL
__device__ unsigned char (*varFunction)(unsigned char, unsigned char,
                                        unsigned char, unsigned char,
                                        unsigned char, unsigned char,
                                        unsigned char, unsigned char,
                                        unsigned char, float x) = NULL;

__device__ unsigned char ComputeBox(unsigned char ul,  // upper left
                                    unsigned char um,  // upper middle
                                    unsigned char ur,  // upper right
                                    unsigned char ml,  // middle left
                                    unsigned char mm,  // middle...middle
                                    unsigned char mr,  // middle right
                                    unsigned char ll,  // lower left
                                    unsigned char lm,  // lower middle
                                    unsigned char lr,  // lower right
                                    float fscale) {
  short Sum = (short)(ul + um + ur + ml + mm + mr + ll + lm + lr) / 9;
  Sum *= fscale;
  return (unsigned char)((Sum < 0) ? 0 : ((Sum > 255) ? 255 : Sum));
}
__device__ unsigned char Threshold(unsigned char in, float thresh) {
  if (in > thresh) {
    return 0xFF;
  } else {
    return 0;
  }
}

// Declare function tables, one for the point function chosen, one for the
// block function chosen.  The number of entries is determined by the
// enum in FunctionPointers_kernels.h
__device__ blockFunction_t blockFunction_table[LAST_BLOCK_FILTER];
__device__ pointFunction_t pointFunction_table[LAST_POINT_FILTER];

// Declare device side function pointers.  We retrieve them later with
// hipMemcpyFromSymbol to set our function tables above in some
// particular order specified at runtime.
__device__ blockFunction_t pComputeSobel = ComputeSobel;
__device__ blockFunction_t pComputeBox = ComputeBox;
__device__ pointFunction_t pComputeThreshold = Threshold;

// Allocate host side tables to mirror the device side, and later, we
// fill these tables with the function pointers.  This lets us send
// the pointers to the kernel on invocation, as a method of choosing
// which function to run.
blockFunction_t h_blockFunction_table[2];
pointFunction_t h_pointFunction_table[2];

// Perform a filter operation on the data, using shared memory
// The actual operation performed is
// determined by the function pointer "blockFunction" and selected
// by the integer argument "blockOperation" and has access
// to an apron around the current pixel being processed.
// Following the block operation, a per-pixel operation,
// pointed to by pPointFunction is performed before the final
// pixel is produced.
__global__ void SobelShared(uchar4 *pSobelOriginal, unsigned short SobelPitch,
#ifndef FIXED_BLOCKWIDTH
                            short BlockWidth, short SharedPitch,
#endif
                            short w, short h, float fScale, int blockOperation,
                            pointFunction_t pPointFunction,
                            hipTextureObject_t tex) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  short u = 4 * blockIdx.x * BlockWidth;
  short v = blockIdx.y * blockDim.y + threadIdx.y;
  short ib;

  int SharedIdx = threadIdx.y * SharedPitch;

  for (ib = threadIdx.x; ib < BlockWidth + 2 * RADIUS; ib += blockDim.x) {
    LocalBlock[SharedIdx + 4 * ib + 0] = tex2D<unsigned char>(
        tex, (float)(u + 4 * ib - RADIUS + 0), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 1] = tex2D<unsigned char>(
        tex, (float)(u + 4 * ib - RADIUS + 1), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 2] = tex2D<unsigned char>(
        tex, (float)(u + 4 * ib - RADIUS + 2), (float)(v - RADIUS));
    LocalBlock[SharedIdx + 4 * ib + 3] = tex2D<unsigned char>(
        tex, (float)(u + 4 * ib - RADIUS + 3), (float)(v - RADIUS));
  }

  if (threadIdx.y < RADIUS * 2) {
    //
    // copy trailing RADIUS*2 rows of pixels into shared
    //
    SharedIdx = (blockDim.y + threadIdx.y) * SharedPitch;

    for (ib = threadIdx.x; ib < BlockWidth + 2 * RADIUS; ib += blockDim.x) {
      LocalBlock[SharedIdx + 4 * ib + 0] =
          tex2D<unsigned char>(tex, (float)(u + 4 * ib - RADIUS + 0),
                               (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 1] =
          tex2D<unsigned char>(tex, (float)(u + 4 * ib - RADIUS + 1),
                               (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 2] =
          tex2D<unsigned char>(tex, (float)(u + 4 * ib - RADIUS + 2),
                               (float)(v + blockDim.y - RADIUS));
      LocalBlock[SharedIdx + 4 * ib + 3] =
          tex2D<unsigned char>(tex, (float)(u + 4 * ib - RADIUS + 3),
                               (float)(v + blockDim.y - RADIUS));
    }
  }

  cg::sync(cta);

  u >>= 2;  // index as uchar4 from here
  uchar4 *pSobel = (uchar4 *)(((char *)pSobelOriginal) + v * SobelPitch);
  SharedIdx = threadIdx.y * SharedPitch;

  blockFunction = blockFunction_table[blockOperation];

  for (ib = threadIdx.x; ib < BlockWidth; ib += blockDim.x) {
    uchar4 out;

    unsigned char pix00 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 0];
    unsigned char pix01 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 1];
    unsigned char pix02 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 2];
    unsigned char pix10 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 0];
    unsigned char pix11 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 1];
    unsigned char pix12 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 2];
    unsigned char pix20 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 0];
    unsigned char pix21 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 1];
    unsigned char pix22 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 2];

    out.x = (*blockFunction)(pix00, pix01, pix02, pix10, pix11, pix12, pix20,
                             pix21, pix22, fScale);

    pix00 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 3];
    pix10 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 3];
    pix20 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 3];
    out.y = (*blockFunction)(pix01, pix02, pix00, pix11, pix12, pix10, pix21,
                             pix22, pix20, fScale);

    pix01 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 4];
    pix11 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 4];
    pix21 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 4];
    out.z = (*blockFunction)(pix02, pix00, pix01, pix12, pix10, pix11, pix22,
                             pix20, pix21, fScale);

    pix02 = LocalBlock[SharedIdx + 4 * ib + 0 * SharedPitch + 5];
    pix12 = LocalBlock[SharedIdx + 4 * ib + 1 * SharedPitch + 5];
    pix22 = LocalBlock[SharedIdx + 4 * ib + 2 * SharedPitch + 5];
    out.w = (*blockFunction)(pix00, pix01, pix02, pix10, pix11, pix12, pix20,
                             pix21, pix22, fScale);

    if (pPointFunction != NULL) {
      out.x = (*pPointFunction)(out.x, THRESHOLD);
      out.y = (*pPointFunction)(out.y, THRESHOLD);
      out.z = (*pPointFunction)(out.z, THRESHOLD);
      out.w = (*pPointFunction)(out.w, THRESHOLD);
    }

    if (u + ib < w / 4 && v < h) {
      pSobel[u + ib] = out;
    }
  }

  cg::sync(cta);
}

__global__ void SobelCopyImage(Pixel *pSobelOriginal, unsigned int Pitch, int w,
                               int h, float fscale, hipTextureObject_t tex) {
  unsigned char *pSobel =
      (unsigned char *)(((char *)pSobelOriginal) + blockIdx.x * Pitch);

  for (int i = threadIdx.x; i < w; i += blockDim.x) {
    pSobel[i] = min(
        max((tex2D<unsigned char>(tex, (float)i, (float)blockIdx.x) * fscale),
            0.f),
        255.f);
  }
}

// Perform block and pointer filtering using texture lookups.
// The block and point operations are determined by the
// input argument (see comment above for "SobelShared" function)
__global__ void SobelTex(Pixel *pSobelOriginal, unsigned int Pitch, int w,
                         int h, float fScale, int blockOperation,
                         pointFunction_t pPointOperation,
                         hipTextureObject_t tex) {
  unsigned char *pSobel =
      (unsigned char *)(((char *)pSobelOriginal) + blockIdx.x * Pitch);
  unsigned char tmp = 0;

  for (int i = threadIdx.x; i < w; i += blockDim.x) {
    unsigned char pix00 =
        tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x - 1);
    unsigned char pix01 =
        tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x - 1);
    unsigned char pix02 =
        tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x - 1);
    unsigned char pix10 =
        tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x + 0);
    unsigned char pix11 =
        tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x + 0);
    unsigned char pix12 =
        tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x + 0);
    unsigned char pix20 =
        tex2D<unsigned char>(tex, (float)i - 1, (float)blockIdx.x + 1);
    unsigned char pix21 =
        tex2D<unsigned char>(tex, (float)i + 0, (float)blockIdx.x + 1);
    unsigned char pix22 =
        tex2D<unsigned char>(tex, (float)i + 1, (float)blockIdx.x + 1);
    tmp = (*(blockFunction_table[blockOperation]))(
        pix00, pix01, pix02, pix10, pix11, pix12, pix20, pix21, pix22, fScale);

    if (pPointOperation != NULL) {
      tmp = (*pPointOperation)(tmp, 150.0);
    }

    pSobel[i] = tmp;
  }
}

extern "C" void setupTexture(int iw, int ih, Pixel *data, int Bpp) {
  hipChannelFormatDesc desc;

  if (Bpp == 1) {
    desc = hipCreateChannelDesc<unsigned char>();
  } else {
    desc = hipCreateChannelDesc<uchar4>();
  }

  checkCudaErrors(hipMallocArray(&array, &desc, iw, ih));
  checkCudaErrors(hipMemcpy2DToArray(
      array, 0, 0, data, iw * Bpp * sizeof(Pixel), iw * Bpp * sizeof(Pixel), ih,
      hipMemcpyHostToDevice));

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = array;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));
}

extern "C" void deleteTexture(void) {
  checkCudaErrors(hipFreeArray(array));
  checkCudaErrors(hipDestroyTextureObject(tex));
}

// Copy the pointers from the function tables to the host side
void setupFunctionTables() {
  // Dynamically assign the function table.
  // Copy the function pointers to their appropriate locations according to the
  // enum
  checkCudaErrors(hipMemcpyFromSymbol(&h_blockFunction_table[SOBEL_FILTER],
                                       HIP_SYMBOL(pComputeSobel), sizeof(blockFunction_t)));
  checkCudaErrors(hipMemcpyFromSymbol(&h_blockFunction_table[BOX_FILTER],
                                       HIP_SYMBOL(pComputeBox), sizeof(blockFunction_t)));

  // do the same for the point function, where the 2nd function is NULL ("no-op"
  // filter, skipped in kernel code)
  checkCudaErrors(hipMemcpyFromSymbol(&h_pointFunction_table[THRESHOLD_FILTER],
                                       HIP_SYMBOL(pComputeThreshold),
                                       sizeof(pointFunction_t)));
  h_pointFunction_table[NULL_FILTER] = NULL;

  // now copy the function tables back to the device, so if we wish we can use
  // an index into the table to choose them
  // We have now set the order in the function table according to our enum.
  checkCudaErrors(
      hipMemcpyToSymbol(HIP_SYMBOL(blockFunction_table), h_blockFunction_table,
                         sizeof(blockFunction_t) * LAST_BLOCK_FILTER));
  checkCudaErrors(
      hipMemcpyToSymbol(HIP_SYMBOL(pointFunction_table), h_pointFunction_table,
                         sizeof(pointFunction_t) * LAST_POINT_FILTER));
}

// Wrapper for the __global__ call that sets up the texture and threads
// Below two methods for selecting the image processing function to run are
// shown.
// BlockOperation is an integer kernel argument used as an index into the
// blockFunction_table on the device side
// pPointOp is itself a function pointer passed as a kernel argument, retrieved
// from a host side copy of the function table
extern "C" void sobelFilter(Pixel *odata, int iw, int ih,
                            enum SobelDisplayMode mode, float fScale,
                            int blockOperation, int pointOperation) {
  pointFunction_t pPointOp = h_pointFunction_table[pointOperation];

  switch (mode) {
    case SOBELDISPLAY_IMAGE:
      SobelCopyImage<<<ih, 384>>>(odata, iw, iw, ih, fScale, tex);
      break;

    case SOBELDISPLAY_SOBELTEX:
      SobelTex<<<ih, 384>>>(odata, iw, iw, ih, fScale, blockOperation, pPointOp,
                            tex);
      break;

    case SOBELDISPLAY_SOBELSHARED: {
      dim3 threads(16, 4);
#ifndef FIXED_BLOCKWIDTH
      int BlockWidth = 80;  // must be divisible by 16 for coalescing
#endif
      dim3 blocks = dim3(iw / (4 * BlockWidth) + (0 != iw % (4 * BlockWidth)),
                         ih / threads.y + (0 != ih % threads.y));
      int SharedPitch = ~0x3f & (4 * (BlockWidth + 2 * RADIUS) + 0x3f);
      int sharedMem = SharedPitch * (threads.y + 2 * RADIUS);

      // for the shared kernel, width must be divisible by 4
      iw &= ~3;

      SobelShared<<<blocks, threads, sharedMem>>>(
          (uchar4 *)odata, iw,
#ifndef FIXED_BLOCKWIDTH
          BlockWidth, SharedPitch,
#endif
          iw, ih, fScale, blockOperation, pPointOp, tex);
    } break;
  }
}
