#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample demonstrates two adaptive image denoising techniques:
 * KNN and NLM, based on computation of both geometric and color distance
 * between texels. While both techniques are already implemented in the
 * DirectX SDK using shaders, massively speeded up variation
 * of the latter technique, taking advantage of shared memory, is implemented
 * in addition to DirectX counterparts.
 * See supplied whitepaper for more explanations.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>
#include "imageDenoising.h"

////////////////////////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////////////////////////
float Max(float x, float y) { return (x > y) ? x : y; }

float Min(float x, float y) { return (x < y) ? x : y; }

int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

__device__ float lerpf(float a, float b, float c) { return a + (b - a) * c; }

__device__ float vecLen(float4 a, float4 b) {
  return ((b.x - a.x) * (b.x - a.x) + (b.y - a.y) * (b.y - a.y) +
          (b.z - a.z) * (b.z - a.z));
}

__device__ TColor make_color(float r, float g, float b, float a) {
  return ((int)(a * 255.0f) << 24) | ((int)(b * 255.0f) << 16) |
         ((int)(g * 255.0f) << 8) | ((int)(r * 255.0f) << 0);
}

////////////////////////////////////////////////////////////////////////////////
// Global data handlers and parameters
////////////////////////////////////////////////////////////////////////////////
// Texture object and channel descriptor for image texture
hipTextureObject_t texImage;
hipChannelFormatDesc uchar4tex = hipCreateChannelDesc<uchar4>();

// CUDA array descriptor
hipArray *a_Src;

////////////////////////////////////////////////////////////////////////////////
// Filtering kernels
////////////////////////////////////////////////////////////////////////////////
#include "imageDenoising_copy_kernel.cuh"
#include "imageDenoising_knn_kernel.cuh"
#include "imageDenoising_nlm_kernel.cuh"
#include "imageDenoising_nlm2_kernel.cuh"

extern "C" hipError_t CUDA_MallocArray(uchar4 **h_Src, int imageW,
                                        int imageH) {
  hipError_t error;

  error = hipMallocArray(&a_Src, &uchar4tex, imageW, imageH);
  error = hipMemcpy2DToArray(a_Src, 0, 0, *h_Src, sizeof(uchar4) * imageW,
                              sizeof(uchar4) * imageW, imageH,
                              hipMemcpyHostToDevice);

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = a_Src;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(hipCreateTextureObject(&texImage, &texRes, &texDescr, NULL));

  return error;
}

extern "C" hipError_t CUDA_FreeArray() { return hipFreeArray(a_Src); }
