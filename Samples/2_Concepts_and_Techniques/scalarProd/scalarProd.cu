#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample calculates scalar products of a
 * given set of input vector pairs
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on CPU
///////////////////////////////////////////////////////////////////////////////
extern "C" void scalarProdCPU(float *h_C, float *h_A, float *h_B, int vectorN,
                              int elementN);

///////////////////////////////////////////////////////////////////////////////
// Calculate scalar products of VectorN vectors of ElementN elements on GPU
///////////////////////////////////////////////////////////////////////////////
#include "scalarProd_kernel.cuh"

////////////////////////////////////////////////////////////////////////////////
// Helper function, returning uniformly distributed
// random float in [low, high] range
////////////////////////////////////////////////////////////////////////////////
float RandFloat(float low, float high) {
  float t = (float)rand() / (float)RAND_MAX;
  return (1.0f - t) * low + t * high;
}

///////////////////////////////////////////////////////////////////////////////
// Data configuration
///////////////////////////////////////////////////////////////////////////////

// Total number of input vector pairs; arbitrary
const int VECTOR_N = 256;
// Number of elements per vector; arbitrary,
// but strongly preferred to be a multiple of warp size
// to meet memory coalescing constraints
const int ELEMENT_N = 4096;
// Total number of data elements
const int DATA_N = VECTOR_N * ELEMENT_N;

const int DATA_SZ = DATA_N * sizeof(float);
const int RESULT_SZ = VECTOR_N * sizeof(float);

///////////////////////////////////////////////////////////////////////////////
// Main program
///////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  float *h_A, *h_B, *h_C_CPU, *h_C_GPU;
  float *d_A, *d_B, *d_C;
  double delta, ref, sum_delta, sum_ref, L1norm;
  StopWatchInterface *hTimer = NULL;
  int i;

  printf("%s Starting...\n\n", argv[0]);

  // use command-line specified CUDA device, otherwise use device with highest
  // Gflops/s
  findCudaDevice(argc, (const char **)argv);

  sdkCreateTimer(&hTimer);

  printf("Initializing data...\n");
  printf("...allocating CPU memory.\n");
  h_A = (float *)malloc(DATA_SZ);
  h_B = (float *)malloc(DATA_SZ);
  h_C_CPU = (float *)malloc(RESULT_SZ);
  h_C_GPU = (float *)malloc(RESULT_SZ);

  printf("...allocating GPU memory.\n");
  checkCudaErrors(hipMalloc((void **)&d_A, DATA_SZ));
  checkCudaErrors(hipMalloc((void **)&d_B, DATA_SZ));
  checkCudaErrors(hipMalloc((void **)&d_C, RESULT_SZ));

  printf("...generating input data in CPU mem.\n");
  srand(123);

  // Generating input data on CPU
  for (i = 0; i < DATA_N; i++) {
    h_A[i] = RandFloat(0.0f, 1.0f);
    h_B[i] = RandFloat(0.0f, 1.0f);
  }

  printf("...copying input data to GPU mem.\n");
  // Copy options data to GPU memory for further processing
  checkCudaErrors(hipMemcpy(d_A, h_A, DATA_SZ, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_B, h_B, DATA_SZ, hipMemcpyHostToDevice));
  printf("Data init done.\n");

  printf("Executing GPU kernel...\n");
  checkCudaErrors(hipDeviceSynchronize());
  sdkResetTimer(&hTimer);
  sdkStartTimer(&hTimer);
  scalarProdGPU<<<128, 256>>>(d_C, d_A, d_B, VECTOR_N, ELEMENT_N);
  getLastCudaError("scalarProdGPU() execution failed\n");
  checkCudaErrors(hipDeviceSynchronize());
  sdkStopTimer(&hTimer);
  printf("GPU time: %f msecs.\n", sdkGetTimerValue(&hTimer));

  printf("Reading back GPU result...\n");
  // Read back GPU results to compare them to CPU results
  checkCudaErrors(hipMemcpy(h_C_GPU, d_C, RESULT_SZ, hipMemcpyDeviceToHost));

  printf("Checking GPU results...\n");
  printf("..running CPU scalar product calculation\n");
  scalarProdCPU(h_C_CPU, h_A, h_B, VECTOR_N, ELEMENT_N);

  printf("...comparing the results\n");
  // Calculate max absolute difference and L1 distance
  // between CPU and GPU results
  sum_delta = 0;
  sum_ref = 0;

  for (i = 0; i < VECTOR_N; i++) {
    delta = fabs(h_C_GPU[i] - h_C_CPU[i]);
    ref = h_C_CPU[i];
    sum_delta += delta;
    sum_ref += ref;
  }

  L1norm = sum_delta / sum_ref;

  printf("Shutting down...\n");
  checkCudaErrors(hipFree(d_C));
  checkCudaErrors(hipFree(d_B));
  checkCudaErrors(hipFree(d_A));
  free(h_C_GPU);
  free(h_C_CPU);
  free(h_B);
  free(h_A);
  sdkDeleteTimer(&hTimer);

  printf("L1 error: %E\n", L1norm);
  printf((L1norm < 1e-6) ? "Test passed\n" : "Test failed!\n");
  exit(L1norm < 1e-6 ? EXIT_SUCCESS : EXIT_FAILURE);
}
