#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <assert.h>
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
#include <hip/hip_runtime_api.h>
#include "scan_common.h"

// All three kernels run 512 threads per workgroup
// Must be a power of two
#define THREADBLOCK_SIZE 256

////////////////////////////////////////////////////////////////////////////////
// Basic scan codelets
////////////////////////////////////////////////////////////////////////////////
// Naive inclusive scan: O(N * log2(N)) operations
// Allocate 2 * 'size' local memory, initialize the first half
// with 'size' zeros avoiding if(pos >= offset) condition evaluation
// and saving instructions
inline __device__ uint scan1Inclusive(uint idata, volatile uint *s_Data,
                                      uint size, cg::thread_block cta) {
  uint pos = 2 * threadIdx.x - (threadIdx.x & (size - 1));
  s_Data[pos] = 0;
  pos += size;
  s_Data[pos] = idata;

  for (uint offset = 1; offset < size; offset <<= 1) {
    cg::sync(cta);
    uint t = s_Data[pos] + s_Data[pos - offset];
    cg::sync(cta);
    s_Data[pos] = t;
  }

  return s_Data[pos];
}

inline __device__ uint scan1Exclusive(uint idata, volatile uint *s_Data,
                                      uint size, cg::thread_block cta) {
  return scan1Inclusive(idata, s_Data, size, cta) - idata;
}

inline __device__ uint4 scan4Inclusive(uint4 idata4, volatile uint *s_Data,
                                       uint size, cg::thread_block cta) {
  // Level-0 inclusive scan
  idata4.y += idata4.x;
  idata4.z += idata4.y;
  idata4.w += idata4.z;

  // Level-1 exclusive scan
  uint oval = scan1Exclusive(idata4.w, s_Data, size / 4, cta);

  idata4.x += oval;
  idata4.y += oval;
  idata4.z += oval;
  idata4.w += oval;

  return idata4;
}

// Exclusive vector scan: the array to be scanned is stored
// in local thread memory scope as uint4
inline __device__ uint4 scan4Exclusive(uint4 idata4, volatile uint *s_Data,
                                       uint size, cg::thread_block cta) {
  uint4 odata4 = scan4Inclusive(idata4, s_Data, size, cta);
  odata4.x -= idata4.x;
  odata4.y -= idata4.y;
  odata4.z -= idata4.z;
  odata4.w -= idata4.w;
  return odata4;
}

////////////////////////////////////////////////////////////////////////////////
// Scan kernels
////////////////////////////////////////////////////////////////////////////////
__global__ void scanExclusiveShared(uint4 *d_Dst, uint4 *d_Src, uint size) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

  uint pos = blockIdx.x * blockDim.x + threadIdx.x;

  // Load data
  uint4 idata4 = d_Src[pos];

  // Calculate exclusive scan
  uint4 odata4 = scan4Exclusive(idata4, s_Data, size, cta);

  // Write back
  d_Dst[pos] = odata4;
}

// Exclusive scan of top elements of bottom-level scans (4 * THREADBLOCK_SIZE)
__global__ void scanExclusiveShared2(uint *d_Buf, uint *d_Dst, uint *d_Src,
                                     uint N, uint arrayLength) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ uint s_Data[2 * THREADBLOCK_SIZE];

  // Skip loads and stores for inactive threads of last threadblock (pos >= N)
  uint pos = blockIdx.x * blockDim.x + threadIdx.x;

  // Load top elements
  // Convert results of bottom-level scan back to inclusive
  uint idata = 0;

  if (pos < N)
    idata = d_Dst[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos] +
            d_Src[(4 * THREADBLOCK_SIZE) - 1 + (4 * THREADBLOCK_SIZE) * pos];

  // Compute
  uint odata = scan1Exclusive(idata, s_Data, arrayLength, cta);

  // Avoid out-of-bound access
  if (pos < N) {
    d_Buf[pos] = odata;
  }
}

// Final step of large-array scan: combine basic inclusive scan with exclusive
// scan of top elements of input arrays
__global__ void uniformUpdate(uint4 *d_Data, uint *d_Buffer) {
  // Handle to thread block group
  cg::thread_block cta = cg::this_thread_block();
  __shared__ uint buf;
  uint pos = blockIdx.x * blockDim.x + threadIdx.x;

  if (threadIdx.x == 0) {
    buf = d_Buffer[blockIdx.x];
  }

  cg::sync(cta);

  uint4 data4 = d_Data[pos];
  data4.x += buf;
  data4.y += buf;
  data4.z += buf;
  data4.w += buf;
  d_Data[pos] = data4;
}

////////////////////////////////////////////////////////////////////////////////
// Interface function
////////////////////////////////////////////////////////////////////////////////
// Derived as 32768 (max power-of-two gridDim.x) * 4 * THREADBLOCK_SIZE
// Due to scanExclusiveShared<<<>>>() 1D block addressing
extern "C" const uint MAX_BATCH_ELEMENTS = 64 * 1048576;
extern "C" const uint MIN_SHORT_ARRAY_SIZE = 4;
extern "C" const uint MAX_SHORT_ARRAY_SIZE = 4 * THREADBLOCK_SIZE;
extern "C" const uint MIN_LARGE_ARRAY_SIZE = 8 * THREADBLOCK_SIZE;
extern "C" const uint MAX_LARGE_ARRAY_SIZE =
    4 * THREADBLOCK_SIZE * THREADBLOCK_SIZE;

// Internal exclusive scan buffer
static uint *d_Buf;

extern "C" void initScan(void) {
  checkCudaErrors(
      hipMalloc((void **)&d_Buf,
                 (MAX_BATCH_ELEMENTS / (4 * THREADBLOCK_SIZE)) * sizeof(uint)));
}

extern "C" void closeScan(void) { checkCudaErrors(hipFree(d_Buf)); }

static uint factorRadix2(uint &log2L, uint L) {
  if (!L) {
    log2L = 0;
    return 0;
  } else {
    for (log2L = 0; (L & 1) == 0; L >>= 1, log2L++)
      ;

    return L;
  }
}

static uint iDivUp(uint dividend, uint divisor) {
  return ((dividend % divisor) == 0) ? (dividend / divisor)
                                     : (dividend / divisor + 1);
}

extern "C" size_t scanExclusiveShort(uint *d_Dst, uint *d_Src, uint batchSize,
                                     uint arrayLength) {
  // Check power-of-two factorization
  uint log2L;
  uint factorizationRemainder = factorRadix2(log2L, arrayLength);
  assert(factorizationRemainder == 1);

  // Check supported size range
  assert((arrayLength >= MIN_SHORT_ARRAY_SIZE) &&
         (arrayLength <= MAX_SHORT_ARRAY_SIZE));

  // Check total batch size limit
  assert((batchSize * arrayLength) <= MAX_BATCH_ELEMENTS);

  // Check all threadblocks to be fully packed with data
  assert((batchSize * arrayLength) % (4 * THREADBLOCK_SIZE) == 0);

  scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
                        THREADBLOCK_SIZE>>>((uint4 *)d_Dst, (uint4 *)d_Src,
                                            arrayLength);
  getLastCudaError("scanExclusiveShared() execution FAILED\n");

  return THREADBLOCK_SIZE;
}

extern "C" size_t scanExclusiveLarge(uint *d_Dst, uint *d_Src, uint batchSize,
                                     uint arrayLength) {
  // Check power-of-two factorization
  uint log2L;
  uint factorizationRemainder = factorRadix2(log2L, arrayLength);
  assert(factorizationRemainder == 1);

  // Check supported size range
  assert((arrayLength >= MIN_LARGE_ARRAY_SIZE) &&
         (arrayLength <= MAX_LARGE_ARRAY_SIZE));

  // Check total batch size limit
  assert((batchSize * arrayLength) <= MAX_BATCH_ELEMENTS);

  scanExclusiveShared<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
                        THREADBLOCK_SIZE>>>((uint4 *)d_Dst, (uint4 *)d_Src,
                                            4 * THREADBLOCK_SIZE);
  getLastCudaError("scanExclusiveShared() execution FAILED\n");

  // Not all threadblocks need to be packed with input data:
  // inactive threads of highest threadblock just don't do global reads and
  // writes
  const uint blockCount2 = iDivUp(
      (batchSize * arrayLength) / (4 * THREADBLOCK_SIZE), THREADBLOCK_SIZE);
  scanExclusiveShared2<<<blockCount2, THREADBLOCK_SIZE>>>(
      (uint *)d_Buf, (uint *)d_Dst, (uint *)d_Src,
      (batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
      arrayLength / (4 * THREADBLOCK_SIZE));
  getLastCudaError("scanExclusiveShared2() execution FAILED\n");

  uniformUpdate<<<(batchSize * arrayLength) / (4 * THREADBLOCK_SIZE),
                  THREADBLOCK_SIZE>>>((uint4 *)d_Dst, (uint *)d_Buf);
  getLastCudaError("uniformUpdate() execution FAILED\n");

  return THREADBLOCK_SIZE;
}
