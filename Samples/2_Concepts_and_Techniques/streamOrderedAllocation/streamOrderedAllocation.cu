#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
 * This sample demonstrates stream ordered memory allocation on a GPU using
 * hipMallocAsync and cudaMemPool family of APIs.
 *
 * basicStreamOrderedAllocation(): demonstrates stream ordered allocation using
 * hipMallocAsync/hipFreeAsync APIs with default settings.
 *
 * streamOrderedAllocationPostSync(): demonstrates if there's a synchronization
 * in between allocations, then setting the release threshold on the pool will
 * make sure the synchronize will not free memory back to the OS.
 */

// System includes
#include <assert.h>
#include <stdio.h>
#include <climits>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#define MAX_ITER 20

/* Add two vectors on the GPU */
__global__ void vectorAddGPU(const float *a, const float *b, float *c, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < N) {
    c[idx] = a[idx] + b[idx];
  }
}

int basicStreamOrderedAllocation(const int dev, const int nelem, const float *a,
                                 const float *b, float *c) {
  float *d_a, *d_b, *d_c;  // Device buffers
  float errorNorm, refNorm, ref, diff;
  size_t bytes = nelem * sizeof(float);

  hipStream_t stream;
  printf("Starting basicStreamOrderedAllocation()\n");
  checkCudaErrors(hipSetDevice(dev));
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

  checkCudaErrors(hipMallocAsync(&d_a, bytes, stream));
  checkCudaErrors(hipMallocAsync(&d_b, bytes, stream));
  checkCudaErrors(hipMallocAsync(&d_c, bytes, stream));
  checkCudaErrors(
      hipMemcpyAsync(d_a, a, bytes, hipMemcpyHostToDevice, stream));
  checkCudaErrors(
      hipMemcpyAsync(d_b, b, bytes, hipMemcpyHostToDevice, stream));

  dim3 block(256);
  dim3 grid((unsigned int)ceil(nelem / (float)block.x));
  vectorAddGPU<<<grid, block, 0, stream>>>(d_a, d_b, d_c, nelem);

  checkCudaErrors(hipFreeAsync(d_a, stream));
  checkCudaErrors(hipFreeAsync(d_b, stream));
  checkCudaErrors(
      hipMemcpyAsync(c, d_c, bytes, hipMemcpyDeviceToHost, stream));
  checkCudaErrors(hipFreeAsync(d_c, stream));
  checkCudaErrors(hipStreamSynchronize(stream));

  /* Compare the results */
  printf("> Checking the results from vectorAddGPU() ...\n");
  errorNorm = 0.f;
  refNorm = 0.f;

  for (int n = 0; n < nelem; n++) {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff * diff;
    refNorm += ref * ref;
  }

  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);
  if (errorNorm / refNorm < 1.e-6f)
    printf("basicStreamOrderedAllocation PASSED\n");

  checkCudaErrors(hipStreamDestroy(stream));

  return errorNorm / refNorm < 1.e-6f ? EXIT_SUCCESS : EXIT_FAILURE;
}

// streamOrderedAllocationPostSync(): demonstrates If the application wants the
// memory to persist in the pool beyond synchronization, then it sets the
// release threshold on the pool. This way, when the application reaches the
// "steady state", it is no longer allocating/freeing memory from the OS.
int streamOrderedAllocationPostSync(const int dev, const int nelem,
                                    const float *a, const float *b, float *c) {
  float *d_a, *d_b, *d_c;  // Device buffers
  float errorNorm, refNorm, ref, diff;
  size_t bytes = nelem * sizeof(float);

  hipStream_t stream;
  hipMemPool_t memPool;
  hipEvent_t start, end;
  printf("Starting streamOrderedAllocationPostSync()\n");
  checkCudaErrors(hipSetDevice(dev));
  checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  checkCudaErrors(hipEventCreate(&start));
  checkCudaErrors(hipEventCreate(&end));

  checkCudaErrors(hipDeviceGetDefaultMemPool(&memPool, dev));
  uint64_t thresholdVal = ULONG_MAX;
  // set high release threshold on the default pool so that hipFreeAsync will
  // not actually release memory to the system. By default, the release
  // threshold for a memory pool is set to zero. This implies that the CUDA
  // driver is allowed to release a memory chunk back to the system as long as
  // it does not contain any active suballocations.
  checkCudaErrors(hipMemPoolSetAttribute(
      memPool, hipMemPoolAttrReleaseThreshold, (void *)&thresholdVal));

  // Record the start event
  checkCudaErrors(hipEventRecord(start, stream));
  for (int i = 0; i < MAX_ITER; i++) {
    checkCudaErrors(hipMallocAsync(&d_a, bytes, stream));
    checkCudaErrors(hipMallocAsync(&d_b, bytes, stream));
    checkCudaErrors(hipMallocAsync(&d_c, bytes, stream));
    checkCudaErrors(
        hipMemcpyAsync(d_a, a, bytes, hipMemcpyHostToDevice, stream));
    checkCudaErrors(
        hipMemcpyAsync(d_b, b, bytes, hipMemcpyHostToDevice, stream));

    dim3 block(256);
    dim3 grid((unsigned int)ceil(nelem / (float)block.x));
    vectorAddGPU<<<grid, block, 0, stream>>>(d_a, d_b, d_c, nelem);

    checkCudaErrors(hipFreeAsync(d_a, stream));
    checkCudaErrors(hipFreeAsync(d_b, stream));
    checkCudaErrors(
        hipMemcpyAsync(c, d_c, bytes, hipMemcpyDeviceToHost, stream));
    checkCudaErrors(hipFreeAsync(d_c, stream));
    checkCudaErrors(hipStreamSynchronize(stream));
  }
  checkCudaErrors(hipEventRecord(end, stream));
  // Wait for the end event to complete
  checkCudaErrors(hipEventSynchronize(end));

  float msecTotal = 0.0f;
  checkCudaErrors(hipEventElapsedTime(&msecTotal, start, end));
  printf("Total elapsed time = %f ms over %d iterations\n", msecTotal,
         MAX_ITER);

  /* Compare the results */
  printf("> Checking the results from vectorAddGPU() ...\n");
  errorNorm = 0.f;
  refNorm = 0.f;

  for (int n = 0; n < nelem; n++) {
    ref = a[n] + b[n];
    diff = c[n] - ref;
    errorNorm += diff * diff;
    refNorm += ref * ref;
  }

  errorNorm = (float)sqrt((double)errorNorm);
  refNorm = (float)sqrt((double)refNorm);
  if (errorNorm / refNorm < 1.e-6f)
    printf("streamOrderedAllocationPostSync PASSED\n");

  checkCudaErrors(hipStreamDestroy(stream));

  return errorNorm / refNorm < 1.e-6f ? EXIT_SUCCESS : EXIT_FAILURE;
}

int main(int argc, char **argv) {
  int nelem;
  int dev = 0;  // use default device 0
  size_t bytes;
  float *a, *b, *c;  // Host

  if (checkCmdLineFlag(argc, (const char **)argv, "help")) {
    printf("Usage:  streamOrderedAllocation [OPTION]\n\n");
    printf("Options:\n");
    printf("  --device=[device #]  Specify the device to be used\n");
    return EXIT_SUCCESS;
  }

  dev = findCudaDevice(argc, (const char **)argv);

  int isMemPoolSupported = 0;
  checkCudaErrors(hipDeviceGetAttribute(&isMemPoolSupported,
                                         hipDeviceAttributeMemoryPoolsSupported, dev));
  if (!isMemPoolSupported) {
    printf("Waiving execution as device does not support Memory Pools\n");
    exit(EXIT_WAIVED);
  }

  // Allocate CPU memory.
  nelem = 1048576;
  bytes = nelem * sizeof(float);

  a = (float *)malloc(bytes);
  b = (float *)malloc(bytes);
  c = (float *)malloc(bytes);
  /* Initialize the vectors. */
  for (int n = 0; n < nelem; n++) {
    a[n] = rand() / (float)RAND_MAX;
    b[n] = rand() / (float)RAND_MAX;
  }

  int ret1 = basicStreamOrderedAllocation(dev, nelem, a, b, c);
  int ret2 = streamOrderedAllocationPostSync(dev, nelem, a, b, c);

  /* Memory clean up */
  free(a);
  free(b);
  free(c);

  return ((ret1 == EXIT_SUCCESS && ret2 == EXIT_SUCCESS) ? EXIT_SUCCESS
                                                         : EXIT_FAILURE);
}
