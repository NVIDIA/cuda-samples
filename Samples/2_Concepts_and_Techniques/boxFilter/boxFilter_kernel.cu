#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#ifndef _BOXFILTER_KERNEL_CH_
#define _BOXFILTER_KERNEL_CH_

#include <hip/hip_vector_types.h>
#include <helper_functions.h>

hipTextureObject_t tex;
hipTextureObject_t texTempArray;
hipTextureObject_t rgbaTex;
hipTextureObject_t rgbaTexTempArray;
hipArray *d_array, *d_tempArray;

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host
// call returns an error
#define checkCudaErrors(err) __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line) {
  if (hipSuccess != err) {
    fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n", file, line,
            (int)err, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

/*
  Perform a fast box filter using the sliding window method.

  As the kernel moves from left to right, we add in the contribution of the
  new sample on the right, and subtract the value of the exiting sample on the
  left. This only requires 2 adds and a mul per output value, independent of the
  filter radius. The box filter is separable, so to perform a 2D box filter we
  perform the filter in the x direction, followed by the same filter in the y
  direction. Applying multiple iterations of the box filter converges towards a
  Gaussian blur. Using CUDA, rows or columns of the image are processed in
  parallel. This version duplicates edge pixels.

  Note that the x (row) pass suffers from uncoalesced global memory reads,
  since each thread is reading from a different row. For this reason it is
  better to use texture lookups for the x pass.
  The y (column) pass is perfectly coalesced.

  Parameters
  id - pointer to input data in global memory
  od - pointer to output data in global memory
  w  - image width
  h  - image height
  r  - filter radius

  e.g. for r = 2, w = 8:

  0 1 2 3 4 5 6 7
  x - -
  - x - -
  - - x - -
    - - x - -
      - - x - -
        - - x - -
          - - x -
            - - x
*/

// process row
__device__ void d_boxfilter_x(float *id, float *od, int w, int h, int r) {
  float scale = 1.0f / (float)((r << 1) + 1);

  float t;
  // do left edge
  t = id[0] * r;

  for (int x = 0; x < (r + 1); x++) {
    t += id[x];
  }

  od[0] = t * scale;

  for (int x = 1; x < (r + 1); x++) {
    t += id[x + r];
    t -= id[0];
    od[x] = t * scale;
  }

  // main loop
  for (int x = (r + 1); x < w - r; x++) {
    t += id[x + r];
    t -= id[x - r - 1];
    od[x] = t * scale;
  }

  // do right edge
  for (int x = w - r; x < w; x++) {
    t += id[w - 1];
    t -= id[x - r - 1];
    od[x] = t * scale;
  }
}

// process column
__device__ void d_boxfilter_y(float *id, float *od, int w, int h, int r) {
  float scale = 1.0f / (float)((r << 1) + 1);

  float t;
  // do left edge
  t = id[0] * r;

  for (int y = 0; y < (r + 1); y++) {
    t += id[y * w];
  }

  od[0] = t * scale;

  for (int y = 1; y < (r + 1); y++) {
    t += id[(y + r) * w];
    t -= id[0];
    od[y * w] = t * scale;
  }

  // main loop
  for (int y = (r + 1); y < (h - r); y++) {
    t += id[(y + r) * w];
    t -= id[((y - r) * w) - w];
    od[y * w] = t * scale;
  }

  // do right edge
  for (int y = h - r; y < h; y++) {
    t += id[(h - 1) * w];
    t -= id[((y - r) * w) - w];
    od[y * w] = t * scale;
  }
}

__global__ void d_boxfilter_x_global(float *id, float *od, int w, int h,
                                     int r) {
  unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;
  d_boxfilter_x(&id[y * w], &od[y * w], w, h, r);
}

__global__ void d_boxfilter_y_global(float *id, float *od, int w, int h,
                                     int r) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  d_boxfilter_y(&id[x], &od[x], w, h, r);
}

// texture version
// texture fetches automatically clamp to edge of image
__global__ void d_boxfilter_x_tex(float *od, int w, int h, int r,
                                  hipTextureObject_t tex) {
  float scale = 1.0f / (float)((r << 1) + 1);
  unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;

  float t = 0.0f;

  for (int x = -r; x <= r; x++) {
    t += tex2D<float>(tex, x, y);
  }

  od[y * w] = t * scale;

  for (int x = 1; x < w; x++) {
    t += tex2D<float>(tex, x + r, y);
    t -= tex2D<float>(tex, x - r - 1, y);
    od[y * w + x] = t * scale;
  }
}

__global__ void d_boxfilter_y_tex(float *od, int w, int h, int r,
                                  hipTextureObject_t tex) {
  float scale = 1.0f / (float)((r << 1) + 1);
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

  float t = 0.0f;

  for (int y = -r; y <= r; y++) {
    t += tex2D<float>(tex, x, y);
  }

  od[x] = t * scale;

  for (int y = 1; y < h; y++) {
    t += tex2D<float>(tex, x, y + r);
    t -= tex2D<float>(tex, x, y - r - 1);
    od[y * w + x] = t * scale;
  }
}

// RGBA version
// reads from 32-bit unsigned int array holding 8-bit RGBA

// convert floating point rgba color to 32-bit integer
__device__ unsigned int rgbaFloatToInt(float4 rgba) {
  rgba.x = __saturatef(rgba.x);  // clamp to [0.0, 1.0]
  rgba.y = __saturatef(rgba.y);
  rgba.z = __saturatef(rgba.z);
  rgba.w = __saturatef(rgba.w);
  return ((unsigned int)(rgba.w * 255.0f) << 24) |
         ((unsigned int)(rgba.z * 255.0f) << 16) |
         ((unsigned int)(rgba.y * 255.0f) << 8) |
         ((unsigned int)(rgba.x * 255.0f));
}

__device__ float4 rgbaIntToFloat(unsigned int c) {
  float4 rgba;
  rgba.x = (c & 0xff) * 0.003921568627f;          //  /255.0f;
  rgba.y = ((c >> 8) & 0xff) * 0.003921568627f;   //  /255.0f;
  rgba.z = ((c >> 16) & 0xff) * 0.003921568627f;  //  /255.0f;
  rgba.w = ((c >> 24) & 0xff) * 0.003921568627f;  //  /255.0f;
  return rgba;
}

// row pass using texture lookups
__global__ void d_boxfilter_rgba_x(unsigned int *od, int w, int h, int r,
                                   hipTextureObject_t rgbaTex) {
  float scale = 1.0f / (float)((r << 1) + 1);
  unsigned int y = blockIdx.x * blockDim.x + threadIdx.x;

  // as long as address is always less than height, we do work
  if (y < h) {
    float4 t = make_float4(0.0f);

    for (int x = -r; x <= r; x++) {
      t += tex2D<float4>(rgbaTex, x, y);
    }

    od[y * w] = rgbaFloatToInt(t * scale);

    for (int x = 1; x < w; x++) {
      t += tex2D<float4>(rgbaTex, x + r, y);
      t -= tex2D<float4>(rgbaTex, x - r - 1, y);
      od[y * w + x] = rgbaFloatToInt(t * scale);
    }
  }
}

// column pass using coalesced global memory reads
__global__ void d_boxfilter_rgba_y(unsigned int *id, unsigned int *od, int w,
                                   int h, int r) {
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  id = &id[x];
  od = &od[x];

  float scale = 1.0f / (float)((r << 1) + 1);

  float4 t;
  // do left edge
  t = rgbaIntToFloat(id[0]) * r;

  for (int y = 0; y < (r + 1); y++) {
    t += rgbaIntToFloat(id[y * w]);
  }

  od[0] = rgbaFloatToInt(t * scale);

  for (int y = 1; y < (r + 1); y++) {
    t += rgbaIntToFloat(id[(y + r) * w]);
    t -= rgbaIntToFloat(id[0]);
    od[y * w] = rgbaFloatToInt(t * scale);
  }

  // main loop
  for (int y = (r + 1); y < (h - r); y++) {
    t += rgbaIntToFloat(id[(y + r) * w]);
    t -= rgbaIntToFloat(id[((y - r) * w) - w]);
    od[y * w] = rgbaFloatToInt(t * scale);
  }

  // do right edge
  for (int y = h - r; y < h; y++) {
    t += rgbaIntToFloat(id[(h - 1) * w]);
    t -= rgbaIntToFloat(id[((y - r) * w) - w]);
    od[y * w] = rgbaFloatToInt(t * scale);
  }
}

extern "C" void initTexture(int width, int height, void *pImage, bool useRGBA) {
  // copy image data to array
  hipChannelFormatDesc channelDesc;
  if (useRGBA) {
    channelDesc =
        hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
  } else {
    channelDesc =
        hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
  }
  checkCudaErrors(hipMallocArray(&d_array, &channelDesc, width, height));

  size_t bytesPerElem = (useRGBA ? sizeof(uchar4) : sizeof(float));
  checkCudaErrors(hipMemcpy2DToArray(
      d_array, 0, 0, pImage, width * bytesPerElem, width * bytesPerElem, height,
      hipMemcpyHostToDevice));

  checkCudaErrors(hipMallocArray(&d_tempArray, &channelDesc, width, height));

  // set texture parameters
  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_array;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(hipCreateTextureObject(&rgbaTex, &texRes, &texDescr, NULL));

  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_tempArray;

  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModeLinear;
  texDescr.addressMode[0] = hipAddressModeClamp;
  texDescr.addressMode[1] = hipAddressModeClamp;
  texDescr.readMode = hipReadModeNormalizedFloat;

  checkCudaErrors(
      hipCreateTextureObject(&rgbaTexTempArray, &texRes, &texDescr, NULL));

  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_array;

  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(hipCreateTextureObject(&tex, &texRes, &texDescr, NULL));

  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = d_tempArray;

  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = true;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.addressMode[1] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&texTempArray, &texRes, &texDescr, NULL));
}

extern "C" void freeTextures() {
  checkCudaErrors(hipDestroyTextureObject(tex));
  checkCudaErrors(hipDestroyTextureObject(texTempArray));
  checkCudaErrors(hipDestroyTextureObject(rgbaTex));
  checkCudaErrors(hipDestroyTextureObject(rgbaTexTempArray));
  checkCudaErrors(hipFreeArray(d_array));
  checkCudaErrors(hipFreeArray(d_tempArray));
}

/*
    Perform 2D box filter on image using CUDA

    Parameters:
    d_src  - pointer to input image in device memory
    d_temp - pointer to temporary storage in device memory
    d_dest - pointer to destination image in device memory
    width  - image width
    height - image height
    radius - filter radius
    iterations - number of iterations

*/
extern "C" double boxFilter(float *d_src, float *d_temp, float *d_dest,
                            int width, int height, int radius, int iterations,
                            int nthreads, StopWatchInterface *timer) {
  // var for kernel timing
  double dKernelTime = 0.0;

  // sync host and start computation timer_kernel
  checkCudaErrors(hipDeviceSynchronize());

  for (int i = 0; i < iterations; i++) {
    sdkResetTimer(&timer);
    // use texture for horizontal pass
    if (iterations > 1) {
      d_boxfilter_x_tex<<<height / nthreads, nthreads, 0>>>(
          d_temp, width, height, radius, texTempArray);
    } else {
      d_boxfilter_x_tex<<<height / nthreads, nthreads, 0>>>(
          d_temp, width, height, radius, tex);
    }

    d_boxfilter_y_global<<<width / nthreads, nthreads, 0>>>(
        d_temp, d_dest, width, height, radius);

    // sync host and stop computation timer_kernel
    checkCudaErrors(hipDeviceSynchronize());
    dKernelTime += sdkGetTimerValue(&timer);

    if (iterations > 1) {
      // copy result back from global memory to array
      checkCudaErrors(hipMemcpy2DToArray(
          d_tempArray, 0, 0, d_dest, width * sizeof(float),
          width * sizeof(float), height, hipMemcpyDeviceToDevice));
    }
  }

  return ((dKernelTime / 1000.) / (double)iterations);
}

// RGBA version
extern "C" double boxFilterRGBA(unsigned int *d_src, unsigned int *d_temp,
                                unsigned int *d_dest, int width, int height,
                                int radius, int iterations, int nthreads,
                                StopWatchInterface *timer) {
  // var for kernel computation timing
  double dKernelTime;

  for (int i = 0; i < iterations; i++) {
    // sync host and start kernel computation timer_kernel
    dKernelTime = 0.0;
    checkCudaErrors(hipDeviceSynchronize());
    sdkResetTimer(&timer);

    // use texture for horizontal pass
    if (iterations > 1) {
      d_boxfilter_rgba_x<<<height / nthreads, nthreads, 0>>>(
          d_temp, width, height, radius, rgbaTexTempArray);
    } else {
      d_boxfilter_rgba_x<<<height / nthreads, nthreads, 0>>>(
          d_temp, width, height, radius, rgbaTex);
    }

    d_boxfilter_rgba_y<<<width / nthreads, nthreads, 0>>>(d_temp, d_dest, width,
                                                          height, radius);

    // sync host and stop computation timer_kernel
    checkCudaErrors(hipDeviceSynchronize());
    dKernelTime += sdkGetTimerValue(&timer);

    if (iterations > 1) {
      // copy result back from global memory to array
      checkCudaErrors(hipMemcpy2DToArray(
          d_tempArray, 0, 0, d_dest, width * sizeof(unsigned int),
          width * sizeof(unsigned int), height, hipMemcpyDeviceToDevice));
    }
  }

  return ((dKernelTime / 1000.) / (double)iterations);
}

#endif  // #ifndef _BOXFILTER_KERNEL_H_
