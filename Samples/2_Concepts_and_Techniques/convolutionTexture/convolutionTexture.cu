#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime_api.h>

#include "convolutionTexture_common.h"

////////////////////////////////////////////////////////////////////////////////
// GPU-specific defines
////////////////////////////////////////////////////////////////////////////////
// Maps to a single instruction on G8x / G9x / G10x
#define IMAD(a, b, c) (__mul24((a), (b)) + (c))

// Use unrolled innermost convolution loop
#define UNROLL_INNER 1

// Round a / b to nearest higher integer value
inline int iDivUp(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

// Align a to nearest higher multiple of b
inline int iAlignUp(int a, int b) { return (a % b != 0) ? (a - a % b + b) : a; }

////////////////////////////////////////////////////////////////////////////////
// Convolution kernel and input array storage
////////////////////////////////////////////////////////////////////////////////
__constant__ float c_Kernel[KERNEL_LENGTH];

extern "C" void setConvolutionKernel(float *h_Kernel) {
  hipMemcpyToSymbol(HIP_SYMBOL(c_Kernel), h_Kernel, KERNEL_LENGTH * sizeof(float));
}

////////////////////////////////////////////////////////////////////////////////
// Loop unrolling templates, needed for best performance
////////////////////////////////////////////////////////////////////////////////
template <int i>
__device__ float convolutionRow(float x, float y, hipTextureObject_t texSrc) {
  return tex2D<float>(texSrc, x + (float)(KERNEL_RADIUS - i), y) * c_Kernel[i] +
         convolutionRow<i - 1>(x, y, texSrc);
}

template <>
__device__ float convolutionRow<-1>(float x, float y,
                                    hipTextureObject_t texSrc) {
  return 0;
}

template <int i>
__device__ float convolutionColumn(float x, float y,
                                   hipTextureObject_t texSrc) {
  return tex2D<float>(texSrc, x, y + (float)(KERNEL_RADIUS - i)) * c_Kernel[i] +
         convolutionColumn<i - 1>(x, y, texSrc);
}

template <>
__device__ float convolutionColumn<-1>(float x, float y,
                                       hipTextureObject_t texSrc) {
  return 0;
}

////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionRowsKernel(float *d_Dst, int imageW, int imageH,
                                      hipTextureObject_t texSrc) {
  const int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
  const int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
  const float x = (float)ix + 0.5f;
  const float y = (float)iy + 0.5f;

  if (ix >= imageW || iy >= imageH) {
    return;
  }

  float sum = 0;

#if (UNROLL_INNER)
  sum = convolutionRow<2 * KERNEL_RADIUS>(x, y, texSrc);
#else

  for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++) {
    sum += tex2D<float>(texSrc, x + (float)k, y) * c_Kernel[KERNEL_RADIUS - k];
  }

#endif

  d_Dst[IMAD(iy, imageW, ix)] = sum;
}

extern "C" void convolutionRowsGPU(float *d_Dst, hipArray *a_Src, int imageW,
                                   int imageH, hipTextureObject_t texSrc) {
  dim3 threads(16, 12);
  dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

  convolutionRowsKernel<<<blocks, threads>>>(d_Dst, imageW, imageH, texSrc);
  getLastCudaError("convolutionRowsKernel() execution failed\n");
}

////////////////////////////////////////////////////////////////////////////////
// Column convolution filter
////////////////////////////////////////////////////////////////////////////////
__global__ void convolutionColumnsKernel(float *d_Dst, int imageW, int imageH,
                                         hipTextureObject_t texSrc) {
  const int ix = IMAD(blockDim.x, blockIdx.x, threadIdx.x);
  const int iy = IMAD(blockDim.y, blockIdx.y, threadIdx.y);
  const float x = (float)ix + 0.5f;
  const float y = (float)iy + 0.5f;

  if (ix >= imageW || iy >= imageH) {
    return;
  }

  float sum = 0;

#if (UNROLL_INNER)
  sum = convolutionColumn<2 * KERNEL_RADIUS>(x, y, texSrc);
#else

  for (int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++) {
    sum += tex2D<float>(texSrc, x, y + (float)k) * c_Kernel[KERNEL_RADIUS - k];
  }

#endif

  d_Dst[IMAD(iy, imageW, ix)] = sum;
}

extern "C" void convolutionColumnsGPU(float *d_Dst, hipArray *a_Src,
                                      int imageW, int imageH,
                                      hipTextureObject_t texSrc) {
  dim3 threads(16, 12);
  dim3 blocks(iDivUp(imageW, threads.x), iDivUp(imageH, threads.y));

  convolutionColumnsKernel<<<blocks, threads>>>(d_Dst, imageW, imageH, texSrc);
  getLastCudaError("convolutionColumnsKernel() execution failed\n");
}
