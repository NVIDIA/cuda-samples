#include "hip/hip_runtime.h"
/* Copyright (c) 2022, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

/*
  Parallel reduction

  This sample shows how to perform a reduction operation on an array of values
  to produce a single value in a single kernel (as opposed to two or more
  kernel calls as shown in the "reduction" CUDA Sample).  Single-pass
  reduction requires global atomic instructions (Compute Capability 1.1 or
  later) and the __threadfence() intrinsic (CUDA 2.2 or later).

  Reductions are a very common computation in parallel algorithms.  Any time
  an array of values needs to be reduced to a single value using a binary
  associative operator, a reduction can be used.  Example applications include
  statistics computations such as mean and standard deviation, and image
  processing applications such as finding the total luminance of an
  image.

  This code performs sum reductions, but any associative operator such as
  min() or max() could also be used.

  It assumes the input size is a power of 2.

  COMMAND LINE ARGUMENTS

  "--shmoo":         Test performance for 1 to 32M elements with each of the 
                     7 different kernels
  "--n=<N>":         Specify the number of elements to reduce (default 1048576)
  "--threads=<N>":   Specify the number of threads per block (default 128)
  "--maxblocks=<N>": Specify the maximum number of thread blocks to launch 
                     (kernel 6 only, default 64)
  "--cpufinal":      Read back the per-block results and do final sum of block 
                     sums on CPU (default false)
  "--cputhresh=<N>": The threshold of number of blocks sums below which to 
                     perform a CPU final reduction (default 1)
  "--multipass":     Use a multipass reduction instead of a single-pass reduction
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#define VERSION_MAJOR (CUDART_VERSION / 1000)
#define VERSION_MINOR (CUDART_VERSION % 100) / 10

const char *sSDKsample = "threadFenceReduction";

#if CUDART_VERSION >= 2020
#include "threadFenceReduction_kernel.cuh"
#else
#pragma comment(user, "CUDA 2.2 is required to build for threadFenceReduction")
#endif

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
bool runTest(int argc, char **argv);

extern "C" {
void reduce(int size, int threads, int blocks, float *d_idata, float *d_odata);
void reduceSinglePass(int size, int threads, int blocks, float *d_idata,
                      float *d_odata);
}

#if CUDART_VERSION < 2020
void reduce(int size, int threads, int blocks, float *d_idata, float *d_odata) {
  printf("reduce(), compiler not supported, aborting tests\n");
}

void reduceSinglePass(int size, int threads, int blocks, float *d_idata,
                      float *d_odata) {
  printf("reduceSinglePass(), compiler not supported, aborting tests\n");
}
#endif

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
  hipDeviceProp_t deviceProp;
  deviceProp.major = 0;
  deviceProp.minor = 0;
  int dev;

  printf("%s Starting...\n\n", sSDKsample);

  dev = findCudaDevice(argc, (const char **)argv);

  checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));

  printf("GPU Device supports SM %d.%d compute capability\n\n",
         deviceProp.major, deviceProp.minor);

  bool bTestResult = false;

#if CUDART_VERSION >= 2020
  bTestResult = runTest(argc, argv);
#else
  print_NVCC_min_spec(sSDKsample, "2.2", "Version 185");
  exit(EXIT_SUCCESS);
#endif

  exit(bTestResult ? EXIT_SUCCESS : EXIT_FAILURE);
}

////////////////////////////////////////////////////////////////////////////////
//! Compute sum reduction on CPU
//! We use Kahan summation for an accurate sum of large arrays.
//! http://en.wikipedia.org/wiki/Kahan_summation_algorithm
//!
//! @param data       pointer to input data
//! @param size       number of input data elements
////////////////////////////////////////////////////////////////////////////////
template <class T>
T reduceCPU(T *data, int size) {
  T sum = data[0];
  T c = (T)0.0;

  for (int i = 1; i < size; i++) {
    T y = data[i] - c;
    T t = sum + y;
    c = (t - sum) - y;
    sum = t;
  }

  return sum;
}

unsigned int nextPow2(unsigned int x) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

////////////////////////////////////////////////////////////////////////////////
// Compute the number of threads and blocks to use for the reduction
// We set threads / block to the minimum of maxThreads and n/2.
////////////////////////////////////////////////////////////////////////////////
void getNumBlocksAndThreads(int n, int maxBlocks, int maxThreads, int &blocks,
                            int &threads) {
  if (n == 1) {
    threads = 1;
    blocks = 1;
  } else {
    threads = (n < maxThreads * 2) ? nextPow2(n / 2) : maxThreads;
    blocks = max(1, n / (threads * 2));
  }

  blocks = min(maxBlocks, blocks);
}

////////////////////////////////////////////////////////////////////////////////
// This function performs a reduction of the input data multiple times and
// measures the average reduction time.
////////////////////////////////////////////////////////////////////////////////
float benchmarkReduce(int n, int numThreads, int numBlocks, int maxThreads,
                      int maxBlocks, int testIterations, bool multiPass,
                      bool cpuFinalReduction, int cpuFinalThreshold,
                      StopWatchInterface *timer, float *h_odata, float *d_idata,
                      float *d_odata) {
  float gpu_result = 0;
  bool bNeedReadback = true;
  hipError_t error;

  for (int i = 0; i < testIterations; ++i) {
    gpu_result = 0;
    unsigned int retCnt = 0;
    error = setRetirementCount(retCnt);
    checkCudaErrors(error);

    hipDeviceSynchronize();
    sdkStartTimer(&timer);

    if (multiPass) {
      // execute the kernel
      reduce(n, numThreads, numBlocks, d_idata, d_odata);

      // check if kernel execution generated an error
      getLastCudaError("Kernel execution failed");

      if (cpuFinalReduction) {
        // sum partial sums from each block on CPU
        // copy result from device to host
        error = hipMemcpy(h_odata, d_odata, numBlocks * sizeof(float),
                           hipMemcpyDeviceToHost);
        checkCudaErrors(error);

        for (int i = 0; i < numBlocks; i++) {
          gpu_result += h_odata[i];
        }

        bNeedReadback = false;
      } else {
        // sum partial block sums on GPU
        int s = numBlocks;

        while (s > cpuFinalThreshold) {
          int threads = 0, blocks = 0;
          getNumBlocksAndThreads(s, maxBlocks, maxThreads, blocks, threads);

          reduce(s, threads, blocks, d_odata, d_odata);

          s = s / (threads * 2);
        }

        if (s > 1) {
          // copy result from device to host
          error = hipMemcpy(h_odata, d_odata, s * sizeof(float),
                             hipMemcpyDeviceToHost);
          checkCudaErrors(error);

          for (int i = 0; i < s; i++) {
            gpu_result += h_odata[i];
          }

          bNeedReadback = false;
        }
      }
    } else {
      getLastCudaError("Kernel execution failed");

      // execute the kernel
      reduceSinglePass(n, numThreads, numBlocks, d_idata, d_odata);

      // check if kernel execution generated an error
      getLastCudaError("Kernel execution failed");
    }

    hipDeviceSynchronize();
    sdkStopTimer(&timer);
  }

  if (bNeedReadback) {
    // copy final sum from device to host
    error =
        hipMemcpy(&gpu_result, d_odata, sizeof(float), hipMemcpyDeviceToHost);
    checkCudaErrors(error);
  }

  return gpu_result;
}

////////////////////////////////////////////////////////////////////////////////
// This function calls benchmarkReduce multiple times for a range of array sizes
// and prints a report in CSV (comma-separated value) format that can be used
// for generating a "shmoo" plot showing the performance for each kernel
// variation over a wide range of input sizes.
////////////////////////////////////////////////////////////////////////////////
void shmoo(int minN, int maxN, int maxThreads, int maxBlocks) {
  // create random input data on CPU
  unsigned int bytes = maxN * sizeof(float);

  float *h_idata = (float *)malloc(bytes);

  for (int i = 0; i < maxN; i++) {
    // Keep the numbers small so we don't get truncation error in the sum
    h_idata[i] = (rand() & 0xFF) / (float)RAND_MAX;
  }

  int maxNumBlocks = min(65535, maxN / maxThreads);

  // allocate mem for the result on host side
  float *h_odata = (float *)malloc(maxNumBlocks * sizeof(float));

  // allocate device memory and data
  float *d_idata = NULL;
  float *d_odata = NULL;

  checkCudaErrors(hipMalloc((void **)&d_idata, bytes));
  checkCudaErrors(hipMalloc((void **)&d_odata, maxNumBlocks * sizeof(float)));

  // copy data directly to device memory
  checkCudaErrors(hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_odata, h_idata, maxNumBlocks * sizeof(float),
                             hipMemcpyHostToDevice));

  // warm-up
  reduce(maxN, maxThreads, maxNumBlocks, d_idata, d_odata);
  int testIterations = 100;

  StopWatchInterface *timer = NULL;
  sdkCreateTimer(&timer);

  // print headers
  printf("N, %d blocks one pass, %d blocks multipass\n", maxBlocks, maxBlocks);

  for (int i = minN; i <= maxN; i *= 2) {
    printf("%d, ", i);

    for (int multiPass = 0; multiPass <= 1; multiPass++) {
      sdkResetTimer(&timer);
      int numBlocks = 0;
      int numThreads = 0;
      getNumBlocksAndThreads(i, maxBlocks, maxThreads, numBlocks, numThreads);

      benchmarkReduce(i, numThreads, numBlocks, maxThreads, maxBlocks,
                      testIterations, multiPass == 1, false, 1, timer, h_odata,
                      d_idata, d_odata);

      float reduceTime = sdkGetAverageTimerValue(&timer);
      printf("%f%s", reduceTime, multiPass == 0 ? ", " : "\n");
    }
  }

  printf("\n");

  // cleanup
  sdkDeleteTimer(&timer);
  free(h_idata);
  free(h_odata);

  hipFree(d_idata);
  hipFree(d_odata);
}

////////////////////////////////////////////////////////////////////////////////
// The main function which runs the reduction test.
////////////////////////////////////////////////////////////////////////////////
bool runTest(int argc, char **argv) {
  int size = 1 << 20;    // number of elements to reduce
  int maxThreads = 128;  // number of threads per block
  int maxBlocks = 64;
  bool cpuFinalReduction = false;
  int cpuFinalThreshold = 1;
  bool multipass = false;
  bool bTestResult = false;

  if (checkCmdLineFlag(argc, (const char **)argv, "n")) {
    size = getCmdLineArgumentInt(argc, (const char **)argv, "n");
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "threads")) {
    maxThreads = getCmdLineArgumentInt(argc, (const char **)argv, "threads");
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "maxblocks")) {
    maxBlocks = getCmdLineArgumentInt(argc, (const char **)argv, "maxblocks");
  }

  printf("%d elements\n", size);
  printf("%d threads (max)\n", maxThreads);

  cpuFinalReduction = checkCmdLineFlag(argc, (const char **)argv, "cpufinal");
  multipass = checkCmdLineFlag(argc, (const char **)argv, "multipass");

  if (checkCmdLineFlag(argc, (const char **)argv, "cputhresh")) {
    cpuFinalThreshold =
        getCmdLineArgumentInt(argc, (const char **)argv, "cputhresh");
  }

  bool runShmoo = checkCmdLineFlag(argc, (const char **)argv, "shmoo");

  if (runShmoo) {
    shmoo(1, 33554432, maxThreads, maxBlocks);
  } else {
    // create random input data on CPU
    unsigned int bytes = size * sizeof(float);

    float *h_idata = (float *)malloc(bytes);

    for (int i = 0; i < size; i++) {
      // Keep the numbers small so we don't get truncation error in the sum
      h_idata[i] = (rand() & 0xFF) / (float)RAND_MAX;
    }

    int numBlocks = 0;
    int numThreads = 0;
    getNumBlocksAndThreads(size, maxBlocks, maxThreads, numBlocks, numThreads);

    if (numBlocks == 1) {
      cpuFinalThreshold = 1;
    }

    // allocate mem for the result on host side
    float *h_odata = (float *)malloc(numBlocks * sizeof(float));

    printf("%d blocks\n", numBlocks);

    // allocate device memory and data
    float *d_idata = NULL;
    float *d_odata = NULL;

    checkCudaErrors(hipMalloc((void **)&d_idata, bytes));
    checkCudaErrors(hipMalloc((void **)&d_odata, numBlocks * sizeof(float)));

    // copy data directly to device memory
    checkCudaErrors(
        hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_odata, h_idata, numBlocks * sizeof(float),
                               hipMemcpyHostToDevice));

    // warm-up
    reduce(size, numThreads, numBlocks, d_idata, d_odata);
    int testIterations = 100;

    StopWatchInterface *timer = 0;
    sdkCreateTimer(&timer);

    float gpu_result = 0;

    gpu_result =
        benchmarkReduce(size, numThreads, numBlocks, maxThreads, maxBlocks,
                        testIterations, multipass, cpuFinalReduction,
                        cpuFinalThreshold, timer, h_odata, d_idata, d_odata);

    float reduceTime = sdkGetAverageTimerValue(&timer);
    printf("Average time: %f ms\n", reduceTime);
    printf("Bandwidth:    %f GB/s\n\n",
           (size * sizeof(int)) / (reduceTime * 1.0e6));

    // compute reference solution
    float cpu_result = reduceCPU<float>(h_idata, size);

    printf("GPU result = %0.12f\n", gpu_result);
    printf("CPU result = %0.12f\n", cpu_result);

    double threshold = 1e-8 * size;
    double diff = abs((double)gpu_result - (double)cpu_result);
    bTestResult = (diff < threshold);

    // cleanup
    sdkDeleteTimer(&timer);

    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);
  }

  return bTestResult;
}
